#include "hip/hip_runtime.h"
#include <stdio.h>

#include "include/inc.h"

class CUDAMatrix {
private:
    float *mat, *dev_mat;
    int row, col;
    size_t size;

public:
    CUDAMatrix(int r, int c) : row(r), col(c), size(sizeof(float) * r * c) { mat = (float *) malloc(size); hipMalloc((void **) &dev_mat, size); }
    ~CUDAMatrix() { free(mat), hipFree(dev_mat); }
    float *devPtr() { return dev_mat; }
    void cpyToDevice() { hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice); }
    void cpyToHost() { hipMemcpy(mat, dev_mat, size, hipMemcpyDeviceToHost); }
    float &at(int i, int j) { return mat[i * col + j]; }
    const float &at(int i, int j) const { return mat[i * col + j]; }
    void print() {
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", mat[i * col + j]);
            }
            printf("\n");
        }
    }
    void fillRand() {
        for (int i = 0; i < row; i++)
            for (int j = 0; j < col; j++)
                at(i, j) = rand() % 100;
    }
    void fill(float val) {
        for (int i = 0; i < row; i++)
            for (int j = 0; j < col; j++)
                at(i, j) = val;
    }
};

float equal(float a, float b) { return abs(a - b) < 0.001; }

float multiTest(void (*calc)(float *, float *, float *), int round = 100, bool check = true) {
    srand(time(nullptr));
    // fprintf(stderr, "srand seed: %ld\n", time(nullptr));
    // srand(1666362747);
    static const int warmupGoal = round / 10;
    CUDAMatrix A(K, M), B(K, N), C(M, N), ans(M, N);
    double totTime = 0;
    A.fillRand(), B.fillRand();
    if (check) {
        ans.fill(0);
        for (int i = 0; i < M; i++)
            for (int j = 0; j < N; j++)
                for (int k = 0; k < K; k++) 
                    ans.at(i, j) += A.at(k, i) * B.at(k, j);
    }  
    A.cpyToDevice(), B.cpyToDevice();
    for (int i = 0; i < warmupGoal; i++) calc(A.devPtr(), B.devPtr(), C.devPtr());
    for (int i = 0; i < round; i++) {
        // A.fillRand(), B.fillRand();
        // A.cpyToDevice(), B.cpyToDevice();
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        calc(A.devPtr(), B.devPtr(), C.devPtr());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float ti;
        hipEventElapsedTime(&ti, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        totTime += ti;
        // C.cpyToHost();
        // printf("%f\n", ti);
    }
    C.cpyToHost();
    if (check) {
        for (int i = 0; i < M; i++)
            for (int j = 0; j < N; j++)
                if (!equal(ans.at(i, j), C.at(i, j)))
                    return fprintf(stderr, "Error with (%.2f - %.2f) = %.2f at (%d, %d)\n", ans.at(i, j), C.at(i, j), (ans.at(i, j) - C.at(i, j)), i, j), -1;
    }
    double FLOPs = (((long long int) (M)) * N * K + M * N) * 2;
    double runtime = totTime / round;
    float ret = double(FLOPs) / runtime / 1.0e6;
    if (ret > 25000) fprintf(stderr, "Cuda Error: %s\n", hipGetErrorString(hipGetLastError()));
    return ret;
}

void run_tune(float *A, float *B, float *C) {
    if (SHARED_SIZE >= (48 << 10)) hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel0), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_SIZE);
    main_kernel0<<<BLOCK_NUM, THREAD_NUM, SHARED_SIZE>>>(A, B, C);
}

int main() {
    float tune = multiTest(run_tune, 100, false);
    printf("tune GFLOPs: %f\n", tune);
    return 0;
}