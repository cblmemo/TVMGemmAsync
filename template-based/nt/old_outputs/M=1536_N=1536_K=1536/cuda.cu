
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[96];
  __shared__ float A_shared[768];
  __shared__ float B_shared[4096];
  for (int i_3_init = 0; i_3_init < 12; ++i_3_init) {
    Y_local[(i_3_init * 4)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 48)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 1)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 49)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 2)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 50)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 3)] = 0.000000e+00f;
    Y_local[((i_3_init * 4) + 51)] = 0.000000e+00f;
  }
  for (int k_0 = 0; k_0 < 96; ++k_0) {
    __syncthreads();
    *(float2*)(A_shared + (((int)threadIdx.x) * 2)) = *(float2*)(A + ((((k_0 * 24576) + ((((int)threadIdx.x) / 24) * 1536)) + ((((int)blockIdx.x) / 6) * 48)) + ((((int)threadIdx.x) % 24) * 2)));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(A + ((((k_0 * 24576) + ((((((int)threadIdx.x) * 2) + 256) / 48) * 1536)) + ((((int)blockIdx.x) / 6) * 48)) + (((((int)threadIdx.x) * 2) + 16) % 48)));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(A + ((((k_0 * 24576) + ((((((int)threadIdx.x) * 2) + 512) / 48) * 1536)) + ((((int)blockIdx.x) / 6) * 48)) + (((((int)threadIdx.x) * 2) + 32) % 48)));
    *(float4*)(B_shared + (((int)threadIdx.x) * 4)) = *(float4*)(B + ((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 3072));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 6144));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 9216));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 12288));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 15360));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 18432));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(B + (((((k_0 * 24576) + ((((int)threadIdx.x) >> 6) * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 21504));
    __syncthreads();
    for (int k_1 = 0; k_1 < 16; ++k_1) {
      for (int i_3 = 0; i_3 < 12; ++i_3) {
        Y_local[(i_3 * 4)] = (Y_local[(i_3 * 4)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4))]));
        Y_local[((i_3 * 4) + 48)] = (Y_local[((i_3 * 4) + 48)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 128)]));
        Y_local[((i_3 * 4) + 1)] = (Y_local[((i_3 * 4) + 1)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 1)]));
        Y_local[((i_3 * 4) + 49)] = (Y_local[((i_3 * 4) + 49)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 129)]));
        Y_local[((i_3 * 4) + 2)] = (Y_local[((i_3 * 4) + 2)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 2)]));
        Y_local[((i_3 * 4) + 50)] = (Y_local[((i_3 * 4) + 50)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 130)]));
        Y_local[((i_3 * 4) + 3)] = (Y_local[((i_3 * 4) + 3)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 3)]));
        Y_local[((i_3 * 4) + 51)] = (Y_local[((i_3 * 4) + 51)] + (A_shared[(((k_1 * 48) + ((((int)threadIdx.x) >> 5) * 12)) + i_3)] * B_shared[(((k_1 * 256) + ((((int)threadIdx.x) & 31) * 4)) + 131)]));
      }
    }
  }
  for (int ax0 = 0; ax0 < 12; ++ax0) {
    Y[((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4))] = Y_local[(ax0 * 4)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 128)] = Y_local[((ax0 * 4) + 48)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 1)] = Y_local[((ax0 * 4) + 1)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 129)] = Y_local[((ax0 * 4) + 49)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 2)] = Y_local[((ax0 * 4) + 2)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 130)] = Y_local[((ax0 * 4) + 50)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 3)] = Y_local[((ax0 * 4) + 3)];
    Y[(((((((((int)blockIdx.x) / 6) * 73728) + ((((int)threadIdx.x) >> 5) * 18432)) + (ax0 * 1536)) + ((((int)blockIdx.x) % 6) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + 131)] = Y_local[((ax0 * 4) + 51)];
  }
}


