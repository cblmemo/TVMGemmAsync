
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(112) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[56];
  __shared__ float A_shared[1568];
  __shared__ float B_shared[3136];
  for (int i_3_init = 0; i_3_init < 4; ++i_3_init) {
    for (int j_3_init = 0; j_3_init < 2; ++j_3_init) {
      Y_local[((i_3_init * 2) + j_3_init)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 8)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 16)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 24)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 32)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 40)] = 0.000000e+00f;
      Y_local[(((i_3_init * 2) + j_3_init) + 48)] = 0.000000e+00f;
    }
  }
  for (int k_0 = 0; k_0 < 32; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 14; ++ax0_ax1_fused_0) {
      A_shared[((ax0_ax1_fused_0 * 112) + ((int)threadIdx.x))] = A[(((((k_0 * 25088) + (ax0_ax1_fused_0 * 1792)) + ((((int)threadIdx.x) / 56) * 896)) + ((((int)blockIdx.x) >> 3) * 56)) + (((int)threadIdx.x) % 56))];
    }
    for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 7; ++ax0_ax1_fused_0_1) {
      *(float4*)(B_shared + ((ax0_ax1_fused_0_1 * 448) + (((int)threadIdx.x) * 4))) = *(float4*)(B + (((((k_0 * 25088) + (ax0_ax1_fused_0_1 * 3584)) + ((((int)threadIdx.x) / 28) * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) % 28) * 4)));
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 4; ++k_1) {
      for (int i_3 = 0; i_3 < 4; ++i_3) {
        for (int j_3 = 0; j_3 < 2; ++j_3) {
          for (int k_2 = 0; k_2 < 7; ++k_2) {
            Y_local[((i_3 * 2) + j_3)] = (Y_local[((i_3 * 2) + j_3)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3)]));
            Y_local[(((i_3 * 2) + j_3) + 8)] = (Y_local[(((i_3 * 2) + j_3) + 8)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 16)]));
            Y_local[(((i_3 * 2) + j_3) + 16)] = (Y_local[(((i_3 * 2) + j_3) + 16)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 32)]));
            Y_local[(((i_3 * 2) + j_3) + 24)] = (Y_local[(((i_3 * 2) + j_3) + 24)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 48)]));
            Y_local[(((i_3 * 2) + j_3) + 32)] = (Y_local[(((i_3 * 2) + j_3) + 32)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 64)]));
            Y_local[(((i_3 * 2) + j_3) + 40)] = (Y_local[(((i_3 * 2) + j_3) + 40)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 80)]));
            Y_local[(((i_3 * 2) + j_3) + 48)] = (Y_local[(((i_3 * 2) + j_3) + 48)] + (A_shared[((((k_1 * 392) + (k_2 * 56)) + ((((int)threadIdx.x) >> 3) * 4)) + i_3)] * B_shared[(((((k_1 * 784) + (k_2 * 112)) + ((((int)threadIdx.x) & 7) * 2)) + j_3) + 96)]));
          }
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 4; ++ax0) {
    for (int ax1 = 0; ax1 < 2; ++ax1) {
      Y[(((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1)] = Y_local[((ax0 * 2) + ax1)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 16)] = Y_local[(((ax0 * 2) + ax1) + 8)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 32)] = Y_local[(((ax0 * 2) + ax1) + 16)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 48)] = Y_local[(((ax0 * 2) + ax1) + 24)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 64)] = Y_local[(((ax0 * 2) + ax1) + 32)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 80)] = Y_local[(((ax0 * 2) + ax1) + 40)];
      Y[((((((((((int)blockIdx.x) >> 3) * 50176) + ((((int)threadIdx.x) >> 3) * 3584)) + (ax0 * 896)) + ((((int)blockIdx.x) & 7) * 112)) + ((((int)threadIdx.x) & 7) * 2)) + ax1) + 96)] = Y_local[(((ax0 * 2) + ax1) + 48)];
    }
  }
}


