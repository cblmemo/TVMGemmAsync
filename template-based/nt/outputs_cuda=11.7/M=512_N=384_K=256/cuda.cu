
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[24];
  float A_shared_dyn_local[4];
  float B_shared_dyn_local[24];
  Y_local[0] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)threadIdx.x) >> 4) * 512) + ((((int)blockIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 3; ++ax0_ax1_fused_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) >> 4) * 512) + (((((((int)blockIdx.x) & 7) * 3) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 6) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2) & 3) * 4)) + 8192) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + ax0_ax1_fused_2))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)threadIdx.x) >> 4) * 512) + ((((int)blockIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 3; ++ax0_ax1_fused_2_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) >> 4) * 512) + (((((((int)blockIdx.x) & 7) * 3) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_1) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_1) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 6) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_1) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_1) & 3) * 4)) + 12288) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) >> 4) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + ax0_ax1_fused_2_1) + 3072))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)threadIdx.x) >> 4) * 512) + ((((int)blockIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 3; ++ax0_ax1_fused_2_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) >> 4) * 512) + (((((((int)blockIdx.x) & 7) * 3) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_2) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_2) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 6) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_2) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_2) & 3) * 4)) + 16384) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) >> 4) * 384) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + ax0_ax1_fused_2_2) + 6144))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax1_1_s = 0; ax1_1_s < 4; ++ax1_1_s) {
    if (ax1_1_s < 2) {
      A_shared_dyn_local[ax1_1_s] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s)];
    }
  }
  for (int ax1_0 = 0; ax1_0 < 3; ++ax1_0) {
    *(float4*)(B_shared_dyn_local + (ax1_0 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0 * 4)) >> 6) * 64) + (((ax1_0 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0 * 4)) >> 3)) & 7) * 4)) + 2048) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int k_0 = 0; k_0 < 29; ++k_0) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((k_0 + 3) & 3) * 2048) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((k_0 * 4096) + ((((int)threadIdx.x) >> 4) * 512)) + ((((int)blockIdx.x) >> 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 12288))), "n"(16)
    );
  }
    for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 3; ++ax0_ax1_fused_2_3) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((k_0 + 3) & 3) * 4096) + ((((int)threadIdx.x) >> 4) * 512)) + (((((((int)blockIdx.x) & 7) * 3) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_3) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_3) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 6) + ((((((int)threadIdx.x) & 15) * 3) + ax0_ax1_fused_2_3) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_3) & 3) * 4)) + 8192) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((k_0 * 3072) + ((((int)threadIdx.x) >> 4) * 384)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) & 15) * 3)) + ax0_ax1_fused_2_3) + 9216))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int ax1_1_s_1 = 0; ax1_1_s_1 < 4; ++ax1_1_s_1) {
      if (ax1_1_s_1 < 2) {
        A_shared_dyn_local[(ax1_1_s_1 + 2)] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_1) + 64)];
      }
    }
    for (int ax1_0_1 = 0; ax1_0_1 < 3; ++ax1_0_1) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_1 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_1 * 4)) >> 6) * 64)) + (((ax1_0_1 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_1 * 4)) >> 3)) & 7) * 4)) + 2176) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s = 0; i_2_1_s < 4; ++i_2_1_s) {
      if (i_2_1_s < 2) {
        Y_local[(i_2_1_s * 12)] = (Y_local[(i_2_1_s * 12)] + (A_shared_dyn_local[i_2_1_s] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_1 = 0; i_2_1_s_1 < 4; ++i_2_1_s_1) {
      if (i_2_1_s_1 < 2) {
        Y_local[((i_2_1_s_1 * 12) + 1)] = (Y_local[((i_2_1_s_1 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_1] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_2 = 0; i_2_1_s_2 < 4; ++i_2_1_s_2) {
      if (i_2_1_s_2 < 2) {
        Y_local[((i_2_1_s_2 * 12) + 2)] = (Y_local[((i_2_1_s_2 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_2] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_3 = 0; i_2_1_s_3 < 4; ++i_2_1_s_3) {
      if (i_2_1_s_3 < 2) {
        Y_local[((i_2_1_s_3 * 12) + 3)] = (Y_local[((i_2_1_s_3 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_3] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_4 = 0; i_2_1_s_4 < 4; ++i_2_1_s_4) {
      if (i_2_1_s_4 < 2) {
        Y_local[((i_2_1_s_4 * 12) + 4)] = (Y_local[((i_2_1_s_4 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_4] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_5 = 0; i_2_1_s_5 < 4; ++i_2_1_s_5) {
      if (i_2_1_s_5 < 2) {
        Y_local[((i_2_1_s_5 * 12) + 5)] = (Y_local[((i_2_1_s_5 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_5] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_6 = 0; i_2_1_s_6 < 4; ++i_2_1_s_6) {
      if (i_2_1_s_6 < 2) {
        Y_local[((i_2_1_s_6 * 12) + 6)] = (Y_local[((i_2_1_s_6 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_6] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_7 = 0; i_2_1_s_7 < 4; ++i_2_1_s_7) {
      if (i_2_1_s_7 < 2) {
        Y_local[((i_2_1_s_7 * 12) + 7)] = (Y_local[((i_2_1_s_7 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_7] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_8 = 0; i_2_1_s_8 < 4; ++i_2_1_s_8) {
      if (i_2_1_s_8 < 2) {
        Y_local[((i_2_1_s_8 * 12) + 8)] = (Y_local[((i_2_1_s_8 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_8] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_9 = 0; i_2_1_s_9 < 4; ++i_2_1_s_9) {
      if (i_2_1_s_9 < 2) {
        Y_local[((i_2_1_s_9 * 12) + 9)] = (Y_local[((i_2_1_s_9 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_9] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_10 = 0; i_2_1_s_10 < 4; ++i_2_1_s_10) {
      if (i_2_1_s_10 < 2) {
        Y_local[((i_2_1_s_10 * 12) + 10)] = (Y_local[((i_2_1_s_10 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_10] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_11 = 0; i_2_1_s_11 < 4; ++i_2_1_s_11) {
      if (i_2_1_s_11 < 2) {
        Y_local[((i_2_1_s_11 * 12) + 11)] = (Y_local[((i_2_1_s_11 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_11] * B_shared_dyn_local[11]));
      }
    }
    for (int ax1_1_s_2 = 0; ax1_1_s_2 < 4; ++ax1_1_s_2) {
      if (ax1_1_s_2 < 2) {
        A_shared_dyn_local[ax1_1_s_2] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_2) + 128)];
      }
    }
    for (int ax1_0_2 = 0; ax1_0_2 < 3; ++ax1_0_2) {
      *(float4*)(B_shared_dyn_local + (ax1_0_2 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_2 * 4)) >> 6) * 64)) + (((ax1_0_2 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_2 * 4)) >> 3)) & 7) * 4)) + 2304) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_12 = 0; i_2_1_s_12 < 4; ++i_2_1_s_12) {
      if (i_2_1_s_12 < 2) {
        Y_local[(i_2_1_s_12 * 12)] = (Y_local[(i_2_1_s_12 * 12)] + (A_shared_dyn_local[(i_2_1_s_12 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_13 = 0; i_2_1_s_13 < 4; ++i_2_1_s_13) {
      if (i_2_1_s_13 < 2) {
        Y_local[((i_2_1_s_13 * 12) + 1)] = (Y_local[((i_2_1_s_13 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_13 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_14 = 0; i_2_1_s_14 < 4; ++i_2_1_s_14) {
      if (i_2_1_s_14 < 2) {
        Y_local[((i_2_1_s_14 * 12) + 2)] = (Y_local[((i_2_1_s_14 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_14 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_15 = 0; i_2_1_s_15 < 4; ++i_2_1_s_15) {
      if (i_2_1_s_15 < 2) {
        Y_local[((i_2_1_s_15 * 12) + 3)] = (Y_local[((i_2_1_s_15 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_15 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int i_2_1_s_16 = 0; i_2_1_s_16 < 4; ++i_2_1_s_16) {
      if (i_2_1_s_16 < 2) {
        Y_local[((i_2_1_s_16 * 12) + 4)] = (Y_local[((i_2_1_s_16 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_16 + 2)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_17 = 0; i_2_1_s_17 < 4; ++i_2_1_s_17) {
      if (i_2_1_s_17 < 2) {
        Y_local[((i_2_1_s_17 * 12) + 5)] = (Y_local[((i_2_1_s_17 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_17 + 2)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_18 = 0; i_2_1_s_18 < 4; ++i_2_1_s_18) {
      if (i_2_1_s_18 < 2) {
        Y_local[((i_2_1_s_18 * 12) + 6)] = (Y_local[((i_2_1_s_18 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_18 + 2)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_19 = 0; i_2_1_s_19 < 4; ++i_2_1_s_19) {
      if (i_2_1_s_19 < 2) {
        Y_local[((i_2_1_s_19 * 12) + 7)] = (Y_local[((i_2_1_s_19 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_19 + 2)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_20 = 0; i_2_1_s_20 < 4; ++i_2_1_s_20) {
      if (i_2_1_s_20 < 2) {
        Y_local[((i_2_1_s_20 * 12) + 8)] = (Y_local[((i_2_1_s_20 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_20 + 2)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_21 = 0; i_2_1_s_21 < 4; ++i_2_1_s_21) {
      if (i_2_1_s_21 < 2) {
        Y_local[((i_2_1_s_21 * 12) + 9)] = (Y_local[((i_2_1_s_21 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_21 + 2)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_22 = 0; i_2_1_s_22 < 4; ++i_2_1_s_22) {
      if (i_2_1_s_22 < 2) {
        Y_local[((i_2_1_s_22 * 12) + 10)] = (Y_local[((i_2_1_s_22 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_22 + 2)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_23 = 0; i_2_1_s_23 < 4; ++i_2_1_s_23) {
      if (i_2_1_s_23 < 2) {
        Y_local[((i_2_1_s_23 * 12) + 11)] = (Y_local[((i_2_1_s_23 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_23 + 2)] * B_shared_dyn_local[23]));
      }
    }
    for (int ax1_1_s_3 = 0; ax1_1_s_3 < 4; ++ax1_1_s_3) {
      if (ax1_1_s_3 < 2) {
        A_shared_dyn_local[(ax1_1_s_3 + 2)] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_3) + 192)];
      }
    }
    for (int ax1_0_3 = 0; ax1_0_3 < 3; ++ax1_0_3) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_3 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_3 * 4)) >> 6) * 64)) + (((ax1_0_3 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_3 * 4)) >> 3)) & 7) * 4)) + 2432) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_24 = 0; i_2_1_s_24 < 4; ++i_2_1_s_24) {
      if (i_2_1_s_24 < 2) {
        Y_local[(i_2_1_s_24 * 12)] = (Y_local[(i_2_1_s_24 * 12)] + (A_shared_dyn_local[i_2_1_s_24] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_25 = 0; i_2_1_s_25 < 4; ++i_2_1_s_25) {
      if (i_2_1_s_25 < 2) {
        Y_local[((i_2_1_s_25 * 12) + 1)] = (Y_local[((i_2_1_s_25 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_25] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_26 = 0; i_2_1_s_26 < 4; ++i_2_1_s_26) {
      if (i_2_1_s_26 < 2) {
        Y_local[((i_2_1_s_26 * 12) + 2)] = (Y_local[((i_2_1_s_26 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_26] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_27 = 0; i_2_1_s_27 < 4; ++i_2_1_s_27) {
      if (i_2_1_s_27 < 2) {
        Y_local[((i_2_1_s_27 * 12) + 3)] = (Y_local[((i_2_1_s_27 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_27] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_28 = 0; i_2_1_s_28 < 4; ++i_2_1_s_28) {
      if (i_2_1_s_28 < 2) {
        Y_local[((i_2_1_s_28 * 12) + 4)] = (Y_local[((i_2_1_s_28 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_28] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_29 = 0; i_2_1_s_29 < 4; ++i_2_1_s_29) {
      if (i_2_1_s_29 < 2) {
        Y_local[((i_2_1_s_29 * 12) + 5)] = (Y_local[((i_2_1_s_29 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_29] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_30 = 0; i_2_1_s_30 < 4; ++i_2_1_s_30) {
      if (i_2_1_s_30 < 2) {
        Y_local[((i_2_1_s_30 * 12) + 6)] = (Y_local[((i_2_1_s_30 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_30] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_31 = 0; i_2_1_s_31 < 4; ++i_2_1_s_31) {
      if (i_2_1_s_31 < 2) {
        Y_local[((i_2_1_s_31 * 12) + 7)] = (Y_local[((i_2_1_s_31 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_31] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_32 = 0; i_2_1_s_32 < 4; ++i_2_1_s_32) {
      if (i_2_1_s_32 < 2) {
        Y_local[((i_2_1_s_32 * 12) + 8)] = (Y_local[((i_2_1_s_32 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_32] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_33 = 0; i_2_1_s_33 < 4; ++i_2_1_s_33) {
      if (i_2_1_s_33 < 2) {
        Y_local[((i_2_1_s_33 * 12) + 9)] = (Y_local[((i_2_1_s_33 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_33] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_34 = 0; i_2_1_s_34 < 4; ++i_2_1_s_34) {
      if (i_2_1_s_34 < 2) {
        Y_local[((i_2_1_s_34 * 12) + 10)] = (Y_local[((i_2_1_s_34 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_34] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_35 = 0; i_2_1_s_35 < 4; ++i_2_1_s_35) {
      if (i_2_1_s_35 < 2) {
        Y_local[((i_2_1_s_35 * 12) + 11)] = (Y_local[((i_2_1_s_35 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_35] * B_shared_dyn_local[11]));
      }
    }
    for (int ax1_1_s_4 = 0; ax1_1_s_4 < 4; ++ax1_1_s_4) {
      if (ax1_1_s_4 < 2) {
        A_shared_dyn_local[ax1_1_s_4] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_4) + 256)];
      }
    }
    for (int ax1_0_4 = 0; ax1_0_4 < 3; ++ax1_0_4) {
      *(float4*)(B_shared_dyn_local + (ax1_0_4 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_4 * 4)) >> 6) * 64)) + (((ax1_0_4 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_4 * 4)) >> 3)) & 7) * 4)) + 2560) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_36 = 0; i_2_1_s_36 < 4; ++i_2_1_s_36) {
      if (i_2_1_s_36 < 2) {
        Y_local[(i_2_1_s_36 * 12)] = (Y_local[(i_2_1_s_36 * 12)] + (A_shared_dyn_local[(i_2_1_s_36 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_37 = 0; i_2_1_s_37 < 4; ++i_2_1_s_37) {
      if (i_2_1_s_37 < 2) {
        Y_local[((i_2_1_s_37 * 12) + 1)] = (Y_local[((i_2_1_s_37 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_37 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_38 = 0; i_2_1_s_38 < 4; ++i_2_1_s_38) {
      if (i_2_1_s_38 < 2) {
        Y_local[((i_2_1_s_38 * 12) + 2)] = (Y_local[((i_2_1_s_38 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_38 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_39 = 0; i_2_1_s_39 < 4; ++i_2_1_s_39) {
      if (i_2_1_s_39 < 2) {
        Y_local[((i_2_1_s_39 * 12) + 3)] = (Y_local[((i_2_1_s_39 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_39 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int i_2_1_s_40 = 0; i_2_1_s_40 < 4; ++i_2_1_s_40) {
      if (i_2_1_s_40 < 2) {
        Y_local[((i_2_1_s_40 * 12) + 4)] = (Y_local[((i_2_1_s_40 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_40 + 2)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_41 = 0; i_2_1_s_41 < 4; ++i_2_1_s_41) {
      if (i_2_1_s_41 < 2) {
        Y_local[((i_2_1_s_41 * 12) + 5)] = (Y_local[((i_2_1_s_41 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_41 + 2)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_42 = 0; i_2_1_s_42 < 4; ++i_2_1_s_42) {
      if (i_2_1_s_42 < 2) {
        Y_local[((i_2_1_s_42 * 12) + 6)] = (Y_local[((i_2_1_s_42 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_42 + 2)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_43 = 0; i_2_1_s_43 < 4; ++i_2_1_s_43) {
      if (i_2_1_s_43 < 2) {
        Y_local[((i_2_1_s_43 * 12) + 7)] = (Y_local[((i_2_1_s_43 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_43 + 2)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_44 = 0; i_2_1_s_44 < 4; ++i_2_1_s_44) {
      if (i_2_1_s_44 < 2) {
        Y_local[((i_2_1_s_44 * 12) + 8)] = (Y_local[((i_2_1_s_44 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_44 + 2)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_45 = 0; i_2_1_s_45 < 4; ++i_2_1_s_45) {
      if (i_2_1_s_45 < 2) {
        Y_local[((i_2_1_s_45 * 12) + 9)] = (Y_local[((i_2_1_s_45 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_45 + 2)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_46 = 0; i_2_1_s_46 < 4; ++i_2_1_s_46) {
      if (i_2_1_s_46 < 2) {
        Y_local[((i_2_1_s_46 * 12) + 10)] = (Y_local[((i_2_1_s_46 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_46 + 2)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_47 = 0; i_2_1_s_47 < 4; ++i_2_1_s_47) {
      if (i_2_1_s_47 < 2) {
        Y_local[((i_2_1_s_47 * 12) + 11)] = (Y_local[((i_2_1_s_47 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_47 + 2)] * B_shared_dyn_local[23]));
      }
    }
    for (int ax1_1_s_5 = 0; ax1_1_s_5 < 4; ++ax1_1_s_5) {
      if (ax1_1_s_5 < 2) {
        A_shared_dyn_local[(ax1_1_s_5 + 2)] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_5) + 320)];
      }
    }
    for (int ax1_0_5 = 0; ax1_0_5 < 3; ++ax1_0_5) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_5 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_5 * 4)) >> 6) * 64)) + (((ax1_0_5 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_5 * 4)) >> 3)) & 7) * 4)) + 2688) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_48 = 0; i_2_1_s_48 < 4; ++i_2_1_s_48) {
      if (i_2_1_s_48 < 2) {
        Y_local[(i_2_1_s_48 * 12)] = (Y_local[(i_2_1_s_48 * 12)] + (A_shared_dyn_local[i_2_1_s_48] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_49 = 0; i_2_1_s_49 < 4; ++i_2_1_s_49) {
      if (i_2_1_s_49 < 2) {
        Y_local[((i_2_1_s_49 * 12) + 1)] = (Y_local[((i_2_1_s_49 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_49] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_50 = 0; i_2_1_s_50 < 4; ++i_2_1_s_50) {
      if (i_2_1_s_50 < 2) {
        Y_local[((i_2_1_s_50 * 12) + 2)] = (Y_local[((i_2_1_s_50 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_50] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_51 = 0; i_2_1_s_51 < 4; ++i_2_1_s_51) {
      if (i_2_1_s_51 < 2) {
        Y_local[((i_2_1_s_51 * 12) + 3)] = (Y_local[((i_2_1_s_51 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_51] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_52 = 0; i_2_1_s_52 < 4; ++i_2_1_s_52) {
      if (i_2_1_s_52 < 2) {
        Y_local[((i_2_1_s_52 * 12) + 4)] = (Y_local[((i_2_1_s_52 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_52] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_53 = 0; i_2_1_s_53 < 4; ++i_2_1_s_53) {
      if (i_2_1_s_53 < 2) {
        Y_local[((i_2_1_s_53 * 12) + 5)] = (Y_local[((i_2_1_s_53 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_53] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_54 = 0; i_2_1_s_54 < 4; ++i_2_1_s_54) {
      if (i_2_1_s_54 < 2) {
        Y_local[((i_2_1_s_54 * 12) + 6)] = (Y_local[((i_2_1_s_54 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_54] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_55 = 0; i_2_1_s_55 < 4; ++i_2_1_s_55) {
      if (i_2_1_s_55 < 2) {
        Y_local[((i_2_1_s_55 * 12) + 7)] = (Y_local[((i_2_1_s_55 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_55] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_56 = 0; i_2_1_s_56 < 4; ++i_2_1_s_56) {
      if (i_2_1_s_56 < 2) {
        Y_local[((i_2_1_s_56 * 12) + 8)] = (Y_local[((i_2_1_s_56 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_56] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_57 = 0; i_2_1_s_57 < 4; ++i_2_1_s_57) {
      if (i_2_1_s_57 < 2) {
        Y_local[((i_2_1_s_57 * 12) + 9)] = (Y_local[((i_2_1_s_57 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_57] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_58 = 0; i_2_1_s_58 < 4; ++i_2_1_s_58) {
      if (i_2_1_s_58 < 2) {
        Y_local[((i_2_1_s_58 * 12) + 10)] = (Y_local[((i_2_1_s_58 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_58] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_59 = 0; i_2_1_s_59 < 4; ++i_2_1_s_59) {
      if (i_2_1_s_59 < 2) {
        Y_local[((i_2_1_s_59 * 12) + 11)] = (Y_local[((i_2_1_s_59 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_59] * B_shared_dyn_local[11]));
      }
    }
    for (int ax1_1_s_6 = 0; ax1_1_s_6 < 4; ++ax1_1_s_6) {
      if (ax1_1_s_6 < 2) {
        A_shared_dyn_local[ax1_1_s_6] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_6) + 384)];
      }
    }
    for (int ax1_0_6 = 0; ax1_0_6 < 3; ++ax1_0_6) {
      *(float4*)(B_shared_dyn_local + (ax1_0_6 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_6 * 4)) >> 6) * 64)) + (((ax1_0_6 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_6 * 4)) >> 3)) & 7) * 4)) + 2816) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_60 = 0; i_2_1_s_60 < 4; ++i_2_1_s_60) {
      if (i_2_1_s_60 < 2) {
        Y_local[(i_2_1_s_60 * 12)] = (Y_local[(i_2_1_s_60 * 12)] + (A_shared_dyn_local[(i_2_1_s_60 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_61 = 0; i_2_1_s_61 < 4; ++i_2_1_s_61) {
      if (i_2_1_s_61 < 2) {
        Y_local[((i_2_1_s_61 * 12) + 1)] = (Y_local[((i_2_1_s_61 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_61 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_62 = 0; i_2_1_s_62 < 4; ++i_2_1_s_62) {
      if (i_2_1_s_62 < 2) {
        Y_local[((i_2_1_s_62 * 12) + 2)] = (Y_local[((i_2_1_s_62 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_62 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_63 = 0; i_2_1_s_63 < 4; ++i_2_1_s_63) {
      if (i_2_1_s_63 < 2) {
        Y_local[((i_2_1_s_63 * 12) + 3)] = (Y_local[((i_2_1_s_63 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_63 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int i_2_1_s_64 = 0; i_2_1_s_64 < 4; ++i_2_1_s_64) {
      if (i_2_1_s_64 < 2) {
        Y_local[((i_2_1_s_64 * 12) + 4)] = (Y_local[((i_2_1_s_64 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_64 + 2)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_65 = 0; i_2_1_s_65 < 4; ++i_2_1_s_65) {
      if (i_2_1_s_65 < 2) {
        Y_local[((i_2_1_s_65 * 12) + 5)] = (Y_local[((i_2_1_s_65 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_65 + 2)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_66 = 0; i_2_1_s_66 < 4; ++i_2_1_s_66) {
      if (i_2_1_s_66 < 2) {
        Y_local[((i_2_1_s_66 * 12) + 6)] = (Y_local[((i_2_1_s_66 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_66 + 2)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_67 = 0; i_2_1_s_67 < 4; ++i_2_1_s_67) {
      if (i_2_1_s_67 < 2) {
        Y_local[((i_2_1_s_67 * 12) + 7)] = (Y_local[((i_2_1_s_67 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_67 + 2)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_68 = 0; i_2_1_s_68 < 4; ++i_2_1_s_68) {
      if (i_2_1_s_68 < 2) {
        Y_local[((i_2_1_s_68 * 12) + 8)] = (Y_local[((i_2_1_s_68 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_68 + 2)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_69 = 0; i_2_1_s_69 < 4; ++i_2_1_s_69) {
      if (i_2_1_s_69 < 2) {
        Y_local[((i_2_1_s_69 * 12) + 9)] = (Y_local[((i_2_1_s_69 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_69 + 2)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_70 = 0; i_2_1_s_70 < 4; ++i_2_1_s_70) {
      if (i_2_1_s_70 < 2) {
        Y_local[((i_2_1_s_70 * 12) + 10)] = (Y_local[((i_2_1_s_70 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_70 + 2)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_71 = 0; i_2_1_s_71 < 4; ++i_2_1_s_71) {
      if (i_2_1_s_71 < 2) {
        Y_local[((i_2_1_s_71 * 12) + 11)] = (Y_local[((i_2_1_s_71 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_71 + 2)] * B_shared_dyn_local[23]));
      }
    }
    for (int ax1_1_s_7 = 0; ax1_1_s_7 < 4; ++ax1_1_s_7) {
      if (ax1_1_s_7 < 2) {
        A_shared_dyn_local[(ax1_1_s_7 + 2)] = ((float*)buf_dyn_shmem)[((((((((k_0 & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_7) + 448)];
      }
    }
    for (int ax1_0_7 = 0; ax1_0_7 < 3; ++ax1_0_7) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_7 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_7 * 4)) >> 6) * 64)) + (((ax1_0_7 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_7 * 4)) >> 3)) & 7) * 4)) + 2944) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_72 = 0; i_2_1_s_72 < 4; ++i_2_1_s_72) {
      if (i_2_1_s_72 < 2) {
        Y_local[(i_2_1_s_72 * 12)] = (Y_local[(i_2_1_s_72 * 12)] + (A_shared_dyn_local[i_2_1_s_72] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_73 = 0; i_2_1_s_73 < 4; ++i_2_1_s_73) {
      if (i_2_1_s_73 < 2) {
        Y_local[((i_2_1_s_73 * 12) + 1)] = (Y_local[((i_2_1_s_73 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_73] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_74 = 0; i_2_1_s_74 < 4; ++i_2_1_s_74) {
      if (i_2_1_s_74 < 2) {
        Y_local[((i_2_1_s_74 * 12) + 2)] = (Y_local[((i_2_1_s_74 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_74] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_75 = 0; i_2_1_s_75 < 4; ++i_2_1_s_75) {
      if (i_2_1_s_75 < 2) {
        Y_local[((i_2_1_s_75 * 12) + 3)] = (Y_local[((i_2_1_s_75 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_75] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_76 = 0; i_2_1_s_76 < 4; ++i_2_1_s_76) {
      if (i_2_1_s_76 < 2) {
        Y_local[((i_2_1_s_76 * 12) + 4)] = (Y_local[((i_2_1_s_76 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_76] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_77 = 0; i_2_1_s_77 < 4; ++i_2_1_s_77) {
      if (i_2_1_s_77 < 2) {
        Y_local[((i_2_1_s_77 * 12) + 5)] = (Y_local[((i_2_1_s_77 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_77] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_78 = 0; i_2_1_s_78 < 4; ++i_2_1_s_78) {
      if (i_2_1_s_78 < 2) {
        Y_local[((i_2_1_s_78 * 12) + 6)] = (Y_local[((i_2_1_s_78 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_78] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_79 = 0; i_2_1_s_79 < 4; ++i_2_1_s_79) {
      if (i_2_1_s_79 < 2) {
        Y_local[((i_2_1_s_79 * 12) + 7)] = (Y_local[((i_2_1_s_79 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_79] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_80 = 0; i_2_1_s_80 < 4; ++i_2_1_s_80) {
      if (i_2_1_s_80 < 2) {
        Y_local[((i_2_1_s_80 * 12) + 8)] = (Y_local[((i_2_1_s_80 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_80] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_81 = 0; i_2_1_s_81 < 4; ++i_2_1_s_81) {
      if (i_2_1_s_81 < 2) {
        Y_local[((i_2_1_s_81 * 12) + 9)] = (Y_local[((i_2_1_s_81 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_81] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_82 = 0; i_2_1_s_82 < 4; ++i_2_1_s_82) {
      if (i_2_1_s_82 < 2) {
        Y_local[((i_2_1_s_82 * 12) + 10)] = (Y_local[((i_2_1_s_82 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_82] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_83 = 0; i_2_1_s_83 < 4; ++i_2_1_s_83) {
      if (i_2_1_s_83 < 2) {
        Y_local[((i_2_1_s_83 * 12) + 11)] = (Y_local[((i_2_1_s_83 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_83] * B_shared_dyn_local[11]));
      }
    }
    for (int ax1_1_s_8 = 0; ax1_1_s_8 < 4; ++ax1_1_s_8) {
      if (ax1_1_s_8 < 2) {
        A_shared_dyn_local[ax1_1_s_8] = ((float*)buf_dyn_shmem)[((((((((k_0 + 1) & 3) * 512) + (((((int)threadIdx.x) & 63) >> 5) * 32)) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_8)];
      }
    }
    for (int ax1_0_8 = 0; ax1_0_8 < 3; ++ax1_0_8) {
      *(float4*)(B_shared_dyn_local + (ax1_0_8 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 + 1) & 3) * 1024) + (((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_8 * 4)) >> 6) * 64)) + (((ax1_0_8 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_8 * 4)) >> 3)) & 7) * 4)) + 2048) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
    }
    for (int i_2_1_s_84 = 0; i_2_1_s_84 < 4; ++i_2_1_s_84) {
      if (i_2_1_s_84 < 2) {
        Y_local[(i_2_1_s_84 * 12)] = (Y_local[(i_2_1_s_84 * 12)] + (A_shared_dyn_local[(i_2_1_s_84 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_85 = 0; i_2_1_s_85 < 4; ++i_2_1_s_85) {
      if (i_2_1_s_85 < 2) {
        Y_local[((i_2_1_s_85 * 12) + 1)] = (Y_local[((i_2_1_s_85 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_85 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_86 = 0; i_2_1_s_86 < 4; ++i_2_1_s_86) {
      if (i_2_1_s_86 < 2) {
        Y_local[((i_2_1_s_86 * 12) + 2)] = (Y_local[((i_2_1_s_86 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_86 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_87 = 0; i_2_1_s_87 < 4; ++i_2_1_s_87) {
      if (i_2_1_s_87 < 2) {
        Y_local[((i_2_1_s_87 * 12) + 3)] = (Y_local[((i_2_1_s_87 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_87 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int i_2_1_s_88 = 0; i_2_1_s_88 < 4; ++i_2_1_s_88) {
      if (i_2_1_s_88 < 2) {
        Y_local[((i_2_1_s_88 * 12) + 4)] = (Y_local[((i_2_1_s_88 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_88 + 2)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_89 = 0; i_2_1_s_89 < 4; ++i_2_1_s_89) {
      if (i_2_1_s_89 < 2) {
        Y_local[((i_2_1_s_89 * 12) + 5)] = (Y_local[((i_2_1_s_89 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_89 + 2)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_90 = 0; i_2_1_s_90 < 4; ++i_2_1_s_90) {
      if (i_2_1_s_90 < 2) {
        Y_local[((i_2_1_s_90 * 12) + 6)] = (Y_local[((i_2_1_s_90 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_90 + 2)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_91 = 0; i_2_1_s_91 < 4; ++i_2_1_s_91) {
      if (i_2_1_s_91 < 2) {
        Y_local[((i_2_1_s_91 * 12) + 7)] = (Y_local[((i_2_1_s_91 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_91 + 2)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_92 = 0; i_2_1_s_92 < 4; ++i_2_1_s_92) {
      if (i_2_1_s_92 < 2) {
        Y_local[((i_2_1_s_92 * 12) + 8)] = (Y_local[((i_2_1_s_92 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_92 + 2)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_93 = 0; i_2_1_s_93 < 4; ++i_2_1_s_93) {
      if (i_2_1_s_93 < 2) {
        Y_local[((i_2_1_s_93 * 12) + 9)] = (Y_local[((i_2_1_s_93 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_93 + 2)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_94 = 0; i_2_1_s_94 < 4; ++i_2_1_s_94) {
      if (i_2_1_s_94 < 2) {
        Y_local[((i_2_1_s_94 * 12) + 10)] = (Y_local[((i_2_1_s_94 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_94 + 2)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_95 = 0; i_2_1_s_95 < 4; ++i_2_1_s_95) {
      if (i_2_1_s_95 < 2) {
        Y_local[((i_2_1_s_95 * 12) + 11)] = (Y_local[((i_2_1_s_95 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_95 + 2)] * B_shared_dyn_local[23]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax1_1_s_9 = 0; ax1_1_s_9 < 4; ++ax1_1_s_9) {
    if (ax1_1_s_9 < 2) {
      A_shared_dyn_local[(ax1_1_s_9 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_9) + 576)];
    }
  }
  for (int ax1_0_9 = 0; ax1_0_9 < 3; ++ax1_0_9) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_9 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_9 * 4)) >> 6) * 64) + (((ax1_0_9 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_9 * 4)) >> 3)) & 7) * 4)) + 3200) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_96 = 0; i_2_1_s_96 < 4; ++i_2_1_s_96) {
    if (i_2_1_s_96 < 2) {
      Y_local[(i_2_1_s_96 * 12)] = (Y_local[(i_2_1_s_96 * 12)] + (A_shared_dyn_local[i_2_1_s_96] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_97 = 0; i_2_1_s_97 < 4; ++i_2_1_s_97) {
    if (i_2_1_s_97 < 2) {
      Y_local[((i_2_1_s_97 * 12) + 1)] = (Y_local[((i_2_1_s_97 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_97] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_98 = 0; i_2_1_s_98 < 4; ++i_2_1_s_98) {
    if (i_2_1_s_98 < 2) {
      Y_local[((i_2_1_s_98 * 12) + 2)] = (Y_local[((i_2_1_s_98 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_98] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_99 = 0; i_2_1_s_99 < 4; ++i_2_1_s_99) {
    if (i_2_1_s_99 < 2) {
      Y_local[((i_2_1_s_99 * 12) + 3)] = (Y_local[((i_2_1_s_99 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_99] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_100 = 0; i_2_1_s_100 < 4; ++i_2_1_s_100) {
    if (i_2_1_s_100 < 2) {
      Y_local[((i_2_1_s_100 * 12) + 4)] = (Y_local[((i_2_1_s_100 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_100] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_101 = 0; i_2_1_s_101 < 4; ++i_2_1_s_101) {
    if (i_2_1_s_101 < 2) {
      Y_local[((i_2_1_s_101 * 12) + 5)] = (Y_local[((i_2_1_s_101 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_101] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_102 = 0; i_2_1_s_102 < 4; ++i_2_1_s_102) {
    if (i_2_1_s_102 < 2) {
      Y_local[((i_2_1_s_102 * 12) + 6)] = (Y_local[((i_2_1_s_102 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_102] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_103 = 0; i_2_1_s_103 < 4; ++i_2_1_s_103) {
    if (i_2_1_s_103 < 2) {
      Y_local[((i_2_1_s_103 * 12) + 7)] = (Y_local[((i_2_1_s_103 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_103] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_104 = 0; i_2_1_s_104 < 4; ++i_2_1_s_104) {
    if (i_2_1_s_104 < 2) {
      Y_local[((i_2_1_s_104 * 12) + 8)] = (Y_local[((i_2_1_s_104 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_104] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_105 = 0; i_2_1_s_105 < 4; ++i_2_1_s_105) {
    if (i_2_1_s_105 < 2) {
      Y_local[((i_2_1_s_105 * 12) + 9)] = (Y_local[((i_2_1_s_105 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_105] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_106 = 0; i_2_1_s_106 < 4; ++i_2_1_s_106) {
    if (i_2_1_s_106 < 2) {
      Y_local[((i_2_1_s_106 * 12) + 10)] = (Y_local[((i_2_1_s_106 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_106] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_107 = 0; i_2_1_s_107 < 4; ++i_2_1_s_107) {
    if (i_2_1_s_107 < 2) {
      Y_local[((i_2_1_s_107 * 12) + 11)] = (Y_local[((i_2_1_s_107 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_107] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_10 = 0; ax1_1_s_10 < 4; ++ax1_1_s_10) {
    if (ax1_1_s_10 < 2) {
      A_shared_dyn_local[ax1_1_s_10] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_10) + 640)];
    }
  }
  for (int ax1_0_10 = 0; ax1_0_10 < 3; ++ax1_0_10) {
    *(float4*)(B_shared_dyn_local + (ax1_0_10 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_10 * 4)) >> 6) * 64) + (((ax1_0_10 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_10 * 4)) >> 3)) & 7) * 4)) + 3328) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_108 = 0; i_2_1_s_108 < 4; ++i_2_1_s_108) {
    if (i_2_1_s_108 < 2) {
      Y_local[(i_2_1_s_108 * 12)] = (Y_local[(i_2_1_s_108 * 12)] + (A_shared_dyn_local[(i_2_1_s_108 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_109 = 0; i_2_1_s_109 < 4; ++i_2_1_s_109) {
    if (i_2_1_s_109 < 2) {
      Y_local[((i_2_1_s_109 * 12) + 1)] = (Y_local[((i_2_1_s_109 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_109 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_110 = 0; i_2_1_s_110 < 4; ++i_2_1_s_110) {
    if (i_2_1_s_110 < 2) {
      Y_local[((i_2_1_s_110 * 12) + 2)] = (Y_local[((i_2_1_s_110 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_110 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_111 = 0; i_2_1_s_111 < 4; ++i_2_1_s_111) {
    if (i_2_1_s_111 < 2) {
      Y_local[((i_2_1_s_111 * 12) + 3)] = (Y_local[((i_2_1_s_111 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_111 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_112 = 0; i_2_1_s_112 < 4; ++i_2_1_s_112) {
    if (i_2_1_s_112 < 2) {
      Y_local[((i_2_1_s_112 * 12) + 4)] = (Y_local[((i_2_1_s_112 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_112 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_113 = 0; i_2_1_s_113 < 4; ++i_2_1_s_113) {
    if (i_2_1_s_113 < 2) {
      Y_local[((i_2_1_s_113 * 12) + 5)] = (Y_local[((i_2_1_s_113 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_113 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_114 = 0; i_2_1_s_114 < 4; ++i_2_1_s_114) {
    if (i_2_1_s_114 < 2) {
      Y_local[((i_2_1_s_114 * 12) + 6)] = (Y_local[((i_2_1_s_114 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_114 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_115 = 0; i_2_1_s_115 < 4; ++i_2_1_s_115) {
    if (i_2_1_s_115 < 2) {
      Y_local[((i_2_1_s_115 * 12) + 7)] = (Y_local[((i_2_1_s_115 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_115 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_116 = 0; i_2_1_s_116 < 4; ++i_2_1_s_116) {
    if (i_2_1_s_116 < 2) {
      Y_local[((i_2_1_s_116 * 12) + 8)] = (Y_local[((i_2_1_s_116 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_116 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_117 = 0; i_2_1_s_117 < 4; ++i_2_1_s_117) {
    if (i_2_1_s_117 < 2) {
      Y_local[((i_2_1_s_117 * 12) + 9)] = (Y_local[((i_2_1_s_117 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_117 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_118 = 0; i_2_1_s_118 < 4; ++i_2_1_s_118) {
    if (i_2_1_s_118 < 2) {
      Y_local[((i_2_1_s_118 * 12) + 10)] = (Y_local[((i_2_1_s_118 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_118 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_119 = 0; i_2_1_s_119 < 4; ++i_2_1_s_119) {
    if (i_2_1_s_119 < 2) {
      Y_local[((i_2_1_s_119 * 12) + 11)] = (Y_local[((i_2_1_s_119 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_119 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_11 = 0; ax1_1_s_11 < 4; ++ax1_1_s_11) {
    if (ax1_1_s_11 < 2) {
      A_shared_dyn_local[(ax1_1_s_11 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_11) + 704)];
    }
  }
  for (int ax1_0_11 = 0; ax1_0_11 < 3; ++ax1_0_11) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_11 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_11 * 4)) >> 6) * 64) + (((ax1_0_11 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_11 * 4)) >> 3)) & 7) * 4)) + 3456) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_120 = 0; i_2_1_s_120 < 4; ++i_2_1_s_120) {
    if (i_2_1_s_120 < 2) {
      Y_local[(i_2_1_s_120 * 12)] = (Y_local[(i_2_1_s_120 * 12)] + (A_shared_dyn_local[i_2_1_s_120] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_121 = 0; i_2_1_s_121 < 4; ++i_2_1_s_121) {
    if (i_2_1_s_121 < 2) {
      Y_local[((i_2_1_s_121 * 12) + 1)] = (Y_local[((i_2_1_s_121 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_121] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_122 = 0; i_2_1_s_122 < 4; ++i_2_1_s_122) {
    if (i_2_1_s_122 < 2) {
      Y_local[((i_2_1_s_122 * 12) + 2)] = (Y_local[((i_2_1_s_122 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_122] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_123 = 0; i_2_1_s_123 < 4; ++i_2_1_s_123) {
    if (i_2_1_s_123 < 2) {
      Y_local[((i_2_1_s_123 * 12) + 3)] = (Y_local[((i_2_1_s_123 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_123] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_124 = 0; i_2_1_s_124 < 4; ++i_2_1_s_124) {
    if (i_2_1_s_124 < 2) {
      Y_local[((i_2_1_s_124 * 12) + 4)] = (Y_local[((i_2_1_s_124 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_124] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_125 = 0; i_2_1_s_125 < 4; ++i_2_1_s_125) {
    if (i_2_1_s_125 < 2) {
      Y_local[((i_2_1_s_125 * 12) + 5)] = (Y_local[((i_2_1_s_125 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_125] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_126 = 0; i_2_1_s_126 < 4; ++i_2_1_s_126) {
    if (i_2_1_s_126 < 2) {
      Y_local[((i_2_1_s_126 * 12) + 6)] = (Y_local[((i_2_1_s_126 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_126] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_127 = 0; i_2_1_s_127 < 4; ++i_2_1_s_127) {
    if (i_2_1_s_127 < 2) {
      Y_local[((i_2_1_s_127 * 12) + 7)] = (Y_local[((i_2_1_s_127 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_127] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_128 = 0; i_2_1_s_128 < 4; ++i_2_1_s_128) {
    if (i_2_1_s_128 < 2) {
      Y_local[((i_2_1_s_128 * 12) + 8)] = (Y_local[((i_2_1_s_128 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_128] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_129 = 0; i_2_1_s_129 < 4; ++i_2_1_s_129) {
    if (i_2_1_s_129 < 2) {
      Y_local[((i_2_1_s_129 * 12) + 9)] = (Y_local[((i_2_1_s_129 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_129] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_130 = 0; i_2_1_s_130 < 4; ++i_2_1_s_130) {
    if (i_2_1_s_130 < 2) {
      Y_local[((i_2_1_s_130 * 12) + 10)] = (Y_local[((i_2_1_s_130 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_130] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_131 = 0; i_2_1_s_131 < 4; ++i_2_1_s_131) {
    if (i_2_1_s_131 < 2) {
      Y_local[((i_2_1_s_131 * 12) + 11)] = (Y_local[((i_2_1_s_131 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_131] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_12 = 0; ax1_1_s_12 < 4; ++ax1_1_s_12) {
    if (ax1_1_s_12 < 2) {
      A_shared_dyn_local[ax1_1_s_12] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_12) + 768)];
    }
  }
  for (int ax1_0_12 = 0; ax1_0_12 < 3; ++ax1_0_12) {
    *(float4*)(B_shared_dyn_local + (ax1_0_12 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_12 * 4)) >> 6) * 64) + (((ax1_0_12 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_12 * 4)) >> 3)) & 7) * 4)) + 3584) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_132 = 0; i_2_1_s_132 < 4; ++i_2_1_s_132) {
    if (i_2_1_s_132 < 2) {
      Y_local[(i_2_1_s_132 * 12)] = (Y_local[(i_2_1_s_132 * 12)] + (A_shared_dyn_local[(i_2_1_s_132 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_133 = 0; i_2_1_s_133 < 4; ++i_2_1_s_133) {
    if (i_2_1_s_133 < 2) {
      Y_local[((i_2_1_s_133 * 12) + 1)] = (Y_local[((i_2_1_s_133 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_133 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_134 = 0; i_2_1_s_134 < 4; ++i_2_1_s_134) {
    if (i_2_1_s_134 < 2) {
      Y_local[((i_2_1_s_134 * 12) + 2)] = (Y_local[((i_2_1_s_134 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_134 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_135 = 0; i_2_1_s_135 < 4; ++i_2_1_s_135) {
    if (i_2_1_s_135 < 2) {
      Y_local[((i_2_1_s_135 * 12) + 3)] = (Y_local[((i_2_1_s_135 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_135 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_136 = 0; i_2_1_s_136 < 4; ++i_2_1_s_136) {
    if (i_2_1_s_136 < 2) {
      Y_local[((i_2_1_s_136 * 12) + 4)] = (Y_local[((i_2_1_s_136 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_136 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_137 = 0; i_2_1_s_137 < 4; ++i_2_1_s_137) {
    if (i_2_1_s_137 < 2) {
      Y_local[((i_2_1_s_137 * 12) + 5)] = (Y_local[((i_2_1_s_137 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_137 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_138 = 0; i_2_1_s_138 < 4; ++i_2_1_s_138) {
    if (i_2_1_s_138 < 2) {
      Y_local[((i_2_1_s_138 * 12) + 6)] = (Y_local[((i_2_1_s_138 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_138 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_139 = 0; i_2_1_s_139 < 4; ++i_2_1_s_139) {
    if (i_2_1_s_139 < 2) {
      Y_local[((i_2_1_s_139 * 12) + 7)] = (Y_local[((i_2_1_s_139 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_139 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_140 = 0; i_2_1_s_140 < 4; ++i_2_1_s_140) {
    if (i_2_1_s_140 < 2) {
      Y_local[((i_2_1_s_140 * 12) + 8)] = (Y_local[((i_2_1_s_140 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_140 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_141 = 0; i_2_1_s_141 < 4; ++i_2_1_s_141) {
    if (i_2_1_s_141 < 2) {
      Y_local[((i_2_1_s_141 * 12) + 9)] = (Y_local[((i_2_1_s_141 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_141 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_142 = 0; i_2_1_s_142 < 4; ++i_2_1_s_142) {
    if (i_2_1_s_142 < 2) {
      Y_local[((i_2_1_s_142 * 12) + 10)] = (Y_local[((i_2_1_s_142 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_142 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_143 = 0; i_2_1_s_143 < 4; ++i_2_1_s_143) {
    if (i_2_1_s_143 < 2) {
      Y_local[((i_2_1_s_143 * 12) + 11)] = (Y_local[((i_2_1_s_143 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_143 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_13 = 0; ax1_1_s_13 < 4; ++ax1_1_s_13) {
    if (ax1_1_s_13 < 2) {
      A_shared_dyn_local[(ax1_1_s_13 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_13) + 832)];
    }
  }
  for (int ax1_0_13 = 0; ax1_0_13 < 3; ++ax1_0_13) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_13 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_13 * 4)) >> 6) * 64) + (((ax1_0_13 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_13 * 4)) >> 3)) & 7) * 4)) + 3712) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_144 = 0; i_2_1_s_144 < 4; ++i_2_1_s_144) {
    if (i_2_1_s_144 < 2) {
      Y_local[(i_2_1_s_144 * 12)] = (Y_local[(i_2_1_s_144 * 12)] + (A_shared_dyn_local[i_2_1_s_144] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_145 = 0; i_2_1_s_145 < 4; ++i_2_1_s_145) {
    if (i_2_1_s_145 < 2) {
      Y_local[((i_2_1_s_145 * 12) + 1)] = (Y_local[((i_2_1_s_145 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_145] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_146 = 0; i_2_1_s_146 < 4; ++i_2_1_s_146) {
    if (i_2_1_s_146 < 2) {
      Y_local[((i_2_1_s_146 * 12) + 2)] = (Y_local[((i_2_1_s_146 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_146] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_147 = 0; i_2_1_s_147 < 4; ++i_2_1_s_147) {
    if (i_2_1_s_147 < 2) {
      Y_local[((i_2_1_s_147 * 12) + 3)] = (Y_local[((i_2_1_s_147 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_147] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_148 = 0; i_2_1_s_148 < 4; ++i_2_1_s_148) {
    if (i_2_1_s_148 < 2) {
      Y_local[((i_2_1_s_148 * 12) + 4)] = (Y_local[((i_2_1_s_148 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_148] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_149 = 0; i_2_1_s_149 < 4; ++i_2_1_s_149) {
    if (i_2_1_s_149 < 2) {
      Y_local[((i_2_1_s_149 * 12) + 5)] = (Y_local[((i_2_1_s_149 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_149] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_150 = 0; i_2_1_s_150 < 4; ++i_2_1_s_150) {
    if (i_2_1_s_150 < 2) {
      Y_local[((i_2_1_s_150 * 12) + 6)] = (Y_local[((i_2_1_s_150 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_150] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_151 = 0; i_2_1_s_151 < 4; ++i_2_1_s_151) {
    if (i_2_1_s_151 < 2) {
      Y_local[((i_2_1_s_151 * 12) + 7)] = (Y_local[((i_2_1_s_151 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_151] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_152 = 0; i_2_1_s_152 < 4; ++i_2_1_s_152) {
    if (i_2_1_s_152 < 2) {
      Y_local[((i_2_1_s_152 * 12) + 8)] = (Y_local[((i_2_1_s_152 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_152] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_153 = 0; i_2_1_s_153 < 4; ++i_2_1_s_153) {
    if (i_2_1_s_153 < 2) {
      Y_local[((i_2_1_s_153 * 12) + 9)] = (Y_local[((i_2_1_s_153 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_153] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_154 = 0; i_2_1_s_154 < 4; ++i_2_1_s_154) {
    if (i_2_1_s_154 < 2) {
      Y_local[((i_2_1_s_154 * 12) + 10)] = (Y_local[((i_2_1_s_154 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_154] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_155 = 0; i_2_1_s_155 < 4; ++i_2_1_s_155) {
    if (i_2_1_s_155 < 2) {
      Y_local[((i_2_1_s_155 * 12) + 11)] = (Y_local[((i_2_1_s_155 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_155] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_14 = 0; ax1_1_s_14 < 4; ++ax1_1_s_14) {
    if (ax1_1_s_14 < 2) {
      A_shared_dyn_local[ax1_1_s_14] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_14) + 896)];
    }
  }
  for (int ax1_0_14 = 0; ax1_0_14 < 3; ++ax1_0_14) {
    *(float4*)(B_shared_dyn_local + (ax1_0_14 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_14 * 4)) >> 6) * 64) + (((ax1_0_14 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_14 * 4)) >> 3)) & 7) * 4)) + 3840) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_156 = 0; i_2_1_s_156 < 4; ++i_2_1_s_156) {
    if (i_2_1_s_156 < 2) {
      Y_local[(i_2_1_s_156 * 12)] = (Y_local[(i_2_1_s_156 * 12)] + (A_shared_dyn_local[(i_2_1_s_156 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_157 = 0; i_2_1_s_157 < 4; ++i_2_1_s_157) {
    if (i_2_1_s_157 < 2) {
      Y_local[((i_2_1_s_157 * 12) + 1)] = (Y_local[((i_2_1_s_157 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_157 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_158 = 0; i_2_1_s_158 < 4; ++i_2_1_s_158) {
    if (i_2_1_s_158 < 2) {
      Y_local[((i_2_1_s_158 * 12) + 2)] = (Y_local[((i_2_1_s_158 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_158 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_159 = 0; i_2_1_s_159 < 4; ++i_2_1_s_159) {
    if (i_2_1_s_159 < 2) {
      Y_local[((i_2_1_s_159 * 12) + 3)] = (Y_local[((i_2_1_s_159 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_159 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_160 = 0; i_2_1_s_160 < 4; ++i_2_1_s_160) {
    if (i_2_1_s_160 < 2) {
      Y_local[((i_2_1_s_160 * 12) + 4)] = (Y_local[((i_2_1_s_160 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_160 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_161 = 0; i_2_1_s_161 < 4; ++i_2_1_s_161) {
    if (i_2_1_s_161 < 2) {
      Y_local[((i_2_1_s_161 * 12) + 5)] = (Y_local[((i_2_1_s_161 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_161 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_162 = 0; i_2_1_s_162 < 4; ++i_2_1_s_162) {
    if (i_2_1_s_162 < 2) {
      Y_local[((i_2_1_s_162 * 12) + 6)] = (Y_local[((i_2_1_s_162 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_162 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_163 = 0; i_2_1_s_163 < 4; ++i_2_1_s_163) {
    if (i_2_1_s_163 < 2) {
      Y_local[((i_2_1_s_163 * 12) + 7)] = (Y_local[((i_2_1_s_163 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_163 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_164 = 0; i_2_1_s_164 < 4; ++i_2_1_s_164) {
    if (i_2_1_s_164 < 2) {
      Y_local[((i_2_1_s_164 * 12) + 8)] = (Y_local[((i_2_1_s_164 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_164 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_165 = 0; i_2_1_s_165 < 4; ++i_2_1_s_165) {
    if (i_2_1_s_165 < 2) {
      Y_local[((i_2_1_s_165 * 12) + 9)] = (Y_local[((i_2_1_s_165 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_165 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_166 = 0; i_2_1_s_166 < 4; ++i_2_1_s_166) {
    if (i_2_1_s_166 < 2) {
      Y_local[((i_2_1_s_166 * 12) + 10)] = (Y_local[((i_2_1_s_166 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_166 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_167 = 0; i_2_1_s_167 < 4; ++i_2_1_s_167) {
    if (i_2_1_s_167 < 2) {
      Y_local[((i_2_1_s_167 * 12) + 11)] = (Y_local[((i_2_1_s_167 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_167 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_15 = 0; ax1_1_s_15 < 4; ++ax1_1_s_15) {
    if (ax1_1_s_15 < 2) {
      A_shared_dyn_local[(ax1_1_s_15 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_15) + 960)];
    }
  }
  for (int ax1_0_15 = 0; ax1_0_15 < 3; ++ax1_0_15) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_15 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_15 * 4)) >> 6) * 64) + (((ax1_0_15 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_15 * 4)) >> 3)) & 7) * 4)) + 3968) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_168 = 0; i_2_1_s_168 < 4; ++i_2_1_s_168) {
    if (i_2_1_s_168 < 2) {
      Y_local[(i_2_1_s_168 * 12)] = (Y_local[(i_2_1_s_168 * 12)] + (A_shared_dyn_local[i_2_1_s_168] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_169 = 0; i_2_1_s_169 < 4; ++i_2_1_s_169) {
    if (i_2_1_s_169 < 2) {
      Y_local[((i_2_1_s_169 * 12) + 1)] = (Y_local[((i_2_1_s_169 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_169] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_170 = 0; i_2_1_s_170 < 4; ++i_2_1_s_170) {
    if (i_2_1_s_170 < 2) {
      Y_local[((i_2_1_s_170 * 12) + 2)] = (Y_local[((i_2_1_s_170 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_170] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_171 = 0; i_2_1_s_171 < 4; ++i_2_1_s_171) {
    if (i_2_1_s_171 < 2) {
      Y_local[((i_2_1_s_171 * 12) + 3)] = (Y_local[((i_2_1_s_171 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_171] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_172 = 0; i_2_1_s_172 < 4; ++i_2_1_s_172) {
    if (i_2_1_s_172 < 2) {
      Y_local[((i_2_1_s_172 * 12) + 4)] = (Y_local[((i_2_1_s_172 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_172] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_173 = 0; i_2_1_s_173 < 4; ++i_2_1_s_173) {
    if (i_2_1_s_173 < 2) {
      Y_local[((i_2_1_s_173 * 12) + 5)] = (Y_local[((i_2_1_s_173 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_173] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_174 = 0; i_2_1_s_174 < 4; ++i_2_1_s_174) {
    if (i_2_1_s_174 < 2) {
      Y_local[((i_2_1_s_174 * 12) + 6)] = (Y_local[((i_2_1_s_174 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_174] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_175 = 0; i_2_1_s_175 < 4; ++i_2_1_s_175) {
    if (i_2_1_s_175 < 2) {
      Y_local[((i_2_1_s_175 * 12) + 7)] = (Y_local[((i_2_1_s_175 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_175] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_176 = 0; i_2_1_s_176 < 4; ++i_2_1_s_176) {
    if (i_2_1_s_176 < 2) {
      Y_local[((i_2_1_s_176 * 12) + 8)] = (Y_local[((i_2_1_s_176 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_176] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_177 = 0; i_2_1_s_177 < 4; ++i_2_1_s_177) {
    if (i_2_1_s_177 < 2) {
      Y_local[((i_2_1_s_177 * 12) + 9)] = (Y_local[((i_2_1_s_177 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_177] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_178 = 0; i_2_1_s_178 < 4; ++i_2_1_s_178) {
    if (i_2_1_s_178 < 2) {
      Y_local[((i_2_1_s_178 * 12) + 10)] = (Y_local[((i_2_1_s_178 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_178] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_179 = 0; i_2_1_s_179 < 4; ++i_2_1_s_179) {
    if (i_2_1_s_179 < 2) {
      Y_local[((i_2_1_s_179 * 12) + 11)] = (Y_local[((i_2_1_s_179 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_179] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_16 = 0; ax1_1_s_16 < 4; ++ax1_1_s_16) {
    if (ax1_1_s_16 < 2) {
      A_shared_dyn_local[ax1_1_s_16] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_16) + 1024)];
    }
  }
  for (int ax1_0_16 = 0; ax1_0_16 < 3; ++ax1_0_16) {
    *(float4*)(B_shared_dyn_local + (ax1_0_16 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_16 * 4)) >> 6) * 64) + (((ax1_0_16 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_16 * 4)) >> 3)) & 7) * 4)) + 4096) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_180 = 0; i_2_1_s_180 < 4; ++i_2_1_s_180) {
    if (i_2_1_s_180 < 2) {
      Y_local[(i_2_1_s_180 * 12)] = (Y_local[(i_2_1_s_180 * 12)] + (A_shared_dyn_local[(i_2_1_s_180 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_181 = 0; i_2_1_s_181 < 4; ++i_2_1_s_181) {
    if (i_2_1_s_181 < 2) {
      Y_local[((i_2_1_s_181 * 12) + 1)] = (Y_local[((i_2_1_s_181 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_181 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_182 = 0; i_2_1_s_182 < 4; ++i_2_1_s_182) {
    if (i_2_1_s_182 < 2) {
      Y_local[((i_2_1_s_182 * 12) + 2)] = (Y_local[((i_2_1_s_182 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_182 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_183 = 0; i_2_1_s_183 < 4; ++i_2_1_s_183) {
    if (i_2_1_s_183 < 2) {
      Y_local[((i_2_1_s_183 * 12) + 3)] = (Y_local[((i_2_1_s_183 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_183 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_184 = 0; i_2_1_s_184 < 4; ++i_2_1_s_184) {
    if (i_2_1_s_184 < 2) {
      Y_local[((i_2_1_s_184 * 12) + 4)] = (Y_local[((i_2_1_s_184 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_184 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_185 = 0; i_2_1_s_185 < 4; ++i_2_1_s_185) {
    if (i_2_1_s_185 < 2) {
      Y_local[((i_2_1_s_185 * 12) + 5)] = (Y_local[((i_2_1_s_185 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_185 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_186 = 0; i_2_1_s_186 < 4; ++i_2_1_s_186) {
    if (i_2_1_s_186 < 2) {
      Y_local[((i_2_1_s_186 * 12) + 6)] = (Y_local[((i_2_1_s_186 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_186 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_187 = 0; i_2_1_s_187 < 4; ++i_2_1_s_187) {
    if (i_2_1_s_187 < 2) {
      Y_local[((i_2_1_s_187 * 12) + 7)] = (Y_local[((i_2_1_s_187 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_187 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_188 = 0; i_2_1_s_188 < 4; ++i_2_1_s_188) {
    if (i_2_1_s_188 < 2) {
      Y_local[((i_2_1_s_188 * 12) + 8)] = (Y_local[((i_2_1_s_188 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_188 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_189 = 0; i_2_1_s_189 < 4; ++i_2_1_s_189) {
    if (i_2_1_s_189 < 2) {
      Y_local[((i_2_1_s_189 * 12) + 9)] = (Y_local[((i_2_1_s_189 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_189 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_190 = 0; i_2_1_s_190 < 4; ++i_2_1_s_190) {
    if (i_2_1_s_190 < 2) {
      Y_local[((i_2_1_s_190 * 12) + 10)] = (Y_local[((i_2_1_s_190 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_190 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_191 = 0; i_2_1_s_191 < 4; ++i_2_1_s_191) {
    if (i_2_1_s_191 < 2) {
      Y_local[((i_2_1_s_191 * 12) + 11)] = (Y_local[((i_2_1_s_191 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_191 + 2)] * B_shared_dyn_local[23]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax1_1_s_17 = 0; ax1_1_s_17 < 4; ++ax1_1_s_17) {
    if (ax1_1_s_17 < 2) {
      A_shared_dyn_local[(ax1_1_s_17 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_17) + 1088)];
    }
  }
  for (int ax1_0_17 = 0; ax1_0_17 < 3; ++ax1_0_17) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_17 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_17 * 4)) >> 6) * 64) + (((ax1_0_17 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_17 * 4)) >> 3)) & 7) * 4)) + 4224) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_192 = 0; i_2_1_s_192 < 4; ++i_2_1_s_192) {
    if (i_2_1_s_192 < 2) {
      Y_local[(i_2_1_s_192 * 12)] = (Y_local[(i_2_1_s_192 * 12)] + (A_shared_dyn_local[i_2_1_s_192] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_193 = 0; i_2_1_s_193 < 4; ++i_2_1_s_193) {
    if (i_2_1_s_193 < 2) {
      Y_local[((i_2_1_s_193 * 12) + 1)] = (Y_local[((i_2_1_s_193 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_193] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_194 = 0; i_2_1_s_194 < 4; ++i_2_1_s_194) {
    if (i_2_1_s_194 < 2) {
      Y_local[((i_2_1_s_194 * 12) + 2)] = (Y_local[((i_2_1_s_194 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_194] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_195 = 0; i_2_1_s_195 < 4; ++i_2_1_s_195) {
    if (i_2_1_s_195 < 2) {
      Y_local[((i_2_1_s_195 * 12) + 3)] = (Y_local[((i_2_1_s_195 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_195] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_196 = 0; i_2_1_s_196 < 4; ++i_2_1_s_196) {
    if (i_2_1_s_196 < 2) {
      Y_local[((i_2_1_s_196 * 12) + 4)] = (Y_local[((i_2_1_s_196 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_196] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_197 = 0; i_2_1_s_197 < 4; ++i_2_1_s_197) {
    if (i_2_1_s_197 < 2) {
      Y_local[((i_2_1_s_197 * 12) + 5)] = (Y_local[((i_2_1_s_197 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_197] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_198 = 0; i_2_1_s_198 < 4; ++i_2_1_s_198) {
    if (i_2_1_s_198 < 2) {
      Y_local[((i_2_1_s_198 * 12) + 6)] = (Y_local[((i_2_1_s_198 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_198] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_199 = 0; i_2_1_s_199 < 4; ++i_2_1_s_199) {
    if (i_2_1_s_199 < 2) {
      Y_local[((i_2_1_s_199 * 12) + 7)] = (Y_local[((i_2_1_s_199 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_199] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_200 = 0; i_2_1_s_200 < 4; ++i_2_1_s_200) {
    if (i_2_1_s_200 < 2) {
      Y_local[((i_2_1_s_200 * 12) + 8)] = (Y_local[((i_2_1_s_200 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_200] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_201 = 0; i_2_1_s_201 < 4; ++i_2_1_s_201) {
    if (i_2_1_s_201 < 2) {
      Y_local[((i_2_1_s_201 * 12) + 9)] = (Y_local[((i_2_1_s_201 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_201] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_202 = 0; i_2_1_s_202 < 4; ++i_2_1_s_202) {
    if (i_2_1_s_202 < 2) {
      Y_local[((i_2_1_s_202 * 12) + 10)] = (Y_local[((i_2_1_s_202 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_202] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_203 = 0; i_2_1_s_203 < 4; ++i_2_1_s_203) {
    if (i_2_1_s_203 < 2) {
      Y_local[((i_2_1_s_203 * 12) + 11)] = (Y_local[((i_2_1_s_203 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_203] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_18 = 0; ax1_1_s_18 < 4; ++ax1_1_s_18) {
    if (ax1_1_s_18 < 2) {
      A_shared_dyn_local[ax1_1_s_18] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_18) + 1152)];
    }
  }
  for (int ax1_0_18 = 0; ax1_0_18 < 3; ++ax1_0_18) {
    *(float4*)(B_shared_dyn_local + (ax1_0_18 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_18 * 4)) >> 6) * 64) + (((ax1_0_18 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_18 * 4)) >> 3)) & 7) * 4)) + 4352) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_204 = 0; i_2_1_s_204 < 4; ++i_2_1_s_204) {
    if (i_2_1_s_204 < 2) {
      Y_local[(i_2_1_s_204 * 12)] = (Y_local[(i_2_1_s_204 * 12)] + (A_shared_dyn_local[(i_2_1_s_204 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_205 = 0; i_2_1_s_205 < 4; ++i_2_1_s_205) {
    if (i_2_1_s_205 < 2) {
      Y_local[((i_2_1_s_205 * 12) + 1)] = (Y_local[((i_2_1_s_205 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_205 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_206 = 0; i_2_1_s_206 < 4; ++i_2_1_s_206) {
    if (i_2_1_s_206 < 2) {
      Y_local[((i_2_1_s_206 * 12) + 2)] = (Y_local[((i_2_1_s_206 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_206 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_207 = 0; i_2_1_s_207 < 4; ++i_2_1_s_207) {
    if (i_2_1_s_207 < 2) {
      Y_local[((i_2_1_s_207 * 12) + 3)] = (Y_local[((i_2_1_s_207 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_207 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_208 = 0; i_2_1_s_208 < 4; ++i_2_1_s_208) {
    if (i_2_1_s_208 < 2) {
      Y_local[((i_2_1_s_208 * 12) + 4)] = (Y_local[((i_2_1_s_208 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_208 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_209 = 0; i_2_1_s_209 < 4; ++i_2_1_s_209) {
    if (i_2_1_s_209 < 2) {
      Y_local[((i_2_1_s_209 * 12) + 5)] = (Y_local[((i_2_1_s_209 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_209 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_210 = 0; i_2_1_s_210 < 4; ++i_2_1_s_210) {
    if (i_2_1_s_210 < 2) {
      Y_local[((i_2_1_s_210 * 12) + 6)] = (Y_local[((i_2_1_s_210 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_210 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_211 = 0; i_2_1_s_211 < 4; ++i_2_1_s_211) {
    if (i_2_1_s_211 < 2) {
      Y_local[((i_2_1_s_211 * 12) + 7)] = (Y_local[((i_2_1_s_211 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_211 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_212 = 0; i_2_1_s_212 < 4; ++i_2_1_s_212) {
    if (i_2_1_s_212 < 2) {
      Y_local[((i_2_1_s_212 * 12) + 8)] = (Y_local[((i_2_1_s_212 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_212 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_213 = 0; i_2_1_s_213 < 4; ++i_2_1_s_213) {
    if (i_2_1_s_213 < 2) {
      Y_local[((i_2_1_s_213 * 12) + 9)] = (Y_local[((i_2_1_s_213 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_213 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_214 = 0; i_2_1_s_214 < 4; ++i_2_1_s_214) {
    if (i_2_1_s_214 < 2) {
      Y_local[((i_2_1_s_214 * 12) + 10)] = (Y_local[((i_2_1_s_214 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_214 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_215 = 0; i_2_1_s_215 < 4; ++i_2_1_s_215) {
    if (i_2_1_s_215 < 2) {
      Y_local[((i_2_1_s_215 * 12) + 11)] = (Y_local[((i_2_1_s_215 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_215 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_19 = 0; ax1_1_s_19 < 4; ++ax1_1_s_19) {
    if (ax1_1_s_19 < 2) {
      A_shared_dyn_local[(ax1_1_s_19 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_19) + 1216)];
    }
  }
  for (int ax1_0_19 = 0; ax1_0_19 < 3; ++ax1_0_19) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_19 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_19 * 4)) >> 6) * 64) + (((ax1_0_19 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_19 * 4)) >> 3)) & 7) * 4)) + 4480) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_216 = 0; i_2_1_s_216 < 4; ++i_2_1_s_216) {
    if (i_2_1_s_216 < 2) {
      Y_local[(i_2_1_s_216 * 12)] = (Y_local[(i_2_1_s_216 * 12)] + (A_shared_dyn_local[i_2_1_s_216] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_217 = 0; i_2_1_s_217 < 4; ++i_2_1_s_217) {
    if (i_2_1_s_217 < 2) {
      Y_local[((i_2_1_s_217 * 12) + 1)] = (Y_local[((i_2_1_s_217 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_217] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_218 = 0; i_2_1_s_218 < 4; ++i_2_1_s_218) {
    if (i_2_1_s_218 < 2) {
      Y_local[((i_2_1_s_218 * 12) + 2)] = (Y_local[((i_2_1_s_218 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_218] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_219 = 0; i_2_1_s_219 < 4; ++i_2_1_s_219) {
    if (i_2_1_s_219 < 2) {
      Y_local[((i_2_1_s_219 * 12) + 3)] = (Y_local[((i_2_1_s_219 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_219] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_220 = 0; i_2_1_s_220 < 4; ++i_2_1_s_220) {
    if (i_2_1_s_220 < 2) {
      Y_local[((i_2_1_s_220 * 12) + 4)] = (Y_local[((i_2_1_s_220 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_220] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_221 = 0; i_2_1_s_221 < 4; ++i_2_1_s_221) {
    if (i_2_1_s_221 < 2) {
      Y_local[((i_2_1_s_221 * 12) + 5)] = (Y_local[((i_2_1_s_221 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_221] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_222 = 0; i_2_1_s_222 < 4; ++i_2_1_s_222) {
    if (i_2_1_s_222 < 2) {
      Y_local[((i_2_1_s_222 * 12) + 6)] = (Y_local[((i_2_1_s_222 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_222] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_223 = 0; i_2_1_s_223 < 4; ++i_2_1_s_223) {
    if (i_2_1_s_223 < 2) {
      Y_local[((i_2_1_s_223 * 12) + 7)] = (Y_local[((i_2_1_s_223 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_223] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_224 = 0; i_2_1_s_224 < 4; ++i_2_1_s_224) {
    if (i_2_1_s_224 < 2) {
      Y_local[((i_2_1_s_224 * 12) + 8)] = (Y_local[((i_2_1_s_224 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_224] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_225 = 0; i_2_1_s_225 < 4; ++i_2_1_s_225) {
    if (i_2_1_s_225 < 2) {
      Y_local[((i_2_1_s_225 * 12) + 9)] = (Y_local[((i_2_1_s_225 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_225] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_226 = 0; i_2_1_s_226 < 4; ++i_2_1_s_226) {
    if (i_2_1_s_226 < 2) {
      Y_local[((i_2_1_s_226 * 12) + 10)] = (Y_local[((i_2_1_s_226 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_226] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_227 = 0; i_2_1_s_227 < 4; ++i_2_1_s_227) {
    if (i_2_1_s_227 < 2) {
      Y_local[((i_2_1_s_227 * 12) + 11)] = (Y_local[((i_2_1_s_227 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_227] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_20 = 0; ax1_1_s_20 < 4; ++ax1_1_s_20) {
    if (ax1_1_s_20 < 2) {
      A_shared_dyn_local[ax1_1_s_20] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_20) + 1280)];
    }
  }
  for (int ax1_0_20 = 0; ax1_0_20 < 3; ++ax1_0_20) {
    *(float4*)(B_shared_dyn_local + (ax1_0_20 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_20 * 4)) >> 6) * 64) + (((ax1_0_20 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_20 * 4)) >> 3)) & 7) * 4)) + 4608) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_228 = 0; i_2_1_s_228 < 4; ++i_2_1_s_228) {
    if (i_2_1_s_228 < 2) {
      Y_local[(i_2_1_s_228 * 12)] = (Y_local[(i_2_1_s_228 * 12)] + (A_shared_dyn_local[(i_2_1_s_228 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_229 = 0; i_2_1_s_229 < 4; ++i_2_1_s_229) {
    if (i_2_1_s_229 < 2) {
      Y_local[((i_2_1_s_229 * 12) + 1)] = (Y_local[((i_2_1_s_229 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_229 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_230 = 0; i_2_1_s_230 < 4; ++i_2_1_s_230) {
    if (i_2_1_s_230 < 2) {
      Y_local[((i_2_1_s_230 * 12) + 2)] = (Y_local[((i_2_1_s_230 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_230 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_231 = 0; i_2_1_s_231 < 4; ++i_2_1_s_231) {
    if (i_2_1_s_231 < 2) {
      Y_local[((i_2_1_s_231 * 12) + 3)] = (Y_local[((i_2_1_s_231 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_231 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_232 = 0; i_2_1_s_232 < 4; ++i_2_1_s_232) {
    if (i_2_1_s_232 < 2) {
      Y_local[((i_2_1_s_232 * 12) + 4)] = (Y_local[((i_2_1_s_232 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_232 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_233 = 0; i_2_1_s_233 < 4; ++i_2_1_s_233) {
    if (i_2_1_s_233 < 2) {
      Y_local[((i_2_1_s_233 * 12) + 5)] = (Y_local[((i_2_1_s_233 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_233 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_234 = 0; i_2_1_s_234 < 4; ++i_2_1_s_234) {
    if (i_2_1_s_234 < 2) {
      Y_local[((i_2_1_s_234 * 12) + 6)] = (Y_local[((i_2_1_s_234 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_234 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_235 = 0; i_2_1_s_235 < 4; ++i_2_1_s_235) {
    if (i_2_1_s_235 < 2) {
      Y_local[((i_2_1_s_235 * 12) + 7)] = (Y_local[((i_2_1_s_235 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_235 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_236 = 0; i_2_1_s_236 < 4; ++i_2_1_s_236) {
    if (i_2_1_s_236 < 2) {
      Y_local[((i_2_1_s_236 * 12) + 8)] = (Y_local[((i_2_1_s_236 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_236 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_237 = 0; i_2_1_s_237 < 4; ++i_2_1_s_237) {
    if (i_2_1_s_237 < 2) {
      Y_local[((i_2_1_s_237 * 12) + 9)] = (Y_local[((i_2_1_s_237 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_237 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_238 = 0; i_2_1_s_238 < 4; ++i_2_1_s_238) {
    if (i_2_1_s_238 < 2) {
      Y_local[((i_2_1_s_238 * 12) + 10)] = (Y_local[((i_2_1_s_238 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_238 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_239 = 0; i_2_1_s_239 < 4; ++i_2_1_s_239) {
    if (i_2_1_s_239 < 2) {
      Y_local[((i_2_1_s_239 * 12) + 11)] = (Y_local[((i_2_1_s_239 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_239 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_21 = 0; ax1_1_s_21 < 4; ++ax1_1_s_21) {
    if (ax1_1_s_21 < 2) {
      A_shared_dyn_local[(ax1_1_s_21 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_21) + 1344)];
    }
  }
  for (int ax1_0_21 = 0; ax1_0_21 < 3; ++ax1_0_21) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_21 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_21 * 4)) >> 6) * 64) + (((ax1_0_21 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_21 * 4)) >> 3)) & 7) * 4)) + 4736) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_240 = 0; i_2_1_s_240 < 4; ++i_2_1_s_240) {
    if (i_2_1_s_240 < 2) {
      Y_local[(i_2_1_s_240 * 12)] = (Y_local[(i_2_1_s_240 * 12)] + (A_shared_dyn_local[i_2_1_s_240] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_241 = 0; i_2_1_s_241 < 4; ++i_2_1_s_241) {
    if (i_2_1_s_241 < 2) {
      Y_local[((i_2_1_s_241 * 12) + 1)] = (Y_local[((i_2_1_s_241 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_241] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_242 = 0; i_2_1_s_242 < 4; ++i_2_1_s_242) {
    if (i_2_1_s_242 < 2) {
      Y_local[((i_2_1_s_242 * 12) + 2)] = (Y_local[((i_2_1_s_242 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_242] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_243 = 0; i_2_1_s_243 < 4; ++i_2_1_s_243) {
    if (i_2_1_s_243 < 2) {
      Y_local[((i_2_1_s_243 * 12) + 3)] = (Y_local[((i_2_1_s_243 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_243] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_244 = 0; i_2_1_s_244 < 4; ++i_2_1_s_244) {
    if (i_2_1_s_244 < 2) {
      Y_local[((i_2_1_s_244 * 12) + 4)] = (Y_local[((i_2_1_s_244 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_244] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_245 = 0; i_2_1_s_245 < 4; ++i_2_1_s_245) {
    if (i_2_1_s_245 < 2) {
      Y_local[((i_2_1_s_245 * 12) + 5)] = (Y_local[((i_2_1_s_245 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_245] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_246 = 0; i_2_1_s_246 < 4; ++i_2_1_s_246) {
    if (i_2_1_s_246 < 2) {
      Y_local[((i_2_1_s_246 * 12) + 6)] = (Y_local[((i_2_1_s_246 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_246] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_247 = 0; i_2_1_s_247 < 4; ++i_2_1_s_247) {
    if (i_2_1_s_247 < 2) {
      Y_local[((i_2_1_s_247 * 12) + 7)] = (Y_local[((i_2_1_s_247 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_247] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_248 = 0; i_2_1_s_248 < 4; ++i_2_1_s_248) {
    if (i_2_1_s_248 < 2) {
      Y_local[((i_2_1_s_248 * 12) + 8)] = (Y_local[((i_2_1_s_248 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_248] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_249 = 0; i_2_1_s_249 < 4; ++i_2_1_s_249) {
    if (i_2_1_s_249 < 2) {
      Y_local[((i_2_1_s_249 * 12) + 9)] = (Y_local[((i_2_1_s_249 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_249] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_250 = 0; i_2_1_s_250 < 4; ++i_2_1_s_250) {
    if (i_2_1_s_250 < 2) {
      Y_local[((i_2_1_s_250 * 12) + 10)] = (Y_local[((i_2_1_s_250 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_250] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_251 = 0; i_2_1_s_251 < 4; ++i_2_1_s_251) {
    if (i_2_1_s_251 < 2) {
      Y_local[((i_2_1_s_251 * 12) + 11)] = (Y_local[((i_2_1_s_251 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_251] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_22 = 0; ax1_1_s_22 < 4; ++ax1_1_s_22) {
    if (ax1_1_s_22 < 2) {
      A_shared_dyn_local[ax1_1_s_22] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_22) + 1408)];
    }
  }
  for (int ax1_0_22 = 0; ax1_0_22 < 3; ++ax1_0_22) {
    *(float4*)(B_shared_dyn_local + (ax1_0_22 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_22 * 4)) >> 6) * 64) + (((ax1_0_22 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_22 * 4)) >> 3)) & 7) * 4)) + 4864) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_252 = 0; i_2_1_s_252 < 4; ++i_2_1_s_252) {
    if (i_2_1_s_252 < 2) {
      Y_local[(i_2_1_s_252 * 12)] = (Y_local[(i_2_1_s_252 * 12)] + (A_shared_dyn_local[(i_2_1_s_252 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_253 = 0; i_2_1_s_253 < 4; ++i_2_1_s_253) {
    if (i_2_1_s_253 < 2) {
      Y_local[((i_2_1_s_253 * 12) + 1)] = (Y_local[((i_2_1_s_253 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_253 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_254 = 0; i_2_1_s_254 < 4; ++i_2_1_s_254) {
    if (i_2_1_s_254 < 2) {
      Y_local[((i_2_1_s_254 * 12) + 2)] = (Y_local[((i_2_1_s_254 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_254 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_255 = 0; i_2_1_s_255 < 4; ++i_2_1_s_255) {
    if (i_2_1_s_255 < 2) {
      Y_local[((i_2_1_s_255 * 12) + 3)] = (Y_local[((i_2_1_s_255 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_255 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_256 = 0; i_2_1_s_256 < 4; ++i_2_1_s_256) {
    if (i_2_1_s_256 < 2) {
      Y_local[((i_2_1_s_256 * 12) + 4)] = (Y_local[((i_2_1_s_256 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_256 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_257 = 0; i_2_1_s_257 < 4; ++i_2_1_s_257) {
    if (i_2_1_s_257 < 2) {
      Y_local[((i_2_1_s_257 * 12) + 5)] = (Y_local[((i_2_1_s_257 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_257 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_258 = 0; i_2_1_s_258 < 4; ++i_2_1_s_258) {
    if (i_2_1_s_258 < 2) {
      Y_local[((i_2_1_s_258 * 12) + 6)] = (Y_local[((i_2_1_s_258 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_258 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_259 = 0; i_2_1_s_259 < 4; ++i_2_1_s_259) {
    if (i_2_1_s_259 < 2) {
      Y_local[((i_2_1_s_259 * 12) + 7)] = (Y_local[((i_2_1_s_259 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_259 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_260 = 0; i_2_1_s_260 < 4; ++i_2_1_s_260) {
    if (i_2_1_s_260 < 2) {
      Y_local[((i_2_1_s_260 * 12) + 8)] = (Y_local[((i_2_1_s_260 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_260 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_261 = 0; i_2_1_s_261 < 4; ++i_2_1_s_261) {
    if (i_2_1_s_261 < 2) {
      Y_local[((i_2_1_s_261 * 12) + 9)] = (Y_local[((i_2_1_s_261 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_261 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_262 = 0; i_2_1_s_262 < 4; ++i_2_1_s_262) {
    if (i_2_1_s_262 < 2) {
      Y_local[((i_2_1_s_262 * 12) + 10)] = (Y_local[((i_2_1_s_262 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_262 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_263 = 0; i_2_1_s_263 < 4; ++i_2_1_s_263) {
    if (i_2_1_s_263 < 2) {
      Y_local[((i_2_1_s_263 * 12) + 11)] = (Y_local[((i_2_1_s_263 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_263 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_23 = 0; ax1_1_s_23 < 4; ++ax1_1_s_23) {
    if (ax1_1_s_23 < 2) {
      A_shared_dyn_local[(ax1_1_s_23 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_23) + 1472)];
    }
  }
  for (int ax1_0_23 = 0; ax1_0_23 < 3; ++ax1_0_23) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_23 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_23 * 4)) >> 6) * 64) + (((ax1_0_23 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_23 * 4)) >> 3)) & 7) * 4)) + 4992) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_264 = 0; i_2_1_s_264 < 4; ++i_2_1_s_264) {
    if (i_2_1_s_264 < 2) {
      Y_local[(i_2_1_s_264 * 12)] = (Y_local[(i_2_1_s_264 * 12)] + (A_shared_dyn_local[i_2_1_s_264] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_265 = 0; i_2_1_s_265 < 4; ++i_2_1_s_265) {
    if (i_2_1_s_265 < 2) {
      Y_local[((i_2_1_s_265 * 12) + 1)] = (Y_local[((i_2_1_s_265 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_265] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_266 = 0; i_2_1_s_266 < 4; ++i_2_1_s_266) {
    if (i_2_1_s_266 < 2) {
      Y_local[((i_2_1_s_266 * 12) + 2)] = (Y_local[((i_2_1_s_266 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_266] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_267 = 0; i_2_1_s_267 < 4; ++i_2_1_s_267) {
    if (i_2_1_s_267 < 2) {
      Y_local[((i_2_1_s_267 * 12) + 3)] = (Y_local[((i_2_1_s_267 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_267] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_268 = 0; i_2_1_s_268 < 4; ++i_2_1_s_268) {
    if (i_2_1_s_268 < 2) {
      Y_local[((i_2_1_s_268 * 12) + 4)] = (Y_local[((i_2_1_s_268 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_268] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_269 = 0; i_2_1_s_269 < 4; ++i_2_1_s_269) {
    if (i_2_1_s_269 < 2) {
      Y_local[((i_2_1_s_269 * 12) + 5)] = (Y_local[((i_2_1_s_269 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_269] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_270 = 0; i_2_1_s_270 < 4; ++i_2_1_s_270) {
    if (i_2_1_s_270 < 2) {
      Y_local[((i_2_1_s_270 * 12) + 6)] = (Y_local[((i_2_1_s_270 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_270] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_271 = 0; i_2_1_s_271 < 4; ++i_2_1_s_271) {
    if (i_2_1_s_271 < 2) {
      Y_local[((i_2_1_s_271 * 12) + 7)] = (Y_local[((i_2_1_s_271 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_271] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_272 = 0; i_2_1_s_272 < 4; ++i_2_1_s_272) {
    if (i_2_1_s_272 < 2) {
      Y_local[((i_2_1_s_272 * 12) + 8)] = (Y_local[((i_2_1_s_272 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_272] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_273 = 0; i_2_1_s_273 < 4; ++i_2_1_s_273) {
    if (i_2_1_s_273 < 2) {
      Y_local[((i_2_1_s_273 * 12) + 9)] = (Y_local[((i_2_1_s_273 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_273] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_274 = 0; i_2_1_s_274 < 4; ++i_2_1_s_274) {
    if (i_2_1_s_274 < 2) {
      Y_local[((i_2_1_s_274 * 12) + 10)] = (Y_local[((i_2_1_s_274 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_274] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_275 = 0; i_2_1_s_275 < 4; ++i_2_1_s_275) {
    if (i_2_1_s_275 < 2) {
      Y_local[((i_2_1_s_275 * 12) + 11)] = (Y_local[((i_2_1_s_275 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_275] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_24 = 0; ax1_1_s_24 < 4; ++ax1_1_s_24) {
    if (ax1_1_s_24 < 2) {
      A_shared_dyn_local[ax1_1_s_24] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_24) + 1536)];
    }
  }
  for (int ax1_0_24 = 0; ax1_0_24 < 3; ++ax1_0_24) {
    *(float4*)(B_shared_dyn_local + (ax1_0_24 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_24 * 4)) >> 6) * 64) + (((ax1_0_24 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_24 * 4)) >> 3)) & 7) * 4)) + 5120) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_276 = 0; i_2_1_s_276 < 4; ++i_2_1_s_276) {
    if (i_2_1_s_276 < 2) {
      Y_local[(i_2_1_s_276 * 12)] = (Y_local[(i_2_1_s_276 * 12)] + (A_shared_dyn_local[(i_2_1_s_276 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_277 = 0; i_2_1_s_277 < 4; ++i_2_1_s_277) {
    if (i_2_1_s_277 < 2) {
      Y_local[((i_2_1_s_277 * 12) + 1)] = (Y_local[((i_2_1_s_277 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_277 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_278 = 0; i_2_1_s_278 < 4; ++i_2_1_s_278) {
    if (i_2_1_s_278 < 2) {
      Y_local[((i_2_1_s_278 * 12) + 2)] = (Y_local[((i_2_1_s_278 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_278 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_279 = 0; i_2_1_s_279 < 4; ++i_2_1_s_279) {
    if (i_2_1_s_279 < 2) {
      Y_local[((i_2_1_s_279 * 12) + 3)] = (Y_local[((i_2_1_s_279 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_279 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_280 = 0; i_2_1_s_280 < 4; ++i_2_1_s_280) {
    if (i_2_1_s_280 < 2) {
      Y_local[((i_2_1_s_280 * 12) + 4)] = (Y_local[((i_2_1_s_280 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_280 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_281 = 0; i_2_1_s_281 < 4; ++i_2_1_s_281) {
    if (i_2_1_s_281 < 2) {
      Y_local[((i_2_1_s_281 * 12) + 5)] = (Y_local[((i_2_1_s_281 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_281 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_282 = 0; i_2_1_s_282 < 4; ++i_2_1_s_282) {
    if (i_2_1_s_282 < 2) {
      Y_local[((i_2_1_s_282 * 12) + 6)] = (Y_local[((i_2_1_s_282 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_282 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_283 = 0; i_2_1_s_283 < 4; ++i_2_1_s_283) {
    if (i_2_1_s_283 < 2) {
      Y_local[((i_2_1_s_283 * 12) + 7)] = (Y_local[((i_2_1_s_283 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_283 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_284 = 0; i_2_1_s_284 < 4; ++i_2_1_s_284) {
    if (i_2_1_s_284 < 2) {
      Y_local[((i_2_1_s_284 * 12) + 8)] = (Y_local[((i_2_1_s_284 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_284 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_285 = 0; i_2_1_s_285 < 4; ++i_2_1_s_285) {
    if (i_2_1_s_285 < 2) {
      Y_local[((i_2_1_s_285 * 12) + 9)] = (Y_local[((i_2_1_s_285 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_285 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_286 = 0; i_2_1_s_286 < 4; ++i_2_1_s_286) {
    if (i_2_1_s_286 < 2) {
      Y_local[((i_2_1_s_286 * 12) + 10)] = (Y_local[((i_2_1_s_286 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_286 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_287 = 0; i_2_1_s_287 < 4; ++i_2_1_s_287) {
    if (i_2_1_s_287 < 2) {
      Y_local[((i_2_1_s_287 * 12) + 11)] = (Y_local[((i_2_1_s_287 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_287 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_25 = 0; ax1_1_s_25 < 4; ++ax1_1_s_25) {
    if (ax1_1_s_25 < 2) {
      A_shared_dyn_local[(ax1_1_s_25 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_25) + 1600)];
    }
  }
  for (int ax1_0_25 = 0; ax1_0_25 < 3; ++ax1_0_25) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_25 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_25 * 4)) >> 6) * 64) + (((ax1_0_25 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_25 * 4)) >> 3)) & 7) * 4)) + 5248) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_288 = 0; i_2_1_s_288 < 4; ++i_2_1_s_288) {
    if (i_2_1_s_288 < 2) {
      Y_local[(i_2_1_s_288 * 12)] = (Y_local[(i_2_1_s_288 * 12)] + (A_shared_dyn_local[i_2_1_s_288] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_289 = 0; i_2_1_s_289 < 4; ++i_2_1_s_289) {
    if (i_2_1_s_289 < 2) {
      Y_local[((i_2_1_s_289 * 12) + 1)] = (Y_local[((i_2_1_s_289 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_289] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_290 = 0; i_2_1_s_290 < 4; ++i_2_1_s_290) {
    if (i_2_1_s_290 < 2) {
      Y_local[((i_2_1_s_290 * 12) + 2)] = (Y_local[((i_2_1_s_290 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_290] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_291 = 0; i_2_1_s_291 < 4; ++i_2_1_s_291) {
    if (i_2_1_s_291 < 2) {
      Y_local[((i_2_1_s_291 * 12) + 3)] = (Y_local[((i_2_1_s_291 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_291] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_292 = 0; i_2_1_s_292 < 4; ++i_2_1_s_292) {
    if (i_2_1_s_292 < 2) {
      Y_local[((i_2_1_s_292 * 12) + 4)] = (Y_local[((i_2_1_s_292 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_292] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_293 = 0; i_2_1_s_293 < 4; ++i_2_1_s_293) {
    if (i_2_1_s_293 < 2) {
      Y_local[((i_2_1_s_293 * 12) + 5)] = (Y_local[((i_2_1_s_293 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_293] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_294 = 0; i_2_1_s_294 < 4; ++i_2_1_s_294) {
    if (i_2_1_s_294 < 2) {
      Y_local[((i_2_1_s_294 * 12) + 6)] = (Y_local[((i_2_1_s_294 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_294] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_295 = 0; i_2_1_s_295 < 4; ++i_2_1_s_295) {
    if (i_2_1_s_295 < 2) {
      Y_local[((i_2_1_s_295 * 12) + 7)] = (Y_local[((i_2_1_s_295 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_295] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_296 = 0; i_2_1_s_296 < 4; ++i_2_1_s_296) {
    if (i_2_1_s_296 < 2) {
      Y_local[((i_2_1_s_296 * 12) + 8)] = (Y_local[((i_2_1_s_296 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_296] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_297 = 0; i_2_1_s_297 < 4; ++i_2_1_s_297) {
    if (i_2_1_s_297 < 2) {
      Y_local[((i_2_1_s_297 * 12) + 9)] = (Y_local[((i_2_1_s_297 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_297] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_298 = 0; i_2_1_s_298 < 4; ++i_2_1_s_298) {
    if (i_2_1_s_298 < 2) {
      Y_local[((i_2_1_s_298 * 12) + 10)] = (Y_local[((i_2_1_s_298 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_298] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_299 = 0; i_2_1_s_299 < 4; ++i_2_1_s_299) {
    if (i_2_1_s_299 < 2) {
      Y_local[((i_2_1_s_299 * 12) + 11)] = (Y_local[((i_2_1_s_299 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_299] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_26 = 0; ax1_1_s_26 < 4; ++ax1_1_s_26) {
    if (ax1_1_s_26 < 2) {
      A_shared_dyn_local[ax1_1_s_26] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_26) + 1664)];
    }
  }
  for (int ax1_0_26 = 0; ax1_0_26 < 3; ++ax1_0_26) {
    *(float4*)(B_shared_dyn_local + (ax1_0_26 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_26 * 4)) >> 6) * 64) + (((ax1_0_26 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_26 * 4)) >> 3)) & 7) * 4)) + 5376) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_300 = 0; i_2_1_s_300 < 4; ++i_2_1_s_300) {
    if (i_2_1_s_300 < 2) {
      Y_local[(i_2_1_s_300 * 12)] = (Y_local[(i_2_1_s_300 * 12)] + (A_shared_dyn_local[(i_2_1_s_300 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_301 = 0; i_2_1_s_301 < 4; ++i_2_1_s_301) {
    if (i_2_1_s_301 < 2) {
      Y_local[((i_2_1_s_301 * 12) + 1)] = (Y_local[((i_2_1_s_301 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_301 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_302 = 0; i_2_1_s_302 < 4; ++i_2_1_s_302) {
    if (i_2_1_s_302 < 2) {
      Y_local[((i_2_1_s_302 * 12) + 2)] = (Y_local[((i_2_1_s_302 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_302 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_303 = 0; i_2_1_s_303 < 4; ++i_2_1_s_303) {
    if (i_2_1_s_303 < 2) {
      Y_local[((i_2_1_s_303 * 12) + 3)] = (Y_local[((i_2_1_s_303 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_303 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_304 = 0; i_2_1_s_304 < 4; ++i_2_1_s_304) {
    if (i_2_1_s_304 < 2) {
      Y_local[((i_2_1_s_304 * 12) + 4)] = (Y_local[((i_2_1_s_304 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_304 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_305 = 0; i_2_1_s_305 < 4; ++i_2_1_s_305) {
    if (i_2_1_s_305 < 2) {
      Y_local[((i_2_1_s_305 * 12) + 5)] = (Y_local[((i_2_1_s_305 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_305 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_306 = 0; i_2_1_s_306 < 4; ++i_2_1_s_306) {
    if (i_2_1_s_306 < 2) {
      Y_local[((i_2_1_s_306 * 12) + 6)] = (Y_local[((i_2_1_s_306 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_306 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_307 = 0; i_2_1_s_307 < 4; ++i_2_1_s_307) {
    if (i_2_1_s_307 < 2) {
      Y_local[((i_2_1_s_307 * 12) + 7)] = (Y_local[((i_2_1_s_307 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_307 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_308 = 0; i_2_1_s_308 < 4; ++i_2_1_s_308) {
    if (i_2_1_s_308 < 2) {
      Y_local[((i_2_1_s_308 * 12) + 8)] = (Y_local[((i_2_1_s_308 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_308 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_309 = 0; i_2_1_s_309 < 4; ++i_2_1_s_309) {
    if (i_2_1_s_309 < 2) {
      Y_local[((i_2_1_s_309 * 12) + 9)] = (Y_local[((i_2_1_s_309 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_309 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_310 = 0; i_2_1_s_310 < 4; ++i_2_1_s_310) {
    if (i_2_1_s_310 < 2) {
      Y_local[((i_2_1_s_310 * 12) + 10)] = (Y_local[((i_2_1_s_310 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_310 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_311 = 0; i_2_1_s_311 < 4; ++i_2_1_s_311) {
    if (i_2_1_s_311 < 2) {
      Y_local[((i_2_1_s_311 * 12) + 11)] = (Y_local[((i_2_1_s_311 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_311 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_27 = 0; ax1_1_s_27 < 4; ++ax1_1_s_27) {
    if (ax1_1_s_27 < 2) {
      A_shared_dyn_local[(ax1_1_s_27 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_27) + 1728)];
    }
  }
  for (int ax1_0_27 = 0; ax1_0_27 < 3; ++ax1_0_27) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_27 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_27 * 4)) >> 6) * 64) + (((ax1_0_27 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_27 * 4)) >> 3)) & 7) * 4)) + 5504) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_312 = 0; i_2_1_s_312 < 4; ++i_2_1_s_312) {
    if (i_2_1_s_312 < 2) {
      Y_local[(i_2_1_s_312 * 12)] = (Y_local[(i_2_1_s_312 * 12)] + (A_shared_dyn_local[i_2_1_s_312] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_313 = 0; i_2_1_s_313 < 4; ++i_2_1_s_313) {
    if (i_2_1_s_313 < 2) {
      Y_local[((i_2_1_s_313 * 12) + 1)] = (Y_local[((i_2_1_s_313 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_313] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_314 = 0; i_2_1_s_314 < 4; ++i_2_1_s_314) {
    if (i_2_1_s_314 < 2) {
      Y_local[((i_2_1_s_314 * 12) + 2)] = (Y_local[((i_2_1_s_314 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_314] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_315 = 0; i_2_1_s_315 < 4; ++i_2_1_s_315) {
    if (i_2_1_s_315 < 2) {
      Y_local[((i_2_1_s_315 * 12) + 3)] = (Y_local[((i_2_1_s_315 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_315] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_316 = 0; i_2_1_s_316 < 4; ++i_2_1_s_316) {
    if (i_2_1_s_316 < 2) {
      Y_local[((i_2_1_s_316 * 12) + 4)] = (Y_local[((i_2_1_s_316 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_316] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_317 = 0; i_2_1_s_317 < 4; ++i_2_1_s_317) {
    if (i_2_1_s_317 < 2) {
      Y_local[((i_2_1_s_317 * 12) + 5)] = (Y_local[((i_2_1_s_317 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_317] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_318 = 0; i_2_1_s_318 < 4; ++i_2_1_s_318) {
    if (i_2_1_s_318 < 2) {
      Y_local[((i_2_1_s_318 * 12) + 6)] = (Y_local[((i_2_1_s_318 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_318] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_319 = 0; i_2_1_s_319 < 4; ++i_2_1_s_319) {
    if (i_2_1_s_319 < 2) {
      Y_local[((i_2_1_s_319 * 12) + 7)] = (Y_local[((i_2_1_s_319 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_319] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_320 = 0; i_2_1_s_320 < 4; ++i_2_1_s_320) {
    if (i_2_1_s_320 < 2) {
      Y_local[((i_2_1_s_320 * 12) + 8)] = (Y_local[((i_2_1_s_320 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_320] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_321 = 0; i_2_1_s_321 < 4; ++i_2_1_s_321) {
    if (i_2_1_s_321 < 2) {
      Y_local[((i_2_1_s_321 * 12) + 9)] = (Y_local[((i_2_1_s_321 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_321] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_322 = 0; i_2_1_s_322 < 4; ++i_2_1_s_322) {
    if (i_2_1_s_322 < 2) {
      Y_local[((i_2_1_s_322 * 12) + 10)] = (Y_local[((i_2_1_s_322 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_322] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_323 = 0; i_2_1_s_323 < 4; ++i_2_1_s_323) {
    if (i_2_1_s_323 < 2) {
      Y_local[((i_2_1_s_323 * 12) + 11)] = (Y_local[((i_2_1_s_323 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_323] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_28 = 0; ax1_1_s_28 < 4; ++ax1_1_s_28) {
    if (ax1_1_s_28 < 2) {
      A_shared_dyn_local[ax1_1_s_28] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_28) + 1792)];
    }
  }
  for (int ax1_0_28 = 0; ax1_0_28 < 3; ++ax1_0_28) {
    *(float4*)(B_shared_dyn_local + (ax1_0_28 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_28 * 4)) >> 6) * 64) + (((ax1_0_28 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_28 * 4)) >> 3)) & 7) * 4)) + 5632) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_324 = 0; i_2_1_s_324 < 4; ++i_2_1_s_324) {
    if (i_2_1_s_324 < 2) {
      Y_local[(i_2_1_s_324 * 12)] = (Y_local[(i_2_1_s_324 * 12)] + (A_shared_dyn_local[(i_2_1_s_324 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_325 = 0; i_2_1_s_325 < 4; ++i_2_1_s_325) {
    if (i_2_1_s_325 < 2) {
      Y_local[((i_2_1_s_325 * 12) + 1)] = (Y_local[((i_2_1_s_325 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_325 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_326 = 0; i_2_1_s_326 < 4; ++i_2_1_s_326) {
    if (i_2_1_s_326 < 2) {
      Y_local[((i_2_1_s_326 * 12) + 2)] = (Y_local[((i_2_1_s_326 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_326 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_327 = 0; i_2_1_s_327 < 4; ++i_2_1_s_327) {
    if (i_2_1_s_327 < 2) {
      Y_local[((i_2_1_s_327 * 12) + 3)] = (Y_local[((i_2_1_s_327 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_327 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_328 = 0; i_2_1_s_328 < 4; ++i_2_1_s_328) {
    if (i_2_1_s_328 < 2) {
      Y_local[((i_2_1_s_328 * 12) + 4)] = (Y_local[((i_2_1_s_328 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_328 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_329 = 0; i_2_1_s_329 < 4; ++i_2_1_s_329) {
    if (i_2_1_s_329 < 2) {
      Y_local[((i_2_1_s_329 * 12) + 5)] = (Y_local[((i_2_1_s_329 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_329 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_330 = 0; i_2_1_s_330 < 4; ++i_2_1_s_330) {
    if (i_2_1_s_330 < 2) {
      Y_local[((i_2_1_s_330 * 12) + 6)] = (Y_local[((i_2_1_s_330 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_330 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_331 = 0; i_2_1_s_331 < 4; ++i_2_1_s_331) {
    if (i_2_1_s_331 < 2) {
      Y_local[((i_2_1_s_331 * 12) + 7)] = (Y_local[((i_2_1_s_331 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_331 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_332 = 0; i_2_1_s_332 < 4; ++i_2_1_s_332) {
    if (i_2_1_s_332 < 2) {
      Y_local[((i_2_1_s_332 * 12) + 8)] = (Y_local[((i_2_1_s_332 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_332 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_333 = 0; i_2_1_s_333 < 4; ++i_2_1_s_333) {
    if (i_2_1_s_333 < 2) {
      Y_local[((i_2_1_s_333 * 12) + 9)] = (Y_local[((i_2_1_s_333 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_333 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_334 = 0; i_2_1_s_334 < 4; ++i_2_1_s_334) {
    if (i_2_1_s_334 < 2) {
      Y_local[((i_2_1_s_334 * 12) + 10)] = (Y_local[((i_2_1_s_334 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_334 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_335 = 0; i_2_1_s_335 < 4; ++i_2_1_s_335) {
    if (i_2_1_s_335 < 2) {
      Y_local[((i_2_1_s_335 * 12) + 11)] = (Y_local[((i_2_1_s_335 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_335 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_29 = 0; ax1_1_s_29 < 4; ++ax1_1_s_29) {
    if (ax1_1_s_29 < 2) {
      A_shared_dyn_local[(ax1_1_s_29 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_29) + 1856)];
    }
  }
  for (int ax1_0_29 = 0; ax1_0_29 < 3; ++ax1_0_29) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_29 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_29 * 4)) >> 6) * 64) + (((ax1_0_29 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_29 * 4)) >> 3)) & 7) * 4)) + 5760) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_336 = 0; i_2_1_s_336 < 4; ++i_2_1_s_336) {
    if (i_2_1_s_336 < 2) {
      Y_local[(i_2_1_s_336 * 12)] = (Y_local[(i_2_1_s_336 * 12)] + (A_shared_dyn_local[i_2_1_s_336] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_337 = 0; i_2_1_s_337 < 4; ++i_2_1_s_337) {
    if (i_2_1_s_337 < 2) {
      Y_local[((i_2_1_s_337 * 12) + 1)] = (Y_local[((i_2_1_s_337 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_337] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_338 = 0; i_2_1_s_338 < 4; ++i_2_1_s_338) {
    if (i_2_1_s_338 < 2) {
      Y_local[((i_2_1_s_338 * 12) + 2)] = (Y_local[((i_2_1_s_338 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_338] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_339 = 0; i_2_1_s_339 < 4; ++i_2_1_s_339) {
    if (i_2_1_s_339 < 2) {
      Y_local[((i_2_1_s_339 * 12) + 3)] = (Y_local[((i_2_1_s_339 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_339] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_340 = 0; i_2_1_s_340 < 4; ++i_2_1_s_340) {
    if (i_2_1_s_340 < 2) {
      Y_local[((i_2_1_s_340 * 12) + 4)] = (Y_local[((i_2_1_s_340 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_340] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_341 = 0; i_2_1_s_341 < 4; ++i_2_1_s_341) {
    if (i_2_1_s_341 < 2) {
      Y_local[((i_2_1_s_341 * 12) + 5)] = (Y_local[((i_2_1_s_341 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_341] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_342 = 0; i_2_1_s_342 < 4; ++i_2_1_s_342) {
    if (i_2_1_s_342 < 2) {
      Y_local[((i_2_1_s_342 * 12) + 6)] = (Y_local[((i_2_1_s_342 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_342] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_343 = 0; i_2_1_s_343 < 4; ++i_2_1_s_343) {
    if (i_2_1_s_343 < 2) {
      Y_local[((i_2_1_s_343 * 12) + 7)] = (Y_local[((i_2_1_s_343 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_343] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_344 = 0; i_2_1_s_344 < 4; ++i_2_1_s_344) {
    if (i_2_1_s_344 < 2) {
      Y_local[((i_2_1_s_344 * 12) + 8)] = (Y_local[((i_2_1_s_344 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_344] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_345 = 0; i_2_1_s_345 < 4; ++i_2_1_s_345) {
    if (i_2_1_s_345 < 2) {
      Y_local[((i_2_1_s_345 * 12) + 9)] = (Y_local[((i_2_1_s_345 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_345] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_346 = 0; i_2_1_s_346 < 4; ++i_2_1_s_346) {
    if (i_2_1_s_346 < 2) {
      Y_local[((i_2_1_s_346 * 12) + 10)] = (Y_local[((i_2_1_s_346 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_346] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_347 = 0; i_2_1_s_347 < 4; ++i_2_1_s_347) {
    if (i_2_1_s_347 < 2) {
      Y_local[((i_2_1_s_347 * 12) + 11)] = (Y_local[((i_2_1_s_347 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_347] * B_shared_dyn_local[11]));
    }
  }
  for (int ax1_1_s_30 = 0; ax1_1_s_30 < 4; ++ax1_1_s_30) {
    if (ax1_1_s_30 < 2) {
      A_shared_dyn_local[ax1_1_s_30] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_30) + 1920)];
    }
  }
  for (int ax1_0_30 = 0; ax1_0_30 < 3; ++ax1_0_30) {
    *(float4*)(B_shared_dyn_local + (ax1_0_30 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_30 * 4)) >> 6) * 64) + (((ax1_0_30 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_30 * 4)) >> 3)) & 7) * 4)) + 5888) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_348 = 0; i_2_1_s_348 < 4; ++i_2_1_s_348) {
    if (i_2_1_s_348 < 2) {
      Y_local[(i_2_1_s_348 * 12)] = (Y_local[(i_2_1_s_348 * 12)] + (A_shared_dyn_local[(i_2_1_s_348 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_349 = 0; i_2_1_s_349 < 4; ++i_2_1_s_349) {
    if (i_2_1_s_349 < 2) {
      Y_local[((i_2_1_s_349 * 12) + 1)] = (Y_local[((i_2_1_s_349 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_349 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_350 = 0; i_2_1_s_350 < 4; ++i_2_1_s_350) {
    if (i_2_1_s_350 < 2) {
      Y_local[((i_2_1_s_350 * 12) + 2)] = (Y_local[((i_2_1_s_350 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_350 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_351 = 0; i_2_1_s_351 < 4; ++i_2_1_s_351) {
    if (i_2_1_s_351 < 2) {
      Y_local[((i_2_1_s_351 * 12) + 3)] = (Y_local[((i_2_1_s_351 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_351 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_352 = 0; i_2_1_s_352 < 4; ++i_2_1_s_352) {
    if (i_2_1_s_352 < 2) {
      Y_local[((i_2_1_s_352 * 12) + 4)] = (Y_local[((i_2_1_s_352 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_352 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_353 = 0; i_2_1_s_353 < 4; ++i_2_1_s_353) {
    if (i_2_1_s_353 < 2) {
      Y_local[((i_2_1_s_353 * 12) + 5)] = (Y_local[((i_2_1_s_353 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_353 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_354 = 0; i_2_1_s_354 < 4; ++i_2_1_s_354) {
    if (i_2_1_s_354 < 2) {
      Y_local[((i_2_1_s_354 * 12) + 6)] = (Y_local[((i_2_1_s_354 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_354 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_355 = 0; i_2_1_s_355 < 4; ++i_2_1_s_355) {
    if (i_2_1_s_355 < 2) {
      Y_local[((i_2_1_s_355 * 12) + 7)] = (Y_local[((i_2_1_s_355 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_355 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_356 = 0; i_2_1_s_356 < 4; ++i_2_1_s_356) {
    if (i_2_1_s_356 < 2) {
      Y_local[((i_2_1_s_356 * 12) + 8)] = (Y_local[((i_2_1_s_356 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_356 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_357 = 0; i_2_1_s_357 < 4; ++i_2_1_s_357) {
    if (i_2_1_s_357 < 2) {
      Y_local[((i_2_1_s_357 * 12) + 9)] = (Y_local[((i_2_1_s_357 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_357 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_358 = 0; i_2_1_s_358 < 4; ++i_2_1_s_358) {
    if (i_2_1_s_358 < 2) {
      Y_local[((i_2_1_s_358 * 12) + 10)] = (Y_local[((i_2_1_s_358 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_358 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_359 = 0; i_2_1_s_359 < 4; ++i_2_1_s_359) {
    if (i_2_1_s_359 < 2) {
      Y_local[((i_2_1_s_359 * 12) + 11)] = (Y_local[((i_2_1_s_359 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_359 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_1_s_31 = 0; ax1_1_s_31 < 4; ++ax1_1_s_31) {
    if (ax1_1_s_31 < 2) {
      A_shared_dyn_local[(ax1_1_s_31 + 2)] = ((float*)buf_dyn_shmem)[((((((((((int)threadIdx.x) & 63) >> 5) * 32) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + (((((int)threadIdx.x) & 3) >> 1) * 2)) + ax1_1_s_31) + 1984)];
    }
  }
  for (int ax1_0_31 = 0; ax1_0_31 < 3; ++ax1_0_31) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_31 * 4) + 12)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((((((int)blockIdx.x) & 7) * 48) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_31 * 4)) >> 6) * 64) + (((ax1_0_31 + (((int)threadIdx.x) & 1)) & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 6) + ((((int)threadIdx.x) >> 6) * 3)) + ((((((int)threadIdx.x) & 1) * 12) + (ax1_0_31 * 4)) >> 3)) & 7) * 4)) + 6016) - ((((((int)blockIdx.x) & 7) * 48) >> 6) * 64)));
  }
  for (int i_2_1_s_360 = 0; i_2_1_s_360 < 4; ++i_2_1_s_360) {
    if (i_2_1_s_360 < 2) {
      Y_local[(i_2_1_s_360 * 12)] = (Y_local[(i_2_1_s_360 * 12)] + (A_shared_dyn_local[i_2_1_s_360] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_361 = 0; i_2_1_s_361 < 4; ++i_2_1_s_361) {
    if (i_2_1_s_361 < 2) {
      Y_local[((i_2_1_s_361 * 12) + 1)] = (Y_local[((i_2_1_s_361 * 12) + 1)] + (A_shared_dyn_local[i_2_1_s_361] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_362 = 0; i_2_1_s_362 < 4; ++i_2_1_s_362) {
    if (i_2_1_s_362 < 2) {
      Y_local[((i_2_1_s_362 * 12) + 2)] = (Y_local[((i_2_1_s_362 * 12) + 2)] + (A_shared_dyn_local[i_2_1_s_362] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_363 = 0; i_2_1_s_363 < 4; ++i_2_1_s_363) {
    if (i_2_1_s_363 < 2) {
      Y_local[((i_2_1_s_363 * 12) + 3)] = (Y_local[((i_2_1_s_363 * 12) + 3)] + (A_shared_dyn_local[i_2_1_s_363] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_364 = 0; i_2_1_s_364 < 4; ++i_2_1_s_364) {
    if (i_2_1_s_364 < 2) {
      Y_local[((i_2_1_s_364 * 12) + 4)] = (Y_local[((i_2_1_s_364 * 12) + 4)] + (A_shared_dyn_local[i_2_1_s_364] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_365 = 0; i_2_1_s_365 < 4; ++i_2_1_s_365) {
    if (i_2_1_s_365 < 2) {
      Y_local[((i_2_1_s_365 * 12) + 5)] = (Y_local[((i_2_1_s_365 * 12) + 5)] + (A_shared_dyn_local[i_2_1_s_365] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_366 = 0; i_2_1_s_366 < 4; ++i_2_1_s_366) {
    if (i_2_1_s_366 < 2) {
      Y_local[((i_2_1_s_366 * 12) + 6)] = (Y_local[((i_2_1_s_366 * 12) + 6)] + (A_shared_dyn_local[i_2_1_s_366] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_367 = 0; i_2_1_s_367 < 4; ++i_2_1_s_367) {
    if (i_2_1_s_367 < 2) {
      Y_local[((i_2_1_s_367 * 12) + 7)] = (Y_local[((i_2_1_s_367 * 12) + 7)] + (A_shared_dyn_local[i_2_1_s_367] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_368 = 0; i_2_1_s_368 < 4; ++i_2_1_s_368) {
    if (i_2_1_s_368 < 2) {
      Y_local[((i_2_1_s_368 * 12) + 8)] = (Y_local[((i_2_1_s_368 * 12) + 8)] + (A_shared_dyn_local[i_2_1_s_368] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_369 = 0; i_2_1_s_369 < 4; ++i_2_1_s_369) {
    if (i_2_1_s_369 < 2) {
      Y_local[((i_2_1_s_369 * 12) + 9)] = (Y_local[((i_2_1_s_369 * 12) + 9)] + (A_shared_dyn_local[i_2_1_s_369] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_370 = 0; i_2_1_s_370 < 4; ++i_2_1_s_370) {
    if (i_2_1_s_370 < 2) {
      Y_local[((i_2_1_s_370 * 12) + 10)] = (Y_local[((i_2_1_s_370 * 12) + 10)] + (A_shared_dyn_local[i_2_1_s_370] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_371 = 0; i_2_1_s_371 < 4; ++i_2_1_s_371) {
    if (i_2_1_s_371 < 2) {
      Y_local[((i_2_1_s_371 * 12) + 11)] = (Y_local[((i_2_1_s_371 * 12) + 11)] + (A_shared_dyn_local[i_2_1_s_371] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_372 = 0; i_2_1_s_372 < 4; ++i_2_1_s_372) {
    if (i_2_1_s_372 < 2) {
      Y_local[(i_2_1_s_372 * 12)] = (Y_local[(i_2_1_s_372 * 12)] + (A_shared_dyn_local[(i_2_1_s_372 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_373 = 0; i_2_1_s_373 < 4; ++i_2_1_s_373) {
    if (i_2_1_s_373 < 2) {
      Y_local[((i_2_1_s_373 * 12) + 1)] = (Y_local[((i_2_1_s_373 * 12) + 1)] + (A_shared_dyn_local[(i_2_1_s_373 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_374 = 0; i_2_1_s_374 < 4; ++i_2_1_s_374) {
    if (i_2_1_s_374 < 2) {
      Y_local[((i_2_1_s_374 * 12) + 2)] = (Y_local[((i_2_1_s_374 * 12) + 2)] + (A_shared_dyn_local[(i_2_1_s_374 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_375 = 0; i_2_1_s_375 < 4; ++i_2_1_s_375) {
    if (i_2_1_s_375 < 2) {
      Y_local[((i_2_1_s_375 * 12) + 3)] = (Y_local[((i_2_1_s_375 * 12) + 3)] + (A_shared_dyn_local[(i_2_1_s_375 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_376 = 0; i_2_1_s_376 < 4; ++i_2_1_s_376) {
    if (i_2_1_s_376 < 2) {
      Y_local[((i_2_1_s_376 * 12) + 4)] = (Y_local[((i_2_1_s_376 * 12) + 4)] + (A_shared_dyn_local[(i_2_1_s_376 + 2)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_377 = 0; i_2_1_s_377 < 4; ++i_2_1_s_377) {
    if (i_2_1_s_377 < 2) {
      Y_local[((i_2_1_s_377 * 12) + 5)] = (Y_local[((i_2_1_s_377 * 12) + 5)] + (A_shared_dyn_local[(i_2_1_s_377 + 2)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_378 = 0; i_2_1_s_378 < 4; ++i_2_1_s_378) {
    if (i_2_1_s_378 < 2) {
      Y_local[((i_2_1_s_378 * 12) + 6)] = (Y_local[((i_2_1_s_378 * 12) + 6)] + (A_shared_dyn_local[(i_2_1_s_378 + 2)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_379 = 0; i_2_1_s_379 < 4; ++i_2_1_s_379) {
    if (i_2_1_s_379 < 2) {
      Y_local[((i_2_1_s_379 * 12) + 7)] = (Y_local[((i_2_1_s_379 * 12) + 7)] + (A_shared_dyn_local[(i_2_1_s_379 + 2)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_380 = 0; i_2_1_s_380 < 4; ++i_2_1_s_380) {
    if (i_2_1_s_380 < 2) {
      Y_local[((i_2_1_s_380 * 12) + 8)] = (Y_local[((i_2_1_s_380 * 12) + 8)] + (A_shared_dyn_local[(i_2_1_s_380 + 2)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_381 = 0; i_2_1_s_381 < 4; ++i_2_1_s_381) {
    if (i_2_1_s_381 < 2) {
      Y_local[((i_2_1_s_381 * 12) + 9)] = (Y_local[((i_2_1_s_381 * 12) + 9)] + (A_shared_dyn_local[(i_2_1_s_381 + 2)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_382 = 0; i_2_1_s_382 < 4; ++i_2_1_s_382) {
    if (i_2_1_s_382 < 2) {
      Y_local[((i_2_1_s_382 * 12) + 10)] = (Y_local[((i_2_1_s_382 * 12) + 10)] + (A_shared_dyn_local[(i_2_1_s_382 + 2)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_383 = 0; i_2_1_s_383 < 4; ++i_2_1_s_383) {
    if (i_2_1_s_383 < 2) {
      Y_local[((i_2_1_s_383 * 12) + 11)] = (Y_local[((i_2_1_s_383 * 12) + 11)] + (A_shared_dyn_local[(i_2_1_s_383 + 2)] * B_shared_dyn_local[23]));
    }
  }
  for (int ax1_0_32 = 0; ax1_0_32 < 3; ++ax1_0_32) {
    *(float4*)(Y + (((((((((int)blockIdx.x) >> 3) * 24576) + (((((int)threadIdx.x) & 63) >> 1) * 768)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_32 * 4))) = *(float4*)(Y_local + (ax1_0_32 * 4));
  }
  for (int ax1_0_33 = 0; ax1_0_33 < 3; ++ax1_0_33) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 24576) + (((((int)threadIdx.x) & 63) >> 1) * 768)) + ((((int)blockIdx.x) & 7) * 48)) + ((((int)threadIdx.x) >> 6) * 24)) + ((((int)threadIdx.x) & 1) * 12)) + (ax1_0_33 * 4)) + 384)) = *(float4*)(Y_local + ((ax1_0_33 * 4) + 12));
  }
}


