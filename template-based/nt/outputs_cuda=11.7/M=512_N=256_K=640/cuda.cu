
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[16];
  float A_shared_dyn_local[4];
  float B_shared_dyn_local[16];
  Y_local[0] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) >> 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 4; ++ax0_ax1_fused_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 3) * 192) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + (ax0_ax1_fused_2 * 4)) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax0_ax1_fused_2))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + 1024)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) >> 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 4096))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 4; ++ax0_ax1_fused_2_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 3) * 192) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + (ax0_ax1_fused_2_1 * 4)) + 5632)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax0_ax1_fused_2_1) + 2048))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 3) * 128) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)threadIdx.x) >> 3) * 512) + ((((int)blockIdx.x) >> 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 8192))), "n"(16)
    );
  }
  for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 4; ++ax0_ax1_fused_2_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 3) * 192) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + (ax0_ax1_fused_2_2 * 4)) + 7168)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) >> 3) * 256) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax0_ax1_fused_2_2) + 4096))), "n"(4)
    );
  }
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax1_1_s = 0; ax1_1_s < 4; ++ax1_1_s) {
    if (ax1_1_s < 2) {
      A_shared_dyn_local[ax1_1_s] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s)];
    }
  }
  for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) {
    *(float4*)(B_shared_dyn_local + (ax1_0 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0 * 4)) >> 6) * 64) + (ax1_0 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1024) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int k_0 = 0; k_0 < 77; ++k_0) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((k_0 + 3) & 3) * 1024) + ((((int)threadIdx.x) >> 3) * 128)) + ((((int)threadIdx.x) & 1) * 64)) + (((((int)threadIdx.x) & 7) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((k_0 * 4096) + ((((int)threadIdx.x) >> 3) * 512)) + ((((int)blockIdx.x) >> 3) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + 12288))), "n"(16)
    );
  }
    for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 4; ++ax0_ax1_fused_2_3) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((k_0 + 3) & 3) * 1536) + ((((int)threadIdx.x) >> 3) * 192)) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 7) >> 1) * 16)) + (ax0_ax1_fused_2_3 * 4)) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((k_0 * 2048) + ((((int)threadIdx.x) >> 3) * 256)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) & 7) * 4)) + ax0_ax1_fused_2_3) + 6144))), "n"(4)
    );
  }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int ax1_1_s_1 = 0; ax1_1_s_1 < 4; ++ax1_1_s_1) {
      if (ax1_1_s_1 < 2) {
        A_shared_dyn_local[(ax1_1_s_1 + 2)] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_1) + 32)];
      }
    }
    for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_1 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_1 * 4)) >> 6) * 64)) + (ax1_0_1 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1072) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s = 0; i_2_1_s < 4; ++i_2_1_s) {
      if (i_2_1_s < 2) {
        Y_local[(i_2_1_s * 8)] = (Y_local[(i_2_1_s * 8)] + (A_shared_dyn_local[i_2_1_s] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_1 = 0; i_2_1_s_1 < 4; ++i_2_1_s_1) {
      if (i_2_1_s_1 < 2) {
        Y_local[((i_2_1_s_1 * 8) + 1)] = (Y_local[((i_2_1_s_1 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_1] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_2 = 0; i_2_1_s_2 < 4; ++i_2_1_s_2) {
      if (i_2_1_s_2 < 2) {
        Y_local[((i_2_1_s_2 * 8) + 2)] = (Y_local[((i_2_1_s_2 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_2] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_3 = 0; i_2_1_s_3 < 4; ++i_2_1_s_3) {
      if (i_2_1_s_3 < 2) {
        Y_local[((i_2_1_s_3 * 8) + 3)] = (Y_local[((i_2_1_s_3 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_3] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_4 = 0; i_2_1_s_4 < 4; ++i_2_1_s_4) {
      if (i_2_1_s_4 < 2) {
        Y_local[((i_2_1_s_4 * 8) + 4)] = (Y_local[((i_2_1_s_4 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_4] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_5 = 0; i_2_1_s_5 < 4; ++i_2_1_s_5) {
      if (i_2_1_s_5 < 2) {
        Y_local[((i_2_1_s_5 * 8) + 5)] = (Y_local[((i_2_1_s_5 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_5] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_6 = 0; i_2_1_s_6 < 4; ++i_2_1_s_6) {
      if (i_2_1_s_6 < 2) {
        Y_local[((i_2_1_s_6 * 8) + 6)] = (Y_local[((i_2_1_s_6 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_6] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_7 = 0; i_2_1_s_7 < 4; ++i_2_1_s_7) {
      if (i_2_1_s_7 < 2) {
        Y_local[((i_2_1_s_7 * 8) + 7)] = (Y_local[((i_2_1_s_7 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_7] * B_shared_dyn_local[7]));
      }
    }
    for (int ax1_1_s_2 = 0; ax1_1_s_2 < 4; ++ax1_1_s_2) {
      if (ax1_1_s_2 < 2) {
        A_shared_dyn_local[ax1_1_s_2] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_2) + 64)];
      }
    }
    for (int ax1_0_2 = 0; ax1_0_2 < 2; ++ax1_0_2) {
      *(float4*)(B_shared_dyn_local + (ax1_0_2 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_2 * 4)) >> 6) * 64)) + (ax1_0_2 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1120) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_8 = 0; i_2_1_s_8 < 4; ++i_2_1_s_8) {
      if (i_2_1_s_8 < 2) {
        Y_local[(i_2_1_s_8 * 8)] = (Y_local[(i_2_1_s_8 * 8)] + (A_shared_dyn_local[(i_2_1_s_8 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_9 = 0; i_2_1_s_9 < 4; ++i_2_1_s_9) {
      if (i_2_1_s_9 < 2) {
        Y_local[((i_2_1_s_9 * 8) + 1)] = (Y_local[((i_2_1_s_9 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_9 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_10 = 0; i_2_1_s_10 < 4; ++i_2_1_s_10) {
      if (i_2_1_s_10 < 2) {
        Y_local[((i_2_1_s_10 * 8) + 2)] = (Y_local[((i_2_1_s_10 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_10 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_11 = 0; i_2_1_s_11 < 4; ++i_2_1_s_11) {
      if (i_2_1_s_11 < 2) {
        Y_local[((i_2_1_s_11 * 8) + 3)] = (Y_local[((i_2_1_s_11 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_11 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_12 = 0; i_2_1_s_12 < 4; ++i_2_1_s_12) {
      if (i_2_1_s_12 < 2) {
        Y_local[((i_2_1_s_12 * 8) + 4)] = (Y_local[((i_2_1_s_12 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_12 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_13 = 0; i_2_1_s_13 < 4; ++i_2_1_s_13) {
      if (i_2_1_s_13 < 2) {
        Y_local[((i_2_1_s_13 * 8) + 5)] = (Y_local[((i_2_1_s_13 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_13 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_14 = 0; i_2_1_s_14 < 4; ++i_2_1_s_14) {
      if (i_2_1_s_14 < 2) {
        Y_local[((i_2_1_s_14 * 8) + 6)] = (Y_local[((i_2_1_s_14 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_14 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_15 = 0; i_2_1_s_15 < 4; ++i_2_1_s_15) {
      if (i_2_1_s_15 < 2) {
        Y_local[((i_2_1_s_15 * 8) + 7)] = (Y_local[((i_2_1_s_15 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_15 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax1_1_s_3 = 0; ax1_1_s_3 < 4; ++ax1_1_s_3) {
      if (ax1_1_s_3 < 2) {
        A_shared_dyn_local[(ax1_1_s_3 + 2)] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_3) + 96)];
      }
    }
    for (int ax1_0_3 = 0; ax1_0_3 < 2; ++ax1_0_3) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_3 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_3 * 4)) >> 6) * 64)) + (ax1_0_3 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1168) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_16 = 0; i_2_1_s_16 < 4; ++i_2_1_s_16) {
      if (i_2_1_s_16 < 2) {
        Y_local[(i_2_1_s_16 * 8)] = (Y_local[(i_2_1_s_16 * 8)] + (A_shared_dyn_local[i_2_1_s_16] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_17 = 0; i_2_1_s_17 < 4; ++i_2_1_s_17) {
      if (i_2_1_s_17 < 2) {
        Y_local[((i_2_1_s_17 * 8) + 1)] = (Y_local[((i_2_1_s_17 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_17] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_18 = 0; i_2_1_s_18 < 4; ++i_2_1_s_18) {
      if (i_2_1_s_18 < 2) {
        Y_local[((i_2_1_s_18 * 8) + 2)] = (Y_local[((i_2_1_s_18 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_18] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_19 = 0; i_2_1_s_19 < 4; ++i_2_1_s_19) {
      if (i_2_1_s_19 < 2) {
        Y_local[((i_2_1_s_19 * 8) + 3)] = (Y_local[((i_2_1_s_19 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_19] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_20 = 0; i_2_1_s_20 < 4; ++i_2_1_s_20) {
      if (i_2_1_s_20 < 2) {
        Y_local[((i_2_1_s_20 * 8) + 4)] = (Y_local[((i_2_1_s_20 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_20] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_21 = 0; i_2_1_s_21 < 4; ++i_2_1_s_21) {
      if (i_2_1_s_21 < 2) {
        Y_local[((i_2_1_s_21 * 8) + 5)] = (Y_local[((i_2_1_s_21 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_21] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_22 = 0; i_2_1_s_22 < 4; ++i_2_1_s_22) {
      if (i_2_1_s_22 < 2) {
        Y_local[((i_2_1_s_22 * 8) + 6)] = (Y_local[((i_2_1_s_22 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_22] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_23 = 0; i_2_1_s_23 < 4; ++i_2_1_s_23) {
      if (i_2_1_s_23 < 2) {
        Y_local[((i_2_1_s_23 * 8) + 7)] = (Y_local[((i_2_1_s_23 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_23] * B_shared_dyn_local[7]));
      }
    }
    for (int ax1_1_s_4 = 0; ax1_1_s_4 < 4; ++ax1_1_s_4) {
      if (ax1_1_s_4 < 2) {
        A_shared_dyn_local[ax1_1_s_4] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_4) + 128)];
      }
    }
    for (int ax1_0_4 = 0; ax1_0_4 < 2; ++ax1_0_4) {
      *(float4*)(B_shared_dyn_local + (ax1_0_4 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_4 * 4)) >> 6) * 64)) + (ax1_0_4 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1216) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_24 = 0; i_2_1_s_24 < 4; ++i_2_1_s_24) {
      if (i_2_1_s_24 < 2) {
        Y_local[(i_2_1_s_24 * 8)] = (Y_local[(i_2_1_s_24 * 8)] + (A_shared_dyn_local[(i_2_1_s_24 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_25 = 0; i_2_1_s_25 < 4; ++i_2_1_s_25) {
      if (i_2_1_s_25 < 2) {
        Y_local[((i_2_1_s_25 * 8) + 1)] = (Y_local[((i_2_1_s_25 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_25 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_26 = 0; i_2_1_s_26 < 4; ++i_2_1_s_26) {
      if (i_2_1_s_26 < 2) {
        Y_local[((i_2_1_s_26 * 8) + 2)] = (Y_local[((i_2_1_s_26 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_26 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_27 = 0; i_2_1_s_27 < 4; ++i_2_1_s_27) {
      if (i_2_1_s_27 < 2) {
        Y_local[((i_2_1_s_27 * 8) + 3)] = (Y_local[((i_2_1_s_27 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_27 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_28 = 0; i_2_1_s_28 < 4; ++i_2_1_s_28) {
      if (i_2_1_s_28 < 2) {
        Y_local[((i_2_1_s_28 * 8) + 4)] = (Y_local[((i_2_1_s_28 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_28 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_29 = 0; i_2_1_s_29 < 4; ++i_2_1_s_29) {
      if (i_2_1_s_29 < 2) {
        Y_local[((i_2_1_s_29 * 8) + 5)] = (Y_local[((i_2_1_s_29 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_29 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_30 = 0; i_2_1_s_30 < 4; ++i_2_1_s_30) {
      if (i_2_1_s_30 < 2) {
        Y_local[((i_2_1_s_30 * 8) + 6)] = (Y_local[((i_2_1_s_30 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_30 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_31 = 0; i_2_1_s_31 < 4; ++i_2_1_s_31) {
      if (i_2_1_s_31 < 2) {
        Y_local[((i_2_1_s_31 * 8) + 7)] = (Y_local[((i_2_1_s_31 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_31 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax1_1_s_5 = 0; ax1_1_s_5 < 4; ++ax1_1_s_5) {
      if (ax1_1_s_5 < 2) {
        A_shared_dyn_local[(ax1_1_s_5 + 2)] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_5) + 160)];
      }
    }
    for (int ax1_0_5 = 0; ax1_0_5 < 2; ++ax1_0_5) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_5 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_5 * 4)) >> 6) * 64)) + (ax1_0_5 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1264) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_32 = 0; i_2_1_s_32 < 4; ++i_2_1_s_32) {
      if (i_2_1_s_32 < 2) {
        Y_local[(i_2_1_s_32 * 8)] = (Y_local[(i_2_1_s_32 * 8)] + (A_shared_dyn_local[i_2_1_s_32] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_33 = 0; i_2_1_s_33 < 4; ++i_2_1_s_33) {
      if (i_2_1_s_33 < 2) {
        Y_local[((i_2_1_s_33 * 8) + 1)] = (Y_local[((i_2_1_s_33 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_33] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_34 = 0; i_2_1_s_34 < 4; ++i_2_1_s_34) {
      if (i_2_1_s_34 < 2) {
        Y_local[((i_2_1_s_34 * 8) + 2)] = (Y_local[((i_2_1_s_34 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_34] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_35 = 0; i_2_1_s_35 < 4; ++i_2_1_s_35) {
      if (i_2_1_s_35 < 2) {
        Y_local[((i_2_1_s_35 * 8) + 3)] = (Y_local[((i_2_1_s_35 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_35] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_36 = 0; i_2_1_s_36 < 4; ++i_2_1_s_36) {
      if (i_2_1_s_36 < 2) {
        Y_local[((i_2_1_s_36 * 8) + 4)] = (Y_local[((i_2_1_s_36 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_36] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_37 = 0; i_2_1_s_37 < 4; ++i_2_1_s_37) {
      if (i_2_1_s_37 < 2) {
        Y_local[((i_2_1_s_37 * 8) + 5)] = (Y_local[((i_2_1_s_37 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_37] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_38 = 0; i_2_1_s_38 < 4; ++i_2_1_s_38) {
      if (i_2_1_s_38 < 2) {
        Y_local[((i_2_1_s_38 * 8) + 6)] = (Y_local[((i_2_1_s_38 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_38] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_39 = 0; i_2_1_s_39 < 4; ++i_2_1_s_39) {
      if (i_2_1_s_39 < 2) {
        Y_local[((i_2_1_s_39 * 8) + 7)] = (Y_local[((i_2_1_s_39 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_39] * B_shared_dyn_local[7]));
      }
    }
    for (int ax1_1_s_6 = 0; ax1_1_s_6 < 4; ++ax1_1_s_6) {
      if (ax1_1_s_6 < 2) {
        A_shared_dyn_local[ax1_1_s_6] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_6) + 192)];
      }
    }
    for (int ax1_0_6 = 0; ax1_0_6 < 2; ++ax1_0_6) {
      *(float4*)(B_shared_dyn_local + (ax1_0_6 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_6 * 4)) >> 6) * 64)) + (ax1_0_6 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1312) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_40 = 0; i_2_1_s_40 < 4; ++i_2_1_s_40) {
      if (i_2_1_s_40 < 2) {
        Y_local[(i_2_1_s_40 * 8)] = (Y_local[(i_2_1_s_40 * 8)] + (A_shared_dyn_local[(i_2_1_s_40 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_41 = 0; i_2_1_s_41 < 4; ++i_2_1_s_41) {
      if (i_2_1_s_41 < 2) {
        Y_local[((i_2_1_s_41 * 8) + 1)] = (Y_local[((i_2_1_s_41 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_41 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_42 = 0; i_2_1_s_42 < 4; ++i_2_1_s_42) {
      if (i_2_1_s_42 < 2) {
        Y_local[((i_2_1_s_42 * 8) + 2)] = (Y_local[((i_2_1_s_42 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_42 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_43 = 0; i_2_1_s_43 < 4; ++i_2_1_s_43) {
      if (i_2_1_s_43 < 2) {
        Y_local[((i_2_1_s_43 * 8) + 3)] = (Y_local[((i_2_1_s_43 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_43 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_44 = 0; i_2_1_s_44 < 4; ++i_2_1_s_44) {
      if (i_2_1_s_44 < 2) {
        Y_local[((i_2_1_s_44 * 8) + 4)] = (Y_local[((i_2_1_s_44 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_44 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_45 = 0; i_2_1_s_45 < 4; ++i_2_1_s_45) {
      if (i_2_1_s_45 < 2) {
        Y_local[((i_2_1_s_45 * 8) + 5)] = (Y_local[((i_2_1_s_45 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_45 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_46 = 0; i_2_1_s_46 < 4; ++i_2_1_s_46) {
      if (i_2_1_s_46 < 2) {
        Y_local[((i_2_1_s_46 * 8) + 6)] = (Y_local[((i_2_1_s_46 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_46 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_47 = 0; i_2_1_s_47 < 4; ++i_2_1_s_47) {
      if (i_2_1_s_47 < 2) {
        Y_local[((i_2_1_s_47 * 8) + 7)] = (Y_local[((i_2_1_s_47 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_47 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax1_1_s_7 = 0; ax1_1_s_7 < 4; ++ax1_1_s_7) {
      if (ax1_1_s_7 < 2) {
        A_shared_dyn_local[(ax1_1_s_7 + 2)] = ((float*)buf_dyn_shmem)[(((((((k_0 & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_7) + 224)];
      }
    }
    for (int ax1_0_7 = 0; ax1_0_7 < 2; ++ax1_0_7) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_7 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_7 * 4)) >> 6) * 64)) + (ax1_0_7 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1360) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_48 = 0; i_2_1_s_48 < 4; ++i_2_1_s_48) {
      if (i_2_1_s_48 < 2) {
        Y_local[(i_2_1_s_48 * 8)] = (Y_local[(i_2_1_s_48 * 8)] + (A_shared_dyn_local[i_2_1_s_48] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_49 = 0; i_2_1_s_49 < 4; ++i_2_1_s_49) {
      if (i_2_1_s_49 < 2) {
        Y_local[((i_2_1_s_49 * 8) + 1)] = (Y_local[((i_2_1_s_49 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_49] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_50 = 0; i_2_1_s_50 < 4; ++i_2_1_s_50) {
      if (i_2_1_s_50 < 2) {
        Y_local[((i_2_1_s_50 * 8) + 2)] = (Y_local[((i_2_1_s_50 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_50] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_51 = 0; i_2_1_s_51 < 4; ++i_2_1_s_51) {
      if (i_2_1_s_51 < 2) {
        Y_local[((i_2_1_s_51 * 8) + 3)] = (Y_local[((i_2_1_s_51 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_51] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_52 = 0; i_2_1_s_52 < 4; ++i_2_1_s_52) {
      if (i_2_1_s_52 < 2) {
        Y_local[((i_2_1_s_52 * 8) + 4)] = (Y_local[((i_2_1_s_52 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_52] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_53 = 0; i_2_1_s_53 < 4; ++i_2_1_s_53) {
      if (i_2_1_s_53 < 2) {
        Y_local[((i_2_1_s_53 * 8) + 5)] = (Y_local[((i_2_1_s_53 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_53] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_54 = 0; i_2_1_s_54 < 4; ++i_2_1_s_54) {
      if (i_2_1_s_54 < 2) {
        Y_local[((i_2_1_s_54 * 8) + 6)] = (Y_local[((i_2_1_s_54 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_54] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_55 = 0; i_2_1_s_55 < 4; ++i_2_1_s_55) {
      if (i_2_1_s_55 < 2) {
        Y_local[((i_2_1_s_55 * 8) + 7)] = (Y_local[((i_2_1_s_55 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_55] * B_shared_dyn_local[7]));
      }
    }
    for (int ax1_1_s_8 = 0; ax1_1_s_8 < 4; ++ax1_1_s_8) {
      if (ax1_1_s_8 < 2) {
        A_shared_dyn_local[ax1_1_s_8] = ((float*)buf_dyn_shmem)[(((((((k_0 + 1) & 3) * 256) + (((((int)threadIdx.x) & 7) >> 2) * 16)) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_8)];
      }
    }
    for (int ax1_0_8 = 0; ax1_0_8 < 2; ++ax1_0_8) {
      *(float4*)(B_shared_dyn_local + (ax1_0_8 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((((k_0 + 1) & 3) * 384) + (((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_8 * 4)) >> 6) * 64)) + (ax1_0_8 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1024) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
    }
    for (int i_2_1_s_56 = 0; i_2_1_s_56 < 4; ++i_2_1_s_56) {
      if (i_2_1_s_56 < 2) {
        Y_local[(i_2_1_s_56 * 8)] = (Y_local[(i_2_1_s_56 * 8)] + (A_shared_dyn_local[(i_2_1_s_56 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_57 = 0; i_2_1_s_57 < 4; ++i_2_1_s_57) {
      if (i_2_1_s_57 < 2) {
        Y_local[((i_2_1_s_57 * 8) + 1)] = (Y_local[((i_2_1_s_57 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_57 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_58 = 0; i_2_1_s_58 < 4; ++i_2_1_s_58) {
      if (i_2_1_s_58 < 2) {
        Y_local[((i_2_1_s_58 * 8) + 2)] = (Y_local[((i_2_1_s_58 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_58 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_59 = 0; i_2_1_s_59 < 4; ++i_2_1_s_59) {
      if (i_2_1_s_59 < 2) {
        Y_local[((i_2_1_s_59 * 8) + 3)] = (Y_local[((i_2_1_s_59 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_59 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_60 = 0; i_2_1_s_60 < 4; ++i_2_1_s_60) {
      if (i_2_1_s_60 < 2) {
        Y_local[((i_2_1_s_60 * 8) + 4)] = (Y_local[((i_2_1_s_60 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_60 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_61 = 0; i_2_1_s_61 < 4; ++i_2_1_s_61) {
      if (i_2_1_s_61 < 2) {
        Y_local[((i_2_1_s_61 * 8) + 5)] = (Y_local[((i_2_1_s_61 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_61 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_62 = 0; i_2_1_s_62 < 4; ++i_2_1_s_62) {
      if (i_2_1_s_62 < 2) {
        Y_local[((i_2_1_s_62 * 8) + 6)] = (Y_local[((i_2_1_s_62 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_62 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_63 = 0; i_2_1_s_63 < 4; ++i_2_1_s_63) {
      if (i_2_1_s_63 < 2) {
        Y_local[((i_2_1_s_63 * 8) + 7)] = (Y_local[((i_2_1_s_63 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_63 + 2)] * B_shared_dyn_local[15]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax1_1_s_9 = 0; ax1_1_s_9 < 4; ++ax1_1_s_9) {
    if (ax1_1_s_9 < 2) {
      A_shared_dyn_local[(ax1_1_s_9 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_9) + 288)];
    }
  }
  for (int ax1_0_9 = 0; ax1_0_9 < 2; ++ax1_0_9) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_9 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_9 * 4)) >> 6) * 64) + (ax1_0_9 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1456) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_64 = 0; i_2_1_s_64 < 4; ++i_2_1_s_64) {
    if (i_2_1_s_64 < 2) {
      Y_local[(i_2_1_s_64 * 8)] = (Y_local[(i_2_1_s_64 * 8)] + (A_shared_dyn_local[i_2_1_s_64] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_65 = 0; i_2_1_s_65 < 4; ++i_2_1_s_65) {
    if (i_2_1_s_65 < 2) {
      Y_local[((i_2_1_s_65 * 8) + 1)] = (Y_local[((i_2_1_s_65 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_65] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_66 = 0; i_2_1_s_66 < 4; ++i_2_1_s_66) {
    if (i_2_1_s_66 < 2) {
      Y_local[((i_2_1_s_66 * 8) + 2)] = (Y_local[((i_2_1_s_66 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_66] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_67 = 0; i_2_1_s_67 < 4; ++i_2_1_s_67) {
    if (i_2_1_s_67 < 2) {
      Y_local[((i_2_1_s_67 * 8) + 3)] = (Y_local[((i_2_1_s_67 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_67] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_68 = 0; i_2_1_s_68 < 4; ++i_2_1_s_68) {
    if (i_2_1_s_68 < 2) {
      Y_local[((i_2_1_s_68 * 8) + 4)] = (Y_local[((i_2_1_s_68 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_68] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_69 = 0; i_2_1_s_69 < 4; ++i_2_1_s_69) {
    if (i_2_1_s_69 < 2) {
      Y_local[((i_2_1_s_69 * 8) + 5)] = (Y_local[((i_2_1_s_69 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_69] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_70 = 0; i_2_1_s_70 < 4; ++i_2_1_s_70) {
    if (i_2_1_s_70 < 2) {
      Y_local[((i_2_1_s_70 * 8) + 6)] = (Y_local[((i_2_1_s_70 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_70] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_71 = 0; i_2_1_s_71 < 4; ++i_2_1_s_71) {
    if (i_2_1_s_71 < 2) {
      Y_local[((i_2_1_s_71 * 8) + 7)] = (Y_local[((i_2_1_s_71 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_71] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_10 = 0; ax1_1_s_10 < 4; ++ax1_1_s_10) {
    if (ax1_1_s_10 < 2) {
      A_shared_dyn_local[ax1_1_s_10] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_10) + 320)];
    }
  }
  for (int ax1_0_10 = 0; ax1_0_10 < 2; ++ax1_0_10) {
    *(float4*)(B_shared_dyn_local + (ax1_0_10 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_10 * 4)) >> 6) * 64) + (ax1_0_10 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1504) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_72 = 0; i_2_1_s_72 < 4; ++i_2_1_s_72) {
    if (i_2_1_s_72 < 2) {
      Y_local[(i_2_1_s_72 * 8)] = (Y_local[(i_2_1_s_72 * 8)] + (A_shared_dyn_local[(i_2_1_s_72 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_73 = 0; i_2_1_s_73 < 4; ++i_2_1_s_73) {
    if (i_2_1_s_73 < 2) {
      Y_local[((i_2_1_s_73 * 8) + 1)] = (Y_local[((i_2_1_s_73 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_73 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_74 = 0; i_2_1_s_74 < 4; ++i_2_1_s_74) {
    if (i_2_1_s_74 < 2) {
      Y_local[((i_2_1_s_74 * 8) + 2)] = (Y_local[((i_2_1_s_74 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_74 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_75 = 0; i_2_1_s_75 < 4; ++i_2_1_s_75) {
    if (i_2_1_s_75 < 2) {
      Y_local[((i_2_1_s_75 * 8) + 3)] = (Y_local[((i_2_1_s_75 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_75 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_76 = 0; i_2_1_s_76 < 4; ++i_2_1_s_76) {
    if (i_2_1_s_76 < 2) {
      Y_local[((i_2_1_s_76 * 8) + 4)] = (Y_local[((i_2_1_s_76 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_76 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_77 = 0; i_2_1_s_77 < 4; ++i_2_1_s_77) {
    if (i_2_1_s_77 < 2) {
      Y_local[((i_2_1_s_77 * 8) + 5)] = (Y_local[((i_2_1_s_77 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_77 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_78 = 0; i_2_1_s_78 < 4; ++i_2_1_s_78) {
    if (i_2_1_s_78 < 2) {
      Y_local[((i_2_1_s_78 * 8) + 6)] = (Y_local[((i_2_1_s_78 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_78 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_79 = 0; i_2_1_s_79 < 4; ++i_2_1_s_79) {
    if (i_2_1_s_79 < 2) {
      Y_local[((i_2_1_s_79 * 8) + 7)] = (Y_local[((i_2_1_s_79 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_79 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_11 = 0; ax1_1_s_11 < 4; ++ax1_1_s_11) {
    if (ax1_1_s_11 < 2) {
      A_shared_dyn_local[(ax1_1_s_11 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_11) + 352)];
    }
  }
  for (int ax1_0_11 = 0; ax1_0_11 < 2; ++ax1_0_11) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_11 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_11 * 4)) >> 6) * 64) + (ax1_0_11 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1552) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_80 = 0; i_2_1_s_80 < 4; ++i_2_1_s_80) {
    if (i_2_1_s_80 < 2) {
      Y_local[(i_2_1_s_80 * 8)] = (Y_local[(i_2_1_s_80 * 8)] + (A_shared_dyn_local[i_2_1_s_80] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_81 = 0; i_2_1_s_81 < 4; ++i_2_1_s_81) {
    if (i_2_1_s_81 < 2) {
      Y_local[((i_2_1_s_81 * 8) + 1)] = (Y_local[((i_2_1_s_81 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_81] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_82 = 0; i_2_1_s_82 < 4; ++i_2_1_s_82) {
    if (i_2_1_s_82 < 2) {
      Y_local[((i_2_1_s_82 * 8) + 2)] = (Y_local[((i_2_1_s_82 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_82] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_83 = 0; i_2_1_s_83 < 4; ++i_2_1_s_83) {
    if (i_2_1_s_83 < 2) {
      Y_local[((i_2_1_s_83 * 8) + 3)] = (Y_local[((i_2_1_s_83 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_83] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_84 = 0; i_2_1_s_84 < 4; ++i_2_1_s_84) {
    if (i_2_1_s_84 < 2) {
      Y_local[((i_2_1_s_84 * 8) + 4)] = (Y_local[((i_2_1_s_84 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_84] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_85 = 0; i_2_1_s_85 < 4; ++i_2_1_s_85) {
    if (i_2_1_s_85 < 2) {
      Y_local[((i_2_1_s_85 * 8) + 5)] = (Y_local[((i_2_1_s_85 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_85] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_86 = 0; i_2_1_s_86 < 4; ++i_2_1_s_86) {
    if (i_2_1_s_86 < 2) {
      Y_local[((i_2_1_s_86 * 8) + 6)] = (Y_local[((i_2_1_s_86 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_86] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_87 = 0; i_2_1_s_87 < 4; ++i_2_1_s_87) {
    if (i_2_1_s_87 < 2) {
      Y_local[((i_2_1_s_87 * 8) + 7)] = (Y_local[((i_2_1_s_87 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_87] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_12 = 0; ax1_1_s_12 < 4; ++ax1_1_s_12) {
    if (ax1_1_s_12 < 2) {
      A_shared_dyn_local[ax1_1_s_12] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_12) + 384)];
    }
  }
  for (int ax1_0_12 = 0; ax1_0_12 < 2; ++ax1_0_12) {
    *(float4*)(B_shared_dyn_local + (ax1_0_12 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_12 * 4)) >> 6) * 64) + (ax1_0_12 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1600) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_88 = 0; i_2_1_s_88 < 4; ++i_2_1_s_88) {
    if (i_2_1_s_88 < 2) {
      Y_local[(i_2_1_s_88 * 8)] = (Y_local[(i_2_1_s_88 * 8)] + (A_shared_dyn_local[(i_2_1_s_88 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_89 = 0; i_2_1_s_89 < 4; ++i_2_1_s_89) {
    if (i_2_1_s_89 < 2) {
      Y_local[((i_2_1_s_89 * 8) + 1)] = (Y_local[((i_2_1_s_89 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_89 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_90 = 0; i_2_1_s_90 < 4; ++i_2_1_s_90) {
    if (i_2_1_s_90 < 2) {
      Y_local[((i_2_1_s_90 * 8) + 2)] = (Y_local[((i_2_1_s_90 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_90 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_91 = 0; i_2_1_s_91 < 4; ++i_2_1_s_91) {
    if (i_2_1_s_91 < 2) {
      Y_local[((i_2_1_s_91 * 8) + 3)] = (Y_local[((i_2_1_s_91 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_91 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_92 = 0; i_2_1_s_92 < 4; ++i_2_1_s_92) {
    if (i_2_1_s_92 < 2) {
      Y_local[((i_2_1_s_92 * 8) + 4)] = (Y_local[((i_2_1_s_92 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_92 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_93 = 0; i_2_1_s_93 < 4; ++i_2_1_s_93) {
    if (i_2_1_s_93 < 2) {
      Y_local[((i_2_1_s_93 * 8) + 5)] = (Y_local[((i_2_1_s_93 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_93 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_94 = 0; i_2_1_s_94 < 4; ++i_2_1_s_94) {
    if (i_2_1_s_94 < 2) {
      Y_local[((i_2_1_s_94 * 8) + 6)] = (Y_local[((i_2_1_s_94 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_94 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_95 = 0; i_2_1_s_95 < 4; ++i_2_1_s_95) {
    if (i_2_1_s_95 < 2) {
      Y_local[((i_2_1_s_95 * 8) + 7)] = (Y_local[((i_2_1_s_95 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_95 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_13 = 0; ax1_1_s_13 < 4; ++ax1_1_s_13) {
    if (ax1_1_s_13 < 2) {
      A_shared_dyn_local[(ax1_1_s_13 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_13) + 416)];
    }
  }
  for (int ax1_0_13 = 0; ax1_0_13 < 2; ++ax1_0_13) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_13 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_13 * 4)) >> 6) * 64) + (ax1_0_13 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1648) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_96 = 0; i_2_1_s_96 < 4; ++i_2_1_s_96) {
    if (i_2_1_s_96 < 2) {
      Y_local[(i_2_1_s_96 * 8)] = (Y_local[(i_2_1_s_96 * 8)] + (A_shared_dyn_local[i_2_1_s_96] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_97 = 0; i_2_1_s_97 < 4; ++i_2_1_s_97) {
    if (i_2_1_s_97 < 2) {
      Y_local[((i_2_1_s_97 * 8) + 1)] = (Y_local[((i_2_1_s_97 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_97] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_98 = 0; i_2_1_s_98 < 4; ++i_2_1_s_98) {
    if (i_2_1_s_98 < 2) {
      Y_local[((i_2_1_s_98 * 8) + 2)] = (Y_local[((i_2_1_s_98 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_98] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_99 = 0; i_2_1_s_99 < 4; ++i_2_1_s_99) {
    if (i_2_1_s_99 < 2) {
      Y_local[((i_2_1_s_99 * 8) + 3)] = (Y_local[((i_2_1_s_99 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_99] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_100 = 0; i_2_1_s_100 < 4; ++i_2_1_s_100) {
    if (i_2_1_s_100 < 2) {
      Y_local[((i_2_1_s_100 * 8) + 4)] = (Y_local[((i_2_1_s_100 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_100] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_101 = 0; i_2_1_s_101 < 4; ++i_2_1_s_101) {
    if (i_2_1_s_101 < 2) {
      Y_local[((i_2_1_s_101 * 8) + 5)] = (Y_local[((i_2_1_s_101 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_101] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_102 = 0; i_2_1_s_102 < 4; ++i_2_1_s_102) {
    if (i_2_1_s_102 < 2) {
      Y_local[((i_2_1_s_102 * 8) + 6)] = (Y_local[((i_2_1_s_102 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_102] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_103 = 0; i_2_1_s_103 < 4; ++i_2_1_s_103) {
    if (i_2_1_s_103 < 2) {
      Y_local[((i_2_1_s_103 * 8) + 7)] = (Y_local[((i_2_1_s_103 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_103] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_14 = 0; ax1_1_s_14 < 4; ++ax1_1_s_14) {
    if (ax1_1_s_14 < 2) {
      A_shared_dyn_local[ax1_1_s_14] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_14) + 448)];
    }
  }
  for (int ax1_0_14 = 0; ax1_0_14 < 2; ++ax1_0_14) {
    *(float4*)(B_shared_dyn_local + (ax1_0_14 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_14 * 4)) >> 6) * 64) + (ax1_0_14 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1696) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_104 = 0; i_2_1_s_104 < 4; ++i_2_1_s_104) {
    if (i_2_1_s_104 < 2) {
      Y_local[(i_2_1_s_104 * 8)] = (Y_local[(i_2_1_s_104 * 8)] + (A_shared_dyn_local[(i_2_1_s_104 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_105 = 0; i_2_1_s_105 < 4; ++i_2_1_s_105) {
    if (i_2_1_s_105 < 2) {
      Y_local[((i_2_1_s_105 * 8) + 1)] = (Y_local[((i_2_1_s_105 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_105 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_106 = 0; i_2_1_s_106 < 4; ++i_2_1_s_106) {
    if (i_2_1_s_106 < 2) {
      Y_local[((i_2_1_s_106 * 8) + 2)] = (Y_local[((i_2_1_s_106 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_106 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_107 = 0; i_2_1_s_107 < 4; ++i_2_1_s_107) {
    if (i_2_1_s_107 < 2) {
      Y_local[((i_2_1_s_107 * 8) + 3)] = (Y_local[((i_2_1_s_107 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_107 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_108 = 0; i_2_1_s_108 < 4; ++i_2_1_s_108) {
    if (i_2_1_s_108 < 2) {
      Y_local[((i_2_1_s_108 * 8) + 4)] = (Y_local[((i_2_1_s_108 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_108 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_109 = 0; i_2_1_s_109 < 4; ++i_2_1_s_109) {
    if (i_2_1_s_109 < 2) {
      Y_local[((i_2_1_s_109 * 8) + 5)] = (Y_local[((i_2_1_s_109 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_109 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_110 = 0; i_2_1_s_110 < 4; ++i_2_1_s_110) {
    if (i_2_1_s_110 < 2) {
      Y_local[((i_2_1_s_110 * 8) + 6)] = (Y_local[((i_2_1_s_110 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_110 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_111 = 0; i_2_1_s_111 < 4; ++i_2_1_s_111) {
    if (i_2_1_s_111 < 2) {
      Y_local[((i_2_1_s_111 * 8) + 7)] = (Y_local[((i_2_1_s_111 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_111 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_15 = 0; ax1_1_s_15 < 4; ++ax1_1_s_15) {
    if (ax1_1_s_15 < 2) {
      A_shared_dyn_local[(ax1_1_s_15 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_15) + 480)];
    }
  }
  for (int ax1_0_15 = 0; ax1_0_15 < 2; ++ax1_0_15) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_15 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_15 * 4)) >> 6) * 64) + (ax1_0_15 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1744) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_112 = 0; i_2_1_s_112 < 4; ++i_2_1_s_112) {
    if (i_2_1_s_112 < 2) {
      Y_local[(i_2_1_s_112 * 8)] = (Y_local[(i_2_1_s_112 * 8)] + (A_shared_dyn_local[i_2_1_s_112] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_113 = 0; i_2_1_s_113 < 4; ++i_2_1_s_113) {
    if (i_2_1_s_113 < 2) {
      Y_local[((i_2_1_s_113 * 8) + 1)] = (Y_local[((i_2_1_s_113 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_113] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_114 = 0; i_2_1_s_114 < 4; ++i_2_1_s_114) {
    if (i_2_1_s_114 < 2) {
      Y_local[((i_2_1_s_114 * 8) + 2)] = (Y_local[((i_2_1_s_114 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_114] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_115 = 0; i_2_1_s_115 < 4; ++i_2_1_s_115) {
    if (i_2_1_s_115 < 2) {
      Y_local[((i_2_1_s_115 * 8) + 3)] = (Y_local[((i_2_1_s_115 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_115] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_116 = 0; i_2_1_s_116 < 4; ++i_2_1_s_116) {
    if (i_2_1_s_116 < 2) {
      Y_local[((i_2_1_s_116 * 8) + 4)] = (Y_local[((i_2_1_s_116 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_116] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_117 = 0; i_2_1_s_117 < 4; ++i_2_1_s_117) {
    if (i_2_1_s_117 < 2) {
      Y_local[((i_2_1_s_117 * 8) + 5)] = (Y_local[((i_2_1_s_117 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_117] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_118 = 0; i_2_1_s_118 < 4; ++i_2_1_s_118) {
    if (i_2_1_s_118 < 2) {
      Y_local[((i_2_1_s_118 * 8) + 6)] = (Y_local[((i_2_1_s_118 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_118] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_119 = 0; i_2_1_s_119 < 4; ++i_2_1_s_119) {
    if (i_2_1_s_119 < 2) {
      Y_local[((i_2_1_s_119 * 8) + 7)] = (Y_local[((i_2_1_s_119 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_119] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_16 = 0; ax1_1_s_16 < 4; ++ax1_1_s_16) {
    if (ax1_1_s_16 < 2) {
      A_shared_dyn_local[ax1_1_s_16] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_16) + 512)];
    }
  }
  for (int ax1_0_16 = 0; ax1_0_16 < 2; ++ax1_0_16) {
    *(float4*)(B_shared_dyn_local + (ax1_0_16 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_16 * 4)) >> 6) * 64) + (ax1_0_16 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1792) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_120 = 0; i_2_1_s_120 < 4; ++i_2_1_s_120) {
    if (i_2_1_s_120 < 2) {
      Y_local[(i_2_1_s_120 * 8)] = (Y_local[(i_2_1_s_120 * 8)] + (A_shared_dyn_local[(i_2_1_s_120 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_121 = 0; i_2_1_s_121 < 4; ++i_2_1_s_121) {
    if (i_2_1_s_121 < 2) {
      Y_local[((i_2_1_s_121 * 8) + 1)] = (Y_local[((i_2_1_s_121 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_121 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_122 = 0; i_2_1_s_122 < 4; ++i_2_1_s_122) {
    if (i_2_1_s_122 < 2) {
      Y_local[((i_2_1_s_122 * 8) + 2)] = (Y_local[((i_2_1_s_122 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_122 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_123 = 0; i_2_1_s_123 < 4; ++i_2_1_s_123) {
    if (i_2_1_s_123 < 2) {
      Y_local[((i_2_1_s_123 * 8) + 3)] = (Y_local[((i_2_1_s_123 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_123 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_124 = 0; i_2_1_s_124 < 4; ++i_2_1_s_124) {
    if (i_2_1_s_124 < 2) {
      Y_local[((i_2_1_s_124 * 8) + 4)] = (Y_local[((i_2_1_s_124 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_124 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_125 = 0; i_2_1_s_125 < 4; ++i_2_1_s_125) {
    if (i_2_1_s_125 < 2) {
      Y_local[((i_2_1_s_125 * 8) + 5)] = (Y_local[((i_2_1_s_125 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_125 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_126 = 0; i_2_1_s_126 < 4; ++i_2_1_s_126) {
    if (i_2_1_s_126 < 2) {
      Y_local[((i_2_1_s_126 * 8) + 6)] = (Y_local[((i_2_1_s_126 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_126 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_127 = 0; i_2_1_s_127 < 4; ++i_2_1_s_127) {
    if (i_2_1_s_127 < 2) {
      Y_local[((i_2_1_s_127 * 8) + 7)] = (Y_local[((i_2_1_s_127 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_127 + 2)] * B_shared_dyn_local[15]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax1_1_s_17 = 0; ax1_1_s_17 < 4; ++ax1_1_s_17) {
    if (ax1_1_s_17 < 2) {
      A_shared_dyn_local[(ax1_1_s_17 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_17) + 544)];
    }
  }
  for (int ax1_0_17 = 0; ax1_0_17 < 2; ++ax1_0_17) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_17 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_17 * 4)) >> 6) * 64) + (ax1_0_17 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1840) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_128 = 0; i_2_1_s_128 < 4; ++i_2_1_s_128) {
    if (i_2_1_s_128 < 2) {
      Y_local[(i_2_1_s_128 * 8)] = (Y_local[(i_2_1_s_128 * 8)] + (A_shared_dyn_local[i_2_1_s_128] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_129 = 0; i_2_1_s_129 < 4; ++i_2_1_s_129) {
    if (i_2_1_s_129 < 2) {
      Y_local[((i_2_1_s_129 * 8) + 1)] = (Y_local[((i_2_1_s_129 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_129] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_130 = 0; i_2_1_s_130 < 4; ++i_2_1_s_130) {
    if (i_2_1_s_130 < 2) {
      Y_local[((i_2_1_s_130 * 8) + 2)] = (Y_local[((i_2_1_s_130 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_130] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_131 = 0; i_2_1_s_131 < 4; ++i_2_1_s_131) {
    if (i_2_1_s_131 < 2) {
      Y_local[((i_2_1_s_131 * 8) + 3)] = (Y_local[((i_2_1_s_131 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_131] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_132 = 0; i_2_1_s_132 < 4; ++i_2_1_s_132) {
    if (i_2_1_s_132 < 2) {
      Y_local[((i_2_1_s_132 * 8) + 4)] = (Y_local[((i_2_1_s_132 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_132] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_133 = 0; i_2_1_s_133 < 4; ++i_2_1_s_133) {
    if (i_2_1_s_133 < 2) {
      Y_local[((i_2_1_s_133 * 8) + 5)] = (Y_local[((i_2_1_s_133 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_133] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_134 = 0; i_2_1_s_134 < 4; ++i_2_1_s_134) {
    if (i_2_1_s_134 < 2) {
      Y_local[((i_2_1_s_134 * 8) + 6)] = (Y_local[((i_2_1_s_134 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_134] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_135 = 0; i_2_1_s_135 < 4; ++i_2_1_s_135) {
    if (i_2_1_s_135 < 2) {
      Y_local[((i_2_1_s_135 * 8) + 7)] = (Y_local[((i_2_1_s_135 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_135] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_18 = 0; ax1_1_s_18 < 4; ++ax1_1_s_18) {
    if (ax1_1_s_18 < 2) {
      A_shared_dyn_local[ax1_1_s_18] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_18) + 576)];
    }
  }
  for (int ax1_0_18 = 0; ax1_0_18 < 2; ++ax1_0_18) {
    *(float4*)(B_shared_dyn_local + (ax1_0_18 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_18 * 4)) >> 6) * 64) + (ax1_0_18 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1888) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_136 = 0; i_2_1_s_136 < 4; ++i_2_1_s_136) {
    if (i_2_1_s_136 < 2) {
      Y_local[(i_2_1_s_136 * 8)] = (Y_local[(i_2_1_s_136 * 8)] + (A_shared_dyn_local[(i_2_1_s_136 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_137 = 0; i_2_1_s_137 < 4; ++i_2_1_s_137) {
    if (i_2_1_s_137 < 2) {
      Y_local[((i_2_1_s_137 * 8) + 1)] = (Y_local[((i_2_1_s_137 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_137 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_138 = 0; i_2_1_s_138 < 4; ++i_2_1_s_138) {
    if (i_2_1_s_138 < 2) {
      Y_local[((i_2_1_s_138 * 8) + 2)] = (Y_local[((i_2_1_s_138 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_138 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_139 = 0; i_2_1_s_139 < 4; ++i_2_1_s_139) {
    if (i_2_1_s_139 < 2) {
      Y_local[((i_2_1_s_139 * 8) + 3)] = (Y_local[((i_2_1_s_139 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_139 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_140 = 0; i_2_1_s_140 < 4; ++i_2_1_s_140) {
    if (i_2_1_s_140 < 2) {
      Y_local[((i_2_1_s_140 * 8) + 4)] = (Y_local[((i_2_1_s_140 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_140 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_141 = 0; i_2_1_s_141 < 4; ++i_2_1_s_141) {
    if (i_2_1_s_141 < 2) {
      Y_local[((i_2_1_s_141 * 8) + 5)] = (Y_local[((i_2_1_s_141 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_141 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_142 = 0; i_2_1_s_142 < 4; ++i_2_1_s_142) {
    if (i_2_1_s_142 < 2) {
      Y_local[((i_2_1_s_142 * 8) + 6)] = (Y_local[((i_2_1_s_142 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_142 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_143 = 0; i_2_1_s_143 < 4; ++i_2_1_s_143) {
    if (i_2_1_s_143 < 2) {
      Y_local[((i_2_1_s_143 * 8) + 7)] = (Y_local[((i_2_1_s_143 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_143 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_19 = 0; ax1_1_s_19 < 4; ++ax1_1_s_19) {
    if (ax1_1_s_19 < 2) {
      A_shared_dyn_local[(ax1_1_s_19 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_19) + 608)];
    }
  }
  for (int ax1_0_19 = 0; ax1_0_19 < 2; ++ax1_0_19) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_19 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_19 * 4)) >> 6) * 64) + (ax1_0_19 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1936) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_144 = 0; i_2_1_s_144 < 4; ++i_2_1_s_144) {
    if (i_2_1_s_144 < 2) {
      Y_local[(i_2_1_s_144 * 8)] = (Y_local[(i_2_1_s_144 * 8)] + (A_shared_dyn_local[i_2_1_s_144] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_145 = 0; i_2_1_s_145 < 4; ++i_2_1_s_145) {
    if (i_2_1_s_145 < 2) {
      Y_local[((i_2_1_s_145 * 8) + 1)] = (Y_local[((i_2_1_s_145 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_145] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_146 = 0; i_2_1_s_146 < 4; ++i_2_1_s_146) {
    if (i_2_1_s_146 < 2) {
      Y_local[((i_2_1_s_146 * 8) + 2)] = (Y_local[((i_2_1_s_146 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_146] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_147 = 0; i_2_1_s_147 < 4; ++i_2_1_s_147) {
    if (i_2_1_s_147 < 2) {
      Y_local[((i_2_1_s_147 * 8) + 3)] = (Y_local[((i_2_1_s_147 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_147] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_148 = 0; i_2_1_s_148 < 4; ++i_2_1_s_148) {
    if (i_2_1_s_148 < 2) {
      Y_local[((i_2_1_s_148 * 8) + 4)] = (Y_local[((i_2_1_s_148 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_148] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_149 = 0; i_2_1_s_149 < 4; ++i_2_1_s_149) {
    if (i_2_1_s_149 < 2) {
      Y_local[((i_2_1_s_149 * 8) + 5)] = (Y_local[((i_2_1_s_149 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_149] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_150 = 0; i_2_1_s_150 < 4; ++i_2_1_s_150) {
    if (i_2_1_s_150 < 2) {
      Y_local[((i_2_1_s_150 * 8) + 6)] = (Y_local[((i_2_1_s_150 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_150] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_151 = 0; i_2_1_s_151 < 4; ++i_2_1_s_151) {
    if (i_2_1_s_151 < 2) {
      Y_local[((i_2_1_s_151 * 8) + 7)] = (Y_local[((i_2_1_s_151 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_151] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_20 = 0; ax1_1_s_20 < 4; ++ax1_1_s_20) {
    if (ax1_1_s_20 < 2) {
      A_shared_dyn_local[ax1_1_s_20] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_20) + 640)];
    }
  }
  for (int ax1_0_20 = 0; ax1_0_20 < 2; ++ax1_0_20) {
    *(float4*)(B_shared_dyn_local + (ax1_0_20 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_20 * 4)) >> 6) * 64) + (ax1_0_20 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 1984) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_152 = 0; i_2_1_s_152 < 4; ++i_2_1_s_152) {
    if (i_2_1_s_152 < 2) {
      Y_local[(i_2_1_s_152 * 8)] = (Y_local[(i_2_1_s_152 * 8)] + (A_shared_dyn_local[(i_2_1_s_152 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_153 = 0; i_2_1_s_153 < 4; ++i_2_1_s_153) {
    if (i_2_1_s_153 < 2) {
      Y_local[((i_2_1_s_153 * 8) + 1)] = (Y_local[((i_2_1_s_153 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_153 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_154 = 0; i_2_1_s_154 < 4; ++i_2_1_s_154) {
    if (i_2_1_s_154 < 2) {
      Y_local[((i_2_1_s_154 * 8) + 2)] = (Y_local[((i_2_1_s_154 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_154 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_155 = 0; i_2_1_s_155 < 4; ++i_2_1_s_155) {
    if (i_2_1_s_155 < 2) {
      Y_local[((i_2_1_s_155 * 8) + 3)] = (Y_local[((i_2_1_s_155 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_155 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_156 = 0; i_2_1_s_156 < 4; ++i_2_1_s_156) {
    if (i_2_1_s_156 < 2) {
      Y_local[((i_2_1_s_156 * 8) + 4)] = (Y_local[((i_2_1_s_156 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_156 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_157 = 0; i_2_1_s_157 < 4; ++i_2_1_s_157) {
    if (i_2_1_s_157 < 2) {
      Y_local[((i_2_1_s_157 * 8) + 5)] = (Y_local[((i_2_1_s_157 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_157 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_158 = 0; i_2_1_s_158 < 4; ++i_2_1_s_158) {
    if (i_2_1_s_158 < 2) {
      Y_local[((i_2_1_s_158 * 8) + 6)] = (Y_local[((i_2_1_s_158 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_158 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_159 = 0; i_2_1_s_159 < 4; ++i_2_1_s_159) {
    if (i_2_1_s_159 < 2) {
      Y_local[((i_2_1_s_159 * 8) + 7)] = (Y_local[((i_2_1_s_159 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_159 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_21 = 0; ax1_1_s_21 < 4; ++ax1_1_s_21) {
    if (ax1_1_s_21 < 2) {
      A_shared_dyn_local[(ax1_1_s_21 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_21) + 672)];
    }
  }
  for (int ax1_0_21 = 0; ax1_0_21 < 2; ++ax1_0_21) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_21 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_21 * 4)) >> 6) * 64) + (ax1_0_21 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2032) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_160 = 0; i_2_1_s_160 < 4; ++i_2_1_s_160) {
    if (i_2_1_s_160 < 2) {
      Y_local[(i_2_1_s_160 * 8)] = (Y_local[(i_2_1_s_160 * 8)] + (A_shared_dyn_local[i_2_1_s_160] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_161 = 0; i_2_1_s_161 < 4; ++i_2_1_s_161) {
    if (i_2_1_s_161 < 2) {
      Y_local[((i_2_1_s_161 * 8) + 1)] = (Y_local[((i_2_1_s_161 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_161] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_162 = 0; i_2_1_s_162 < 4; ++i_2_1_s_162) {
    if (i_2_1_s_162 < 2) {
      Y_local[((i_2_1_s_162 * 8) + 2)] = (Y_local[((i_2_1_s_162 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_162] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_163 = 0; i_2_1_s_163 < 4; ++i_2_1_s_163) {
    if (i_2_1_s_163 < 2) {
      Y_local[((i_2_1_s_163 * 8) + 3)] = (Y_local[((i_2_1_s_163 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_163] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_164 = 0; i_2_1_s_164 < 4; ++i_2_1_s_164) {
    if (i_2_1_s_164 < 2) {
      Y_local[((i_2_1_s_164 * 8) + 4)] = (Y_local[((i_2_1_s_164 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_164] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_165 = 0; i_2_1_s_165 < 4; ++i_2_1_s_165) {
    if (i_2_1_s_165 < 2) {
      Y_local[((i_2_1_s_165 * 8) + 5)] = (Y_local[((i_2_1_s_165 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_165] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_166 = 0; i_2_1_s_166 < 4; ++i_2_1_s_166) {
    if (i_2_1_s_166 < 2) {
      Y_local[((i_2_1_s_166 * 8) + 6)] = (Y_local[((i_2_1_s_166 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_166] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_167 = 0; i_2_1_s_167 < 4; ++i_2_1_s_167) {
    if (i_2_1_s_167 < 2) {
      Y_local[((i_2_1_s_167 * 8) + 7)] = (Y_local[((i_2_1_s_167 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_167] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_22 = 0; ax1_1_s_22 < 4; ++ax1_1_s_22) {
    if (ax1_1_s_22 < 2) {
      A_shared_dyn_local[ax1_1_s_22] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_22) + 704)];
    }
  }
  for (int ax1_0_22 = 0; ax1_0_22 < 2; ++ax1_0_22) {
    *(float4*)(B_shared_dyn_local + (ax1_0_22 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_22 * 4)) >> 6) * 64) + (ax1_0_22 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2080) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_168 = 0; i_2_1_s_168 < 4; ++i_2_1_s_168) {
    if (i_2_1_s_168 < 2) {
      Y_local[(i_2_1_s_168 * 8)] = (Y_local[(i_2_1_s_168 * 8)] + (A_shared_dyn_local[(i_2_1_s_168 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_169 = 0; i_2_1_s_169 < 4; ++i_2_1_s_169) {
    if (i_2_1_s_169 < 2) {
      Y_local[((i_2_1_s_169 * 8) + 1)] = (Y_local[((i_2_1_s_169 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_169 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_170 = 0; i_2_1_s_170 < 4; ++i_2_1_s_170) {
    if (i_2_1_s_170 < 2) {
      Y_local[((i_2_1_s_170 * 8) + 2)] = (Y_local[((i_2_1_s_170 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_170 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_171 = 0; i_2_1_s_171 < 4; ++i_2_1_s_171) {
    if (i_2_1_s_171 < 2) {
      Y_local[((i_2_1_s_171 * 8) + 3)] = (Y_local[((i_2_1_s_171 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_171 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_172 = 0; i_2_1_s_172 < 4; ++i_2_1_s_172) {
    if (i_2_1_s_172 < 2) {
      Y_local[((i_2_1_s_172 * 8) + 4)] = (Y_local[((i_2_1_s_172 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_172 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_173 = 0; i_2_1_s_173 < 4; ++i_2_1_s_173) {
    if (i_2_1_s_173 < 2) {
      Y_local[((i_2_1_s_173 * 8) + 5)] = (Y_local[((i_2_1_s_173 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_173 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_174 = 0; i_2_1_s_174 < 4; ++i_2_1_s_174) {
    if (i_2_1_s_174 < 2) {
      Y_local[((i_2_1_s_174 * 8) + 6)] = (Y_local[((i_2_1_s_174 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_174 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_175 = 0; i_2_1_s_175 < 4; ++i_2_1_s_175) {
    if (i_2_1_s_175 < 2) {
      Y_local[((i_2_1_s_175 * 8) + 7)] = (Y_local[((i_2_1_s_175 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_175 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_23 = 0; ax1_1_s_23 < 4; ++ax1_1_s_23) {
    if (ax1_1_s_23 < 2) {
      A_shared_dyn_local[(ax1_1_s_23 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_23) + 736)];
    }
  }
  for (int ax1_0_23 = 0; ax1_0_23 < 2; ++ax1_0_23) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_23 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_23 * 4)) >> 6) * 64) + (ax1_0_23 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2128) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_176 = 0; i_2_1_s_176 < 4; ++i_2_1_s_176) {
    if (i_2_1_s_176 < 2) {
      Y_local[(i_2_1_s_176 * 8)] = (Y_local[(i_2_1_s_176 * 8)] + (A_shared_dyn_local[i_2_1_s_176] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_177 = 0; i_2_1_s_177 < 4; ++i_2_1_s_177) {
    if (i_2_1_s_177 < 2) {
      Y_local[((i_2_1_s_177 * 8) + 1)] = (Y_local[((i_2_1_s_177 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_177] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_178 = 0; i_2_1_s_178 < 4; ++i_2_1_s_178) {
    if (i_2_1_s_178 < 2) {
      Y_local[((i_2_1_s_178 * 8) + 2)] = (Y_local[((i_2_1_s_178 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_178] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_179 = 0; i_2_1_s_179 < 4; ++i_2_1_s_179) {
    if (i_2_1_s_179 < 2) {
      Y_local[((i_2_1_s_179 * 8) + 3)] = (Y_local[((i_2_1_s_179 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_179] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_180 = 0; i_2_1_s_180 < 4; ++i_2_1_s_180) {
    if (i_2_1_s_180 < 2) {
      Y_local[((i_2_1_s_180 * 8) + 4)] = (Y_local[((i_2_1_s_180 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_180] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_181 = 0; i_2_1_s_181 < 4; ++i_2_1_s_181) {
    if (i_2_1_s_181 < 2) {
      Y_local[((i_2_1_s_181 * 8) + 5)] = (Y_local[((i_2_1_s_181 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_181] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_182 = 0; i_2_1_s_182 < 4; ++i_2_1_s_182) {
    if (i_2_1_s_182 < 2) {
      Y_local[((i_2_1_s_182 * 8) + 6)] = (Y_local[((i_2_1_s_182 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_182] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_183 = 0; i_2_1_s_183 < 4; ++i_2_1_s_183) {
    if (i_2_1_s_183 < 2) {
      Y_local[((i_2_1_s_183 * 8) + 7)] = (Y_local[((i_2_1_s_183 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_183] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_24 = 0; ax1_1_s_24 < 4; ++ax1_1_s_24) {
    if (ax1_1_s_24 < 2) {
      A_shared_dyn_local[ax1_1_s_24] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_24) + 768)];
    }
  }
  for (int ax1_0_24 = 0; ax1_0_24 < 2; ++ax1_0_24) {
    *(float4*)(B_shared_dyn_local + (ax1_0_24 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_24 * 4)) >> 6) * 64) + (ax1_0_24 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2176) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_184 = 0; i_2_1_s_184 < 4; ++i_2_1_s_184) {
    if (i_2_1_s_184 < 2) {
      Y_local[(i_2_1_s_184 * 8)] = (Y_local[(i_2_1_s_184 * 8)] + (A_shared_dyn_local[(i_2_1_s_184 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_185 = 0; i_2_1_s_185 < 4; ++i_2_1_s_185) {
    if (i_2_1_s_185 < 2) {
      Y_local[((i_2_1_s_185 * 8) + 1)] = (Y_local[((i_2_1_s_185 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_185 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_186 = 0; i_2_1_s_186 < 4; ++i_2_1_s_186) {
    if (i_2_1_s_186 < 2) {
      Y_local[((i_2_1_s_186 * 8) + 2)] = (Y_local[((i_2_1_s_186 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_186 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_187 = 0; i_2_1_s_187 < 4; ++i_2_1_s_187) {
    if (i_2_1_s_187 < 2) {
      Y_local[((i_2_1_s_187 * 8) + 3)] = (Y_local[((i_2_1_s_187 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_187 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_188 = 0; i_2_1_s_188 < 4; ++i_2_1_s_188) {
    if (i_2_1_s_188 < 2) {
      Y_local[((i_2_1_s_188 * 8) + 4)] = (Y_local[((i_2_1_s_188 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_188 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_189 = 0; i_2_1_s_189 < 4; ++i_2_1_s_189) {
    if (i_2_1_s_189 < 2) {
      Y_local[((i_2_1_s_189 * 8) + 5)] = (Y_local[((i_2_1_s_189 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_189 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_190 = 0; i_2_1_s_190 < 4; ++i_2_1_s_190) {
    if (i_2_1_s_190 < 2) {
      Y_local[((i_2_1_s_190 * 8) + 6)] = (Y_local[((i_2_1_s_190 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_190 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_191 = 0; i_2_1_s_191 < 4; ++i_2_1_s_191) {
    if (i_2_1_s_191 < 2) {
      Y_local[((i_2_1_s_191 * 8) + 7)] = (Y_local[((i_2_1_s_191 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_191 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_25 = 0; ax1_1_s_25 < 4; ++ax1_1_s_25) {
    if (ax1_1_s_25 < 2) {
      A_shared_dyn_local[(ax1_1_s_25 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_25) + 800)];
    }
  }
  for (int ax1_0_25 = 0; ax1_0_25 < 2; ++ax1_0_25) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_25 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_25 * 4)) >> 6) * 64) + (ax1_0_25 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2224) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_192 = 0; i_2_1_s_192 < 4; ++i_2_1_s_192) {
    if (i_2_1_s_192 < 2) {
      Y_local[(i_2_1_s_192 * 8)] = (Y_local[(i_2_1_s_192 * 8)] + (A_shared_dyn_local[i_2_1_s_192] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_193 = 0; i_2_1_s_193 < 4; ++i_2_1_s_193) {
    if (i_2_1_s_193 < 2) {
      Y_local[((i_2_1_s_193 * 8) + 1)] = (Y_local[((i_2_1_s_193 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_193] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_194 = 0; i_2_1_s_194 < 4; ++i_2_1_s_194) {
    if (i_2_1_s_194 < 2) {
      Y_local[((i_2_1_s_194 * 8) + 2)] = (Y_local[((i_2_1_s_194 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_194] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_195 = 0; i_2_1_s_195 < 4; ++i_2_1_s_195) {
    if (i_2_1_s_195 < 2) {
      Y_local[((i_2_1_s_195 * 8) + 3)] = (Y_local[((i_2_1_s_195 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_195] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_196 = 0; i_2_1_s_196 < 4; ++i_2_1_s_196) {
    if (i_2_1_s_196 < 2) {
      Y_local[((i_2_1_s_196 * 8) + 4)] = (Y_local[((i_2_1_s_196 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_196] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_197 = 0; i_2_1_s_197 < 4; ++i_2_1_s_197) {
    if (i_2_1_s_197 < 2) {
      Y_local[((i_2_1_s_197 * 8) + 5)] = (Y_local[((i_2_1_s_197 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_197] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_198 = 0; i_2_1_s_198 < 4; ++i_2_1_s_198) {
    if (i_2_1_s_198 < 2) {
      Y_local[((i_2_1_s_198 * 8) + 6)] = (Y_local[((i_2_1_s_198 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_198] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_199 = 0; i_2_1_s_199 < 4; ++i_2_1_s_199) {
    if (i_2_1_s_199 < 2) {
      Y_local[((i_2_1_s_199 * 8) + 7)] = (Y_local[((i_2_1_s_199 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_199] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_26 = 0; ax1_1_s_26 < 4; ++ax1_1_s_26) {
    if (ax1_1_s_26 < 2) {
      A_shared_dyn_local[ax1_1_s_26] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_26) + 832)];
    }
  }
  for (int ax1_0_26 = 0; ax1_0_26 < 2; ++ax1_0_26) {
    *(float4*)(B_shared_dyn_local + (ax1_0_26 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_26 * 4)) >> 6) * 64) + (ax1_0_26 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2272) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_200 = 0; i_2_1_s_200 < 4; ++i_2_1_s_200) {
    if (i_2_1_s_200 < 2) {
      Y_local[(i_2_1_s_200 * 8)] = (Y_local[(i_2_1_s_200 * 8)] + (A_shared_dyn_local[(i_2_1_s_200 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_201 = 0; i_2_1_s_201 < 4; ++i_2_1_s_201) {
    if (i_2_1_s_201 < 2) {
      Y_local[((i_2_1_s_201 * 8) + 1)] = (Y_local[((i_2_1_s_201 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_201 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_202 = 0; i_2_1_s_202 < 4; ++i_2_1_s_202) {
    if (i_2_1_s_202 < 2) {
      Y_local[((i_2_1_s_202 * 8) + 2)] = (Y_local[((i_2_1_s_202 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_202 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_203 = 0; i_2_1_s_203 < 4; ++i_2_1_s_203) {
    if (i_2_1_s_203 < 2) {
      Y_local[((i_2_1_s_203 * 8) + 3)] = (Y_local[((i_2_1_s_203 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_203 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_204 = 0; i_2_1_s_204 < 4; ++i_2_1_s_204) {
    if (i_2_1_s_204 < 2) {
      Y_local[((i_2_1_s_204 * 8) + 4)] = (Y_local[((i_2_1_s_204 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_204 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_205 = 0; i_2_1_s_205 < 4; ++i_2_1_s_205) {
    if (i_2_1_s_205 < 2) {
      Y_local[((i_2_1_s_205 * 8) + 5)] = (Y_local[((i_2_1_s_205 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_205 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_206 = 0; i_2_1_s_206 < 4; ++i_2_1_s_206) {
    if (i_2_1_s_206 < 2) {
      Y_local[((i_2_1_s_206 * 8) + 6)] = (Y_local[((i_2_1_s_206 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_206 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_207 = 0; i_2_1_s_207 < 4; ++i_2_1_s_207) {
    if (i_2_1_s_207 < 2) {
      Y_local[((i_2_1_s_207 * 8) + 7)] = (Y_local[((i_2_1_s_207 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_207 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_27 = 0; ax1_1_s_27 < 4; ++ax1_1_s_27) {
    if (ax1_1_s_27 < 2) {
      A_shared_dyn_local[(ax1_1_s_27 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_27) + 864)];
    }
  }
  for (int ax1_0_27 = 0; ax1_0_27 < 2; ++ax1_0_27) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_27 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_27 * 4)) >> 6) * 64) + (ax1_0_27 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2320) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_208 = 0; i_2_1_s_208 < 4; ++i_2_1_s_208) {
    if (i_2_1_s_208 < 2) {
      Y_local[(i_2_1_s_208 * 8)] = (Y_local[(i_2_1_s_208 * 8)] + (A_shared_dyn_local[i_2_1_s_208] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_209 = 0; i_2_1_s_209 < 4; ++i_2_1_s_209) {
    if (i_2_1_s_209 < 2) {
      Y_local[((i_2_1_s_209 * 8) + 1)] = (Y_local[((i_2_1_s_209 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_209] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_210 = 0; i_2_1_s_210 < 4; ++i_2_1_s_210) {
    if (i_2_1_s_210 < 2) {
      Y_local[((i_2_1_s_210 * 8) + 2)] = (Y_local[((i_2_1_s_210 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_210] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_211 = 0; i_2_1_s_211 < 4; ++i_2_1_s_211) {
    if (i_2_1_s_211 < 2) {
      Y_local[((i_2_1_s_211 * 8) + 3)] = (Y_local[((i_2_1_s_211 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_211] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_212 = 0; i_2_1_s_212 < 4; ++i_2_1_s_212) {
    if (i_2_1_s_212 < 2) {
      Y_local[((i_2_1_s_212 * 8) + 4)] = (Y_local[((i_2_1_s_212 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_212] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_213 = 0; i_2_1_s_213 < 4; ++i_2_1_s_213) {
    if (i_2_1_s_213 < 2) {
      Y_local[((i_2_1_s_213 * 8) + 5)] = (Y_local[((i_2_1_s_213 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_213] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_214 = 0; i_2_1_s_214 < 4; ++i_2_1_s_214) {
    if (i_2_1_s_214 < 2) {
      Y_local[((i_2_1_s_214 * 8) + 6)] = (Y_local[((i_2_1_s_214 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_214] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_215 = 0; i_2_1_s_215 < 4; ++i_2_1_s_215) {
    if (i_2_1_s_215 < 2) {
      Y_local[((i_2_1_s_215 * 8) + 7)] = (Y_local[((i_2_1_s_215 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_215] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_28 = 0; ax1_1_s_28 < 4; ++ax1_1_s_28) {
    if (ax1_1_s_28 < 2) {
      A_shared_dyn_local[ax1_1_s_28] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_28) + 896)];
    }
  }
  for (int ax1_0_28 = 0; ax1_0_28 < 2; ++ax1_0_28) {
    *(float4*)(B_shared_dyn_local + (ax1_0_28 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_28 * 4)) >> 6) * 64) + (ax1_0_28 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2368) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_216 = 0; i_2_1_s_216 < 4; ++i_2_1_s_216) {
    if (i_2_1_s_216 < 2) {
      Y_local[(i_2_1_s_216 * 8)] = (Y_local[(i_2_1_s_216 * 8)] + (A_shared_dyn_local[(i_2_1_s_216 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_217 = 0; i_2_1_s_217 < 4; ++i_2_1_s_217) {
    if (i_2_1_s_217 < 2) {
      Y_local[((i_2_1_s_217 * 8) + 1)] = (Y_local[((i_2_1_s_217 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_217 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_218 = 0; i_2_1_s_218 < 4; ++i_2_1_s_218) {
    if (i_2_1_s_218 < 2) {
      Y_local[((i_2_1_s_218 * 8) + 2)] = (Y_local[((i_2_1_s_218 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_218 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_219 = 0; i_2_1_s_219 < 4; ++i_2_1_s_219) {
    if (i_2_1_s_219 < 2) {
      Y_local[((i_2_1_s_219 * 8) + 3)] = (Y_local[((i_2_1_s_219 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_219 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_220 = 0; i_2_1_s_220 < 4; ++i_2_1_s_220) {
    if (i_2_1_s_220 < 2) {
      Y_local[((i_2_1_s_220 * 8) + 4)] = (Y_local[((i_2_1_s_220 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_220 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_221 = 0; i_2_1_s_221 < 4; ++i_2_1_s_221) {
    if (i_2_1_s_221 < 2) {
      Y_local[((i_2_1_s_221 * 8) + 5)] = (Y_local[((i_2_1_s_221 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_221 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_222 = 0; i_2_1_s_222 < 4; ++i_2_1_s_222) {
    if (i_2_1_s_222 < 2) {
      Y_local[((i_2_1_s_222 * 8) + 6)] = (Y_local[((i_2_1_s_222 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_222 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_223 = 0; i_2_1_s_223 < 4; ++i_2_1_s_223) {
    if (i_2_1_s_223 < 2) {
      Y_local[((i_2_1_s_223 * 8) + 7)] = (Y_local[((i_2_1_s_223 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_223 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_29 = 0; ax1_1_s_29 < 4; ++ax1_1_s_29) {
    if (ax1_1_s_29 < 2) {
      A_shared_dyn_local[(ax1_1_s_29 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_29) + 928)];
    }
  }
  for (int ax1_0_29 = 0; ax1_0_29 < 2; ++ax1_0_29) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_29 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_29 * 4)) >> 6) * 64) + (ax1_0_29 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2416) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_224 = 0; i_2_1_s_224 < 4; ++i_2_1_s_224) {
    if (i_2_1_s_224 < 2) {
      Y_local[(i_2_1_s_224 * 8)] = (Y_local[(i_2_1_s_224 * 8)] + (A_shared_dyn_local[i_2_1_s_224] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_225 = 0; i_2_1_s_225 < 4; ++i_2_1_s_225) {
    if (i_2_1_s_225 < 2) {
      Y_local[((i_2_1_s_225 * 8) + 1)] = (Y_local[((i_2_1_s_225 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_225] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_226 = 0; i_2_1_s_226 < 4; ++i_2_1_s_226) {
    if (i_2_1_s_226 < 2) {
      Y_local[((i_2_1_s_226 * 8) + 2)] = (Y_local[((i_2_1_s_226 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_226] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_227 = 0; i_2_1_s_227 < 4; ++i_2_1_s_227) {
    if (i_2_1_s_227 < 2) {
      Y_local[((i_2_1_s_227 * 8) + 3)] = (Y_local[((i_2_1_s_227 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_227] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_228 = 0; i_2_1_s_228 < 4; ++i_2_1_s_228) {
    if (i_2_1_s_228 < 2) {
      Y_local[((i_2_1_s_228 * 8) + 4)] = (Y_local[((i_2_1_s_228 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_228] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_229 = 0; i_2_1_s_229 < 4; ++i_2_1_s_229) {
    if (i_2_1_s_229 < 2) {
      Y_local[((i_2_1_s_229 * 8) + 5)] = (Y_local[((i_2_1_s_229 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_229] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_230 = 0; i_2_1_s_230 < 4; ++i_2_1_s_230) {
    if (i_2_1_s_230 < 2) {
      Y_local[((i_2_1_s_230 * 8) + 6)] = (Y_local[((i_2_1_s_230 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_230] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_231 = 0; i_2_1_s_231 < 4; ++i_2_1_s_231) {
    if (i_2_1_s_231 < 2) {
      Y_local[((i_2_1_s_231 * 8) + 7)] = (Y_local[((i_2_1_s_231 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_231] * B_shared_dyn_local[7]));
    }
  }
  for (int ax1_1_s_30 = 0; ax1_1_s_30 < 4; ++ax1_1_s_30) {
    if (ax1_1_s_30 < 2) {
      A_shared_dyn_local[ax1_1_s_30] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_30) + 960)];
    }
  }
  for (int ax1_0_30 = 0; ax1_0_30 < 2; ++ax1_0_30) {
    *(float4*)(B_shared_dyn_local + (ax1_0_30 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_30 * 4)) >> 6) * 64) + (ax1_0_30 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2464) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_232 = 0; i_2_1_s_232 < 4; ++i_2_1_s_232) {
    if (i_2_1_s_232 < 2) {
      Y_local[(i_2_1_s_232 * 8)] = (Y_local[(i_2_1_s_232 * 8)] + (A_shared_dyn_local[(i_2_1_s_232 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_233 = 0; i_2_1_s_233 < 4; ++i_2_1_s_233) {
    if (i_2_1_s_233 < 2) {
      Y_local[((i_2_1_s_233 * 8) + 1)] = (Y_local[((i_2_1_s_233 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_233 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_234 = 0; i_2_1_s_234 < 4; ++i_2_1_s_234) {
    if (i_2_1_s_234 < 2) {
      Y_local[((i_2_1_s_234 * 8) + 2)] = (Y_local[((i_2_1_s_234 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_234 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_235 = 0; i_2_1_s_235 < 4; ++i_2_1_s_235) {
    if (i_2_1_s_235 < 2) {
      Y_local[((i_2_1_s_235 * 8) + 3)] = (Y_local[((i_2_1_s_235 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_235 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_236 = 0; i_2_1_s_236 < 4; ++i_2_1_s_236) {
    if (i_2_1_s_236 < 2) {
      Y_local[((i_2_1_s_236 * 8) + 4)] = (Y_local[((i_2_1_s_236 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_236 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_237 = 0; i_2_1_s_237 < 4; ++i_2_1_s_237) {
    if (i_2_1_s_237 < 2) {
      Y_local[((i_2_1_s_237 * 8) + 5)] = (Y_local[((i_2_1_s_237 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_237 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_238 = 0; i_2_1_s_238 < 4; ++i_2_1_s_238) {
    if (i_2_1_s_238 < 2) {
      Y_local[((i_2_1_s_238 * 8) + 6)] = (Y_local[((i_2_1_s_238 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_238 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_239 = 0; i_2_1_s_239 < 4; ++i_2_1_s_239) {
    if (i_2_1_s_239 < 2) {
      Y_local[((i_2_1_s_239 * 8) + 7)] = (Y_local[((i_2_1_s_239 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_239 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_1_s_31 = 0; ax1_1_s_31 < 4; ++ax1_1_s_31) {
    if (ax1_1_s_31 < 2) {
      A_shared_dyn_local[(ax1_1_s_31 + 2)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 7) >> 2) * 16) + (((((int)threadIdx.x) & 31) >> 3) * 4)) + ((((int)threadIdx.x) & 1) * 2)) + ax1_1_s_31) + 992)];
    }
  }
  for (int ax1_0_31 = 0; ax1_0_31 < 2; ++ax1_0_31) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_31 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((((int)blockIdx.x) & 7) * 32) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_31 * 4)) >> 6) * 64) + (ax1_0_31 * 32)) + ((((((((int)blockIdx.x) & 7) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + ((((int)threadIdx.x) & 3) >> 1)) & 7) * 4)) + 2512) - ((((int)blockIdx.x) & 1) * 16)) - (((((int)blockIdx.x) & 7) >> 1) * 64)));
  }
  for (int i_2_1_s_240 = 0; i_2_1_s_240 < 4; ++i_2_1_s_240) {
    if (i_2_1_s_240 < 2) {
      Y_local[(i_2_1_s_240 * 8)] = (Y_local[(i_2_1_s_240 * 8)] + (A_shared_dyn_local[i_2_1_s_240] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_241 = 0; i_2_1_s_241 < 4; ++i_2_1_s_241) {
    if (i_2_1_s_241 < 2) {
      Y_local[((i_2_1_s_241 * 8) + 1)] = (Y_local[((i_2_1_s_241 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_241] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_242 = 0; i_2_1_s_242 < 4; ++i_2_1_s_242) {
    if (i_2_1_s_242 < 2) {
      Y_local[((i_2_1_s_242 * 8) + 2)] = (Y_local[((i_2_1_s_242 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_242] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_243 = 0; i_2_1_s_243 < 4; ++i_2_1_s_243) {
    if (i_2_1_s_243 < 2) {
      Y_local[((i_2_1_s_243 * 8) + 3)] = (Y_local[((i_2_1_s_243 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_243] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_244 = 0; i_2_1_s_244 < 4; ++i_2_1_s_244) {
    if (i_2_1_s_244 < 2) {
      Y_local[((i_2_1_s_244 * 8) + 4)] = (Y_local[((i_2_1_s_244 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_244] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_245 = 0; i_2_1_s_245 < 4; ++i_2_1_s_245) {
    if (i_2_1_s_245 < 2) {
      Y_local[((i_2_1_s_245 * 8) + 5)] = (Y_local[((i_2_1_s_245 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_245] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_246 = 0; i_2_1_s_246 < 4; ++i_2_1_s_246) {
    if (i_2_1_s_246 < 2) {
      Y_local[((i_2_1_s_246 * 8) + 6)] = (Y_local[((i_2_1_s_246 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_246] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_247 = 0; i_2_1_s_247 < 4; ++i_2_1_s_247) {
    if (i_2_1_s_247 < 2) {
      Y_local[((i_2_1_s_247 * 8) + 7)] = (Y_local[((i_2_1_s_247 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_247] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_248 = 0; i_2_1_s_248 < 4; ++i_2_1_s_248) {
    if (i_2_1_s_248 < 2) {
      Y_local[(i_2_1_s_248 * 8)] = (Y_local[(i_2_1_s_248 * 8)] + (A_shared_dyn_local[(i_2_1_s_248 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_249 = 0; i_2_1_s_249 < 4; ++i_2_1_s_249) {
    if (i_2_1_s_249 < 2) {
      Y_local[((i_2_1_s_249 * 8) + 1)] = (Y_local[((i_2_1_s_249 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_249 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_250 = 0; i_2_1_s_250 < 4; ++i_2_1_s_250) {
    if (i_2_1_s_250 < 2) {
      Y_local[((i_2_1_s_250 * 8) + 2)] = (Y_local[((i_2_1_s_250 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_250 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_251 = 0; i_2_1_s_251 < 4; ++i_2_1_s_251) {
    if (i_2_1_s_251 < 2) {
      Y_local[((i_2_1_s_251 * 8) + 3)] = (Y_local[((i_2_1_s_251 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_251 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_252 = 0; i_2_1_s_252 < 4; ++i_2_1_s_252) {
    if (i_2_1_s_252 < 2) {
      Y_local[((i_2_1_s_252 * 8) + 4)] = (Y_local[((i_2_1_s_252 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_252 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_253 = 0; i_2_1_s_253 < 4; ++i_2_1_s_253) {
    if (i_2_1_s_253 < 2) {
      Y_local[((i_2_1_s_253 * 8) + 5)] = (Y_local[((i_2_1_s_253 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_253 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_254 = 0; i_2_1_s_254 < 4; ++i_2_1_s_254) {
    if (i_2_1_s_254 < 2) {
      Y_local[((i_2_1_s_254 * 8) + 6)] = (Y_local[((i_2_1_s_254 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_254 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_255 = 0; i_2_1_s_255 < 4; ++i_2_1_s_255) {
    if (i_2_1_s_255 < 2) {
      Y_local[((i_2_1_s_255 * 8) + 7)] = (Y_local[((i_2_1_s_255 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_255 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_0_32 = 0; ax1_0_32 < 2; ++ax1_0_32) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 8192) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_32 * 4))) = *(float4*)(Y_local + (ax1_0_32 * 4));
  }
  for (int ax1_0_33 = 0; ax1_0_33 < 2; ++ax1_0_33) {
    *(float4*)(Y + (((((((((((int)blockIdx.x) >> 3) * 8192) + (((((int)threadIdx.x) & 31) >> 2) * 1024)) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)blockIdx.x) & 7) * 32)) + ((((int)threadIdx.x) >> 5) * 16)) + (((((int)threadIdx.x) & 3) >> 1) * 8)) + (ax1_0_33 * 4)) + 256)) = *(float4*)(Y_local + ((ax1_0_33 * 4) + 8));
  }
}


