
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(125) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[128];
  float A_shared_dyn_local[16];
  float B_shared_dyn_local[32];
  Y_local[0] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[32] = 0.000000e+00f;
  Y_local[48] = 0.000000e+00f;
  Y_local[64] = 0.000000e+00f;
  Y_local[80] = 0.000000e+00f;
  Y_local[96] = 0.000000e+00f;
  Y_local[112] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[33] = 0.000000e+00f;
  Y_local[49] = 0.000000e+00f;
  Y_local[65] = 0.000000e+00f;
  Y_local[81] = 0.000000e+00f;
  Y_local[97] = 0.000000e+00f;
  Y_local[113] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[34] = 0.000000e+00f;
  Y_local[50] = 0.000000e+00f;
  Y_local[66] = 0.000000e+00f;
  Y_local[82] = 0.000000e+00f;
  Y_local[98] = 0.000000e+00f;
  Y_local[114] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[35] = 0.000000e+00f;
  Y_local[51] = 0.000000e+00f;
  Y_local[67] = 0.000000e+00f;
  Y_local[83] = 0.000000e+00f;
  Y_local[99] = 0.000000e+00f;
  Y_local[115] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[36] = 0.000000e+00f;
  Y_local[52] = 0.000000e+00f;
  Y_local[68] = 0.000000e+00f;
  Y_local[84] = 0.000000e+00f;
  Y_local[100] = 0.000000e+00f;
  Y_local[116] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[37] = 0.000000e+00f;
  Y_local[53] = 0.000000e+00f;
  Y_local[69] = 0.000000e+00f;
  Y_local[85] = 0.000000e+00f;
  Y_local[101] = 0.000000e+00f;
  Y_local[117] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[38] = 0.000000e+00f;
  Y_local[54] = 0.000000e+00f;
  Y_local[70] = 0.000000e+00f;
  Y_local[86] = 0.000000e+00f;
  Y_local[102] = 0.000000e+00f;
  Y_local[118] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;
  Y_local[39] = 0.000000e+00f;
  Y_local[55] = 0.000000e+00f;
  Y_local[71] = 0.000000e+00f;
  Y_local[87] = 0.000000e+00f;
  Y_local[103] = 0.000000e+00f;
  Y_local[119] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[24] = 0.000000e+00f;
  Y_local[40] = 0.000000e+00f;
  Y_local[56] = 0.000000e+00f;
  Y_local[72] = 0.000000e+00f;
  Y_local[88] = 0.000000e+00f;
  Y_local[104] = 0.000000e+00f;
  Y_local[120] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[25] = 0.000000e+00f;
  Y_local[41] = 0.000000e+00f;
  Y_local[57] = 0.000000e+00f;
  Y_local[73] = 0.000000e+00f;
  Y_local[89] = 0.000000e+00f;
  Y_local[105] = 0.000000e+00f;
  Y_local[121] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[26] = 0.000000e+00f;
  Y_local[42] = 0.000000e+00f;
  Y_local[58] = 0.000000e+00f;
  Y_local[74] = 0.000000e+00f;
  Y_local[90] = 0.000000e+00f;
  Y_local[106] = 0.000000e+00f;
  Y_local[122] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[27] = 0.000000e+00f;
  Y_local[43] = 0.000000e+00f;
  Y_local[59] = 0.000000e+00f;
  Y_local[75] = 0.000000e+00f;
  Y_local[91] = 0.000000e+00f;
  Y_local[107] = 0.000000e+00f;
  Y_local[123] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[28] = 0.000000e+00f;
  Y_local[44] = 0.000000e+00f;
  Y_local[60] = 0.000000e+00f;
  Y_local[76] = 0.000000e+00f;
  Y_local[92] = 0.000000e+00f;
  Y_local[108] = 0.000000e+00f;
  Y_local[124] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[29] = 0.000000e+00f;
  Y_local[45] = 0.000000e+00f;
  Y_local[61] = 0.000000e+00f;
  Y_local[77] = 0.000000e+00f;
  Y_local[93] = 0.000000e+00f;
  Y_local[109] = 0.000000e+00f;
  Y_local[125] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[30] = 0.000000e+00f;
  Y_local[46] = 0.000000e+00f;
  Y_local[62] = 0.000000e+00f;
  Y_local[78] = 0.000000e+00f;
  Y_local[94] = 0.000000e+00f;
  Y_local[110] = 0.000000e+00f;
  Y_local[126] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[31] = 0.000000e+00f;
  Y_local[47] = 0.000000e+00f;
  Y_local[63] = 0.000000e+00f;
  Y_local[79] = 0.000000e+00f;
  Y_local[95] = 0.000000e+00f;
  Y_local[111] = 0.000000e+00f;
  Y_local[127] = 0.000000e+00f;
  for (int ax0_ax1_fused_2 = 0; ax0_ax1_fused_2 < 10; ++ax0_ax1_fused_2) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 40) + (ax0_ax1_fused_2 * 4)) + 7680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)threadIdx.x) / 20) * 1000) + ((((int)blockIdx.x) / 25) * 200)) + ((((int)threadIdx.x) % 20) * 10)) + ax0_ax1_fused_2))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_1 = 0; ax0_ax1_fused_2_1 < 4; ++ax0_ax1_fused_2_1) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((int)threadIdx.x) * 16) + (ax0_ax1_fused_2_1 * 4))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) / 20) * 2000) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax0_ax1_fused_2_1))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_2 = 0; ax0_ax1_fused_2_2 < 10; ++ax0_ax1_fused_2_2) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 40) + (ax0_ax1_fused_2_2 * 4)) + 12480)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)threadIdx.x) / 20) * 1000) + ((((int)blockIdx.x) / 25) * 200)) + ((((int)threadIdx.x) % 20) * 10)) + ax0_ax1_fused_2_2) + 6000))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_3 = 0; ax0_ax1_fused_2_3 < 4; ++ax0_ax1_fused_2_3) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 16) + (ax0_ax1_fused_2_3 * 4)) + 1920)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) / 20) * 2000) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax0_ax1_fused_2_3) + 12000))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_4 = 0; ax0_ax1_fused_2_4 < 10; ++ax0_ax1_fused_2_4) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 40) + (ax0_ax1_fused_2_4 * 4)) + 17280)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)threadIdx.x) / 20) * 1000) + ((((int)blockIdx.x) / 25) * 200)) + ((((int)threadIdx.x) % 20) * 10)) + ax0_ax1_fused_2_4) + 12000))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_5 = 0; ax0_ax1_fused_2_5 < 4; ++ax0_ax1_fused_2_5) {
    if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 16) + (ax0_ax1_fused_2_5 * 4)) + 3840)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) / 20) * 2000) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax0_ax1_fused_2_5) + 24000))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) {
    *(float4*)(A_shared_dyn_local + (ax1_0 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0 * 4)) + 1920));
  }
  for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
    *(float4*)(B_shared_dyn_local + (ax1_0_1 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((int)threadIdx.x) / 25) * 16) + (ax1_0_1 * 4)));
  }
  for (int k_0 = 0; k_0 < 497; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_2_6 = 0; ax0_ax1_fused_2_6 < 10; ++ax0_ax1_fused_2_6) {
      if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((k_0 + 3) & 3) * 4800) + (((int)threadIdx.x) * 40)) + (ax0_ax1_fused_2_6 * 4)) + 7680)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((k_0 * 6000) + ((((int)threadIdx.x) / 20) * 1000)) + ((((int)blockIdx.x) / 25) * 200)) + ((((int)threadIdx.x) % 20) * 10)) + ax0_ax1_fused_2_6) + 18000))), "n"(4)
    );
  }
      }
    }
    for (int ax0_ax1_fused_2_7 = 0; ax0_ax1_fused_2_7 < 4; ++ax0_ax1_fused_2_7) {
      if (((int)threadIdx.x) < 120) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((k_0 + 3) & 3) * 1920) + (((int)threadIdx.x) * 16)) + (ax0_ax1_fused_2_7 * 4))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((k_0 * 12000) + ((((int)threadIdx.x) / 20) * 2000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) % 20) * 4)) + ax0_ax1_fused_2_7) + 36000))), "n"(4)
    );
  }
      }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 2;");

    __syncthreads();
    for (int ax1_0_2 = 0; ax1_0_2 < 2; ++ax1_0_2) {
      *(float4*)(A_shared_dyn_local + ((ax1_0_2 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_2 * 4)) + 2120));
    }
    for (int ax1_0_3 = 0; ax1_0_3 < 4; ++ax1_0_3) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_3 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_3 * 4)) + 80));
    }
    for (int i_2_1 = 0; i_2_1 < 4; ++i_2_1) {
      Y_local[(i_2_1 * 16)] = (Y_local[(i_2_1 * 16)] + (A_shared_dyn_local[i_2_1] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_1 = 0; i_2_1_1 < 4; ++i_2_1_1) {
      Y_local[((i_2_1_1 * 16) + 64)] = (Y_local[((i_2_1_1 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_1 + 4)] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_2 = 0; i_2_1_2 < 4; ++i_2_1_2) {
      Y_local[((i_2_1_2 * 16) + 1)] = (Y_local[((i_2_1_2 * 16) + 1)] + (A_shared_dyn_local[i_2_1_2] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_3 = 0; i_2_1_3 < 4; ++i_2_1_3) {
      Y_local[((i_2_1_3 * 16) + 65)] = (Y_local[((i_2_1_3 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_3 + 4)] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_4 = 0; i_2_1_4 < 4; ++i_2_1_4) {
      Y_local[((i_2_1_4 * 16) + 2)] = (Y_local[((i_2_1_4 * 16) + 2)] + (A_shared_dyn_local[i_2_1_4] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_5 = 0; i_2_1_5 < 4; ++i_2_1_5) {
      Y_local[((i_2_1_5 * 16) + 66)] = (Y_local[((i_2_1_5 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_5 + 4)] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_6 = 0; i_2_1_6 < 4; ++i_2_1_6) {
      Y_local[((i_2_1_6 * 16) + 3)] = (Y_local[((i_2_1_6 * 16) + 3)] + (A_shared_dyn_local[i_2_1_6] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_7 = 0; i_2_1_7 < 4; ++i_2_1_7) {
      Y_local[((i_2_1_7 * 16) + 67)] = (Y_local[((i_2_1_7 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_7 + 4)] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_8 = 0; i_2_1_8 < 4; ++i_2_1_8) {
      Y_local[((i_2_1_8 * 16) + 4)] = (Y_local[((i_2_1_8 * 16) + 4)] + (A_shared_dyn_local[i_2_1_8] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_9 = 0; i_2_1_9 < 4; ++i_2_1_9) {
      Y_local[((i_2_1_9 * 16) + 68)] = (Y_local[((i_2_1_9 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_9 + 4)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_10 = 0; i_2_1_10 < 4; ++i_2_1_10) {
      Y_local[((i_2_1_10 * 16) + 5)] = (Y_local[((i_2_1_10 * 16) + 5)] + (A_shared_dyn_local[i_2_1_10] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_11 = 0; i_2_1_11 < 4; ++i_2_1_11) {
      Y_local[((i_2_1_11 * 16) + 69)] = (Y_local[((i_2_1_11 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_11 + 4)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_12 = 0; i_2_1_12 < 4; ++i_2_1_12) {
      Y_local[((i_2_1_12 * 16) + 6)] = (Y_local[((i_2_1_12 * 16) + 6)] + (A_shared_dyn_local[i_2_1_12] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_13 = 0; i_2_1_13 < 4; ++i_2_1_13) {
      Y_local[((i_2_1_13 * 16) + 70)] = (Y_local[((i_2_1_13 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_13 + 4)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_14 = 0; i_2_1_14 < 4; ++i_2_1_14) {
      Y_local[((i_2_1_14 * 16) + 7)] = (Y_local[((i_2_1_14 * 16) + 7)] + (A_shared_dyn_local[i_2_1_14] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_15 = 0; i_2_1_15 < 4; ++i_2_1_15) {
      Y_local[((i_2_1_15 * 16) + 71)] = (Y_local[((i_2_1_15 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_15 + 4)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_16 = 0; i_2_1_16 < 4; ++i_2_1_16) {
      Y_local[((i_2_1_16 * 16) + 8)] = (Y_local[((i_2_1_16 * 16) + 8)] + (A_shared_dyn_local[i_2_1_16] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_17 = 0; i_2_1_17 < 4; ++i_2_1_17) {
      Y_local[((i_2_1_17 * 16) + 72)] = (Y_local[((i_2_1_17 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_17 + 4)] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_18 = 0; i_2_1_18 < 4; ++i_2_1_18) {
      Y_local[((i_2_1_18 * 16) + 9)] = (Y_local[((i_2_1_18 * 16) + 9)] + (A_shared_dyn_local[i_2_1_18] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_19 = 0; i_2_1_19 < 4; ++i_2_1_19) {
      Y_local[((i_2_1_19 * 16) + 73)] = (Y_local[((i_2_1_19 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_19 + 4)] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_20 = 0; i_2_1_20 < 4; ++i_2_1_20) {
      Y_local[((i_2_1_20 * 16) + 10)] = (Y_local[((i_2_1_20 * 16) + 10)] + (A_shared_dyn_local[i_2_1_20] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_21 = 0; i_2_1_21 < 4; ++i_2_1_21) {
      Y_local[((i_2_1_21 * 16) + 74)] = (Y_local[((i_2_1_21 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_21 + 4)] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_22 = 0; i_2_1_22 < 4; ++i_2_1_22) {
      Y_local[((i_2_1_22 * 16) + 11)] = (Y_local[((i_2_1_22 * 16) + 11)] + (A_shared_dyn_local[i_2_1_22] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_23 = 0; i_2_1_23 < 4; ++i_2_1_23) {
      Y_local[((i_2_1_23 * 16) + 75)] = (Y_local[((i_2_1_23 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_23 + 4)] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_24 = 0; i_2_1_24 < 4; ++i_2_1_24) {
      Y_local[((i_2_1_24 * 16) + 12)] = (Y_local[((i_2_1_24 * 16) + 12)] + (A_shared_dyn_local[i_2_1_24] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_25 = 0; i_2_1_25 < 4; ++i_2_1_25) {
      Y_local[((i_2_1_25 * 16) + 76)] = (Y_local[((i_2_1_25 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_25 + 4)] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_26 = 0; i_2_1_26 < 4; ++i_2_1_26) {
      Y_local[((i_2_1_26 * 16) + 13)] = (Y_local[((i_2_1_26 * 16) + 13)] + (A_shared_dyn_local[i_2_1_26] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_27 = 0; i_2_1_27 < 4; ++i_2_1_27) {
      Y_local[((i_2_1_27 * 16) + 77)] = (Y_local[((i_2_1_27 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_27 + 4)] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_28 = 0; i_2_1_28 < 4; ++i_2_1_28) {
      Y_local[((i_2_1_28 * 16) + 14)] = (Y_local[((i_2_1_28 * 16) + 14)] + (A_shared_dyn_local[i_2_1_28] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_29 = 0; i_2_1_29 < 4; ++i_2_1_29) {
      Y_local[((i_2_1_29 * 16) + 78)] = (Y_local[((i_2_1_29 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_29 + 4)] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_30 = 0; i_2_1_30 < 4; ++i_2_1_30) {
      Y_local[((i_2_1_30 * 16) + 15)] = (Y_local[((i_2_1_30 * 16) + 15)] + (A_shared_dyn_local[i_2_1_30] * B_shared_dyn_local[15]));
    }
    for (int i_2_1_31 = 0; i_2_1_31 < 4; ++i_2_1_31) {
      Y_local[((i_2_1_31 * 16) + 79)] = (Y_local[((i_2_1_31 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_31 + 4)] * B_shared_dyn_local[15]));
    }
    for (int ax1_0_4 = 0; ax1_0_4 < 2; ++ax1_0_4) {
      *(float4*)(A_shared_dyn_local + (ax1_0_4 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_4 * 4)) + 2320));
    }
    for (int ax1_0_5 = 0; ax1_0_5 < 4; ++ax1_0_5) {
      *(float4*)(B_shared_dyn_local + (ax1_0_5 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_5 * 4)) + 160));
    }
    for (int i_2_1_32 = 0; i_2_1_32 < 4; ++i_2_1_32) {
      Y_local[(i_2_1_32 * 16)] = (Y_local[(i_2_1_32 * 16)] + (A_shared_dyn_local[(i_2_1_32 + 8)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_33 = 0; i_2_1_33 < 4; ++i_2_1_33) {
      Y_local[((i_2_1_33 * 16) + 64)] = (Y_local[((i_2_1_33 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_33 + 12)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_34 = 0; i_2_1_34 < 4; ++i_2_1_34) {
      Y_local[((i_2_1_34 * 16) + 1)] = (Y_local[((i_2_1_34 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_34 + 8)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_35 = 0; i_2_1_35 < 4; ++i_2_1_35) {
      Y_local[((i_2_1_35 * 16) + 65)] = (Y_local[((i_2_1_35 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_35 + 12)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_36 = 0; i_2_1_36 < 4; ++i_2_1_36) {
      Y_local[((i_2_1_36 * 16) + 2)] = (Y_local[((i_2_1_36 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_36 + 8)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_37 = 0; i_2_1_37 < 4; ++i_2_1_37) {
      Y_local[((i_2_1_37 * 16) + 66)] = (Y_local[((i_2_1_37 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_37 + 12)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_38 = 0; i_2_1_38 < 4; ++i_2_1_38) {
      Y_local[((i_2_1_38 * 16) + 3)] = (Y_local[((i_2_1_38 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_38 + 8)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_39 = 0; i_2_1_39 < 4; ++i_2_1_39) {
      Y_local[((i_2_1_39 * 16) + 67)] = (Y_local[((i_2_1_39 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_39 + 12)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_40 = 0; i_2_1_40 < 4; ++i_2_1_40) {
      Y_local[((i_2_1_40 * 16) + 4)] = (Y_local[((i_2_1_40 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_40 + 8)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_41 = 0; i_2_1_41 < 4; ++i_2_1_41) {
      Y_local[((i_2_1_41 * 16) + 68)] = (Y_local[((i_2_1_41 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_41 + 12)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_42 = 0; i_2_1_42 < 4; ++i_2_1_42) {
      Y_local[((i_2_1_42 * 16) + 5)] = (Y_local[((i_2_1_42 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_42 + 8)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_43 = 0; i_2_1_43 < 4; ++i_2_1_43) {
      Y_local[((i_2_1_43 * 16) + 69)] = (Y_local[((i_2_1_43 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_43 + 12)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_44 = 0; i_2_1_44 < 4; ++i_2_1_44) {
      Y_local[((i_2_1_44 * 16) + 6)] = (Y_local[((i_2_1_44 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_44 + 8)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_45 = 0; i_2_1_45 < 4; ++i_2_1_45) {
      Y_local[((i_2_1_45 * 16) + 70)] = (Y_local[((i_2_1_45 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_45 + 12)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_46 = 0; i_2_1_46 < 4; ++i_2_1_46) {
      Y_local[((i_2_1_46 * 16) + 7)] = (Y_local[((i_2_1_46 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_46 + 8)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_47 = 0; i_2_1_47 < 4; ++i_2_1_47) {
      Y_local[((i_2_1_47 * 16) + 71)] = (Y_local[((i_2_1_47 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_47 + 12)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_48 = 0; i_2_1_48 < 4; ++i_2_1_48) {
      Y_local[((i_2_1_48 * 16) + 8)] = (Y_local[((i_2_1_48 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_48 + 8)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_49 = 0; i_2_1_49 < 4; ++i_2_1_49) {
      Y_local[((i_2_1_49 * 16) + 72)] = (Y_local[((i_2_1_49 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_49 + 12)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_50 = 0; i_2_1_50 < 4; ++i_2_1_50) {
      Y_local[((i_2_1_50 * 16) + 9)] = (Y_local[((i_2_1_50 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_50 + 8)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_51 = 0; i_2_1_51 < 4; ++i_2_1_51) {
      Y_local[((i_2_1_51 * 16) + 73)] = (Y_local[((i_2_1_51 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_51 + 12)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_52 = 0; i_2_1_52 < 4; ++i_2_1_52) {
      Y_local[((i_2_1_52 * 16) + 10)] = (Y_local[((i_2_1_52 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_52 + 8)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_53 = 0; i_2_1_53 < 4; ++i_2_1_53) {
      Y_local[((i_2_1_53 * 16) + 74)] = (Y_local[((i_2_1_53 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_53 + 12)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_54 = 0; i_2_1_54 < 4; ++i_2_1_54) {
      Y_local[((i_2_1_54 * 16) + 11)] = (Y_local[((i_2_1_54 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_54 + 8)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_55 = 0; i_2_1_55 < 4; ++i_2_1_55) {
      Y_local[((i_2_1_55 * 16) + 75)] = (Y_local[((i_2_1_55 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_55 + 12)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_56 = 0; i_2_1_56 < 4; ++i_2_1_56) {
      Y_local[((i_2_1_56 * 16) + 12)] = (Y_local[((i_2_1_56 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_56 + 8)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_57 = 0; i_2_1_57 < 4; ++i_2_1_57) {
      Y_local[((i_2_1_57 * 16) + 76)] = (Y_local[((i_2_1_57 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_57 + 12)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_58 = 0; i_2_1_58 < 4; ++i_2_1_58) {
      Y_local[((i_2_1_58 * 16) + 13)] = (Y_local[((i_2_1_58 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_58 + 8)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_59 = 0; i_2_1_59 < 4; ++i_2_1_59) {
      Y_local[((i_2_1_59 * 16) + 77)] = (Y_local[((i_2_1_59 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_59 + 12)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_60 = 0; i_2_1_60 < 4; ++i_2_1_60) {
      Y_local[((i_2_1_60 * 16) + 14)] = (Y_local[((i_2_1_60 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_60 + 8)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_61 = 0; i_2_1_61 < 4; ++i_2_1_61) {
      Y_local[((i_2_1_61 * 16) + 78)] = (Y_local[((i_2_1_61 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_61 + 12)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_62 = 0; i_2_1_62 < 4; ++i_2_1_62) {
      Y_local[((i_2_1_62 * 16) + 15)] = (Y_local[((i_2_1_62 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_62 + 8)] * B_shared_dyn_local[31]));
    }
    for (int i_2_1_63 = 0; i_2_1_63 < 4; ++i_2_1_63) {
      Y_local[((i_2_1_63 * 16) + 79)] = (Y_local[((i_2_1_63 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_63 + 12)] * B_shared_dyn_local[31]));
    }
    for (int ax1_0_6 = 0; ax1_0_6 < 2; ++ax1_0_6) {
      *(float4*)(A_shared_dyn_local + ((ax1_0_6 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_6 * 4)) + 2520));
    }
    for (int ax1_0_7 = 0; ax1_0_7 < 4; ++ax1_0_7) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_7 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_7 * 4)) + 240));
    }
    for (int i_2_1_64 = 0; i_2_1_64 < 4; ++i_2_1_64) {
      Y_local[(i_2_1_64 * 16)] = (Y_local[(i_2_1_64 * 16)] + (A_shared_dyn_local[i_2_1_64] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_65 = 0; i_2_1_65 < 4; ++i_2_1_65) {
      Y_local[((i_2_1_65 * 16) + 64)] = (Y_local[((i_2_1_65 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_65 + 4)] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_66 = 0; i_2_1_66 < 4; ++i_2_1_66) {
      Y_local[((i_2_1_66 * 16) + 1)] = (Y_local[((i_2_1_66 * 16) + 1)] + (A_shared_dyn_local[i_2_1_66] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_67 = 0; i_2_1_67 < 4; ++i_2_1_67) {
      Y_local[((i_2_1_67 * 16) + 65)] = (Y_local[((i_2_1_67 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_67 + 4)] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_68 = 0; i_2_1_68 < 4; ++i_2_1_68) {
      Y_local[((i_2_1_68 * 16) + 2)] = (Y_local[((i_2_1_68 * 16) + 2)] + (A_shared_dyn_local[i_2_1_68] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_69 = 0; i_2_1_69 < 4; ++i_2_1_69) {
      Y_local[((i_2_1_69 * 16) + 66)] = (Y_local[((i_2_1_69 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_69 + 4)] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_70 = 0; i_2_1_70 < 4; ++i_2_1_70) {
      Y_local[((i_2_1_70 * 16) + 3)] = (Y_local[((i_2_1_70 * 16) + 3)] + (A_shared_dyn_local[i_2_1_70] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_71 = 0; i_2_1_71 < 4; ++i_2_1_71) {
      Y_local[((i_2_1_71 * 16) + 67)] = (Y_local[((i_2_1_71 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_71 + 4)] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_72 = 0; i_2_1_72 < 4; ++i_2_1_72) {
      Y_local[((i_2_1_72 * 16) + 4)] = (Y_local[((i_2_1_72 * 16) + 4)] + (A_shared_dyn_local[i_2_1_72] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_73 = 0; i_2_1_73 < 4; ++i_2_1_73) {
      Y_local[((i_2_1_73 * 16) + 68)] = (Y_local[((i_2_1_73 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_73 + 4)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_74 = 0; i_2_1_74 < 4; ++i_2_1_74) {
      Y_local[((i_2_1_74 * 16) + 5)] = (Y_local[((i_2_1_74 * 16) + 5)] + (A_shared_dyn_local[i_2_1_74] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_75 = 0; i_2_1_75 < 4; ++i_2_1_75) {
      Y_local[((i_2_1_75 * 16) + 69)] = (Y_local[((i_2_1_75 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_75 + 4)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_76 = 0; i_2_1_76 < 4; ++i_2_1_76) {
      Y_local[((i_2_1_76 * 16) + 6)] = (Y_local[((i_2_1_76 * 16) + 6)] + (A_shared_dyn_local[i_2_1_76] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_77 = 0; i_2_1_77 < 4; ++i_2_1_77) {
      Y_local[((i_2_1_77 * 16) + 70)] = (Y_local[((i_2_1_77 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_77 + 4)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_78 = 0; i_2_1_78 < 4; ++i_2_1_78) {
      Y_local[((i_2_1_78 * 16) + 7)] = (Y_local[((i_2_1_78 * 16) + 7)] + (A_shared_dyn_local[i_2_1_78] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_79 = 0; i_2_1_79 < 4; ++i_2_1_79) {
      Y_local[((i_2_1_79 * 16) + 71)] = (Y_local[((i_2_1_79 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_79 + 4)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_80 = 0; i_2_1_80 < 4; ++i_2_1_80) {
      Y_local[((i_2_1_80 * 16) + 8)] = (Y_local[((i_2_1_80 * 16) + 8)] + (A_shared_dyn_local[i_2_1_80] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_81 = 0; i_2_1_81 < 4; ++i_2_1_81) {
      Y_local[((i_2_1_81 * 16) + 72)] = (Y_local[((i_2_1_81 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_81 + 4)] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_82 = 0; i_2_1_82 < 4; ++i_2_1_82) {
      Y_local[((i_2_1_82 * 16) + 9)] = (Y_local[((i_2_1_82 * 16) + 9)] + (A_shared_dyn_local[i_2_1_82] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_83 = 0; i_2_1_83 < 4; ++i_2_1_83) {
      Y_local[((i_2_1_83 * 16) + 73)] = (Y_local[((i_2_1_83 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_83 + 4)] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_84 = 0; i_2_1_84 < 4; ++i_2_1_84) {
      Y_local[((i_2_1_84 * 16) + 10)] = (Y_local[((i_2_1_84 * 16) + 10)] + (A_shared_dyn_local[i_2_1_84] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_85 = 0; i_2_1_85 < 4; ++i_2_1_85) {
      Y_local[((i_2_1_85 * 16) + 74)] = (Y_local[((i_2_1_85 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_85 + 4)] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_86 = 0; i_2_1_86 < 4; ++i_2_1_86) {
      Y_local[((i_2_1_86 * 16) + 11)] = (Y_local[((i_2_1_86 * 16) + 11)] + (A_shared_dyn_local[i_2_1_86] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_87 = 0; i_2_1_87 < 4; ++i_2_1_87) {
      Y_local[((i_2_1_87 * 16) + 75)] = (Y_local[((i_2_1_87 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_87 + 4)] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_88 = 0; i_2_1_88 < 4; ++i_2_1_88) {
      Y_local[((i_2_1_88 * 16) + 12)] = (Y_local[((i_2_1_88 * 16) + 12)] + (A_shared_dyn_local[i_2_1_88] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_89 = 0; i_2_1_89 < 4; ++i_2_1_89) {
      Y_local[((i_2_1_89 * 16) + 76)] = (Y_local[((i_2_1_89 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_89 + 4)] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_90 = 0; i_2_1_90 < 4; ++i_2_1_90) {
      Y_local[((i_2_1_90 * 16) + 13)] = (Y_local[((i_2_1_90 * 16) + 13)] + (A_shared_dyn_local[i_2_1_90] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_91 = 0; i_2_1_91 < 4; ++i_2_1_91) {
      Y_local[((i_2_1_91 * 16) + 77)] = (Y_local[((i_2_1_91 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_91 + 4)] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_92 = 0; i_2_1_92 < 4; ++i_2_1_92) {
      Y_local[((i_2_1_92 * 16) + 14)] = (Y_local[((i_2_1_92 * 16) + 14)] + (A_shared_dyn_local[i_2_1_92] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_93 = 0; i_2_1_93 < 4; ++i_2_1_93) {
      Y_local[((i_2_1_93 * 16) + 78)] = (Y_local[((i_2_1_93 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_93 + 4)] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_94 = 0; i_2_1_94 < 4; ++i_2_1_94) {
      Y_local[((i_2_1_94 * 16) + 15)] = (Y_local[((i_2_1_94 * 16) + 15)] + (A_shared_dyn_local[i_2_1_94] * B_shared_dyn_local[15]));
    }
    for (int i_2_1_95 = 0; i_2_1_95 < 4; ++i_2_1_95) {
      Y_local[((i_2_1_95 * 16) + 79)] = (Y_local[((i_2_1_95 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_95 + 4)] * B_shared_dyn_local[15]));
    }
    for (int ax1_0_8 = 0; ax1_0_8 < 2; ++ax1_0_8) {
      *(float4*)(A_shared_dyn_local + (ax1_0_8 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_8 * 4)) + 2720));
    }
    for (int ax1_0_9 = 0; ax1_0_9 < 4; ++ax1_0_9) {
      *(float4*)(B_shared_dyn_local + (ax1_0_9 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_9 * 4)) + 320));
    }
    for (int i_2_1_96 = 0; i_2_1_96 < 4; ++i_2_1_96) {
      Y_local[(i_2_1_96 * 16)] = (Y_local[(i_2_1_96 * 16)] + (A_shared_dyn_local[(i_2_1_96 + 8)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_97 = 0; i_2_1_97 < 4; ++i_2_1_97) {
      Y_local[((i_2_1_97 * 16) + 64)] = (Y_local[((i_2_1_97 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_97 + 12)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_98 = 0; i_2_1_98 < 4; ++i_2_1_98) {
      Y_local[((i_2_1_98 * 16) + 1)] = (Y_local[((i_2_1_98 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_98 + 8)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_99 = 0; i_2_1_99 < 4; ++i_2_1_99) {
      Y_local[((i_2_1_99 * 16) + 65)] = (Y_local[((i_2_1_99 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_99 + 12)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_100 = 0; i_2_1_100 < 4; ++i_2_1_100) {
      Y_local[((i_2_1_100 * 16) + 2)] = (Y_local[((i_2_1_100 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_100 + 8)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_101 = 0; i_2_1_101 < 4; ++i_2_1_101) {
      Y_local[((i_2_1_101 * 16) + 66)] = (Y_local[((i_2_1_101 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_101 + 12)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_102 = 0; i_2_1_102 < 4; ++i_2_1_102) {
      Y_local[((i_2_1_102 * 16) + 3)] = (Y_local[((i_2_1_102 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_102 + 8)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_103 = 0; i_2_1_103 < 4; ++i_2_1_103) {
      Y_local[((i_2_1_103 * 16) + 67)] = (Y_local[((i_2_1_103 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_103 + 12)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_104 = 0; i_2_1_104 < 4; ++i_2_1_104) {
      Y_local[((i_2_1_104 * 16) + 4)] = (Y_local[((i_2_1_104 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_104 + 8)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_105 = 0; i_2_1_105 < 4; ++i_2_1_105) {
      Y_local[((i_2_1_105 * 16) + 68)] = (Y_local[((i_2_1_105 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_105 + 12)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_106 = 0; i_2_1_106 < 4; ++i_2_1_106) {
      Y_local[((i_2_1_106 * 16) + 5)] = (Y_local[((i_2_1_106 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_106 + 8)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_107 = 0; i_2_1_107 < 4; ++i_2_1_107) {
      Y_local[((i_2_1_107 * 16) + 69)] = (Y_local[((i_2_1_107 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_107 + 12)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_108 = 0; i_2_1_108 < 4; ++i_2_1_108) {
      Y_local[((i_2_1_108 * 16) + 6)] = (Y_local[((i_2_1_108 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_108 + 8)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_109 = 0; i_2_1_109 < 4; ++i_2_1_109) {
      Y_local[((i_2_1_109 * 16) + 70)] = (Y_local[((i_2_1_109 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_109 + 12)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_110 = 0; i_2_1_110 < 4; ++i_2_1_110) {
      Y_local[((i_2_1_110 * 16) + 7)] = (Y_local[((i_2_1_110 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_110 + 8)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_111 = 0; i_2_1_111 < 4; ++i_2_1_111) {
      Y_local[((i_2_1_111 * 16) + 71)] = (Y_local[((i_2_1_111 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_111 + 12)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_112 = 0; i_2_1_112 < 4; ++i_2_1_112) {
      Y_local[((i_2_1_112 * 16) + 8)] = (Y_local[((i_2_1_112 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_112 + 8)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_113 = 0; i_2_1_113 < 4; ++i_2_1_113) {
      Y_local[((i_2_1_113 * 16) + 72)] = (Y_local[((i_2_1_113 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_113 + 12)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_114 = 0; i_2_1_114 < 4; ++i_2_1_114) {
      Y_local[((i_2_1_114 * 16) + 9)] = (Y_local[((i_2_1_114 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_114 + 8)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_115 = 0; i_2_1_115 < 4; ++i_2_1_115) {
      Y_local[((i_2_1_115 * 16) + 73)] = (Y_local[((i_2_1_115 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_115 + 12)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_116 = 0; i_2_1_116 < 4; ++i_2_1_116) {
      Y_local[((i_2_1_116 * 16) + 10)] = (Y_local[((i_2_1_116 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_116 + 8)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_117 = 0; i_2_1_117 < 4; ++i_2_1_117) {
      Y_local[((i_2_1_117 * 16) + 74)] = (Y_local[((i_2_1_117 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_117 + 12)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_118 = 0; i_2_1_118 < 4; ++i_2_1_118) {
      Y_local[((i_2_1_118 * 16) + 11)] = (Y_local[((i_2_1_118 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_118 + 8)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_119 = 0; i_2_1_119 < 4; ++i_2_1_119) {
      Y_local[((i_2_1_119 * 16) + 75)] = (Y_local[((i_2_1_119 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_119 + 12)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_120 = 0; i_2_1_120 < 4; ++i_2_1_120) {
      Y_local[((i_2_1_120 * 16) + 12)] = (Y_local[((i_2_1_120 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_120 + 8)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_121 = 0; i_2_1_121 < 4; ++i_2_1_121) {
      Y_local[((i_2_1_121 * 16) + 76)] = (Y_local[((i_2_1_121 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_121 + 12)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_122 = 0; i_2_1_122 < 4; ++i_2_1_122) {
      Y_local[((i_2_1_122 * 16) + 13)] = (Y_local[((i_2_1_122 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_122 + 8)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_123 = 0; i_2_1_123 < 4; ++i_2_1_123) {
      Y_local[((i_2_1_123 * 16) + 77)] = (Y_local[((i_2_1_123 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_123 + 12)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_124 = 0; i_2_1_124 < 4; ++i_2_1_124) {
      Y_local[((i_2_1_124 * 16) + 14)] = (Y_local[((i_2_1_124 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_124 + 8)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_125 = 0; i_2_1_125 < 4; ++i_2_1_125) {
      Y_local[((i_2_1_125 * 16) + 78)] = (Y_local[((i_2_1_125 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_125 + 12)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_126 = 0; i_2_1_126 < 4; ++i_2_1_126) {
      Y_local[((i_2_1_126 * 16) + 15)] = (Y_local[((i_2_1_126 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_126 + 8)] * B_shared_dyn_local[31]));
    }
    for (int i_2_1_127 = 0; i_2_1_127 < 4; ++i_2_1_127) {
      Y_local[((i_2_1_127 * 16) + 79)] = (Y_local[((i_2_1_127 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_127 + 12)] * B_shared_dyn_local[31]));
    }
    for (int ax1_0_10 = 0; ax1_0_10 < 2; ++ax1_0_10) {
      *(float4*)(A_shared_dyn_local + ((ax1_0_10 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_10 * 4)) + 2920));
    }
    for (int ax1_0_11 = 0; ax1_0_11 < 4; ++ax1_0_11) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_11 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_11 * 4)) + 400));
    }
    for (int i_2_1_128 = 0; i_2_1_128 < 4; ++i_2_1_128) {
      Y_local[(i_2_1_128 * 16)] = (Y_local[(i_2_1_128 * 16)] + (A_shared_dyn_local[i_2_1_128] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_129 = 0; i_2_1_129 < 4; ++i_2_1_129) {
      Y_local[((i_2_1_129 * 16) + 64)] = (Y_local[((i_2_1_129 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_129 + 4)] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_130 = 0; i_2_1_130 < 4; ++i_2_1_130) {
      Y_local[((i_2_1_130 * 16) + 1)] = (Y_local[((i_2_1_130 * 16) + 1)] + (A_shared_dyn_local[i_2_1_130] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_131 = 0; i_2_1_131 < 4; ++i_2_1_131) {
      Y_local[((i_2_1_131 * 16) + 65)] = (Y_local[((i_2_1_131 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_131 + 4)] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_132 = 0; i_2_1_132 < 4; ++i_2_1_132) {
      Y_local[((i_2_1_132 * 16) + 2)] = (Y_local[((i_2_1_132 * 16) + 2)] + (A_shared_dyn_local[i_2_1_132] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_133 = 0; i_2_1_133 < 4; ++i_2_1_133) {
      Y_local[((i_2_1_133 * 16) + 66)] = (Y_local[((i_2_1_133 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_133 + 4)] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_134 = 0; i_2_1_134 < 4; ++i_2_1_134) {
      Y_local[((i_2_1_134 * 16) + 3)] = (Y_local[((i_2_1_134 * 16) + 3)] + (A_shared_dyn_local[i_2_1_134] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_135 = 0; i_2_1_135 < 4; ++i_2_1_135) {
      Y_local[((i_2_1_135 * 16) + 67)] = (Y_local[((i_2_1_135 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_135 + 4)] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_136 = 0; i_2_1_136 < 4; ++i_2_1_136) {
      Y_local[((i_2_1_136 * 16) + 4)] = (Y_local[((i_2_1_136 * 16) + 4)] + (A_shared_dyn_local[i_2_1_136] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_137 = 0; i_2_1_137 < 4; ++i_2_1_137) {
      Y_local[((i_2_1_137 * 16) + 68)] = (Y_local[((i_2_1_137 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_137 + 4)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_138 = 0; i_2_1_138 < 4; ++i_2_1_138) {
      Y_local[((i_2_1_138 * 16) + 5)] = (Y_local[((i_2_1_138 * 16) + 5)] + (A_shared_dyn_local[i_2_1_138] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_139 = 0; i_2_1_139 < 4; ++i_2_1_139) {
      Y_local[((i_2_1_139 * 16) + 69)] = (Y_local[((i_2_1_139 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_139 + 4)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_140 = 0; i_2_1_140 < 4; ++i_2_1_140) {
      Y_local[((i_2_1_140 * 16) + 6)] = (Y_local[((i_2_1_140 * 16) + 6)] + (A_shared_dyn_local[i_2_1_140] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_141 = 0; i_2_1_141 < 4; ++i_2_1_141) {
      Y_local[((i_2_1_141 * 16) + 70)] = (Y_local[((i_2_1_141 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_141 + 4)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_142 = 0; i_2_1_142 < 4; ++i_2_1_142) {
      Y_local[((i_2_1_142 * 16) + 7)] = (Y_local[((i_2_1_142 * 16) + 7)] + (A_shared_dyn_local[i_2_1_142] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_143 = 0; i_2_1_143 < 4; ++i_2_1_143) {
      Y_local[((i_2_1_143 * 16) + 71)] = (Y_local[((i_2_1_143 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_143 + 4)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_144 = 0; i_2_1_144 < 4; ++i_2_1_144) {
      Y_local[((i_2_1_144 * 16) + 8)] = (Y_local[((i_2_1_144 * 16) + 8)] + (A_shared_dyn_local[i_2_1_144] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_145 = 0; i_2_1_145 < 4; ++i_2_1_145) {
      Y_local[((i_2_1_145 * 16) + 72)] = (Y_local[((i_2_1_145 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_145 + 4)] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_146 = 0; i_2_1_146 < 4; ++i_2_1_146) {
      Y_local[((i_2_1_146 * 16) + 9)] = (Y_local[((i_2_1_146 * 16) + 9)] + (A_shared_dyn_local[i_2_1_146] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_147 = 0; i_2_1_147 < 4; ++i_2_1_147) {
      Y_local[((i_2_1_147 * 16) + 73)] = (Y_local[((i_2_1_147 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_147 + 4)] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_148 = 0; i_2_1_148 < 4; ++i_2_1_148) {
      Y_local[((i_2_1_148 * 16) + 10)] = (Y_local[((i_2_1_148 * 16) + 10)] + (A_shared_dyn_local[i_2_1_148] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_149 = 0; i_2_1_149 < 4; ++i_2_1_149) {
      Y_local[((i_2_1_149 * 16) + 74)] = (Y_local[((i_2_1_149 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_149 + 4)] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_150 = 0; i_2_1_150 < 4; ++i_2_1_150) {
      Y_local[((i_2_1_150 * 16) + 11)] = (Y_local[((i_2_1_150 * 16) + 11)] + (A_shared_dyn_local[i_2_1_150] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_151 = 0; i_2_1_151 < 4; ++i_2_1_151) {
      Y_local[((i_2_1_151 * 16) + 75)] = (Y_local[((i_2_1_151 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_151 + 4)] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_152 = 0; i_2_1_152 < 4; ++i_2_1_152) {
      Y_local[((i_2_1_152 * 16) + 12)] = (Y_local[((i_2_1_152 * 16) + 12)] + (A_shared_dyn_local[i_2_1_152] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_153 = 0; i_2_1_153 < 4; ++i_2_1_153) {
      Y_local[((i_2_1_153 * 16) + 76)] = (Y_local[((i_2_1_153 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_153 + 4)] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_154 = 0; i_2_1_154 < 4; ++i_2_1_154) {
      Y_local[((i_2_1_154 * 16) + 13)] = (Y_local[((i_2_1_154 * 16) + 13)] + (A_shared_dyn_local[i_2_1_154] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_155 = 0; i_2_1_155 < 4; ++i_2_1_155) {
      Y_local[((i_2_1_155 * 16) + 77)] = (Y_local[((i_2_1_155 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_155 + 4)] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_156 = 0; i_2_1_156 < 4; ++i_2_1_156) {
      Y_local[((i_2_1_156 * 16) + 14)] = (Y_local[((i_2_1_156 * 16) + 14)] + (A_shared_dyn_local[i_2_1_156] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_157 = 0; i_2_1_157 < 4; ++i_2_1_157) {
      Y_local[((i_2_1_157 * 16) + 78)] = (Y_local[((i_2_1_157 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_157 + 4)] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_158 = 0; i_2_1_158 < 4; ++i_2_1_158) {
      Y_local[((i_2_1_158 * 16) + 15)] = (Y_local[((i_2_1_158 * 16) + 15)] + (A_shared_dyn_local[i_2_1_158] * B_shared_dyn_local[15]));
    }
    for (int i_2_1_159 = 0; i_2_1_159 < 4; ++i_2_1_159) {
      Y_local[((i_2_1_159 * 16) + 79)] = (Y_local[((i_2_1_159 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_159 + 4)] * B_shared_dyn_local[15]));
    }
    for (int ax1_0_12 = 0; ax1_0_12 < 2; ++ax1_0_12) {
      *(float4*)(A_shared_dyn_local + (ax1_0_12 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 + 1) & 3) * 1200) + ((((int)threadIdx.x) % 25) * 8)) + (ax1_0_12 * 4)) + 1920));
    }
    for (int ax1_0_13 = 0; ax1_0_13 < 4; ++ax1_0_13) {
      *(float4*)(B_shared_dyn_local + (ax1_0_13 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((k_0 + 1) & 3) * 480) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_13 * 4)));
    }
    for (int i_2_1_160 = 0; i_2_1_160 < 4; ++i_2_1_160) {
      Y_local[(i_2_1_160 * 16)] = (Y_local[(i_2_1_160 * 16)] + (A_shared_dyn_local[(i_2_1_160 + 8)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_161 = 0; i_2_1_161 < 4; ++i_2_1_161) {
      Y_local[((i_2_1_161 * 16) + 64)] = (Y_local[((i_2_1_161 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_161 + 12)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_162 = 0; i_2_1_162 < 4; ++i_2_1_162) {
      Y_local[((i_2_1_162 * 16) + 1)] = (Y_local[((i_2_1_162 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_162 + 8)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_163 = 0; i_2_1_163 < 4; ++i_2_1_163) {
      Y_local[((i_2_1_163 * 16) + 65)] = (Y_local[((i_2_1_163 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_163 + 12)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_164 = 0; i_2_1_164 < 4; ++i_2_1_164) {
      Y_local[((i_2_1_164 * 16) + 2)] = (Y_local[((i_2_1_164 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_164 + 8)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_165 = 0; i_2_1_165 < 4; ++i_2_1_165) {
      Y_local[((i_2_1_165 * 16) + 66)] = (Y_local[((i_2_1_165 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_165 + 12)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_166 = 0; i_2_1_166 < 4; ++i_2_1_166) {
      Y_local[((i_2_1_166 * 16) + 3)] = (Y_local[((i_2_1_166 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_166 + 8)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_167 = 0; i_2_1_167 < 4; ++i_2_1_167) {
      Y_local[((i_2_1_167 * 16) + 67)] = (Y_local[((i_2_1_167 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_167 + 12)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_168 = 0; i_2_1_168 < 4; ++i_2_1_168) {
      Y_local[((i_2_1_168 * 16) + 4)] = (Y_local[((i_2_1_168 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_168 + 8)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_169 = 0; i_2_1_169 < 4; ++i_2_1_169) {
      Y_local[((i_2_1_169 * 16) + 68)] = (Y_local[((i_2_1_169 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_169 + 12)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_170 = 0; i_2_1_170 < 4; ++i_2_1_170) {
      Y_local[((i_2_1_170 * 16) + 5)] = (Y_local[((i_2_1_170 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_170 + 8)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_171 = 0; i_2_1_171 < 4; ++i_2_1_171) {
      Y_local[((i_2_1_171 * 16) + 69)] = (Y_local[((i_2_1_171 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_171 + 12)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_172 = 0; i_2_1_172 < 4; ++i_2_1_172) {
      Y_local[((i_2_1_172 * 16) + 6)] = (Y_local[((i_2_1_172 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_172 + 8)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_173 = 0; i_2_1_173 < 4; ++i_2_1_173) {
      Y_local[((i_2_1_173 * 16) + 70)] = (Y_local[((i_2_1_173 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_173 + 12)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_174 = 0; i_2_1_174 < 4; ++i_2_1_174) {
      Y_local[((i_2_1_174 * 16) + 7)] = (Y_local[((i_2_1_174 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_174 + 8)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_175 = 0; i_2_1_175 < 4; ++i_2_1_175) {
      Y_local[((i_2_1_175 * 16) + 71)] = (Y_local[((i_2_1_175 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_175 + 12)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_176 = 0; i_2_1_176 < 4; ++i_2_1_176) {
      Y_local[((i_2_1_176 * 16) + 8)] = (Y_local[((i_2_1_176 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_176 + 8)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_177 = 0; i_2_1_177 < 4; ++i_2_1_177) {
      Y_local[((i_2_1_177 * 16) + 72)] = (Y_local[((i_2_1_177 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_177 + 12)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_178 = 0; i_2_1_178 < 4; ++i_2_1_178) {
      Y_local[((i_2_1_178 * 16) + 9)] = (Y_local[((i_2_1_178 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_178 + 8)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_179 = 0; i_2_1_179 < 4; ++i_2_1_179) {
      Y_local[((i_2_1_179 * 16) + 73)] = (Y_local[((i_2_1_179 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_179 + 12)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_180 = 0; i_2_1_180 < 4; ++i_2_1_180) {
      Y_local[((i_2_1_180 * 16) + 10)] = (Y_local[((i_2_1_180 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_180 + 8)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_181 = 0; i_2_1_181 < 4; ++i_2_1_181) {
      Y_local[((i_2_1_181 * 16) + 74)] = (Y_local[((i_2_1_181 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_181 + 12)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_182 = 0; i_2_1_182 < 4; ++i_2_1_182) {
      Y_local[((i_2_1_182 * 16) + 11)] = (Y_local[((i_2_1_182 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_182 + 8)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_183 = 0; i_2_1_183 < 4; ++i_2_1_183) {
      Y_local[((i_2_1_183 * 16) + 75)] = (Y_local[((i_2_1_183 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_183 + 12)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_184 = 0; i_2_1_184 < 4; ++i_2_1_184) {
      Y_local[((i_2_1_184 * 16) + 12)] = (Y_local[((i_2_1_184 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_184 + 8)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_185 = 0; i_2_1_185 < 4; ++i_2_1_185) {
      Y_local[((i_2_1_185 * 16) + 76)] = (Y_local[((i_2_1_185 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_185 + 12)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_186 = 0; i_2_1_186 < 4; ++i_2_1_186) {
      Y_local[((i_2_1_186 * 16) + 13)] = (Y_local[((i_2_1_186 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_186 + 8)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_187 = 0; i_2_1_187 < 4; ++i_2_1_187) {
      Y_local[((i_2_1_187 * 16) + 77)] = (Y_local[((i_2_1_187 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_187 + 12)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_188 = 0; i_2_1_188 < 4; ++i_2_1_188) {
      Y_local[((i_2_1_188 * 16) + 14)] = (Y_local[((i_2_1_188 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_188 + 8)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_189 = 0; i_2_1_189 < 4; ++i_2_1_189) {
      Y_local[((i_2_1_189 * 16) + 78)] = (Y_local[((i_2_1_189 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_189 + 12)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_190 = 0; i_2_1_190 < 4; ++i_2_1_190) {
      Y_local[((i_2_1_190 * 16) + 15)] = (Y_local[((i_2_1_190 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_190 + 8)] * B_shared_dyn_local[31]));
    }
    for (int i_2_1_191 = 0; i_2_1_191 < 4; ++i_2_1_191) {
      Y_local[((i_2_1_191 * 16) + 79)] = (Y_local[((i_2_1_191 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_191 + 12)] * B_shared_dyn_local[31]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax1_0_14 = 0; ax1_0_14 < 2; ++ax1_0_14) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_14 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_14 * 4)) + 3320));
  }
  for (int ax1_0_15 = 0; ax1_0_15 < 4; ++ax1_0_15) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_15 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_15 * 4)) + 560));
  }
  for (int i_2_1_192 = 0; i_2_1_192 < 4; ++i_2_1_192) {
    Y_local[(i_2_1_192 * 16)] = (Y_local[(i_2_1_192 * 16)] + (A_shared_dyn_local[i_2_1_192] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_193 = 0; i_2_1_193 < 4; ++i_2_1_193) {
    Y_local[((i_2_1_193 * 16) + 64)] = (Y_local[((i_2_1_193 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_193 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_194 = 0; i_2_1_194 < 4; ++i_2_1_194) {
    Y_local[((i_2_1_194 * 16) + 1)] = (Y_local[((i_2_1_194 * 16) + 1)] + (A_shared_dyn_local[i_2_1_194] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_195 = 0; i_2_1_195 < 4; ++i_2_1_195) {
    Y_local[((i_2_1_195 * 16) + 65)] = (Y_local[((i_2_1_195 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_195 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_196 = 0; i_2_1_196 < 4; ++i_2_1_196) {
    Y_local[((i_2_1_196 * 16) + 2)] = (Y_local[((i_2_1_196 * 16) + 2)] + (A_shared_dyn_local[i_2_1_196] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_197 = 0; i_2_1_197 < 4; ++i_2_1_197) {
    Y_local[((i_2_1_197 * 16) + 66)] = (Y_local[((i_2_1_197 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_197 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_198 = 0; i_2_1_198 < 4; ++i_2_1_198) {
    Y_local[((i_2_1_198 * 16) + 3)] = (Y_local[((i_2_1_198 * 16) + 3)] + (A_shared_dyn_local[i_2_1_198] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_199 = 0; i_2_1_199 < 4; ++i_2_1_199) {
    Y_local[((i_2_1_199 * 16) + 67)] = (Y_local[((i_2_1_199 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_199 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_200 = 0; i_2_1_200 < 4; ++i_2_1_200) {
    Y_local[((i_2_1_200 * 16) + 4)] = (Y_local[((i_2_1_200 * 16) + 4)] + (A_shared_dyn_local[i_2_1_200] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_201 = 0; i_2_1_201 < 4; ++i_2_1_201) {
    Y_local[((i_2_1_201 * 16) + 68)] = (Y_local[((i_2_1_201 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_201 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_202 = 0; i_2_1_202 < 4; ++i_2_1_202) {
    Y_local[((i_2_1_202 * 16) + 5)] = (Y_local[((i_2_1_202 * 16) + 5)] + (A_shared_dyn_local[i_2_1_202] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_203 = 0; i_2_1_203 < 4; ++i_2_1_203) {
    Y_local[((i_2_1_203 * 16) + 69)] = (Y_local[((i_2_1_203 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_203 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_204 = 0; i_2_1_204 < 4; ++i_2_1_204) {
    Y_local[((i_2_1_204 * 16) + 6)] = (Y_local[((i_2_1_204 * 16) + 6)] + (A_shared_dyn_local[i_2_1_204] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_205 = 0; i_2_1_205 < 4; ++i_2_1_205) {
    Y_local[((i_2_1_205 * 16) + 70)] = (Y_local[((i_2_1_205 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_205 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_206 = 0; i_2_1_206 < 4; ++i_2_1_206) {
    Y_local[((i_2_1_206 * 16) + 7)] = (Y_local[((i_2_1_206 * 16) + 7)] + (A_shared_dyn_local[i_2_1_206] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_207 = 0; i_2_1_207 < 4; ++i_2_1_207) {
    Y_local[((i_2_1_207 * 16) + 71)] = (Y_local[((i_2_1_207 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_207 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_208 = 0; i_2_1_208 < 4; ++i_2_1_208) {
    Y_local[((i_2_1_208 * 16) + 8)] = (Y_local[((i_2_1_208 * 16) + 8)] + (A_shared_dyn_local[i_2_1_208] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_209 = 0; i_2_1_209 < 4; ++i_2_1_209) {
    Y_local[((i_2_1_209 * 16) + 72)] = (Y_local[((i_2_1_209 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_209 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_210 = 0; i_2_1_210 < 4; ++i_2_1_210) {
    Y_local[((i_2_1_210 * 16) + 9)] = (Y_local[((i_2_1_210 * 16) + 9)] + (A_shared_dyn_local[i_2_1_210] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_211 = 0; i_2_1_211 < 4; ++i_2_1_211) {
    Y_local[((i_2_1_211 * 16) + 73)] = (Y_local[((i_2_1_211 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_211 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_212 = 0; i_2_1_212 < 4; ++i_2_1_212) {
    Y_local[((i_2_1_212 * 16) + 10)] = (Y_local[((i_2_1_212 * 16) + 10)] + (A_shared_dyn_local[i_2_1_212] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_213 = 0; i_2_1_213 < 4; ++i_2_1_213) {
    Y_local[((i_2_1_213 * 16) + 74)] = (Y_local[((i_2_1_213 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_213 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_214 = 0; i_2_1_214 < 4; ++i_2_1_214) {
    Y_local[((i_2_1_214 * 16) + 11)] = (Y_local[((i_2_1_214 * 16) + 11)] + (A_shared_dyn_local[i_2_1_214] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_215 = 0; i_2_1_215 < 4; ++i_2_1_215) {
    Y_local[((i_2_1_215 * 16) + 75)] = (Y_local[((i_2_1_215 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_215 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_216 = 0; i_2_1_216 < 4; ++i_2_1_216) {
    Y_local[((i_2_1_216 * 16) + 12)] = (Y_local[((i_2_1_216 * 16) + 12)] + (A_shared_dyn_local[i_2_1_216] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_217 = 0; i_2_1_217 < 4; ++i_2_1_217) {
    Y_local[((i_2_1_217 * 16) + 76)] = (Y_local[((i_2_1_217 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_217 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_218 = 0; i_2_1_218 < 4; ++i_2_1_218) {
    Y_local[((i_2_1_218 * 16) + 13)] = (Y_local[((i_2_1_218 * 16) + 13)] + (A_shared_dyn_local[i_2_1_218] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_219 = 0; i_2_1_219 < 4; ++i_2_1_219) {
    Y_local[((i_2_1_219 * 16) + 77)] = (Y_local[((i_2_1_219 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_219 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_220 = 0; i_2_1_220 < 4; ++i_2_1_220) {
    Y_local[((i_2_1_220 * 16) + 14)] = (Y_local[((i_2_1_220 * 16) + 14)] + (A_shared_dyn_local[i_2_1_220] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_221 = 0; i_2_1_221 < 4; ++i_2_1_221) {
    Y_local[((i_2_1_221 * 16) + 78)] = (Y_local[((i_2_1_221 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_221 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_222 = 0; i_2_1_222 < 4; ++i_2_1_222) {
    Y_local[((i_2_1_222 * 16) + 15)] = (Y_local[((i_2_1_222 * 16) + 15)] + (A_shared_dyn_local[i_2_1_222] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_223 = 0; i_2_1_223 < 4; ++i_2_1_223) {
    Y_local[((i_2_1_223 * 16) + 79)] = (Y_local[((i_2_1_223 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_223 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_16 = 0; ax1_0_16 < 2; ++ax1_0_16) {
    *(float4*)(A_shared_dyn_local + (ax1_0_16 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_16 * 4)) + 3520));
  }
  for (int ax1_0_17 = 0; ax1_0_17 < 4; ++ax1_0_17) {
    *(float4*)(B_shared_dyn_local + (ax1_0_17 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_17 * 4)) + 640));
  }
  for (int i_2_1_224 = 0; i_2_1_224 < 4; ++i_2_1_224) {
    Y_local[(i_2_1_224 * 16)] = (Y_local[(i_2_1_224 * 16)] + (A_shared_dyn_local[(i_2_1_224 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_225 = 0; i_2_1_225 < 4; ++i_2_1_225) {
    Y_local[((i_2_1_225 * 16) + 64)] = (Y_local[((i_2_1_225 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_225 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_226 = 0; i_2_1_226 < 4; ++i_2_1_226) {
    Y_local[((i_2_1_226 * 16) + 1)] = (Y_local[((i_2_1_226 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_226 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_227 = 0; i_2_1_227 < 4; ++i_2_1_227) {
    Y_local[((i_2_1_227 * 16) + 65)] = (Y_local[((i_2_1_227 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_227 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_228 = 0; i_2_1_228 < 4; ++i_2_1_228) {
    Y_local[((i_2_1_228 * 16) + 2)] = (Y_local[((i_2_1_228 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_228 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_229 = 0; i_2_1_229 < 4; ++i_2_1_229) {
    Y_local[((i_2_1_229 * 16) + 66)] = (Y_local[((i_2_1_229 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_229 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_230 = 0; i_2_1_230 < 4; ++i_2_1_230) {
    Y_local[((i_2_1_230 * 16) + 3)] = (Y_local[((i_2_1_230 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_230 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_231 = 0; i_2_1_231 < 4; ++i_2_1_231) {
    Y_local[((i_2_1_231 * 16) + 67)] = (Y_local[((i_2_1_231 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_231 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_232 = 0; i_2_1_232 < 4; ++i_2_1_232) {
    Y_local[((i_2_1_232 * 16) + 4)] = (Y_local[((i_2_1_232 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_232 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_233 = 0; i_2_1_233 < 4; ++i_2_1_233) {
    Y_local[((i_2_1_233 * 16) + 68)] = (Y_local[((i_2_1_233 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_233 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_234 = 0; i_2_1_234 < 4; ++i_2_1_234) {
    Y_local[((i_2_1_234 * 16) + 5)] = (Y_local[((i_2_1_234 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_234 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_235 = 0; i_2_1_235 < 4; ++i_2_1_235) {
    Y_local[((i_2_1_235 * 16) + 69)] = (Y_local[((i_2_1_235 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_235 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_236 = 0; i_2_1_236 < 4; ++i_2_1_236) {
    Y_local[((i_2_1_236 * 16) + 6)] = (Y_local[((i_2_1_236 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_236 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_237 = 0; i_2_1_237 < 4; ++i_2_1_237) {
    Y_local[((i_2_1_237 * 16) + 70)] = (Y_local[((i_2_1_237 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_237 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_238 = 0; i_2_1_238 < 4; ++i_2_1_238) {
    Y_local[((i_2_1_238 * 16) + 7)] = (Y_local[((i_2_1_238 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_238 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_239 = 0; i_2_1_239 < 4; ++i_2_1_239) {
    Y_local[((i_2_1_239 * 16) + 71)] = (Y_local[((i_2_1_239 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_239 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_240 = 0; i_2_1_240 < 4; ++i_2_1_240) {
    Y_local[((i_2_1_240 * 16) + 8)] = (Y_local[((i_2_1_240 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_240 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_241 = 0; i_2_1_241 < 4; ++i_2_1_241) {
    Y_local[((i_2_1_241 * 16) + 72)] = (Y_local[((i_2_1_241 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_241 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_242 = 0; i_2_1_242 < 4; ++i_2_1_242) {
    Y_local[((i_2_1_242 * 16) + 9)] = (Y_local[((i_2_1_242 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_242 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_243 = 0; i_2_1_243 < 4; ++i_2_1_243) {
    Y_local[((i_2_1_243 * 16) + 73)] = (Y_local[((i_2_1_243 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_243 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_244 = 0; i_2_1_244 < 4; ++i_2_1_244) {
    Y_local[((i_2_1_244 * 16) + 10)] = (Y_local[((i_2_1_244 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_244 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_245 = 0; i_2_1_245 < 4; ++i_2_1_245) {
    Y_local[((i_2_1_245 * 16) + 74)] = (Y_local[((i_2_1_245 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_245 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_246 = 0; i_2_1_246 < 4; ++i_2_1_246) {
    Y_local[((i_2_1_246 * 16) + 11)] = (Y_local[((i_2_1_246 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_246 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_247 = 0; i_2_1_247 < 4; ++i_2_1_247) {
    Y_local[((i_2_1_247 * 16) + 75)] = (Y_local[((i_2_1_247 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_247 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_248 = 0; i_2_1_248 < 4; ++i_2_1_248) {
    Y_local[((i_2_1_248 * 16) + 12)] = (Y_local[((i_2_1_248 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_248 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_249 = 0; i_2_1_249 < 4; ++i_2_1_249) {
    Y_local[((i_2_1_249 * 16) + 76)] = (Y_local[((i_2_1_249 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_249 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_250 = 0; i_2_1_250 < 4; ++i_2_1_250) {
    Y_local[((i_2_1_250 * 16) + 13)] = (Y_local[((i_2_1_250 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_250 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_251 = 0; i_2_1_251 < 4; ++i_2_1_251) {
    Y_local[((i_2_1_251 * 16) + 77)] = (Y_local[((i_2_1_251 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_251 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_252 = 0; i_2_1_252 < 4; ++i_2_1_252) {
    Y_local[((i_2_1_252 * 16) + 14)] = (Y_local[((i_2_1_252 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_252 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_253 = 0; i_2_1_253 < 4; ++i_2_1_253) {
    Y_local[((i_2_1_253 * 16) + 78)] = (Y_local[((i_2_1_253 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_253 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_254 = 0; i_2_1_254 < 4; ++i_2_1_254) {
    Y_local[((i_2_1_254 * 16) + 15)] = (Y_local[((i_2_1_254 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_254 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_255 = 0; i_2_1_255 < 4; ++i_2_1_255) {
    Y_local[((i_2_1_255 * 16) + 79)] = (Y_local[((i_2_1_255 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_255 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_18 = 0; ax1_0_18 < 2; ++ax1_0_18) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_18 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_18 * 4)) + 3720));
  }
  for (int ax1_0_19 = 0; ax1_0_19 < 4; ++ax1_0_19) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_19 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_19 * 4)) + 720));
  }
  for (int i_2_1_256 = 0; i_2_1_256 < 4; ++i_2_1_256) {
    Y_local[(i_2_1_256 * 16)] = (Y_local[(i_2_1_256 * 16)] + (A_shared_dyn_local[i_2_1_256] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_257 = 0; i_2_1_257 < 4; ++i_2_1_257) {
    Y_local[((i_2_1_257 * 16) + 64)] = (Y_local[((i_2_1_257 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_257 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_258 = 0; i_2_1_258 < 4; ++i_2_1_258) {
    Y_local[((i_2_1_258 * 16) + 1)] = (Y_local[((i_2_1_258 * 16) + 1)] + (A_shared_dyn_local[i_2_1_258] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_259 = 0; i_2_1_259 < 4; ++i_2_1_259) {
    Y_local[((i_2_1_259 * 16) + 65)] = (Y_local[((i_2_1_259 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_259 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_260 = 0; i_2_1_260 < 4; ++i_2_1_260) {
    Y_local[((i_2_1_260 * 16) + 2)] = (Y_local[((i_2_1_260 * 16) + 2)] + (A_shared_dyn_local[i_2_1_260] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_261 = 0; i_2_1_261 < 4; ++i_2_1_261) {
    Y_local[((i_2_1_261 * 16) + 66)] = (Y_local[((i_2_1_261 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_261 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_262 = 0; i_2_1_262 < 4; ++i_2_1_262) {
    Y_local[((i_2_1_262 * 16) + 3)] = (Y_local[((i_2_1_262 * 16) + 3)] + (A_shared_dyn_local[i_2_1_262] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_263 = 0; i_2_1_263 < 4; ++i_2_1_263) {
    Y_local[((i_2_1_263 * 16) + 67)] = (Y_local[((i_2_1_263 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_263 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_264 = 0; i_2_1_264 < 4; ++i_2_1_264) {
    Y_local[((i_2_1_264 * 16) + 4)] = (Y_local[((i_2_1_264 * 16) + 4)] + (A_shared_dyn_local[i_2_1_264] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_265 = 0; i_2_1_265 < 4; ++i_2_1_265) {
    Y_local[((i_2_1_265 * 16) + 68)] = (Y_local[((i_2_1_265 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_265 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_266 = 0; i_2_1_266 < 4; ++i_2_1_266) {
    Y_local[((i_2_1_266 * 16) + 5)] = (Y_local[((i_2_1_266 * 16) + 5)] + (A_shared_dyn_local[i_2_1_266] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_267 = 0; i_2_1_267 < 4; ++i_2_1_267) {
    Y_local[((i_2_1_267 * 16) + 69)] = (Y_local[((i_2_1_267 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_267 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_268 = 0; i_2_1_268 < 4; ++i_2_1_268) {
    Y_local[((i_2_1_268 * 16) + 6)] = (Y_local[((i_2_1_268 * 16) + 6)] + (A_shared_dyn_local[i_2_1_268] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_269 = 0; i_2_1_269 < 4; ++i_2_1_269) {
    Y_local[((i_2_1_269 * 16) + 70)] = (Y_local[((i_2_1_269 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_269 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_270 = 0; i_2_1_270 < 4; ++i_2_1_270) {
    Y_local[((i_2_1_270 * 16) + 7)] = (Y_local[((i_2_1_270 * 16) + 7)] + (A_shared_dyn_local[i_2_1_270] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_271 = 0; i_2_1_271 < 4; ++i_2_1_271) {
    Y_local[((i_2_1_271 * 16) + 71)] = (Y_local[((i_2_1_271 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_271 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_272 = 0; i_2_1_272 < 4; ++i_2_1_272) {
    Y_local[((i_2_1_272 * 16) + 8)] = (Y_local[((i_2_1_272 * 16) + 8)] + (A_shared_dyn_local[i_2_1_272] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_273 = 0; i_2_1_273 < 4; ++i_2_1_273) {
    Y_local[((i_2_1_273 * 16) + 72)] = (Y_local[((i_2_1_273 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_273 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_274 = 0; i_2_1_274 < 4; ++i_2_1_274) {
    Y_local[((i_2_1_274 * 16) + 9)] = (Y_local[((i_2_1_274 * 16) + 9)] + (A_shared_dyn_local[i_2_1_274] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_275 = 0; i_2_1_275 < 4; ++i_2_1_275) {
    Y_local[((i_2_1_275 * 16) + 73)] = (Y_local[((i_2_1_275 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_275 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_276 = 0; i_2_1_276 < 4; ++i_2_1_276) {
    Y_local[((i_2_1_276 * 16) + 10)] = (Y_local[((i_2_1_276 * 16) + 10)] + (A_shared_dyn_local[i_2_1_276] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_277 = 0; i_2_1_277 < 4; ++i_2_1_277) {
    Y_local[((i_2_1_277 * 16) + 74)] = (Y_local[((i_2_1_277 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_277 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_278 = 0; i_2_1_278 < 4; ++i_2_1_278) {
    Y_local[((i_2_1_278 * 16) + 11)] = (Y_local[((i_2_1_278 * 16) + 11)] + (A_shared_dyn_local[i_2_1_278] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_279 = 0; i_2_1_279 < 4; ++i_2_1_279) {
    Y_local[((i_2_1_279 * 16) + 75)] = (Y_local[((i_2_1_279 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_279 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_280 = 0; i_2_1_280 < 4; ++i_2_1_280) {
    Y_local[((i_2_1_280 * 16) + 12)] = (Y_local[((i_2_1_280 * 16) + 12)] + (A_shared_dyn_local[i_2_1_280] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_281 = 0; i_2_1_281 < 4; ++i_2_1_281) {
    Y_local[((i_2_1_281 * 16) + 76)] = (Y_local[((i_2_1_281 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_281 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_282 = 0; i_2_1_282 < 4; ++i_2_1_282) {
    Y_local[((i_2_1_282 * 16) + 13)] = (Y_local[((i_2_1_282 * 16) + 13)] + (A_shared_dyn_local[i_2_1_282] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_283 = 0; i_2_1_283 < 4; ++i_2_1_283) {
    Y_local[((i_2_1_283 * 16) + 77)] = (Y_local[((i_2_1_283 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_283 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_284 = 0; i_2_1_284 < 4; ++i_2_1_284) {
    Y_local[((i_2_1_284 * 16) + 14)] = (Y_local[((i_2_1_284 * 16) + 14)] + (A_shared_dyn_local[i_2_1_284] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_285 = 0; i_2_1_285 < 4; ++i_2_1_285) {
    Y_local[((i_2_1_285 * 16) + 78)] = (Y_local[((i_2_1_285 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_285 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_286 = 0; i_2_1_286 < 4; ++i_2_1_286) {
    Y_local[((i_2_1_286 * 16) + 15)] = (Y_local[((i_2_1_286 * 16) + 15)] + (A_shared_dyn_local[i_2_1_286] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_287 = 0; i_2_1_287 < 4; ++i_2_1_287) {
    Y_local[((i_2_1_287 * 16) + 79)] = (Y_local[((i_2_1_287 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_287 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_20 = 0; ax1_0_20 < 2; ++ax1_0_20) {
    *(float4*)(A_shared_dyn_local + (ax1_0_20 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_20 * 4)) + 3920));
  }
  for (int ax1_0_21 = 0; ax1_0_21 < 4; ++ax1_0_21) {
    *(float4*)(B_shared_dyn_local + (ax1_0_21 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_21 * 4)) + 800));
  }
  for (int i_2_1_288 = 0; i_2_1_288 < 4; ++i_2_1_288) {
    Y_local[(i_2_1_288 * 16)] = (Y_local[(i_2_1_288 * 16)] + (A_shared_dyn_local[(i_2_1_288 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_289 = 0; i_2_1_289 < 4; ++i_2_1_289) {
    Y_local[((i_2_1_289 * 16) + 64)] = (Y_local[((i_2_1_289 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_289 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_290 = 0; i_2_1_290 < 4; ++i_2_1_290) {
    Y_local[((i_2_1_290 * 16) + 1)] = (Y_local[((i_2_1_290 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_290 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_291 = 0; i_2_1_291 < 4; ++i_2_1_291) {
    Y_local[((i_2_1_291 * 16) + 65)] = (Y_local[((i_2_1_291 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_291 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_292 = 0; i_2_1_292 < 4; ++i_2_1_292) {
    Y_local[((i_2_1_292 * 16) + 2)] = (Y_local[((i_2_1_292 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_292 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_293 = 0; i_2_1_293 < 4; ++i_2_1_293) {
    Y_local[((i_2_1_293 * 16) + 66)] = (Y_local[((i_2_1_293 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_293 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_294 = 0; i_2_1_294 < 4; ++i_2_1_294) {
    Y_local[((i_2_1_294 * 16) + 3)] = (Y_local[((i_2_1_294 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_294 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_295 = 0; i_2_1_295 < 4; ++i_2_1_295) {
    Y_local[((i_2_1_295 * 16) + 67)] = (Y_local[((i_2_1_295 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_295 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_296 = 0; i_2_1_296 < 4; ++i_2_1_296) {
    Y_local[((i_2_1_296 * 16) + 4)] = (Y_local[((i_2_1_296 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_296 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_297 = 0; i_2_1_297 < 4; ++i_2_1_297) {
    Y_local[((i_2_1_297 * 16) + 68)] = (Y_local[((i_2_1_297 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_297 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_298 = 0; i_2_1_298 < 4; ++i_2_1_298) {
    Y_local[((i_2_1_298 * 16) + 5)] = (Y_local[((i_2_1_298 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_298 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_299 = 0; i_2_1_299 < 4; ++i_2_1_299) {
    Y_local[((i_2_1_299 * 16) + 69)] = (Y_local[((i_2_1_299 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_299 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_300 = 0; i_2_1_300 < 4; ++i_2_1_300) {
    Y_local[((i_2_1_300 * 16) + 6)] = (Y_local[((i_2_1_300 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_300 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_301 = 0; i_2_1_301 < 4; ++i_2_1_301) {
    Y_local[((i_2_1_301 * 16) + 70)] = (Y_local[((i_2_1_301 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_301 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_302 = 0; i_2_1_302 < 4; ++i_2_1_302) {
    Y_local[((i_2_1_302 * 16) + 7)] = (Y_local[((i_2_1_302 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_302 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_303 = 0; i_2_1_303 < 4; ++i_2_1_303) {
    Y_local[((i_2_1_303 * 16) + 71)] = (Y_local[((i_2_1_303 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_303 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_304 = 0; i_2_1_304 < 4; ++i_2_1_304) {
    Y_local[((i_2_1_304 * 16) + 8)] = (Y_local[((i_2_1_304 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_304 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_305 = 0; i_2_1_305 < 4; ++i_2_1_305) {
    Y_local[((i_2_1_305 * 16) + 72)] = (Y_local[((i_2_1_305 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_305 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_306 = 0; i_2_1_306 < 4; ++i_2_1_306) {
    Y_local[((i_2_1_306 * 16) + 9)] = (Y_local[((i_2_1_306 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_306 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_307 = 0; i_2_1_307 < 4; ++i_2_1_307) {
    Y_local[((i_2_1_307 * 16) + 73)] = (Y_local[((i_2_1_307 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_307 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_308 = 0; i_2_1_308 < 4; ++i_2_1_308) {
    Y_local[((i_2_1_308 * 16) + 10)] = (Y_local[((i_2_1_308 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_308 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_309 = 0; i_2_1_309 < 4; ++i_2_1_309) {
    Y_local[((i_2_1_309 * 16) + 74)] = (Y_local[((i_2_1_309 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_309 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_310 = 0; i_2_1_310 < 4; ++i_2_1_310) {
    Y_local[((i_2_1_310 * 16) + 11)] = (Y_local[((i_2_1_310 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_310 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_311 = 0; i_2_1_311 < 4; ++i_2_1_311) {
    Y_local[((i_2_1_311 * 16) + 75)] = (Y_local[((i_2_1_311 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_311 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_312 = 0; i_2_1_312 < 4; ++i_2_1_312) {
    Y_local[((i_2_1_312 * 16) + 12)] = (Y_local[((i_2_1_312 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_312 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_313 = 0; i_2_1_313 < 4; ++i_2_1_313) {
    Y_local[((i_2_1_313 * 16) + 76)] = (Y_local[((i_2_1_313 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_313 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_314 = 0; i_2_1_314 < 4; ++i_2_1_314) {
    Y_local[((i_2_1_314 * 16) + 13)] = (Y_local[((i_2_1_314 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_314 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_315 = 0; i_2_1_315 < 4; ++i_2_1_315) {
    Y_local[((i_2_1_315 * 16) + 77)] = (Y_local[((i_2_1_315 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_315 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_316 = 0; i_2_1_316 < 4; ++i_2_1_316) {
    Y_local[((i_2_1_316 * 16) + 14)] = (Y_local[((i_2_1_316 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_316 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_317 = 0; i_2_1_317 < 4; ++i_2_1_317) {
    Y_local[((i_2_1_317 * 16) + 78)] = (Y_local[((i_2_1_317 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_317 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_318 = 0; i_2_1_318 < 4; ++i_2_1_318) {
    Y_local[((i_2_1_318 * 16) + 15)] = (Y_local[((i_2_1_318 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_318 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_319 = 0; i_2_1_319 < 4; ++i_2_1_319) {
    Y_local[((i_2_1_319 * 16) + 79)] = (Y_local[((i_2_1_319 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_319 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_22 = 0; ax1_0_22 < 2; ++ax1_0_22) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_22 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_22 * 4)) + 4120));
  }
  for (int ax1_0_23 = 0; ax1_0_23 < 4; ++ax1_0_23) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_23 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_23 * 4)) + 880));
  }
  for (int i_2_1_320 = 0; i_2_1_320 < 4; ++i_2_1_320) {
    Y_local[(i_2_1_320 * 16)] = (Y_local[(i_2_1_320 * 16)] + (A_shared_dyn_local[i_2_1_320] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_321 = 0; i_2_1_321 < 4; ++i_2_1_321) {
    Y_local[((i_2_1_321 * 16) + 64)] = (Y_local[((i_2_1_321 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_321 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_322 = 0; i_2_1_322 < 4; ++i_2_1_322) {
    Y_local[((i_2_1_322 * 16) + 1)] = (Y_local[((i_2_1_322 * 16) + 1)] + (A_shared_dyn_local[i_2_1_322] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_323 = 0; i_2_1_323 < 4; ++i_2_1_323) {
    Y_local[((i_2_1_323 * 16) + 65)] = (Y_local[((i_2_1_323 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_323 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_324 = 0; i_2_1_324 < 4; ++i_2_1_324) {
    Y_local[((i_2_1_324 * 16) + 2)] = (Y_local[((i_2_1_324 * 16) + 2)] + (A_shared_dyn_local[i_2_1_324] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_325 = 0; i_2_1_325 < 4; ++i_2_1_325) {
    Y_local[((i_2_1_325 * 16) + 66)] = (Y_local[((i_2_1_325 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_325 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_326 = 0; i_2_1_326 < 4; ++i_2_1_326) {
    Y_local[((i_2_1_326 * 16) + 3)] = (Y_local[((i_2_1_326 * 16) + 3)] + (A_shared_dyn_local[i_2_1_326] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_327 = 0; i_2_1_327 < 4; ++i_2_1_327) {
    Y_local[((i_2_1_327 * 16) + 67)] = (Y_local[((i_2_1_327 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_327 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_328 = 0; i_2_1_328 < 4; ++i_2_1_328) {
    Y_local[((i_2_1_328 * 16) + 4)] = (Y_local[((i_2_1_328 * 16) + 4)] + (A_shared_dyn_local[i_2_1_328] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_329 = 0; i_2_1_329 < 4; ++i_2_1_329) {
    Y_local[((i_2_1_329 * 16) + 68)] = (Y_local[((i_2_1_329 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_329 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_330 = 0; i_2_1_330 < 4; ++i_2_1_330) {
    Y_local[((i_2_1_330 * 16) + 5)] = (Y_local[((i_2_1_330 * 16) + 5)] + (A_shared_dyn_local[i_2_1_330] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_331 = 0; i_2_1_331 < 4; ++i_2_1_331) {
    Y_local[((i_2_1_331 * 16) + 69)] = (Y_local[((i_2_1_331 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_331 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_332 = 0; i_2_1_332 < 4; ++i_2_1_332) {
    Y_local[((i_2_1_332 * 16) + 6)] = (Y_local[((i_2_1_332 * 16) + 6)] + (A_shared_dyn_local[i_2_1_332] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_333 = 0; i_2_1_333 < 4; ++i_2_1_333) {
    Y_local[((i_2_1_333 * 16) + 70)] = (Y_local[((i_2_1_333 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_333 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_334 = 0; i_2_1_334 < 4; ++i_2_1_334) {
    Y_local[((i_2_1_334 * 16) + 7)] = (Y_local[((i_2_1_334 * 16) + 7)] + (A_shared_dyn_local[i_2_1_334] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_335 = 0; i_2_1_335 < 4; ++i_2_1_335) {
    Y_local[((i_2_1_335 * 16) + 71)] = (Y_local[((i_2_1_335 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_335 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_336 = 0; i_2_1_336 < 4; ++i_2_1_336) {
    Y_local[((i_2_1_336 * 16) + 8)] = (Y_local[((i_2_1_336 * 16) + 8)] + (A_shared_dyn_local[i_2_1_336] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_337 = 0; i_2_1_337 < 4; ++i_2_1_337) {
    Y_local[((i_2_1_337 * 16) + 72)] = (Y_local[((i_2_1_337 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_337 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_338 = 0; i_2_1_338 < 4; ++i_2_1_338) {
    Y_local[((i_2_1_338 * 16) + 9)] = (Y_local[((i_2_1_338 * 16) + 9)] + (A_shared_dyn_local[i_2_1_338] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_339 = 0; i_2_1_339 < 4; ++i_2_1_339) {
    Y_local[((i_2_1_339 * 16) + 73)] = (Y_local[((i_2_1_339 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_339 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_340 = 0; i_2_1_340 < 4; ++i_2_1_340) {
    Y_local[((i_2_1_340 * 16) + 10)] = (Y_local[((i_2_1_340 * 16) + 10)] + (A_shared_dyn_local[i_2_1_340] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_341 = 0; i_2_1_341 < 4; ++i_2_1_341) {
    Y_local[((i_2_1_341 * 16) + 74)] = (Y_local[((i_2_1_341 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_341 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_342 = 0; i_2_1_342 < 4; ++i_2_1_342) {
    Y_local[((i_2_1_342 * 16) + 11)] = (Y_local[((i_2_1_342 * 16) + 11)] + (A_shared_dyn_local[i_2_1_342] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_343 = 0; i_2_1_343 < 4; ++i_2_1_343) {
    Y_local[((i_2_1_343 * 16) + 75)] = (Y_local[((i_2_1_343 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_343 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_344 = 0; i_2_1_344 < 4; ++i_2_1_344) {
    Y_local[((i_2_1_344 * 16) + 12)] = (Y_local[((i_2_1_344 * 16) + 12)] + (A_shared_dyn_local[i_2_1_344] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_345 = 0; i_2_1_345 < 4; ++i_2_1_345) {
    Y_local[((i_2_1_345 * 16) + 76)] = (Y_local[((i_2_1_345 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_345 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_346 = 0; i_2_1_346 < 4; ++i_2_1_346) {
    Y_local[((i_2_1_346 * 16) + 13)] = (Y_local[((i_2_1_346 * 16) + 13)] + (A_shared_dyn_local[i_2_1_346] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_347 = 0; i_2_1_347 < 4; ++i_2_1_347) {
    Y_local[((i_2_1_347 * 16) + 77)] = (Y_local[((i_2_1_347 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_347 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_348 = 0; i_2_1_348 < 4; ++i_2_1_348) {
    Y_local[((i_2_1_348 * 16) + 14)] = (Y_local[((i_2_1_348 * 16) + 14)] + (A_shared_dyn_local[i_2_1_348] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_349 = 0; i_2_1_349 < 4; ++i_2_1_349) {
    Y_local[((i_2_1_349 * 16) + 78)] = (Y_local[((i_2_1_349 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_349 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_350 = 0; i_2_1_350 < 4; ++i_2_1_350) {
    Y_local[((i_2_1_350 * 16) + 15)] = (Y_local[((i_2_1_350 * 16) + 15)] + (A_shared_dyn_local[i_2_1_350] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_351 = 0; i_2_1_351 < 4; ++i_2_1_351) {
    Y_local[((i_2_1_351 * 16) + 79)] = (Y_local[((i_2_1_351 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_351 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_24 = 0; ax1_0_24 < 2; ++ax1_0_24) {
    *(float4*)(A_shared_dyn_local + (ax1_0_24 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_24 * 4)) + 4320));
  }
  for (int ax1_0_25 = 0; ax1_0_25 < 4; ++ax1_0_25) {
    *(float4*)(B_shared_dyn_local + (ax1_0_25 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_25 * 4)) + 960));
  }
  for (int i_2_1_352 = 0; i_2_1_352 < 4; ++i_2_1_352) {
    Y_local[(i_2_1_352 * 16)] = (Y_local[(i_2_1_352 * 16)] + (A_shared_dyn_local[(i_2_1_352 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_353 = 0; i_2_1_353 < 4; ++i_2_1_353) {
    Y_local[((i_2_1_353 * 16) + 64)] = (Y_local[((i_2_1_353 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_353 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_354 = 0; i_2_1_354 < 4; ++i_2_1_354) {
    Y_local[((i_2_1_354 * 16) + 1)] = (Y_local[((i_2_1_354 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_354 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_355 = 0; i_2_1_355 < 4; ++i_2_1_355) {
    Y_local[((i_2_1_355 * 16) + 65)] = (Y_local[((i_2_1_355 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_355 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_356 = 0; i_2_1_356 < 4; ++i_2_1_356) {
    Y_local[((i_2_1_356 * 16) + 2)] = (Y_local[((i_2_1_356 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_356 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_357 = 0; i_2_1_357 < 4; ++i_2_1_357) {
    Y_local[((i_2_1_357 * 16) + 66)] = (Y_local[((i_2_1_357 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_357 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_358 = 0; i_2_1_358 < 4; ++i_2_1_358) {
    Y_local[((i_2_1_358 * 16) + 3)] = (Y_local[((i_2_1_358 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_358 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_359 = 0; i_2_1_359 < 4; ++i_2_1_359) {
    Y_local[((i_2_1_359 * 16) + 67)] = (Y_local[((i_2_1_359 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_359 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_360 = 0; i_2_1_360 < 4; ++i_2_1_360) {
    Y_local[((i_2_1_360 * 16) + 4)] = (Y_local[((i_2_1_360 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_360 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_361 = 0; i_2_1_361 < 4; ++i_2_1_361) {
    Y_local[((i_2_1_361 * 16) + 68)] = (Y_local[((i_2_1_361 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_361 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_362 = 0; i_2_1_362 < 4; ++i_2_1_362) {
    Y_local[((i_2_1_362 * 16) + 5)] = (Y_local[((i_2_1_362 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_362 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_363 = 0; i_2_1_363 < 4; ++i_2_1_363) {
    Y_local[((i_2_1_363 * 16) + 69)] = (Y_local[((i_2_1_363 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_363 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_364 = 0; i_2_1_364 < 4; ++i_2_1_364) {
    Y_local[((i_2_1_364 * 16) + 6)] = (Y_local[((i_2_1_364 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_364 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_365 = 0; i_2_1_365 < 4; ++i_2_1_365) {
    Y_local[((i_2_1_365 * 16) + 70)] = (Y_local[((i_2_1_365 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_365 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_366 = 0; i_2_1_366 < 4; ++i_2_1_366) {
    Y_local[((i_2_1_366 * 16) + 7)] = (Y_local[((i_2_1_366 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_366 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_367 = 0; i_2_1_367 < 4; ++i_2_1_367) {
    Y_local[((i_2_1_367 * 16) + 71)] = (Y_local[((i_2_1_367 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_367 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_368 = 0; i_2_1_368 < 4; ++i_2_1_368) {
    Y_local[((i_2_1_368 * 16) + 8)] = (Y_local[((i_2_1_368 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_368 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_369 = 0; i_2_1_369 < 4; ++i_2_1_369) {
    Y_local[((i_2_1_369 * 16) + 72)] = (Y_local[((i_2_1_369 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_369 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_370 = 0; i_2_1_370 < 4; ++i_2_1_370) {
    Y_local[((i_2_1_370 * 16) + 9)] = (Y_local[((i_2_1_370 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_370 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_371 = 0; i_2_1_371 < 4; ++i_2_1_371) {
    Y_local[((i_2_1_371 * 16) + 73)] = (Y_local[((i_2_1_371 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_371 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_372 = 0; i_2_1_372 < 4; ++i_2_1_372) {
    Y_local[((i_2_1_372 * 16) + 10)] = (Y_local[((i_2_1_372 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_372 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_373 = 0; i_2_1_373 < 4; ++i_2_1_373) {
    Y_local[((i_2_1_373 * 16) + 74)] = (Y_local[((i_2_1_373 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_373 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_374 = 0; i_2_1_374 < 4; ++i_2_1_374) {
    Y_local[((i_2_1_374 * 16) + 11)] = (Y_local[((i_2_1_374 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_374 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_375 = 0; i_2_1_375 < 4; ++i_2_1_375) {
    Y_local[((i_2_1_375 * 16) + 75)] = (Y_local[((i_2_1_375 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_375 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_376 = 0; i_2_1_376 < 4; ++i_2_1_376) {
    Y_local[((i_2_1_376 * 16) + 12)] = (Y_local[((i_2_1_376 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_376 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_377 = 0; i_2_1_377 < 4; ++i_2_1_377) {
    Y_local[((i_2_1_377 * 16) + 76)] = (Y_local[((i_2_1_377 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_377 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_378 = 0; i_2_1_378 < 4; ++i_2_1_378) {
    Y_local[((i_2_1_378 * 16) + 13)] = (Y_local[((i_2_1_378 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_378 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_379 = 0; i_2_1_379 < 4; ++i_2_1_379) {
    Y_local[((i_2_1_379 * 16) + 77)] = (Y_local[((i_2_1_379 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_379 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_380 = 0; i_2_1_380 < 4; ++i_2_1_380) {
    Y_local[((i_2_1_380 * 16) + 14)] = (Y_local[((i_2_1_380 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_380 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_381 = 0; i_2_1_381 < 4; ++i_2_1_381) {
    Y_local[((i_2_1_381 * 16) + 78)] = (Y_local[((i_2_1_381 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_381 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_382 = 0; i_2_1_382 < 4; ++i_2_1_382) {
    Y_local[((i_2_1_382 * 16) + 15)] = (Y_local[((i_2_1_382 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_382 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_383 = 0; i_2_1_383 < 4; ++i_2_1_383) {
    Y_local[((i_2_1_383 * 16) + 79)] = (Y_local[((i_2_1_383 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_383 + 12)] * B_shared_dyn_local[31]));
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax1_0_26 = 0; ax1_0_26 < 2; ++ax1_0_26) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_26 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_26 * 4)) + 4520));
  }
  for (int ax1_0_27 = 0; ax1_0_27 < 4; ++ax1_0_27) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_27 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_27 * 4)) + 1040));
  }
  for (int i_2_1_384 = 0; i_2_1_384 < 4; ++i_2_1_384) {
    Y_local[(i_2_1_384 * 16)] = (Y_local[(i_2_1_384 * 16)] + (A_shared_dyn_local[i_2_1_384] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_385 = 0; i_2_1_385 < 4; ++i_2_1_385) {
    Y_local[((i_2_1_385 * 16) + 64)] = (Y_local[((i_2_1_385 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_385 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_386 = 0; i_2_1_386 < 4; ++i_2_1_386) {
    Y_local[((i_2_1_386 * 16) + 1)] = (Y_local[((i_2_1_386 * 16) + 1)] + (A_shared_dyn_local[i_2_1_386] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_387 = 0; i_2_1_387 < 4; ++i_2_1_387) {
    Y_local[((i_2_1_387 * 16) + 65)] = (Y_local[((i_2_1_387 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_387 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_388 = 0; i_2_1_388 < 4; ++i_2_1_388) {
    Y_local[((i_2_1_388 * 16) + 2)] = (Y_local[((i_2_1_388 * 16) + 2)] + (A_shared_dyn_local[i_2_1_388] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_389 = 0; i_2_1_389 < 4; ++i_2_1_389) {
    Y_local[((i_2_1_389 * 16) + 66)] = (Y_local[((i_2_1_389 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_389 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_390 = 0; i_2_1_390 < 4; ++i_2_1_390) {
    Y_local[((i_2_1_390 * 16) + 3)] = (Y_local[((i_2_1_390 * 16) + 3)] + (A_shared_dyn_local[i_2_1_390] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_391 = 0; i_2_1_391 < 4; ++i_2_1_391) {
    Y_local[((i_2_1_391 * 16) + 67)] = (Y_local[((i_2_1_391 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_391 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_392 = 0; i_2_1_392 < 4; ++i_2_1_392) {
    Y_local[((i_2_1_392 * 16) + 4)] = (Y_local[((i_2_1_392 * 16) + 4)] + (A_shared_dyn_local[i_2_1_392] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_393 = 0; i_2_1_393 < 4; ++i_2_1_393) {
    Y_local[((i_2_1_393 * 16) + 68)] = (Y_local[((i_2_1_393 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_393 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_394 = 0; i_2_1_394 < 4; ++i_2_1_394) {
    Y_local[((i_2_1_394 * 16) + 5)] = (Y_local[((i_2_1_394 * 16) + 5)] + (A_shared_dyn_local[i_2_1_394] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_395 = 0; i_2_1_395 < 4; ++i_2_1_395) {
    Y_local[((i_2_1_395 * 16) + 69)] = (Y_local[((i_2_1_395 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_395 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_396 = 0; i_2_1_396 < 4; ++i_2_1_396) {
    Y_local[((i_2_1_396 * 16) + 6)] = (Y_local[((i_2_1_396 * 16) + 6)] + (A_shared_dyn_local[i_2_1_396] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_397 = 0; i_2_1_397 < 4; ++i_2_1_397) {
    Y_local[((i_2_1_397 * 16) + 70)] = (Y_local[((i_2_1_397 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_397 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_398 = 0; i_2_1_398 < 4; ++i_2_1_398) {
    Y_local[((i_2_1_398 * 16) + 7)] = (Y_local[((i_2_1_398 * 16) + 7)] + (A_shared_dyn_local[i_2_1_398] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_399 = 0; i_2_1_399 < 4; ++i_2_1_399) {
    Y_local[((i_2_1_399 * 16) + 71)] = (Y_local[((i_2_1_399 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_399 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_400 = 0; i_2_1_400 < 4; ++i_2_1_400) {
    Y_local[((i_2_1_400 * 16) + 8)] = (Y_local[((i_2_1_400 * 16) + 8)] + (A_shared_dyn_local[i_2_1_400] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_401 = 0; i_2_1_401 < 4; ++i_2_1_401) {
    Y_local[((i_2_1_401 * 16) + 72)] = (Y_local[((i_2_1_401 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_401 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_402 = 0; i_2_1_402 < 4; ++i_2_1_402) {
    Y_local[((i_2_1_402 * 16) + 9)] = (Y_local[((i_2_1_402 * 16) + 9)] + (A_shared_dyn_local[i_2_1_402] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_403 = 0; i_2_1_403 < 4; ++i_2_1_403) {
    Y_local[((i_2_1_403 * 16) + 73)] = (Y_local[((i_2_1_403 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_403 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_404 = 0; i_2_1_404 < 4; ++i_2_1_404) {
    Y_local[((i_2_1_404 * 16) + 10)] = (Y_local[((i_2_1_404 * 16) + 10)] + (A_shared_dyn_local[i_2_1_404] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_405 = 0; i_2_1_405 < 4; ++i_2_1_405) {
    Y_local[((i_2_1_405 * 16) + 74)] = (Y_local[((i_2_1_405 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_405 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_406 = 0; i_2_1_406 < 4; ++i_2_1_406) {
    Y_local[((i_2_1_406 * 16) + 11)] = (Y_local[((i_2_1_406 * 16) + 11)] + (A_shared_dyn_local[i_2_1_406] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_407 = 0; i_2_1_407 < 4; ++i_2_1_407) {
    Y_local[((i_2_1_407 * 16) + 75)] = (Y_local[((i_2_1_407 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_407 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_408 = 0; i_2_1_408 < 4; ++i_2_1_408) {
    Y_local[((i_2_1_408 * 16) + 12)] = (Y_local[((i_2_1_408 * 16) + 12)] + (A_shared_dyn_local[i_2_1_408] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_409 = 0; i_2_1_409 < 4; ++i_2_1_409) {
    Y_local[((i_2_1_409 * 16) + 76)] = (Y_local[((i_2_1_409 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_409 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_410 = 0; i_2_1_410 < 4; ++i_2_1_410) {
    Y_local[((i_2_1_410 * 16) + 13)] = (Y_local[((i_2_1_410 * 16) + 13)] + (A_shared_dyn_local[i_2_1_410] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_411 = 0; i_2_1_411 < 4; ++i_2_1_411) {
    Y_local[((i_2_1_411 * 16) + 77)] = (Y_local[((i_2_1_411 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_411 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_412 = 0; i_2_1_412 < 4; ++i_2_1_412) {
    Y_local[((i_2_1_412 * 16) + 14)] = (Y_local[((i_2_1_412 * 16) + 14)] + (A_shared_dyn_local[i_2_1_412] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_413 = 0; i_2_1_413 < 4; ++i_2_1_413) {
    Y_local[((i_2_1_413 * 16) + 78)] = (Y_local[((i_2_1_413 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_413 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_414 = 0; i_2_1_414 < 4; ++i_2_1_414) {
    Y_local[((i_2_1_414 * 16) + 15)] = (Y_local[((i_2_1_414 * 16) + 15)] + (A_shared_dyn_local[i_2_1_414] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_415 = 0; i_2_1_415 < 4; ++i_2_1_415) {
    Y_local[((i_2_1_415 * 16) + 79)] = (Y_local[((i_2_1_415 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_415 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_28 = 0; ax1_0_28 < 2; ++ax1_0_28) {
    *(float4*)(A_shared_dyn_local + (ax1_0_28 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_28 * 4)) + 4720));
  }
  for (int ax1_0_29 = 0; ax1_0_29 < 4; ++ax1_0_29) {
    *(float4*)(B_shared_dyn_local + (ax1_0_29 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_29 * 4)) + 1120));
  }
  for (int i_2_1_416 = 0; i_2_1_416 < 4; ++i_2_1_416) {
    Y_local[(i_2_1_416 * 16)] = (Y_local[(i_2_1_416 * 16)] + (A_shared_dyn_local[(i_2_1_416 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_417 = 0; i_2_1_417 < 4; ++i_2_1_417) {
    Y_local[((i_2_1_417 * 16) + 64)] = (Y_local[((i_2_1_417 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_417 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_418 = 0; i_2_1_418 < 4; ++i_2_1_418) {
    Y_local[((i_2_1_418 * 16) + 1)] = (Y_local[((i_2_1_418 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_418 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_419 = 0; i_2_1_419 < 4; ++i_2_1_419) {
    Y_local[((i_2_1_419 * 16) + 65)] = (Y_local[((i_2_1_419 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_419 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_420 = 0; i_2_1_420 < 4; ++i_2_1_420) {
    Y_local[((i_2_1_420 * 16) + 2)] = (Y_local[((i_2_1_420 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_420 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_421 = 0; i_2_1_421 < 4; ++i_2_1_421) {
    Y_local[((i_2_1_421 * 16) + 66)] = (Y_local[((i_2_1_421 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_421 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_422 = 0; i_2_1_422 < 4; ++i_2_1_422) {
    Y_local[((i_2_1_422 * 16) + 3)] = (Y_local[((i_2_1_422 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_422 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_423 = 0; i_2_1_423 < 4; ++i_2_1_423) {
    Y_local[((i_2_1_423 * 16) + 67)] = (Y_local[((i_2_1_423 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_423 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_424 = 0; i_2_1_424 < 4; ++i_2_1_424) {
    Y_local[((i_2_1_424 * 16) + 4)] = (Y_local[((i_2_1_424 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_424 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_425 = 0; i_2_1_425 < 4; ++i_2_1_425) {
    Y_local[((i_2_1_425 * 16) + 68)] = (Y_local[((i_2_1_425 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_425 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_426 = 0; i_2_1_426 < 4; ++i_2_1_426) {
    Y_local[((i_2_1_426 * 16) + 5)] = (Y_local[((i_2_1_426 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_426 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_427 = 0; i_2_1_427 < 4; ++i_2_1_427) {
    Y_local[((i_2_1_427 * 16) + 69)] = (Y_local[((i_2_1_427 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_427 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_428 = 0; i_2_1_428 < 4; ++i_2_1_428) {
    Y_local[((i_2_1_428 * 16) + 6)] = (Y_local[((i_2_1_428 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_428 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_429 = 0; i_2_1_429 < 4; ++i_2_1_429) {
    Y_local[((i_2_1_429 * 16) + 70)] = (Y_local[((i_2_1_429 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_429 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_430 = 0; i_2_1_430 < 4; ++i_2_1_430) {
    Y_local[((i_2_1_430 * 16) + 7)] = (Y_local[((i_2_1_430 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_430 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_431 = 0; i_2_1_431 < 4; ++i_2_1_431) {
    Y_local[((i_2_1_431 * 16) + 71)] = (Y_local[((i_2_1_431 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_431 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_432 = 0; i_2_1_432 < 4; ++i_2_1_432) {
    Y_local[((i_2_1_432 * 16) + 8)] = (Y_local[((i_2_1_432 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_432 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_433 = 0; i_2_1_433 < 4; ++i_2_1_433) {
    Y_local[((i_2_1_433 * 16) + 72)] = (Y_local[((i_2_1_433 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_433 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_434 = 0; i_2_1_434 < 4; ++i_2_1_434) {
    Y_local[((i_2_1_434 * 16) + 9)] = (Y_local[((i_2_1_434 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_434 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_435 = 0; i_2_1_435 < 4; ++i_2_1_435) {
    Y_local[((i_2_1_435 * 16) + 73)] = (Y_local[((i_2_1_435 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_435 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_436 = 0; i_2_1_436 < 4; ++i_2_1_436) {
    Y_local[((i_2_1_436 * 16) + 10)] = (Y_local[((i_2_1_436 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_436 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_437 = 0; i_2_1_437 < 4; ++i_2_1_437) {
    Y_local[((i_2_1_437 * 16) + 74)] = (Y_local[((i_2_1_437 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_437 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_438 = 0; i_2_1_438 < 4; ++i_2_1_438) {
    Y_local[((i_2_1_438 * 16) + 11)] = (Y_local[((i_2_1_438 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_438 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_439 = 0; i_2_1_439 < 4; ++i_2_1_439) {
    Y_local[((i_2_1_439 * 16) + 75)] = (Y_local[((i_2_1_439 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_439 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_440 = 0; i_2_1_440 < 4; ++i_2_1_440) {
    Y_local[((i_2_1_440 * 16) + 12)] = (Y_local[((i_2_1_440 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_440 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_441 = 0; i_2_1_441 < 4; ++i_2_1_441) {
    Y_local[((i_2_1_441 * 16) + 76)] = (Y_local[((i_2_1_441 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_441 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_442 = 0; i_2_1_442 < 4; ++i_2_1_442) {
    Y_local[((i_2_1_442 * 16) + 13)] = (Y_local[((i_2_1_442 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_442 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_443 = 0; i_2_1_443 < 4; ++i_2_1_443) {
    Y_local[((i_2_1_443 * 16) + 77)] = (Y_local[((i_2_1_443 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_443 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_444 = 0; i_2_1_444 < 4; ++i_2_1_444) {
    Y_local[((i_2_1_444 * 16) + 14)] = (Y_local[((i_2_1_444 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_444 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_445 = 0; i_2_1_445 < 4; ++i_2_1_445) {
    Y_local[((i_2_1_445 * 16) + 78)] = (Y_local[((i_2_1_445 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_445 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_446 = 0; i_2_1_446 < 4; ++i_2_1_446) {
    Y_local[((i_2_1_446 * 16) + 15)] = (Y_local[((i_2_1_446 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_446 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_447 = 0; i_2_1_447 < 4; ++i_2_1_447) {
    Y_local[((i_2_1_447 * 16) + 79)] = (Y_local[((i_2_1_447 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_447 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_30 = 0; ax1_0_30 < 2; ++ax1_0_30) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_30 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_30 * 4)) + 4920));
  }
  for (int ax1_0_31 = 0; ax1_0_31 < 4; ++ax1_0_31) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_31 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_31 * 4)) + 1200));
  }
  for (int i_2_1_448 = 0; i_2_1_448 < 4; ++i_2_1_448) {
    Y_local[(i_2_1_448 * 16)] = (Y_local[(i_2_1_448 * 16)] + (A_shared_dyn_local[i_2_1_448] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_449 = 0; i_2_1_449 < 4; ++i_2_1_449) {
    Y_local[((i_2_1_449 * 16) + 64)] = (Y_local[((i_2_1_449 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_449 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_450 = 0; i_2_1_450 < 4; ++i_2_1_450) {
    Y_local[((i_2_1_450 * 16) + 1)] = (Y_local[((i_2_1_450 * 16) + 1)] + (A_shared_dyn_local[i_2_1_450] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_451 = 0; i_2_1_451 < 4; ++i_2_1_451) {
    Y_local[((i_2_1_451 * 16) + 65)] = (Y_local[((i_2_1_451 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_451 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_452 = 0; i_2_1_452 < 4; ++i_2_1_452) {
    Y_local[((i_2_1_452 * 16) + 2)] = (Y_local[((i_2_1_452 * 16) + 2)] + (A_shared_dyn_local[i_2_1_452] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_453 = 0; i_2_1_453 < 4; ++i_2_1_453) {
    Y_local[((i_2_1_453 * 16) + 66)] = (Y_local[((i_2_1_453 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_453 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_454 = 0; i_2_1_454 < 4; ++i_2_1_454) {
    Y_local[((i_2_1_454 * 16) + 3)] = (Y_local[((i_2_1_454 * 16) + 3)] + (A_shared_dyn_local[i_2_1_454] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_455 = 0; i_2_1_455 < 4; ++i_2_1_455) {
    Y_local[((i_2_1_455 * 16) + 67)] = (Y_local[((i_2_1_455 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_455 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_456 = 0; i_2_1_456 < 4; ++i_2_1_456) {
    Y_local[((i_2_1_456 * 16) + 4)] = (Y_local[((i_2_1_456 * 16) + 4)] + (A_shared_dyn_local[i_2_1_456] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_457 = 0; i_2_1_457 < 4; ++i_2_1_457) {
    Y_local[((i_2_1_457 * 16) + 68)] = (Y_local[((i_2_1_457 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_457 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_458 = 0; i_2_1_458 < 4; ++i_2_1_458) {
    Y_local[((i_2_1_458 * 16) + 5)] = (Y_local[((i_2_1_458 * 16) + 5)] + (A_shared_dyn_local[i_2_1_458] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_459 = 0; i_2_1_459 < 4; ++i_2_1_459) {
    Y_local[((i_2_1_459 * 16) + 69)] = (Y_local[((i_2_1_459 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_459 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_460 = 0; i_2_1_460 < 4; ++i_2_1_460) {
    Y_local[((i_2_1_460 * 16) + 6)] = (Y_local[((i_2_1_460 * 16) + 6)] + (A_shared_dyn_local[i_2_1_460] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_461 = 0; i_2_1_461 < 4; ++i_2_1_461) {
    Y_local[((i_2_1_461 * 16) + 70)] = (Y_local[((i_2_1_461 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_461 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_462 = 0; i_2_1_462 < 4; ++i_2_1_462) {
    Y_local[((i_2_1_462 * 16) + 7)] = (Y_local[((i_2_1_462 * 16) + 7)] + (A_shared_dyn_local[i_2_1_462] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_463 = 0; i_2_1_463 < 4; ++i_2_1_463) {
    Y_local[((i_2_1_463 * 16) + 71)] = (Y_local[((i_2_1_463 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_463 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_464 = 0; i_2_1_464 < 4; ++i_2_1_464) {
    Y_local[((i_2_1_464 * 16) + 8)] = (Y_local[((i_2_1_464 * 16) + 8)] + (A_shared_dyn_local[i_2_1_464] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_465 = 0; i_2_1_465 < 4; ++i_2_1_465) {
    Y_local[((i_2_1_465 * 16) + 72)] = (Y_local[((i_2_1_465 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_465 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_466 = 0; i_2_1_466 < 4; ++i_2_1_466) {
    Y_local[((i_2_1_466 * 16) + 9)] = (Y_local[((i_2_1_466 * 16) + 9)] + (A_shared_dyn_local[i_2_1_466] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_467 = 0; i_2_1_467 < 4; ++i_2_1_467) {
    Y_local[((i_2_1_467 * 16) + 73)] = (Y_local[((i_2_1_467 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_467 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_468 = 0; i_2_1_468 < 4; ++i_2_1_468) {
    Y_local[((i_2_1_468 * 16) + 10)] = (Y_local[((i_2_1_468 * 16) + 10)] + (A_shared_dyn_local[i_2_1_468] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_469 = 0; i_2_1_469 < 4; ++i_2_1_469) {
    Y_local[((i_2_1_469 * 16) + 74)] = (Y_local[((i_2_1_469 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_469 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_470 = 0; i_2_1_470 < 4; ++i_2_1_470) {
    Y_local[((i_2_1_470 * 16) + 11)] = (Y_local[((i_2_1_470 * 16) + 11)] + (A_shared_dyn_local[i_2_1_470] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_471 = 0; i_2_1_471 < 4; ++i_2_1_471) {
    Y_local[((i_2_1_471 * 16) + 75)] = (Y_local[((i_2_1_471 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_471 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_472 = 0; i_2_1_472 < 4; ++i_2_1_472) {
    Y_local[((i_2_1_472 * 16) + 12)] = (Y_local[((i_2_1_472 * 16) + 12)] + (A_shared_dyn_local[i_2_1_472] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_473 = 0; i_2_1_473 < 4; ++i_2_1_473) {
    Y_local[((i_2_1_473 * 16) + 76)] = (Y_local[((i_2_1_473 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_473 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_474 = 0; i_2_1_474 < 4; ++i_2_1_474) {
    Y_local[((i_2_1_474 * 16) + 13)] = (Y_local[((i_2_1_474 * 16) + 13)] + (A_shared_dyn_local[i_2_1_474] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_475 = 0; i_2_1_475 < 4; ++i_2_1_475) {
    Y_local[((i_2_1_475 * 16) + 77)] = (Y_local[((i_2_1_475 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_475 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_476 = 0; i_2_1_476 < 4; ++i_2_1_476) {
    Y_local[((i_2_1_476 * 16) + 14)] = (Y_local[((i_2_1_476 * 16) + 14)] + (A_shared_dyn_local[i_2_1_476] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_477 = 0; i_2_1_477 < 4; ++i_2_1_477) {
    Y_local[((i_2_1_477 * 16) + 78)] = (Y_local[((i_2_1_477 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_477 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_478 = 0; i_2_1_478 < 4; ++i_2_1_478) {
    Y_local[((i_2_1_478 * 16) + 15)] = (Y_local[((i_2_1_478 * 16) + 15)] + (A_shared_dyn_local[i_2_1_478] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_479 = 0; i_2_1_479 < 4; ++i_2_1_479) {
    Y_local[((i_2_1_479 * 16) + 79)] = (Y_local[((i_2_1_479 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_479 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_32 = 0; ax1_0_32 < 2; ++ax1_0_32) {
    *(float4*)(A_shared_dyn_local + (ax1_0_32 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_32 * 4)) + 5120));
  }
  for (int ax1_0_33 = 0; ax1_0_33 < 4; ++ax1_0_33) {
    *(float4*)(B_shared_dyn_local + (ax1_0_33 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_33 * 4)) + 1280));
  }
  for (int i_2_1_480 = 0; i_2_1_480 < 4; ++i_2_1_480) {
    Y_local[(i_2_1_480 * 16)] = (Y_local[(i_2_1_480 * 16)] + (A_shared_dyn_local[(i_2_1_480 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_481 = 0; i_2_1_481 < 4; ++i_2_1_481) {
    Y_local[((i_2_1_481 * 16) + 64)] = (Y_local[((i_2_1_481 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_481 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_482 = 0; i_2_1_482 < 4; ++i_2_1_482) {
    Y_local[((i_2_1_482 * 16) + 1)] = (Y_local[((i_2_1_482 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_482 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_483 = 0; i_2_1_483 < 4; ++i_2_1_483) {
    Y_local[((i_2_1_483 * 16) + 65)] = (Y_local[((i_2_1_483 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_483 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_484 = 0; i_2_1_484 < 4; ++i_2_1_484) {
    Y_local[((i_2_1_484 * 16) + 2)] = (Y_local[((i_2_1_484 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_484 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_485 = 0; i_2_1_485 < 4; ++i_2_1_485) {
    Y_local[((i_2_1_485 * 16) + 66)] = (Y_local[((i_2_1_485 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_485 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_486 = 0; i_2_1_486 < 4; ++i_2_1_486) {
    Y_local[((i_2_1_486 * 16) + 3)] = (Y_local[((i_2_1_486 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_486 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_487 = 0; i_2_1_487 < 4; ++i_2_1_487) {
    Y_local[((i_2_1_487 * 16) + 67)] = (Y_local[((i_2_1_487 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_487 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_488 = 0; i_2_1_488 < 4; ++i_2_1_488) {
    Y_local[((i_2_1_488 * 16) + 4)] = (Y_local[((i_2_1_488 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_488 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_489 = 0; i_2_1_489 < 4; ++i_2_1_489) {
    Y_local[((i_2_1_489 * 16) + 68)] = (Y_local[((i_2_1_489 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_489 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_490 = 0; i_2_1_490 < 4; ++i_2_1_490) {
    Y_local[((i_2_1_490 * 16) + 5)] = (Y_local[((i_2_1_490 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_490 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_491 = 0; i_2_1_491 < 4; ++i_2_1_491) {
    Y_local[((i_2_1_491 * 16) + 69)] = (Y_local[((i_2_1_491 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_491 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_492 = 0; i_2_1_492 < 4; ++i_2_1_492) {
    Y_local[((i_2_1_492 * 16) + 6)] = (Y_local[((i_2_1_492 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_492 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_493 = 0; i_2_1_493 < 4; ++i_2_1_493) {
    Y_local[((i_2_1_493 * 16) + 70)] = (Y_local[((i_2_1_493 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_493 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_494 = 0; i_2_1_494 < 4; ++i_2_1_494) {
    Y_local[((i_2_1_494 * 16) + 7)] = (Y_local[((i_2_1_494 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_494 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_495 = 0; i_2_1_495 < 4; ++i_2_1_495) {
    Y_local[((i_2_1_495 * 16) + 71)] = (Y_local[((i_2_1_495 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_495 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_496 = 0; i_2_1_496 < 4; ++i_2_1_496) {
    Y_local[((i_2_1_496 * 16) + 8)] = (Y_local[((i_2_1_496 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_496 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_497 = 0; i_2_1_497 < 4; ++i_2_1_497) {
    Y_local[((i_2_1_497 * 16) + 72)] = (Y_local[((i_2_1_497 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_497 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_498 = 0; i_2_1_498 < 4; ++i_2_1_498) {
    Y_local[((i_2_1_498 * 16) + 9)] = (Y_local[((i_2_1_498 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_498 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_499 = 0; i_2_1_499 < 4; ++i_2_1_499) {
    Y_local[((i_2_1_499 * 16) + 73)] = (Y_local[((i_2_1_499 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_499 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_500 = 0; i_2_1_500 < 4; ++i_2_1_500) {
    Y_local[((i_2_1_500 * 16) + 10)] = (Y_local[((i_2_1_500 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_500 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_501 = 0; i_2_1_501 < 4; ++i_2_1_501) {
    Y_local[((i_2_1_501 * 16) + 74)] = (Y_local[((i_2_1_501 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_501 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_502 = 0; i_2_1_502 < 4; ++i_2_1_502) {
    Y_local[((i_2_1_502 * 16) + 11)] = (Y_local[((i_2_1_502 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_502 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_503 = 0; i_2_1_503 < 4; ++i_2_1_503) {
    Y_local[((i_2_1_503 * 16) + 75)] = (Y_local[((i_2_1_503 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_503 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_504 = 0; i_2_1_504 < 4; ++i_2_1_504) {
    Y_local[((i_2_1_504 * 16) + 12)] = (Y_local[((i_2_1_504 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_504 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_505 = 0; i_2_1_505 < 4; ++i_2_1_505) {
    Y_local[((i_2_1_505 * 16) + 76)] = (Y_local[((i_2_1_505 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_505 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_506 = 0; i_2_1_506 < 4; ++i_2_1_506) {
    Y_local[((i_2_1_506 * 16) + 13)] = (Y_local[((i_2_1_506 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_506 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_507 = 0; i_2_1_507 < 4; ++i_2_1_507) {
    Y_local[((i_2_1_507 * 16) + 77)] = (Y_local[((i_2_1_507 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_507 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_508 = 0; i_2_1_508 < 4; ++i_2_1_508) {
    Y_local[((i_2_1_508 * 16) + 14)] = (Y_local[((i_2_1_508 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_508 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_509 = 0; i_2_1_509 < 4; ++i_2_1_509) {
    Y_local[((i_2_1_509 * 16) + 78)] = (Y_local[((i_2_1_509 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_509 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_510 = 0; i_2_1_510 < 4; ++i_2_1_510) {
    Y_local[((i_2_1_510 * 16) + 15)] = (Y_local[((i_2_1_510 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_510 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_511 = 0; i_2_1_511 < 4; ++i_2_1_511) {
    Y_local[((i_2_1_511 * 16) + 79)] = (Y_local[((i_2_1_511 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_511 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_34 = 0; ax1_0_34 < 2; ++ax1_0_34) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_34 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_34 * 4)) + 5320));
  }
  for (int ax1_0_35 = 0; ax1_0_35 < 4; ++ax1_0_35) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_35 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_35 * 4)) + 1360));
  }
  for (int i_2_1_512 = 0; i_2_1_512 < 4; ++i_2_1_512) {
    Y_local[(i_2_1_512 * 16)] = (Y_local[(i_2_1_512 * 16)] + (A_shared_dyn_local[i_2_1_512] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_513 = 0; i_2_1_513 < 4; ++i_2_1_513) {
    Y_local[((i_2_1_513 * 16) + 64)] = (Y_local[((i_2_1_513 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_513 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_514 = 0; i_2_1_514 < 4; ++i_2_1_514) {
    Y_local[((i_2_1_514 * 16) + 1)] = (Y_local[((i_2_1_514 * 16) + 1)] + (A_shared_dyn_local[i_2_1_514] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_515 = 0; i_2_1_515 < 4; ++i_2_1_515) {
    Y_local[((i_2_1_515 * 16) + 65)] = (Y_local[((i_2_1_515 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_515 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_516 = 0; i_2_1_516 < 4; ++i_2_1_516) {
    Y_local[((i_2_1_516 * 16) + 2)] = (Y_local[((i_2_1_516 * 16) + 2)] + (A_shared_dyn_local[i_2_1_516] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_517 = 0; i_2_1_517 < 4; ++i_2_1_517) {
    Y_local[((i_2_1_517 * 16) + 66)] = (Y_local[((i_2_1_517 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_517 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_518 = 0; i_2_1_518 < 4; ++i_2_1_518) {
    Y_local[((i_2_1_518 * 16) + 3)] = (Y_local[((i_2_1_518 * 16) + 3)] + (A_shared_dyn_local[i_2_1_518] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_519 = 0; i_2_1_519 < 4; ++i_2_1_519) {
    Y_local[((i_2_1_519 * 16) + 67)] = (Y_local[((i_2_1_519 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_519 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_520 = 0; i_2_1_520 < 4; ++i_2_1_520) {
    Y_local[((i_2_1_520 * 16) + 4)] = (Y_local[((i_2_1_520 * 16) + 4)] + (A_shared_dyn_local[i_2_1_520] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_521 = 0; i_2_1_521 < 4; ++i_2_1_521) {
    Y_local[((i_2_1_521 * 16) + 68)] = (Y_local[((i_2_1_521 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_521 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_522 = 0; i_2_1_522 < 4; ++i_2_1_522) {
    Y_local[((i_2_1_522 * 16) + 5)] = (Y_local[((i_2_1_522 * 16) + 5)] + (A_shared_dyn_local[i_2_1_522] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_523 = 0; i_2_1_523 < 4; ++i_2_1_523) {
    Y_local[((i_2_1_523 * 16) + 69)] = (Y_local[((i_2_1_523 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_523 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_524 = 0; i_2_1_524 < 4; ++i_2_1_524) {
    Y_local[((i_2_1_524 * 16) + 6)] = (Y_local[((i_2_1_524 * 16) + 6)] + (A_shared_dyn_local[i_2_1_524] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_525 = 0; i_2_1_525 < 4; ++i_2_1_525) {
    Y_local[((i_2_1_525 * 16) + 70)] = (Y_local[((i_2_1_525 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_525 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_526 = 0; i_2_1_526 < 4; ++i_2_1_526) {
    Y_local[((i_2_1_526 * 16) + 7)] = (Y_local[((i_2_1_526 * 16) + 7)] + (A_shared_dyn_local[i_2_1_526] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_527 = 0; i_2_1_527 < 4; ++i_2_1_527) {
    Y_local[((i_2_1_527 * 16) + 71)] = (Y_local[((i_2_1_527 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_527 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_528 = 0; i_2_1_528 < 4; ++i_2_1_528) {
    Y_local[((i_2_1_528 * 16) + 8)] = (Y_local[((i_2_1_528 * 16) + 8)] + (A_shared_dyn_local[i_2_1_528] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_529 = 0; i_2_1_529 < 4; ++i_2_1_529) {
    Y_local[((i_2_1_529 * 16) + 72)] = (Y_local[((i_2_1_529 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_529 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_530 = 0; i_2_1_530 < 4; ++i_2_1_530) {
    Y_local[((i_2_1_530 * 16) + 9)] = (Y_local[((i_2_1_530 * 16) + 9)] + (A_shared_dyn_local[i_2_1_530] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_531 = 0; i_2_1_531 < 4; ++i_2_1_531) {
    Y_local[((i_2_1_531 * 16) + 73)] = (Y_local[((i_2_1_531 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_531 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_532 = 0; i_2_1_532 < 4; ++i_2_1_532) {
    Y_local[((i_2_1_532 * 16) + 10)] = (Y_local[((i_2_1_532 * 16) + 10)] + (A_shared_dyn_local[i_2_1_532] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_533 = 0; i_2_1_533 < 4; ++i_2_1_533) {
    Y_local[((i_2_1_533 * 16) + 74)] = (Y_local[((i_2_1_533 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_533 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_534 = 0; i_2_1_534 < 4; ++i_2_1_534) {
    Y_local[((i_2_1_534 * 16) + 11)] = (Y_local[((i_2_1_534 * 16) + 11)] + (A_shared_dyn_local[i_2_1_534] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_535 = 0; i_2_1_535 < 4; ++i_2_1_535) {
    Y_local[((i_2_1_535 * 16) + 75)] = (Y_local[((i_2_1_535 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_535 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_536 = 0; i_2_1_536 < 4; ++i_2_1_536) {
    Y_local[((i_2_1_536 * 16) + 12)] = (Y_local[((i_2_1_536 * 16) + 12)] + (A_shared_dyn_local[i_2_1_536] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_537 = 0; i_2_1_537 < 4; ++i_2_1_537) {
    Y_local[((i_2_1_537 * 16) + 76)] = (Y_local[((i_2_1_537 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_537 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_538 = 0; i_2_1_538 < 4; ++i_2_1_538) {
    Y_local[((i_2_1_538 * 16) + 13)] = (Y_local[((i_2_1_538 * 16) + 13)] + (A_shared_dyn_local[i_2_1_538] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_539 = 0; i_2_1_539 < 4; ++i_2_1_539) {
    Y_local[((i_2_1_539 * 16) + 77)] = (Y_local[((i_2_1_539 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_539 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_540 = 0; i_2_1_540 < 4; ++i_2_1_540) {
    Y_local[((i_2_1_540 * 16) + 14)] = (Y_local[((i_2_1_540 * 16) + 14)] + (A_shared_dyn_local[i_2_1_540] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_541 = 0; i_2_1_541 < 4; ++i_2_1_541) {
    Y_local[((i_2_1_541 * 16) + 78)] = (Y_local[((i_2_1_541 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_541 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_542 = 0; i_2_1_542 < 4; ++i_2_1_542) {
    Y_local[((i_2_1_542 * 16) + 15)] = (Y_local[((i_2_1_542 * 16) + 15)] + (A_shared_dyn_local[i_2_1_542] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_543 = 0; i_2_1_543 < 4; ++i_2_1_543) {
    Y_local[((i_2_1_543 * 16) + 79)] = (Y_local[((i_2_1_543 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_543 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_36 = 0; ax1_0_36 < 2; ++ax1_0_36) {
    *(float4*)(A_shared_dyn_local + (ax1_0_36 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_36 * 4)) + 5520));
  }
  for (int ax1_0_37 = 0; ax1_0_37 < 4; ++ax1_0_37) {
    *(float4*)(B_shared_dyn_local + (ax1_0_37 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_37 * 4)) + 1440));
  }
  for (int i_2_1_544 = 0; i_2_1_544 < 4; ++i_2_1_544) {
    Y_local[(i_2_1_544 * 16)] = (Y_local[(i_2_1_544 * 16)] + (A_shared_dyn_local[(i_2_1_544 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_545 = 0; i_2_1_545 < 4; ++i_2_1_545) {
    Y_local[((i_2_1_545 * 16) + 64)] = (Y_local[((i_2_1_545 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_545 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_546 = 0; i_2_1_546 < 4; ++i_2_1_546) {
    Y_local[((i_2_1_546 * 16) + 1)] = (Y_local[((i_2_1_546 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_546 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_547 = 0; i_2_1_547 < 4; ++i_2_1_547) {
    Y_local[((i_2_1_547 * 16) + 65)] = (Y_local[((i_2_1_547 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_547 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_548 = 0; i_2_1_548 < 4; ++i_2_1_548) {
    Y_local[((i_2_1_548 * 16) + 2)] = (Y_local[((i_2_1_548 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_548 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_549 = 0; i_2_1_549 < 4; ++i_2_1_549) {
    Y_local[((i_2_1_549 * 16) + 66)] = (Y_local[((i_2_1_549 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_549 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_550 = 0; i_2_1_550 < 4; ++i_2_1_550) {
    Y_local[((i_2_1_550 * 16) + 3)] = (Y_local[((i_2_1_550 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_550 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_551 = 0; i_2_1_551 < 4; ++i_2_1_551) {
    Y_local[((i_2_1_551 * 16) + 67)] = (Y_local[((i_2_1_551 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_551 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_552 = 0; i_2_1_552 < 4; ++i_2_1_552) {
    Y_local[((i_2_1_552 * 16) + 4)] = (Y_local[((i_2_1_552 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_552 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_553 = 0; i_2_1_553 < 4; ++i_2_1_553) {
    Y_local[((i_2_1_553 * 16) + 68)] = (Y_local[((i_2_1_553 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_553 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_554 = 0; i_2_1_554 < 4; ++i_2_1_554) {
    Y_local[((i_2_1_554 * 16) + 5)] = (Y_local[((i_2_1_554 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_554 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_555 = 0; i_2_1_555 < 4; ++i_2_1_555) {
    Y_local[((i_2_1_555 * 16) + 69)] = (Y_local[((i_2_1_555 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_555 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_556 = 0; i_2_1_556 < 4; ++i_2_1_556) {
    Y_local[((i_2_1_556 * 16) + 6)] = (Y_local[((i_2_1_556 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_556 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_557 = 0; i_2_1_557 < 4; ++i_2_1_557) {
    Y_local[((i_2_1_557 * 16) + 70)] = (Y_local[((i_2_1_557 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_557 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_558 = 0; i_2_1_558 < 4; ++i_2_1_558) {
    Y_local[((i_2_1_558 * 16) + 7)] = (Y_local[((i_2_1_558 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_558 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_559 = 0; i_2_1_559 < 4; ++i_2_1_559) {
    Y_local[((i_2_1_559 * 16) + 71)] = (Y_local[((i_2_1_559 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_559 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_560 = 0; i_2_1_560 < 4; ++i_2_1_560) {
    Y_local[((i_2_1_560 * 16) + 8)] = (Y_local[((i_2_1_560 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_560 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_561 = 0; i_2_1_561 < 4; ++i_2_1_561) {
    Y_local[((i_2_1_561 * 16) + 72)] = (Y_local[((i_2_1_561 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_561 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_562 = 0; i_2_1_562 < 4; ++i_2_1_562) {
    Y_local[((i_2_1_562 * 16) + 9)] = (Y_local[((i_2_1_562 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_562 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_563 = 0; i_2_1_563 < 4; ++i_2_1_563) {
    Y_local[((i_2_1_563 * 16) + 73)] = (Y_local[((i_2_1_563 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_563 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_564 = 0; i_2_1_564 < 4; ++i_2_1_564) {
    Y_local[((i_2_1_564 * 16) + 10)] = (Y_local[((i_2_1_564 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_564 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_565 = 0; i_2_1_565 < 4; ++i_2_1_565) {
    Y_local[((i_2_1_565 * 16) + 74)] = (Y_local[((i_2_1_565 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_565 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_566 = 0; i_2_1_566 < 4; ++i_2_1_566) {
    Y_local[((i_2_1_566 * 16) + 11)] = (Y_local[((i_2_1_566 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_566 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_567 = 0; i_2_1_567 < 4; ++i_2_1_567) {
    Y_local[((i_2_1_567 * 16) + 75)] = (Y_local[((i_2_1_567 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_567 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_568 = 0; i_2_1_568 < 4; ++i_2_1_568) {
    Y_local[((i_2_1_568 * 16) + 12)] = (Y_local[((i_2_1_568 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_568 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_569 = 0; i_2_1_569 < 4; ++i_2_1_569) {
    Y_local[((i_2_1_569 * 16) + 76)] = (Y_local[((i_2_1_569 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_569 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_570 = 0; i_2_1_570 < 4; ++i_2_1_570) {
    Y_local[((i_2_1_570 * 16) + 13)] = (Y_local[((i_2_1_570 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_570 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_571 = 0; i_2_1_571 < 4; ++i_2_1_571) {
    Y_local[((i_2_1_571 * 16) + 77)] = (Y_local[((i_2_1_571 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_571 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_572 = 0; i_2_1_572 < 4; ++i_2_1_572) {
    Y_local[((i_2_1_572 * 16) + 14)] = (Y_local[((i_2_1_572 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_572 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_573 = 0; i_2_1_573 < 4; ++i_2_1_573) {
    Y_local[((i_2_1_573 * 16) + 78)] = (Y_local[((i_2_1_573 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_573 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_574 = 0; i_2_1_574 < 4; ++i_2_1_574) {
    Y_local[((i_2_1_574 * 16) + 15)] = (Y_local[((i_2_1_574 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_574 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_575 = 0; i_2_1_575 < 4; ++i_2_1_575) {
    Y_local[((i_2_1_575 * 16) + 79)] = (Y_local[((i_2_1_575 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_575 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_38 = 0; ax1_0_38 < 2; ++ax1_0_38) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_38 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_38 * 4)) + 5720));
  }
  for (int ax1_0_39 = 0; ax1_0_39 < 4; ++ax1_0_39) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_39 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_39 * 4)) + 1520));
  }
  for (int i_2_1_576 = 0; i_2_1_576 < 4; ++i_2_1_576) {
    Y_local[(i_2_1_576 * 16)] = (Y_local[(i_2_1_576 * 16)] + (A_shared_dyn_local[i_2_1_576] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_577 = 0; i_2_1_577 < 4; ++i_2_1_577) {
    Y_local[((i_2_1_577 * 16) + 64)] = (Y_local[((i_2_1_577 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_577 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_578 = 0; i_2_1_578 < 4; ++i_2_1_578) {
    Y_local[((i_2_1_578 * 16) + 1)] = (Y_local[((i_2_1_578 * 16) + 1)] + (A_shared_dyn_local[i_2_1_578] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_579 = 0; i_2_1_579 < 4; ++i_2_1_579) {
    Y_local[((i_2_1_579 * 16) + 65)] = (Y_local[((i_2_1_579 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_579 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_580 = 0; i_2_1_580 < 4; ++i_2_1_580) {
    Y_local[((i_2_1_580 * 16) + 2)] = (Y_local[((i_2_1_580 * 16) + 2)] + (A_shared_dyn_local[i_2_1_580] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_581 = 0; i_2_1_581 < 4; ++i_2_1_581) {
    Y_local[((i_2_1_581 * 16) + 66)] = (Y_local[((i_2_1_581 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_581 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_582 = 0; i_2_1_582 < 4; ++i_2_1_582) {
    Y_local[((i_2_1_582 * 16) + 3)] = (Y_local[((i_2_1_582 * 16) + 3)] + (A_shared_dyn_local[i_2_1_582] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_583 = 0; i_2_1_583 < 4; ++i_2_1_583) {
    Y_local[((i_2_1_583 * 16) + 67)] = (Y_local[((i_2_1_583 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_583 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_584 = 0; i_2_1_584 < 4; ++i_2_1_584) {
    Y_local[((i_2_1_584 * 16) + 4)] = (Y_local[((i_2_1_584 * 16) + 4)] + (A_shared_dyn_local[i_2_1_584] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_585 = 0; i_2_1_585 < 4; ++i_2_1_585) {
    Y_local[((i_2_1_585 * 16) + 68)] = (Y_local[((i_2_1_585 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_585 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_586 = 0; i_2_1_586 < 4; ++i_2_1_586) {
    Y_local[((i_2_1_586 * 16) + 5)] = (Y_local[((i_2_1_586 * 16) + 5)] + (A_shared_dyn_local[i_2_1_586] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_587 = 0; i_2_1_587 < 4; ++i_2_1_587) {
    Y_local[((i_2_1_587 * 16) + 69)] = (Y_local[((i_2_1_587 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_587 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_588 = 0; i_2_1_588 < 4; ++i_2_1_588) {
    Y_local[((i_2_1_588 * 16) + 6)] = (Y_local[((i_2_1_588 * 16) + 6)] + (A_shared_dyn_local[i_2_1_588] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_589 = 0; i_2_1_589 < 4; ++i_2_1_589) {
    Y_local[((i_2_1_589 * 16) + 70)] = (Y_local[((i_2_1_589 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_589 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_590 = 0; i_2_1_590 < 4; ++i_2_1_590) {
    Y_local[((i_2_1_590 * 16) + 7)] = (Y_local[((i_2_1_590 * 16) + 7)] + (A_shared_dyn_local[i_2_1_590] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_591 = 0; i_2_1_591 < 4; ++i_2_1_591) {
    Y_local[((i_2_1_591 * 16) + 71)] = (Y_local[((i_2_1_591 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_591 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_592 = 0; i_2_1_592 < 4; ++i_2_1_592) {
    Y_local[((i_2_1_592 * 16) + 8)] = (Y_local[((i_2_1_592 * 16) + 8)] + (A_shared_dyn_local[i_2_1_592] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_593 = 0; i_2_1_593 < 4; ++i_2_1_593) {
    Y_local[((i_2_1_593 * 16) + 72)] = (Y_local[((i_2_1_593 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_593 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_594 = 0; i_2_1_594 < 4; ++i_2_1_594) {
    Y_local[((i_2_1_594 * 16) + 9)] = (Y_local[((i_2_1_594 * 16) + 9)] + (A_shared_dyn_local[i_2_1_594] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_595 = 0; i_2_1_595 < 4; ++i_2_1_595) {
    Y_local[((i_2_1_595 * 16) + 73)] = (Y_local[((i_2_1_595 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_595 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_596 = 0; i_2_1_596 < 4; ++i_2_1_596) {
    Y_local[((i_2_1_596 * 16) + 10)] = (Y_local[((i_2_1_596 * 16) + 10)] + (A_shared_dyn_local[i_2_1_596] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_597 = 0; i_2_1_597 < 4; ++i_2_1_597) {
    Y_local[((i_2_1_597 * 16) + 74)] = (Y_local[((i_2_1_597 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_597 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_598 = 0; i_2_1_598 < 4; ++i_2_1_598) {
    Y_local[((i_2_1_598 * 16) + 11)] = (Y_local[((i_2_1_598 * 16) + 11)] + (A_shared_dyn_local[i_2_1_598] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_599 = 0; i_2_1_599 < 4; ++i_2_1_599) {
    Y_local[((i_2_1_599 * 16) + 75)] = (Y_local[((i_2_1_599 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_599 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_600 = 0; i_2_1_600 < 4; ++i_2_1_600) {
    Y_local[((i_2_1_600 * 16) + 12)] = (Y_local[((i_2_1_600 * 16) + 12)] + (A_shared_dyn_local[i_2_1_600] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_601 = 0; i_2_1_601 < 4; ++i_2_1_601) {
    Y_local[((i_2_1_601 * 16) + 76)] = (Y_local[((i_2_1_601 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_601 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_602 = 0; i_2_1_602 < 4; ++i_2_1_602) {
    Y_local[((i_2_1_602 * 16) + 13)] = (Y_local[((i_2_1_602 * 16) + 13)] + (A_shared_dyn_local[i_2_1_602] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_603 = 0; i_2_1_603 < 4; ++i_2_1_603) {
    Y_local[((i_2_1_603 * 16) + 77)] = (Y_local[((i_2_1_603 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_603 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_604 = 0; i_2_1_604 < 4; ++i_2_1_604) {
    Y_local[((i_2_1_604 * 16) + 14)] = (Y_local[((i_2_1_604 * 16) + 14)] + (A_shared_dyn_local[i_2_1_604] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_605 = 0; i_2_1_605 < 4; ++i_2_1_605) {
    Y_local[((i_2_1_605 * 16) + 78)] = (Y_local[((i_2_1_605 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_605 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_606 = 0; i_2_1_606 < 4; ++i_2_1_606) {
    Y_local[((i_2_1_606 * 16) + 15)] = (Y_local[((i_2_1_606 * 16) + 15)] + (A_shared_dyn_local[i_2_1_606] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_607 = 0; i_2_1_607 < 4; ++i_2_1_607) {
    Y_local[((i_2_1_607 * 16) + 79)] = (Y_local[((i_2_1_607 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_607 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_40 = 0; ax1_0_40 < 2; ++ax1_0_40) {
    *(float4*)(A_shared_dyn_local + (ax1_0_40 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_40 * 4)) + 5920));
  }
  for (int ax1_0_41 = 0; ax1_0_41 < 4; ++ax1_0_41) {
    *(float4*)(B_shared_dyn_local + (ax1_0_41 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_41 * 4)) + 1600));
  }
  for (int i_2_1_608 = 0; i_2_1_608 < 4; ++i_2_1_608) {
    Y_local[(i_2_1_608 * 16)] = (Y_local[(i_2_1_608 * 16)] + (A_shared_dyn_local[(i_2_1_608 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_609 = 0; i_2_1_609 < 4; ++i_2_1_609) {
    Y_local[((i_2_1_609 * 16) + 64)] = (Y_local[((i_2_1_609 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_609 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_610 = 0; i_2_1_610 < 4; ++i_2_1_610) {
    Y_local[((i_2_1_610 * 16) + 1)] = (Y_local[((i_2_1_610 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_610 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_611 = 0; i_2_1_611 < 4; ++i_2_1_611) {
    Y_local[((i_2_1_611 * 16) + 65)] = (Y_local[((i_2_1_611 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_611 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_612 = 0; i_2_1_612 < 4; ++i_2_1_612) {
    Y_local[((i_2_1_612 * 16) + 2)] = (Y_local[((i_2_1_612 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_612 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_613 = 0; i_2_1_613 < 4; ++i_2_1_613) {
    Y_local[((i_2_1_613 * 16) + 66)] = (Y_local[((i_2_1_613 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_613 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_614 = 0; i_2_1_614 < 4; ++i_2_1_614) {
    Y_local[((i_2_1_614 * 16) + 3)] = (Y_local[((i_2_1_614 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_614 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_615 = 0; i_2_1_615 < 4; ++i_2_1_615) {
    Y_local[((i_2_1_615 * 16) + 67)] = (Y_local[((i_2_1_615 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_615 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_616 = 0; i_2_1_616 < 4; ++i_2_1_616) {
    Y_local[((i_2_1_616 * 16) + 4)] = (Y_local[((i_2_1_616 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_616 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_617 = 0; i_2_1_617 < 4; ++i_2_1_617) {
    Y_local[((i_2_1_617 * 16) + 68)] = (Y_local[((i_2_1_617 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_617 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_618 = 0; i_2_1_618 < 4; ++i_2_1_618) {
    Y_local[((i_2_1_618 * 16) + 5)] = (Y_local[((i_2_1_618 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_618 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_619 = 0; i_2_1_619 < 4; ++i_2_1_619) {
    Y_local[((i_2_1_619 * 16) + 69)] = (Y_local[((i_2_1_619 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_619 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_620 = 0; i_2_1_620 < 4; ++i_2_1_620) {
    Y_local[((i_2_1_620 * 16) + 6)] = (Y_local[((i_2_1_620 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_620 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_621 = 0; i_2_1_621 < 4; ++i_2_1_621) {
    Y_local[((i_2_1_621 * 16) + 70)] = (Y_local[((i_2_1_621 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_621 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_622 = 0; i_2_1_622 < 4; ++i_2_1_622) {
    Y_local[((i_2_1_622 * 16) + 7)] = (Y_local[((i_2_1_622 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_622 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_623 = 0; i_2_1_623 < 4; ++i_2_1_623) {
    Y_local[((i_2_1_623 * 16) + 71)] = (Y_local[((i_2_1_623 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_623 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_624 = 0; i_2_1_624 < 4; ++i_2_1_624) {
    Y_local[((i_2_1_624 * 16) + 8)] = (Y_local[((i_2_1_624 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_624 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_625 = 0; i_2_1_625 < 4; ++i_2_1_625) {
    Y_local[((i_2_1_625 * 16) + 72)] = (Y_local[((i_2_1_625 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_625 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_626 = 0; i_2_1_626 < 4; ++i_2_1_626) {
    Y_local[((i_2_1_626 * 16) + 9)] = (Y_local[((i_2_1_626 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_626 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_627 = 0; i_2_1_627 < 4; ++i_2_1_627) {
    Y_local[((i_2_1_627 * 16) + 73)] = (Y_local[((i_2_1_627 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_627 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_628 = 0; i_2_1_628 < 4; ++i_2_1_628) {
    Y_local[((i_2_1_628 * 16) + 10)] = (Y_local[((i_2_1_628 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_628 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_629 = 0; i_2_1_629 < 4; ++i_2_1_629) {
    Y_local[((i_2_1_629 * 16) + 74)] = (Y_local[((i_2_1_629 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_629 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_630 = 0; i_2_1_630 < 4; ++i_2_1_630) {
    Y_local[((i_2_1_630 * 16) + 11)] = (Y_local[((i_2_1_630 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_630 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_631 = 0; i_2_1_631 < 4; ++i_2_1_631) {
    Y_local[((i_2_1_631 * 16) + 75)] = (Y_local[((i_2_1_631 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_631 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_632 = 0; i_2_1_632 < 4; ++i_2_1_632) {
    Y_local[((i_2_1_632 * 16) + 12)] = (Y_local[((i_2_1_632 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_632 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_633 = 0; i_2_1_633 < 4; ++i_2_1_633) {
    Y_local[((i_2_1_633 * 16) + 76)] = (Y_local[((i_2_1_633 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_633 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_634 = 0; i_2_1_634 < 4; ++i_2_1_634) {
    Y_local[((i_2_1_634 * 16) + 13)] = (Y_local[((i_2_1_634 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_634 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_635 = 0; i_2_1_635 < 4; ++i_2_1_635) {
    Y_local[((i_2_1_635 * 16) + 77)] = (Y_local[((i_2_1_635 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_635 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_636 = 0; i_2_1_636 < 4; ++i_2_1_636) {
    Y_local[((i_2_1_636 * 16) + 14)] = (Y_local[((i_2_1_636 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_636 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_637 = 0; i_2_1_637 < 4; ++i_2_1_637) {
    Y_local[((i_2_1_637 * 16) + 78)] = (Y_local[((i_2_1_637 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_637 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_638 = 0; i_2_1_638 < 4; ++i_2_1_638) {
    Y_local[((i_2_1_638 * 16) + 15)] = (Y_local[((i_2_1_638 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_638 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_639 = 0; i_2_1_639 < 4; ++i_2_1_639) {
    Y_local[((i_2_1_639 * 16) + 79)] = (Y_local[((i_2_1_639 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_639 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_42 = 0; ax1_0_42 < 2; ++ax1_0_42) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_42 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_42 * 4)) + 6120));
  }
  for (int ax1_0_43 = 0; ax1_0_43 < 4; ++ax1_0_43) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_43 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_43 * 4)) + 1680));
  }
  for (int i_2_1_640 = 0; i_2_1_640 < 4; ++i_2_1_640) {
    Y_local[(i_2_1_640 * 16)] = (Y_local[(i_2_1_640 * 16)] + (A_shared_dyn_local[i_2_1_640] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_641 = 0; i_2_1_641 < 4; ++i_2_1_641) {
    Y_local[((i_2_1_641 * 16) + 64)] = (Y_local[((i_2_1_641 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_641 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_642 = 0; i_2_1_642 < 4; ++i_2_1_642) {
    Y_local[((i_2_1_642 * 16) + 1)] = (Y_local[((i_2_1_642 * 16) + 1)] + (A_shared_dyn_local[i_2_1_642] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_643 = 0; i_2_1_643 < 4; ++i_2_1_643) {
    Y_local[((i_2_1_643 * 16) + 65)] = (Y_local[((i_2_1_643 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_643 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_644 = 0; i_2_1_644 < 4; ++i_2_1_644) {
    Y_local[((i_2_1_644 * 16) + 2)] = (Y_local[((i_2_1_644 * 16) + 2)] + (A_shared_dyn_local[i_2_1_644] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_645 = 0; i_2_1_645 < 4; ++i_2_1_645) {
    Y_local[((i_2_1_645 * 16) + 66)] = (Y_local[((i_2_1_645 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_645 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_646 = 0; i_2_1_646 < 4; ++i_2_1_646) {
    Y_local[((i_2_1_646 * 16) + 3)] = (Y_local[((i_2_1_646 * 16) + 3)] + (A_shared_dyn_local[i_2_1_646] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_647 = 0; i_2_1_647 < 4; ++i_2_1_647) {
    Y_local[((i_2_1_647 * 16) + 67)] = (Y_local[((i_2_1_647 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_647 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_648 = 0; i_2_1_648 < 4; ++i_2_1_648) {
    Y_local[((i_2_1_648 * 16) + 4)] = (Y_local[((i_2_1_648 * 16) + 4)] + (A_shared_dyn_local[i_2_1_648] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_649 = 0; i_2_1_649 < 4; ++i_2_1_649) {
    Y_local[((i_2_1_649 * 16) + 68)] = (Y_local[((i_2_1_649 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_649 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_650 = 0; i_2_1_650 < 4; ++i_2_1_650) {
    Y_local[((i_2_1_650 * 16) + 5)] = (Y_local[((i_2_1_650 * 16) + 5)] + (A_shared_dyn_local[i_2_1_650] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_651 = 0; i_2_1_651 < 4; ++i_2_1_651) {
    Y_local[((i_2_1_651 * 16) + 69)] = (Y_local[((i_2_1_651 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_651 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_652 = 0; i_2_1_652 < 4; ++i_2_1_652) {
    Y_local[((i_2_1_652 * 16) + 6)] = (Y_local[((i_2_1_652 * 16) + 6)] + (A_shared_dyn_local[i_2_1_652] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_653 = 0; i_2_1_653 < 4; ++i_2_1_653) {
    Y_local[((i_2_1_653 * 16) + 70)] = (Y_local[((i_2_1_653 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_653 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_654 = 0; i_2_1_654 < 4; ++i_2_1_654) {
    Y_local[((i_2_1_654 * 16) + 7)] = (Y_local[((i_2_1_654 * 16) + 7)] + (A_shared_dyn_local[i_2_1_654] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_655 = 0; i_2_1_655 < 4; ++i_2_1_655) {
    Y_local[((i_2_1_655 * 16) + 71)] = (Y_local[((i_2_1_655 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_655 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_656 = 0; i_2_1_656 < 4; ++i_2_1_656) {
    Y_local[((i_2_1_656 * 16) + 8)] = (Y_local[((i_2_1_656 * 16) + 8)] + (A_shared_dyn_local[i_2_1_656] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_657 = 0; i_2_1_657 < 4; ++i_2_1_657) {
    Y_local[((i_2_1_657 * 16) + 72)] = (Y_local[((i_2_1_657 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_657 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_658 = 0; i_2_1_658 < 4; ++i_2_1_658) {
    Y_local[((i_2_1_658 * 16) + 9)] = (Y_local[((i_2_1_658 * 16) + 9)] + (A_shared_dyn_local[i_2_1_658] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_659 = 0; i_2_1_659 < 4; ++i_2_1_659) {
    Y_local[((i_2_1_659 * 16) + 73)] = (Y_local[((i_2_1_659 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_659 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_660 = 0; i_2_1_660 < 4; ++i_2_1_660) {
    Y_local[((i_2_1_660 * 16) + 10)] = (Y_local[((i_2_1_660 * 16) + 10)] + (A_shared_dyn_local[i_2_1_660] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_661 = 0; i_2_1_661 < 4; ++i_2_1_661) {
    Y_local[((i_2_1_661 * 16) + 74)] = (Y_local[((i_2_1_661 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_661 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_662 = 0; i_2_1_662 < 4; ++i_2_1_662) {
    Y_local[((i_2_1_662 * 16) + 11)] = (Y_local[((i_2_1_662 * 16) + 11)] + (A_shared_dyn_local[i_2_1_662] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_663 = 0; i_2_1_663 < 4; ++i_2_1_663) {
    Y_local[((i_2_1_663 * 16) + 75)] = (Y_local[((i_2_1_663 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_663 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_664 = 0; i_2_1_664 < 4; ++i_2_1_664) {
    Y_local[((i_2_1_664 * 16) + 12)] = (Y_local[((i_2_1_664 * 16) + 12)] + (A_shared_dyn_local[i_2_1_664] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_665 = 0; i_2_1_665 < 4; ++i_2_1_665) {
    Y_local[((i_2_1_665 * 16) + 76)] = (Y_local[((i_2_1_665 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_665 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_666 = 0; i_2_1_666 < 4; ++i_2_1_666) {
    Y_local[((i_2_1_666 * 16) + 13)] = (Y_local[((i_2_1_666 * 16) + 13)] + (A_shared_dyn_local[i_2_1_666] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_667 = 0; i_2_1_667 < 4; ++i_2_1_667) {
    Y_local[((i_2_1_667 * 16) + 77)] = (Y_local[((i_2_1_667 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_667 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_668 = 0; i_2_1_668 < 4; ++i_2_1_668) {
    Y_local[((i_2_1_668 * 16) + 14)] = (Y_local[((i_2_1_668 * 16) + 14)] + (A_shared_dyn_local[i_2_1_668] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_669 = 0; i_2_1_669 < 4; ++i_2_1_669) {
    Y_local[((i_2_1_669 * 16) + 78)] = (Y_local[((i_2_1_669 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_669 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_670 = 0; i_2_1_670 < 4; ++i_2_1_670) {
    Y_local[((i_2_1_670 * 16) + 15)] = (Y_local[((i_2_1_670 * 16) + 15)] + (A_shared_dyn_local[i_2_1_670] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_671 = 0; i_2_1_671 < 4; ++i_2_1_671) {
    Y_local[((i_2_1_671 * 16) + 79)] = (Y_local[((i_2_1_671 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_671 + 4)] * B_shared_dyn_local[15]));
  }
  for (int ax1_0_44 = 0; ax1_0_44 < 2; ++ax1_0_44) {
    *(float4*)(A_shared_dyn_local + (ax1_0_44 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_44 * 4)) + 6320));
  }
  for (int ax1_0_45 = 0; ax1_0_45 < 4; ++ax1_0_45) {
    *(float4*)(B_shared_dyn_local + (ax1_0_45 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_45 * 4)) + 1760));
  }
  for (int i_2_1_672 = 0; i_2_1_672 < 4; ++i_2_1_672) {
    Y_local[(i_2_1_672 * 16)] = (Y_local[(i_2_1_672 * 16)] + (A_shared_dyn_local[(i_2_1_672 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_673 = 0; i_2_1_673 < 4; ++i_2_1_673) {
    Y_local[((i_2_1_673 * 16) + 64)] = (Y_local[((i_2_1_673 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_673 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_674 = 0; i_2_1_674 < 4; ++i_2_1_674) {
    Y_local[((i_2_1_674 * 16) + 1)] = (Y_local[((i_2_1_674 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_674 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_675 = 0; i_2_1_675 < 4; ++i_2_1_675) {
    Y_local[((i_2_1_675 * 16) + 65)] = (Y_local[((i_2_1_675 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_675 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_676 = 0; i_2_1_676 < 4; ++i_2_1_676) {
    Y_local[((i_2_1_676 * 16) + 2)] = (Y_local[((i_2_1_676 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_676 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_677 = 0; i_2_1_677 < 4; ++i_2_1_677) {
    Y_local[((i_2_1_677 * 16) + 66)] = (Y_local[((i_2_1_677 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_677 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_678 = 0; i_2_1_678 < 4; ++i_2_1_678) {
    Y_local[((i_2_1_678 * 16) + 3)] = (Y_local[((i_2_1_678 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_678 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_679 = 0; i_2_1_679 < 4; ++i_2_1_679) {
    Y_local[((i_2_1_679 * 16) + 67)] = (Y_local[((i_2_1_679 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_679 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_680 = 0; i_2_1_680 < 4; ++i_2_1_680) {
    Y_local[((i_2_1_680 * 16) + 4)] = (Y_local[((i_2_1_680 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_680 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_681 = 0; i_2_1_681 < 4; ++i_2_1_681) {
    Y_local[((i_2_1_681 * 16) + 68)] = (Y_local[((i_2_1_681 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_681 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_682 = 0; i_2_1_682 < 4; ++i_2_1_682) {
    Y_local[((i_2_1_682 * 16) + 5)] = (Y_local[((i_2_1_682 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_682 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_683 = 0; i_2_1_683 < 4; ++i_2_1_683) {
    Y_local[((i_2_1_683 * 16) + 69)] = (Y_local[((i_2_1_683 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_683 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_684 = 0; i_2_1_684 < 4; ++i_2_1_684) {
    Y_local[((i_2_1_684 * 16) + 6)] = (Y_local[((i_2_1_684 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_684 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_685 = 0; i_2_1_685 < 4; ++i_2_1_685) {
    Y_local[((i_2_1_685 * 16) + 70)] = (Y_local[((i_2_1_685 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_685 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_686 = 0; i_2_1_686 < 4; ++i_2_1_686) {
    Y_local[((i_2_1_686 * 16) + 7)] = (Y_local[((i_2_1_686 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_686 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_687 = 0; i_2_1_687 < 4; ++i_2_1_687) {
    Y_local[((i_2_1_687 * 16) + 71)] = (Y_local[((i_2_1_687 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_687 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_688 = 0; i_2_1_688 < 4; ++i_2_1_688) {
    Y_local[((i_2_1_688 * 16) + 8)] = (Y_local[((i_2_1_688 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_688 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_689 = 0; i_2_1_689 < 4; ++i_2_1_689) {
    Y_local[((i_2_1_689 * 16) + 72)] = (Y_local[((i_2_1_689 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_689 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_690 = 0; i_2_1_690 < 4; ++i_2_1_690) {
    Y_local[((i_2_1_690 * 16) + 9)] = (Y_local[((i_2_1_690 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_690 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_691 = 0; i_2_1_691 < 4; ++i_2_1_691) {
    Y_local[((i_2_1_691 * 16) + 73)] = (Y_local[((i_2_1_691 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_691 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_692 = 0; i_2_1_692 < 4; ++i_2_1_692) {
    Y_local[((i_2_1_692 * 16) + 10)] = (Y_local[((i_2_1_692 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_692 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_693 = 0; i_2_1_693 < 4; ++i_2_1_693) {
    Y_local[((i_2_1_693 * 16) + 74)] = (Y_local[((i_2_1_693 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_693 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_694 = 0; i_2_1_694 < 4; ++i_2_1_694) {
    Y_local[((i_2_1_694 * 16) + 11)] = (Y_local[((i_2_1_694 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_694 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_695 = 0; i_2_1_695 < 4; ++i_2_1_695) {
    Y_local[((i_2_1_695 * 16) + 75)] = (Y_local[((i_2_1_695 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_695 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_696 = 0; i_2_1_696 < 4; ++i_2_1_696) {
    Y_local[((i_2_1_696 * 16) + 12)] = (Y_local[((i_2_1_696 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_696 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_697 = 0; i_2_1_697 < 4; ++i_2_1_697) {
    Y_local[((i_2_1_697 * 16) + 76)] = (Y_local[((i_2_1_697 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_697 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_698 = 0; i_2_1_698 < 4; ++i_2_1_698) {
    Y_local[((i_2_1_698 * 16) + 13)] = (Y_local[((i_2_1_698 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_698 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_699 = 0; i_2_1_699 < 4; ++i_2_1_699) {
    Y_local[((i_2_1_699 * 16) + 77)] = (Y_local[((i_2_1_699 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_699 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_700 = 0; i_2_1_700 < 4; ++i_2_1_700) {
    Y_local[((i_2_1_700 * 16) + 14)] = (Y_local[((i_2_1_700 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_700 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_701 = 0; i_2_1_701 < 4; ++i_2_1_701) {
    Y_local[((i_2_1_701 * 16) + 78)] = (Y_local[((i_2_1_701 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_701 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_702 = 0; i_2_1_702 < 4; ++i_2_1_702) {
    Y_local[((i_2_1_702 * 16) + 15)] = (Y_local[((i_2_1_702 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_702 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_703 = 0; i_2_1_703 < 4; ++i_2_1_703) {
    Y_local[((i_2_1_703 * 16) + 79)] = (Y_local[((i_2_1_703 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_703 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_46 = 0; ax1_0_46 < 2; ++ax1_0_46) {
    *(float4*)(A_shared_dyn_local + ((ax1_0_46 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) % 25) * 8) + (ax1_0_46 * 4)) + 6520));
  }
  for (int ax1_0_47 = 0; ax1_0_47 < 4; ++ax1_0_47) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_47 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((int)threadIdx.x) / 25) * 16) + (ax1_0_47 * 4)) + 1840));
  }
  for (int i_2_1_704 = 0; i_2_1_704 < 4; ++i_2_1_704) {
    Y_local[(i_2_1_704 * 16)] = (Y_local[(i_2_1_704 * 16)] + (A_shared_dyn_local[i_2_1_704] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_705 = 0; i_2_1_705 < 4; ++i_2_1_705) {
    Y_local[((i_2_1_705 * 16) + 64)] = (Y_local[((i_2_1_705 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_705 + 4)] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_706 = 0; i_2_1_706 < 4; ++i_2_1_706) {
    Y_local[((i_2_1_706 * 16) + 1)] = (Y_local[((i_2_1_706 * 16) + 1)] + (A_shared_dyn_local[i_2_1_706] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_707 = 0; i_2_1_707 < 4; ++i_2_1_707) {
    Y_local[((i_2_1_707 * 16) + 65)] = (Y_local[((i_2_1_707 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_707 + 4)] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_708 = 0; i_2_1_708 < 4; ++i_2_1_708) {
    Y_local[((i_2_1_708 * 16) + 2)] = (Y_local[((i_2_1_708 * 16) + 2)] + (A_shared_dyn_local[i_2_1_708] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_709 = 0; i_2_1_709 < 4; ++i_2_1_709) {
    Y_local[((i_2_1_709 * 16) + 66)] = (Y_local[((i_2_1_709 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_709 + 4)] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_710 = 0; i_2_1_710 < 4; ++i_2_1_710) {
    Y_local[((i_2_1_710 * 16) + 3)] = (Y_local[((i_2_1_710 * 16) + 3)] + (A_shared_dyn_local[i_2_1_710] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_711 = 0; i_2_1_711 < 4; ++i_2_1_711) {
    Y_local[((i_2_1_711 * 16) + 67)] = (Y_local[((i_2_1_711 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_711 + 4)] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_712 = 0; i_2_1_712 < 4; ++i_2_1_712) {
    Y_local[((i_2_1_712 * 16) + 4)] = (Y_local[((i_2_1_712 * 16) + 4)] + (A_shared_dyn_local[i_2_1_712] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_713 = 0; i_2_1_713 < 4; ++i_2_1_713) {
    Y_local[((i_2_1_713 * 16) + 68)] = (Y_local[((i_2_1_713 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_713 + 4)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_714 = 0; i_2_1_714 < 4; ++i_2_1_714) {
    Y_local[((i_2_1_714 * 16) + 5)] = (Y_local[((i_2_1_714 * 16) + 5)] + (A_shared_dyn_local[i_2_1_714] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_715 = 0; i_2_1_715 < 4; ++i_2_1_715) {
    Y_local[((i_2_1_715 * 16) + 69)] = (Y_local[((i_2_1_715 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_715 + 4)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_716 = 0; i_2_1_716 < 4; ++i_2_1_716) {
    Y_local[((i_2_1_716 * 16) + 6)] = (Y_local[((i_2_1_716 * 16) + 6)] + (A_shared_dyn_local[i_2_1_716] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_717 = 0; i_2_1_717 < 4; ++i_2_1_717) {
    Y_local[((i_2_1_717 * 16) + 70)] = (Y_local[((i_2_1_717 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_717 + 4)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_718 = 0; i_2_1_718 < 4; ++i_2_1_718) {
    Y_local[((i_2_1_718 * 16) + 7)] = (Y_local[((i_2_1_718 * 16) + 7)] + (A_shared_dyn_local[i_2_1_718] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_719 = 0; i_2_1_719 < 4; ++i_2_1_719) {
    Y_local[((i_2_1_719 * 16) + 71)] = (Y_local[((i_2_1_719 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_719 + 4)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_720 = 0; i_2_1_720 < 4; ++i_2_1_720) {
    Y_local[((i_2_1_720 * 16) + 8)] = (Y_local[((i_2_1_720 * 16) + 8)] + (A_shared_dyn_local[i_2_1_720] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_721 = 0; i_2_1_721 < 4; ++i_2_1_721) {
    Y_local[((i_2_1_721 * 16) + 72)] = (Y_local[((i_2_1_721 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_721 + 4)] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_722 = 0; i_2_1_722 < 4; ++i_2_1_722) {
    Y_local[((i_2_1_722 * 16) + 9)] = (Y_local[((i_2_1_722 * 16) + 9)] + (A_shared_dyn_local[i_2_1_722] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_723 = 0; i_2_1_723 < 4; ++i_2_1_723) {
    Y_local[((i_2_1_723 * 16) + 73)] = (Y_local[((i_2_1_723 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_723 + 4)] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_724 = 0; i_2_1_724 < 4; ++i_2_1_724) {
    Y_local[((i_2_1_724 * 16) + 10)] = (Y_local[((i_2_1_724 * 16) + 10)] + (A_shared_dyn_local[i_2_1_724] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_725 = 0; i_2_1_725 < 4; ++i_2_1_725) {
    Y_local[((i_2_1_725 * 16) + 74)] = (Y_local[((i_2_1_725 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_725 + 4)] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_726 = 0; i_2_1_726 < 4; ++i_2_1_726) {
    Y_local[((i_2_1_726 * 16) + 11)] = (Y_local[((i_2_1_726 * 16) + 11)] + (A_shared_dyn_local[i_2_1_726] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_727 = 0; i_2_1_727 < 4; ++i_2_1_727) {
    Y_local[((i_2_1_727 * 16) + 75)] = (Y_local[((i_2_1_727 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_727 + 4)] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_728 = 0; i_2_1_728 < 4; ++i_2_1_728) {
    Y_local[((i_2_1_728 * 16) + 12)] = (Y_local[((i_2_1_728 * 16) + 12)] + (A_shared_dyn_local[i_2_1_728] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_729 = 0; i_2_1_729 < 4; ++i_2_1_729) {
    Y_local[((i_2_1_729 * 16) + 76)] = (Y_local[((i_2_1_729 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_729 + 4)] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_730 = 0; i_2_1_730 < 4; ++i_2_1_730) {
    Y_local[((i_2_1_730 * 16) + 13)] = (Y_local[((i_2_1_730 * 16) + 13)] + (A_shared_dyn_local[i_2_1_730] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_731 = 0; i_2_1_731 < 4; ++i_2_1_731) {
    Y_local[((i_2_1_731 * 16) + 77)] = (Y_local[((i_2_1_731 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_731 + 4)] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_732 = 0; i_2_1_732 < 4; ++i_2_1_732) {
    Y_local[((i_2_1_732 * 16) + 14)] = (Y_local[((i_2_1_732 * 16) + 14)] + (A_shared_dyn_local[i_2_1_732] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_733 = 0; i_2_1_733 < 4; ++i_2_1_733) {
    Y_local[((i_2_1_733 * 16) + 78)] = (Y_local[((i_2_1_733 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_733 + 4)] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_734 = 0; i_2_1_734 < 4; ++i_2_1_734) {
    Y_local[((i_2_1_734 * 16) + 15)] = (Y_local[((i_2_1_734 * 16) + 15)] + (A_shared_dyn_local[i_2_1_734] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_735 = 0; i_2_1_735 < 4; ++i_2_1_735) {
    Y_local[((i_2_1_735 * 16) + 79)] = (Y_local[((i_2_1_735 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_735 + 4)] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_736 = 0; i_2_1_736 < 4; ++i_2_1_736) {
    Y_local[(i_2_1_736 * 16)] = (Y_local[(i_2_1_736 * 16)] + (A_shared_dyn_local[(i_2_1_736 + 8)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_737 = 0; i_2_1_737 < 4; ++i_2_1_737) {
    Y_local[((i_2_1_737 * 16) + 64)] = (Y_local[((i_2_1_737 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_737 + 12)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_738 = 0; i_2_1_738 < 4; ++i_2_1_738) {
    Y_local[((i_2_1_738 * 16) + 1)] = (Y_local[((i_2_1_738 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_738 + 8)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_739 = 0; i_2_1_739 < 4; ++i_2_1_739) {
    Y_local[((i_2_1_739 * 16) + 65)] = (Y_local[((i_2_1_739 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_739 + 12)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_740 = 0; i_2_1_740 < 4; ++i_2_1_740) {
    Y_local[((i_2_1_740 * 16) + 2)] = (Y_local[((i_2_1_740 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_740 + 8)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_741 = 0; i_2_1_741 < 4; ++i_2_1_741) {
    Y_local[((i_2_1_741 * 16) + 66)] = (Y_local[((i_2_1_741 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_741 + 12)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_742 = 0; i_2_1_742 < 4; ++i_2_1_742) {
    Y_local[((i_2_1_742 * 16) + 3)] = (Y_local[((i_2_1_742 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_742 + 8)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_743 = 0; i_2_1_743 < 4; ++i_2_1_743) {
    Y_local[((i_2_1_743 * 16) + 67)] = (Y_local[((i_2_1_743 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_743 + 12)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_744 = 0; i_2_1_744 < 4; ++i_2_1_744) {
    Y_local[((i_2_1_744 * 16) + 4)] = (Y_local[((i_2_1_744 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_744 + 8)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_745 = 0; i_2_1_745 < 4; ++i_2_1_745) {
    Y_local[((i_2_1_745 * 16) + 68)] = (Y_local[((i_2_1_745 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_745 + 12)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_746 = 0; i_2_1_746 < 4; ++i_2_1_746) {
    Y_local[((i_2_1_746 * 16) + 5)] = (Y_local[((i_2_1_746 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_746 + 8)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_747 = 0; i_2_1_747 < 4; ++i_2_1_747) {
    Y_local[((i_2_1_747 * 16) + 69)] = (Y_local[((i_2_1_747 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_747 + 12)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_748 = 0; i_2_1_748 < 4; ++i_2_1_748) {
    Y_local[((i_2_1_748 * 16) + 6)] = (Y_local[((i_2_1_748 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_748 + 8)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_749 = 0; i_2_1_749 < 4; ++i_2_1_749) {
    Y_local[((i_2_1_749 * 16) + 70)] = (Y_local[((i_2_1_749 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_749 + 12)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_750 = 0; i_2_1_750 < 4; ++i_2_1_750) {
    Y_local[((i_2_1_750 * 16) + 7)] = (Y_local[((i_2_1_750 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_750 + 8)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_751 = 0; i_2_1_751 < 4; ++i_2_1_751) {
    Y_local[((i_2_1_751 * 16) + 71)] = (Y_local[((i_2_1_751 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_751 + 12)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_752 = 0; i_2_1_752 < 4; ++i_2_1_752) {
    Y_local[((i_2_1_752 * 16) + 8)] = (Y_local[((i_2_1_752 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_752 + 8)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_753 = 0; i_2_1_753 < 4; ++i_2_1_753) {
    Y_local[((i_2_1_753 * 16) + 72)] = (Y_local[((i_2_1_753 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_753 + 12)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_754 = 0; i_2_1_754 < 4; ++i_2_1_754) {
    Y_local[((i_2_1_754 * 16) + 9)] = (Y_local[((i_2_1_754 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_754 + 8)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_755 = 0; i_2_1_755 < 4; ++i_2_1_755) {
    Y_local[((i_2_1_755 * 16) + 73)] = (Y_local[((i_2_1_755 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_755 + 12)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_756 = 0; i_2_1_756 < 4; ++i_2_1_756) {
    Y_local[((i_2_1_756 * 16) + 10)] = (Y_local[((i_2_1_756 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_756 + 8)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_757 = 0; i_2_1_757 < 4; ++i_2_1_757) {
    Y_local[((i_2_1_757 * 16) + 74)] = (Y_local[((i_2_1_757 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_757 + 12)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_758 = 0; i_2_1_758 < 4; ++i_2_1_758) {
    Y_local[((i_2_1_758 * 16) + 11)] = (Y_local[((i_2_1_758 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_758 + 8)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_759 = 0; i_2_1_759 < 4; ++i_2_1_759) {
    Y_local[((i_2_1_759 * 16) + 75)] = (Y_local[((i_2_1_759 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_759 + 12)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_760 = 0; i_2_1_760 < 4; ++i_2_1_760) {
    Y_local[((i_2_1_760 * 16) + 12)] = (Y_local[((i_2_1_760 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_760 + 8)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_761 = 0; i_2_1_761 < 4; ++i_2_1_761) {
    Y_local[((i_2_1_761 * 16) + 76)] = (Y_local[((i_2_1_761 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_761 + 12)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_762 = 0; i_2_1_762 < 4; ++i_2_1_762) {
    Y_local[((i_2_1_762 * 16) + 13)] = (Y_local[((i_2_1_762 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_762 + 8)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_763 = 0; i_2_1_763 < 4; ++i_2_1_763) {
    Y_local[((i_2_1_763 * 16) + 77)] = (Y_local[((i_2_1_763 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_763 + 12)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_764 = 0; i_2_1_764 < 4; ++i_2_1_764) {
    Y_local[((i_2_1_764 * 16) + 14)] = (Y_local[((i_2_1_764 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_764 + 8)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_765 = 0; i_2_1_765 < 4; ++i_2_1_765) {
    Y_local[((i_2_1_765 * 16) + 78)] = (Y_local[((i_2_1_765 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_765 + 12)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_766 = 0; i_2_1_766 < 4; ++i_2_1_766) {
    Y_local[((i_2_1_766 * 16) + 15)] = (Y_local[((i_2_1_766 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_766 + 8)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_767 = 0; i_2_1_767 < 4; ++i_2_1_767) {
    Y_local[((i_2_1_767 * 16) + 79)] = (Y_local[((i_2_1_767 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_767 + 12)] * B_shared_dyn_local[31]));
  }
  for (int ax1_0_48 = 0; ax1_0_48 < 4; ++ax1_0_48) {
    *(float4*)(Y + ((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_48 * 4))) = *(float4*)(Y_local + (ax1_0_48 * 4));
  }
  for (int ax1_0_49 = 0; ax1_0_49 < 4; ++ax1_0_49) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_49 * 4)) + 2000)) = *(float4*)(Y_local + ((ax1_0_49 * 4) + 16));
  }
  for (int ax1_0_50 = 0; ax1_0_50 < 4; ++ax1_0_50) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_50 * 4)) + 4000)) = *(float4*)(Y_local + ((ax1_0_50 * 4) + 32));
  }
  for (int ax1_0_51 = 0; ax1_0_51 < 4; ++ax1_0_51) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_51 * 4)) + 6000)) = *(float4*)(Y_local + ((ax1_0_51 * 4) + 48));
  }
  for (int ax1_0_52 = 0; ax1_0_52 < 4; ++ax1_0_52) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_52 * 4)) + 8000)) = *(float4*)(Y_local + ((ax1_0_52 * 4) + 64));
  }
  for (int ax1_0_53 = 0; ax1_0_53 < 4; ++ax1_0_53) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_53 * 4)) + 10000)) = *(float4*)(Y_local + ((ax1_0_53 * 4) + 80));
  }
  for (int ax1_0_54 = 0; ax1_0_54 < 4; ++ax1_0_54) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_54 * 4)) + 12000)) = *(float4*)(Y_local + ((ax1_0_54 * 4) + 96));
  }
  for (int ax1_0_55 = 0; ax1_0_55 < 4; ++ax1_0_55) {
    *(float4*)(Y + (((((((((int)blockIdx.x) / 25) * 400000) + ((((int)threadIdx.x) % 25) * 16000)) + ((((int)blockIdx.x) % 25) * 80)) + ((((int)threadIdx.x) / 25) * 16)) + (ax1_0_55 * 4)) + 14000)) = *(float4*)(Y_local + ((ax1_0_55 * 4) + 112));
  }
}


