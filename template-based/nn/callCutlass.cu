#include "hip/hip_runtime.h"
#include "cutlass/gemm/device/gemm.h"
#include "include/inc.h"

using CutlassBestPerf = cutlass::gemm::device::Gemm<
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor,
    float, cutlass::layout::RowMajor,
    float,
    cutlass::arch::OpClassSimt,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<128, 64, 8>, // MODIFY THIS LINE
    cutlass::gemm::GemmShape<64, 32, 8>,  // MODIFY THIS LINE
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        float,
        1,
        float,
        float
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
    2,                                    // MODIFY THIS LINE
    1,
    1,
    true,
    cutlass::arch::OpMultiplyAdd
    >;

void run_bestPerf(float *A, float *B, float *C) {
    int lda = K;
    int ldb = N;
    int ldc = M;
    CutlassBestPerf gemm_operator;
    CutlassBestPerf::Arguments args(
        {M, N, K},
        {A, lda},
        {B, ldb},
        {C, ldc},
        {C, ldc},
        {1, 0}
    );
    gemm_operator(args);
}

class CUDAMatrix {
private:
    float *mat, *dev_mat;
    int row, col;
    size_t size;

public:
    CUDAMatrix(int r, int c) : row(r), col(c), size(sizeof(float) * r * c) { mat = (float *) malloc(size); hipMalloc((void **) &dev_mat, size); }
    ~CUDAMatrix() { free(mat), hipFree(dev_mat); }
    float *devPtr() { return dev_mat; }
    void cpyToDevice() { hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice); }
    void cpyToHost() { hipMemcpy(mat, dev_mat, size, hipMemcpyDeviceToHost); }
    float &at(int i, int j) { return mat[i * col + j]; }
    const float &at(int i, int j) const { return mat[i * col + j]; }
    void print() {
        for (int i = 0; i < row; i++) {
            for (int j = 0; j < col; j++) {
                printf("%.2f ", mat[i * col + j]);
            }
            printf("\n");
        }
    }
    void fillRand() {
        for (int i = 0; i < row; i++)
            for (int j = 0; j < col; j++)
                at(i, j) = rand() % 100;
    }
    void fill(float val) {
        for (int i = 0; i < row; i++)
            for (int j = 0; j < col; j++)
                at(i, j) = val;
    }
};

int main() {
    int round = 100;
    int warmupGoal = round / 10;
    double totTime = 0;
    CUDAMatrix A(M, K), B(K, N), C(M, N);
    A.fillRand(), B.fillRand();
    A.cpyToDevice(), B.cpyToDevice();
    for (int i = 0; i < warmupGoal; i++) run_bestPerf(A.devPtr(), B.devPtr(), C.devPtr());
    for (int i = 0; i < round; i++) {
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        run_bestPerf(A.devPtr(), B.devPtr(), C.devPtr());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float ti;
        hipEventElapsedTime(&ti, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        totTime += ti;
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
    }
    C.cpyToHost();
    double FLOPs = (((long long int) (M)) * N * K + M * N) * 2;
    double runtime = totTime / round;
    printf("cutlass GFLOPs: %lf\n", double(FLOPs) / runtime / 1.0e6);
}