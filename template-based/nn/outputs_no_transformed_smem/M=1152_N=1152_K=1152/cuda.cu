
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(216) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[96];
  float A_shared_dyn_local[12];
  float B_shared_dyn_local[32];
  Y_local[0] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[32] = 0.000000e+00f;
  Y_local[48] = 0.000000e+00f;
  Y_local[64] = 0.000000e+00f;
  Y_local[80] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[33] = 0.000000e+00f;
  Y_local[49] = 0.000000e+00f;
  Y_local[65] = 0.000000e+00f;
  Y_local[81] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[34] = 0.000000e+00f;
  Y_local[50] = 0.000000e+00f;
  Y_local[66] = 0.000000e+00f;
  Y_local[82] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[35] = 0.000000e+00f;
  Y_local[51] = 0.000000e+00f;
  Y_local[67] = 0.000000e+00f;
  Y_local[83] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[36] = 0.000000e+00f;
  Y_local[52] = 0.000000e+00f;
  Y_local[68] = 0.000000e+00f;
  Y_local[84] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[37] = 0.000000e+00f;
  Y_local[53] = 0.000000e+00f;
  Y_local[69] = 0.000000e+00f;
  Y_local[85] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[38] = 0.000000e+00f;
  Y_local[54] = 0.000000e+00f;
  Y_local[70] = 0.000000e+00f;
  Y_local[86] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;
  Y_local[39] = 0.000000e+00f;
  Y_local[55] = 0.000000e+00f;
  Y_local[71] = 0.000000e+00f;
  Y_local[87] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[24] = 0.000000e+00f;
  Y_local[40] = 0.000000e+00f;
  Y_local[56] = 0.000000e+00f;
  Y_local[72] = 0.000000e+00f;
  Y_local[88] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[25] = 0.000000e+00f;
  Y_local[41] = 0.000000e+00f;
  Y_local[57] = 0.000000e+00f;
  Y_local[73] = 0.000000e+00f;
  Y_local[89] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[26] = 0.000000e+00f;
  Y_local[42] = 0.000000e+00f;
  Y_local[58] = 0.000000e+00f;
  Y_local[74] = 0.000000e+00f;
  Y_local[90] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[27] = 0.000000e+00f;
  Y_local[43] = 0.000000e+00f;
  Y_local[59] = 0.000000e+00f;
  Y_local[75] = 0.000000e+00f;
  Y_local[91] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[28] = 0.000000e+00f;
  Y_local[44] = 0.000000e+00f;
  Y_local[60] = 0.000000e+00f;
  Y_local[76] = 0.000000e+00f;
  Y_local[92] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[29] = 0.000000e+00f;
  Y_local[45] = 0.000000e+00f;
  Y_local[61] = 0.000000e+00f;
  Y_local[77] = 0.000000e+00f;
  Y_local[93] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[30] = 0.000000e+00f;
  Y_local[46] = 0.000000e+00f;
  Y_local[62] = 0.000000e+00f;
  Y_local[78] = 0.000000e+00f;
  Y_local[94] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[31] = 0.000000e+00f;
  Y_local[47] = 0.000000e+00f;
  Y_local[63] = 0.000000e+00f;
  Y_local[79] = 0.000000e+00f;
  Y_local[95] = 0.000000e+00f;
  for (int ax0_ax1_fused_2_s = 0; ax0_ax1_fused_2_s < 3; ++ax0_ax1_fused_2_s) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((int)threadIdx.x) * 12) + (ax0_ax1_fused_2_s * 4))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) >> 3) * 165888) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s) >> 2) * 1152)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s) & 3)))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_s_1 = 0; ax0_ax1_fused_2_s_1 < 3; ++ax0_ax1_fused_2_s_1) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) / 48) * 768) + (((((((int)blockIdx.x) & 7) * 9) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_1) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_1) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 2) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_1) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_1) & 3) * 4)) + 11520) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) / 48) * 1152) + ((((int)blockIdx.x) & 7) * 144)) + ((((int)threadIdx.x) % 48) * 3)) + ax0_ax1_fused_2_s_1))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_2 = 0; ax0_ax1_fused_2_s_2 < 3; ++ax0_ax1_fused_2_s_2) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 12) + (ax0_ax1_fused_2_s_2 * 4)) + 2304)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 3) * 165888) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_2) >> 2) * 1152)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_2) & 3)) + 4))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_s_3 = 0; ax0_ax1_fused_2_s_3 < 3; ++ax0_ax1_fused_2_s_3) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) / 48) * 768) + (((((((int)blockIdx.x) & 7) * 9) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_3) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_3) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 2) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_3) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_3) & 3) * 4)) + 14592) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) / 48) * 1152) + ((((int)blockIdx.x) & 7) * 144)) + ((((int)threadIdx.x) % 48) * 3)) + ax0_ax1_fused_2_s_3) + 4608))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_4 = 0; ax0_ax1_fused_2_s_4 < 3; ++ax0_ax1_fused_2_s_4) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 12) + (ax0_ax1_fused_2_s_4 * 4)) + 4608)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 3) * 165888) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_4) >> 2) * 1152)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_4) & 3)) + 8))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_s_5 = 0; ax0_ax1_fused_2_s_5 < 3; ++ax0_ax1_fused_2_s_5) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) / 48) * 768) + (((((((int)blockIdx.x) & 7) * 9) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_5) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_5) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 2) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_5) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_5) & 3) * 4)) + 17664) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) / 48) * 1152) + ((((int)blockIdx.x) & 7) * 144)) + ((((int)threadIdx.x) % 48) * 3)) + ax0_ax1_fused_2_s_5) + 9216))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_6 = 0; ax0_ax1_fused_2_s_6 < 3; ++ax0_ax1_fused_2_s_6) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((int)threadIdx.x) * 12) + (ax0_ax1_fused_2_s_6 * 4)) + 6912)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 3) * 165888) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_6) >> 2) * 1152)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_6) & 3)) + 12))), "n"(4)
    );
  }
    }
  }
  for (int ax0_ax1_fused_2_s_7 = 0; ax0_ax1_fused_2_s_7 < 3; ++ax0_ax1_fused_2_s_7) {
    if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((int)threadIdx.x) / 48) * 768) + (((((((int)blockIdx.x) & 7) * 9) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_7) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_7) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 2) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_7) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_7) & 3) * 4)) + 20736) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((((int)threadIdx.x) / 48) * 1152) + ((((int)blockIdx.x) & 7) * 144)) + ((((int)threadIdx.x) % 48) * 3)) + ax0_ax1_fused_2_s_7) + 13824))), "n"(4)
    );
  }
    }
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

  __syncthreads();
  for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
    for (int ax0_1_s = 0; ax0_1_s < 4; ++ax0_1_s) {
      if (((ax0_0 * 2) + (ax0_1_s >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0 * 4) + ax0_1_s)] = ((float*)buf_dyn_shmem)[(((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0 * 16)) + (ax0_1_s * 4))];
      }
    }
  }
  for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {
    *(float4*)(B_shared_dyn_local + (ax1_0 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0 * 4)) >> 6) * 64) + ((ax1_0 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0 >> 1)) & 7) * 4)) + 2880) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int k_0 = 0; k_0 < 284; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_2_s_8 = 0; ax0_ax1_fused_2_s_8 < 3; ++ax0_ax1_fused_2_s_8) {
      if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((k_0 + 4) % 5) * 2304) + (((int)threadIdx.x) * 12)) + (ax0_ax1_fused_2_s_8 * 4))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 3) * 165888) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_8) >> 2) * 1152)) + (k_0 * 4)) + (((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_8) & 3)) + 16))), "n"(4)
    );
  }
      }
    }
    for (int ax0_ax1_fused_2_s_9 = 0; ax0_ax1_fused_2_s_9 < 3; ++ax0_ax1_fused_2_s_9) {
      if (((int)threadIdx.x) < 192) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((((k_0 + 4) % 5) * 3072) + ((((int)threadIdx.x) / 48) * 768)) + (((((((int)blockIdx.x) & 7) * 9) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_9) >> 4)) >> 2) * 256)) + (((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_9) & 7) >> 2) * 128)) + (((((((int)blockIdx.x) & 7) * 2) + ((((((int)threadIdx.x) % 48) * 3) + ax0_ax1_fused_2_s_9) >> 3)) & 7) * 16)) + ((((((int)threadIdx.x) * 3) + ax0_ax1_fused_2_s_9) & 3) * 4)) + 11520) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 256))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((k_0 * 4608) + ((((int)threadIdx.x) / 48) * 1152)) + ((((int)blockIdx.x) & 7) * 144)) + ((((int)threadIdx.x) % 48) * 3)) + ax0_ax1_fused_2_s_9) + 18432))), "n"(4)
    );
  }
      }
    }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1) {
      for (int ax0_1_s_1 = 0; ax0_1_s_1 < 4; ++ax0_1_s_1) {
        if (((ax0_0_1 * 2) + (ax0_1_s_1 >> 1)) < 3) {
          A_shared_dyn_local[(((ax0_0_1 * 4) + ax0_1_s_1) + 6)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 576) + ((((int)threadIdx.x) / 27) * 72)) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_1 * 16)) + (ax0_1_s_1 * 4)) + 1)];
        }
      }
    }
    for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_1 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 768) + ((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_1 * 4)) >> 6) * 64)) + ((ax1_0_1 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_1 >> 1)) & 7) * 4)) + 3072) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
    }
    for (int i_2_1_s = 0; i_2_1_s < 4; ++i_2_1_s) {
      Y_local[(i_2_1_s * 16)] = (Y_local[(i_2_1_s * 16)] + (A_shared_dyn_local[i_2_1_s] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_1 = 0; i_2_1_s_1 < 4; ++i_2_1_s_1) {
      if (i_2_1_s_1 < 2) {
        Y_local[((i_2_1_s_1 * 16) + 64)] = (Y_local[((i_2_1_s_1 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_1 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_2 = 0; i_2_1_s_2 < 4; ++i_2_1_s_2) {
      Y_local[((i_2_1_s_2 * 16) + 1)] = (Y_local[((i_2_1_s_2 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_2] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_3 = 0; i_2_1_s_3 < 4; ++i_2_1_s_3) {
      if (i_2_1_s_3 < 2) {
        Y_local[((i_2_1_s_3 * 16) + 65)] = (Y_local[((i_2_1_s_3 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_3 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_4 = 0; i_2_1_s_4 < 4; ++i_2_1_s_4) {
      Y_local[((i_2_1_s_4 * 16) + 2)] = (Y_local[((i_2_1_s_4 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_4] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_5 = 0; i_2_1_s_5 < 4; ++i_2_1_s_5) {
      if (i_2_1_s_5 < 2) {
        Y_local[((i_2_1_s_5 * 16) + 66)] = (Y_local[((i_2_1_s_5 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_5 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_6 = 0; i_2_1_s_6 < 4; ++i_2_1_s_6) {
      Y_local[((i_2_1_s_6 * 16) + 3)] = (Y_local[((i_2_1_s_6 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_6] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_7 = 0; i_2_1_s_7 < 4; ++i_2_1_s_7) {
      if (i_2_1_s_7 < 2) {
        Y_local[((i_2_1_s_7 * 16) + 67)] = (Y_local[((i_2_1_s_7 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_7 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_8 = 0; i_2_1_s_8 < 4; ++i_2_1_s_8) {
      Y_local[((i_2_1_s_8 * 16) + 4)] = (Y_local[((i_2_1_s_8 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_8] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_9 = 0; i_2_1_s_9 < 4; ++i_2_1_s_9) {
      if (i_2_1_s_9 < 2) {
        Y_local[((i_2_1_s_9 * 16) + 68)] = (Y_local[((i_2_1_s_9 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_9 + 4)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_10 = 0; i_2_1_s_10 < 4; ++i_2_1_s_10) {
      Y_local[((i_2_1_s_10 * 16) + 5)] = (Y_local[((i_2_1_s_10 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_10] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_11 = 0; i_2_1_s_11 < 4; ++i_2_1_s_11) {
      if (i_2_1_s_11 < 2) {
        Y_local[((i_2_1_s_11 * 16) + 69)] = (Y_local[((i_2_1_s_11 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_11 + 4)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_12 = 0; i_2_1_s_12 < 4; ++i_2_1_s_12) {
      Y_local[((i_2_1_s_12 * 16) + 6)] = (Y_local[((i_2_1_s_12 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_12] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_13 = 0; i_2_1_s_13 < 4; ++i_2_1_s_13) {
      if (i_2_1_s_13 < 2) {
        Y_local[((i_2_1_s_13 * 16) + 70)] = (Y_local[((i_2_1_s_13 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_13 + 4)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_14 = 0; i_2_1_s_14 < 4; ++i_2_1_s_14) {
      Y_local[((i_2_1_s_14 * 16) + 7)] = (Y_local[((i_2_1_s_14 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_14] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_15 = 0; i_2_1_s_15 < 4; ++i_2_1_s_15) {
      if (i_2_1_s_15 < 2) {
        Y_local[((i_2_1_s_15 * 16) + 71)] = (Y_local[((i_2_1_s_15 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_15 + 4)] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_16 = 0; i_2_1_s_16 < 4; ++i_2_1_s_16) {
      Y_local[((i_2_1_s_16 * 16) + 8)] = (Y_local[((i_2_1_s_16 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_16] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_s_17 = 0; i_2_1_s_17 < 4; ++i_2_1_s_17) {
      if (i_2_1_s_17 < 2) {
        Y_local[((i_2_1_s_17 * 16) + 72)] = (Y_local[((i_2_1_s_17 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_17 + 4)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_18 = 0; i_2_1_s_18 < 4; ++i_2_1_s_18) {
      Y_local[((i_2_1_s_18 * 16) + 9)] = (Y_local[((i_2_1_s_18 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_18] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_s_19 = 0; i_2_1_s_19 < 4; ++i_2_1_s_19) {
      if (i_2_1_s_19 < 2) {
        Y_local[((i_2_1_s_19 * 16) + 73)] = (Y_local[((i_2_1_s_19 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_19 + 4)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_20 = 0; i_2_1_s_20 < 4; ++i_2_1_s_20) {
      Y_local[((i_2_1_s_20 * 16) + 10)] = (Y_local[((i_2_1_s_20 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_20] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_s_21 = 0; i_2_1_s_21 < 4; ++i_2_1_s_21) {
      if (i_2_1_s_21 < 2) {
        Y_local[((i_2_1_s_21 * 16) + 74)] = (Y_local[((i_2_1_s_21 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_21 + 4)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_22 = 0; i_2_1_s_22 < 4; ++i_2_1_s_22) {
      Y_local[((i_2_1_s_22 * 16) + 11)] = (Y_local[((i_2_1_s_22 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_22] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_s_23 = 0; i_2_1_s_23 < 4; ++i_2_1_s_23) {
      if (i_2_1_s_23 < 2) {
        Y_local[((i_2_1_s_23 * 16) + 75)] = (Y_local[((i_2_1_s_23 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_23 + 4)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_24 = 0; i_2_1_s_24 < 4; ++i_2_1_s_24) {
      Y_local[((i_2_1_s_24 * 16) + 12)] = (Y_local[((i_2_1_s_24 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_24] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_s_25 = 0; i_2_1_s_25 < 4; ++i_2_1_s_25) {
      if (i_2_1_s_25 < 2) {
        Y_local[((i_2_1_s_25 * 16) + 76)] = (Y_local[((i_2_1_s_25 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_25 + 4)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_26 = 0; i_2_1_s_26 < 4; ++i_2_1_s_26) {
      Y_local[((i_2_1_s_26 * 16) + 13)] = (Y_local[((i_2_1_s_26 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_26] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_s_27 = 0; i_2_1_s_27 < 4; ++i_2_1_s_27) {
      if (i_2_1_s_27 < 2) {
        Y_local[((i_2_1_s_27 * 16) + 77)] = (Y_local[((i_2_1_s_27 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_27 + 4)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_28 = 0; i_2_1_s_28 < 4; ++i_2_1_s_28) {
      Y_local[((i_2_1_s_28 * 16) + 14)] = (Y_local[((i_2_1_s_28 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_28] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_s_29 = 0; i_2_1_s_29 < 4; ++i_2_1_s_29) {
      if (i_2_1_s_29 < 2) {
        Y_local[((i_2_1_s_29 * 16) + 78)] = (Y_local[((i_2_1_s_29 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_29 + 4)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_30 = 0; i_2_1_s_30 < 4; ++i_2_1_s_30) {
      Y_local[((i_2_1_s_30 * 16) + 15)] = (Y_local[((i_2_1_s_30 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_30] * B_shared_dyn_local[15]));
    }
    for (int i_2_1_s_31 = 0; i_2_1_s_31 < 4; ++i_2_1_s_31) {
      if (i_2_1_s_31 < 2) {
        Y_local[((i_2_1_s_31 * 16) + 79)] = (Y_local[((i_2_1_s_31 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_31 + 4)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {
      for (int ax0_1_s_2 = 0; ax0_1_s_2 < 4; ++ax0_1_s_2) {
        if (((ax0_0_2 * 2) + (ax0_1_s_2 >> 1)) < 3) {
          A_shared_dyn_local[((ax0_0_2 * 4) + ax0_1_s_2)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 576) + ((((int)threadIdx.x) / 27) * 72)) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_2 * 16)) + (ax0_1_s_2 * 4)) + 2)];
        }
      }
    }
    for (int ax1_0_2 = 0; ax1_0_2 < 4; ++ax1_0_2) {
      *(float4*)(B_shared_dyn_local + (ax1_0_2 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 768) + ((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_2 * 4)) >> 6) * 64)) + ((ax1_0_2 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_2 >> 1)) & 7) * 4)) + 3264) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
    }
    for (int i_2_1_s_32 = 0; i_2_1_s_32 < 4; ++i_2_1_s_32) {
      Y_local[(i_2_1_s_32 * 16)] = (Y_local[(i_2_1_s_32 * 16)] + (A_shared_dyn_local[(i_2_1_s_32 + 6)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_s_33 = 0; i_2_1_s_33 < 4; ++i_2_1_s_33) {
      if (i_2_1_s_33 < 2) {
        Y_local[((i_2_1_s_33 * 16) + 64)] = (Y_local[((i_2_1_s_33 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_33 + 10)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_34 = 0; i_2_1_s_34 < 4; ++i_2_1_s_34) {
      Y_local[((i_2_1_s_34 * 16) + 1)] = (Y_local[((i_2_1_s_34 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_34 + 6)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_s_35 = 0; i_2_1_s_35 < 4; ++i_2_1_s_35) {
      if (i_2_1_s_35 < 2) {
        Y_local[((i_2_1_s_35 * 16) + 65)] = (Y_local[((i_2_1_s_35 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_35 + 10)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_36 = 0; i_2_1_s_36 < 4; ++i_2_1_s_36) {
      Y_local[((i_2_1_s_36 * 16) + 2)] = (Y_local[((i_2_1_s_36 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_36 + 6)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_s_37 = 0; i_2_1_s_37 < 4; ++i_2_1_s_37) {
      if (i_2_1_s_37 < 2) {
        Y_local[((i_2_1_s_37 * 16) + 66)] = (Y_local[((i_2_1_s_37 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_37 + 10)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_38 = 0; i_2_1_s_38 < 4; ++i_2_1_s_38) {
      Y_local[((i_2_1_s_38 * 16) + 3)] = (Y_local[((i_2_1_s_38 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_38 + 6)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_s_39 = 0; i_2_1_s_39 < 4; ++i_2_1_s_39) {
      if (i_2_1_s_39 < 2) {
        Y_local[((i_2_1_s_39 * 16) + 67)] = (Y_local[((i_2_1_s_39 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_39 + 10)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_40 = 0; i_2_1_s_40 < 4; ++i_2_1_s_40) {
      Y_local[((i_2_1_s_40 * 16) + 4)] = (Y_local[((i_2_1_s_40 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_40 + 6)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_s_41 = 0; i_2_1_s_41 < 4; ++i_2_1_s_41) {
      if (i_2_1_s_41 < 2) {
        Y_local[((i_2_1_s_41 * 16) + 68)] = (Y_local[((i_2_1_s_41 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_41 + 10)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_42 = 0; i_2_1_s_42 < 4; ++i_2_1_s_42) {
      Y_local[((i_2_1_s_42 * 16) + 5)] = (Y_local[((i_2_1_s_42 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_42 + 6)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_s_43 = 0; i_2_1_s_43 < 4; ++i_2_1_s_43) {
      if (i_2_1_s_43 < 2) {
        Y_local[((i_2_1_s_43 * 16) + 69)] = (Y_local[((i_2_1_s_43 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_43 + 10)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_44 = 0; i_2_1_s_44 < 4; ++i_2_1_s_44) {
      Y_local[((i_2_1_s_44 * 16) + 6)] = (Y_local[((i_2_1_s_44 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_44 + 6)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_s_45 = 0; i_2_1_s_45 < 4; ++i_2_1_s_45) {
      if (i_2_1_s_45 < 2) {
        Y_local[((i_2_1_s_45 * 16) + 70)] = (Y_local[((i_2_1_s_45 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_45 + 10)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_46 = 0; i_2_1_s_46 < 4; ++i_2_1_s_46) {
      Y_local[((i_2_1_s_46 * 16) + 7)] = (Y_local[((i_2_1_s_46 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_46 + 6)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_s_47 = 0; i_2_1_s_47 < 4; ++i_2_1_s_47) {
      if (i_2_1_s_47 < 2) {
        Y_local[((i_2_1_s_47 * 16) + 71)] = (Y_local[((i_2_1_s_47 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_47 + 10)] * B_shared_dyn_local[23]));
      }
    }
    for (int i_2_1_s_48 = 0; i_2_1_s_48 < 4; ++i_2_1_s_48) {
      Y_local[((i_2_1_s_48 * 16) + 8)] = (Y_local[((i_2_1_s_48 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_48 + 6)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_s_49 = 0; i_2_1_s_49 < 4; ++i_2_1_s_49) {
      if (i_2_1_s_49 < 2) {
        Y_local[((i_2_1_s_49 * 16) + 72)] = (Y_local[((i_2_1_s_49 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_49 + 10)] * B_shared_dyn_local[24]));
      }
    }
    for (int i_2_1_s_50 = 0; i_2_1_s_50 < 4; ++i_2_1_s_50) {
      Y_local[((i_2_1_s_50 * 16) + 9)] = (Y_local[((i_2_1_s_50 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_50 + 6)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_s_51 = 0; i_2_1_s_51 < 4; ++i_2_1_s_51) {
      if (i_2_1_s_51 < 2) {
        Y_local[((i_2_1_s_51 * 16) + 73)] = (Y_local[((i_2_1_s_51 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_51 + 10)] * B_shared_dyn_local[25]));
      }
    }
    for (int i_2_1_s_52 = 0; i_2_1_s_52 < 4; ++i_2_1_s_52) {
      Y_local[((i_2_1_s_52 * 16) + 10)] = (Y_local[((i_2_1_s_52 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_52 + 6)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_s_53 = 0; i_2_1_s_53 < 4; ++i_2_1_s_53) {
      if (i_2_1_s_53 < 2) {
        Y_local[((i_2_1_s_53 * 16) + 74)] = (Y_local[((i_2_1_s_53 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_53 + 10)] * B_shared_dyn_local[26]));
      }
    }
    for (int i_2_1_s_54 = 0; i_2_1_s_54 < 4; ++i_2_1_s_54) {
      Y_local[((i_2_1_s_54 * 16) + 11)] = (Y_local[((i_2_1_s_54 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_54 + 6)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_s_55 = 0; i_2_1_s_55 < 4; ++i_2_1_s_55) {
      if (i_2_1_s_55 < 2) {
        Y_local[((i_2_1_s_55 * 16) + 75)] = (Y_local[((i_2_1_s_55 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_55 + 10)] * B_shared_dyn_local[27]));
      }
    }
    for (int i_2_1_s_56 = 0; i_2_1_s_56 < 4; ++i_2_1_s_56) {
      Y_local[((i_2_1_s_56 * 16) + 12)] = (Y_local[((i_2_1_s_56 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_56 + 6)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_s_57 = 0; i_2_1_s_57 < 4; ++i_2_1_s_57) {
      if (i_2_1_s_57 < 2) {
        Y_local[((i_2_1_s_57 * 16) + 76)] = (Y_local[((i_2_1_s_57 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_57 + 10)] * B_shared_dyn_local[28]));
      }
    }
    for (int i_2_1_s_58 = 0; i_2_1_s_58 < 4; ++i_2_1_s_58) {
      Y_local[((i_2_1_s_58 * 16) + 13)] = (Y_local[((i_2_1_s_58 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_58 + 6)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_s_59 = 0; i_2_1_s_59 < 4; ++i_2_1_s_59) {
      if (i_2_1_s_59 < 2) {
        Y_local[((i_2_1_s_59 * 16) + 77)] = (Y_local[((i_2_1_s_59 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_59 + 10)] * B_shared_dyn_local[29]));
      }
    }
    for (int i_2_1_s_60 = 0; i_2_1_s_60 < 4; ++i_2_1_s_60) {
      Y_local[((i_2_1_s_60 * 16) + 14)] = (Y_local[((i_2_1_s_60 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_60 + 6)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_s_61 = 0; i_2_1_s_61 < 4; ++i_2_1_s_61) {
      if (i_2_1_s_61 < 2) {
        Y_local[((i_2_1_s_61 * 16) + 78)] = (Y_local[((i_2_1_s_61 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_61 + 10)] * B_shared_dyn_local[30]));
      }
    }
    for (int i_2_1_s_62 = 0; i_2_1_s_62 < 4; ++i_2_1_s_62) {
      Y_local[((i_2_1_s_62 * 16) + 15)] = (Y_local[((i_2_1_s_62 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_62 + 6)] * B_shared_dyn_local[31]));
    }
    for (int i_2_1_s_63 = 0; i_2_1_s_63 < 4; ++i_2_1_s_63) {
      if (i_2_1_s_63 < 2) {
        Y_local[((i_2_1_s_63 * 16) + 79)] = (Y_local[((i_2_1_s_63 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_63 + 10)] * B_shared_dyn_local[31]));
      }
    }
    for (int ax0_0_3 = 0; ax0_0_3 < 2; ++ax0_0_3) {
      for (int ax0_1_s_3 = 0; ax0_1_s_3 < 4; ++ax0_1_s_3) {
        if (((ax0_0_3 * 2) + (ax0_1_s_3 >> 1)) < 3) {
          A_shared_dyn_local[(((ax0_0_3 * 4) + ax0_1_s_3) + 6)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 576) + ((((int)threadIdx.x) / 27) * 72)) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_3 * 16)) + (ax0_1_s_3 * 4)) + 3)];
        }
      }
    }
    for (int ax1_0_3 = 0; ax1_0_3 < 4; ++ax1_0_3) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_3 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 768) + ((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_3 * 4)) >> 6) * 64)) + ((ax1_0_3 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_3 >> 1)) & 7) * 4)) + 3456) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
    }
    for (int i_2_1_s_64 = 0; i_2_1_s_64 < 4; ++i_2_1_s_64) {
      Y_local[(i_2_1_s_64 * 16)] = (Y_local[(i_2_1_s_64 * 16)] + (A_shared_dyn_local[i_2_1_s_64] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_65 = 0; i_2_1_s_65 < 4; ++i_2_1_s_65) {
      if (i_2_1_s_65 < 2) {
        Y_local[((i_2_1_s_65 * 16) + 64)] = (Y_local[((i_2_1_s_65 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_65 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_66 = 0; i_2_1_s_66 < 4; ++i_2_1_s_66) {
      Y_local[((i_2_1_s_66 * 16) + 1)] = (Y_local[((i_2_1_s_66 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_66] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_67 = 0; i_2_1_s_67 < 4; ++i_2_1_s_67) {
      if (i_2_1_s_67 < 2) {
        Y_local[((i_2_1_s_67 * 16) + 65)] = (Y_local[((i_2_1_s_67 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_67 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_68 = 0; i_2_1_s_68 < 4; ++i_2_1_s_68) {
      Y_local[((i_2_1_s_68 * 16) + 2)] = (Y_local[((i_2_1_s_68 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_68] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_69 = 0; i_2_1_s_69 < 4; ++i_2_1_s_69) {
      if (i_2_1_s_69 < 2) {
        Y_local[((i_2_1_s_69 * 16) + 66)] = (Y_local[((i_2_1_s_69 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_69 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_70 = 0; i_2_1_s_70 < 4; ++i_2_1_s_70) {
      Y_local[((i_2_1_s_70 * 16) + 3)] = (Y_local[((i_2_1_s_70 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_70] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_71 = 0; i_2_1_s_71 < 4; ++i_2_1_s_71) {
      if (i_2_1_s_71 < 2) {
        Y_local[((i_2_1_s_71 * 16) + 67)] = (Y_local[((i_2_1_s_71 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_71 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_72 = 0; i_2_1_s_72 < 4; ++i_2_1_s_72) {
      Y_local[((i_2_1_s_72 * 16) + 4)] = (Y_local[((i_2_1_s_72 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_72] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_73 = 0; i_2_1_s_73 < 4; ++i_2_1_s_73) {
      if (i_2_1_s_73 < 2) {
        Y_local[((i_2_1_s_73 * 16) + 68)] = (Y_local[((i_2_1_s_73 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_73 + 4)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_74 = 0; i_2_1_s_74 < 4; ++i_2_1_s_74) {
      Y_local[((i_2_1_s_74 * 16) + 5)] = (Y_local[((i_2_1_s_74 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_74] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_75 = 0; i_2_1_s_75 < 4; ++i_2_1_s_75) {
      if (i_2_1_s_75 < 2) {
        Y_local[((i_2_1_s_75 * 16) + 69)] = (Y_local[((i_2_1_s_75 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_75 + 4)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_76 = 0; i_2_1_s_76 < 4; ++i_2_1_s_76) {
      Y_local[((i_2_1_s_76 * 16) + 6)] = (Y_local[((i_2_1_s_76 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_76] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_77 = 0; i_2_1_s_77 < 4; ++i_2_1_s_77) {
      if (i_2_1_s_77 < 2) {
        Y_local[((i_2_1_s_77 * 16) + 70)] = (Y_local[((i_2_1_s_77 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_77 + 4)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_78 = 0; i_2_1_s_78 < 4; ++i_2_1_s_78) {
      Y_local[((i_2_1_s_78 * 16) + 7)] = (Y_local[((i_2_1_s_78 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_78] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_79 = 0; i_2_1_s_79 < 4; ++i_2_1_s_79) {
      if (i_2_1_s_79 < 2) {
        Y_local[((i_2_1_s_79 * 16) + 71)] = (Y_local[((i_2_1_s_79 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_79 + 4)] * B_shared_dyn_local[7]));
      }
    }
    for (int i_2_1_s_80 = 0; i_2_1_s_80 < 4; ++i_2_1_s_80) {
      Y_local[((i_2_1_s_80 * 16) + 8)] = (Y_local[((i_2_1_s_80 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_80] * B_shared_dyn_local[8]));
    }
    for (int i_2_1_s_81 = 0; i_2_1_s_81 < 4; ++i_2_1_s_81) {
      if (i_2_1_s_81 < 2) {
        Y_local[((i_2_1_s_81 * 16) + 72)] = (Y_local[((i_2_1_s_81 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_81 + 4)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_82 = 0; i_2_1_s_82 < 4; ++i_2_1_s_82) {
      Y_local[((i_2_1_s_82 * 16) + 9)] = (Y_local[((i_2_1_s_82 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_82] * B_shared_dyn_local[9]));
    }
    for (int i_2_1_s_83 = 0; i_2_1_s_83 < 4; ++i_2_1_s_83) {
      if (i_2_1_s_83 < 2) {
        Y_local[((i_2_1_s_83 * 16) + 73)] = (Y_local[((i_2_1_s_83 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_83 + 4)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_84 = 0; i_2_1_s_84 < 4; ++i_2_1_s_84) {
      Y_local[((i_2_1_s_84 * 16) + 10)] = (Y_local[((i_2_1_s_84 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_84] * B_shared_dyn_local[10]));
    }
    for (int i_2_1_s_85 = 0; i_2_1_s_85 < 4; ++i_2_1_s_85) {
      if (i_2_1_s_85 < 2) {
        Y_local[((i_2_1_s_85 * 16) + 74)] = (Y_local[((i_2_1_s_85 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_85 + 4)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_86 = 0; i_2_1_s_86 < 4; ++i_2_1_s_86) {
      Y_local[((i_2_1_s_86 * 16) + 11)] = (Y_local[((i_2_1_s_86 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_86] * B_shared_dyn_local[11]));
    }
    for (int i_2_1_s_87 = 0; i_2_1_s_87 < 4; ++i_2_1_s_87) {
      if (i_2_1_s_87 < 2) {
        Y_local[((i_2_1_s_87 * 16) + 75)] = (Y_local[((i_2_1_s_87 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_87 + 4)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_88 = 0; i_2_1_s_88 < 4; ++i_2_1_s_88) {
      Y_local[((i_2_1_s_88 * 16) + 12)] = (Y_local[((i_2_1_s_88 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_88] * B_shared_dyn_local[12]));
    }
    for (int i_2_1_s_89 = 0; i_2_1_s_89 < 4; ++i_2_1_s_89) {
      if (i_2_1_s_89 < 2) {
        Y_local[((i_2_1_s_89 * 16) + 76)] = (Y_local[((i_2_1_s_89 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_89 + 4)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_90 = 0; i_2_1_s_90 < 4; ++i_2_1_s_90) {
      Y_local[((i_2_1_s_90 * 16) + 13)] = (Y_local[((i_2_1_s_90 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_90] * B_shared_dyn_local[13]));
    }
    for (int i_2_1_s_91 = 0; i_2_1_s_91 < 4; ++i_2_1_s_91) {
      if (i_2_1_s_91 < 2) {
        Y_local[((i_2_1_s_91 * 16) + 77)] = (Y_local[((i_2_1_s_91 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_91 + 4)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_92 = 0; i_2_1_s_92 < 4; ++i_2_1_s_92) {
      Y_local[((i_2_1_s_92 * 16) + 14)] = (Y_local[((i_2_1_s_92 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_92] * B_shared_dyn_local[14]));
    }
    for (int i_2_1_s_93 = 0; i_2_1_s_93 < 4; ++i_2_1_s_93) {
      if (i_2_1_s_93 < 2) {
        Y_local[((i_2_1_s_93 * 16) + 78)] = (Y_local[((i_2_1_s_93 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_93 + 4)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_94 = 0; i_2_1_s_94 < 4; ++i_2_1_s_94) {
      Y_local[((i_2_1_s_94 * 16) + 15)] = (Y_local[((i_2_1_s_94 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_94] * B_shared_dyn_local[15]));
    }
    for (int i_2_1_s_95 = 0; i_2_1_s_95 < 4; ++i_2_1_s_95) {
      if (i_2_1_s_95 < 2) {
        Y_local[((i_2_1_s_95 * 16) + 79)] = (Y_local[((i_2_1_s_95 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_95 + 4)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax0_0_4 = 0; ax0_0_4 < 2; ++ax0_0_4) {
      for (int ax0_1_s_4 = 0; ax0_1_s_4 < 4; ++ax0_1_s_4) {
        if (((ax0_0_4 * 2) + (ax0_1_s_4 >> 1)) < 3) {
          A_shared_dyn_local[((ax0_0_4 * 4) + ax0_1_s_4)] = ((float*)buf_dyn_shmem)[(((((((k_0 + 1) % 5) * 576) + ((((int)threadIdx.x) / 27) * 72)) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_4 * 16)) + (ax0_1_s_4 * 4))];
        }
      }
    }
    for (int ax1_0_4 = 0; ax1_0_4 < 4; ++ax1_0_4) {
      *(float4*)(B_shared_dyn_local + (ax1_0_4 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((k_0 + 1) % 5) * 768) + ((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_4 * 4)) >> 6) * 64)) + ((ax1_0_4 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_4 >> 1)) & 7) * 4)) + 2880) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
    }
    for (int i_2_1_s_96 = 0; i_2_1_s_96 < 4; ++i_2_1_s_96) {
      Y_local[(i_2_1_s_96 * 16)] = (Y_local[(i_2_1_s_96 * 16)] + (A_shared_dyn_local[(i_2_1_s_96 + 6)] * B_shared_dyn_local[16]));
    }
    for (int i_2_1_s_97 = 0; i_2_1_s_97 < 4; ++i_2_1_s_97) {
      if (i_2_1_s_97 < 2) {
        Y_local[((i_2_1_s_97 * 16) + 64)] = (Y_local[((i_2_1_s_97 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_97 + 10)] * B_shared_dyn_local[16]));
      }
    }
    for (int i_2_1_s_98 = 0; i_2_1_s_98 < 4; ++i_2_1_s_98) {
      Y_local[((i_2_1_s_98 * 16) + 1)] = (Y_local[((i_2_1_s_98 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_98 + 6)] * B_shared_dyn_local[17]));
    }
    for (int i_2_1_s_99 = 0; i_2_1_s_99 < 4; ++i_2_1_s_99) {
      if (i_2_1_s_99 < 2) {
        Y_local[((i_2_1_s_99 * 16) + 65)] = (Y_local[((i_2_1_s_99 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_99 + 10)] * B_shared_dyn_local[17]));
      }
    }
    for (int i_2_1_s_100 = 0; i_2_1_s_100 < 4; ++i_2_1_s_100) {
      Y_local[((i_2_1_s_100 * 16) + 2)] = (Y_local[((i_2_1_s_100 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_100 + 6)] * B_shared_dyn_local[18]));
    }
    for (int i_2_1_s_101 = 0; i_2_1_s_101 < 4; ++i_2_1_s_101) {
      if (i_2_1_s_101 < 2) {
        Y_local[((i_2_1_s_101 * 16) + 66)] = (Y_local[((i_2_1_s_101 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_101 + 10)] * B_shared_dyn_local[18]));
      }
    }
    for (int i_2_1_s_102 = 0; i_2_1_s_102 < 4; ++i_2_1_s_102) {
      Y_local[((i_2_1_s_102 * 16) + 3)] = (Y_local[((i_2_1_s_102 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_102 + 6)] * B_shared_dyn_local[19]));
    }
    for (int i_2_1_s_103 = 0; i_2_1_s_103 < 4; ++i_2_1_s_103) {
      if (i_2_1_s_103 < 2) {
        Y_local[((i_2_1_s_103 * 16) + 67)] = (Y_local[((i_2_1_s_103 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_103 + 10)] * B_shared_dyn_local[19]));
      }
    }
    for (int i_2_1_s_104 = 0; i_2_1_s_104 < 4; ++i_2_1_s_104) {
      Y_local[((i_2_1_s_104 * 16) + 4)] = (Y_local[((i_2_1_s_104 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_104 + 6)] * B_shared_dyn_local[20]));
    }
    for (int i_2_1_s_105 = 0; i_2_1_s_105 < 4; ++i_2_1_s_105) {
      if (i_2_1_s_105 < 2) {
        Y_local[((i_2_1_s_105 * 16) + 68)] = (Y_local[((i_2_1_s_105 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_105 + 10)] * B_shared_dyn_local[20]));
      }
    }
    for (int i_2_1_s_106 = 0; i_2_1_s_106 < 4; ++i_2_1_s_106) {
      Y_local[((i_2_1_s_106 * 16) + 5)] = (Y_local[((i_2_1_s_106 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_106 + 6)] * B_shared_dyn_local[21]));
    }
    for (int i_2_1_s_107 = 0; i_2_1_s_107 < 4; ++i_2_1_s_107) {
      if (i_2_1_s_107 < 2) {
        Y_local[((i_2_1_s_107 * 16) + 69)] = (Y_local[((i_2_1_s_107 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_107 + 10)] * B_shared_dyn_local[21]));
      }
    }
    for (int i_2_1_s_108 = 0; i_2_1_s_108 < 4; ++i_2_1_s_108) {
      Y_local[((i_2_1_s_108 * 16) + 6)] = (Y_local[((i_2_1_s_108 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_108 + 6)] * B_shared_dyn_local[22]));
    }
    for (int i_2_1_s_109 = 0; i_2_1_s_109 < 4; ++i_2_1_s_109) {
      if (i_2_1_s_109 < 2) {
        Y_local[((i_2_1_s_109 * 16) + 70)] = (Y_local[((i_2_1_s_109 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_109 + 10)] * B_shared_dyn_local[22]));
      }
    }
    for (int i_2_1_s_110 = 0; i_2_1_s_110 < 4; ++i_2_1_s_110) {
      Y_local[((i_2_1_s_110 * 16) + 7)] = (Y_local[((i_2_1_s_110 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_110 + 6)] * B_shared_dyn_local[23]));
    }
    for (int i_2_1_s_111 = 0; i_2_1_s_111 < 4; ++i_2_1_s_111) {
      if (i_2_1_s_111 < 2) {
        Y_local[((i_2_1_s_111 * 16) + 71)] = (Y_local[((i_2_1_s_111 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_111 + 10)] * B_shared_dyn_local[23]));
      }
    }
    for (int i_2_1_s_112 = 0; i_2_1_s_112 < 4; ++i_2_1_s_112) {
      Y_local[((i_2_1_s_112 * 16) + 8)] = (Y_local[((i_2_1_s_112 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_112 + 6)] * B_shared_dyn_local[24]));
    }
    for (int i_2_1_s_113 = 0; i_2_1_s_113 < 4; ++i_2_1_s_113) {
      if (i_2_1_s_113 < 2) {
        Y_local[((i_2_1_s_113 * 16) + 72)] = (Y_local[((i_2_1_s_113 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_113 + 10)] * B_shared_dyn_local[24]));
      }
    }
    for (int i_2_1_s_114 = 0; i_2_1_s_114 < 4; ++i_2_1_s_114) {
      Y_local[((i_2_1_s_114 * 16) + 9)] = (Y_local[((i_2_1_s_114 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_114 + 6)] * B_shared_dyn_local[25]));
    }
    for (int i_2_1_s_115 = 0; i_2_1_s_115 < 4; ++i_2_1_s_115) {
      if (i_2_1_s_115 < 2) {
        Y_local[((i_2_1_s_115 * 16) + 73)] = (Y_local[((i_2_1_s_115 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_115 + 10)] * B_shared_dyn_local[25]));
      }
    }
    for (int i_2_1_s_116 = 0; i_2_1_s_116 < 4; ++i_2_1_s_116) {
      Y_local[((i_2_1_s_116 * 16) + 10)] = (Y_local[((i_2_1_s_116 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_116 + 6)] * B_shared_dyn_local[26]));
    }
    for (int i_2_1_s_117 = 0; i_2_1_s_117 < 4; ++i_2_1_s_117) {
      if (i_2_1_s_117 < 2) {
        Y_local[((i_2_1_s_117 * 16) + 74)] = (Y_local[((i_2_1_s_117 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_117 + 10)] * B_shared_dyn_local[26]));
      }
    }
    for (int i_2_1_s_118 = 0; i_2_1_s_118 < 4; ++i_2_1_s_118) {
      Y_local[((i_2_1_s_118 * 16) + 11)] = (Y_local[((i_2_1_s_118 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_118 + 6)] * B_shared_dyn_local[27]));
    }
    for (int i_2_1_s_119 = 0; i_2_1_s_119 < 4; ++i_2_1_s_119) {
      if (i_2_1_s_119 < 2) {
        Y_local[((i_2_1_s_119 * 16) + 75)] = (Y_local[((i_2_1_s_119 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_119 + 10)] * B_shared_dyn_local[27]));
      }
    }
    for (int i_2_1_s_120 = 0; i_2_1_s_120 < 4; ++i_2_1_s_120) {
      Y_local[((i_2_1_s_120 * 16) + 12)] = (Y_local[((i_2_1_s_120 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_120 + 6)] * B_shared_dyn_local[28]));
    }
    for (int i_2_1_s_121 = 0; i_2_1_s_121 < 4; ++i_2_1_s_121) {
      if (i_2_1_s_121 < 2) {
        Y_local[((i_2_1_s_121 * 16) + 76)] = (Y_local[((i_2_1_s_121 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_121 + 10)] * B_shared_dyn_local[28]));
      }
    }
    for (int i_2_1_s_122 = 0; i_2_1_s_122 < 4; ++i_2_1_s_122) {
      Y_local[((i_2_1_s_122 * 16) + 13)] = (Y_local[((i_2_1_s_122 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_122 + 6)] * B_shared_dyn_local[29]));
    }
    for (int i_2_1_s_123 = 0; i_2_1_s_123 < 4; ++i_2_1_s_123) {
      if (i_2_1_s_123 < 2) {
        Y_local[((i_2_1_s_123 * 16) + 77)] = (Y_local[((i_2_1_s_123 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_123 + 10)] * B_shared_dyn_local[29]));
      }
    }
    for (int i_2_1_s_124 = 0; i_2_1_s_124 < 4; ++i_2_1_s_124) {
      Y_local[((i_2_1_s_124 * 16) + 14)] = (Y_local[((i_2_1_s_124 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_124 + 6)] * B_shared_dyn_local[30]));
    }
    for (int i_2_1_s_125 = 0; i_2_1_s_125 < 4; ++i_2_1_s_125) {
      if (i_2_1_s_125 < 2) {
        Y_local[((i_2_1_s_125 * 16) + 78)] = (Y_local[((i_2_1_s_125 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_125 + 10)] * B_shared_dyn_local[30]));
      }
    }
    for (int i_2_1_s_126 = 0; i_2_1_s_126 < 4; ++i_2_1_s_126) {
      Y_local[((i_2_1_s_126 * 16) + 15)] = (Y_local[((i_2_1_s_126 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_126 + 6)] * B_shared_dyn_local[31]));
    }
    for (int i_2_1_s_127 = 0; i_2_1_s_127 < 4; ++i_2_1_s_127) {
      if (i_2_1_s_127 < 2) {
        Y_local[((i_2_1_s_127 * 16) + 79)] = (Y_local[((i_2_1_s_127 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_127 + 10)] * B_shared_dyn_local[31]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax0_0_5 = 0; ax0_0_5 < 2; ++ax0_0_5) {
    for (int ax0_1_s_5 = 0; ax0_1_s_5 < 4; ++ax0_1_s_5) {
      if (((ax0_0_5 * 2) + (ax0_1_s_5 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_5 * 4) + ax0_1_s_5) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_5 * 16)) + (ax0_1_s_5 * 4)) + 2305)];
      }
    }
  }
  for (int ax1_0_5 = 0; ax1_0_5 < 4; ++ax1_0_5) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_5 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_5 * 4)) >> 6) * 64) + ((ax1_0_5 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_5 >> 1)) & 7) * 4)) + 6144) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_128 = 0; i_2_1_s_128 < 4; ++i_2_1_s_128) {
    Y_local[(i_2_1_s_128 * 16)] = (Y_local[(i_2_1_s_128 * 16)] + (A_shared_dyn_local[i_2_1_s_128] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_129 = 0; i_2_1_s_129 < 4; ++i_2_1_s_129) {
    if (i_2_1_s_129 < 2) {
      Y_local[((i_2_1_s_129 * 16) + 64)] = (Y_local[((i_2_1_s_129 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_129 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_130 = 0; i_2_1_s_130 < 4; ++i_2_1_s_130) {
    Y_local[((i_2_1_s_130 * 16) + 1)] = (Y_local[((i_2_1_s_130 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_130] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_131 = 0; i_2_1_s_131 < 4; ++i_2_1_s_131) {
    if (i_2_1_s_131 < 2) {
      Y_local[((i_2_1_s_131 * 16) + 65)] = (Y_local[((i_2_1_s_131 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_131 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_132 = 0; i_2_1_s_132 < 4; ++i_2_1_s_132) {
    Y_local[((i_2_1_s_132 * 16) + 2)] = (Y_local[((i_2_1_s_132 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_132] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_133 = 0; i_2_1_s_133 < 4; ++i_2_1_s_133) {
    if (i_2_1_s_133 < 2) {
      Y_local[((i_2_1_s_133 * 16) + 66)] = (Y_local[((i_2_1_s_133 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_133 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_134 = 0; i_2_1_s_134 < 4; ++i_2_1_s_134) {
    Y_local[((i_2_1_s_134 * 16) + 3)] = (Y_local[((i_2_1_s_134 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_134] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_135 = 0; i_2_1_s_135 < 4; ++i_2_1_s_135) {
    if (i_2_1_s_135 < 2) {
      Y_local[((i_2_1_s_135 * 16) + 67)] = (Y_local[((i_2_1_s_135 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_135 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_136 = 0; i_2_1_s_136 < 4; ++i_2_1_s_136) {
    Y_local[((i_2_1_s_136 * 16) + 4)] = (Y_local[((i_2_1_s_136 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_136] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_137 = 0; i_2_1_s_137 < 4; ++i_2_1_s_137) {
    if (i_2_1_s_137 < 2) {
      Y_local[((i_2_1_s_137 * 16) + 68)] = (Y_local[((i_2_1_s_137 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_137 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_138 = 0; i_2_1_s_138 < 4; ++i_2_1_s_138) {
    Y_local[((i_2_1_s_138 * 16) + 5)] = (Y_local[((i_2_1_s_138 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_138] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_139 = 0; i_2_1_s_139 < 4; ++i_2_1_s_139) {
    if (i_2_1_s_139 < 2) {
      Y_local[((i_2_1_s_139 * 16) + 69)] = (Y_local[((i_2_1_s_139 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_139 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_140 = 0; i_2_1_s_140 < 4; ++i_2_1_s_140) {
    Y_local[((i_2_1_s_140 * 16) + 6)] = (Y_local[((i_2_1_s_140 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_140] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_141 = 0; i_2_1_s_141 < 4; ++i_2_1_s_141) {
    if (i_2_1_s_141 < 2) {
      Y_local[((i_2_1_s_141 * 16) + 70)] = (Y_local[((i_2_1_s_141 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_141 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_142 = 0; i_2_1_s_142 < 4; ++i_2_1_s_142) {
    Y_local[((i_2_1_s_142 * 16) + 7)] = (Y_local[((i_2_1_s_142 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_142] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_143 = 0; i_2_1_s_143 < 4; ++i_2_1_s_143) {
    if (i_2_1_s_143 < 2) {
      Y_local[((i_2_1_s_143 * 16) + 71)] = (Y_local[((i_2_1_s_143 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_143 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_144 = 0; i_2_1_s_144 < 4; ++i_2_1_s_144) {
    Y_local[((i_2_1_s_144 * 16) + 8)] = (Y_local[((i_2_1_s_144 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_144] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_145 = 0; i_2_1_s_145 < 4; ++i_2_1_s_145) {
    if (i_2_1_s_145 < 2) {
      Y_local[((i_2_1_s_145 * 16) + 72)] = (Y_local[((i_2_1_s_145 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_145 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_146 = 0; i_2_1_s_146 < 4; ++i_2_1_s_146) {
    Y_local[((i_2_1_s_146 * 16) + 9)] = (Y_local[((i_2_1_s_146 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_146] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_147 = 0; i_2_1_s_147 < 4; ++i_2_1_s_147) {
    if (i_2_1_s_147 < 2) {
      Y_local[((i_2_1_s_147 * 16) + 73)] = (Y_local[((i_2_1_s_147 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_147 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_148 = 0; i_2_1_s_148 < 4; ++i_2_1_s_148) {
    Y_local[((i_2_1_s_148 * 16) + 10)] = (Y_local[((i_2_1_s_148 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_148] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_149 = 0; i_2_1_s_149 < 4; ++i_2_1_s_149) {
    if (i_2_1_s_149 < 2) {
      Y_local[((i_2_1_s_149 * 16) + 74)] = (Y_local[((i_2_1_s_149 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_149 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_150 = 0; i_2_1_s_150 < 4; ++i_2_1_s_150) {
    Y_local[((i_2_1_s_150 * 16) + 11)] = (Y_local[((i_2_1_s_150 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_150] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_151 = 0; i_2_1_s_151 < 4; ++i_2_1_s_151) {
    if (i_2_1_s_151 < 2) {
      Y_local[((i_2_1_s_151 * 16) + 75)] = (Y_local[((i_2_1_s_151 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_151 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_152 = 0; i_2_1_s_152 < 4; ++i_2_1_s_152) {
    Y_local[((i_2_1_s_152 * 16) + 12)] = (Y_local[((i_2_1_s_152 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_152] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_153 = 0; i_2_1_s_153 < 4; ++i_2_1_s_153) {
    if (i_2_1_s_153 < 2) {
      Y_local[((i_2_1_s_153 * 16) + 76)] = (Y_local[((i_2_1_s_153 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_153 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_154 = 0; i_2_1_s_154 < 4; ++i_2_1_s_154) {
    Y_local[((i_2_1_s_154 * 16) + 13)] = (Y_local[((i_2_1_s_154 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_154] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_155 = 0; i_2_1_s_155 < 4; ++i_2_1_s_155) {
    if (i_2_1_s_155 < 2) {
      Y_local[((i_2_1_s_155 * 16) + 77)] = (Y_local[((i_2_1_s_155 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_155 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_156 = 0; i_2_1_s_156 < 4; ++i_2_1_s_156) {
    Y_local[((i_2_1_s_156 * 16) + 14)] = (Y_local[((i_2_1_s_156 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_156] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_157 = 0; i_2_1_s_157 < 4; ++i_2_1_s_157) {
    if (i_2_1_s_157 < 2) {
      Y_local[((i_2_1_s_157 * 16) + 78)] = (Y_local[((i_2_1_s_157 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_157 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_158 = 0; i_2_1_s_158 < 4; ++i_2_1_s_158) {
    Y_local[((i_2_1_s_158 * 16) + 15)] = (Y_local[((i_2_1_s_158 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_158] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_159 = 0; i_2_1_s_159 < 4; ++i_2_1_s_159) {
    if (i_2_1_s_159 < 2) {
      Y_local[((i_2_1_s_159 * 16) + 79)] = (Y_local[((i_2_1_s_159 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_159 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_6 = 0; ax0_0_6 < 2; ++ax0_0_6) {
    for (int ax0_1_s_6 = 0; ax0_1_s_6 < 4; ++ax0_1_s_6) {
      if (((ax0_0_6 * 2) + (ax0_1_s_6 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_6 * 4) + ax0_1_s_6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_6 * 16)) + (ax0_1_s_6 * 4)) + 2306)];
      }
    }
  }
  for (int ax1_0_6 = 0; ax1_0_6 < 4; ++ax1_0_6) {
    *(float4*)(B_shared_dyn_local + (ax1_0_6 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_6 * 4)) >> 6) * 64) + ((ax1_0_6 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_6 >> 1)) & 7) * 4)) + 6336) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_160 = 0; i_2_1_s_160 < 4; ++i_2_1_s_160) {
    Y_local[(i_2_1_s_160 * 16)] = (Y_local[(i_2_1_s_160 * 16)] + (A_shared_dyn_local[(i_2_1_s_160 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_161 = 0; i_2_1_s_161 < 4; ++i_2_1_s_161) {
    if (i_2_1_s_161 < 2) {
      Y_local[((i_2_1_s_161 * 16) + 64)] = (Y_local[((i_2_1_s_161 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_161 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_162 = 0; i_2_1_s_162 < 4; ++i_2_1_s_162) {
    Y_local[((i_2_1_s_162 * 16) + 1)] = (Y_local[((i_2_1_s_162 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_162 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_163 = 0; i_2_1_s_163 < 4; ++i_2_1_s_163) {
    if (i_2_1_s_163 < 2) {
      Y_local[((i_2_1_s_163 * 16) + 65)] = (Y_local[((i_2_1_s_163 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_163 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_164 = 0; i_2_1_s_164 < 4; ++i_2_1_s_164) {
    Y_local[((i_2_1_s_164 * 16) + 2)] = (Y_local[((i_2_1_s_164 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_164 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_165 = 0; i_2_1_s_165 < 4; ++i_2_1_s_165) {
    if (i_2_1_s_165 < 2) {
      Y_local[((i_2_1_s_165 * 16) + 66)] = (Y_local[((i_2_1_s_165 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_165 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_166 = 0; i_2_1_s_166 < 4; ++i_2_1_s_166) {
    Y_local[((i_2_1_s_166 * 16) + 3)] = (Y_local[((i_2_1_s_166 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_166 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_167 = 0; i_2_1_s_167 < 4; ++i_2_1_s_167) {
    if (i_2_1_s_167 < 2) {
      Y_local[((i_2_1_s_167 * 16) + 67)] = (Y_local[((i_2_1_s_167 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_167 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_168 = 0; i_2_1_s_168 < 4; ++i_2_1_s_168) {
    Y_local[((i_2_1_s_168 * 16) + 4)] = (Y_local[((i_2_1_s_168 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_168 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_169 = 0; i_2_1_s_169 < 4; ++i_2_1_s_169) {
    if (i_2_1_s_169 < 2) {
      Y_local[((i_2_1_s_169 * 16) + 68)] = (Y_local[((i_2_1_s_169 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_169 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_170 = 0; i_2_1_s_170 < 4; ++i_2_1_s_170) {
    Y_local[((i_2_1_s_170 * 16) + 5)] = (Y_local[((i_2_1_s_170 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_170 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_171 = 0; i_2_1_s_171 < 4; ++i_2_1_s_171) {
    if (i_2_1_s_171 < 2) {
      Y_local[((i_2_1_s_171 * 16) + 69)] = (Y_local[((i_2_1_s_171 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_171 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_172 = 0; i_2_1_s_172 < 4; ++i_2_1_s_172) {
    Y_local[((i_2_1_s_172 * 16) + 6)] = (Y_local[((i_2_1_s_172 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_172 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_173 = 0; i_2_1_s_173 < 4; ++i_2_1_s_173) {
    if (i_2_1_s_173 < 2) {
      Y_local[((i_2_1_s_173 * 16) + 70)] = (Y_local[((i_2_1_s_173 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_173 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_174 = 0; i_2_1_s_174 < 4; ++i_2_1_s_174) {
    Y_local[((i_2_1_s_174 * 16) + 7)] = (Y_local[((i_2_1_s_174 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_174 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_175 = 0; i_2_1_s_175 < 4; ++i_2_1_s_175) {
    if (i_2_1_s_175 < 2) {
      Y_local[((i_2_1_s_175 * 16) + 71)] = (Y_local[((i_2_1_s_175 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_175 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_176 = 0; i_2_1_s_176 < 4; ++i_2_1_s_176) {
    Y_local[((i_2_1_s_176 * 16) + 8)] = (Y_local[((i_2_1_s_176 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_176 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_177 = 0; i_2_1_s_177 < 4; ++i_2_1_s_177) {
    if (i_2_1_s_177 < 2) {
      Y_local[((i_2_1_s_177 * 16) + 72)] = (Y_local[((i_2_1_s_177 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_177 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_178 = 0; i_2_1_s_178 < 4; ++i_2_1_s_178) {
    Y_local[((i_2_1_s_178 * 16) + 9)] = (Y_local[((i_2_1_s_178 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_178 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_179 = 0; i_2_1_s_179 < 4; ++i_2_1_s_179) {
    if (i_2_1_s_179 < 2) {
      Y_local[((i_2_1_s_179 * 16) + 73)] = (Y_local[((i_2_1_s_179 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_179 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_180 = 0; i_2_1_s_180 < 4; ++i_2_1_s_180) {
    Y_local[((i_2_1_s_180 * 16) + 10)] = (Y_local[((i_2_1_s_180 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_180 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_181 = 0; i_2_1_s_181 < 4; ++i_2_1_s_181) {
    if (i_2_1_s_181 < 2) {
      Y_local[((i_2_1_s_181 * 16) + 74)] = (Y_local[((i_2_1_s_181 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_181 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_182 = 0; i_2_1_s_182 < 4; ++i_2_1_s_182) {
    Y_local[((i_2_1_s_182 * 16) + 11)] = (Y_local[((i_2_1_s_182 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_182 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_183 = 0; i_2_1_s_183 < 4; ++i_2_1_s_183) {
    if (i_2_1_s_183 < 2) {
      Y_local[((i_2_1_s_183 * 16) + 75)] = (Y_local[((i_2_1_s_183 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_183 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_184 = 0; i_2_1_s_184 < 4; ++i_2_1_s_184) {
    Y_local[((i_2_1_s_184 * 16) + 12)] = (Y_local[((i_2_1_s_184 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_184 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_185 = 0; i_2_1_s_185 < 4; ++i_2_1_s_185) {
    if (i_2_1_s_185 < 2) {
      Y_local[((i_2_1_s_185 * 16) + 76)] = (Y_local[((i_2_1_s_185 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_185 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_186 = 0; i_2_1_s_186 < 4; ++i_2_1_s_186) {
    Y_local[((i_2_1_s_186 * 16) + 13)] = (Y_local[((i_2_1_s_186 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_186 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_187 = 0; i_2_1_s_187 < 4; ++i_2_1_s_187) {
    if (i_2_1_s_187 < 2) {
      Y_local[((i_2_1_s_187 * 16) + 77)] = (Y_local[((i_2_1_s_187 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_187 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_188 = 0; i_2_1_s_188 < 4; ++i_2_1_s_188) {
    Y_local[((i_2_1_s_188 * 16) + 14)] = (Y_local[((i_2_1_s_188 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_188 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_189 = 0; i_2_1_s_189 < 4; ++i_2_1_s_189) {
    if (i_2_1_s_189 < 2) {
      Y_local[((i_2_1_s_189 * 16) + 78)] = (Y_local[((i_2_1_s_189 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_189 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_190 = 0; i_2_1_s_190 < 4; ++i_2_1_s_190) {
    Y_local[((i_2_1_s_190 * 16) + 15)] = (Y_local[((i_2_1_s_190 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_190 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_191 = 0; i_2_1_s_191 < 4; ++i_2_1_s_191) {
    if (i_2_1_s_191 < 2) {
      Y_local[((i_2_1_s_191 * 16) + 79)] = (Y_local[((i_2_1_s_191 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_191 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax0_0_7 = 0; ax0_0_7 < 2; ++ax0_0_7) {
    for (int ax0_1_s_7 = 0; ax0_1_s_7 < 4; ++ax0_1_s_7) {
      if (((ax0_0_7 * 2) + (ax0_1_s_7 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_7 * 4) + ax0_1_s_7) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_7 * 16)) + (ax0_1_s_7 * 4)) + 2307)];
      }
    }
  }
  for (int ax1_0_7 = 0; ax1_0_7 < 4; ++ax1_0_7) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_7 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_7 * 4)) >> 6) * 64) + ((ax1_0_7 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_7 >> 1)) & 7) * 4)) + 6528) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_192 = 0; i_2_1_s_192 < 4; ++i_2_1_s_192) {
    Y_local[(i_2_1_s_192 * 16)] = (Y_local[(i_2_1_s_192 * 16)] + (A_shared_dyn_local[i_2_1_s_192] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_193 = 0; i_2_1_s_193 < 4; ++i_2_1_s_193) {
    if (i_2_1_s_193 < 2) {
      Y_local[((i_2_1_s_193 * 16) + 64)] = (Y_local[((i_2_1_s_193 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_193 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_194 = 0; i_2_1_s_194 < 4; ++i_2_1_s_194) {
    Y_local[((i_2_1_s_194 * 16) + 1)] = (Y_local[((i_2_1_s_194 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_194] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_195 = 0; i_2_1_s_195 < 4; ++i_2_1_s_195) {
    if (i_2_1_s_195 < 2) {
      Y_local[((i_2_1_s_195 * 16) + 65)] = (Y_local[((i_2_1_s_195 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_195 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_196 = 0; i_2_1_s_196 < 4; ++i_2_1_s_196) {
    Y_local[((i_2_1_s_196 * 16) + 2)] = (Y_local[((i_2_1_s_196 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_196] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_197 = 0; i_2_1_s_197 < 4; ++i_2_1_s_197) {
    if (i_2_1_s_197 < 2) {
      Y_local[((i_2_1_s_197 * 16) + 66)] = (Y_local[((i_2_1_s_197 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_197 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_198 = 0; i_2_1_s_198 < 4; ++i_2_1_s_198) {
    Y_local[((i_2_1_s_198 * 16) + 3)] = (Y_local[((i_2_1_s_198 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_198] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_199 = 0; i_2_1_s_199 < 4; ++i_2_1_s_199) {
    if (i_2_1_s_199 < 2) {
      Y_local[((i_2_1_s_199 * 16) + 67)] = (Y_local[((i_2_1_s_199 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_199 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_200 = 0; i_2_1_s_200 < 4; ++i_2_1_s_200) {
    Y_local[((i_2_1_s_200 * 16) + 4)] = (Y_local[((i_2_1_s_200 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_200] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_201 = 0; i_2_1_s_201 < 4; ++i_2_1_s_201) {
    if (i_2_1_s_201 < 2) {
      Y_local[((i_2_1_s_201 * 16) + 68)] = (Y_local[((i_2_1_s_201 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_201 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_202 = 0; i_2_1_s_202 < 4; ++i_2_1_s_202) {
    Y_local[((i_2_1_s_202 * 16) + 5)] = (Y_local[((i_2_1_s_202 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_202] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_203 = 0; i_2_1_s_203 < 4; ++i_2_1_s_203) {
    if (i_2_1_s_203 < 2) {
      Y_local[((i_2_1_s_203 * 16) + 69)] = (Y_local[((i_2_1_s_203 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_203 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_204 = 0; i_2_1_s_204 < 4; ++i_2_1_s_204) {
    Y_local[((i_2_1_s_204 * 16) + 6)] = (Y_local[((i_2_1_s_204 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_204] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_205 = 0; i_2_1_s_205 < 4; ++i_2_1_s_205) {
    if (i_2_1_s_205 < 2) {
      Y_local[((i_2_1_s_205 * 16) + 70)] = (Y_local[((i_2_1_s_205 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_205 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_206 = 0; i_2_1_s_206 < 4; ++i_2_1_s_206) {
    Y_local[((i_2_1_s_206 * 16) + 7)] = (Y_local[((i_2_1_s_206 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_206] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_207 = 0; i_2_1_s_207 < 4; ++i_2_1_s_207) {
    if (i_2_1_s_207 < 2) {
      Y_local[((i_2_1_s_207 * 16) + 71)] = (Y_local[((i_2_1_s_207 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_207 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_208 = 0; i_2_1_s_208 < 4; ++i_2_1_s_208) {
    Y_local[((i_2_1_s_208 * 16) + 8)] = (Y_local[((i_2_1_s_208 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_208] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_209 = 0; i_2_1_s_209 < 4; ++i_2_1_s_209) {
    if (i_2_1_s_209 < 2) {
      Y_local[((i_2_1_s_209 * 16) + 72)] = (Y_local[((i_2_1_s_209 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_209 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_210 = 0; i_2_1_s_210 < 4; ++i_2_1_s_210) {
    Y_local[((i_2_1_s_210 * 16) + 9)] = (Y_local[((i_2_1_s_210 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_210] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_211 = 0; i_2_1_s_211 < 4; ++i_2_1_s_211) {
    if (i_2_1_s_211 < 2) {
      Y_local[((i_2_1_s_211 * 16) + 73)] = (Y_local[((i_2_1_s_211 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_211 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_212 = 0; i_2_1_s_212 < 4; ++i_2_1_s_212) {
    Y_local[((i_2_1_s_212 * 16) + 10)] = (Y_local[((i_2_1_s_212 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_212] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_213 = 0; i_2_1_s_213 < 4; ++i_2_1_s_213) {
    if (i_2_1_s_213 < 2) {
      Y_local[((i_2_1_s_213 * 16) + 74)] = (Y_local[((i_2_1_s_213 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_213 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_214 = 0; i_2_1_s_214 < 4; ++i_2_1_s_214) {
    Y_local[((i_2_1_s_214 * 16) + 11)] = (Y_local[((i_2_1_s_214 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_214] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_215 = 0; i_2_1_s_215 < 4; ++i_2_1_s_215) {
    if (i_2_1_s_215 < 2) {
      Y_local[((i_2_1_s_215 * 16) + 75)] = (Y_local[((i_2_1_s_215 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_215 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_216 = 0; i_2_1_s_216 < 4; ++i_2_1_s_216) {
    Y_local[((i_2_1_s_216 * 16) + 12)] = (Y_local[((i_2_1_s_216 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_216] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_217 = 0; i_2_1_s_217 < 4; ++i_2_1_s_217) {
    if (i_2_1_s_217 < 2) {
      Y_local[((i_2_1_s_217 * 16) + 76)] = (Y_local[((i_2_1_s_217 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_217 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_218 = 0; i_2_1_s_218 < 4; ++i_2_1_s_218) {
    Y_local[((i_2_1_s_218 * 16) + 13)] = (Y_local[((i_2_1_s_218 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_218] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_219 = 0; i_2_1_s_219 < 4; ++i_2_1_s_219) {
    if (i_2_1_s_219 < 2) {
      Y_local[((i_2_1_s_219 * 16) + 77)] = (Y_local[((i_2_1_s_219 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_219 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_220 = 0; i_2_1_s_220 < 4; ++i_2_1_s_220) {
    Y_local[((i_2_1_s_220 * 16) + 14)] = (Y_local[((i_2_1_s_220 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_220] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_221 = 0; i_2_1_s_221 < 4; ++i_2_1_s_221) {
    if (i_2_1_s_221 < 2) {
      Y_local[((i_2_1_s_221 * 16) + 78)] = (Y_local[((i_2_1_s_221 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_221 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_222 = 0; i_2_1_s_222 < 4; ++i_2_1_s_222) {
    Y_local[((i_2_1_s_222 * 16) + 15)] = (Y_local[((i_2_1_s_222 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_222] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_223 = 0; i_2_1_s_223 < 4; ++i_2_1_s_223) {
    if (i_2_1_s_223 < 2) {
      Y_local[((i_2_1_s_223 * 16) + 79)] = (Y_local[((i_2_1_s_223 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_223 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_8 = 0; ax0_0_8 < 2; ++ax0_0_8) {
    for (int ax0_1_s_8 = 0; ax0_1_s_8 < 4; ++ax0_1_s_8) {
      if (((ax0_0_8 * 2) + (ax0_1_s_8 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_8 * 4) + ax0_1_s_8)] = ((float*)buf_dyn_shmem)[(((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_8 * 16)) + (ax0_1_s_8 * 4))];
      }
    }
  }
  for (int ax1_0_8 = 0; ax1_0_8 < 4; ++ax1_0_8) {
    *(float4*)(B_shared_dyn_local + (ax1_0_8 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_8 * 4)) >> 6) * 64) + ((ax1_0_8 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_8 >> 1)) & 7) * 4)) + 2880) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_224 = 0; i_2_1_s_224 < 4; ++i_2_1_s_224) {
    Y_local[(i_2_1_s_224 * 16)] = (Y_local[(i_2_1_s_224 * 16)] + (A_shared_dyn_local[(i_2_1_s_224 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_225 = 0; i_2_1_s_225 < 4; ++i_2_1_s_225) {
    if (i_2_1_s_225 < 2) {
      Y_local[((i_2_1_s_225 * 16) + 64)] = (Y_local[((i_2_1_s_225 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_225 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_226 = 0; i_2_1_s_226 < 4; ++i_2_1_s_226) {
    Y_local[((i_2_1_s_226 * 16) + 1)] = (Y_local[((i_2_1_s_226 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_226 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_227 = 0; i_2_1_s_227 < 4; ++i_2_1_s_227) {
    if (i_2_1_s_227 < 2) {
      Y_local[((i_2_1_s_227 * 16) + 65)] = (Y_local[((i_2_1_s_227 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_227 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_228 = 0; i_2_1_s_228 < 4; ++i_2_1_s_228) {
    Y_local[((i_2_1_s_228 * 16) + 2)] = (Y_local[((i_2_1_s_228 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_228 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_229 = 0; i_2_1_s_229 < 4; ++i_2_1_s_229) {
    if (i_2_1_s_229 < 2) {
      Y_local[((i_2_1_s_229 * 16) + 66)] = (Y_local[((i_2_1_s_229 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_229 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_230 = 0; i_2_1_s_230 < 4; ++i_2_1_s_230) {
    Y_local[((i_2_1_s_230 * 16) + 3)] = (Y_local[((i_2_1_s_230 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_230 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_231 = 0; i_2_1_s_231 < 4; ++i_2_1_s_231) {
    if (i_2_1_s_231 < 2) {
      Y_local[((i_2_1_s_231 * 16) + 67)] = (Y_local[((i_2_1_s_231 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_231 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_232 = 0; i_2_1_s_232 < 4; ++i_2_1_s_232) {
    Y_local[((i_2_1_s_232 * 16) + 4)] = (Y_local[((i_2_1_s_232 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_232 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_233 = 0; i_2_1_s_233 < 4; ++i_2_1_s_233) {
    if (i_2_1_s_233 < 2) {
      Y_local[((i_2_1_s_233 * 16) + 68)] = (Y_local[((i_2_1_s_233 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_233 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_234 = 0; i_2_1_s_234 < 4; ++i_2_1_s_234) {
    Y_local[((i_2_1_s_234 * 16) + 5)] = (Y_local[((i_2_1_s_234 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_234 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_235 = 0; i_2_1_s_235 < 4; ++i_2_1_s_235) {
    if (i_2_1_s_235 < 2) {
      Y_local[((i_2_1_s_235 * 16) + 69)] = (Y_local[((i_2_1_s_235 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_235 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_236 = 0; i_2_1_s_236 < 4; ++i_2_1_s_236) {
    Y_local[((i_2_1_s_236 * 16) + 6)] = (Y_local[((i_2_1_s_236 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_236 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_237 = 0; i_2_1_s_237 < 4; ++i_2_1_s_237) {
    if (i_2_1_s_237 < 2) {
      Y_local[((i_2_1_s_237 * 16) + 70)] = (Y_local[((i_2_1_s_237 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_237 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_238 = 0; i_2_1_s_238 < 4; ++i_2_1_s_238) {
    Y_local[((i_2_1_s_238 * 16) + 7)] = (Y_local[((i_2_1_s_238 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_238 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_239 = 0; i_2_1_s_239 < 4; ++i_2_1_s_239) {
    if (i_2_1_s_239 < 2) {
      Y_local[((i_2_1_s_239 * 16) + 71)] = (Y_local[((i_2_1_s_239 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_239 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_240 = 0; i_2_1_s_240 < 4; ++i_2_1_s_240) {
    Y_local[((i_2_1_s_240 * 16) + 8)] = (Y_local[((i_2_1_s_240 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_240 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_241 = 0; i_2_1_s_241 < 4; ++i_2_1_s_241) {
    if (i_2_1_s_241 < 2) {
      Y_local[((i_2_1_s_241 * 16) + 72)] = (Y_local[((i_2_1_s_241 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_241 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_242 = 0; i_2_1_s_242 < 4; ++i_2_1_s_242) {
    Y_local[((i_2_1_s_242 * 16) + 9)] = (Y_local[((i_2_1_s_242 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_242 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_243 = 0; i_2_1_s_243 < 4; ++i_2_1_s_243) {
    if (i_2_1_s_243 < 2) {
      Y_local[((i_2_1_s_243 * 16) + 73)] = (Y_local[((i_2_1_s_243 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_243 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_244 = 0; i_2_1_s_244 < 4; ++i_2_1_s_244) {
    Y_local[((i_2_1_s_244 * 16) + 10)] = (Y_local[((i_2_1_s_244 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_244 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_245 = 0; i_2_1_s_245 < 4; ++i_2_1_s_245) {
    if (i_2_1_s_245 < 2) {
      Y_local[((i_2_1_s_245 * 16) + 74)] = (Y_local[((i_2_1_s_245 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_245 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_246 = 0; i_2_1_s_246 < 4; ++i_2_1_s_246) {
    Y_local[((i_2_1_s_246 * 16) + 11)] = (Y_local[((i_2_1_s_246 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_246 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_247 = 0; i_2_1_s_247 < 4; ++i_2_1_s_247) {
    if (i_2_1_s_247 < 2) {
      Y_local[((i_2_1_s_247 * 16) + 75)] = (Y_local[((i_2_1_s_247 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_247 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_248 = 0; i_2_1_s_248 < 4; ++i_2_1_s_248) {
    Y_local[((i_2_1_s_248 * 16) + 12)] = (Y_local[((i_2_1_s_248 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_248 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_249 = 0; i_2_1_s_249 < 4; ++i_2_1_s_249) {
    if (i_2_1_s_249 < 2) {
      Y_local[((i_2_1_s_249 * 16) + 76)] = (Y_local[((i_2_1_s_249 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_249 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_250 = 0; i_2_1_s_250 < 4; ++i_2_1_s_250) {
    Y_local[((i_2_1_s_250 * 16) + 13)] = (Y_local[((i_2_1_s_250 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_250 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_251 = 0; i_2_1_s_251 < 4; ++i_2_1_s_251) {
    if (i_2_1_s_251 < 2) {
      Y_local[((i_2_1_s_251 * 16) + 77)] = (Y_local[((i_2_1_s_251 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_251 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_252 = 0; i_2_1_s_252 < 4; ++i_2_1_s_252) {
    Y_local[((i_2_1_s_252 * 16) + 14)] = (Y_local[((i_2_1_s_252 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_252 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_253 = 0; i_2_1_s_253 < 4; ++i_2_1_s_253) {
    if (i_2_1_s_253 < 2) {
      Y_local[((i_2_1_s_253 * 16) + 78)] = (Y_local[((i_2_1_s_253 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_253 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_254 = 0; i_2_1_s_254 < 4; ++i_2_1_s_254) {
    Y_local[((i_2_1_s_254 * 16) + 15)] = (Y_local[((i_2_1_s_254 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_254 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_255 = 0; i_2_1_s_255 < 4; ++i_2_1_s_255) {
    if (i_2_1_s_255 < 2) {
      Y_local[((i_2_1_s_255 * 16) + 79)] = (Y_local[((i_2_1_s_255 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_255 + 10)] * B_shared_dyn_local[31]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax0_0_9 = 0; ax0_0_9 < 2; ++ax0_0_9) {
    for (int ax0_1_s_9 = 0; ax0_1_s_9 < 4; ++ax0_1_s_9) {
      if (((ax0_0_9 * 2) + (ax0_1_s_9 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_9 * 4) + ax0_1_s_9) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_9 * 16)) + (ax0_1_s_9 * 4)) + 1)];
      }
    }
  }
  for (int ax1_0_9 = 0; ax1_0_9 < 4; ++ax1_0_9) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_9 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_9 * 4)) >> 6) * 64) + ((ax1_0_9 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_9 >> 1)) & 7) * 4)) + 3072) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_256 = 0; i_2_1_s_256 < 4; ++i_2_1_s_256) {
    Y_local[(i_2_1_s_256 * 16)] = (Y_local[(i_2_1_s_256 * 16)] + (A_shared_dyn_local[i_2_1_s_256] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_257 = 0; i_2_1_s_257 < 4; ++i_2_1_s_257) {
    if (i_2_1_s_257 < 2) {
      Y_local[((i_2_1_s_257 * 16) + 64)] = (Y_local[((i_2_1_s_257 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_257 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_258 = 0; i_2_1_s_258 < 4; ++i_2_1_s_258) {
    Y_local[((i_2_1_s_258 * 16) + 1)] = (Y_local[((i_2_1_s_258 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_258] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_259 = 0; i_2_1_s_259 < 4; ++i_2_1_s_259) {
    if (i_2_1_s_259 < 2) {
      Y_local[((i_2_1_s_259 * 16) + 65)] = (Y_local[((i_2_1_s_259 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_259 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_260 = 0; i_2_1_s_260 < 4; ++i_2_1_s_260) {
    Y_local[((i_2_1_s_260 * 16) + 2)] = (Y_local[((i_2_1_s_260 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_260] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_261 = 0; i_2_1_s_261 < 4; ++i_2_1_s_261) {
    if (i_2_1_s_261 < 2) {
      Y_local[((i_2_1_s_261 * 16) + 66)] = (Y_local[((i_2_1_s_261 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_261 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_262 = 0; i_2_1_s_262 < 4; ++i_2_1_s_262) {
    Y_local[((i_2_1_s_262 * 16) + 3)] = (Y_local[((i_2_1_s_262 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_262] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_263 = 0; i_2_1_s_263 < 4; ++i_2_1_s_263) {
    if (i_2_1_s_263 < 2) {
      Y_local[((i_2_1_s_263 * 16) + 67)] = (Y_local[((i_2_1_s_263 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_263 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_264 = 0; i_2_1_s_264 < 4; ++i_2_1_s_264) {
    Y_local[((i_2_1_s_264 * 16) + 4)] = (Y_local[((i_2_1_s_264 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_264] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_265 = 0; i_2_1_s_265 < 4; ++i_2_1_s_265) {
    if (i_2_1_s_265 < 2) {
      Y_local[((i_2_1_s_265 * 16) + 68)] = (Y_local[((i_2_1_s_265 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_265 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_266 = 0; i_2_1_s_266 < 4; ++i_2_1_s_266) {
    Y_local[((i_2_1_s_266 * 16) + 5)] = (Y_local[((i_2_1_s_266 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_266] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_267 = 0; i_2_1_s_267 < 4; ++i_2_1_s_267) {
    if (i_2_1_s_267 < 2) {
      Y_local[((i_2_1_s_267 * 16) + 69)] = (Y_local[((i_2_1_s_267 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_267 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_268 = 0; i_2_1_s_268 < 4; ++i_2_1_s_268) {
    Y_local[((i_2_1_s_268 * 16) + 6)] = (Y_local[((i_2_1_s_268 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_268] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_269 = 0; i_2_1_s_269 < 4; ++i_2_1_s_269) {
    if (i_2_1_s_269 < 2) {
      Y_local[((i_2_1_s_269 * 16) + 70)] = (Y_local[((i_2_1_s_269 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_269 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_270 = 0; i_2_1_s_270 < 4; ++i_2_1_s_270) {
    Y_local[((i_2_1_s_270 * 16) + 7)] = (Y_local[((i_2_1_s_270 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_270] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_271 = 0; i_2_1_s_271 < 4; ++i_2_1_s_271) {
    if (i_2_1_s_271 < 2) {
      Y_local[((i_2_1_s_271 * 16) + 71)] = (Y_local[((i_2_1_s_271 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_271 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_272 = 0; i_2_1_s_272 < 4; ++i_2_1_s_272) {
    Y_local[((i_2_1_s_272 * 16) + 8)] = (Y_local[((i_2_1_s_272 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_272] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_273 = 0; i_2_1_s_273 < 4; ++i_2_1_s_273) {
    if (i_2_1_s_273 < 2) {
      Y_local[((i_2_1_s_273 * 16) + 72)] = (Y_local[((i_2_1_s_273 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_273 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_274 = 0; i_2_1_s_274 < 4; ++i_2_1_s_274) {
    Y_local[((i_2_1_s_274 * 16) + 9)] = (Y_local[((i_2_1_s_274 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_274] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_275 = 0; i_2_1_s_275 < 4; ++i_2_1_s_275) {
    if (i_2_1_s_275 < 2) {
      Y_local[((i_2_1_s_275 * 16) + 73)] = (Y_local[((i_2_1_s_275 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_275 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_276 = 0; i_2_1_s_276 < 4; ++i_2_1_s_276) {
    Y_local[((i_2_1_s_276 * 16) + 10)] = (Y_local[((i_2_1_s_276 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_276] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_277 = 0; i_2_1_s_277 < 4; ++i_2_1_s_277) {
    if (i_2_1_s_277 < 2) {
      Y_local[((i_2_1_s_277 * 16) + 74)] = (Y_local[((i_2_1_s_277 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_277 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_278 = 0; i_2_1_s_278 < 4; ++i_2_1_s_278) {
    Y_local[((i_2_1_s_278 * 16) + 11)] = (Y_local[((i_2_1_s_278 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_278] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_279 = 0; i_2_1_s_279 < 4; ++i_2_1_s_279) {
    if (i_2_1_s_279 < 2) {
      Y_local[((i_2_1_s_279 * 16) + 75)] = (Y_local[((i_2_1_s_279 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_279 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_280 = 0; i_2_1_s_280 < 4; ++i_2_1_s_280) {
    Y_local[((i_2_1_s_280 * 16) + 12)] = (Y_local[((i_2_1_s_280 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_280] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_281 = 0; i_2_1_s_281 < 4; ++i_2_1_s_281) {
    if (i_2_1_s_281 < 2) {
      Y_local[((i_2_1_s_281 * 16) + 76)] = (Y_local[((i_2_1_s_281 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_281 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_282 = 0; i_2_1_s_282 < 4; ++i_2_1_s_282) {
    Y_local[((i_2_1_s_282 * 16) + 13)] = (Y_local[((i_2_1_s_282 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_282] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_283 = 0; i_2_1_s_283 < 4; ++i_2_1_s_283) {
    if (i_2_1_s_283 < 2) {
      Y_local[((i_2_1_s_283 * 16) + 77)] = (Y_local[((i_2_1_s_283 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_283 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_284 = 0; i_2_1_s_284 < 4; ++i_2_1_s_284) {
    Y_local[((i_2_1_s_284 * 16) + 14)] = (Y_local[((i_2_1_s_284 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_284] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_285 = 0; i_2_1_s_285 < 4; ++i_2_1_s_285) {
    if (i_2_1_s_285 < 2) {
      Y_local[((i_2_1_s_285 * 16) + 78)] = (Y_local[((i_2_1_s_285 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_285 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_286 = 0; i_2_1_s_286 < 4; ++i_2_1_s_286) {
    Y_local[((i_2_1_s_286 * 16) + 15)] = (Y_local[((i_2_1_s_286 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_286] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_287 = 0; i_2_1_s_287 < 4; ++i_2_1_s_287) {
    if (i_2_1_s_287 < 2) {
      Y_local[((i_2_1_s_287 * 16) + 79)] = (Y_local[((i_2_1_s_287 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_287 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_10 = 0; ax0_0_10 < 2; ++ax0_0_10) {
    for (int ax0_1_s_10 = 0; ax0_1_s_10 < 4; ++ax0_1_s_10) {
      if (((ax0_0_10 * 2) + (ax0_1_s_10 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_10 * 4) + ax0_1_s_10)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_10 * 16)) + (ax0_1_s_10 * 4)) + 2)];
      }
    }
  }
  for (int ax1_0_10 = 0; ax1_0_10 < 4; ++ax1_0_10) {
    *(float4*)(B_shared_dyn_local + (ax1_0_10 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_10 * 4)) >> 6) * 64) + ((ax1_0_10 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_10 >> 1)) & 7) * 4)) + 3264) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_288 = 0; i_2_1_s_288 < 4; ++i_2_1_s_288) {
    Y_local[(i_2_1_s_288 * 16)] = (Y_local[(i_2_1_s_288 * 16)] + (A_shared_dyn_local[(i_2_1_s_288 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_289 = 0; i_2_1_s_289 < 4; ++i_2_1_s_289) {
    if (i_2_1_s_289 < 2) {
      Y_local[((i_2_1_s_289 * 16) + 64)] = (Y_local[((i_2_1_s_289 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_289 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_290 = 0; i_2_1_s_290 < 4; ++i_2_1_s_290) {
    Y_local[((i_2_1_s_290 * 16) + 1)] = (Y_local[((i_2_1_s_290 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_290 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_291 = 0; i_2_1_s_291 < 4; ++i_2_1_s_291) {
    if (i_2_1_s_291 < 2) {
      Y_local[((i_2_1_s_291 * 16) + 65)] = (Y_local[((i_2_1_s_291 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_291 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_292 = 0; i_2_1_s_292 < 4; ++i_2_1_s_292) {
    Y_local[((i_2_1_s_292 * 16) + 2)] = (Y_local[((i_2_1_s_292 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_292 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_293 = 0; i_2_1_s_293 < 4; ++i_2_1_s_293) {
    if (i_2_1_s_293 < 2) {
      Y_local[((i_2_1_s_293 * 16) + 66)] = (Y_local[((i_2_1_s_293 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_293 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_294 = 0; i_2_1_s_294 < 4; ++i_2_1_s_294) {
    Y_local[((i_2_1_s_294 * 16) + 3)] = (Y_local[((i_2_1_s_294 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_294 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_295 = 0; i_2_1_s_295 < 4; ++i_2_1_s_295) {
    if (i_2_1_s_295 < 2) {
      Y_local[((i_2_1_s_295 * 16) + 67)] = (Y_local[((i_2_1_s_295 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_295 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_296 = 0; i_2_1_s_296 < 4; ++i_2_1_s_296) {
    Y_local[((i_2_1_s_296 * 16) + 4)] = (Y_local[((i_2_1_s_296 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_296 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_297 = 0; i_2_1_s_297 < 4; ++i_2_1_s_297) {
    if (i_2_1_s_297 < 2) {
      Y_local[((i_2_1_s_297 * 16) + 68)] = (Y_local[((i_2_1_s_297 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_297 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_298 = 0; i_2_1_s_298 < 4; ++i_2_1_s_298) {
    Y_local[((i_2_1_s_298 * 16) + 5)] = (Y_local[((i_2_1_s_298 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_298 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_299 = 0; i_2_1_s_299 < 4; ++i_2_1_s_299) {
    if (i_2_1_s_299 < 2) {
      Y_local[((i_2_1_s_299 * 16) + 69)] = (Y_local[((i_2_1_s_299 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_299 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_300 = 0; i_2_1_s_300 < 4; ++i_2_1_s_300) {
    Y_local[((i_2_1_s_300 * 16) + 6)] = (Y_local[((i_2_1_s_300 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_300 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_301 = 0; i_2_1_s_301 < 4; ++i_2_1_s_301) {
    if (i_2_1_s_301 < 2) {
      Y_local[((i_2_1_s_301 * 16) + 70)] = (Y_local[((i_2_1_s_301 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_301 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_302 = 0; i_2_1_s_302 < 4; ++i_2_1_s_302) {
    Y_local[((i_2_1_s_302 * 16) + 7)] = (Y_local[((i_2_1_s_302 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_302 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_303 = 0; i_2_1_s_303 < 4; ++i_2_1_s_303) {
    if (i_2_1_s_303 < 2) {
      Y_local[((i_2_1_s_303 * 16) + 71)] = (Y_local[((i_2_1_s_303 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_303 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_304 = 0; i_2_1_s_304 < 4; ++i_2_1_s_304) {
    Y_local[((i_2_1_s_304 * 16) + 8)] = (Y_local[((i_2_1_s_304 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_304 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_305 = 0; i_2_1_s_305 < 4; ++i_2_1_s_305) {
    if (i_2_1_s_305 < 2) {
      Y_local[((i_2_1_s_305 * 16) + 72)] = (Y_local[((i_2_1_s_305 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_305 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_306 = 0; i_2_1_s_306 < 4; ++i_2_1_s_306) {
    Y_local[((i_2_1_s_306 * 16) + 9)] = (Y_local[((i_2_1_s_306 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_306 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_307 = 0; i_2_1_s_307 < 4; ++i_2_1_s_307) {
    if (i_2_1_s_307 < 2) {
      Y_local[((i_2_1_s_307 * 16) + 73)] = (Y_local[((i_2_1_s_307 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_307 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_308 = 0; i_2_1_s_308 < 4; ++i_2_1_s_308) {
    Y_local[((i_2_1_s_308 * 16) + 10)] = (Y_local[((i_2_1_s_308 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_308 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_309 = 0; i_2_1_s_309 < 4; ++i_2_1_s_309) {
    if (i_2_1_s_309 < 2) {
      Y_local[((i_2_1_s_309 * 16) + 74)] = (Y_local[((i_2_1_s_309 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_309 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_310 = 0; i_2_1_s_310 < 4; ++i_2_1_s_310) {
    Y_local[((i_2_1_s_310 * 16) + 11)] = (Y_local[((i_2_1_s_310 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_310 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_311 = 0; i_2_1_s_311 < 4; ++i_2_1_s_311) {
    if (i_2_1_s_311 < 2) {
      Y_local[((i_2_1_s_311 * 16) + 75)] = (Y_local[((i_2_1_s_311 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_311 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_312 = 0; i_2_1_s_312 < 4; ++i_2_1_s_312) {
    Y_local[((i_2_1_s_312 * 16) + 12)] = (Y_local[((i_2_1_s_312 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_312 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_313 = 0; i_2_1_s_313 < 4; ++i_2_1_s_313) {
    if (i_2_1_s_313 < 2) {
      Y_local[((i_2_1_s_313 * 16) + 76)] = (Y_local[((i_2_1_s_313 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_313 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_314 = 0; i_2_1_s_314 < 4; ++i_2_1_s_314) {
    Y_local[((i_2_1_s_314 * 16) + 13)] = (Y_local[((i_2_1_s_314 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_314 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_315 = 0; i_2_1_s_315 < 4; ++i_2_1_s_315) {
    if (i_2_1_s_315 < 2) {
      Y_local[((i_2_1_s_315 * 16) + 77)] = (Y_local[((i_2_1_s_315 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_315 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_316 = 0; i_2_1_s_316 < 4; ++i_2_1_s_316) {
    Y_local[((i_2_1_s_316 * 16) + 14)] = (Y_local[((i_2_1_s_316 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_316 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_317 = 0; i_2_1_s_317 < 4; ++i_2_1_s_317) {
    if (i_2_1_s_317 < 2) {
      Y_local[((i_2_1_s_317 * 16) + 78)] = (Y_local[((i_2_1_s_317 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_317 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_318 = 0; i_2_1_s_318 < 4; ++i_2_1_s_318) {
    Y_local[((i_2_1_s_318 * 16) + 15)] = (Y_local[((i_2_1_s_318 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_318 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_319 = 0; i_2_1_s_319 < 4; ++i_2_1_s_319) {
    if (i_2_1_s_319 < 2) {
      Y_local[((i_2_1_s_319 * 16) + 79)] = (Y_local[((i_2_1_s_319 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_319 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax0_0_11 = 0; ax0_0_11 < 2; ++ax0_0_11) {
    for (int ax0_1_s_11 = 0; ax0_1_s_11 < 4; ++ax0_1_s_11) {
      if (((ax0_0_11 * 2) + (ax0_1_s_11 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_11 * 4) + ax0_1_s_11) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_11 * 16)) + (ax0_1_s_11 * 4)) + 3)];
      }
    }
  }
  for (int ax1_0_11 = 0; ax1_0_11 < 4; ++ax1_0_11) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_11 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_11 * 4)) >> 6) * 64) + ((ax1_0_11 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_11 >> 1)) & 7) * 4)) + 3456) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_320 = 0; i_2_1_s_320 < 4; ++i_2_1_s_320) {
    Y_local[(i_2_1_s_320 * 16)] = (Y_local[(i_2_1_s_320 * 16)] + (A_shared_dyn_local[i_2_1_s_320] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_321 = 0; i_2_1_s_321 < 4; ++i_2_1_s_321) {
    if (i_2_1_s_321 < 2) {
      Y_local[((i_2_1_s_321 * 16) + 64)] = (Y_local[((i_2_1_s_321 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_321 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_322 = 0; i_2_1_s_322 < 4; ++i_2_1_s_322) {
    Y_local[((i_2_1_s_322 * 16) + 1)] = (Y_local[((i_2_1_s_322 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_322] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_323 = 0; i_2_1_s_323 < 4; ++i_2_1_s_323) {
    if (i_2_1_s_323 < 2) {
      Y_local[((i_2_1_s_323 * 16) + 65)] = (Y_local[((i_2_1_s_323 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_323 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_324 = 0; i_2_1_s_324 < 4; ++i_2_1_s_324) {
    Y_local[((i_2_1_s_324 * 16) + 2)] = (Y_local[((i_2_1_s_324 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_324] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_325 = 0; i_2_1_s_325 < 4; ++i_2_1_s_325) {
    if (i_2_1_s_325 < 2) {
      Y_local[((i_2_1_s_325 * 16) + 66)] = (Y_local[((i_2_1_s_325 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_325 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_326 = 0; i_2_1_s_326 < 4; ++i_2_1_s_326) {
    Y_local[((i_2_1_s_326 * 16) + 3)] = (Y_local[((i_2_1_s_326 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_326] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_327 = 0; i_2_1_s_327 < 4; ++i_2_1_s_327) {
    if (i_2_1_s_327 < 2) {
      Y_local[((i_2_1_s_327 * 16) + 67)] = (Y_local[((i_2_1_s_327 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_327 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_328 = 0; i_2_1_s_328 < 4; ++i_2_1_s_328) {
    Y_local[((i_2_1_s_328 * 16) + 4)] = (Y_local[((i_2_1_s_328 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_328] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_329 = 0; i_2_1_s_329 < 4; ++i_2_1_s_329) {
    if (i_2_1_s_329 < 2) {
      Y_local[((i_2_1_s_329 * 16) + 68)] = (Y_local[((i_2_1_s_329 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_329 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_330 = 0; i_2_1_s_330 < 4; ++i_2_1_s_330) {
    Y_local[((i_2_1_s_330 * 16) + 5)] = (Y_local[((i_2_1_s_330 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_330] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_331 = 0; i_2_1_s_331 < 4; ++i_2_1_s_331) {
    if (i_2_1_s_331 < 2) {
      Y_local[((i_2_1_s_331 * 16) + 69)] = (Y_local[((i_2_1_s_331 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_331 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_332 = 0; i_2_1_s_332 < 4; ++i_2_1_s_332) {
    Y_local[((i_2_1_s_332 * 16) + 6)] = (Y_local[((i_2_1_s_332 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_332] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_333 = 0; i_2_1_s_333 < 4; ++i_2_1_s_333) {
    if (i_2_1_s_333 < 2) {
      Y_local[((i_2_1_s_333 * 16) + 70)] = (Y_local[((i_2_1_s_333 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_333 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_334 = 0; i_2_1_s_334 < 4; ++i_2_1_s_334) {
    Y_local[((i_2_1_s_334 * 16) + 7)] = (Y_local[((i_2_1_s_334 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_334] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_335 = 0; i_2_1_s_335 < 4; ++i_2_1_s_335) {
    if (i_2_1_s_335 < 2) {
      Y_local[((i_2_1_s_335 * 16) + 71)] = (Y_local[((i_2_1_s_335 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_335 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_336 = 0; i_2_1_s_336 < 4; ++i_2_1_s_336) {
    Y_local[((i_2_1_s_336 * 16) + 8)] = (Y_local[((i_2_1_s_336 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_336] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_337 = 0; i_2_1_s_337 < 4; ++i_2_1_s_337) {
    if (i_2_1_s_337 < 2) {
      Y_local[((i_2_1_s_337 * 16) + 72)] = (Y_local[((i_2_1_s_337 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_337 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_338 = 0; i_2_1_s_338 < 4; ++i_2_1_s_338) {
    Y_local[((i_2_1_s_338 * 16) + 9)] = (Y_local[((i_2_1_s_338 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_338] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_339 = 0; i_2_1_s_339 < 4; ++i_2_1_s_339) {
    if (i_2_1_s_339 < 2) {
      Y_local[((i_2_1_s_339 * 16) + 73)] = (Y_local[((i_2_1_s_339 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_339 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_340 = 0; i_2_1_s_340 < 4; ++i_2_1_s_340) {
    Y_local[((i_2_1_s_340 * 16) + 10)] = (Y_local[((i_2_1_s_340 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_340] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_341 = 0; i_2_1_s_341 < 4; ++i_2_1_s_341) {
    if (i_2_1_s_341 < 2) {
      Y_local[((i_2_1_s_341 * 16) + 74)] = (Y_local[((i_2_1_s_341 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_341 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_342 = 0; i_2_1_s_342 < 4; ++i_2_1_s_342) {
    Y_local[((i_2_1_s_342 * 16) + 11)] = (Y_local[((i_2_1_s_342 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_342] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_343 = 0; i_2_1_s_343 < 4; ++i_2_1_s_343) {
    if (i_2_1_s_343 < 2) {
      Y_local[((i_2_1_s_343 * 16) + 75)] = (Y_local[((i_2_1_s_343 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_343 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_344 = 0; i_2_1_s_344 < 4; ++i_2_1_s_344) {
    Y_local[((i_2_1_s_344 * 16) + 12)] = (Y_local[((i_2_1_s_344 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_344] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_345 = 0; i_2_1_s_345 < 4; ++i_2_1_s_345) {
    if (i_2_1_s_345 < 2) {
      Y_local[((i_2_1_s_345 * 16) + 76)] = (Y_local[((i_2_1_s_345 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_345 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_346 = 0; i_2_1_s_346 < 4; ++i_2_1_s_346) {
    Y_local[((i_2_1_s_346 * 16) + 13)] = (Y_local[((i_2_1_s_346 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_346] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_347 = 0; i_2_1_s_347 < 4; ++i_2_1_s_347) {
    if (i_2_1_s_347 < 2) {
      Y_local[((i_2_1_s_347 * 16) + 77)] = (Y_local[((i_2_1_s_347 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_347 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_348 = 0; i_2_1_s_348 < 4; ++i_2_1_s_348) {
    Y_local[((i_2_1_s_348 * 16) + 14)] = (Y_local[((i_2_1_s_348 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_348] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_349 = 0; i_2_1_s_349 < 4; ++i_2_1_s_349) {
    if (i_2_1_s_349 < 2) {
      Y_local[((i_2_1_s_349 * 16) + 78)] = (Y_local[((i_2_1_s_349 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_349 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_350 = 0; i_2_1_s_350 < 4; ++i_2_1_s_350) {
    Y_local[((i_2_1_s_350 * 16) + 15)] = (Y_local[((i_2_1_s_350 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_350] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_351 = 0; i_2_1_s_351 < 4; ++i_2_1_s_351) {
    if (i_2_1_s_351 < 2) {
      Y_local[((i_2_1_s_351 * 16) + 79)] = (Y_local[((i_2_1_s_351 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_351 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_12 = 0; ax0_0_12 < 2; ++ax0_0_12) {
    for (int ax0_1_s_12 = 0; ax0_1_s_12 < 4; ++ax0_1_s_12) {
      if (((ax0_0_12 * 2) + (ax0_1_s_12 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_12 * 4) + ax0_1_s_12)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_12 * 16)) + (ax0_1_s_12 * 4)) + 576)];
      }
    }
  }
  for (int ax1_0_12 = 0; ax1_0_12 < 4; ++ax1_0_12) {
    *(float4*)(B_shared_dyn_local + (ax1_0_12 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_12 * 4)) >> 6) * 64) + ((ax1_0_12 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_12 >> 1)) & 7) * 4)) + 3648) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_352 = 0; i_2_1_s_352 < 4; ++i_2_1_s_352) {
    Y_local[(i_2_1_s_352 * 16)] = (Y_local[(i_2_1_s_352 * 16)] + (A_shared_dyn_local[(i_2_1_s_352 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_353 = 0; i_2_1_s_353 < 4; ++i_2_1_s_353) {
    if (i_2_1_s_353 < 2) {
      Y_local[((i_2_1_s_353 * 16) + 64)] = (Y_local[((i_2_1_s_353 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_353 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_354 = 0; i_2_1_s_354 < 4; ++i_2_1_s_354) {
    Y_local[((i_2_1_s_354 * 16) + 1)] = (Y_local[((i_2_1_s_354 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_354 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_355 = 0; i_2_1_s_355 < 4; ++i_2_1_s_355) {
    if (i_2_1_s_355 < 2) {
      Y_local[((i_2_1_s_355 * 16) + 65)] = (Y_local[((i_2_1_s_355 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_355 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_356 = 0; i_2_1_s_356 < 4; ++i_2_1_s_356) {
    Y_local[((i_2_1_s_356 * 16) + 2)] = (Y_local[((i_2_1_s_356 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_356 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_357 = 0; i_2_1_s_357 < 4; ++i_2_1_s_357) {
    if (i_2_1_s_357 < 2) {
      Y_local[((i_2_1_s_357 * 16) + 66)] = (Y_local[((i_2_1_s_357 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_357 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_358 = 0; i_2_1_s_358 < 4; ++i_2_1_s_358) {
    Y_local[((i_2_1_s_358 * 16) + 3)] = (Y_local[((i_2_1_s_358 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_358 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_359 = 0; i_2_1_s_359 < 4; ++i_2_1_s_359) {
    if (i_2_1_s_359 < 2) {
      Y_local[((i_2_1_s_359 * 16) + 67)] = (Y_local[((i_2_1_s_359 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_359 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_360 = 0; i_2_1_s_360 < 4; ++i_2_1_s_360) {
    Y_local[((i_2_1_s_360 * 16) + 4)] = (Y_local[((i_2_1_s_360 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_360 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_361 = 0; i_2_1_s_361 < 4; ++i_2_1_s_361) {
    if (i_2_1_s_361 < 2) {
      Y_local[((i_2_1_s_361 * 16) + 68)] = (Y_local[((i_2_1_s_361 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_361 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_362 = 0; i_2_1_s_362 < 4; ++i_2_1_s_362) {
    Y_local[((i_2_1_s_362 * 16) + 5)] = (Y_local[((i_2_1_s_362 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_362 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_363 = 0; i_2_1_s_363 < 4; ++i_2_1_s_363) {
    if (i_2_1_s_363 < 2) {
      Y_local[((i_2_1_s_363 * 16) + 69)] = (Y_local[((i_2_1_s_363 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_363 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_364 = 0; i_2_1_s_364 < 4; ++i_2_1_s_364) {
    Y_local[((i_2_1_s_364 * 16) + 6)] = (Y_local[((i_2_1_s_364 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_364 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_365 = 0; i_2_1_s_365 < 4; ++i_2_1_s_365) {
    if (i_2_1_s_365 < 2) {
      Y_local[((i_2_1_s_365 * 16) + 70)] = (Y_local[((i_2_1_s_365 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_365 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_366 = 0; i_2_1_s_366 < 4; ++i_2_1_s_366) {
    Y_local[((i_2_1_s_366 * 16) + 7)] = (Y_local[((i_2_1_s_366 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_366 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_367 = 0; i_2_1_s_367 < 4; ++i_2_1_s_367) {
    if (i_2_1_s_367 < 2) {
      Y_local[((i_2_1_s_367 * 16) + 71)] = (Y_local[((i_2_1_s_367 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_367 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_368 = 0; i_2_1_s_368 < 4; ++i_2_1_s_368) {
    Y_local[((i_2_1_s_368 * 16) + 8)] = (Y_local[((i_2_1_s_368 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_368 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_369 = 0; i_2_1_s_369 < 4; ++i_2_1_s_369) {
    if (i_2_1_s_369 < 2) {
      Y_local[((i_2_1_s_369 * 16) + 72)] = (Y_local[((i_2_1_s_369 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_369 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_370 = 0; i_2_1_s_370 < 4; ++i_2_1_s_370) {
    Y_local[((i_2_1_s_370 * 16) + 9)] = (Y_local[((i_2_1_s_370 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_370 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_371 = 0; i_2_1_s_371 < 4; ++i_2_1_s_371) {
    if (i_2_1_s_371 < 2) {
      Y_local[((i_2_1_s_371 * 16) + 73)] = (Y_local[((i_2_1_s_371 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_371 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_372 = 0; i_2_1_s_372 < 4; ++i_2_1_s_372) {
    Y_local[((i_2_1_s_372 * 16) + 10)] = (Y_local[((i_2_1_s_372 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_372 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_373 = 0; i_2_1_s_373 < 4; ++i_2_1_s_373) {
    if (i_2_1_s_373 < 2) {
      Y_local[((i_2_1_s_373 * 16) + 74)] = (Y_local[((i_2_1_s_373 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_373 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_374 = 0; i_2_1_s_374 < 4; ++i_2_1_s_374) {
    Y_local[((i_2_1_s_374 * 16) + 11)] = (Y_local[((i_2_1_s_374 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_374 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_375 = 0; i_2_1_s_375 < 4; ++i_2_1_s_375) {
    if (i_2_1_s_375 < 2) {
      Y_local[((i_2_1_s_375 * 16) + 75)] = (Y_local[((i_2_1_s_375 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_375 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_376 = 0; i_2_1_s_376 < 4; ++i_2_1_s_376) {
    Y_local[((i_2_1_s_376 * 16) + 12)] = (Y_local[((i_2_1_s_376 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_376 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_377 = 0; i_2_1_s_377 < 4; ++i_2_1_s_377) {
    if (i_2_1_s_377 < 2) {
      Y_local[((i_2_1_s_377 * 16) + 76)] = (Y_local[((i_2_1_s_377 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_377 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_378 = 0; i_2_1_s_378 < 4; ++i_2_1_s_378) {
    Y_local[((i_2_1_s_378 * 16) + 13)] = (Y_local[((i_2_1_s_378 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_378 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_379 = 0; i_2_1_s_379 < 4; ++i_2_1_s_379) {
    if (i_2_1_s_379 < 2) {
      Y_local[((i_2_1_s_379 * 16) + 77)] = (Y_local[((i_2_1_s_379 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_379 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_380 = 0; i_2_1_s_380 < 4; ++i_2_1_s_380) {
    Y_local[((i_2_1_s_380 * 16) + 14)] = (Y_local[((i_2_1_s_380 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_380 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_381 = 0; i_2_1_s_381 < 4; ++i_2_1_s_381) {
    if (i_2_1_s_381 < 2) {
      Y_local[((i_2_1_s_381 * 16) + 78)] = (Y_local[((i_2_1_s_381 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_381 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_382 = 0; i_2_1_s_382 < 4; ++i_2_1_s_382) {
    Y_local[((i_2_1_s_382 * 16) + 15)] = (Y_local[((i_2_1_s_382 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_382 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_383 = 0; i_2_1_s_383 < 4; ++i_2_1_s_383) {
    if (i_2_1_s_383 < 2) {
      Y_local[((i_2_1_s_383 * 16) + 79)] = (Y_local[((i_2_1_s_383 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_383 + 10)] * B_shared_dyn_local[31]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax0_0_13 = 0; ax0_0_13 < 2; ++ax0_0_13) {
    for (int ax0_1_s_13 = 0; ax0_1_s_13 < 4; ++ax0_1_s_13) {
      if (((ax0_0_13 * 2) + (ax0_1_s_13 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_13 * 4) + ax0_1_s_13) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_13 * 16)) + (ax0_1_s_13 * 4)) + 577)];
      }
    }
  }
  for (int ax1_0_13 = 0; ax1_0_13 < 4; ++ax1_0_13) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_13 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_13 * 4)) >> 6) * 64) + ((ax1_0_13 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_13 >> 1)) & 7) * 4)) + 3840) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_384 = 0; i_2_1_s_384 < 4; ++i_2_1_s_384) {
    Y_local[(i_2_1_s_384 * 16)] = (Y_local[(i_2_1_s_384 * 16)] + (A_shared_dyn_local[i_2_1_s_384] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_385 = 0; i_2_1_s_385 < 4; ++i_2_1_s_385) {
    if (i_2_1_s_385 < 2) {
      Y_local[((i_2_1_s_385 * 16) + 64)] = (Y_local[((i_2_1_s_385 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_385 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_386 = 0; i_2_1_s_386 < 4; ++i_2_1_s_386) {
    Y_local[((i_2_1_s_386 * 16) + 1)] = (Y_local[((i_2_1_s_386 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_386] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_387 = 0; i_2_1_s_387 < 4; ++i_2_1_s_387) {
    if (i_2_1_s_387 < 2) {
      Y_local[((i_2_1_s_387 * 16) + 65)] = (Y_local[((i_2_1_s_387 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_387 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_388 = 0; i_2_1_s_388 < 4; ++i_2_1_s_388) {
    Y_local[((i_2_1_s_388 * 16) + 2)] = (Y_local[((i_2_1_s_388 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_388] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_389 = 0; i_2_1_s_389 < 4; ++i_2_1_s_389) {
    if (i_2_1_s_389 < 2) {
      Y_local[((i_2_1_s_389 * 16) + 66)] = (Y_local[((i_2_1_s_389 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_389 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_390 = 0; i_2_1_s_390 < 4; ++i_2_1_s_390) {
    Y_local[((i_2_1_s_390 * 16) + 3)] = (Y_local[((i_2_1_s_390 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_390] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_391 = 0; i_2_1_s_391 < 4; ++i_2_1_s_391) {
    if (i_2_1_s_391 < 2) {
      Y_local[((i_2_1_s_391 * 16) + 67)] = (Y_local[((i_2_1_s_391 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_391 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_392 = 0; i_2_1_s_392 < 4; ++i_2_1_s_392) {
    Y_local[((i_2_1_s_392 * 16) + 4)] = (Y_local[((i_2_1_s_392 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_392] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_393 = 0; i_2_1_s_393 < 4; ++i_2_1_s_393) {
    if (i_2_1_s_393 < 2) {
      Y_local[((i_2_1_s_393 * 16) + 68)] = (Y_local[((i_2_1_s_393 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_393 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_394 = 0; i_2_1_s_394 < 4; ++i_2_1_s_394) {
    Y_local[((i_2_1_s_394 * 16) + 5)] = (Y_local[((i_2_1_s_394 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_394] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_395 = 0; i_2_1_s_395 < 4; ++i_2_1_s_395) {
    if (i_2_1_s_395 < 2) {
      Y_local[((i_2_1_s_395 * 16) + 69)] = (Y_local[((i_2_1_s_395 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_395 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_396 = 0; i_2_1_s_396 < 4; ++i_2_1_s_396) {
    Y_local[((i_2_1_s_396 * 16) + 6)] = (Y_local[((i_2_1_s_396 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_396] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_397 = 0; i_2_1_s_397 < 4; ++i_2_1_s_397) {
    if (i_2_1_s_397 < 2) {
      Y_local[((i_2_1_s_397 * 16) + 70)] = (Y_local[((i_2_1_s_397 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_397 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_398 = 0; i_2_1_s_398 < 4; ++i_2_1_s_398) {
    Y_local[((i_2_1_s_398 * 16) + 7)] = (Y_local[((i_2_1_s_398 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_398] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_399 = 0; i_2_1_s_399 < 4; ++i_2_1_s_399) {
    if (i_2_1_s_399 < 2) {
      Y_local[((i_2_1_s_399 * 16) + 71)] = (Y_local[((i_2_1_s_399 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_399 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_400 = 0; i_2_1_s_400 < 4; ++i_2_1_s_400) {
    Y_local[((i_2_1_s_400 * 16) + 8)] = (Y_local[((i_2_1_s_400 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_400] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_401 = 0; i_2_1_s_401 < 4; ++i_2_1_s_401) {
    if (i_2_1_s_401 < 2) {
      Y_local[((i_2_1_s_401 * 16) + 72)] = (Y_local[((i_2_1_s_401 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_401 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_402 = 0; i_2_1_s_402 < 4; ++i_2_1_s_402) {
    Y_local[((i_2_1_s_402 * 16) + 9)] = (Y_local[((i_2_1_s_402 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_402] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_403 = 0; i_2_1_s_403 < 4; ++i_2_1_s_403) {
    if (i_2_1_s_403 < 2) {
      Y_local[((i_2_1_s_403 * 16) + 73)] = (Y_local[((i_2_1_s_403 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_403 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_404 = 0; i_2_1_s_404 < 4; ++i_2_1_s_404) {
    Y_local[((i_2_1_s_404 * 16) + 10)] = (Y_local[((i_2_1_s_404 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_404] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_405 = 0; i_2_1_s_405 < 4; ++i_2_1_s_405) {
    if (i_2_1_s_405 < 2) {
      Y_local[((i_2_1_s_405 * 16) + 74)] = (Y_local[((i_2_1_s_405 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_405 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_406 = 0; i_2_1_s_406 < 4; ++i_2_1_s_406) {
    Y_local[((i_2_1_s_406 * 16) + 11)] = (Y_local[((i_2_1_s_406 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_406] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_407 = 0; i_2_1_s_407 < 4; ++i_2_1_s_407) {
    if (i_2_1_s_407 < 2) {
      Y_local[((i_2_1_s_407 * 16) + 75)] = (Y_local[((i_2_1_s_407 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_407 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_408 = 0; i_2_1_s_408 < 4; ++i_2_1_s_408) {
    Y_local[((i_2_1_s_408 * 16) + 12)] = (Y_local[((i_2_1_s_408 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_408] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_409 = 0; i_2_1_s_409 < 4; ++i_2_1_s_409) {
    if (i_2_1_s_409 < 2) {
      Y_local[((i_2_1_s_409 * 16) + 76)] = (Y_local[((i_2_1_s_409 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_409 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_410 = 0; i_2_1_s_410 < 4; ++i_2_1_s_410) {
    Y_local[((i_2_1_s_410 * 16) + 13)] = (Y_local[((i_2_1_s_410 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_410] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_411 = 0; i_2_1_s_411 < 4; ++i_2_1_s_411) {
    if (i_2_1_s_411 < 2) {
      Y_local[((i_2_1_s_411 * 16) + 77)] = (Y_local[((i_2_1_s_411 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_411 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_412 = 0; i_2_1_s_412 < 4; ++i_2_1_s_412) {
    Y_local[((i_2_1_s_412 * 16) + 14)] = (Y_local[((i_2_1_s_412 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_412] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_413 = 0; i_2_1_s_413 < 4; ++i_2_1_s_413) {
    if (i_2_1_s_413 < 2) {
      Y_local[((i_2_1_s_413 * 16) + 78)] = (Y_local[((i_2_1_s_413 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_413 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_414 = 0; i_2_1_s_414 < 4; ++i_2_1_s_414) {
    Y_local[((i_2_1_s_414 * 16) + 15)] = (Y_local[((i_2_1_s_414 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_414] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_415 = 0; i_2_1_s_415 < 4; ++i_2_1_s_415) {
    if (i_2_1_s_415 < 2) {
      Y_local[((i_2_1_s_415 * 16) + 79)] = (Y_local[((i_2_1_s_415 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_415 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_14 = 0; ax0_0_14 < 2; ++ax0_0_14) {
    for (int ax0_1_s_14 = 0; ax0_1_s_14 < 4; ++ax0_1_s_14) {
      if (((ax0_0_14 * 2) + (ax0_1_s_14 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_14 * 4) + ax0_1_s_14)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_14 * 16)) + (ax0_1_s_14 * 4)) + 578)];
      }
    }
  }
  for (int ax1_0_14 = 0; ax1_0_14 < 4; ++ax1_0_14) {
    *(float4*)(B_shared_dyn_local + (ax1_0_14 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_14 * 4)) >> 6) * 64) + ((ax1_0_14 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_14 >> 1)) & 7) * 4)) + 4032) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_416 = 0; i_2_1_s_416 < 4; ++i_2_1_s_416) {
    Y_local[(i_2_1_s_416 * 16)] = (Y_local[(i_2_1_s_416 * 16)] + (A_shared_dyn_local[(i_2_1_s_416 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_417 = 0; i_2_1_s_417 < 4; ++i_2_1_s_417) {
    if (i_2_1_s_417 < 2) {
      Y_local[((i_2_1_s_417 * 16) + 64)] = (Y_local[((i_2_1_s_417 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_417 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_418 = 0; i_2_1_s_418 < 4; ++i_2_1_s_418) {
    Y_local[((i_2_1_s_418 * 16) + 1)] = (Y_local[((i_2_1_s_418 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_418 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_419 = 0; i_2_1_s_419 < 4; ++i_2_1_s_419) {
    if (i_2_1_s_419 < 2) {
      Y_local[((i_2_1_s_419 * 16) + 65)] = (Y_local[((i_2_1_s_419 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_419 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_420 = 0; i_2_1_s_420 < 4; ++i_2_1_s_420) {
    Y_local[((i_2_1_s_420 * 16) + 2)] = (Y_local[((i_2_1_s_420 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_420 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_421 = 0; i_2_1_s_421 < 4; ++i_2_1_s_421) {
    if (i_2_1_s_421 < 2) {
      Y_local[((i_2_1_s_421 * 16) + 66)] = (Y_local[((i_2_1_s_421 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_421 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_422 = 0; i_2_1_s_422 < 4; ++i_2_1_s_422) {
    Y_local[((i_2_1_s_422 * 16) + 3)] = (Y_local[((i_2_1_s_422 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_422 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_423 = 0; i_2_1_s_423 < 4; ++i_2_1_s_423) {
    if (i_2_1_s_423 < 2) {
      Y_local[((i_2_1_s_423 * 16) + 67)] = (Y_local[((i_2_1_s_423 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_423 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_424 = 0; i_2_1_s_424 < 4; ++i_2_1_s_424) {
    Y_local[((i_2_1_s_424 * 16) + 4)] = (Y_local[((i_2_1_s_424 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_424 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_425 = 0; i_2_1_s_425 < 4; ++i_2_1_s_425) {
    if (i_2_1_s_425 < 2) {
      Y_local[((i_2_1_s_425 * 16) + 68)] = (Y_local[((i_2_1_s_425 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_425 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_426 = 0; i_2_1_s_426 < 4; ++i_2_1_s_426) {
    Y_local[((i_2_1_s_426 * 16) + 5)] = (Y_local[((i_2_1_s_426 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_426 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_427 = 0; i_2_1_s_427 < 4; ++i_2_1_s_427) {
    if (i_2_1_s_427 < 2) {
      Y_local[((i_2_1_s_427 * 16) + 69)] = (Y_local[((i_2_1_s_427 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_427 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_428 = 0; i_2_1_s_428 < 4; ++i_2_1_s_428) {
    Y_local[((i_2_1_s_428 * 16) + 6)] = (Y_local[((i_2_1_s_428 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_428 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_429 = 0; i_2_1_s_429 < 4; ++i_2_1_s_429) {
    if (i_2_1_s_429 < 2) {
      Y_local[((i_2_1_s_429 * 16) + 70)] = (Y_local[((i_2_1_s_429 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_429 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_430 = 0; i_2_1_s_430 < 4; ++i_2_1_s_430) {
    Y_local[((i_2_1_s_430 * 16) + 7)] = (Y_local[((i_2_1_s_430 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_430 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_431 = 0; i_2_1_s_431 < 4; ++i_2_1_s_431) {
    if (i_2_1_s_431 < 2) {
      Y_local[((i_2_1_s_431 * 16) + 71)] = (Y_local[((i_2_1_s_431 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_431 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_432 = 0; i_2_1_s_432 < 4; ++i_2_1_s_432) {
    Y_local[((i_2_1_s_432 * 16) + 8)] = (Y_local[((i_2_1_s_432 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_432 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_433 = 0; i_2_1_s_433 < 4; ++i_2_1_s_433) {
    if (i_2_1_s_433 < 2) {
      Y_local[((i_2_1_s_433 * 16) + 72)] = (Y_local[((i_2_1_s_433 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_433 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_434 = 0; i_2_1_s_434 < 4; ++i_2_1_s_434) {
    Y_local[((i_2_1_s_434 * 16) + 9)] = (Y_local[((i_2_1_s_434 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_434 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_435 = 0; i_2_1_s_435 < 4; ++i_2_1_s_435) {
    if (i_2_1_s_435 < 2) {
      Y_local[((i_2_1_s_435 * 16) + 73)] = (Y_local[((i_2_1_s_435 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_435 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_436 = 0; i_2_1_s_436 < 4; ++i_2_1_s_436) {
    Y_local[((i_2_1_s_436 * 16) + 10)] = (Y_local[((i_2_1_s_436 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_436 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_437 = 0; i_2_1_s_437 < 4; ++i_2_1_s_437) {
    if (i_2_1_s_437 < 2) {
      Y_local[((i_2_1_s_437 * 16) + 74)] = (Y_local[((i_2_1_s_437 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_437 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_438 = 0; i_2_1_s_438 < 4; ++i_2_1_s_438) {
    Y_local[((i_2_1_s_438 * 16) + 11)] = (Y_local[((i_2_1_s_438 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_438 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_439 = 0; i_2_1_s_439 < 4; ++i_2_1_s_439) {
    if (i_2_1_s_439 < 2) {
      Y_local[((i_2_1_s_439 * 16) + 75)] = (Y_local[((i_2_1_s_439 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_439 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_440 = 0; i_2_1_s_440 < 4; ++i_2_1_s_440) {
    Y_local[((i_2_1_s_440 * 16) + 12)] = (Y_local[((i_2_1_s_440 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_440 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_441 = 0; i_2_1_s_441 < 4; ++i_2_1_s_441) {
    if (i_2_1_s_441 < 2) {
      Y_local[((i_2_1_s_441 * 16) + 76)] = (Y_local[((i_2_1_s_441 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_441 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_442 = 0; i_2_1_s_442 < 4; ++i_2_1_s_442) {
    Y_local[((i_2_1_s_442 * 16) + 13)] = (Y_local[((i_2_1_s_442 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_442 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_443 = 0; i_2_1_s_443 < 4; ++i_2_1_s_443) {
    if (i_2_1_s_443 < 2) {
      Y_local[((i_2_1_s_443 * 16) + 77)] = (Y_local[((i_2_1_s_443 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_443 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_444 = 0; i_2_1_s_444 < 4; ++i_2_1_s_444) {
    Y_local[((i_2_1_s_444 * 16) + 14)] = (Y_local[((i_2_1_s_444 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_444 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_445 = 0; i_2_1_s_445 < 4; ++i_2_1_s_445) {
    if (i_2_1_s_445 < 2) {
      Y_local[((i_2_1_s_445 * 16) + 78)] = (Y_local[((i_2_1_s_445 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_445 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_446 = 0; i_2_1_s_446 < 4; ++i_2_1_s_446) {
    Y_local[((i_2_1_s_446 * 16) + 15)] = (Y_local[((i_2_1_s_446 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_446 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_447 = 0; i_2_1_s_447 < 4; ++i_2_1_s_447) {
    if (i_2_1_s_447 < 2) {
      Y_local[((i_2_1_s_447 * 16) + 79)] = (Y_local[((i_2_1_s_447 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_447 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax0_0_15 = 0; ax0_0_15 < 2; ++ax0_0_15) {
    for (int ax0_1_s_15 = 0; ax0_1_s_15 < 4; ++ax0_1_s_15) {
      if (((ax0_0_15 * 2) + (ax0_1_s_15 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_15 * 4) + ax0_1_s_15) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_15 * 16)) + (ax0_1_s_15 * 4)) + 579)];
      }
    }
  }
  for (int ax1_0_15 = 0; ax1_0_15 < 4; ++ax1_0_15) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_15 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_15 * 4)) >> 6) * 64) + ((ax1_0_15 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_15 >> 1)) & 7) * 4)) + 4224) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_448 = 0; i_2_1_s_448 < 4; ++i_2_1_s_448) {
    Y_local[(i_2_1_s_448 * 16)] = (Y_local[(i_2_1_s_448 * 16)] + (A_shared_dyn_local[i_2_1_s_448] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_449 = 0; i_2_1_s_449 < 4; ++i_2_1_s_449) {
    if (i_2_1_s_449 < 2) {
      Y_local[((i_2_1_s_449 * 16) + 64)] = (Y_local[((i_2_1_s_449 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_449 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_450 = 0; i_2_1_s_450 < 4; ++i_2_1_s_450) {
    Y_local[((i_2_1_s_450 * 16) + 1)] = (Y_local[((i_2_1_s_450 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_450] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_451 = 0; i_2_1_s_451 < 4; ++i_2_1_s_451) {
    if (i_2_1_s_451 < 2) {
      Y_local[((i_2_1_s_451 * 16) + 65)] = (Y_local[((i_2_1_s_451 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_451 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_452 = 0; i_2_1_s_452 < 4; ++i_2_1_s_452) {
    Y_local[((i_2_1_s_452 * 16) + 2)] = (Y_local[((i_2_1_s_452 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_452] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_453 = 0; i_2_1_s_453 < 4; ++i_2_1_s_453) {
    if (i_2_1_s_453 < 2) {
      Y_local[((i_2_1_s_453 * 16) + 66)] = (Y_local[((i_2_1_s_453 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_453 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_454 = 0; i_2_1_s_454 < 4; ++i_2_1_s_454) {
    Y_local[((i_2_1_s_454 * 16) + 3)] = (Y_local[((i_2_1_s_454 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_454] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_455 = 0; i_2_1_s_455 < 4; ++i_2_1_s_455) {
    if (i_2_1_s_455 < 2) {
      Y_local[((i_2_1_s_455 * 16) + 67)] = (Y_local[((i_2_1_s_455 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_455 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_456 = 0; i_2_1_s_456 < 4; ++i_2_1_s_456) {
    Y_local[((i_2_1_s_456 * 16) + 4)] = (Y_local[((i_2_1_s_456 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_456] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_457 = 0; i_2_1_s_457 < 4; ++i_2_1_s_457) {
    if (i_2_1_s_457 < 2) {
      Y_local[((i_2_1_s_457 * 16) + 68)] = (Y_local[((i_2_1_s_457 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_457 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_458 = 0; i_2_1_s_458 < 4; ++i_2_1_s_458) {
    Y_local[((i_2_1_s_458 * 16) + 5)] = (Y_local[((i_2_1_s_458 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_458] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_459 = 0; i_2_1_s_459 < 4; ++i_2_1_s_459) {
    if (i_2_1_s_459 < 2) {
      Y_local[((i_2_1_s_459 * 16) + 69)] = (Y_local[((i_2_1_s_459 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_459 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_460 = 0; i_2_1_s_460 < 4; ++i_2_1_s_460) {
    Y_local[((i_2_1_s_460 * 16) + 6)] = (Y_local[((i_2_1_s_460 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_460] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_461 = 0; i_2_1_s_461 < 4; ++i_2_1_s_461) {
    if (i_2_1_s_461 < 2) {
      Y_local[((i_2_1_s_461 * 16) + 70)] = (Y_local[((i_2_1_s_461 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_461 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_462 = 0; i_2_1_s_462 < 4; ++i_2_1_s_462) {
    Y_local[((i_2_1_s_462 * 16) + 7)] = (Y_local[((i_2_1_s_462 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_462] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_463 = 0; i_2_1_s_463 < 4; ++i_2_1_s_463) {
    if (i_2_1_s_463 < 2) {
      Y_local[((i_2_1_s_463 * 16) + 71)] = (Y_local[((i_2_1_s_463 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_463 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_464 = 0; i_2_1_s_464 < 4; ++i_2_1_s_464) {
    Y_local[((i_2_1_s_464 * 16) + 8)] = (Y_local[((i_2_1_s_464 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_464] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_465 = 0; i_2_1_s_465 < 4; ++i_2_1_s_465) {
    if (i_2_1_s_465 < 2) {
      Y_local[((i_2_1_s_465 * 16) + 72)] = (Y_local[((i_2_1_s_465 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_465 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_466 = 0; i_2_1_s_466 < 4; ++i_2_1_s_466) {
    Y_local[((i_2_1_s_466 * 16) + 9)] = (Y_local[((i_2_1_s_466 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_466] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_467 = 0; i_2_1_s_467 < 4; ++i_2_1_s_467) {
    if (i_2_1_s_467 < 2) {
      Y_local[((i_2_1_s_467 * 16) + 73)] = (Y_local[((i_2_1_s_467 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_467 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_468 = 0; i_2_1_s_468 < 4; ++i_2_1_s_468) {
    Y_local[((i_2_1_s_468 * 16) + 10)] = (Y_local[((i_2_1_s_468 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_468] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_469 = 0; i_2_1_s_469 < 4; ++i_2_1_s_469) {
    if (i_2_1_s_469 < 2) {
      Y_local[((i_2_1_s_469 * 16) + 74)] = (Y_local[((i_2_1_s_469 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_469 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_470 = 0; i_2_1_s_470 < 4; ++i_2_1_s_470) {
    Y_local[((i_2_1_s_470 * 16) + 11)] = (Y_local[((i_2_1_s_470 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_470] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_471 = 0; i_2_1_s_471 < 4; ++i_2_1_s_471) {
    if (i_2_1_s_471 < 2) {
      Y_local[((i_2_1_s_471 * 16) + 75)] = (Y_local[((i_2_1_s_471 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_471 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_472 = 0; i_2_1_s_472 < 4; ++i_2_1_s_472) {
    Y_local[((i_2_1_s_472 * 16) + 12)] = (Y_local[((i_2_1_s_472 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_472] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_473 = 0; i_2_1_s_473 < 4; ++i_2_1_s_473) {
    if (i_2_1_s_473 < 2) {
      Y_local[((i_2_1_s_473 * 16) + 76)] = (Y_local[((i_2_1_s_473 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_473 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_474 = 0; i_2_1_s_474 < 4; ++i_2_1_s_474) {
    Y_local[((i_2_1_s_474 * 16) + 13)] = (Y_local[((i_2_1_s_474 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_474] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_475 = 0; i_2_1_s_475 < 4; ++i_2_1_s_475) {
    if (i_2_1_s_475 < 2) {
      Y_local[((i_2_1_s_475 * 16) + 77)] = (Y_local[((i_2_1_s_475 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_475 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_476 = 0; i_2_1_s_476 < 4; ++i_2_1_s_476) {
    Y_local[((i_2_1_s_476 * 16) + 14)] = (Y_local[((i_2_1_s_476 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_476] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_477 = 0; i_2_1_s_477 < 4; ++i_2_1_s_477) {
    if (i_2_1_s_477 < 2) {
      Y_local[((i_2_1_s_477 * 16) + 78)] = (Y_local[((i_2_1_s_477 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_477 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_478 = 0; i_2_1_s_478 < 4; ++i_2_1_s_478) {
    Y_local[((i_2_1_s_478 * 16) + 15)] = (Y_local[((i_2_1_s_478 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_478] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_479 = 0; i_2_1_s_479 < 4; ++i_2_1_s_479) {
    if (i_2_1_s_479 < 2) {
      Y_local[((i_2_1_s_479 * 16) + 79)] = (Y_local[((i_2_1_s_479 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_479 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_16 = 0; ax0_0_16 < 2; ++ax0_0_16) {
    for (int ax0_1_s_16 = 0; ax0_1_s_16 < 4; ++ax0_1_s_16) {
      if (((ax0_0_16 * 2) + (ax0_1_s_16 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_16 * 4) + ax0_1_s_16)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_16 * 16)) + (ax0_1_s_16 * 4)) + 1152)];
      }
    }
  }
  for (int ax1_0_16 = 0; ax1_0_16 < 4; ++ax1_0_16) {
    *(float4*)(B_shared_dyn_local + (ax1_0_16 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_16 * 4)) >> 6) * 64) + ((ax1_0_16 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_16 >> 1)) & 7) * 4)) + 4416) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_480 = 0; i_2_1_s_480 < 4; ++i_2_1_s_480) {
    Y_local[(i_2_1_s_480 * 16)] = (Y_local[(i_2_1_s_480 * 16)] + (A_shared_dyn_local[(i_2_1_s_480 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_481 = 0; i_2_1_s_481 < 4; ++i_2_1_s_481) {
    if (i_2_1_s_481 < 2) {
      Y_local[((i_2_1_s_481 * 16) + 64)] = (Y_local[((i_2_1_s_481 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_481 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_482 = 0; i_2_1_s_482 < 4; ++i_2_1_s_482) {
    Y_local[((i_2_1_s_482 * 16) + 1)] = (Y_local[((i_2_1_s_482 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_482 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_483 = 0; i_2_1_s_483 < 4; ++i_2_1_s_483) {
    if (i_2_1_s_483 < 2) {
      Y_local[((i_2_1_s_483 * 16) + 65)] = (Y_local[((i_2_1_s_483 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_483 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_484 = 0; i_2_1_s_484 < 4; ++i_2_1_s_484) {
    Y_local[((i_2_1_s_484 * 16) + 2)] = (Y_local[((i_2_1_s_484 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_484 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_485 = 0; i_2_1_s_485 < 4; ++i_2_1_s_485) {
    if (i_2_1_s_485 < 2) {
      Y_local[((i_2_1_s_485 * 16) + 66)] = (Y_local[((i_2_1_s_485 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_485 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_486 = 0; i_2_1_s_486 < 4; ++i_2_1_s_486) {
    Y_local[((i_2_1_s_486 * 16) + 3)] = (Y_local[((i_2_1_s_486 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_486 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_487 = 0; i_2_1_s_487 < 4; ++i_2_1_s_487) {
    if (i_2_1_s_487 < 2) {
      Y_local[((i_2_1_s_487 * 16) + 67)] = (Y_local[((i_2_1_s_487 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_487 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_488 = 0; i_2_1_s_488 < 4; ++i_2_1_s_488) {
    Y_local[((i_2_1_s_488 * 16) + 4)] = (Y_local[((i_2_1_s_488 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_488 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_489 = 0; i_2_1_s_489 < 4; ++i_2_1_s_489) {
    if (i_2_1_s_489 < 2) {
      Y_local[((i_2_1_s_489 * 16) + 68)] = (Y_local[((i_2_1_s_489 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_489 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_490 = 0; i_2_1_s_490 < 4; ++i_2_1_s_490) {
    Y_local[((i_2_1_s_490 * 16) + 5)] = (Y_local[((i_2_1_s_490 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_490 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_491 = 0; i_2_1_s_491 < 4; ++i_2_1_s_491) {
    if (i_2_1_s_491 < 2) {
      Y_local[((i_2_1_s_491 * 16) + 69)] = (Y_local[((i_2_1_s_491 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_491 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_492 = 0; i_2_1_s_492 < 4; ++i_2_1_s_492) {
    Y_local[((i_2_1_s_492 * 16) + 6)] = (Y_local[((i_2_1_s_492 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_492 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_493 = 0; i_2_1_s_493 < 4; ++i_2_1_s_493) {
    if (i_2_1_s_493 < 2) {
      Y_local[((i_2_1_s_493 * 16) + 70)] = (Y_local[((i_2_1_s_493 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_493 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_494 = 0; i_2_1_s_494 < 4; ++i_2_1_s_494) {
    Y_local[((i_2_1_s_494 * 16) + 7)] = (Y_local[((i_2_1_s_494 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_494 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_495 = 0; i_2_1_s_495 < 4; ++i_2_1_s_495) {
    if (i_2_1_s_495 < 2) {
      Y_local[((i_2_1_s_495 * 16) + 71)] = (Y_local[((i_2_1_s_495 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_495 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_496 = 0; i_2_1_s_496 < 4; ++i_2_1_s_496) {
    Y_local[((i_2_1_s_496 * 16) + 8)] = (Y_local[((i_2_1_s_496 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_496 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_497 = 0; i_2_1_s_497 < 4; ++i_2_1_s_497) {
    if (i_2_1_s_497 < 2) {
      Y_local[((i_2_1_s_497 * 16) + 72)] = (Y_local[((i_2_1_s_497 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_497 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_498 = 0; i_2_1_s_498 < 4; ++i_2_1_s_498) {
    Y_local[((i_2_1_s_498 * 16) + 9)] = (Y_local[((i_2_1_s_498 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_498 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_499 = 0; i_2_1_s_499 < 4; ++i_2_1_s_499) {
    if (i_2_1_s_499 < 2) {
      Y_local[((i_2_1_s_499 * 16) + 73)] = (Y_local[((i_2_1_s_499 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_499 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_500 = 0; i_2_1_s_500 < 4; ++i_2_1_s_500) {
    Y_local[((i_2_1_s_500 * 16) + 10)] = (Y_local[((i_2_1_s_500 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_500 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_501 = 0; i_2_1_s_501 < 4; ++i_2_1_s_501) {
    if (i_2_1_s_501 < 2) {
      Y_local[((i_2_1_s_501 * 16) + 74)] = (Y_local[((i_2_1_s_501 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_501 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_502 = 0; i_2_1_s_502 < 4; ++i_2_1_s_502) {
    Y_local[((i_2_1_s_502 * 16) + 11)] = (Y_local[((i_2_1_s_502 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_502 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_503 = 0; i_2_1_s_503 < 4; ++i_2_1_s_503) {
    if (i_2_1_s_503 < 2) {
      Y_local[((i_2_1_s_503 * 16) + 75)] = (Y_local[((i_2_1_s_503 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_503 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_504 = 0; i_2_1_s_504 < 4; ++i_2_1_s_504) {
    Y_local[((i_2_1_s_504 * 16) + 12)] = (Y_local[((i_2_1_s_504 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_504 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_505 = 0; i_2_1_s_505 < 4; ++i_2_1_s_505) {
    if (i_2_1_s_505 < 2) {
      Y_local[((i_2_1_s_505 * 16) + 76)] = (Y_local[((i_2_1_s_505 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_505 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_506 = 0; i_2_1_s_506 < 4; ++i_2_1_s_506) {
    Y_local[((i_2_1_s_506 * 16) + 13)] = (Y_local[((i_2_1_s_506 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_506 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_507 = 0; i_2_1_s_507 < 4; ++i_2_1_s_507) {
    if (i_2_1_s_507 < 2) {
      Y_local[((i_2_1_s_507 * 16) + 77)] = (Y_local[((i_2_1_s_507 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_507 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_508 = 0; i_2_1_s_508 < 4; ++i_2_1_s_508) {
    Y_local[((i_2_1_s_508 * 16) + 14)] = (Y_local[((i_2_1_s_508 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_508 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_509 = 0; i_2_1_s_509 < 4; ++i_2_1_s_509) {
    if (i_2_1_s_509 < 2) {
      Y_local[((i_2_1_s_509 * 16) + 78)] = (Y_local[((i_2_1_s_509 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_509 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_510 = 0; i_2_1_s_510 < 4; ++i_2_1_s_510) {
    Y_local[((i_2_1_s_510 * 16) + 15)] = (Y_local[((i_2_1_s_510 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_510 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_511 = 0; i_2_1_s_511 < 4; ++i_2_1_s_511) {
    if (i_2_1_s_511 < 2) {
      Y_local[((i_2_1_s_511 * 16) + 79)] = (Y_local[((i_2_1_s_511 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_511 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax0_0_17 = 0; ax0_0_17 < 2; ++ax0_0_17) {
    for (int ax0_1_s_17 = 0; ax0_1_s_17 < 4; ++ax0_1_s_17) {
      if (((ax0_0_17 * 2) + (ax0_1_s_17 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_17 * 4) + ax0_1_s_17) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_17 * 16)) + (ax0_1_s_17 * 4)) + 1153)];
      }
    }
  }
  for (int ax1_0_17 = 0; ax1_0_17 < 4; ++ax1_0_17) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_17 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_17 * 4)) >> 6) * 64) + ((ax1_0_17 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_17 >> 1)) & 7) * 4)) + 4608) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_512 = 0; i_2_1_s_512 < 4; ++i_2_1_s_512) {
    Y_local[(i_2_1_s_512 * 16)] = (Y_local[(i_2_1_s_512 * 16)] + (A_shared_dyn_local[i_2_1_s_512] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_513 = 0; i_2_1_s_513 < 4; ++i_2_1_s_513) {
    if (i_2_1_s_513 < 2) {
      Y_local[((i_2_1_s_513 * 16) + 64)] = (Y_local[((i_2_1_s_513 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_513 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_514 = 0; i_2_1_s_514 < 4; ++i_2_1_s_514) {
    Y_local[((i_2_1_s_514 * 16) + 1)] = (Y_local[((i_2_1_s_514 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_514] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_515 = 0; i_2_1_s_515 < 4; ++i_2_1_s_515) {
    if (i_2_1_s_515 < 2) {
      Y_local[((i_2_1_s_515 * 16) + 65)] = (Y_local[((i_2_1_s_515 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_515 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_516 = 0; i_2_1_s_516 < 4; ++i_2_1_s_516) {
    Y_local[((i_2_1_s_516 * 16) + 2)] = (Y_local[((i_2_1_s_516 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_516] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_517 = 0; i_2_1_s_517 < 4; ++i_2_1_s_517) {
    if (i_2_1_s_517 < 2) {
      Y_local[((i_2_1_s_517 * 16) + 66)] = (Y_local[((i_2_1_s_517 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_517 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_518 = 0; i_2_1_s_518 < 4; ++i_2_1_s_518) {
    Y_local[((i_2_1_s_518 * 16) + 3)] = (Y_local[((i_2_1_s_518 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_518] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_519 = 0; i_2_1_s_519 < 4; ++i_2_1_s_519) {
    if (i_2_1_s_519 < 2) {
      Y_local[((i_2_1_s_519 * 16) + 67)] = (Y_local[((i_2_1_s_519 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_519 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_520 = 0; i_2_1_s_520 < 4; ++i_2_1_s_520) {
    Y_local[((i_2_1_s_520 * 16) + 4)] = (Y_local[((i_2_1_s_520 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_520] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_521 = 0; i_2_1_s_521 < 4; ++i_2_1_s_521) {
    if (i_2_1_s_521 < 2) {
      Y_local[((i_2_1_s_521 * 16) + 68)] = (Y_local[((i_2_1_s_521 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_521 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_522 = 0; i_2_1_s_522 < 4; ++i_2_1_s_522) {
    Y_local[((i_2_1_s_522 * 16) + 5)] = (Y_local[((i_2_1_s_522 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_522] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_523 = 0; i_2_1_s_523 < 4; ++i_2_1_s_523) {
    if (i_2_1_s_523 < 2) {
      Y_local[((i_2_1_s_523 * 16) + 69)] = (Y_local[((i_2_1_s_523 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_523 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_524 = 0; i_2_1_s_524 < 4; ++i_2_1_s_524) {
    Y_local[((i_2_1_s_524 * 16) + 6)] = (Y_local[((i_2_1_s_524 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_524] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_525 = 0; i_2_1_s_525 < 4; ++i_2_1_s_525) {
    if (i_2_1_s_525 < 2) {
      Y_local[((i_2_1_s_525 * 16) + 70)] = (Y_local[((i_2_1_s_525 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_525 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_526 = 0; i_2_1_s_526 < 4; ++i_2_1_s_526) {
    Y_local[((i_2_1_s_526 * 16) + 7)] = (Y_local[((i_2_1_s_526 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_526] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_527 = 0; i_2_1_s_527 < 4; ++i_2_1_s_527) {
    if (i_2_1_s_527 < 2) {
      Y_local[((i_2_1_s_527 * 16) + 71)] = (Y_local[((i_2_1_s_527 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_527 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_528 = 0; i_2_1_s_528 < 4; ++i_2_1_s_528) {
    Y_local[((i_2_1_s_528 * 16) + 8)] = (Y_local[((i_2_1_s_528 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_528] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_529 = 0; i_2_1_s_529 < 4; ++i_2_1_s_529) {
    if (i_2_1_s_529 < 2) {
      Y_local[((i_2_1_s_529 * 16) + 72)] = (Y_local[((i_2_1_s_529 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_529 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_530 = 0; i_2_1_s_530 < 4; ++i_2_1_s_530) {
    Y_local[((i_2_1_s_530 * 16) + 9)] = (Y_local[((i_2_1_s_530 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_530] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_531 = 0; i_2_1_s_531 < 4; ++i_2_1_s_531) {
    if (i_2_1_s_531 < 2) {
      Y_local[((i_2_1_s_531 * 16) + 73)] = (Y_local[((i_2_1_s_531 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_531 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_532 = 0; i_2_1_s_532 < 4; ++i_2_1_s_532) {
    Y_local[((i_2_1_s_532 * 16) + 10)] = (Y_local[((i_2_1_s_532 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_532] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_533 = 0; i_2_1_s_533 < 4; ++i_2_1_s_533) {
    if (i_2_1_s_533 < 2) {
      Y_local[((i_2_1_s_533 * 16) + 74)] = (Y_local[((i_2_1_s_533 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_533 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_534 = 0; i_2_1_s_534 < 4; ++i_2_1_s_534) {
    Y_local[((i_2_1_s_534 * 16) + 11)] = (Y_local[((i_2_1_s_534 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_534] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_535 = 0; i_2_1_s_535 < 4; ++i_2_1_s_535) {
    if (i_2_1_s_535 < 2) {
      Y_local[((i_2_1_s_535 * 16) + 75)] = (Y_local[((i_2_1_s_535 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_535 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_536 = 0; i_2_1_s_536 < 4; ++i_2_1_s_536) {
    Y_local[((i_2_1_s_536 * 16) + 12)] = (Y_local[((i_2_1_s_536 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_536] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_537 = 0; i_2_1_s_537 < 4; ++i_2_1_s_537) {
    if (i_2_1_s_537 < 2) {
      Y_local[((i_2_1_s_537 * 16) + 76)] = (Y_local[((i_2_1_s_537 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_537 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_538 = 0; i_2_1_s_538 < 4; ++i_2_1_s_538) {
    Y_local[((i_2_1_s_538 * 16) + 13)] = (Y_local[((i_2_1_s_538 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_538] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_539 = 0; i_2_1_s_539 < 4; ++i_2_1_s_539) {
    if (i_2_1_s_539 < 2) {
      Y_local[((i_2_1_s_539 * 16) + 77)] = (Y_local[((i_2_1_s_539 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_539 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_540 = 0; i_2_1_s_540 < 4; ++i_2_1_s_540) {
    Y_local[((i_2_1_s_540 * 16) + 14)] = (Y_local[((i_2_1_s_540 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_540] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_541 = 0; i_2_1_s_541 < 4; ++i_2_1_s_541) {
    if (i_2_1_s_541 < 2) {
      Y_local[((i_2_1_s_541 * 16) + 78)] = (Y_local[((i_2_1_s_541 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_541 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_542 = 0; i_2_1_s_542 < 4; ++i_2_1_s_542) {
    Y_local[((i_2_1_s_542 * 16) + 15)] = (Y_local[((i_2_1_s_542 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_542] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_543 = 0; i_2_1_s_543 < 4; ++i_2_1_s_543) {
    if (i_2_1_s_543 < 2) {
      Y_local[((i_2_1_s_543 * 16) + 79)] = (Y_local[((i_2_1_s_543 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_543 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_0_18 = 0; ax0_0_18 < 2; ++ax0_0_18) {
    for (int ax0_1_s_18 = 0; ax0_1_s_18 < 4; ++ax0_1_s_18) {
      if (((ax0_0_18 * 2) + (ax0_1_s_18 >> 1)) < 3) {
        A_shared_dyn_local[((ax0_0_18 * 4) + ax0_1_s_18)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_18 * 16)) + (ax0_1_s_18 * 4)) + 1154)];
      }
    }
  }
  for (int ax1_0_18 = 0; ax1_0_18 < 4; ++ax1_0_18) {
    *(float4*)(B_shared_dyn_local + (ax1_0_18 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_18 * 4)) >> 6) * 64) + ((ax1_0_18 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_18 >> 1)) & 7) * 4)) + 4800) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_544 = 0; i_2_1_s_544 < 4; ++i_2_1_s_544) {
    Y_local[(i_2_1_s_544 * 16)] = (Y_local[(i_2_1_s_544 * 16)] + (A_shared_dyn_local[(i_2_1_s_544 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_545 = 0; i_2_1_s_545 < 4; ++i_2_1_s_545) {
    if (i_2_1_s_545 < 2) {
      Y_local[((i_2_1_s_545 * 16) + 64)] = (Y_local[((i_2_1_s_545 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_545 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_546 = 0; i_2_1_s_546 < 4; ++i_2_1_s_546) {
    Y_local[((i_2_1_s_546 * 16) + 1)] = (Y_local[((i_2_1_s_546 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_546 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_547 = 0; i_2_1_s_547 < 4; ++i_2_1_s_547) {
    if (i_2_1_s_547 < 2) {
      Y_local[((i_2_1_s_547 * 16) + 65)] = (Y_local[((i_2_1_s_547 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_547 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_548 = 0; i_2_1_s_548 < 4; ++i_2_1_s_548) {
    Y_local[((i_2_1_s_548 * 16) + 2)] = (Y_local[((i_2_1_s_548 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_548 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_549 = 0; i_2_1_s_549 < 4; ++i_2_1_s_549) {
    if (i_2_1_s_549 < 2) {
      Y_local[((i_2_1_s_549 * 16) + 66)] = (Y_local[((i_2_1_s_549 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_549 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_550 = 0; i_2_1_s_550 < 4; ++i_2_1_s_550) {
    Y_local[((i_2_1_s_550 * 16) + 3)] = (Y_local[((i_2_1_s_550 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_550 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_551 = 0; i_2_1_s_551 < 4; ++i_2_1_s_551) {
    if (i_2_1_s_551 < 2) {
      Y_local[((i_2_1_s_551 * 16) + 67)] = (Y_local[((i_2_1_s_551 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_551 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_552 = 0; i_2_1_s_552 < 4; ++i_2_1_s_552) {
    Y_local[((i_2_1_s_552 * 16) + 4)] = (Y_local[((i_2_1_s_552 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_552 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_553 = 0; i_2_1_s_553 < 4; ++i_2_1_s_553) {
    if (i_2_1_s_553 < 2) {
      Y_local[((i_2_1_s_553 * 16) + 68)] = (Y_local[((i_2_1_s_553 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_553 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_554 = 0; i_2_1_s_554 < 4; ++i_2_1_s_554) {
    Y_local[((i_2_1_s_554 * 16) + 5)] = (Y_local[((i_2_1_s_554 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_554 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_555 = 0; i_2_1_s_555 < 4; ++i_2_1_s_555) {
    if (i_2_1_s_555 < 2) {
      Y_local[((i_2_1_s_555 * 16) + 69)] = (Y_local[((i_2_1_s_555 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_555 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_556 = 0; i_2_1_s_556 < 4; ++i_2_1_s_556) {
    Y_local[((i_2_1_s_556 * 16) + 6)] = (Y_local[((i_2_1_s_556 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_556 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_557 = 0; i_2_1_s_557 < 4; ++i_2_1_s_557) {
    if (i_2_1_s_557 < 2) {
      Y_local[((i_2_1_s_557 * 16) + 70)] = (Y_local[((i_2_1_s_557 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_557 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_558 = 0; i_2_1_s_558 < 4; ++i_2_1_s_558) {
    Y_local[((i_2_1_s_558 * 16) + 7)] = (Y_local[((i_2_1_s_558 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_558 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_559 = 0; i_2_1_s_559 < 4; ++i_2_1_s_559) {
    if (i_2_1_s_559 < 2) {
      Y_local[((i_2_1_s_559 * 16) + 71)] = (Y_local[((i_2_1_s_559 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_559 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_560 = 0; i_2_1_s_560 < 4; ++i_2_1_s_560) {
    Y_local[((i_2_1_s_560 * 16) + 8)] = (Y_local[((i_2_1_s_560 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_560 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_561 = 0; i_2_1_s_561 < 4; ++i_2_1_s_561) {
    if (i_2_1_s_561 < 2) {
      Y_local[((i_2_1_s_561 * 16) + 72)] = (Y_local[((i_2_1_s_561 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_561 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_562 = 0; i_2_1_s_562 < 4; ++i_2_1_s_562) {
    Y_local[((i_2_1_s_562 * 16) + 9)] = (Y_local[((i_2_1_s_562 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_562 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_563 = 0; i_2_1_s_563 < 4; ++i_2_1_s_563) {
    if (i_2_1_s_563 < 2) {
      Y_local[((i_2_1_s_563 * 16) + 73)] = (Y_local[((i_2_1_s_563 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_563 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_564 = 0; i_2_1_s_564 < 4; ++i_2_1_s_564) {
    Y_local[((i_2_1_s_564 * 16) + 10)] = (Y_local[((i_2_1_s_564 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_564 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_565 = 0; i_2_1_s_565 < 4; ++i_2_1_s_565) {
    if (i_2_1_s_565 < 2) {
      Y_local[((i_2_1_s_565 * 16) + 74)] = (Y_local[((i_2_1_s_565 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_565 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_566 = 0; i_2_1_s_566 < 4; ++i_2_1_s_566) {
    Y_local[((i_2_1_s_566 * 16) + 11)] = (Y_local[((i_2_1_s_566 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_566 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_567 = 0; i_2_1_s_567 < 4; ++i_2_1_s_567) {
    if (i_2_1_s_567 < 2) {
      Y_local[((i_2_1_s_567 * 16) + 75)] = (Y_local[((i_2_1_s_567 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_567 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_568 = 0; i_2_1_s_568 < 4; ++i_2_1_s_568) {
    Y_local[((i_2_1_s_568 * 16) + 12)] = (Y_local[((i_2_1_s_568 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_568 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_569 = 0; i_2_1_s_569 < 4; ++i_2_1_s_569) {
    if (i_2_1_s_569 < 2) {
      Y_local[((i_2_1_s_569 * 16) + 76)] = (Y_local[((i_2_1_s_569 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_569 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_570 = 0; i_2_1_s_570 < 4; ++i_2_1_s_570) {
    Y_local[((i_2_1_s_570 * 16) + 13)] = (Y_local[((i_2_1_s_570 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_570 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_571 = 0; i_2_1_s_571 < 4; ++i_2_1_s_571) {
    if (i_2_1_s_571 < 2) {
      Y_local[((i_2_1_s_571 * 16) + 77)] = (Y_local[((i_2_1_s_571 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_571 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_572 = 0; i_2_1_s_572 < 4; ++i_2_1_s_572) {
    Y_local[((i_2_1_s_572 * 16) + 14)] = (Y_local[((i_2_1_s_572 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_572 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_573 = 0; i_2_1_s_573 < 4; ++i_2_1_s_573) {
    if (i_2_1_s_573 < 2) {
      Y_local[((i_2_1_s_573 * 16) + 78)] = (Y_local[((i_2_1_s_573 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_573 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_574 = 0; i_2_1_s_574 < 4; ++i_2_1_s_574) {
    Y_local[((i_2_1_s_574 * 16) + 15)] = (Y_local[((i_2_1_s_574 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_574 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_575 = 0; i_2_1_s_575 < 4; ++i_2_1_s_575) {
    if (i_2_1_s_575 < 2) {
      Y_local[((i_2_1_s_575 * 16) + 79)] = (Y_local[((i_2_1_s_575 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_575 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax0_0_19 = 0; ax0_0_19 < 2; ++ax0_0_19) {
    for (int ax0_1_s_19 = 0; ax0_1_s_19 < 4; ++ax0_1_s_19) {
      if (((ax0_0_19 * 2) + (ax0_1_s_19 >> 1)) < 3) {
        A_shared_dyn_local[(((ax0_0_19 * 4) + ax0_1_s_19) + 6)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) / 27) * 72) + ((((int)threadIdx.x) % 3) * 24)) + (ax0_0_19 * 16)) + (ax0_1_s_19 * 4)) + 1155)];
      }
    }
  }
  for (int ax1_0_19 = 0; ax1_0_19 < 4; ++ax1_0_19) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_19 * 4) + 16)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((((((int)blockIdx.x) & 7) * 144) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_19 * 4)) >> 6) * 64) + ((ax1_0_19 & 1) * 32)) + ((((((((int)blockIdx.x) & 7) * 18) + (((((int)threadIdx.x) % 27) / 3) * 2)) + (ax1_0_19 >> 1)) & 7) * 4)) + 4992) - ((((((int)blockIdx.x) & 7) * 144) >> 6) * 64)));
  }
  for (int i_2_1_s_576 = 0; i_2_1_s_576 < 4; ++i_2_1_s_576) {
    Y_local[(i_2_1_s_576 * 16)] = (Y_local[(i_2_1_s_576 * 16)] + (A_shared_dyn_local[i_2_1_s_576] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_577 = 0; i_2_1_s_577 < 4; ++i_2_1_s_577) {
    if (i_2_1_s_577 < 2) {
      Y_local[((i_2_1_s_577 * 16) + 64)] = (Y_local[((i_2_1_s_577 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_577 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_578 = 0; i_2_1_s_578 < 4; ++i_2_1_s_578) {
    Y_local[((i_2_1_s_578 * 16) + 1)] = (Y_local[((i_2_1_s_578 * 16) + 1)] + (A_shared_dyn_local[i_2_1_s_578] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_579 = 0; i_2_1_s_579 < 4; ++i_2_1_s_579) {
    if (i_2_1_s_579 < 2) {
      Y_local[((i_2_1_s_579 * 16) + 65)] = (Y_local[((i_2_1_s_579 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_579 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_580 = 0; i_2_1_s_580 < 4; ++i_2_1_s_580) {
    Y_local[((i_2_1_s_580 * 16) + 2)] = (Y_local[((i_2_1_s_580 * 16) + 2)] + (A_shared_dyn_local[i_2_1_s_580] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_581 = 0; i_2_1_s_581 < 4; ++i_2_1_s_581) {
    if (i_2_1_s_581 < 2) {
      Y_local[((i_2_1_s_581 * 16) + 66)] = (Y_local[((i_2_1_s_581 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_581 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_582 = 0; i_2_1_s_582 < 4; ++i_2_1_s_582) {
    Y_local[((i_2_1_s_582 * 16) + 3)] = (Y_local[((i_2_1_s_582 * 16) + 3)] + (A_shared_dyn_local[i_2_1_s_582] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_583 = 0; i_2_1_s_583 < 4; ++i_2_1_s_583) {
    if (i_2_1_s_583 < 2) {
      Y_local[((i_2_1_s_583 * 16) + 67)] = (Y_local[((i_2_1_s_583 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_583 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_584 = 0; i_2_1_s_584 < 4; ++i_2_1_s_584) {
    Y_local[((i_2_1_s_584 * 16) + 4)] = (Y_local[((i_2_1_s_584 * 16) + 4)] + (A_shared_dyn_local[i_2_1_s_584] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_585 = 0; i_2_1_s_585 < 4; ++i_2_1_s_585) {
    if (i_2_1_s_585 < 2) {
      Y_local[((i_2_1_s_585 * 16) + 68)] = (Y_local[((i_2_1_s_585 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_585 + 4)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_586 = 0; i_2_1_s_586 < 4; ++i_2_1_s_586) {
    Y_local[((i_2_1_s_586 * 16) + 5)] = (Y_local[((i_2_1_s_586 * 16) + 5)] + (A_shared_dyn_local[i_2_1_s_586] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_587 = 0; i_2_1_s_587 < 4; ++i_2_1_s_587) {
    if (i_2_1_s_587 < 2) {
      Y_local[((i_2_1_s_587 * 16) + 69)] = (Y_local[((i_2_1_s_587 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_587 + 4)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_588 = 0; i_2_1_s_588 < 4; ++i_2_1_s_588) {
    Y_local[((i_2_1_s_588 * 16) + 6)] = (Y_local[((i_2_1_s_588 * 16) + 6)] + (A_shared_dyn_local[i_2_1_s_588] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_589 = 0; i_2_1_s_589 < 4; ++i_2_1_s_589) {
    if (i_2_1_s_589 < 2) {
      Y_local[((i_2_1_s_589 * 16) + 70)] = (Y_local[((i_2_1_s_589 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_589 + 4)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_590 = 0; i_2_1_s_590 < 4; ++i_2_1_s_590) {
    Y_local[((i_2_1_s_590 * 16) + 7)] = (Y_local[((i_2_1_s_590 * 16) + 7)] + (A_shared_dyn_local[i_2_1_s_590] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_591 = 0; i_2_1_s_591 < 4; ++i_2_1_s_591) {
    if (i_2_1_s_591 < 2) {
      Y_local[((i_2_1_s_591 * 16) + 71)] = (Y_local[((i_2_1_s_591 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_591 + 4)] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_592 = 0; i_2_1_s_592 < 4; ++i_2_1_s_592) {
    Y_local[((i_2_1_s_592 * 16) + 8)] = (Y_local[((i_2_1_s_592 * 16) + 8)] + (A_shared_dyn_local[i_2_1_s_592] * B_shared_dyn_local[8]));
  }
  for (int i_2_1_s_593 = 0; i_2_1_s_593 < 4; ++i_2_1_s_593) {
    if (i_2_1_s_593 < 2) {
      Y_local[((i_2_1_s_593 * 16) + 72)] = (Y_local[((i_2_1_s_593 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_593 + 4)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_594 = 0; i_2_1_s_594 < 4; ++i_2_1_s_594) {
    Y_local[((i_2_1_s_594 * 16) + 9)] = (Y_local[((i_2_1_s_594 * 16) + 9)] + (A_shared_dyn_local[i_2_1_s_594] * B_shared_dyn_local[9]));
  }
  for (int i_2_1_s_595 = 0; i_2_1_s_595 < 4; ++i_2_1_s_595) {
    if (i_2_1_s_595 < 2) {
      Y_local[((i_2_1_s_595 * 16) + 73)] = (Y_local[((i_2_1_s_595 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_595 + 4)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_596 = 0; i_2_1_s_596 < 4; ++i_2_1_s_596) {
    Y_local[((i_2_1_s_596 * 16) + 10)] = (Y_local[((i_2_1_s_596 * 16) + 10)] + (A_shared_dyn_local[i_2_1_s_596] * B_shared_dyn_local[10]));
  }
  for (int i_2_1_s_597 = 0; i_2_1_s_597 < 4; ++i_2_1_s_597) {
    if (i_2_1_s_597 < 2) {
      Y_local[((i_2_1_s_597 * 16) + 74)] = (Y_local[((i_2_1_s_597 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_597 + 4)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_598 = 0; i_2_1_s_598 < 4; ++i_2_1_s_598) {
    Y_local[((i_2_1_s_598 * 16) + 11)] = (Y_local[((i_2_1_s_598 * 16) + 11)] + (A_shared_dyn_local[i_2_1_s_598] * B_shared_dyn_local[11]));
  }
  for (int i_2_1_s_599 = 0; i_2_1_s_599 < 4; ++i_2_1_s_599) {
    if (i_2_1_s_599 < 2) {
      Y_local[((i_2_1_s_599 * 16) + 75)] = (Y_local[((i_2_1_s_599 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_599 + 4)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_600 = 0; i_2_1_s_600 < 4; ++i_2_1_s_600) {
    Y_local[((i_2_1_s_600 * 16) + 12)] = (Y_local[((i_2_1_s_600 * 16) + 12)] + (A_shared_dyn_local[i_2_1_s_600] * B_shared_dyn_local[12]));
  }
  for (int i_2_1_s_601 = 0; i_2_1_s_601 < 4; ++i_2_1_s_601) {
    if (i_2_1_s_601 < 2) {
      Y_local[((i_2_1_s_601 * 16) + 76)] = (Y_local[((i_2_1_s_601 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_601 + 4)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_602 = 0; i_2_1_s_602 < 4; ++i_2_1_s_602) {
    Y_local[((i_2_1_s_602 * 16) + 13)] = (Y_local[((i_2_1_s_602 * 16) + 13)] + (A_shared_dyn_local[i_2_1_s_602] * B_shared_dyn_local[13]));
  }
  for (int i_2_1_s_603 = 0; i_2_1_s_603 < 4; ++i_2_1_s_603) {
    if (i_2_1_s_603 < 2) {
      Y_local[((i_2_1_s_603 * 16) + 77)] = (Y_local[((i_2_1_s_603 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_603 + 4)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_604 = 0; i_2_1_s_604 < 4; ++i_2_1_s_604) {
    Y_local[((i_2_1_s_604 * 16) + 14)] = (Y_local[((i_2_1_s_604 * 16) + 14)] + (A_shared_dyn_local[i_2_1_s_604] * B_shared_dyn_local[14]));
  }
  for (int i_2_1_s_605 = 0; i_2_1_s_605 < 4; ++i_2_1_s_605) {
    if (i_2_1_s_605 < 2) {
      Y_local[((i_2_1_s_605 * 16) + 78)] = (Y_local[((i_2_1_s_605 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_605 + 4)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_606 = 0; i_2_1_s_606 < 4; ++i_2_1_s_606) {
    Y_local[((i_2_1_s_606 * 16) + 15)] = (Y_local[((i_2_1_s_606 * 16) + 15)] + (A_shared_dyn_local[i_2_1_s_606] * B_shared_dyn_local[15]));
  }
  for (int i_2_1_s_607 = 0; i_2_1_s_607 < 4; ++i_2_1_s_607) {
    if (i_2_1_s_607 < 2) {
      Y_local[((i_2_1_s_607 * 16) + 79)] = (Y_local[((i_2_1_s_607 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_607 + 4)] * B_shared_dyn_local[15]));
    }
  }
  for (int i_2_1_s_608 = 0; i_2_1_s_608 < 4; ++i_2_1_s_608) {
    Y_local[(i_2_1_s_608 * 16)] = (Y_local[(i_2_1_s_608 * 16)] + (A_shared_dyn_local[(i_2_1_s_608 + 6)] * B_shared_dyn_local[16]));
  }
  for (int i_2_1_s_609 = 0; i_2_1_s_609 < 4; ++i_2_1_s_609) {
    if (i_2_1_s_609 < 2) {
      Y_local[((i_2_1_s_609 * 16) + 64)] = (Y_local[((i_2_1_s_609 * 16) + 64)] + (A_shared_dyn_local[(i_2_1_s_609 + 10)] * B_shared_dyn_local[16]));
    }
  }
  for (int i_2_1_s_610 = 0; i_2_1_s_610 < 4; ++i_2_1_s_610) {
    Y_local[((i_2_1_s_610 * 16) + 1)] = (Y_local[((i_2_1_s_610 * 16) + 1)] + (A_shared_dyn_local[(i_2_1_s_610 + 6)] * B_shared_dyn_local[17]));
  }
  for (int i_2_1_s_611 = 0; i_2_1_s_611 < 4; ++i_2_1_s_611) {
    if (i_2_1_s_611 < 2) {
      Y_local[((i_2_1_s_611 * 16) + 65)] = (Y_local[((i_2_1_s_611 * 16) + 65)] + (A_shared_dyn_local[(i_2_1_s_611 + 10)] * B_shared_dyn_local[17]));
    }
  }
  for (int i_2_1_s_612 = 0; i_2_1_s_612 < 4; ++i_2_1_s_612) {
    Y_local[((i_2_1_s_612 * 16) + 2)] = (Y_local[((i_2_1_s_612 * 16) + 2)] + (A_shared_dyn_local[(i_2_1_s_612 + 6)] * B_shared_dyn_local[18]));
  }
  for (int i_2_1_s_613 = 0; i_2_1_s_613 < 4; ++i_2_1_s_613) {
    if (i_2_1_s_613 < 2) {
      Y_local[((i_2_1_s_613 * 16) + 66)] = (Y_local[((i_2_1_s_613 * 16) + 66)] + (A_shared_dyn_local[(i_2_1_s_613 + 10)] * B_shared_dyn_local[18]));
    }
  }
  for (int i_2_1_s_614 = 0; i_2_1_s_614 < 4; ++i_2_1_s_614) {
    Y_local[((i_2_1_s_614 * 16) + 3)] = (Y_local[((i_2_1_s_614 * 16) + 3)] + (A_shared_dyn_local[(i_2_1_s_614 + 6)] * B_shared_dyn_local[19]));
  }
  for (int i_2_1_s_615 = 0; i_2_1_s_615 < 4; ++i_2_1_s_615) {
    if (i_2_1_s_615 < 2) {
      Y_local[((i_2_1_s_615 * 16) + 67)] = (Y_local[((i_2_1_s_615 * 16) + 67)] + (A_shared_dyn_local[(i_2_1_s_615 + 10)] * B_shared_dyn_local[19]));
    }
  }
  for (int i_2_1_s_616 = 0; i_2_1_s_616 < 4; ++i_2_1_s_616) {
    Y_local[((i_2_1_s_616 * 16) + 4)] = (Y_local[((i_2_1_s_616 * 16) + 4)] + (A_shared_dyn_local[(i_2_1_s_616 + 6)] * B_shared_dyn_local[20]));
  }
  for (int i_2_1_s_617 = 0; i_2_1_s_617 < 4; ++i_2_1_s_617) {
    if (i_2_1_s_617 < 2) {
      Y_local[((i_2_1_s_617 * 16) + 68)] = (Y_local[((i_2_1_s_617 * 16) + 68)] + (A_shared_dyn_local[(i_2_1_s_617 + 10)] * B_shared_dyn_local[20]));
    }
  }
  for (int i_2_1_s_618 = 0; i_2_1_s_618 < 4; ++i_2_1_s_618) {
    Y_local[((i_2_1_s_618 * 16) + 5)] = (Y_local[((i_2_1_s_618 * 16) + 5)] + (A_shared_dyn_local[(i_2_1_s_618 + 6)] * B_shared_dyn_local[21]));
  }
  for (int i_2_1_s_619 = 0; i_2_1_s_619 < 4; ++i_2_1_s_619) {
    if (i_2_1_s_619 < 2) {
      Y_local[((i_2_1_s_619 * 16) + 69)] = (Y_local[((i_2_1_s_619 * 16) + 69)] + (A_shared_dyn_local[(i_2_1_s_619 + 10)] * B_shared_dyn_local[21]));
    }
  }
  for (int i_2_1_s_620 = 0; i_2_1_s_620 < 4; ++i_2_1_s_620) {
    Y_local[((i_2_1_s_620 * 16) + 6)] = (Y_local[((i_2_1_s_620 * 16) + 6)] + (A_shared_dyn_local[(i_2_1_s_620 + 6)] * B_shared_dyn_local[22]));
  }
  for (int i_2_1_s_621 = 0; i_2_1_s_621 < 4; ++i_2_1_s_621) {
    if (i_2_1_s_621 < 2) {
      Y_local[((i_2_1_s_621 * 16) + 70)] = (Y_local[((i_2_1_s_621 * 16) + 70)] + (A_shared_dyn_local[(i_2_1_s_621 + 10)] * B_shared_dyn_local[22]));
    }
  }
  for (int i_2_1_s_622 = 0; i_2_1_s_622 < 4; ++i_2_1_s_622) {
    Y_local[((i_2_1_s_622 * 16) + 7)] = (Y_local[((i_2_1_s_622 * 16) + 7)] + (A_shared_dyn_local[(i_2_1_s_622 + 6)] * B_shared_dyn_local[23]));
  }
  for (int i_2_1_s_623 = 0; i_2_1_s_623 < 4; ++i_2_1_s_623) {
    if (i_2_1_s_623 < 2) {
      Y_local[((i_2_1_s_623 * 16) + 71)] = (Y_local[((i_2_1_s_623 * 16) + 71)] + (A_shared_dyn_local[(i_2_1_s_623 + 10)] * B_shared_dyn_local[23]));
    }
  }
  for (int i_2_1_s_624 = 0; i_2_1_s_624 < 4; ++i_2_1_s_624) {
    Y_local[((i_2_1_s_624 * 16) + 8)] = (Y_local[((i_2_1_s_624 * 16) + 8)] + (A_shared_dyn_local[(i_2_1_s_624 + 6)] * B_shared_dyn_local[24]));
  }
  for (int i_2_1_s_625 = 0; i_2_1_s_625 < 4; ++i_2_1_s_625) {
    if (i_2_1_s_625 < 2) {
      Y_local[((i_2_1_s_625 * 16) + 72)] = (Y_local[((i_2_1_s_625 * 16) + 72)] + (A_shared_dyn_local[(i_2_1_s_625 + 10)] * B_shared_dyn_local[24]));
    }
  }
  for (int i_2_1_s_626 = 0; i_2_1_s_626 < 4; ++i_2_1_s_626) {
    Y_local[((i_2_1_s_626 * 16) + 9)] = (Y_local[((i_2_1_s_626 * 16) + 9)] + (A_shared_dyn_local[(i_2_1_s_626 + 6)] * B_shared_dyn_local[25]));
  }
  for (int i_2_1_s_627 = 0; i_2_1_s_627 < 4; ++i_2_1_s_627) {
    if (i_2_1_s_627 < 2) {
      Y_local[((i_2_1_s_627 * 16) + 73)] = (Y_local[((i_2_1_s_627 * 16) + 73)] + (A_shared_dyn_local[(i_2_1_s_627 + 10)] * B_shared_dyn_local[25]));
    }
  }
  for (int i_2_1_s_628 = 0; i_2_1_s_628 < 4; ++i_2_1_s_628) {
    Y_local[((i_2_1_s_628 * 16) + 10)] = (Y_local[((i_2_1_s_628 * 16) + 10)] + (A_shared_dyn_local[(i_2_1_s_628 + 6)] * B_shared_dyn_local[26]));
  }
  for (int i_2_1_s_629 = 0; i_2_1_s_629 < 4; ++i_2_1_s_629) {
    if (i_2_1_s_629 < 2) {
      Y_local[((i_2_1_s_629 * 16) + 74)] = (Y_local[((i_2_1_s_629 * 16) + 74)] + (A_shared_dyn_local[(i_2_1_s_629 + 10)] * B_shared_dyn_local[26]));
    }
  }
  for (int i_2_1_s_630 = 0; i_2_1_s_630 < 4; ++i_2_1_s_630) {
    Y_local[((i_2_1_s_630 * 16) + 11)] = (Y_local[((i_2_1_s_630 * 16) + 11)] + (A_shared_dyn_local[(i_2_1_s_630 + 6)] * B_shared_dyn_local[27]));
  }
  for (int i_2_1_s_631 = 0; i_2_1_s_631 < 4; ++i_2_1_s_631) {
    if (i_2_1_s_631 < 2) {
      Y_local[((i_2_1_s_631 * 16) + 75)] = (Y_local[((i_2_1_s_631 * 16) + 75)] + (A_shared_dyn_local[(i_2_1_s_631 + 10)] * B_shared_dyn_local[27]));
    }
  }
  for (int i_2_1_s_632 = 0; i_2_1_s_632 < 4; ++i_2_1_s_632) {
    Y_local[((i_2_1_s_632 * 16) + 12)] = (Y_local[((i_2_1_s_632 * 16) + 12)] + (A_shared_dyn_local[(i_2_1_s_632 + 6)] * B_shared_dyn_local[28]));
  }
  for (int i_2_1_s_633 = 0; i_2_1_s_633 < 4; ++i_2_1_s_633) {
    if (i_2_1_s_633 < 2) {
      Y_local[((i_2_1_s_633 * 16) + 76)] = (Y_local[((i_2_1_s_633 * 16) + 76)] + (A_shared_dyn_local[(i_2_1_s_633 + 10)] * B_shared_dyn_local[28]));
    }
  }
  for (int i_2_1_s_634 = 0; i_2_1_s_634 < 4; ++i_2_1_s_634) {
    Y_local[((i_2_1_s_634 * 16) + 13)] = (Y_local[((i_2_1_s_634 * 16) + 13)] + (A_shared_dyn_local[(i_2_1_s_634 + 6)] * B_shared_dyn_local[29]));
  }
  for (int i_2_1_s_635 = 0; i_2_1_s_635 < 4; ++i_2_1_s_635) {
    if (i_2_1_s_635 < 2) {
      Y_local[((i_2_1_s_635 * 16) + 77)] = (Y_local[((i_2_1_s_635 * 16) + 77)] + (A_shared_dyn_local[(i_2_1_s_635 + 10)] * B_shared_dyn_local[29]));
    }
  }
  for (int i_2_1_s_636 = 0; i_2_1_s_636 < 4; ++i_2_1_s_636) {
    Y_local[((i_2_1_s_636 * 16) + 14)] = (Y_local[((i_2_1_s_636 * 16) + 14)] + (A_shared_dyn_local[(i_2_1_s_636 + 6)] * B_shared_dyn_local[30]));
  }
  for (int i_2_1_s_637 = 0; i_2_1_s_637 < 4; ++i_2_1_s_637) {
    if (i_2_1_s_637 < 2) {
      Y_local[((i_2_1_s_637 * 16) + 78)] = (Y_local[((i_2_1_s_637 * 16) + 78)] + (A_shared_dyn_local[(i_2_1_s_637 + 10)] * B_shared_dyn_local[30]));
    }
  }
  for (int i_2_1_s_638 = 0; i_2_1_s_638 < 4; ++i_2_1_s_638) {
    Y_local[((i_2_1_s_638 * 16) + 15)] = (Y_local[((i_2_1_s_638 * 16) + 15)] + (A_shared_dyn_local[(i_2_1_s_638 + 6)] * B_shared_dyn_local[31]));
  }
  for (int i_2_1_s_639 = 0; i_2_1_s_639 < 4; ++i_2_1_s_639) {
    if (i_2_1_s_639 < 2) {
      Y_local[((i_2_1_s_639 * 16) + 79)] = (Y_local[((i_2_1_s_639 * 16) + 79)] + (A_shared_dyn_local[(i_2_1_s_639 + 10)] * B_shared_dyn_local[31]));
    }
  }
  for (int ax1_0_20 = 0; ax1_0_20 < 4; ++ax1_0_20) {
    *(float4*)(Y + (((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_20 * 4))) = *(float4*)(Y_local + (ax1_0_20 * 4));
  }
  for (int ax1_0_21 = 0; ax1_0_21 < 4; ++ax1_0_21) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_21 * 4)) + 1152)) = *(float4*)(Y_local + ((ax1_0_21 * 4) + 16));
  }
  for (int ax1_0_22 = 0; ax1_0_22 < 4; ++ax1_0_22) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_22 * 4)) + 2304)) = *(float4*)(Y_local + ((ax1_0_22 * 4) + 32));
  }
  for (int ax1_0_23 = 0; ax1_0_23 < 4; ++ax1_0_23) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_23 * 4)) + 3456)) = *(float4*)(Y_local + ((ax1_0_23 * 4) + 48));
  }
  for (int ax1_0_24 = 0; ax1_0_24 < 4; ++ax1_0_24) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_24 * 4)) + 4608)) = *(float4*)(Y_local + ((ax1_0_24 * 4) + 64));
  }
  for (int ax1_0_25 = 0; ax1_0_25 < 4; ++ax1_0_25) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 3) * 165888) + ((((int)threadIdx.x) / 27) * 20736)) + ((((int)threadIdx.x) % 3) * 6912)) + ((((int)blockIdx.x) & 7) * 144)) + (((((int)threadIdx.x) % 27) / 3) * 16)) + (ax1_0_25 * 4)) + 5760)) = *(float4*)(Y_local + ((ax1_0_25 * 4) + 80));
  }
}


