
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[28];
  float A_shared_dyn_local[14];
  float B_shared_dyn_local[8];
  Y_local[0] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[24] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[25] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[26] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;
  Y_local[27] = 0.000000e+00f;
  for (int ax0_ax1_fused_2_s = 0; ax0_ax1_fused_2_s < 2; ++ax0_ax1_fused_2_s) {
    if (((int)threadIdx.x) < 448) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + (ax0_ax1_fused_2_s * 4)) + 20480)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) >> 2) * 896)) + ((((int)threadIdx.x) & 3) * 2)) + ax0_ax1_fused_2_s))), "n"(4)
    );
  }
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 5) * 256) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)threadIdx.x) & 1) * 8))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 6) * 896) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) & 63) * 2)))), "n"(8)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_1 = 0; ax0_ax1_fused_2_s_1 < 2; ++ax0_ax1_fused_2_s_1) {
    if (((int)threadIdx.x) < 448) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + (ax0_ax1_fused_2_s_1 * 4)) + 29440)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) >> 2) * 896)) + ((((int)threadIdx.x) & 3) * 2)) + ax0_ax1_fused_2_s_1) + 8))), "n"(4)
    );
  }
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 5) * 256) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 6) * 896) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 7168))), "n"(8)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_2 = 0; ax0_ax1_fused_2_s_2 < 2; ++ax0_ax1_fused_2_s_2) {
    if (((int)threadIdx.x) < 448) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + (ax0_ax1_fused_2_s_2 * 4)) + 38400)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) >> 2) * 896)) + ((((int)threadIdx.x) & 3) * 2)) + ax0_ax1_fused_2_s_2) + 16))), "n"(4)
    );
  }
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 5) * 256) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 8192)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 6) * 896) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 14336))), "n"(8)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

  for (int ax0_ax1_fused_2_s_3 = 0; ax0_ax1_fused_2_s_3 < 2; ++ax0_ax1_fused_2_s_3) {
    if (((int)threadIdx.x) < 448) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + (ax0_ax1_fused_2_s_3 * 4)) + 47360)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) >> 2) * 896)) + ((((int)threadIdx.x) & 3) * 2)) + ax0_ax1_fused_2_s_3) + 24))), "n"(4)
    );
  }
    }
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((int)threadIdx.x) >> 5) * 256) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)threadIdx.x) & 1) * 8)) + 12288)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 6) * 896) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 21504))), "n"(8)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

  __syncthreads();
  for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
    for (int ax0_1_s = 0; ax0_1_s < 4; ++ax0_1_s) {
      if (((ax0_0 * 4) + ax0_1_s) < 7) {
        A_shared_dyn_local[((ax0_0 * 4) + ax0_1_s)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0 * 80)) + (ax0_1_s * 20)) + 5120)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)));
  for (int k_0 = 0; k_0 < 108; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_2_s_4 = 0; ax0_ax1_fused_2_s_4 < 2; ++ax0_ax1_fused_2_s_4) {
      if (((int)threadIdx.x) < 448) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((((k_0 + 4) % 5) * 8960) + ((((int)threadIdx.x) >> 2) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + (ax0_ax1_fused_2_s_4 * 4)) + 20480)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((((int)blockIdx.x) / 7) * 100352) + ((((int)threadIdx.x) >> 2) * 896)) + (k_0 * 8)) + ((((int)threadIdx.x) & 3) * 2)) + ax0_ax1_fused_2_s_4) + 32))), "n"(4)
    );
  }
      }
    }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((k_0 + 4) % 5) * 4096) + ((((int)threadIdx.x) >> 5) * 256)) + (((((int)threadIdx.x) & 3) >> 1) * 128)) + (((((int)threadIdx.x) & 31) >> 2) * 16)) + ((((int)threadIdx.x) & 1) * 8))))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0 * 7168) + ((((int)threadIdx.x) >> 6) * 896)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 28672))), "n"(8)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    for (int ax0_0_1 = 0; ax0_0_1 < 2; ++ax0_0_1) {
      for (int ax0_1_s_1 = 0; ax0_1_s_1 < 4; ++ax0_1_s_1) {
        if (((ax0_0_1 * 4) + ax0_1_s_1) < 7) {
          A_shared_dyn_local[(((ax0_0_1 * 4) + ax0_1_s_1) + 7)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_1 * 80)) + (ax0_1_s_1 * 20)) + 5121)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 128));
    for (int i_2_1_s = 0; i_2_1_s < 4; ++i_2_1_s) {
      Y_local[(i_2_1_s * 4)] = (Y_local[(i_2_1_s * 4)] + (A_shared_dyn_local[i_2_1_s] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_1 = 0; i_2_1_s_1 < 4; ++i_2_1_s_1) {
      if (i_2_1_s_1 < 3) {
        Y_local[((i_2_1_s_1 * 4) + 16)] = (Y_local[((i_2_1_s_1 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_1 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_2 = 0; i_2_1_s_2 < 4; ++i_2_1_s_2) {
      Y_local[((i_2_1_s_2 * 4) + 1)] = (Y_local[((i_2_1_s_2 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_2] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_3 = 0; i_2_1_s_3 < 4; ++i_2_1_s_3) {
      if (i_2_1_s_3 < 3) {
        Y_local[((i_2_1_s_3 * 4) + 17)] = (Y_local[((i_2_1_s_3 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_3 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_4 = 0; i_2_1_s_4 < 4; ++i_2_1_s_4) {
      Y_local[((i_2_1_s_4 * 4) + 2)] = (Y_local[((i_2_1_s_4 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_4] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_5 = 0; i_2_1_s_5 < 4; ++i_2_1_s_5) {
      if (i_2_1_s_5 < 3) {
        Y_local[((i_2_1_s_5 * 4) + 18)] = (Y_local[((i_2_1_s_5 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_5 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_6 = 0; i_2_1_s_6 < 4; ++i_2_1_s_6) {
      Y_local[((i_2_1_s_6 * 4) + 3)] = (Y_local[((i_2_1_s_6 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_6] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_7 = 0; i_2_1_s_7 < 4; ++i_2_1_s_7) {
      if (i_2_1_s_7 < 3) {
        Y_local[((i_2_1_s_7 * 4) + 19)] = (Y_local[((i_2_1_s_7 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_7 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int ax0_0_2 = 0; ax0_0_2 < 2; ++ax0_0_2) {
      for (int ax0_1_s_2 = 0; ax0_1_s_2 < 4; ++ax0_1_s_2) {
        if (((ax0_0_2 * 4) + ax0_1_s_2) < 7) {
          A_shared_dyn_local[((ax0_0_2 * 4) + ax0_1_s_2)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_2 * 80)) + (ax0_1_s_2 * 20)) + 5122)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 256));
    for (int i_2_1_s_8 = 0; i_2_1_s_8 < 4; ++i_2_1_s_8) {
      Y_local[(i_2_1_s_8 * 4)] = (Y_local[(i_2_1_s_8 * 4)] + (A_shared_dyn_local[(i_2_1_s_8 + 7)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_9 = 0; i_2_1_s_9 < 4; ++i_2_1_s_9) {
      if (i_2_1_s_9 < 3) {
        Y_local[((i_2_1_s_9 * 4) + 16)] = (Y_local[((i_2_1_s_9 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_9 + 11)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_10 = 0; i_2_1_s_10 < 4; ++i_2_1_s_10) {
      Y_local[((i_2_1_s_10 * 4) + 1)] = (Y_local[((i_2_1_s_10 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_10 + 7)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_11 = 0; i_2_1_s_11 < 4; ++i_2_1_s_11) {
      if (i_2_1_s_11 < 3) {
        Y_local[((i_2_1_s_11 * 4) + 17)] = (Y_local[((i_2_1_s_11 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_11 + 11)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_12 = 0; i_2_1_s_12 < 4; ++i_2_1_s_12) {
      Y_local[((i_2_1_s_12 * 4) + 2)] = (Y_local[((i_2_1_s_12 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_12 + 7)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_13 = 0; i_2_1_s_13 < 4; ++i_2_1_s_13) {
      if (i_2_1_s_13 < 3) {
        Y_local[((i_2_1_s_13 * 4) + 18)] = (Y_local[((i_2_1_s_13 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_13 + 11)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_14 = 0; i_2_1_s_14 < 4; ++i_2_1_s_14) {
      Y_local[((i_2_1_s_14 * 4) + 3)] = (Y_local[((i_2_1_s_14 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_14 + 7)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_15 = 0; i_2_1_s_15 < 4; ++i_2_1_s_15) {
      if (i_2_1_s_15 < 3) {
        Y_local[((i_2_1_s_15 * 4) + 19)] = (Y_local[((i_2_1_s_15 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_15 + 11)] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_0_3 = 0; ax0_0_3 < 2; ++ax0_0_3) {
      for (int ax0_1_s_3 = 0; ax0_1_s_3 < 4; ++ax0_1_s_3) {
        if (((ax0_0_3 * 4) + ax0_1_s_3) < 7) {
          A_shared_dyn_local[(((ax0_0_3 * 4) + ax0_1_s_3) + 7)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_3 * 80)) + (ax0_1_s_3 * 20)) + 5123)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 384));
    for (int i_2_1_s_16 = 0; i_2_1_s_16 < 4; ++i_2_1_s_16) {
      Y_local[(i_2_1_s_16 * 4)] = (Y_local[(i_2_1_s_16 * 4)] + (A_shared_dyn_local[i_2_1_s_16] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_17 = 0; i_2_1_s_17 < 4; ++i_2_1_s_17) {
      if (i_2_1_s_17 < 3) {
        Y_local[((i_2_1_s_17 * 4) + 16)] = (Y_local[((i_2_1_s_17 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_17 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_18 = 0; i_2_1_s_18 < 4; ++i_2_1_s_18) {
      Y_local[((i_2_1_s_18 * 4) + 1)] = (Y_local[((i_2_1_s_18 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_18] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_19 = 0; i_2_1_s_19 < 4; ++i_2_1_s_19) {
      if (i_2_1_s_19 < 3) {
        Y_local[((i_2_1_s_19 * 4) + 17)] = (Y_local[((i_2_1_s_19 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_19 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_20 = 0; i_2_1_s_20 < 4; ++i_2_1_s_20) {
      Y_local[((i_2_1_s_20 * 4) + 2)] = (Y_local[((i_2_1_s_20 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_20] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_21 = 0; i_2_1_s_21 < 4; ++i_2_1_s_21) {
      if (i_2_1_s_21 < 3) {
        Y_local[((i_2_1_s_21 * 4) + 18)] = (Y_local[((i_2_1_s_21 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_21 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_22 = 0; i_2_1_s_22 < 4; ++i_2_1_s_22) {
      Y_local[((i_2_1_s_22 * 4) + 3)] = (Y_local[((i_2_1_s_22 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_22] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_23 = 0; i_2_1_s_23 < 4; ++i_2_1_s_23) {
      if (i_2_1_s_23 < 3) {
        Y_local[((i_2_1_s_23 * 4) + 19)] = (Y_local[((i_2_1_s_23 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_23 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int ax0_0_4 = 0; ax0_0_4 < 2; ++ax0_0_4) {
      for (int ax0_1_s_4 = 0; ax0_1_s_4 < 4; ++ax0_1_s_4) {
        if (((ax0_0_4 * 4) + ax0_1_s_4) < 7) {
          A_shared_dyn_local[((ax0_0_4 * 4) + ax0_1_s_4)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_4 * 80)) + (ax0_1_s_4 * 20)) + 5136)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 512));
    for (int i_2_1_s_24 = 0; i_2_1_s_24 < 4; ++i_2_1_s_24) {
      Y_local[(i_2_1_s_24 * 4)] = (Y_local[(i_2_1_s_24 * 4)] + (A_shared_dyn_local[(i_2_1_s_24 + 7)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_25 = 0; i_2_1_s_25 < 4; ++i_2_1_s_25) {
      if (i_2_1_s_25 < 3) {
        Y_local[((i_2_1_s_25 * 4) + 16)] = (Y_local[((i_2_1_s_25 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_25 + 11)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_26 = 0; i_2_1_s_26 < 4; ++i_2_1_s_26) {
      Y_local[((i_2_1_s_26 * 4) + 1)] = (Y_local[((i_2_1_s_26 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_26 + 7)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_27 = 0; i_2_1_s_27 < 4; ++i_2_1_s_27) {
      if (i_2_1_s_27 < 3) {
        Y_local[((i_2_1_s_27 * 4) + 17)] = (Y_local[((i_2_1_s_27 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_27 + 11)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_28 = 0; i_2_1_s_28 < 4; ++i_2_1_s_28) {
      Y_local[((i_2_1_s_28 * 4) + 2)] = (Y_local[((i_2_1_s_28 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_28 + 7)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_29 = 0; i_2_1_s_29 < 4; ++i_2_1_s_29) {
      if (i_2_1_s_29 < 3) {
        Y_local[((i_2_1_s_29 * 4) + 18)] = (Y_local[((i_2_1_s_29 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_29 + 11)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_30 = 0; i_2_1_s_30 < 4; ++i_2_1_s_30) {
      Y_local[((i_2_1_s_30 * 4) + 3)] = (Y_local[((i_2_1_s_30 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_30 + 7)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_31 = 0; i_2_1_s_31 < 4; ++i_2_1_s_31) {
      if (i_2_1_s_31 < 3) {
        Y_local[((i_2_1_s_31 * 4) + 19)] = (Y_local[((i_2_1_s_31 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_31 + 11)] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_0_5 = 0; ax0_0_5 < 2; ++ax0_0_5) {
      for (int ax0_1_s_5 = 0; ax0_1_s_5 < 4; ++ax0_1_s_5) {
        if (((ax0_0_5 * 4) + ax0_1_s_5) < 7) {
          A_shared_dyn_local[(((ax0_0_5 * 4) + ax0_1_s_5) + 7)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_5 * 80)) + (ax0_1_s_5 * 20)) + 5137)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 640));
    for (int i_2_1_s_32 = 0; i_2_1_s_32 < 4; ++i_2_1_s_32) {
      Y_local[(i_2_1_s_32 * 4)] = (Y_local[(i_2_1_s_32 * 4)] + (A_shared_dyn_local[i_2_1_s_32] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_33 = 0; i_2_1_s_33 < 4; ++i_2_1_s_33) {
      if (i_2_1_s_33 < 3) {
        Y_local[((i_2_1_s_33 * 4) + 16)] = (Y_local[((i_2_1_s_33 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_33 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_34 = 0; i_2_1_s_34 < 4; ++i_2_1_s_34) {
      Y_local[((i_2_1_s_34 * 4) + 1)] = (Y_local[((i_2_1_s_34 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_34] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_35 = 0; i_2_1_s_35 < 4; ++i_2_1_s_35) {
      if (i_2_1_s_35 < 3) {
        Y_local[((i_2_1_s_35 * 4) + 17)] = (Y_local[((i_2_1_s_35 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_35 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_36 = 0; i_2_1_s_36 < 4; ++i_2_1_s_36) {
      Y_local[((i_2_1_s_36 * 4) + 2)] = (Y_local[((i_2_1_s_36 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_36] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_37 = 0; i_2_1_s_37 < 4; ++i_2_1_s_37) {
      if (i_2_1_s_37 < 3) {
        Y_local[((i_2_1_s_37 * 4) + 18)] = (Y_local[((i_2_1_s_37 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_37 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_38 = 0; i_2_1_s_38 < 4; ++i_2_1_s_38) {
      Y_local[((i_2_1_s_38 * 4) + 3)] = (Y_local[((i_2_1_s_38 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_38] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_39 = 0; i_2_1_s_39 < 4; ++i_2_1_s_39) {
      if (i_2_1_s_39 < 3) {
        Y_local[((i_2_1_s_39 * 4) + 19)] = (Y_local[((i_2_1_s_39 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_39 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int ax0_0_6 = 0; ax0_0_6 < 2; ++ax0_0_6) {
      for (int ax0_1_s_6 = 0; ax0_1_s_6 < 4; ++ax0_1_s_6) {
        if (((ax0_0_6 * 4) + ax0_1_s_6) < 7) {
          A_shared_dyn_local[((ax0_0_6 * 4) + ax0_1_s_6)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_6 * 80)) + (ax0_1_s_6 * 20)) + 5138)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 768));
    for (int i_2_1_s_40 = 0; i_2_1_s_40 < 4; ++i_2_1_s_40) {
      Y_local[(i_2_1_s_40 * 4)] = (Y_local[(i_2_1_s_40 * 4)] + (A_shared_dyn_local[(i_2_1_s_40 + 7)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_41 = 0; i_2_1_s_41 < 4; ++i_2_1_s_41) {
      if (i_2_1_s_41 < 3) {
        Y_local[((i_2_1_s_41 * 4) + 16)] = (Y_local[((i_2_1_s_41 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_41 + 11)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_42 = 0; i_2_1_s_42 < 4; ++i_2_1_s_42) {
      Y_local[((i_2_1_s_42 * 4) + 1)] = (Y_local[((i_2_1_s_42 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_42 + 7)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_43 = 0; i_2_1_s_43 < 4; ++i_2_1_s_43) {
      if (i_2_1_s_43 < 3) {
        Y_local[((i_2_1_s_43 * 4) + 17)] = (Y_local[((i_2_1_s_43 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_43 + 11)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_44 = 0; i_2_1_s_44 < 4; ++i_2_1_s_44) {
      Y_local[((i_2_1_s_44 * 4) + 2)] = (Y_local[((i_2_1_s_44 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_44 + 7)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_45 = 0; i_2_1_s_45 < 4; ++i_2_1_s_45) {
      if (i_2_1_s_45 < 3) {
        Y_local[((i_2_1_s_45 * 4) + 18)] = (Y_local[((i_2_1_s_45 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_45 + 11)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_46 = 0; i_2_1_s_46 < 4; ++i_2_1_s_46) {
      Y_local[((i_2_1_s_46 * 4) + 3)] = (Y_local[((i_2_1_s_46 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_46 + 7)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_47 = 0; i_2_1_s_47 < 4; ++i_2_1_s_47) {
      if (i_2_1_s_47 < 3) {
        Y_local[((i_2_1_s_47 * 4) + 19)] = (Y_local[((i_2_1_s_47 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_47 + 11)] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_0_7 = 0; ax0_0_7 < 2; ++ax0_0_7) {
      for (int ax0_1_s_7 = 0; ax0_1_s_7 < 4; ++ax0_1_s_7) {
        if (((ax0_0_7 * 4) + ax0_1_s_7) < 7) {
          A_shared_dyn_local[(((ax0_0_7 * 4) + ax0_1_s_7) + 7)] = ((float*)buf_dyn_shmem)[(((((((k_0 % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_7 * 80)) + (ax0_1_s_7 * 20)) + 5139)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 896));
    for (int i_2_1_s_48 = 0; i_2_1_s_48 < 4; ++i_2_1_s_48) {
      Y_local[(i_2_1_s_48 * 4)] = (Y_local[(i_2_1_s_48 * 4)] + (A_shared_dyn_local[i_2_1_s_48] * B_shared_dyn_local[0]));
    }
    for (int i_2_1_s_49 = 0; i_2_1_s_49 < 4; ++i_2_1_s_49) {
      if (i_2_1_s_49 < 3) {
        Y_local[((i_2_1_s_49 * 4) + 16)] = (Y_local[((i_2_1_s_49 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_49 + 4)] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_50 = 0; i_2_1_s_50 < 4; ++i_2_1_s_50) {
      Y_local[((i_2_1_s_50 * 4) + 1)] = (Y_local[((i_2_1_s_50 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_50] * B_shared_dyn_local[1]));
    }
    for (int i_2_1_s_51 = 0; i_2_1_s_51 < 4; ++i_2_1_s_51) {
      if (i_2_1_s_51 < 3) {
        Y_local[((i_2_1_s_51 * 4) + 17)] = (Y_local[((i_2_1_s_51 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_51 + 4)] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_52 = 0; i_2_1_s_52 < 4; ++i_2_1_s_52) {
      Y_local[((i_2_1_s_52 * 4) + 2)] = (Y_local[((i_2_1_s_52 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_52] * B_shared_dyn_local[2]));
    }
    for (int i_2_1_s_53 = 0; i_2_1_s_53 < 4; ++i_2_1_s_53) {
      if (i_2_1_s_53 < 3) {
        Y_local[((i_2_1_s_53 * 4) + 18)] = (Y_local[((i_2_1_s_53 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_53 + 4)] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_54 = 0; i_2_1_s_54 < 4; ++i_2_1_s_54) {
      Y_local[((i_2_1_s_54 * 4) + 3)] = (Y_local[((i_2_1_s_54 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_54] * B_shared_dyn_local[3]));
    }
    for (int i_2_1_s_55 = 0; i_2_1_s_55 < 4; ++i_2_1_s_55) {
      if (i_2_1_s_55 < 3) {
        Y_local[((i_2_1_s_55 * 4) + 19)] = (Y_local[((i_2_1_s_55 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_55 + 4)] * B_shared_dyn_local[3]));
      }
    }
    for (int ax0_0_8 = 0; ax0_0_8 < 2; ++ax0_0_8) {
      for (int ax0_1_s_8 = 0; ax0_1_s_8 < 4; ++ax0_1_s_8) {
        if (((ax0_0_8 * 4) + ax0_1_s_8) < 7) {
          A_shared_dyn_local[((ax0_0_8 * 4) + ax0_1_s_8)] = ((float*)buf_dyn_shmem)[((((((((k_0 + 1) % 5) * 2240) + (((((int)threadIdx.x) & 63) >> 3) * 280)) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_8 * 80)) + (ax0_1_s_8 * 20)) + 5120)];
        }
      }
    }
    *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((k_0 + 1) % 5) * 1024) + ((((int)threadIdx.x) >> 8) * 64)) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)));
    for (int i_2_1_s_56 = 0; i_2_1_s_56 < 4; ++i_2_1_s_56) {
      Y_local[(i_2_1_s_56 * 4)] = (Y_local[(i_2_1_s_56 * 4)] + (A_shared_dyn_local[(i_2_1_s_56 + 7)] * B_shared_dyn_local[4]));
    }
    for (int i_2_1_s_57 = 0; i_2_1_s_57 < 4; ++i_2_1_s_57) {
      if (i_2_1_s_57 < 3) {
        Y_local[((i_2_1_s_57 * 4) + 16)] = (Y_local[((i_2_1_s_57 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_57 + 11)] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_58 = 0; i_2_1_s_58 < 4; ++i_2_1_s_58) {
      Y_local[((i_2_1_s_58 * 4) + 1)] = (Y_local[((i_2_1_s_58 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_58 + 7)] * B_shared_dyn_local[5]));
    }
    for (int i_2_1_s_59 = 0; i_2_1_s_59 < 4; ++i_2_1_s_59) {
      if (i_2_1_s_59 < 3) {
        Y_local[((i_2_1_s_59 * 4) + 17)] = (Y_local[((i_2_1_s_59 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_59 + 11)] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_60 = 0; i_2_1_s_60 < 4; ++i_2_1_s_60) {
      Y_local[((i_2_1_s_60 * 4) + 2)] = (Y_local[((i_2_1_s_60 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_60 + 7)] * B_shared_dyn_local[6]));
    }
    for (int i_2_1_s_61 = 0; i_2_1_s_61 < 4; ++i_2_1_s_61) {
      if (i_2_1_s_61 < 3) {
        Y_local[((i_2_1_s_61 * 4) + 18)] = (Y_local[((i_2_1_s_61 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_61 + 11)] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_62 = 0; i_2_1_s_62 < 4; ++i_2_1_s_62) {
      Y_local[((i_2_1_s_62 * 4) + 3)] = (Y_local[((i_2_1_s_62 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_62 + 7)] * B_shared_dyn_local[7]));
    }
    for (int i_2_1_s_63 = 0; i_2_1_s_63 < 4; ++i_2_1_s_63) {
      if (i_2_1_s_63 < 3) {
        Y_local[((i_2_1_s_63 * 4) + 19)] = (Y_local[((i_2_1_s_63 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_63 + 11)] * B_shared_dyn_local[7]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax0_0_9 = 0; ax0_0_9 < 2; ++ax0_0_9) {
    for (int ax0_1_s_9 = 0; ax0_1_s_9 < 4; ++ax0_1_s_9) {
      if (((ax0_0_9 * 4) + ax0_1_s_9) < 7) {
        A_shared_dyn_local[(((ax0_0_9 * 4) + ax0_1_s_9) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_9 * 80)) + (ax0_1_s_9 * 20)) + 11841)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3200));
  for (int i_2_1_s_64 = 0; i_2_1_s_64 < 4; ++i_2_1_s_64) {
    Y_local[(i_2_1_s_64 * 4)] = (Y_local[(i_2_1_s_64 * 4)] + (A_shared_dyn_local[i_2_1_s_64] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_65 = 0; i_2_1_s_65 < 4; ++i_2_1_s_65) {
    if (i_2_1_s_65 < 3) {
      Y_local[((i_2_1_s_65 * 4) + 16)] = (Y_local[((i_2_1_s_65 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_65 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_66 = 0; i_2_1_s_66 < 4; ++i_2_1_s_66) {
    Y_local[((i_2_1_s_66 * 4) + 1)] = (Y_local[((i_2_1_s_66 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_66] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_67 = 0; i_2_1_s_67 < 4; ++i_2_1_s_67) {
    if (i_2_1_s_67 < 3) {
      Y_local[((i_2_1_s_67 * 4) + 17)] = (Y_local[((i_2_1_s_67 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_67 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_68 = 0; i_2_1_s_68 < 4; ++i_2_1_s_68) {
    Y_local[((i_2_1_s_68 * 4) + 2)] = (Y_local[((i_2_1_s_68 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_68] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_69 = 0; i_2_1_s_69 < 4; ++i_2_1_s_69) {
    if (i_2_1_s_69 < 3) {
      Y_local[((i_2_1_s_69 * 4) + 18)] = (Y_local[((i_2_1_s_69 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_69 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_70 = 0; i_2_1_s_70 < 4; ++i_2_1_s_70) {
    Y_local[((i_2_1_s_70 * 4) + 3)] = (Y_local[((i_2_1_s_70 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_70] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_71 = 0; i_2_1_s_71 < 4; ++i_2_1_s_71) {
    if (i_2_1_s_71 < 3) {
      Y_local[((i_2_1_s_71 * 4) + 19)] = (Y_local[((i_2_1_s_71 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_71 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_10 = 0; ax0_0_10 < 2; ++ax0_0_10) {
    for (int ax0_1_s_10 = 0; ax0_1_s_10 < 4; ++ax0_1_s_10) {
      if (((ax0_0_10 * 4) + ax0_1_s_10) < 7) {
        A_shared_dyn_local[((ax0_0_10 * 4) + ax0_1_s_10)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_10 * 80)) + (ax0_1_s_10 * 20)) + 11842)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3328));
  for (int i_2_1_s_72 = 0; i_2_1_s_72 < 4; ++i_2_1_s_72) {
    Y_local[(i_2_1_s_72 * 4)] = (Y_local[(i_2_1_s_72 * 4)] + (A_shared_dyn_local[(i_2_1_s_72 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_73 = 0; i_2_1_s_73 < 4; ++i_2_1_s_73) {
    if (i_2_1_s_73 < 3) {
      Y_local[((i_2_1_s_73 * 4) + 16)] = (Y_local[((i_2_1_s_73 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_73 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_74 = 0; i_2_1_s_74 < 4; ++i_2_1_s_74) {
    Y_local[((i_2_1_s_74 * 4) + 1)] = (Y_local[((i_2_1_s_74 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_74 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_75 = 0; i_2_1_s_75 < 4; ++i_2_1_s_75) {
    if (i_2_1_s_75 < 3) {
      Y_local[((i_2_1_s_75 * 4) + 17)] = (Y_local[((i_2_1_s_75 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_75 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_76 = 0; i_2_1_s_76 < 4; ++i_2_1_s_76) {
    Y_local[((i_2_1_s_76 * 4) + 2)] = (Y_local[((i_2_1_s_76 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_76 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_77 = 0; i_2_1_s_77 < 4; ++i_2_1_s_77) {
    if (i_2_1_s_77 < 3) {
      Y_local[((i_2_1_s_77 * 4) + 18)] = (Y_local[((i_2_1_s_77 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_77 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_78 = 0; i_2_1_s_78 < 4; ++i_2_1_s_78) {
    Y_local[((i_2_1_s_78 * 4) + 3)] = (Y_local[((i_2_1_s_78 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_78 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_79 = 0; i_2_1_s_79 < 4; ++i_2_1_s_79) {
    if (i_2_1_s_79 < 3) {
      Y_local[((i_2_1_s_79 * 4) + 19)] = (Y_local[((i_2_1_s_79 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_79 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_11 = 0; ax0_0_11 < 2; ++ax0_0_11) {
    for (int ax0_1_s_11 = 0; ax0_1_s_11 < 4; ++ax0_1_s_11) {
      if (((ax0_0_11 * 4) + ax0_1_s_11) < 7) {
        A_shared_dyn_local[(((ax0_0_11 * 4) + ax0_1_s_11) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_11 * 80)) + (ax0_1_s_11 * 20)) + 11843)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3456));
  for (int i_2_1_s_80 = 0; i_2_1_s_80 < 4; ++i_2_1_s_80) {
    Y_local[(i_2_1_s_80 * 4)] = (Y_local[(i_2_1_s_80 * 4)] + (A_shared_dyn_local[i_2_1_s_80] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_81 = 0; i_2_1_s_81 < 4; ++i_2_1_s_81) {
    if (i_2_1_s_81 < 3) {
      Y_local[((i_2_1_s_81 * 4) + 16)] = (Y_local[((i_2_1_s_81 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_81 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_82 = 0; i_2_1_s_82 < 4; ++i_2_1_s_82) {
    Y_local[((i_2_1_s_82 * 4) + 1)] = (Y_local[((i_2_1_s_82 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_82] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_83 = 0; i_2_1_s_83 < 4; ++i_2_1_s_83) {
    if (i_2_1_s_83 < 3) {
      Y_local[((i_2_1_s_83 * 4) + 17)] = (Y_local[((i_2_1_s_83 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_83 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_84 = 0; i_2_1_s_84 < 4; ++i_2_1_s_84) {
    Y_local[((i_2_1_s_84 * 4) + 2)] = (Y_local[((i_2_1_s_84 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_84] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_85 = 0; i_2_1_s_85 < 4; ++i_2_1_s_85) {
    if (i_2_1_s_85 < 3) {
      Y_local[((i_2_1_s_85 * 4) + 18)] = (Y_local[((i_2_1_s_85 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_85 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_86 = 0; i_2_1_s_86 < 4; ++i_2_1_s_86) {
    Y_local[((i_2_1_s_86 * 4) + 3)] = (Y_local[((i_2_1_s_86 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_86] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_87 = 0; i_2_1_s_87 < 4; ++i_2_1_s_87) {
    if (i_2_1_s_87 < 3) {
      Y_local[((i_2_1_s_87 * 4) + 19)] = (Y_local[((i_2_1_s_87 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_87 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_12 = 0; ax0_0_12 < 2; ++ax0_0_12) {
    for (int ax0_1_s_12 = 0; ax0_1_s_12 < 4; ++ax0_1_s_12) {
      if (((ax0_0_12 * 4) + ax0_1_s_12) < 7) {
        A_shared_dyn_local[((ax0_0_12 * 4) + ax0_1_s_12)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_12 * 80)) + (ax0_1_s_12 * 20)) + 11856)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3584));
  for (int i_2_1_s_88 = 0; i_2_1_s_88 < 4; ++i_2_1_s_88) {
    Y_local[(i_2_1_s_88 * 4)] = (Y_local[(i_2_1_s_88 * 4)] + (A_shared_dyn_local[(i_2_1_s_88 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_89 = 0; i_2_1_s_89 < 4; ++i_2_1_s_89) {
    if (i_2_1_s_89 < 3) {
      Y_local[((i_2_1_s_89 * 4) + 16)] = (Y_local[((i_2_1_s_89 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_89 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_90 = 0; i_2_1_s_90 < 4; ++i_2_1_s_90) {
    Y_local[((i_2_1_s_90 * 4) + 1)] = (Y_local[((i_2_1_s_90 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_90 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_91 = 0; i_2_1_s_91 < 4; ++i_2_1_s_91) {
    if (i_2_1_s_91 < 3) {
      Y_local[((i_2_1_s_91 * 4) + 17)] = (Y_local[((i_2_1_s_91 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_91 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_92 = 0; i_2_1_s_92 < 4; ++i_2_1_s_92) {
    Y_local[((i_2_1_s_92 * 4) + 2)] = (Y_local[((i_2_1_s_92 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_92 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_93 = 0; i_2_1_s_93 < 4; ++i_2_1_s_93) {
    if (i_2_1_s_93 < 3) {
      Y_local[((i_2_1_s_93 * 4) + 18)] = (Y_local[((i_2_1_s_93 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_93 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_94 = 0; i_2_1_s_94 < 4; ++i_2_1_s_94) {
    Y_local[((i_2_1_s_94 * 4) + 3)] = (Y_local[((i_2_1_s_94 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_94 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_95 = 0; i_2_1_s_95 < 4; ++i_2_1_s_95) {
    if (i_2_1_s_95 < 3) {
      Y_local[((i_2_1_s_95 * 4) + 19)] = (Y_local[((i_2_1_s_95 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_95 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_13 = 0; ax0_0_13 < 2; ++ax0_0_13) {
    for (int ax0_1_s_13 = 0; ax0_1_s_13 < 4; ++ax0_1_s_13) {
      if (((ax0_0_13 * 4) + ax0_1_s_13) < 7) {
        A_shared_dyn_local[(((ax0_0_13 * 4) + ax0_1_s_13) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_13 * 80)) + (ax0_1_s_13 * 20)) + 11857)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3712));
  for (int i_2_1_s_96 = 0; i_2_1_s_96 < 4; ++i_2_1_s_96) {
    Y_local[(i_2_1_s_96 * 4)] = (Y_local[(i_2_1_s_96 * 4)] + (A_shared_dyn_local[i_2_1_s_96] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_97 = 0; i_2_1_s_97 < 4; ++i_2_1_s_97) {
    if (i_2_1_s_97 < 3) {
      Y_local[((i_2_1_s_97 * 4) + 16)] = (Y_local[((i_2_1_s_97 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_97 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_98 = 0; i_2_1_s_98 < 4; ++i_2_1_s_98) {
    Y_local[((i_2_1_s_98 * 4) + 1)] = (Y_local[((i_2_1_s_98 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_98] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_99 = 0; i_2_1_s_99 < 4; ++i_2_1_s_99) {
    if (i_2_1_s_99 < 3) {
      Y_local[((i_2_1_s_99 * 4) + 17)] = (Y_local[((i_2_1_s_99 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_99 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_100 = 0; i_2_1_s_100 < 4; ++i_2_1_s_100) {
    Y_local[((i_2_1_s_100 * 4) + 2)] = (Y_local[((i_2_1_s_100 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_100] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_101 = 0; i_2_1_s_101 < 4; ++i_2_1_s_101) {
    if (i_2_1_s_101 < 3) {
      Y_local[((i_2_1_s_101 * 4) + 18)] = (Y_local[((i_2_1_s_101 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_101 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_102 = 0; i_2_1_s_102 < 4; ++i_2_1_s_102) {
    Y_local[((i_2_1_s_102 * 4) + 3)] = (Y_local[((i_2_1_s_102 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_102] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_103 = 0; i_2_1_s_103 < 4; ++i_2_1_s_103) {
    if (i_2_1_s_103 < 3) {
      Y_local[((i_2_1_s_103 * 4) + 19)] = (Y_local[((i_2_1_s_103 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_103 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_14 = 0; ax0_0_14 < 2; ++ax0_0_14) {
    for (int ax0_1_s_14 = 0; ax0_1_s_14 < 4; ++ax0_1_s_14) {
      if (((ax0_0_14 * 4) + ax0_1_s_14) < 7) {
        A_shared_dyn_local[((ax0_0_14 * 4) + ax0_1_s_14)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_14 * 80)) + (ax0_1_s_14 * 20)) + 11858)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3840));
  for (int i_2_1_s_104 = 0; i_2_1_s_104 < 4; ++i_2_1_s_104) {
    Y_local[(i_2_1_s_104 * 4)] = (Y_local[(i_2_1_s_104 * 4)] + (A_shared_dyn_local[(i_2_1_s_104 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_105 = 0; i_2_1_s_105 < 4; ++i_2_1_s_105) {
    if (i_2_1_s_105 < 3) {
      Y_local[((i_2_1_s_105 * 4) + 16)] = (Y_local[((i_2_1_s_105 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_105 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_106 = 0; i_2_1_s_106 < 4; ++i_2_1_s_106) {
    Y_local[((i_2_1_s_106 * 4) + 1)] = (Y_local[((i_2_1_s_106 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_106 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_107 = 0; i_2_1_s_107 < 4; ++i_2_1_s_107) {
    if (i_2_1_s_107 < 3) {
      Y_local[((i_2_1_s_107 * 4) + 17)] = (Y_local[((i_2_1_s_107 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_107 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_108 = 0; i_2_1_s_108 < 4; ++i_2_1_s_108) {
    Y_local[((i_2_1_s_108 * 4) + 2)] = (Y_local[((i_2_1_s_108 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_108 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_109 = 0; i_2_1_s_109 < 4; ++i_2_1_s_109) {
    if (i_2_1_s_109 < 3) {
      Y_local[((i_2_1_s_109 * 4) + 18)] = (Y_local[((i_2_1_s_109 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_109 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_110 = 0; i_2_1_s_110 < 4; ++i_2_1_s_110) {
    Y_local[((i_2_1_s_110 * 4) + 3)] = (Y_local[((i_2_1_s_110 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_110 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_111 = 0; i_2_1_s_111 < 4; ++i_2_1_s_111) {
    if (i_2_1_s_111 < 3) {
      Y_local[((i_2_1_s_111 * 4) + 19)] = (Y_local[((i_2_1_s_111 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_111 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_15 = 0; ax0_0_15 < 2; ++ax0_0_15) {
    for (int ax0_1_s_15 = 0; ax0_1_s_15 < 4; ++ax0_1_s_15) {
      if (((ax0_0_15 * 4) + ax0_1_s_15) < 7) {
        A_shared_dyn_local[(((ax0_0_15 * 4) + ax0_1_s_15) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_15 * 80)) + (ax0_1_s_15 * 20)) + 11859)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 3968));
  for (int i_2_1_s_112 = 0; i_2_1_s_112 < 4; ++i_2_1_s_112) {
    Y_local[(i_2_1_s_112 * 4)] = (Y_local[(i_2_1_s_112 * 4)] + (A_shared_dyn_local[i_2_1_s_112] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_113 = 0; i_2_1_s_113 < 4; ++i_2_1_s_113) {
    if (i_2_1_s_113 < 3) {
      Y_local[((i_2_1_s_113 * 4) + 16)] = (Y_local[((i_2_1_s_113 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_113 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_114 = 0; i_2_1_s_114 < 4; ++i_2_1_s_114) {
    Y_local[((i_2_1_s_114 * 4) + 1)] = (Y_local[((i_2_1_s_114 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_114] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_115 = 0; i_2_1_s_115 < 4; ++i_2_1_s_115) {
    if (i_2_1_s_115 < 3) {
      Y_local[((i_2_1_s_115 * 4) + 17)] = (Y_local[((i_2_1_s_115 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_115 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_116 = 0; i_2_1_s_116 < 4; ++i_2_1_s_116) {
    Y_local[((i_2_1_s_116 * 4) + 2)] = (Y_local[((i_2_1_s_116 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_116] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_117 = 0; i_2_1_s_117 < 4; ++i_2_1_s_117) {
    if (i_2_1_s_117 < 3) {
      Y_local[((i_2_1_s_117 * 4) + 18)] = (Y_local[((i_2_1_s_117 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_117 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_118 = 0; i_2_1_s_118 < 4; ++i_2_1_s_118) {
    Y_local[((i_2_1_s_118 * 4) + 3)] = (Y_local[((i_2_1_s_118 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_118] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_119 = 0; i_2_1_s_119 < 4; ++i_2_1_s_119) {
    if (i_2_1_s_119 < 3) {
      Y_local[((i_2_1_s_119 * 4) + 19)] = (Y_local[((i_2_1_s_119 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_119 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_16 = 0; ax0_0_16 < 2; ++ax0_0_16) {
    for (int ax0_1_s_16 = 0; ax0_1_s_16 < 4; ++ax0_1_s_16) {
      if (((ax0_0_16 * 4) + ax0_1_s_16) < 7) {
        A_shared_dyn_local[((ax0_0_16 * 4) + ax0_1_s_16)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_16 * 80)) + (ax0_1_s_16 * 20)) + 14080)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4096));
  for (int i_2_1_s_120 = 0; i_2_1_s_120 < 4; ++i_2_1_s_120) {
    Y_local[(i_2_1_s_120 * 4)] = (Y_local[(i_2_1_s_120 * 4)] + (A_shared_dyn_local[(i_2_1_s_120 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_121 = 0; i_2_1_s_121 < 4; ++i_2_1_s_121) {
    if (i_2_1_s_121 < 3) {
      Y_local[((i_2_1_s_121 * 4) + 16)] = (Y_local[((i_2_1_s_121 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_121 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_122 = 0; i_2_1_s_122 < 4; ++i_2_1_s_122) {
    Y_local[((i_2_1_s_122 * 4) + 1)] = (Y_local[((i_2_1_s_122 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_122 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_123 = 0; i_2_1_s_123 < 4; ++i_2_1_s_123) {
    if (i_2_1_s_123 < 3) {
      Y_local[((i_2_1_s_123 * 4) + 17)] = (Y_local[((i_2_1_s_123 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_123 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_124 = 0; i_2_1_s_124 < 4; ++i_2_1_s_124) {
    Y_local[((i_2_1_s_124 * 4) + 2)] = (Y_local[((i_2_1_s_124 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_124 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_125 = 0; i_2_1_s_125 < 4; ++i_2_1_s_125) {
    if (i_2_1_s_125 < 3) {
      Y_local[((i_2_1_s_125 * 4) + 18)] = (Y_local[((i_2_1_s_125 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_125 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_126 = 0; i_2_1_s_126 < 4; ++i_2_1_s_126) {
    Y_local[((i_2_1_s_126 * 4) + 3)] = (Y_local[((i_2_1_s_126 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_126 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_127 = 0; i_2_1_s_127 < 4; ++i_2_1_s_127) {
    if (i_2_1_s_127 < 3) {
      Y_local[((i_2_1_s_127 * 4) + 19)] = (Y_local[((i_2_1_s_127 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_127 + 11)] * B_shared_dyn_local[7]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax0_0_17 = 0; ax0_0_17 < 2; ++ax0_0_17) {
    for (int ax0_1_s_17 = 0; ax0_1_s_17 < 4; ++ax0_1_s_17) {
      if (((ax0_0_17 * 4) + ax0_1_s_17) < 7) {
        A_shared_dyn_local[(((ax0_0_17 * 4) + ax0_1_s_17) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_17 * 80)) + (ax0_1_s_17 * 20)) + 14081)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4224));
  for (int i_2_1_s_128 = 0; i_2_1_s_128 < 4; ++i_2_1_s_128) {
    Y_local[(i_2_1_s_128 * 4)] = (Y_local[(i_2_1_s_128 * 4)] + (A_shared_dyn_local[i_2_1_s_128] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_129 = 0; i_2_1_s_129 < 4; ++i_2_1_s_129) {
    if (i_2_1_s_129 < 3) {
      Y_local[((i_2_1_s_129 * 4) + 16)] = (Y_local[((i_2_1_s_129 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_129 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_130 = 0; i_2_1_s_130 < 4; ++i_2_1_s_130) {
    Y_local[((i_2_1_s_130 * 4) + 1)] = (Y_local[((i_2_1_s_130 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_130] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_131 = 0; i_2_1_s_131 < 4; ++i_2_1_s_131) {
    if (i_2_1_s_131 < 3) {
      Y_local[((i_2_1_s_131 * 4) + 17)] = (Y_local[((i_2_1_s_131 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_131 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_132 = 0; i_2_1_s_132 < 4; ++i_2_1_s_132) {
    Y_local[((i_2_1_s_132 * 4) + 2)] = (Y_local[((i_2_1_s_132 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_132] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_133 = 0; i_2_1_s_133 < 4; ++i_2_1_s_133) {
    if (i_2_1_s_133 < 3) {
      Y_local[((i_2_1_s_133 * 4) + 18)] = (Y_local[((i_2_1_s_133 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_133 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_134 = 0; i_2_1_s_134 < 4; ++i_2_1_s_134) {
    Y_local[((i_2_1_s_134 * 4) + 3)] = (Y_local[((i_2_1_s_134 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_134] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_135 = 0; i_2_1_s_135 < 4; ++i_2_1_s_135) {
    if (i_2_1_s_135 < 3) {
      Y_local[((i_2_1_s_135 * 4) + 19)] = (Y_local[((i_2_1_s_135 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_135 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_18 = 0; ax0_0_18 < 2; ++ax0_0_18) {
    for (int ax0_1_s_18 = 0; ax0_1_s_18 < 4; ++ax0_1_s_18) {
      if (((ax0_0_18 * 4) + ax0_1_s_18) < 7) {
        A_shared_dyn_local[((ax0_0_18 * 4) + ax0_1_s_18)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_18 * 80)) + (ax0_1_s_18 * 20)) + 14082)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4352));
  for (int i_2_1_s_136 = 0; i_2_1_s_136 < 4; ++i_2_1_s_136) {
    Y_local[(i_2_1_s_136 * 4)] = (Y_local[(i_2_1_s_136 * 4)] + (A_shared_dyn_local[(i_2_1_s_136 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_137 = 0; i_2_1_s_137 < 4; ++i_2_1_s_137) {
    if (i_2_1_s_137 < 3) {
      Y_local[((i_2_1_s_137 * 4) + 16)] = (Y_local[((i_2_1_s_137 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_137 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_138 = 0; i_2_1_s_138 < 4; ++i_2_1_s_138) {
    Y_local[((i_2_1_s_138 * 4) + 1)] = (Y_local[((i_2_1_s_138 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_138 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_139 = 0; i_2_1_s_139 < 4; ++i_2_1_s_139) {
    if (i_2_1_s_139 < 3) {
      Y_local[((i_2_1_s_139 * 4) + 17)] = (Y_local[((i_2_1_s_139 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_139 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_140 = 0; i_2_1_s_140 < 4; ++i_2_1_s_140) {
    Y_local[((i_2_1_s_140 * 4) + 2)] = (Y_local[((i_2_1_s_140 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_140 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_141 = 0; i_2_1_s_141 < 4; ++i_2_1_s_141) {
    if (i_2_1_s_141 < 3) {
      Y_local[((i_2_1_s_141 * 4) + 18)] = (Y_local[((i_2_1_s_141 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_141 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_142 = 0; i_2_1_s_142 < 4; ++i_2_1_s_142) {
    Y_local[((i_2_1_s_142 * 4) + 3)] = (Y_local[((i_2_1_s_142 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_142 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_143 = 0; i_2_1_s_143 < 4; ++i_2_1_s_143) {
    if (i_2_1_s_143 < 3) {
      Y_local[((i_2_1_s_143 * 4) + 19)] = (Y_local[((i_2_1_s_143 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_143 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_19 = 0; ax0_0_19 < 2; ++ax0_0_19) {
    for (int ax0_1_s_19 = 0; ax0_1_s_19 < 4; ++ax0_1_s_19) {
      if (((ax0_0_19 * 4) + ax0_1_s_19) < 7) {
        A_shared_dyn_local[(((ax0_0_19 * 4) + ax0_1_s_19) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_19 * 80)) + (ax0_1_s_19 * 20)) + 14083)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4480));
  for (int i_2_1_s_144 = 0; i_2_1_s_144 < 4; ++i_2_1_s_144) {
    Y_local[(i_2_1_s_144 * 4)] = (Y_local[(i_2_1_s_144 * 4)] + (A_shared_dyn_local[i_2_1_s_144] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_145 = 0; i_2_1_s_145 < 4; ++i_2_1_s_145) {
    if (i_2_1_s_145 < 3) {
      Y_local[((i_2_1_s_145 * 4) + 16)] = (Y_local[((i_2_1_s_145 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_145 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_146 = 0; i_2_1_s_146 < 4; ++i_2_1_s_146) {
    Y_local[((i_2_1_s_146 * 4) + 1)] = (Y_local[((i_2_1_s_146 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_146] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_147 = 0; i_2_1_s_147 < 4; ++i_2_1_s_147) {
    if (i_2_1_s_147 < 3) {
      Y_local[((i_2_1_s_147 * 4) + 17)] = (Y_local[((i_2_1_s_147 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_147 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_148 = 0; i_2_1_s_148 < 4; ++i_2_1_s_148) {
    Y_local[((i_2_1_s_148 * 4) + 2)] = (Y_local[((i_2_1_s_148 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_148] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_149 = 0; i_2_1_s_149 < 4; ++i_2_1_s_149) {
    if (i_2_1_s_149 < 3) {
      Y_local[((i_2_1_s_149 * 4) + 18)] = (Y_local[((i_2_1_s_149 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_149 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_150 = 0; i_2_1_s_150 < 4; ++i_2_1_s_150) {
    Y_local[((i_2_1_s_150 * 4) + 3)] = (Y_local[((i_2_1_s_150 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_150] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_151 = 0; i_2_1_s_151 < 4; ++i_2_1_s_151) {
    if (i_2_1_s_151 < 3) {
      Y_local[((i_2_1_s_151 * 4) + 19)] = (Y_local[((i_2_1_s_151 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_151 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_20 = 0; ax0_0_20 < 2; ++ax0_0_20) {
    for (int ax0_1_s_20 = 0; ax0_1_s_20 < 4; ++ax0_1_s_20) {
      if (((ax0_0_20 * 4) + ax0_1_s_20) < 7) {
        A_shared_dyn_local[((ax0_0_20 * 4) + ax0_1_s_20)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_20 * 80)) + (ax0_1_s_20 * 20)) + 14096)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4608));
  for (int i_2_1_s_152 = 0; i_2_1_s_152 < 4; ++i_2_1_s_152) {
    Y_local[(i_2_1_s_152 * 4)] = (Y_local[(i_2_1_s_152 * 4)] + (A_shared_dyn_local[(i_2_1_s_152 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_153 = 0; i_2_1_s_153 < 4; ++i_2_1_s_153) {
    if (i_2_1_s_153 < 3) {
      Y_local[((i_2_1_s_153 * 4) + 16)] = (Y_local[((i_2_1_s_153 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_153 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_154 = 0; i_2_1_s_154 < 4; ++i_2_1_s_154) {
    Y_local[((i_2_1_s_154 * 4) + 1)] = (Y_local[((i_2_1_s_154 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_154 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_155 = 0; i_2_1_s_155 < 4; ++i_2_1_s_155) {
    if (i_2_1_s_155 < 3) {
      Y_local[((i_2_1_s_155 * 4) + 17)] = (Y_local[((i_2_1_s_155 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_155 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_156 = 0; i_2_1_s_156 < 4; ++i_2_1_s_156) {
    Y_local[((i_2_1_s_156 * 4) + 2)] = (Y_local[((i_2_1_s_156 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_156 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_157 = 0; i_2_1_s_157 < 4; ++i_2_1_s_157) {
    if (i_2_1_s_157 < 3) {
      Y_local[((i_2_1_s_157 * 4) + 18)] = (Y_local[((i_2_1_s_157 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_157 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_158 = 0; i_2_1_s_158 < 4; ++i_2_1_s_158) {
    Y_local[((i_2_1_s_158 * 4) + 3)] = (Y_local[((i_2_1_s_158 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_158 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_159 = 0; i_2_1_s_159 < 4; ++i_2_1_s_159) {
    if (i_2_1_s_159 < 3) {
      Y_local[((i_2_1_s_159 * 4) + 19)] = (Y_local[((i_2_1_s_159 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_159 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_21 = 0; ax0_0_21 < 2; ++ax0_0_21) {
    for (int ax0_1_s_21 = 0; ax0_1_s_21 < 4; ++ax0_1_s_21) {
      if (((ax0_0_21 * 4) + ax0_1_s_21) < 7) {
        A_shared_dyn_local[(((ax0_0_21 * 4) + ax0_1_s_21) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_21 * 80)) + (ax0_1_s_21 * 20)) + 14097)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4736));
  for (int i_2_1_s_160 = 0; i_2_1_s_160 < 4; ++i_2_1_s_160) {
    Y_local[(i_2_1_s_160 * 4)] = (Y_local[(i_2_1_s_160 * 4)] + (A_shared_dyn_local[i_2_1_s_160] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_161 = 0; i_2_1_s_161 < 4; ++i_2_1_s_161) {
    if (i_2_1_s_161 < 3) {
      Y_local[((i_2_1_s_161 * 4) + 16)] = (Y_local[((i_2_1_s_161 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_161 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_162 = 0; i_2_1_s_162 < 4; ++i_2_1_s_162) {
    Y_local[((i_2_1_s_162 * 4) + 1)] = (Y_local[((i_2_1_s_162 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_162] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_163 = 0; i_2_1_s_163 < 4; ++i_2_1_s_163) {
    if (i_2_1_s_163 < 3) {
      Y_local[((i_2_1_s_163 * 4) + 17)] = (Y_local[((i_2_1_s_163 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_163 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_164 = 0; i_2_1_s_164 < 4; ++i_2_1_s_164) {
    Y_local[((i_2_1_s_164 * 4) + 2)] = (Y_local[((i_2_1_s_164 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_164] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_165 = 0; i_2_1_s_165 < 4; ++i_2_1_s_165) {
    if (i_2_1_s_165 < 3) {
      Y_local[((i_2_1_s_165 * 4) + 18)] = (Y_local[((i_2_1_s_165 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_165 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_166 = 0; i_2_1_s_166 < 4; ++i_2_1_s_166) {
    Y_local[((i_2_1_s_166 * 4) + 3)] = (Y_local[((i_2_1_s_166 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_166] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_167 = 0; i_2_1_s_167 < 4; ++i_2_1_s_167) {
    if (i_2_1_s_167 < 3) {
      Y_local[((i_2_1_s_167 * 4) + 19)] = (Y_local[((i_2_1_s_167 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_167 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_22 = 0; ax0_0_22 < 2; ++ax0_0_22) {
    for (int ax0_1_s_22 = 0; ax0_1_s_22 < 4; ++ax0_1_s_22) {
      if (((ax0_0_22 * 4) + ax0_1_s_22) < 7) {
        A_shared_dyn_local[((ax0_0_22 * 4) + ax0_1_s_22)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_22 * 80)) + (ax0_1_s_22 * 20)) + 14098)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4864));
  for (int i_2_1_s_168 = 0; i_2_1_s_168 < 4; ++i_2_1_s_168) {
    Y_local[(i_2_1_s_168 * 4)] = (Y_local[(i_2_1_s_168 * 4)] + (A_shared_dyn_local[(i_2_1_s_168 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_169 = 0; i_2_1_s_169 < 4; ++i_2_1_s_169) {
    if (i_2_1_s_169 < 3) {
      Y_local[((i_2_1_s_169 * 4) + 16)] = (Y_local[((i_2_1_s_169 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_169 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_170 = 0; i_2_1_s_170 < 4; ++i_2_1_s_170) {
    Y_local[((i_2_1_s_170 * 4) + 1)] = (Y_local[((i_2_1_s_170 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_170 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_171 = 0; i_2_1_s_171 < 4; ++i_2_1_s_171) {
    if (i_2_1_s_171 < 3) {
      Y_local[((i_2_1_s_171 * 4) + 17)] = (Y_local[((i_2_1_s_171 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_171 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_172 = 0; i_2_1_s_172 < 4; ++i_2_1_s_172) {
    Y_local[((i_2_1_s_172 * 4) + 2)] = (Y_local[((i_2_1_s_172 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_172 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_173 = 0; i_2_1_s_173 < 4; ++i_2_1_s_173) {
    if (i_2_1_s_173 < 3) {
      Y_local[((i_2_1_s_173 * 4) + 18)] = (Y_local[((i_2_1_s_173 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_173 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_174 = 0; i_2_1_s_174 < 4; ++i_2_1_s_174) {
    Y_local[((i_2_1_s_174 * 4) + 3)] = (Y_local[((i_2_1_s_174 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_174 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_175 = 0; i_2_1_s_175 < 4; ++i_2_1_s_175) {
    if (i_2_1_s_175 < 3) {
      Y_local[((i_2_1_s_175 * 4) + 19)] = (Y_local[((i_2_1_s_175 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_175 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_23 = 0; ax0_0_23 < 2; ++ax0_0_23) {
    for (int ax0_1_s_23 = 0; ax0_1_s_23 < 4; ++ax0_1_s_23) {
      if (((ax0_0_23 * 4) + ax0_1_s_23) < 7) {
        A_shared_dyn_local[(((ax0_0_23 * 4) + ax0_1_s_23) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_23 * 80)) + (ax0_1_s_23 * 20)) + 14099)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 4992));
  for (int i_2_1_s_176 = 0; i_2_1_s_176 < 4; ++i_2_1_s_176) {
    Y_local[(i_2_1_s_176 * 4)] = (Y_local[(i_2_1_s_176 * 4)] + (A_shared_dyn_local[i_2_1_s_176] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_177 = 0; i_2_1_s_177 < 4; ++i_2_1_s_177) {
    if (i_2_1_s_177 < 3) {
      Y_local[((i_2_1_s_177 * 4) + 16)] = (Y_local[((i_2_1_s_177 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_177 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_178 = 0; i_2_1_s_178 < 4; ++i_2_1_s_178) {
    Y_local[((i_2_1_s_178 * 4) + 1)] = (Y_local[((i_2_1_s_178 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_178] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_179 = 0; i_2_1_s_179 < 4; ++i_2_1_s_179) {
    if (i_2_1_s_179 < 3) {
      Y_local[((i_2_1_s_179 * 4) + 17)] = (Y_local[((i_2_1_s_179 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_179 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_180 = 0; i_2_1_s_180 < 4; ++i_2_1_s_180) {
    Y_local[((i_2_1_s_180 * 4) + 2)] = (Y_local[((i_2_1_s_180 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_180] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_181 = 0; i_2_1_s_181 < 4; ++i_2_1_s_181) {
    if (i_2_1_s_181 < 3) {
      Y_local[((i_2_1_s_181 * 4) + 18)] = (Y_local[((i_2_1_s_181 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_181 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_182 = 0; i_2_1_s_182 < 4; ++i_2_1_s_182) {
    Y_local[((i_2_1_s_182 * 4) + 3)] = (Y_local[((i_2_1_s_182 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_182] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_183 = 0; i_2_1_s_183 < 4; ++i_2_1_s_183) {
    if (i_2_1_s_183 < 3) {
      Y_local[((i_2_1_s_183 * 4) + 19)] = (Y_local[((i_2_1_s_183 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_183 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_24 = 0; ax0_0_24 < 2; ++ax0_0_24) {
    for (int ax0_1_s_24 = 0; ax0_1_s_24 < 4; ++ax0_1_s_24) {
      if (((ax0_0_24 * 4) + ax0_1_s_24) < 7) {
        A_shared_dyn_local[((ax0_0_24 * 4) + ax0_1_s_24)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_24 * 80)) + (ax0_1_s_24 * 20)) + 5120)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + (((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)));
  for (int i_2_1_s_184 = 0; i_2_1_s_184 < 4; ++i_2_1_s_184) {
    Y_local[(i_2_1_s_184 * 4)] = (Y_local[(i_2_1_s_184 * 4)] + (A_shared_dyn_local[(i_2_1_s_184 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_185 = 0; i_2_1_s_185 < 4; ++i_2_1_s_185) {
    if (i_2_1_s_185 < 3) {
      Y_local[((i_2_1_s_185 * 4) + 16)] = (Y_local[((i_2_1_s_185 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_185 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_186 = 0; i_2_1_s_186 < 4; ++i_2_1_s_186) {
    Y_local[((i_2_1_s_186 * 4) + 1)] = (Y_local[((i_2_1_s_186 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_186 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_187 = 0; i_2_1_s_187 < 4; ++i_2_1_s_187) {
    if (i_2_1_s_187 < 3) {
      Y_local[((i_2_1_s_187 * 4) + 17)] = (Y_local[((i_2_1_s_187 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_187 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_188 = 0; i_2_1_s_188 < 4; ++i_2_1_s_188) {
    Y_local[((i_2_1_s_188 * 4) + 2)] = (Y_local[((i_2_1_s_188 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_188 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_189 = 0; i_2_1_s_189 < 4; ++i_2_1_s_189) {
    if (i_2_1_s_189 < 3) {
      Y_local[((i_2_1_s_189 * 4) + 18)] = (Y_local[((i_2_1_s_189 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_189 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_190 = 0; i_2_1_s_190 < 4; ++i_2_1_s_190) {
    Y_local[((i_2_1_s_190 * 4) + 3)] = (Y_local[((i_2_1_s_190 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_190 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_191 = 0; i_2_1_s_191 < 4; ++i_2_1_s_191) {
    if (i_2_1_s_191 < 3) {
      Y_local[((i_2_1_s_191 * 4) + 19)] = (Y_local[((i_2_1_s_191 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_191 + 11)] * B_shared_dyn_local[7]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax0_0_25 = 0; ax0_0_25 < 2; ++ax0_0_25) {
    for (int ax0_1_s_25 = 0; ax0_1_s_25 < 4; ++ax0_1_s_25) {
      if (((ax0_0_25 * 4) + ax0_1_s_25) < 7) {
        A_shared_dyn_local[(((ax0_0_25 * 4) + ax0_1_s_25) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_25 * 80)) + (ax0_1_s_25 * 20)) + 5121)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 128));
  for (int i_2_1_s_192 = 0; i_2_1_s_192 < 4; ++i_2_1_s_192) {
    Y_local[(i_2_1_s_192 * 4)] = (Y_local[(i_2_1_s_192 * 4)] + (A_shared_dyn_local[i_2_1_s_192] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_193 = 0; i_2_1_s_193 < 4; ++i_2_1_s_193) {
    if (i_2_1_s_193 < 3) {
      Y_local[((i_2_1_s_193 * 4) + 16)] = (Y_local[((i_2_1_s_193 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_193 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_194 = 0; i_2_1_s_194 < 4; ++i_2_1_s_194) {
    Y_local[((i_2_1_s_194 * 4) + 1)] = (Y_local[((i_2_1_s_194 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_194] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_195 = 0; i_2_1_s_195 < 4; ++i_2_1_s_195) {
    if (i_2_1_s_195 < 3) {
      Y_local[((i_2_1_s_195 * 4) + 17)] = (Y_local[((i_2_1_s_195 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_195 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_196 = 0; i_2_1_s_196 < 4; ++i_2_1_s_196) {
    Y_local[((i_2_1_s_196 * 4) + 2)] = (Y_local[((i_2_1_s_196 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_196] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_197 = 0; i_2_1_s_197 < 4; ++i_2_1_s_197) {
    if (i_2_1_s_197 < 3) {
      Y_local[((i_2_1_s_197 * 4) + 18)] = (Y_local[((i_2_1_s_197 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_197 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_198 = 0; i_2_1_s_198 < 4; ++i_2_1_s_198) {
    Y_local[((i_2_1_s_198 * 4) + 3)] = (Y_local[((i_2_1_s_198 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_198] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_199 = 0; i_2_1_s_199 < 4; ++i_2_1_s_199) {
    if (i_2_1_s_199 < 3) {
      Y_local[((i_2_1_s_199 * 4) + 19)] = (Y_local[((i_2_1_s_199 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_199 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_26 = 0; ax0_0_26 < 2; ++ax0_0_26) {
    for (int ax0_1_s_26 = 0; ax0_1_s_26 < 4; ++ax0_1_s_26) {
      if (((ax0_0_26 * 4) + ax0_1_s_26) < 7) {
        A_shared_dyn_local[((ax0_0_26 * 4) + ax0_1_s_26)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_26 * 80)) + (ax0_1_s_26 * 20)) + 5122)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 256));
  for (int i_2_1_s_200 = 0; i_2_1_s_200 < 4; ++i_2_1_s_200) {
    Y_local[(i_2_1_s_200 * 4)] = (Y_local[(i_2_1_s_200 * 4)] + (A_shared_dyn_local[(i_2_1_s_200 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_201 = 0; i_2_1_s_201 < 4; ++i_2_1_s_201) {
    if (i_2_1_s_201 < 3) {
      Y_local[((i_2_1_s_201 * 4) + 16)] = (Y_local[((i_2_1_s_201 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_201 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_202 = 0; i_2_1_s_202 < 4; ++i_2_1_s_202) {
    Y_local[((i_2_1_s_202 * 4) + 1)] = (Y_local[((i_2_1_s_202 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_202 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_203 = 0; i_2_1_s_203 < 4; ++i_2_1_s_203) {
    if (i_2_1_s_203 < 3) {
      Y_local[((i_2_1_s_203 * 4) + 17)] = (Y_local[((i_2_1_s_203 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_203 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_204 = 0; i_2_1_s_204 < 4; ++i_2_1_s_204) {
    Y_local[((i_2_1_s_204 * 4) + 2)] = (Y_local[((i_2_1_s_204 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_204 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_205 = 0; i_2_1_s_205 < 4; ++i_2_1_s_205) {
    if (i_2_1_s_205 < 3) {
      Y_local[((i_2_1_s_205 * 4) + 18)] = (Y_local[((i_2_1_s_205 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_205 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_206 = 0; i_2_1_s_206 < 4; ++i_2_1_s_206) {
    Y_local[((i_2_1_s_206 * 4) + 3)] = (Y_local[((i_2_1_s_206 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_206 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_207 = 0; i_2_1_s_207 < 4; ++i_2_1_s_207) {
    if (i_2_1_s_207 < 3) {
      Y_local[((i_2_1_s_207 * 4) + 19)] = (Y_local[((i_2_1_s_207 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_207 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_27 = 0; ax0_0_27 < 2; ++ax0_0_27) {
    for (int ax0_1_s_27 = 0; ax0_1_s_27 < 4; ++ax0_1_s_27) {
      if (((ax0_0_27 * 4) + ax0_1_s_27) < 7) {
        A_shared_dyn_local[(((ax0_0_27 * 4) + ax0_1_s_27) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_27 * 80)) + (ax0_1_s_27 * 20)) + 5123)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 384));
  for (int i_2_1_s_208 = 0; i_2_1_s_208 < 4; ++i_2_1_s_208) {
    Y_local[(i_2_1_s_208 * 4)] = (Y_local[(i_2_1_s_208 * 4)] + (A_shared_dyn_local[i_2_1_s_208] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_209 = 0; i_2_1_s_209 < 4; ++i_2_1_s_209) {
    if (i_2_1_s_209 < 3) {
      Y_local[((i_2_1_s_209 * 4) + 16)] = (Y_local[((i_2_1_s_209 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_209 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_210 = 0; i_2_1_s_210 < 4; ++i_2_1_s_210) {
    Y_local[((i_2_1_s_210 * 4) + 1)] = (Y_local[((i_2_1_s_210 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_210] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_211 = 0; i_2_1_s_211 < 4; ++i_2_1_s_211) {
    if (i_2_1_s_211 < 3) {
      Y_local[((i_2_1_s_211 * 4) + 17)] = (Y_local[((i_2_1_s_211 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_211 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_212 = 0; i_2_1_s_212 < 4; ++i_2_1_s_212) {
    Y_local[((i_2_1_s_212 * 4) + 2)] = (Y_local[((i_2_1_s_212 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_212] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_213 = 0; i_2_1_s_213 < 4; ++i_2_1_s_213) {
    if (i_2_1_s_213 < 3) {
      Y_local[((i_2_1_s_213 * 4) + 18)] = (Y_local[((i_2_1_s_213 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_213 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_214 = 0; i_2_1_s_214 < 4; ++i_2_1_s_214) {
    Y_local[((i_2_1_s_214 * 4) + 3)] = (Y_local[((i_2_1_s_214 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_214] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_215 = 0; i_2_1_s_215 < 4; ++i_2_1_s_215) {
    if (i_2_1_s_215 < 3) {
      Y_local[((i_2_1_s_215 * 4) + 19)] = (Y_local[((i_2_1_s_215 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_215 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_28 = 0; ax0_0_28 < 2; ++ax0_0_28) {
    for (int ax0_1_s_28 = 0; ax0_1_s_28 < 4; ++ax0_1_s_28) {
      if (((ax0_0_28 * 4) + ax0_1_s_28) < 7) {
        A_shared_dyn_local[((ax0_0_28 * 4) + ax0_1_s_28)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_28 * 80)) + (ax0_1_s_28 * 20)) + 5136)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 512));
  for (int i_2_1_s_216 = 0; i_2_1_s_216 < 4; ++i_2_1_s_216) {
    Y_local[(i_2_1_s_216 * 4)] = (Y_local[(i_2_1_s_216 * 4)] + (A_shared_dyn_local[(i_2_1_s_216 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_217 = 0; i_2_1_s_217 < 4; ++i_2_1_s_217) {
    if (i_2_1_s_217 < 3) {
      Y_local[((i_2_1_s_217 * 4) + 16)] = (Y_local[((i_2_1_s_217 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_217 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_218 = 0; i_2_1_s_218 < 4; ++i_2_1_s_218) {
    Y_local[((i_2_1_s_218 * 4) + 1)] = (Y_local[((i_2_1_s_218 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_218 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_219 = 0; i_2_1_s_219 < 4; ++i_2_1_s_219) {
    if (i_2_1_s_219 < 3) {
      Y_local[((i_2_1_s_219 * 4) + 17)] = (Y_local[((i_2_1_s_219 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_219 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_220 = 0; i_2_1_s_220 < 4; ++i_2_1_s_220) {
    Y_local[((i_2_1_s_220 * 4) + 2)] = (Y_local[((i_2_1_s_220 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_220 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_221 = 0; i_2_1_s_221 < 4; ++i_2_1_s_221) {
    if (i_2_1_s_221 < 3) {
      Y_local[((i_2_1_s_221 * 4) + 18)] = (Y_local[((i_2_1_s_221 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_221 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_222 = 0; i_2_1_s_222 < 4; ++i_2_1_s_222) {
    Y_local[((i_2_1_s_222 * 4) + 3)] = (Y_local[((i_2_1_s_222 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_222 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_223 = 0; i_2_1_s_223 < 4; ++i_2_1_s_223) {
    if (i_2_1_s_223 < 3) {
      Y_local[((i_2_1_s_223 * 4) + 19)] = (Y_local[((i_2_1_s_223 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_223 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_29 = 0; ax0_0_29 < 2; ++ax0_0_29) {
    for (int ax0_1_s_29 = 0; ax0_1_s_29 < 4; ++ax0_1_s_29) {
      if (((ax0_0_29 * 4) + ax0_1_s_29) < 7) {
        A_shared_dyn_local[(((ax0_0_29 * 4) + ax0_1_s_29) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_29 * 80)) + (ax0_1_s_29 * 20)) + 5137)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 640));
  for (int i_2_1_s_224 = 0; i_2_1_s_224 < 4; ++i_2_1_s_224) {
    Y_local[(i_2_1_s_224 * 4)] = (Y_local[(i_2_1_s_224 * 4)] + (A_shared_dyn_local[i_2_1_s_224] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_225 = 0; i_2_1_s_225 < 4; ++i_2_1_s_225) {
    if (i_2_1_s_225 < 3) {
      Y_local[((i_2_1_s_225 * 4) + 16)] = (Y_local[((i_2_1_s_225 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_225 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_226 = 0; i_2_1_s_226 < 4; ++i_2_1_s_226) {
    Y_local[((i_2_1_s_226 * 4) + 1)] = (Y_local[((i_2_1_s_226 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_226] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_227 = 0; i_2_1_s_227 < 4; ++i_2_1_s_227) {
    if (i_2_1_s_227 < 3) {
      Y_local[((i_2_1_s_227 * 4) + 17)] = (Y_local[((i_2_1_s_227 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_227 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_228 = 0; i_2_1_s_228 < 4; ++i_2_1_s_228) {
    Y_local[((i_2_1_s_228 * 4) + 2)] = (Y_local[((i_2_1_s_228 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_228] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_229 = 0; i_2_1_s_229 < 4; ++i_2_1_s_229) {
    if (i_2_1_s_229 < 3) {
      Y_local[((i_2_1_s_229 * 4) + 18)] = (Y_local[((i_2_1_s_229 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_229 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_230 = 0; i_2_1_s_230 < 4; ++i_2_1_s_230) {
    Y_local[((i_2_1_s_230 * 4) + 3)] = (Y_local[((i_2_1_s_230 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_230] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_231 = 0; i_2_1_s_231 < 4; ++i_2_1_s_231) {
    if (i_2_1_s_231 < 3) {
      Y_local[((i_2_1_s_231 * 4) + 19)] = (Y_local[((i_2_1_s_231 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_231 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_30 = 0; ax0_0_30 < 2; ++ax0_0_30) {
    for (int ax0_1_s_30 = 0; ax0_1_s_30 < 4; ++ax0_1_s_30) {
      if (((ax0_0_30 * 4) + ax0_1_s_30) < 7) {
        A_shared_dyn_local[((ax0_0_30 * 4) + ax0_1_s_30)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_30 * 80)) + (ax0_1_s_30 * 20)) + 5138)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 768));
  for (int i_2_1_s_232 = 0; i_2_1_s_232 < 4; ++i_2_1_s_232) {
    Y_local[(i_2_1_s_232 * 4)] = (Y_local[(i_2_1_s_232 * 4)] + (A_shared_dyn_local[(i_2_1_s_232 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_233 = 0; i_2_1_s_233 < 4; ++i_2_1_s_233) {
    if (i_2_1_s_233 < 3) {
      Y_local[((i_2_1_s_233 * 4) + 16)] = (Y_local[((i_2_1_s_233 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_233 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_234 = 0; i_2_1_s_234 < 4; ++i_2_1_s_234) {
    Y_local[((i_2_1_s_234 * 4) + 1)] = (Y_local[((i_2_1_s_234 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_234 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_235 = 0; i_2_1_s_235 < 4; ++i_2_1_s_235) {
    if (i_2_1_s_235 < 3) {
      Y_local[((i_2_1_s_235 * 4) + 17)] = (Y_local[((i_2_1_s_235 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_235 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_236 = 0; i_2_1_s_236 < 4; ++i_2_1_s_236) {
    Y_local[((i_2_1_s_236 * 4) + 2)] = (Y_local[((i_2_1_s_236 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_236 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_237 = 0; i_2_1_s_237 < 4; ++i_2_1_s_237) {
    if (i_2_1_s_237 < 3) {
      Y_local[((i_2_1_s_237 * 4) + 18)] = (Y_local[((i_2_1_s_237 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_237 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_238 = 0; i_2_1_s_238 < 4; ++i_2_1_s_238) {
    Y_local[((i_2_1_s_238 * 4) + 3)] = (Y_local[((i_2_1_s_238 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_238 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_239 = 0; i_2_1_s_239 < 4; ++i_2_1_s_239) {
    if (i_2_1_s_239 < 3) {
      Y_local[((i_2_1_s_239 * 4) + 19)] = (Y_local[((i_2_1_s_239 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_239 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_31 = 0; ax0_0_31 < 2; ++ax0_0_31) {
    for (int ax0_1_s_31 = 0; ax0_1_s_31 < 4; ++ax0_1_s_31) {
      if (((ax0_0_31 * 4) + ax0_1_s_31) < 7) {
        A_shared_dyn_local[(((ax0_0_31 * 4) + ax0_1_s_31) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_31 * 80)) + (ax0_1_s_31 * 20)) + 5139)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 896));
  for (int i_2_1_s_240 = 0; i_2_1_s_240 < 4; ++i_2_1_s_240) {
    Y_local[(i_2_1_s_240 * 4)] = (Y_local[(i_2_1_s_240 * 4)] + (A_shared_dyn_local[i_2_1_s_240] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_241 = 0; i_2_1_s_241 < 4; ++i_2_1_s_241) {
    if (i_2_1_s_241 < 3) {
      Y_local[((i_2_1_s_241 * 4) + 16)] = (Y_local[((i_2_1_s_241 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_241 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_242 = 0; i_2_1_s_242 < 4; ++i_2_1_s_242) {
    Y_local[((i_2_1_s_242 * 4) + 1)] = (Y_local[((i_2_1_s_242 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_242] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_243 = 0; i_2_1_s_243 < 4; ++i_2_1_s_243) {
    if (i_2_1_s_243 < 3) {
      Y_local[((i_2_1_s_243 * 4) + 17)] = (Y_local[((i_2_1_s_243 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_243 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_244 = 0; i_2_1_s_244 < 4; ++i_2_1_s_244) {
    Y_local[((i_2_1_s_244 * 4) + 2)] = (Y_local[((i_2_1_s_244 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_244] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_245 = 0; i_2_1_s_245 < 4; ++i_2_1_s_245) {
    if (i_2_1_s_245 < 3) {
      Y_local[((i_2_1_s_245 * 4) + 18)] = (Y_local[((i_2_1_s_245 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_245 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_246 = 0; i_2_1_s_246 < 4; ++i_2_1_s_246) {
    Y_local[((i_2_1_s_246 * 4) + 3)] = (Y_local[((i_2_1_s_246 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_246] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_247 = 0; i_2_1_s_247 < 4; ++i_2_1_s_247) {
    if (i_2_1_s_247 < 3) {
      Y_local[((i_2_1_s_247 * 4) + 19)] = (Y_local[((i_2_1_s_247 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_247 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_32 = 0; ax0_0_32 < 2; ++ax0_0_32) {
    for (int ax0_1_s_32 = 0; ax0_1_s_32 < 4; ++ax0_1_s_32) {
      if (((ax0_0_32 * 4) + ax0_1_s_32) < 7) {
        A_shared_dyn_local[((ax0_0_32 * 4) + ax0_1_s_32)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_32 * 80)) + (ax0_1_s_32 * 20)) + 7360)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1024));
  for (int i_2_1_s_248 = 0; i_2_1_s_248 < 4; ++i_2_1_s_248) {
    Y_local[(i_2_1_s_248 * 4)] = (Y_local[(i_2_1_s_248 * 4)] + (A_shared_dyn_local[(i_2_1_s_248 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_249 = 0; i_2_1_s_249 < 4; ++i_2_1_s_249) {
    if (i_2_1_s_249 < 3) {
      Y_local[((i_2_1_s_249 * 4) + 16)] = (Y_local[((i_2_1_s_249 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_249 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_250 = 0; i_2_1_s_250 < 4; ++i_2_1_s_250) {
    Y_local[((i_2_1_s_250 * 4) + 1)] = (Y_local[((i_2_1_s_250 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_250 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_251 = 0; i_2_1_s_251 < 4; ++i_2_1_s_251) {
    if (i_2_1_s_251 < 3) {
      Y_local[((i_2_1_s_251 * 4) + 17)] = (Y_local[((i_2_1_s_251 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_251 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_252 = 0; i_2_1_s_252 < 4; ++i_2_1_s_252) {
    Y_local[((i_2_1_s_252 * 4) + 2)] = (Y_local[((i_2_1_s_252 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_252 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_253 = 0; i_2_1_s_253 < 4; ++i_2_1_s_253) {
    if (i_2_1_s_253 < 3) {
      Y_local[((i_2_1_s_253 * 4) + 18)] = (Y_local[((i_2_1_s_253 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_253 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_254 = 0; i_2_1_s_254 < 4; ++i_2_1_s_254) {
    Y_local[((i_2_1_s_254 * 4) + 3)] = (Y_local[((i_2_1_s_254 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_254 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_255 = 0; i_2_1_s_255 < 4; ++i_2_1_s_255) {
    if (i_2_1_s_255 < 3) {
      Y_local[((i_2_1_s_255 * 4) + 19)] = (Y_local[((i_2_1_s_255 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_255 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_33 = 0; ax0_0_33 < 2; ++ax0_0_33) {
    for (int ax0_1_s_33 = 0; ax0_1_s_33 < 4; ++ax0_1_s_33) {
      if (((ax0_0_33 * 4) + ax0_1_s_33) < 7) {
        A_shared_dyn_local[(((ax0_0_33 * 4) + ax0_1_s_33) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_33 * 80)) + (ax0_1_s_33 * 20)) + 7361)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1152));
  for (int i_2_1_s_256 = 0; i_2_1_s_256 < 4; ++i_2_1_s_256) {
    Y_local[(i_2_1_s_256 * 4)] = (Y_local[(i_2_1_s_256 * 4)] + (A_shared_dyn_local[i_2_1_s_256] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_257 = 0; i_2_1_s_257 < 4; ++i_2_1_s_257) {
    if (i_2_1_s_257 < 3) {
      Y_local[((i_2_1_s_257 * 4) + 16)] = (Y_local[((i_2_1_s_257 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_257 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_258 = 0; i_2_1_s_258 < 4; ++i_2_1_s_258) {
    Y_local[((i_2_1_s_258 * 4) + 1)] = (Y_local[((i_2_1_s_258 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_258] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_259 = 0; i_2_1_s_259 < 4; ++i_2_1_s_259) {
    if (i_2_1_s_259 < 3) {
      Y_local[((i_2_1_s_259 * 4) + 17)] = (Y_local[((i_2_1_s_259 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_259 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_260 = 0; i_2_1_s_260 < 4; ++i_2_1_s_260) {
    Y_local[((i_2_1_s_260 * 4) + 2)] = (Y_local[((i_2_1_s_260 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_260] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_261 = 0; i_2_1_s_261 < 4; ++i_2_1_s_261) {
    if (i_2_1_s_261 < 3) {
      Y_local[((i_2_1_s_261 * 4) + 18)] = (Y_local[((i_2_1_s_261 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_261 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_262 = 0; i_2_1_s_262 < 4; ++i_2_1_s_262) {
    Y_local[((i_2_1_s_262 * 4) + 3)] = (Y_local[((i_2_1_s_262 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_262] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_263 = 0; i_2_1_s_263 < 4; ++i_2_1_s_263) {
    if (i_2_1_s_263 < 3) {
      Y_local[((i_2_1_s_263 * 4) + 19)] = (Y_local[((i_2_1_s_263 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_263 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_34 = 0; ax0_0_34 < 2; ++ax0_0_34) {
    for (int ax0_1_s_34 = 0; ax0_1_s_34 < 4; ++ax0_1_s_34) {
      if (((ax0_0_34 * 4) + ax0_1_s_34) < 7) {
        A_shared_dyn_local[((ax0_0_34 * 4) + ax0_1_s_34)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_34 * 80)) + (ax0_1_s_34 * 20)) + 7362)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1280));
  for (int i_2_1_s_264 = 0; i_2_1_s_264 < 4; ++i_2_1_s_264) {
    Y_local[(i_2_1_s_264 * 4)] = (Y_local[(i_2_1_s_264 * 4)] + (A_shared_dyn_local[(i_2_1_s_264 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_265 = 0; i_2_1_s_265 < 4; ++i_2_1_s_265) {
    if (i_2_1_s_265 < 3) {
      Y_local[((i_2_1_s_265 * 4) + 16)] = (Y_local[((i_2_1_s_265 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_265 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_266 = 0; i_2_1_s_266 < 4; ++i_2_1_s_266) {
    Y_local[((i_2_1_s_266 * 4) + 1)] = (Y_local[((i_2_1_s_266 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_266 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_267 = 0; i_2_1_s_267 < 4; ++i_2_1_s_267) {
    if (i_2_1_s_267 < 3) {
      Y_local[((i_2_1_s_267 * 4) + 17)] = (Y_local[((i_2_1_s_267 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_267 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_268 = 0; i_2_1_s_268 < 4; ++i_2_1_s_268) {
    Y_local[((i_2_1_s_268 * 4) + 2)] = (Y_local[((i_2_1_s_268 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_268 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_269 = 0; i_2_1_s_269 < 4; ++i_2_1_s_269) {
    if (i_2_1_s_269 < 3) {
      Y_local[((i_2_1_s_269 * 4) + 18)] = (Y_local[((i_2_1_s_269 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_269 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_270 = 0; i_2_1_s_270 < 4; ++i_2_1_s_270) {
    Y_local[((i_2_1_s_270 * 4) + 3)] = (Y_local[((i_2_1_s_270 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_270 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_271 = 0; i_2_1_s_271 < 4; ++i_2_1_s_271) {
    if (i_2_1_s_271 < 3) {
      Y_local[((i_2_1_s_271 * 4) + 19)] = (Y_local[((i_2_1_s_271 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_271 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_35 = 0; ax0_0_35 < 2; ++ax0_0_35) {
    for (int ax0_1_s_35 = 0; ax0_1_s_35 < 4; ++ax0_1_s_35) {
      if (((ax0_0_35 * 4) + ax0_1_s_35) < 7) {
        A_shared_dyn_local[(((ax0_0_35 * 4) + ax0_1_s_35) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_35 * 80)) + (ax0_1_s_35 * 20)) + 7363)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1408));
  for (int i_2_1_s_272 = 0; i_2_1_s_272 < 4; ++i_2_1_s_272) {
    Y_local[(i_2_1_s_272 * 4)] = (Y_local[(i_2_1_s_272 * 4)] + (A_shared_dyn_local[i_2_1_s_272] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_273 = 0; i_2_1_s_273 < 4; ++i_2_1_s_273) {
    if (i_2_1_s_273 < 3) {
      Y_local[((i_2_1_s_273 * 4) + 16)] = (Y_local[((i_2_1_s_273 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_273 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_274 = 0; i_2_1_s_274 < 4; ++i_2_1_s_274) {
    Y_local[((i_2_1_s_274 * 4) + 1)] = (Y_local[((i_2_1_s_274 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_274] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_275 = 0; i_2_1_s_275 < 4; ++i_2_1_s_275) {
    if (i_2_1_s_275 < 3) {
      Y_local[((i_2_1_s_275 * 4) + 17)] = (Y_local[((i_2_1_s_275 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_275 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_276 = 0; i_2_1_s_276 < 4; ++i_2_1_s_276) {
    Y_local[((i_2_1_s_276 * 4) + 2)] = (Y_local[((i_2_1_s_276 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_276] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_277 = 0; i_2_1_s_277 < 4; ++i_2_1_s_277) {
    if (i_2_1_s_277 < 3) {
      Y_local[((i_2_1_s_277 * 4) + 18)] = (Y_local[((i_2_1_s_277 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_277 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_278 = 0; i_2_1_s_278 < 4; ++i_2_1_s_278) {
    Y_local[((i_2_1_s_278 * 4) + 3)] = (Y_local[((i_2_1_s_278 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_278] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_279 = 0; i_2_1_s_279 < 4; ++i_2_1_s_279) {
    if (i_2_1_s_279 < 3) {
      Y_local[((i_2_1_s_279 * 4) + 19)] = (Y_local[((i_2_1_s_279 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_279 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_36 = 0; ax0_0_36 < 2; ++ax0_0_36) {
    for (int ax0_1_s_36 = 0; ax0_1_s_36 < 4; ++ax0_1_s_36) {
      if (((ax0_0_36 * 4) + ax0_1_s_36) < 7) {
        A_shared_dyn_local[((ax0_0_36 * 4) + ax0_1_s_36)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_36 * 80)) + (ax0_1_s_36 * 20)) + 7376)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1536));
  for (int i_2_1_s_280 = 0; i_2_1_s_280 < 4; ++i_2_1_s_280) {
    Y_local[(i_2_1_s_280 * 4)] = (Y_local[(i_2_1_s_280 * 4)] + (A_shared_dyn_local[(i_2_1_s_280 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_281 = 0; i_2_1_s_281 < 4; ++i_2_1_s_281) {
    if (i_2_1_s_281 < 3) {
      Y_local[((i_2_1_s_281 * 4) + 16)] = (Y_local[((i_2_1_s_281 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_281 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_282 = 0; i_2_1_s_282 < 4; ++i_2_1_s_282) {
    Y_local[((i_2_1_s_282 * 4) + 1)] = (Y_local[((i_2_1_s_282 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_282 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_283 = 0; i_2_1_s_283 < 4; ++i_2_1_s_283) {
    if (i_2_1_s_283 < 3) {
      Y_local[((i_2_1_s_283 * 4) + 17)] = (Y_local[((i_2_1_s_283 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_283 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_284 = 0; i_2_1_s_284 < 4; ++i_2_1_s_284) {
    Y_local[((i_2_1_s_284 * 4) + 2)] = (Y_local[((i_2_1_s_284 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_284 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_285 = 0; i_2_1_s_285 < 4; ++i_2_1_s_285) {
    if (i_2_1_s_285 < 3) {
      Y_local[((i_2_1_s_285 * 4) + 18)] = (Y_local[((i_2_1_s_285 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_285 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_286 = 0; i_2_1_s_286 < 4; ++i_2_1_s_286) {
    Y_local[((i_2_1_s_286 * 4) + 3)] = (Y_local[((i_2_1_s_286 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_286 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_287 = 0; i_2_1_s_287 < 4; ++i_2_1_s_287) {
    if (i_2_1_s_287 < 3) {
      Y_local[((i_2_1_s_287 * 4) + 19)] = (Y_local[((i_2_1_s_287 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_287 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_37 = 0; ax0_0_37 < 2; ++ax0_0_37) {
    for (int ax0_1_s_37 = 0; ax0_1_s_37 < 4; ++ax0_1_s_37) {
      if (((ax0_0_37 * 4) + ax0_1_s_37) < 7) {
        A_shared_dyn_local[(((ax0_0_37 * 4) + ax0_1_s_37) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_37 * 80)) + (ax0_1_s_37 * 20)) + 7377)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1664));
  for (int i_2_1_s_288 = 0; i_2_1_s_288 < 4; ++i_2_1_s_288) {
    Y_local[(i_2_1_s_288 * 4)] = (Y_local[(i_2_1_s_288 * 4)] + (A_shared_dyn_local[i_2_1_s_288] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_289 = 0; i_2_1_s_289 < 4; ++i_2_1_s_289) {
    if (i_2_1_s_289 < 3) {
      Y_local[((i_2_1_s_289 * 4) + 16)] = (Y_local[((i_2_1_s_289 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_289 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_290 = 0; i_2_1_s_290 < 4; ++i_2_1_s_290) {
    Y_local[((i_2_1_s_290 * 4) + 1)] = (Y_local[((i_2_1_s_290 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_290] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_291 = 0; i_2_1_s_291 < 4; ++i_2_1_s_291) {
    if (i_2_1_s_291 < 3) {
      Y_local[((i_2_1_s_291 * 4) + 17)] = (Y_local[((i_2_1_s_291 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_291 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_292 = 0; i_2_1_s_292 < 4; ++i_2_1_s_292) {
    Y_local[((i_2_1_s_292 * 4) + 2)] = (Y_local[((i_2_1_s_292 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_292] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_293 = 0; i_2_1_s_293 < 4; ++i_2_1_s_293) {
    if (i_2_1_s_293 < 3) {
      Y_local[((i_2_1_s_293 * 4) + 18)] = (Y_local[((i_2_1_s_293 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_293 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_294 = 0; i_2_1_s_294 < 4; ++i_2_1_s_294) {
    Y_local[((i_2_1_s_294 * 4) + 3)] = (Y_local[((i_2_1_s_294 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_294] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_295 = 0; i_2_1_s_295 < 4; ++i_2_1_s_295) {
    if (i_2_1_s_295 < 3) {
      Y_local[((i_2_1_s_295 * 4) + 19)] = (Y_local[((i_2_1_s_295 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_295 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int ax0_0_38 = 0; ax0_0_38 < 2; ++ax0_0_38) {
    for (int ax0_1_s_38 = 0; ax0_1_s_38 < 4; ++ax0_1_s_38) {
      if (((ax0_0_38 * 4) + ax0_1_s_38) < 7) {
        A_shared_dyn_local[((ax0_0_38 * 4) + ax0_1_s_38)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_38 * 80)) + (ax0_1_s_38 * 20)) + 7378)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 0) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1792));
  for (int i_2_1_s_296 = 0; i_2_1_s_296 < 4; ++i_2_1_s_296) {
    Y_local[(i_2_1_s_296 * 4)] = (Y_local[(i_2_1_s_296 * 4)] + (A_shared_dyn_local[(i_2_1_s_296 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_297 = 0; i_2_1_s_297 < 4; ++i_2_1_s_297) {
    if (i_2_1_s_297 < 3) {
      Y_local[((i_2_1_s_297 * 4) + 16)] = (Y_local[((i_2_1_s_297 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_297 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_298 = 0; i_2_1_s_298 < 4; ++i_2_1_s_298) {
    Y_local[((i_2_1_s_298 * 4) + 1)] = (Y_local[((i_2_1_s_298 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_298 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_299 = 0; i_2_1_s_299 < 4; ++i_2_1_s_299) {
    if (i_2_1_s_299 < 3) {
      Y_local[((i_2_1_s_299 * 4) + 17)] = (Y_local[((i_2_1_s_299 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_299 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_300 = 0; i_2_1_s_300 < 4; ++i_2_1_s_300) {
    Y_local[((i_2_1_s_300 * 4) + 2)] = (Y_local[((i_2_1_s_300 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_300 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_301 = 0; i_2_1_s_301 < 4; ++i_2_1_s_301) {
    if (i_2_1_s_301 < 3) {
      Y_local[((i_2_1_s_301 * 4) + 18)] = (Y_local[((i_2_1_s_301 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_301 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_302 = 0; i_2_1_s_302 < 4; ++i_2_1_s_302) {
    Y_local[((i_2_1_s_302 * 4) + 3)] = (Y_local[((i_2_1_s_302 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_302 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_303 = 0; i_2_1_s_303 < 4; ++i_2_1_s_303) {
    if (i_2_1_s_303 < 3) {
      Y_local[((i_2_1_s_303 * 4) + 19)] = (Y_local[((i_2_1_s_303 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_303 + 11)] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_0_39 = 0; ax0_0_39 < 2; ++ax0_0_39) {
    for (int ax0_1_s_39 = 0; ax0_1_s_39 < 4; ++ax0_1_s_39) {
      if (((ax0_0_39 * 4) + ax0_1_s_39) < 7) {
        A_shared_dyn_local[(((ax0_0_39 * 4) + ax0_1_s_39) + 7)] = ((float*)buf_dyn_shmem)[(((((((((int)threadIdx.x) & 63) >> 3) * 280) + ((((int)threadIdx.x) & 1) * 140)) + (ax0_0_39 * 80)) + (ax0_1_s_39 * 20)) + 7379)];
      }
    }
  }
  *(float4*)(B_shared_dyn_local + 4) = *(float4*)(((float*)buf_dyn_shmem) + ((((((((int)threadIdx.x) >> 8) * 64) + (((((int)threadIdx.x) & 3) >> 1) * 32)) + (((((int)threadIdx.x) & 255) >> 6) * 8)) + (((((int)threadIdx.x) & 7) >> 2) * 4)) + 1920));
  for (int i_2_1_s_304 = 0; i_2_1_s_304 < 4; ++i_2_1_s_304) {
    Y_local[(i_2_1_s_304 * 4)] = (Y_local[(i_2_1_s_304 * 4)] + (A_shared_dyn_local[i_2_1_s_304] * B_shared_dyn_local[0]));
  }
  for (int i_2_1_s_305 = 0; i_2_1_s_305 < 4; ++i_2_1_s_305) {
    if (i_2_1_s_305 < 3) {
      Y_local[((i_2_1_s_305 * 4) + 16)] = (Y_local[((i_2_1_s_305 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_305 + 4)] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_306 = 0; i_2_1_s_306 < 4; ++i_2_1_s_306) {
    Y_local[((i_2_1_s_306 * 4) + 1)] = (Y_local[((i_2_1_s_306 * 4) + 1)] + (A_shared_dyn_local[i_2_1_s_306] * B_shared_dyn_local[1]));
  }
  for (int i_2_1_s_307 = 0; i_2_1_s_307 < 4; ++i_2_1_s_307) {
    if (i_2_1_s_307 < 3) {
      Y_local[((i_2_1_s_307 * 4) + 17)] = (Y_local[((i_2_1_s_307 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_307 + 4)] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_308 = 0; i_2_1_s_308 < 4; ++i_2_1_s_308) {
    Y_local[((i_2_1_s_308 * 4) + 2)] = (Y_local[((i_2_1_s_308 * 4) + 2)] + (A_shared_dyn_local[i_2_1_s_308] * B_shared_dyn_local[2]));
  }
  for (int i_2_1_s_309 = 0; i_2_1_s_309 < 4; ++i_2_1_s_309) {
    if (i_2_1_s_309 < 3) {
      Y_local[((i_2_1_s_309 * 4) + 18)] = (Y_local[((i_2_1_s_309 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_309 + 4)] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_310 = 0; i_2_1_s_310 < 4; ++i_2_1_s_310) {
    Y_local[((i_2_1_s_310 * 4) + 3)] = (Y_local[((i_2_1_s_310 * 4) + 3)] + (A_shared_dyn_local[i_2_1_s_310] * B_shared_dyn_local[3]));
  }
  for (int i_2_1_s_311 = 0; i_2_1_s_311 < 4; ++i_2_1_s_311) {
    if (i_2_1_s_311 < 3) {
      Y_local[((i_2_1_s_311 * 4) + 19)] = (Y_local[((i_2_1_s_311 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_311 + 4)] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_312 = 0; i_2_1_s_312 < 4; ++i_2_1_s_312) {
    Y_local[(i_2_1_s_312 * 4)] = (Y_local[(i_2_1_s_312 * 4)] + (A_shared_dyn_local[(i_2_1_s_312 + 7)] * B_shared_dyn_local[4]));
  }
  for (int i_2_1_s_313 = 0; i_2_1_s_313 < 4; ++i_2_1_s_313) {
    if (i_2_1_s_313 < 3) {
      Y_local[((i_2_1_s_313 * 4) + 16)] = (Y_local[((i_2_1_s_313 * 4) + 16)] + (A_shared_dyn_local[(i_2_1_s_313 + 11)] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_314 = 0; i_2_1_s_314 < 4; ++i_2_1_s_314) {
    Y_local[((i_2_1_s_314 * 4) + 1)] = (Y_local[((i_2_1_s_314 * 4) + 1)] + (A_shared_dyn_local[(i_2_1_s_314 + 7)] * B_shared_dyn_local[5]));
  }
  for (int i_2_1_s_315 = 0; i_2_1_s_315 < 4; ++i_2_1_s_315) {
    if (i_2_1_s_315 < 3) {
      Y_local[((i_2_1_s_315 * 4) + 17)] = (Y_local[((i_2_1_s_315 * 4) + 17)] + (A_shared_dyn_local[(i_2_1_s_315 + 11)] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_316 = 0; i_2_1_s_316 < 4; ++i_2_1_s_316) {
    Y_local[((i_2_1_s_316 * 4) + 2)] = (Y_local[((i_2_1_s_316 * 4) + 2)] + (A_shared_dyn_local[(i_2_1_s_316 + 7)] * B_shared_dyn_local[6]));
  }
  for (int i_2_1_s_317 = 0; i_2_1_s_317 < 4; ++i_2_1_s_317) {
    if (i_2_1_s_317 < 3) {
      Y_local[((i_2_1_s_317 * 4) + 18)] = (Y_local[((i_2_1_s_317 * 4) + 18)] + (A_shared_dyn_local[(i_2_1_s_317 + 11)] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_318 = 0; i_2_1_s_318 < 4; ++i_2_1_s_318) {
    Y_local[((i_2_1_s_318 * 4) + 3)] = (Y_local[((i_2_1_s_318 * 4) + 3)] + (A_shared_dyn_local[(i_2_1_s_318 + 7)] * B_shared_dyn_local[7]));
  }
  for (int i_2_1_s_319 = 0; i_2_1_s_319 < 4; ++i_2_1_s_319) {
    if (i_2_1_s_319 < 3) {
      Y_local[((i_2_1_s_319 * 4) + 19)] = (Y_local[((i_2_1_s_319 * 4) + 19)] + (A_shared_dyn_local[(i_2_1_s_319 + 11)] * B_shared_dyn_local[7]));
    }
  }
  *(float4*)(Y + (((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4))) = *(float4*)(Y_local + 0);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 896)) = *(float4*)(Y_local + 4);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1792)) = *(float4*)(Y_local + 8);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2688)) = *(float4*)(Y_local + 12);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 3584)) = *(float4*)(Y_local + 16);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 4480)) = *(float4*)(Y_local + 20);
  *(float4*)(Y + ((((((((((int)blockIdx.x) / 7) * 100352) + (((((int)threadIdx.x) & 63) >> 3) * 12544)) + ((((int)threadIdx.x) & 1) * 6272)) + ((((int)blockIdx.x) % 7) * 128)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 5376)) = *(float4*)(Y_local + 24);
}


