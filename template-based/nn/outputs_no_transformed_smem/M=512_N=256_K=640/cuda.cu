
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  float Y_local[16];
  float A_shared_dyn_local[4];
  float B_shared_dyn_local[16];
  Y_local[0] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + 10240)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((int)blockIdx.x) >> 2) * 20480) + ((((int)threadIdx.x) >> 2) * 640)) + ((((int)threadIdx.x) & 3) * 2)))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((int)threadIdx.x) >> 4) * 256) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 15) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + ((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + 12800)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 20480) + ((((int)threadIdx.x) >> 2) * 640)) + ((((int)threadIdx.x) & 3) * 2)) + 8))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 15) >> 1) * 16)) + 2048)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 2048))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + 15360)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 20480) + ((((int)threadIdx.x) >> 2) * 640)) + ((((int)threadIdx.x) & 3) * 2)) + 16))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 15) >> 1) * 16)) + 4096)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 4096))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");


  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 2) * 80) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + 17920)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + (((((((int)blockIdx.x) >> 2) * 20480) + ((((int)threadIdx.x) >> 2) * 640)) + ((((int)threadIdx.x) & 3) * 2)) + 24))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 15) >> 1) * 16)) + 6144)))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((((int)threadIdx.x) >> 4) * 256) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 6144))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

  __syncthreads();
  for (int ax0_1_s = 0; ax0_1_s < 4; ++ax0_1_s) {
    if (ax0_1_s < 2) {
      A_shared_dyn_local[ax0_1_s] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s * 20)) + 2560)];
    }
  }
  for (int ax1_0 = 0; ax1_0 < 2; ++ax1_0) {
    *(float4*)(B_shared_dyn_local + (ax1_0 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + (((ax1_0 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)));
  }
  for (int k_0 = 0; k_0 < 76; ++k_0) {
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + (((((((k_0 + 4) % 5) * 2560) + ((((int)threadIdx.x) >> 2) * 80)) + (((((int)threadIdx.x) & 3) >> 1) * 64)) + ((((int)threadIdx.x) & 1) * 8)) + 10240)))
    );
    __asm__ __volatile__(
      "cp.async.ca.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(A + ((((((((int)blockIdx.x) >> 2) * 20480) + ((((int)threadIdx.x) >> 2) * 640)) + (k_0 * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 32))), "n"(8)
    );
  }

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)(buf_dyn_shmem + ((((((k_0 + 4) % 5) * 2048) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)threadIdx.x) & 1) * 128)) + (((((int)threadIdx.x) & 15) >> 1) * 16))))
    );
    __asm__ __volatile__(
      "cp.async.cg.shared.global [%0], [%1], %2;"
       :: "r"(addr), "l"((void*)(B + (((((k_0 * 2048) + ((((int)threadIdx.x) >> 4) * 256)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) & 15) * 4)) + 8192))), "n"(16)
    );
  }
__asm__ __volatile__("cp.async.commit_group;");

__asm__ __volatile__("cp.async.wait_group 3;");

    __syncthreads();
    for (int ax0_1_s_1 = 0; ax0_1_s_1 < 4; ++ax0_1_s_1) {
      if (ax0_1_s_1 < 2) {
        A_shared_dyn_local[(ax0_1_s_1 + 2)] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_1 * 20)) + 2561)];
      }
    }
    for (int ax1_0_1 = 0; ax1_0_1 < 2; ++ax1_0_1) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_1 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_1 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 64));
    }
    for (int i_2_1_s = 0; i_2_1_s < 4; ++i_2_1_s) {
      if (i_2_1_s < 2) {
        Y_local[(i_2_1_s * 8)] = (Y_local[(i_2_1_s * 8)] + (A_shared_dyn_local[i_2_1_s] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_1 = 0; i_2_1_s_1 < 4; ++i_2_1_s_1) {
      if (i_2_1_s_1 < 2) {
        Y_local[((i_2_1_s_1 * 8) + 1)] = (Y_local[((i_2_1_s_1 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_1] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_2 = 0; i_2_1_s_2 < 4; ++i_2_1_s_2) {
      if (i_2_1_s_2 < 2) {
        Y_local[((i_2_1_s_2 * 8) + 2)] = (Y_local[((i_2_1_s_2 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_2] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_3 = 0; i_2_1_s_3 < 4; ++i_2_1_s_3) {
      if (i_2_1_s_3 < 2) {
        Y_local[((i_2_1_s_3 * 8) + 3)] = (Y_local[((i_2_1_s_3 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_3] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_4 = 0; i_2_1_s_4 < 4; ++i_2_1_s_4) {
      if (i_2_1_s_4 < 2) {
        Y_local[((i_2_1_s_4 * 8) + 4)] = (Y_local[((i_2_1_s_4 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_4] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_5 = 0; i_2_1_s_5 < 4; ++i_2_1_s_5) {
      if (i_2_1_s_5 < 2) {
        Y_local[((i_2_1_s_5 * 8) + 5)] = (Y_local[((i_2_1_s_5 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_5] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_6 = 0; i_2_1_s_6 < 4; ++i_2_1_s_6) {
      if (i_2_1_s_6 < 2) {
        Y_local[((i_2_1_s_6 * 8) + 6)] = (Y_local[((i_2_1_s_6 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_6] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_7 = 0; i_2_1_s_7 < 4; ++i_2_1_s_7) {
      if (i_2_1_s_7 < 2) {
        Y_local[((i_2_1_s_7 * 8) + 7)] = (Y_local[((i_2_1_s_7 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_7] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_1_s_2 = 0; ax0_1_s_2 < 4; ++ax0_1_s_2) {
      if (ax0_1_s_2 < 2) {
        A_shared_dyn_local[ax0_1_s_2] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_2 * 20)) + 2562)];
      }
    }
    for (int ax1_0_2 = 0; ax1_0_2 < 2; ++ax1_0_2) {
      *(float4*)(B_shared_dyn_local + (ax1_0_2 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_2 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 128));
    }
    for (int i_2_1_s_8 = 0; i_2_1_s_8 < 4; ++i_2_1_s_8) {
      if (i_2_1_s_8 < 2) {
        Y_local[(i_2_1_s_8 * 8)] = (Y_local[(i_2_1_s_8 * 8)] + (A_shared_dyn_local[(i_2_1_s_8 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_9 = 0; i_2_1_s_9 < 4; ++i_2_1_s_9) {
      if (i_2_1_s_9 < 2) {
        Y_local[((i_2_1_s_9 * 8) + 1)] = (Y_local[((i_2_1_s_9 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_9 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_10 = 0; i_2_1_s_10 < 4; ++i_2_1_s_10) {
      if (i_2_1_s_10 < 2) {
        Y_local[((i_2_1_s_10 * 8) + 2)] = (Y_local[((i_2_1_s_10 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_10 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_11 = 0; i_2_1_s_11 < 4; ++i_2_1_s_11) {
      if (i_2_1_s_11 < 2) {
        Y_local[((i_2_1_s_11 * 8) + 3)] = (Y_local[((i_2_1_s_11 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_11 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_12 = 0; i_2_1_s_12 < 4; ++i_2_1_s_12) {
      if (i_2_1_s_12 < 2) {
        Y_local[((i_2_1_s_12 * 8) + 4)] = (Y_local[((i_2_1_s_12 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_12 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_13 = 0; i_2_1_s_13 < 4; ++i_2_1_s_13) {
      if (i_2_1_s_13 < 2) {
        Y_local[((i_2_1_s_13 * 8) + 5)] = (Y_local[((i_2_1_s_13 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_13 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_14 = 0; i_2_1_s_14 < 4; ++i_2_1_s_14) {
      if (i_2_1_s_14 < 2) {
        Y_local[((i_2_1_s_14 * 8) + 6)] = (Y_local[((i_2_1_s_14 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_14 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_15 = 0; i_2_1_s_15 < 4; ++i_2_1_s_15) {
      if (i_2_1_s_15 < 2) {
        Y_local[((i_2_1_s_15 * 8) + 7)] = (Y_local[((i_2_1_s_15 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_15 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax0_1_s_3 = 0; ax0_1_s_3 < 4; ++ax0_1_s_3) {
      if (ax0_1_s_3 < 2) {
        A_shared_dyn_local[(ax0_1_s_3 + 2)] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_3 * 20)) + 2563)];
      }
    }
    for (int ax1_0_3 = 0; ax1_0_3 < 2; ++ax1_0_3) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_3 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_3 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 192));
    }
    for (int i_2_1_s_16 = 0; i_2_1_s_16 < 4; ++i_2_1_s_16) {
      if (i_2_1_s_16 < 2) {
        Y_local[(i_2_1_s_16 * 8)] = (Y_local[(i_2_1_s_16 * 8)] + (A_shared_dyn_local[i_2_1_s_16] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_17 = 0; i_2_1_s_17 < 4; ++i_2_1_s_17) {
      if (i_2_1_s_17 < 2) {
        Y_local[((i_2_1_s_17 * 8) + 1)] = (Y_local[((i_2_1_s_17 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_17] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_18 = 0; i_2_1_s_18 < 4; ++i_2_1_s_18) {
      if (i_2_1_s_18 < 2) {
        Y_local[((i_2_1_s_18 * 8) + 2)] = (Y_local[((i_2_1_s_18 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_18] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_19 = 0; i_2_1_s_19 < 4; ++i_2_1_s_19) {
      if (i_2_1_s_19 < 2) {
        Y_local[((i_2_1_s_19 * 8) + 3)] = (Y_local[((i_2_1_s_19 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_19] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_20 = 0; i_2_1_s_20 < 4; ++i_2_1_s_20) {
      if (i_2_1_s_20 < 2) {
        Y_local[((i_2_1_s_20 * 8) + 4)] = (Y_local[((i_2_1_s_20 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_20] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_21 = 0; i_2_1_s_21 < 4; ++i_2_1_s_21) {
      if (i_2_1_s_21 < 2) {
        Y_local[((i_2_1_s_21 * 8) + 5)] = (Y_local[((i_2_1_s_21 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_21] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_22 = 0; i_2_1_s_22 < 4; ++i_2_1_s_22) {
      if (i_2_1_s_22 < 2) {
        Y_local[((i_2_1_s_22 * 8) + 6)] = (Y_local[((i_2_1_s_22 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_22] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_23 = 0; i_2_1_s_23 < 4; ++i_2_1_s_23) {
      if (i_2_1_s_23 < 2) {
        Y_local[((i_2_1_s_23 * 8) + 7)] = (Y_local[((i_2_1_s_23 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_23] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_1_s_4 = 0; ax0_1_s_4 < 4; ++ax0_1_s_4) {
      if (ax0_1_s_4 < 2) {
        A_shared_dyn_local[ax0_1_s_4] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_4 * 20)) + 2576)];
      }
    }
    for (int ax1_0_4 = 0; ax1_0_4 < 2; ++ax1_0_4) {
      *(float4*)(B_shared_dyn_local + (ax1_0_4 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_4 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 256));
    }
    for (int i_2_1_s_24 = 0; i_2_1_s_24 < 4; ++i_2_1_s_24) {
      if (i_2_1_s_24 < 2) {
        Y_local[(i_2_1_s_24 * 8)] = (Y_local[(i_2_1_s_24 * 8)] + (A_shared_dyn_local[(i_2_1_s_24 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_25 = 0; i_2_1_s_25 < 4; ++i_2_1_s_25) {
      if (i_2_1_s_25 < 2) {
        Y_local[((i_2_1_s_25 * 8) + 1)] = (Y_local[((i_2_1_s_25 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_25 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_26 = 0; i_2_1_s_26 < 4; ++i_2_1_s_26) {
      if (i_2_1_s_26 < 2) {
        Y_local[((i_2_1_s_26 * 8) + 2)] = (Y_local[((i_2_1_s_26 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_26 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_27 = 0; i_2_1_s_27 < 4; ++i_2_1_s_27) {
      if (i_2_1_s_27 < 2) {
        Y_local[((i_2_1_s_27 * 8) + 3)] = (Y_local[((i_2_1_s_27 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_27 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_28 = 0; i_2_1_s_28 < 4; ++i_2_1_s_28) {
      if (i_2_1_s_28 < 2) {
        Y_local[((i_2_1_s_28 * 8) + 4)] = (Y_local[((i_2_1_s_28 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_28 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_29 = 0; i_2_1_s_29 < 4; ++i_2_1_s_29) {
      if (i_2_1_s_29 < 2) {
        Y_local[((i_2_1_s_29 * 8) + 5)] = (Y_local[((i_2_1_s_29 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_29 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_30 = 0; i_2_1_s_30 < 4; ++i_2_1_s_30) {
      if (i_2_1_s_30 < 2) {
        Y_local[((i_2_1_s_30 * 8) + 6)] = (Y_local[((i_2_1_s_30 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_30 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_31 = 0; i_2_1_s_31 < 4; ++i_2_1_s_31) {
      if (i_2_1_s_31 < 2) {
        Y_local[((i_2_1_s_31 * 8) + 7)] = (Y_local[((i_2_1_s_31 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_31 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax0_1_s_5 = 0; ax0_1_s_5 < 4; ++ax0_1_s_5) {
      if (ax0_1_s_5 < 2) {
        A_shared_dyn_local[(ax0_1_s_5 + 2)] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_5 * 20)) + 2577)];
      }
    }
    for (int ax1_0_5 = 0; ax1_0_5 < 2; ++ax1_0_5) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_5 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_5 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 320));
    }
    for (int i_2_1_s_32 = 0; i_2_1_s_32 < 4; ++i_2_1_s_32) {
      if (i_2_1_s_32 < 2) {
        Y_local[(i_2_1_s_32 * 8)] = (Y_local[(i_2_1_s_32 * 8)] + (A_shared_dyn_local[i_2_1_s_32] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_33 = 0; i_2_1_s_33 < 4; ++i_2_1_s_33) {
      if (i_2_1_s_33 < 2) {
        Y_local[((i_2_1_s_33 * 8) + 1)] = (Y_local[((i_2_1_s_33 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_33] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_34 = 0; i_2_1_s_34 < 4; ++i_2_1_s_34) {
      if (i_2_1_s_34 < 2) {
        Y_local[((i_2_1_s_34 * 8) + 2)] = (Y_local[((i_2_1_s_34 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_34] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_35 = 0; i_2_1_s_35 < 4; ++i_2_1_s_35) {
      if (i_2_1_s_35 < 2) {
        Y_local[((i_2_1_s_35 * 8) + 3)] = (Y_local[((i_2_1_s_35 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_35] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_36 = 0; i_2_1_s_36 < 4; ++i_2_1_s_36) {
      if (i_2_1_s_36 < 2) {
        Y_local[((i_2_1_s_36 * 8) + 4)] = (Y_local[((i_2_1_s_36 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_36] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_37 = 0; i_2_1_s_37 < 4; ++i_2_1_s_37) {
      if (i_2_1_s_37 < 2) {
        Y_local[((i_2_1_s_37 * 8) + 5)] = (Y_local[((i_2_1_s_37 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_37] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_38 = 0; i_2_1_s_38 < 4; ++i_2_1_s_38) {
      if (i_2_1_s_38 < 2) {
        Y_local[((i_2_1_s_38 * 8) + 6)] = (Y_local[((i_2_1_s_38 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_38] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_39 = 0; i_2_1_s_39 < 4; ++i_2_1_s_39) {
      if (i_2_1_s_39 < 2) {
        Y_local[((i_2_1_s_39 * 8) + 7)] = (Y_local[((i_2_1_s_39 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_39] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_1_s_6 = 0; ax0_1_s_6 < 4; ++ax0_1_s_6) {
      if (ax0_1_s_6 < 2) {
        A_shared_dyn_local[ax0_1_s_6] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_6 * 20)) + 2578)];
      }
    }
    for (int ax1_0_6 = 0; ax1_0_6 < 2; ++ax1_0_6) {
      *(float4*)(B_shared_dyn_local + (ax1_0_6 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_6 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 384));
    }
    for (int i_2_1_s_40 = 0; i_2_1_s_40 < 4; ++i_2_1_s_40) {
      if (i_2_1_s_40 < 2) {
        Y_local[(i_2_1_s_40 * 8)] = (Y_local[(i_2_1_s_40 * 8)] + (A_shared_dyn_local[(i_2_1_s_40 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_41 = 0; i_2_1_s_41 < 4; ++i_2_1_s_41) {
      if (i_2_1_s_41 < 2) {
        Y_local[((i_2_1_s_41 * 8) + 1)] = (Y_local[((i_2_1_s_41 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_41 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_42 = 0; i_2_1_s_42 < 4; ++i_2_1_s_42) {
      if (i_2_1_s_42 < 2) {
        Y_local[((i_2_1_s_42 * 8) + 2)] = (Y_local[((i_2_1_s_42 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_42 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_43 = 0; i_2_1_s_43 < 4; ++i_2_1_s_43) {
      if (i_2_1_s_43 < 2) {
        Y_local[((i_2_1_s_43 * 8) + 3)] = (Y_local[((i_2_1_s_43 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_43 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_44 = 0; i_2_1_s_44 < 4; ++i_2_1_s_44) {
      if (i_2_1_s_44 < 2) {
        Y_local[((i_2_1_s_44 * 8) + 4)] = (Y_local[((i_2_1_s_44 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_44 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_45 = 0; i_2_1_s_45 < 4; ++i_2_1_s_45) {
      if (i_2_1_s_45 < 2) {
        Y_local[((i_2_1_s_45 * 8) + 5)] = (Y_local[((i_2_1_s_45 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_45 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_46 = 0; i_2_1_s_46 < 4; ++i_2_1_s_46) {
      if (i_2_1_s_46 < 2) {
        Y_local[((i_2_1_s_46 * 8) + 6)] = (Y_local[((i_2_1_s_46 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_46 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_47 = 0; i_2_1_s_47 < 4; ++i_2_1_s_47) {
      if (i_2_1_s_47 < 2) {
        Y_local[((i_2_1_s_47 * 8) + 7)] = (Y_local[((i_2_1_s_47 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_47 + 2)] * B_shared_dyn_local[15]));
      }
    }
    for (int ax0_1_s_7 = 0; ax0_1_s_7 < 4; ++ax0_1_s_7) {
      if (ax0_1_s_7 < 2) {
        A_shared_dyn_local[(ax0_1_s_7 + 2)] = ((float*)buf_dyn_shmem)[((((((k_0 % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_7 * 20)) + 2579)];
      }
    }
    for (int ax1_0_7 = 0; ax1_0_7 < 2; ++ax1_0_7) {
      *(float4*)(B_shared_dyn_local + ((ax1_0_7 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 % 5) * 512) + (ax1_0_7 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 448));
    }
    for (int i_2_1_s_48 = 0; i_2_1_s_48 < 4; ++i_2_1_s_48) {
      if (i_2_1_s_48 < 2) {
        Y_local[(i_2_1_s_48 * 8)] = (Y_local[(i_2_1_s_48 * 8)] + (A_shared_dyn_local[i_2_1_s_48] * B_shared_dyn_local[0]));
      }
    }
    for (int i_2_1_s_49 = 0; i_2_1_s_49 < 4; ++i_2_1_s_49) {
      if (i_2_1_s_49 < 2) {
        Y_local[((i_2_1_s_49 * 8) + 1)] = (Y_local[((i_2_1_s_49 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_49] * B_shared_dyn_local[1]));
      }
    }
    for (int i_2_1_s_50 = 0; i_2_1_s_50 < 4; ++i_2_1_s_50) {
      if (i_2_1_s_50 < 2) {
        Y_local[((i_2_1_s_50 * 8) + 2)] = (Y_local[((i_2_1_s_50 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_50] * B_shared_dyn_local[2]));
      }
    }
    for (int i_2_1_s_51 = 0; i_2_1_s_51 < 4; ++i_2_1_s_51) {
      if (i_2_1_s_51 < 2) {
        Y_local[((i_2_1_s_51 * 8) + 3)] = (Y_local[((i_2_1_s_51 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_51] * B_shared_dyn_local[3]));
      }
    }
    for (int i_2_1_s_52 = 0; i_2_1_s_52 < 4; ++i_2_1_s_52) {
      if (i_2_1_s_52 < 2) {
        Y_local[((i_2_1_s_52 * 8) + 4)] = (Y_local[((i_2_1_s_52 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_52] * B_shared_dyn_local[4]));
      }
    }
    for (int i_2_1_s_53 = 0; i_2_1_s_53 < 4; ++i_2_1_s_53) {
      if (i_2_1_s_53 < 2) {
        Y_local[((i_2_1_s_53 * 8) + 5)] = (Y_local[((i_2_1_s_53 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_53] * B_shared_dyn_local[5]));
      }
    }
    for (int i_2_1_s_54 = 0; i_2_1_s_54 < 4; ++i_2_1_s_54) {
      if (i_2_1_s_54 < 2) {
        Y_local[((i_2_1_s_54 * 8) + 6)] = (Y_local[((i_2_1_s_54 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_54] * B_shared_dyn_local[6]));
      }
    }
    for (int i_2_1_s_55 = 0; i_2_1_s_55 < 4; ++i_2_1_s_55) {
      if (i_2_1_s_55 < 2) {
        Y_local[((i_2_1_s_55 * 8) + 7)] = (Y_local[((i_2_1_s_55 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_55] * B_shared_dyn_local[7]));
      }
    }
    for (int ax0_1_s_8 = 0; ax0_1_s_8 < 4; ++ax0_1_s_8) {
      if (ax0_1_s_8 < 2) {
        A_shared_dyn_local[ax0_1_s_8] = ((float*)buf_dyn_shmem)[(((((((k_0 + 1) % 5) * 640) + (((((int)threadIdx.x) & 63) >> 3) * 80)) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_8 * 20)) + 2560)];
      }
    }
    for (int ax1_0_8 = 0; ax1_0_8 < 2; ++ax1_0_8) {
      *(float4*)(B_shared_dyn_local + (ax1_0_8 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((((k_0 + 1) % 5) * 512) + (ax1_0_8 * 32)) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)));
    }
    for (int i_2_1_s_56 = 0; i_2_1_s_56 < 4; ++i_2_1_s_56) {
      if (i_2_1_s_56 < 2) {
        Y_local[(i_2_1_s_56 * 8)] = (Y_local[(i_2_1_s_56 * 8)] + (A_shared_dyn_local[(i_2_1_s_56 + 2)] * B_shared_dyn_local[8]));
      }
    }
    for (int i_2_1_s_57 = 0; i_2_1_s_57 < 4; ++i_2_1_s_57) {
      if (i_2_1_s_57 < 2) {
        Y_local[((i_2_1_s_57 * 8) + 1)] = (Y_local[((i_2_1_s_57 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_57 + 2)] * B_shared_dyn_local[9]));
      }
    }
    for (int i_2_1_s_58 = 0; i_2_1_s_58 < 4; ++i_2_1_s_58) {
      if (i_2_1_s_58 < 2) {
        Y_local[((i_2_1_s_58 * 8) + 2)] = (Y_local[((i_2_1_s_58 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_58 + 2)] * B_shared_dyn_local[10]));
      }
    }
    for (int i_2_1_s_59 = 0; i_2_1_s_59 < 4; ++i_2_1_s_59) {
      if (i_2_1_s_59 < 2) {
        Y_local[((i_2_1_s_59 * 8) + 3)] = (Y_local[((i_2_1_s_59 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_59 + 2)] * B_shared_dyn_local[11]));
      }
    }
    for (int i_2_1_s_60 = 0; i_2_1_s_60 < 4; ++i_2_1_s_60) {
      if (i_2_1_s_60 < 2) {
        Y_local[((i_2_1_s_60 * 8) + 4)] = (Y_local[((i_2_1_s_60 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_60 + 2)] * B_shared_dyn_local[12]));
      }
    }
    for (int i_2_1_s_61 = 0; i_2_1_s_61 < 4; ++i_2_1_s_61) {
      if (i_2_1_s_61 < 2) {
        Y_local[((i_2_1_s_61 * 8) + 5)] = (Y_local[((i_2_1_s_61 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_61 + 2)] * B_shared_dyn_local[13]));
      }
    }
    for (int i_2_1_s_62 = 0; i_2_1_s_62 < 4; ++i_2_1_s_62) {
      if (i_2_1_s_62 < 2) {
        Y_local[((i_2_1_s_62 * 8) + 6)] = (Y_local[((i_2_1_s_62 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_62 + 2)] * B_shared_dyn_local[14]));
      }
    }
    for (int i_2_1_s_63 = 0; i_2_1_s_63 < 4; ++i_2_1_s_63) {
      if (i_2_1_s_63 < 2) {
        Y_local[((i_2_1_s_63 * 8) + 7)] = (Y_local[((i_2_1_s_63 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_63 + 2)] * B_shared_dyn_local[15]));
      }
    }
  }
__asm__ __volatile__("cp.async.wait_group 2;");

  __syncthreads();
  for (int ax0_1_s_9 = 0; ax0_1_s_9 < 4; ++ax0_1_s_9) {
    if (ax0_1_s_9 < 2) {
      A_shared_dyn_local[(ax0_1_s_9 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_9 * 20)) + 3201)];
    }
  }
  for (int ax1_0_9 = 0; ax1_0_9 < 2; ++ax1_0_9) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_9 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_9 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 576));
  }
  for (int i_2_1_s_64 = 0; i_2_1_s_64 < 4; ++i_2_1_s_64) {
    if (i_2_1_s_64 < 2) {
      Y_local[(i_2_1_s_64 * 8)] = (Y_local[(i_2_1_s_64 * 8)] + (A_shared_dyn_local[i_2_1_s_64] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_65 = 0; i_2_1_s_65 < 4; ++i_2_1_s_65) {
    if (i_2_1_s_65 < 2) {
      Y_local[((i_2_1_s_65 * 8) + 1)] = (Y_local[((i_2_1_s_65 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_65] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_66 = 0; i_2_1_s_66 < 4; ++i_2_1_s_66) {
    if (i_2_1_s_66 < 2) {
      Y_local[((i_2_1_s_66 * 8) + 2)] = (Y_local[((i_2_1_s_66 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_66] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_67 = 0; i_2_1_s_67 < 4; ++i_2_1_s_67) {
    if (i_2_1_s_67 < 2) {
      Y_local[((i_2_1_s_67 * 8) + 3)] = (Y_local[((i_2_1_s_67 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_67] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_68 = 0; i_2_1_s_68 < 4; ++i_2_1_s_68) {
    if (i_2_1_s_68 < 2) {
      Y_local[((i_2_1_s_68 * 8) + 4)] = (Y_local[((i_2_1_s_68 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_68] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_69 = 0; i_2_1_s_69 < 4; ++i_2_1_s_69) {
    if (i_2_1_s_69 < 2) {
      Y_local[((i_2_1_s_69 * 8) + 5)] = (Y_local[((i_2_1_s_69 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_69] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_70 = 0; i_2_1_s_70 < 4; ++i_2_1_s_70) {
    if (i_2_1_s_70 < 2) {
      Y_local[((i_2_1_s_70 * 8) + 6)] = (Y_local[((i_2_1_s_70 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_70] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_71 = 0; i_2_1_s_71 < 4; ++i_2_1_s_71) {
    if (i_2_1_s_71 < 2) {
      Y_local[((i_2_1_s_71 * 8) + 7)] = (Y_local[((i_2_1_s_71 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_71] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_10 = 0; ax0_1_s_10 < 4; ++ax0_1_s_10) {
    if (ax0_1_s_10 < 2) {
      A_shared_dyn_local[ax0_1_s_10] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_10 * 20)) + 3202)];
    }
  }
  for (int ax1_0_10 = 0; ax1_0_10 < 2; ++ax1_0_10) {
    *(float4*)(B_shared_dyn_local + (ax1_0_10 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_10 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 640));
  }
  for (int i_2_1_s_72 = 0; i_2_1_s_72 < 4; ++i_2_1_s_72) {
    if (i_2_1_s_72 < 2) {
      Y_local[(i_2_1_s_72 * 8)] = (Y_local[(i_2_1_s_72 * 8)] + (A_shared_dyn_local[(i_2_1_s_72 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_73 = 0; i_2_1_s_73 < 4; ++i_2_1_s_73) {
    if (i_2_1_s_73 < 2) {
      Y_local[((i_2_1_s_73 * 8) + 1)] = (Y_local[((i_2_1_s_73 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_73 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_74 = 0; i_2_1_s_74 < 4; ++i_2_1_s_74) {
    if (i_2_1_s_74 < 2) {
      Y_local[((i_2_1_s_74 * 8) + 2)] = (Y_local[((i_2_1_s_74 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_74 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_75 = 0; i_2_1_s_75 < 4; ++i_2_1_s_75) {
    if (i_2_1_s_75 < 2) {
      Y_local[((i_2_1_s_75 * 8) + 3)] = (Y_local[((i_2_1_s_75 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_75 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_76 = 0; i_2_1_s_76 < 4; ++i_2_1_s_76) {
    if (i_2_1_s_76 < 2) {
      Y_local[((i_2_1_s_76 * 8) + 4)] = (Y_local[((i_2_1_s_76 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_76 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_77 = 0; i_2_1_s_77 < 4; ++i_2_1_s_77) {
    if (i_2_1_s_77 < 2) {
      Y_local[((i_2_1_s_77 * 8) + 5)] = (Y_local[((i_2_1_s_77 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_77 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_78 = 0; i_2_1_s_78 < 4; ++i_2_1_s_78) {
    if (i_2_1_s_78 < 2) {
      Y_local[((i_2_1_s_78 * 8) + 6)] = (Y_local[((i_2_1_s_78 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_78 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_79 = 0; i_2_1_s_79 < 4; ++i_2_1_s_79) {
    if (i_2_1_s_79 < 2) {
      Y_local[((i_2_1_s_79 * 8) + 7)] = (Y_local[((i_2_1_s_79 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_79 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_11 = 0; ax0_1_s_11 < 4; ++ax0_1_s_11) {
    if (ax0_1_s_11 < 2) {
      A_shared_dyn_local[(ax0_1_s_11 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_11 * 20)) + 3203)];
    }
  }
  for (int ax1_0_11 = 0; ax1_0_11 < 2; ++ax1_0_11) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_11 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_11 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 704));
  }
  for (int i_2_1_s_80 = 0; i_2_1_s_80 < 4; ++i_2_1_s_80) {
    if (i_2_1_s_80 < 2) {
      Y_local[(i_2_1_s_80 * 8)] = (Y_local[(i_2_1_s_80 * 8)] + (A_shared_dyn_local[i_2_1_s_80] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_81 = 0; i_2_1_s_81 < 4; ++i_2_1_s_81) {
    if (i_2_1_s_81 < 2) {
      Y_local[((i_2_1_s_81 * 8) + 1)] = (Y_local[((i_2_1_s_81 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_81] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_82 = 0; i_2_1_s_82 < 4; ++i_2_1_s_82) {
    if (i_2_1_s_82 < 2) {
      Y_local[((i_2_1_s_82 * 8) + 2)] = (Y_local[((i_2_1_s_82 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_82] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_83 = 0; i_2_1_s_83 < 4; ++i_2_1_s_83) {
    if (i_2_1_s_83 < 2) {
      Y_local[((i_2_1_s_83 * 8) + 3)] = (Y_local[((i_2_1_s_83 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_83] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_84 = 0; i_2_1_s_84 < 4; ++i_2_1_s_84) {
    if (i_2_1_s_84 < 2) {
      Y_local[((i_2_1_s_84 * 8) + 4)] = (Y_local[((i_2_1_s_84 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_84] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_85 = 0; i_2_1_s_85 < 4; ++i_2_1_s_85) {
    if (i_2_1_s_85 < 2) {
      Y_local[((i_2_1_s_85 * 8) + 5)] = (Y_local[((i_2_1_s_85 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_85] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_86 = 0; i_2_1_s_86 < 4; ++i_2_1_s_86) {
    if (i_2_1_s_86 < 2) {
      Y_local[((i_2_1_s_86 * 8) + 6)] = (Y_local[((i_2_1_s_86 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_86] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_87 = 0; i_2_1_s_87 < 4; ++i_2_1_s_87) {
    if (i_2_1_s_87 < 2) {
      Y_local[((i_2_1_s_87 * 8) + 7)] = (Y_local[((i_2_1_s_87 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_87] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_12 = 0; ax0_1_s_12 < 4; ++ax0_1_s_12) {
    if (ax0_1_s_12 < 2) {
      A_shared_dyn_local[ax0_1_s_12] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_12 * 20)) + 3216)];
    }
  }
  for (int ax1_0_12 = 0; ax1_0_12 < 2; ++ax1_0_12) {
    *(float4*)(B_shared_dyn_local + (ax1_0_12 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_12 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 768));
  }
  for (int i_2_1_s_88 = 0; i_2_1_s_88 < 4; ++i_2_1_s_88) {
    if (i_2_1_s_88 < 2) {
      Y_local[(i_2_1_s_88 * 8)] = (Y_local[(i_2_1_s_88 * 8)] + (A_shared_dyn_local[(i_2_1_s_88 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_89 = 0; i_2_1_s_89 < 4; ++i_2_1_s_89) {
    if (i_2_1_s_89 < 2) {
      Y_local[((i_2_1_s_89 * 8) + 1)] = (Y_local[((i_2_1_s_89 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_89 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_90 = 0; i_2_1_s_90 < 4; ++i_2_1_s_90) {
    if (i_2_1_s_90 < 2) {
      Y_local[((i_2_1_s_90 * 8) + 2)] = (Y_local[((i_2_1_s_90 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_90 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_91 = 0; i_2_1_s_91 < 4; ++i_2_1_s_91) {
    if (i_2_1_s_91 < 2) {
      Y_local[((i_2_1_s_91 * 8) + 3)] = (Y_local[((i_2_1_s_91 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_91 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_92 = 0; i_2_1_s_92 < 4; ++i_2_1_s_92) {
    if (i_2_1_s_92 < 2) {
      Y_local[((i_2_1_s_92 * 8) + 4)] = (Y_local[((i_2_1_s_92 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_92 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_93 = 0; i_2_1_s_93 < 4; ++i_2_1_s_93) {
    if (i_2_1_s_93 < 2) {
      Y_local[((i_2_1_s_93 * 8) + 5)] = (Y_local[((i_2_1_s_93 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_93 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_94 = 0; i_2_1_s_94 < 4; ++i_2_1_s_94) {
    if (i_2_1_s_94 < 2) {
      Y_local[((i_2_1_s_94 * 8) + 6)] = (Y_local[((i_2_1_s_94 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_94 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_95 = 0; i_2_1_s_95 < 4; ++i_2_1_s_95) {
    if (i_2_1_s_95 < 2) {
      Y_local[((i_2_1_s_95 * 8) + 7)] = (Y_local[((i_2_1_s_95 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_95 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_13 = 0; ax0_1_s_13 < 4; ++ax0_1_s_13) {
    if (ax0_1_s_13 < 2) {
      A_shared_dyn_local[(ax0_1_s_13 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_13 * 20)) + 3217)];
    }
  }
  for (int ax1_0_13 = 0; ax1_0_13 < 2; ++ax1_0_13) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_13 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_13 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 832));
  }
  for (int i_2_1_s_96 = 0; i_2_1_s_96 < 4; ++i_2_1_s_96) {
    if (i_2_1_s_96 < 2) {
      Y_local[(i_2_1_s_96 * 8)] = (Y_local[(i_2_1_s_96 * 8)] + (A_shared_dyn_local[i_2_1_s_96] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_97 = 0; i_2_1_s_97 < 4; ++i_2_1_s_97) {
    if (i_2_1_s_97 < 2) {
      Y_local[((i_2_1_s_97 * 8) + 1)] = (Y_local[((i_2_1_s_97 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_97] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_98 = 0; i_2_1_s_98 < 4; ++i_2_1_s_98) {
    if (i_2_1_s_98 < 2) {
      Y_local[((i_2_1_s_98 * 8) + 2)] = (Y_local[((i_2_1_s_98 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_98] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_99 = 0; i_2_1_s_99 < 4; ++i_2_1_s_99) {
    if (i_2_1_s_99 < 2) {
      Y_local[((i_2_1_s_99 * 8) + 3)] = (Y_local[((i_2_1_s_99 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_99] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_100 = 0; i_2_1_s_100 < 4; ++i_2_1_s_100) {
    if (i_2_1_s_100 < 2) {
      Y_local[((i_2_1_s_100 * 8) + 4)] = (Y_local[((i_2_1_s_100 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_100] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_101 = 0; i_2_1_s_101 < 4; ++i_2_1_s_101) {
    if (i_2_1_s_101 < 2) {
      Y_local[((i_2_1_s_101 * 8) + 5)] = (Y_local[((i_2_1_s_101 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_101] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_102 = 0; i_2_1_s_102 < 4; ++i_2_1_s_102) {
    if (i_2_1_s_102 < 2) {
      Y_local[((i_2_1_s_102 * 8) + 6)] = (Y_local[((i_2_1_s_102 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_102] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_103 = 0; i_2_1_s_103 < 4; ++i_2_1_s_103) {
    if (i_2_1_s_103 < 2) {
      Y_local[((i_2_1_s_103 * 8) + 7)] = (Y_local[((i_2_1_s_103 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_103] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_14 = 0; ax0_1_s_14 < 4; ++ax0_1_s_14) {
    if (ax0_1_s_14 < 2) {
      A_shared_dyn_local[ax0_1_s_14] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_14 * 20)) + 3218)];
    }
  }
  for (int ax1_0_14 = 0; ax1_0_14 < 2; ++ax1_0_14) {
    *(float4*)(B_shared_dyn_local + (ax1_0_14 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_14 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 896));
  }
  for (int i_2_1_s_104 = 0; i_2_1_s_104 < 4; ++i_2_1_s_104) {
    if (i_2_1_s_104 < 2) {
      Y_local[(i_2_1_s_104 * 8)] = (Y_local[(i_2_1_s_104 * 8)] + (A_shared_dyn_local[(i_2_1_s_104 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_105 = 0; i_2_1_s_105 < 4; ++i_2_1_s_105) {
    if (i_2_1_s_105 < 2) {
      Y_local[((i_2_1_s_105 * 8) + 1)] = (Y_local[((i_2_1_s_105 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_105 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_106 = 0; i_2_1_s_106 < 4; ++i_2_1_s_106) {
    if (i_2_1_s_106 < 2) {
      Y_local[((i_2_1_s_106 * 8) + 2)] = (Y_local[((i_2_1_s_106 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_106 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_107 = 0; i_2_1_s_107 < 4; ++i_2_1_s_107) {
    if (i_2_1_s_107 < 2) {
      Y_local[((i_2_1_s_107 * 8) + 3)] = (Y_local[((i_2_1_s_107 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_107 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_108 = 0; i_2_1_s_108 < 4; ++i_2_1_s_108) {
    if (i_2_1_s_108 < 2) {
      Y_local[((i_2_1_s_108 * 8) + 4)] = (Y_local[((i_2_1_s_108 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_108 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_109 = 0; i_2_1_s_109 < 4; ++i_2_1_s_109) {
    if (i_2_1_s_109 < 2) {
      Y_local[((i_2_1_s_109 * 8) + 5)] = (Y_local[((i_2_1_s_109 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_109 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_110 = 0; i_2_1_s_110 < 4; ++i_2_1_s_110) {
    if (i_2_1_s_110 < 2) {
      Y_local[((i_2_1_s_110 * 8) + 6)] = (Y_local[((i_2_1_s_110 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_110 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_111 = 0; i_2_1_s_111 < 4; ++i_2_1_s_111) {
    if (i_2_1_s_111 < 2) {
      Y_local[((i_2_1_s_111 * 8) + 7)] = (Y_local[((i_2_1_s_111 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_111 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_15 = 0; ax0_1_s_15 < 4; ++ax0_1_s_15) {
    if (ax0_1_s_15 < 2) {
      A_shared_dyn_local[(ax0_1_s_15 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_15 * 20)) + 3219)];
    }
  }
  for (int ax1_0_15 = 0; ax1_0_15 < 2; ++ax1_0_15) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_15 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_15 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 960));
  }
  for (int i_2_1_s_112 = 0; i_2_1_s_112 < 4; ++i_2_1_s_112) {
    if (i_2_1_s_112 < 2) {
      Y_local[(i_2_1_s_112 * 8)] = (Y_local[(i_2_1_s_112 * 8)] + (A_shared_dyn_local[i_2_1_s_112] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_113 = 0; i_2_1_s_113 < 4; ++i_2_1_s_113) {
    if (i_2_1_s_113 < 2) {
      Y_local[((i_2_1_s_113 * 8) + 1)] = (Y_local[((i_2_1_s_113 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_113] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_114 = 0; i_2_1_s_114 < 4; ++i_2_1_s_114) {
    if (i_2_1_s_114 < 2) {
      Y_local[((i_2_1_s_114 * 8) + 2)] = (Y_local[((i_2_1_s_114 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_114] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_115 = 0; i_2_1_s_115 < 4; ++i_2_1_s_115) {
    if (i_2_1_s_115 < 2) {
      Y_local[((i_2_1_s_115 * 8) + 3)] = (Y_local[((i_2_1_s_115 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_115] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_116 = 0; i_2_1_s_116 < 4; ++i_2_1_s_116) {
    if (i_2_1_s_116 < 2) {
      Y_local[((i_2_1_s_116 * 8) + 4)] = (Y_local[((i_2_1_s_116 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_116] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_117 = 0; i_2_1_s_117 < 4; ++i_2_1_s_117) {
    if (i_2_1_s_117 < 2) {
      Y_local[((i_2_1_s_117 * 8) + 5)] = (Y_local[((i_2_1_s_117 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_117] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_118 = 0; i_2_1_s_118 < 4; ++i_2_1_s_118) {
    if (i_2_1_s_118 < 2) {
      Y_local[((i_2_1_s_118 * 8) + 6)] = (Y_local[((i_2_1_s_118 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_118] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_119 = 0; i_2_1_s_119 < 4; ++i_2_1_s_119) {
    if (i_2_1_s_119 < 2) {
      Y_local[((i_2_1_s_119 * 8) + 7)] = (Y_local[((i_2_1_s_119 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_119] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_16 = 0; ax0_1_s_16 < 4; ++ax0_1_s_16) {
    if (ax0_1_s_16 < 2) {
      A_shared_dyn_local[ax0_1_s_16] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_16 * 20)) + 3840)];
    }
  }
  for (int ax1_0_16 = 0; ax1_0_16 < 2; ++ax1_0_16) {
    *(float4*)(B_shared_dyn_local + (ax1_0_16 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_16 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1024));
  }
  for (int i_2_1_s_120 = 0; i_2_1_s_120 < 4; ++i_2_1_s_120) {
    if (i_2_1_s_120 < 2) {
      Y_local[(i_2_1_s_120 * 8)] = (Y_local[(i_2_1_s_120 * 8)] + (A_shared_dyn_local[(i_2_1_s_120 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_121 = 0; i_2_1_s_121 < 4; ++i_2_1_s_121) {
    if (i_2_1_s_121 < 2) {
      Y_local[((i_2_1_s_121 * 8) + 1)] = (Y_local[((i_2_1_s_121 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_121 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_122 = 0; i_2_1_s_122 < 4; ++i_2_1_s_122) {
    if (i_2_1_s_122 < 2) {
      Y_local[((i_2_1_s_122 * 8) + 2)] = (Y_local[((i_2_1_s_122 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_122 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_123 = 0; i_2_1_s_123 < 4; ++i_2_1_s_123) {
    if (i_2_1_s_123 < 2) {
      Y_local[((i_2_1_s_123 * 8) + 3)] = (Y_local[((i_2_1_s_123 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_123 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_124 = 0; i_2_1_s_124 < 4; ++i_2_1_s_124) {
    if (i_2_1_s_124 < 2) {
      Y_local[((i_2_1_s_124 * 8) + 4)] = (Y_local[((i_2_1_s_124 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_124 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_125 = 0; i_2_1_s_125 < 4; ++i_2_1_s_125) {
    if (i_2_1_s_125 < 2) {
      Y_local[((i_2_1_s_125 * 8) + 5)] = (Y_local[((i_2_1_s_125 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_125 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_126 = 0; i_2_1_s_126 < 4; ++i_2_1_s_126) {
    if (i_2_1_s_126 < 2) {
      Y_local[((i_2_1_s_126 * 8) + 6)] = (Y_local[((i_2_1_s_126 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_126 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_127 = 0; i_2_1_s_127 < 4; ++i_2_1_s_127) {
    if (i_2_1_s_127 < 2) {
      Y_local[((i_2_1_s_127 * 8) + 7)] = (Y_local[((i_2_1_s_127 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_127 + 2)] * B_shared_dyn_local[15]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 1;");

  __syncthreads();
  for (int ax0_1_s_17 = 0; ax0_1_s_17 < 4; ++ax0_1_s_17) {
    if (ax0_1_s_17 < 2) {
      A_shared_dyn_local[(ax0_1_s_17 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_17 * 20)) + 3841)];
    }
  }
  for (int ax1_0_17 = 0; ax1_0_17 < 2; ++ax1_0_17) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_17 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_17 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1088));
  }
  for (int i_2_1_s_128 = 0; i_2_1_s_128 < 4; ++i_2_1_s_128) {
    if (i_2_1_s_128 < 2) {
      Y_local[(i_2_1_s_128 * 8)] = (Y_local[(i_2_1_s_128 * 8)] + (A_shared_dyn_local[i_2_1_s_128] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_129 = 0; i_2_1_s_129 < 4; ++i_2_1_s_129) {
    if (i_2_1_s_129 < 2) {
      Y_local[((i_2_1_s_129 * 8) + 1)] = (Y_local[((i_2_1_s_129 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_129] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_130 = 0; i_2_1_s_130 < 4; ++i_2_1_s_130) {
    if (i_2_1_s_130 < 2) {
      Y_local[((i_2_1_s_130 * 8) + 2)] = (Y_local[((i_2_1_s_130 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_130] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_131 = 0; i_2_1_s_131 < 4; ++i_2_1_s_131) {
    if (i_2_1_s_131 < 2) {
      Y_local[((i_2_1_s_131 * 8) + 3)] = (Y_local[((i_2_1_s_131 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_131] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_132 = 0; i_2_1_s_132 < 4; ++i_2_1_s_132) {
    if (i_2_1_s_132 < 2) {
      Y_local[((i_2_1_s_132 * 8) + 4)] = (Y_local[((i_2_1_s_132 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_132] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_133 = 0; i_2_1_s_133 < 4; ++i_2_1_s_133) {
    if (i_2_1_s_133 < 2) {
      Y_local[((i_2_1_s_133 * 8) + 5)] = (Y_local[((i_2_1_s_133 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_133] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_134 = 0; i_2_1_s_134 < 4; ++i_2_1_s_134) {
    if (i_2_1_s_134 < 2) {
      Y_local[((i_2_1_s_134 * 8) + 6)] = (Y_local[((i_2_1_s_134 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_134] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_135 = 0; i_2_1_s_135 < 4; ++i_2_1_s_135) {
    if (i_2_1_s_135 < 2) {
      Y_local[((i_2_1_s_135 * 8) + 7)] = (Y_local[((i_2_1_s_135 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_135] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_18 = 0; ax0_1_s_18 < 4; ++ax0_1_s_18) {
    if (ax0_1_s_18 < 2) {
      A_shared_dyn_local[ax0_1_s_18] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_18 * 20)) + 3842)];
    }
  }
  for (int ax1_0_18 = 0; ax1_0_18 < 2; ++ax1_0_18) {
    *(float4*)(B_shared_dyn_local + (ax1_0_18 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_18 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1152));
  }
  for (int i_2_1_s_136 = 0; i_2_1_s_136 < 4; ++i_2_1_s_136) {
    if (i_2_1_s_136 < 2) {
      Y_local[(i_2_1_s_136 * 8)] = (Y_local[(i_2_1_s_136 * 8)] + (A_shared_dyn_local[(i_2_1_s_136 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_137 = 0; i_2_1_s_137 < 4; ++i_2_1_s_137) {
    if (i_2_1_s_137 < 2) {
      Y_local[((i_2_1_s_137 * 8) + 1)] = (Y_local[((i_2_1_s_137 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_137 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_138 = 0; i_2_1_s_138 < 4; ++i_2_1_s_138) {
    if (i_2_1_s_138 < 2) {
      Y_local[((i_2_1_s_138 * 8) + 2)] = (Y_local[((i_2_1_s_138 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_138 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_139 = 0; i_2_1_s_139 < 4; ++i_2_1_s_139) {
    if (i_2_1_s_139 < 2) {
      Y_local[((i_2_1_s_139 * 8) + 3)] = (Y_local[((i_2_1_s_139 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_139 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_140 = 0; i_2_1_s_140 < 4; ++i_2_1_s_140) {
    if (i_2_1_s_140 < 2) {
      Y_local[((i_2_1_s_140 * 8) + 4)] = (Y_local[((i_2_1_s_140 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_140 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_141 = 0; i_2_1_s_141 < 4; ++i_2_1_s_141) {
    if (i_2_1_s_141 < 2) {
      Y_local[((i_2_1_s_141 * 8) + 5)] = (Y_local[((i_2_1_s_141 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_141 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_142 = 0; i_2_1_s_142 < 4; ++i_2_1_s_142) {
    if (i_2_1_s_142 < 2) {
      Y_local[((i_2_1_s_142 * 8) + 6)] = (Y_local[((i_2_1_s_142 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_142 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_143 = 0; i_2_1_s_143 < 4; ++i_2_1_s_143) {
    if (i_2_1_s_143 < 2) {
      Y_local[((i_2_1_s_143 * 8) + 7)] = (Y_local[((i_2_1_s_143 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_143 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_19 = 0; ax0_1_s_19 < 4; ++ax0_1_s_19) {
    if (ax0_1_s_19 < 2) {
      A_shared_dyn_local[(ax0_1_s_19 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_19 * 20)) + 3843)];
    }
  }
  for (int ax1_0_19 = 0; ax1_0_19 < 2; ++ax1_0_19) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_19 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_19 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1216));
  }
  for (int i_2_1_s_144 = 0; i_2_1_s_144 < 4; ++i_2_1_s_144) {
    if (i_2_1_s_144 < 2) {
      Y_local[(i_2_1_s_144 * 8)] = (Y_local[(i_2_1_s_144 * 8)] + (A_shared_dyn_local[i_2_1_s_144] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_145 = 0; i_2_1_s_145 < 4; ++i_2_1_s_145) {
    if (i_2_1_s_145 < 2) {
      Y_local[((i_2_1_s_145 * 8) + 1)] = (Y_local[((i_2_1_s_145 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_145] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_146 = 0; i_2_1_s_146 < 4; ++i_2_1_s_146) {
    if (i_2_1_s_146 < 2) {
      Y_local[((i_2_1_s_146 * 8) + 2)] = (Y_local[((i_2_1_s_146 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_146] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_147 = 0; i_2_1_s_147 < 4; ++i_2_1_s_147) {
    if (i_2_1_s_147 < 2) {
      Y_local[((i_2_1_s_147 * 8) + 3)] = (Y_local[((i_2_1_s_147 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_147] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_148 = 0; i_2_1_s_148 < 4; ++i_2_1_s_148) {
    if (i_2_1_s_148 < 2) {
      Y_local[((i_2_1_s_148 * 8) + 4)] = (Y_local[((i_2_1_s_148 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_148] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_149 = 0; i_2_1_s_149 < 4; ++i_2_1_s_149) {
    if (i_2_1_s_149 < 2) {
      Y_local[((i_2_1_s_149 * 8) + 5)] = (Y_local[((i_2_1_s_149 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_149] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_150 = 0; i_2_1_s_150 < 4; ++i_2_1_s_150) {
    if (i_2_1_s_150 < 2) {
      Y_local[((i_2_1_s_150 * 8) + 6)] = (Y_local[((i_2_1_s_150 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_150] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_151 = 0; i_2_1_s_151 < 4; ++i_2_1_s_151) {
    if (i_2_1_s_151 < 2) {
      Y_local[((i_2_1_s_151 * 8) + 7)] = (Y_local[((i_2_1_s_151 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_151] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_20 = 0; ax0_1_s_20 < 4; ++ax0_1_s_20) {
    if (ax0_1_s_20 < 2) {
      A_shared_dyn_local[ax0_1_s_20] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_20 * 20)) + 3856)];
    }
  }
  for (int ax1_0_20 = 0; ax1_0_20 < 2; ++ax1_0_20) {
    *(float4*)(B_shared_dyn_local + (ax1_0_20 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_20 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1280));
  }
  for (int i_2_1_s_152 = 0; i_2_1_s_152 < 4; ++i_2_1_s_152) {
    if (i_2_1_s_152 < 2) {
      Y_local[(i_2_1_s_152 * 8)] = (Y_local[(i_2_1_s_152 * 8)] + (A_shared_dyn_local[(i_2_1_s_152 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_153 = 0; i_2_1_s_153 < 4; ++i_2_1_s_153) {
    if (i_2_1_s_153 < 2) {
      Y_local[((i_2_1_s_153 * 8) + 1)] = (Y_local[((i_2_1_s_153 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_153 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_154 = 0; i_2_1_s_154 < 4; ++i_2_1_s_154) {
    if (i_2_1_s_154 < 2) {
      Y_local[((i_2_1_s_154 * 8) + 2)] = (Y_local[((i_2_1_s_154 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_154 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_155 = 0; i_2_1_s_155 < 4; ++i_2_1_s_155) {
    if (i_2_1_s_155 < 2) {
      Y_local[((i_2_1_s_155 * 8) + 3)] = (Y_local[((i_2_1_s_155 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_155 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_156 = 0; i_2_1_s_156 < 4; ++i_2_1_s_156) {
    if (i_2_1_s_156 < 2) {
      Y_local[((i_2_1_s_156 * 8) + 4)] = (Y_local[((i_2_1_s_156 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_156 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_157 = 0; i_2_1_s_157 < 4; ++i_2_1_s_157) {
    if (i_2_1_s_157 < 2) {
      Y_local[((i_2_1_s_157 * 8) + 5)] = (Y_local[((i_2_1_s_157 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_157 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_158 = 0; i_2_1_s_158 < 4; ++i_2_1_s_158) {
    if (i_2_1_s_158 < 2) {
      Y_local[((i_2_1_s_158 * 8) + 6)] = (Y_local[((i_2_1_s_158 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_158 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_159 = 0; i_2_1_s_159 < 4; ++i_2_1_s_159) {
    if (i_2_1_s_159 < 2) {
      Y_local[((i_2_1_s_159 * 8) + 7)] = (Y_local[((i_2_1_s_159 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_159 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_21 = 0; ax0_1_s_21 < 4; ++ax0_1_s_21) {
    if (ax0_1_s_21 < 2) {
      A_shared_dyn_local[(ax0_1_s_21 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_21 * 20)) + 3857)];
    }
  }
  for (int ax1_0_21 = 0; ax1_0_21 < 2; ++ax1_0_21) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_21 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_21 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1344));
  }
  for (int i_2_1_s_160 = 0; i_2_1_s_160 < 4; ++i_2_1_s_160) {
    if (i_2_1_s_160 < 2) {
      Y_local[(i_2_1_s_160 * 8)] = (Y_local[(i_2_1_s_160 * 8)] + (A_shared_dyn_local[i_2_1_s_160] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_161 = 0; i_2_1_s_161 < 4; ++i_2_1_s_161) {
    if (i_2_1_s_161 < 2) {
      Y_local[((i_2_1_s_161 * 8) + 1)] = (Y_local[((i_2_1_s_161 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_161] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_162 = 0; i_2_1_s_162 < 4; ++i_2_1_s_162) {
    if (i_2_1_s_162 < 2) {
      Y_local[((i_2_1_s_162 * 8) + 2)] = (Y_local[((i_2_1_s_162 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_162] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_163 = 0; i_2_1_s_163 < 4; ++i_2_1_s_163) {
    if (i_2_1_s_163 < 2) {
      Y_local[((i_2_1_s_163 * 8) + 3)] = (Y_local[((i_2_1_s_163 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_163] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_164 = 0; i_2_1_s_164 < 4; ++i_2_1_s_164) {
    if (i_2_1_s_164 < 2) {
      Y_local[((i_2_1_s_164 * 8) + 4)] = (Y_local[((i_2_1_s_164 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_164] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_165 = 0; i_2_1_s_165 < 4; ++i_2_1_s_165) {
    if (i_2_1_s_165 < 2) {
      Y_local[((i_2_1_s_165 * 8) + 5)] = (Y_local[((i_2_1_s_165 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_165] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_166 = 0; i_2_1_s_166 < 4; ++i_2_1_s_166) {
    if (i_2_1_s_166 < 2) {
      Y_local[((i_2_1_s_166 * 8) + 6)] = (Y_local[((i_2_1_s_166 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_166] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_167 = 0; i_2_1_s_167 < 4; ++i_2_1_s_167) {
    if (i_2_1_s_167 < 2) {
      Y_local[((i_2_1_s_167 * 8) + 7)] = (Y_local[((i_2_1_s_167 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_167] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_22 = 0; ax0_1_s_22 < 4; ++ax0_1_s_22) {
    if (ax0_1_s_22 < 2) {
      A_shared_dyn_local[ax0_1_s_22] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_22 * 20)) + 3858)];
    }
  }
  for (int ax1_0_22 = 0; ax1_0_22 < 2; ++ax1_0_22) {
    *(float4*)(B_shared_dyn_local + (ax1_0_22 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_22 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1408));
  }
  for (int i_2_1_s_168 = 0; i_2_1_s_168 < 4; ++i_2_1_s_168) {
    if (i_2_1_s_168 < 2) {
      Y_local[(i_2_1_s_168 * 8)] = (Y_local[(i_2_1_s_168 * 8)] + (A_shared_dyn_local[(i_2_1_s_168 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_169 = 0; i_2_1_s_169 < 4; ++i_2_1_s_169) {
    if (i_2_1_s_169 < 2) {
      Y_local[((i_2_1_s_169 * 8) + 1)] = (Y_local[((i_2_1_s_169 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_169 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_170 = 0; i_2_1_s_170 < 4; ++i_2_1_s_170) {
    if (i_2_1_s_170 < 2) {
      Y_local[((i_2_1_s_170 * 8) + 2)] = (Y_local[((i_2_1_s_170 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_170 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_171 = 0; i_2_1_s_171 < 4; ++i_2_1_s_171) {
    if (i_2_1_s_171 < 2) {
      Y_local[((i_2_1_s_171 * 8) + 3)] = (Y_local[((i_2_1_s_171 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_171 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_172 = 0; i_2_1_s_172 < 4; ++i_2_1_s_172) {
    if (i_2_1_s_172 < 2) {
      Y_local[((i_2_1_s_172 * 8) + 4)] = (Y_local[((i_2_1_s_172 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_172 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_173 = 0; i_2_1_s_173 < 4; ++i_2_1_s_173) {
    if (i_2_1_s_173 < 2) {
      Y_local[((i_2_1_s_173 * 8) + 5)] = (Y_local[((i_2_1_s_173 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_173 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_174 = 0; i_2_1_s_174 < 4; ++i_2_1_s_174) {
    if (i_2_1_s_174 < 2) {
      Y_local[((i_2_1_s_174 * 8) + 6)] = (Y_local[((i_2_1_s_174 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_174 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_175 = 0; i_2_1_s_175 < 4; ++i_2_1_s_175) {
    if (i_2_1_s_175 < 2) {
      Y_local[((i_2_1_s_175 * 8) + 7)] = (Y_local[((i_2_1_s_175 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_175 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_23 = 0; ax0_1_s_23 < 4; ++ax0_1_s_23) {
    if (ax0_1_s_23 < 2) {
      A_shared_dyn_local[(ax0_1_s_23 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_23 * 20)) + 3859)];
    }
  }
  for (int ax1_0_23 = 0; ax1_0_23 < 2; ++ax1_0_23) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_23 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_23 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1472));
  }
  for (int i_2_1_s_176 = 0; i_2_1_s_176 < 4; ++i_2_1_s_176) {
    if (i_2_1_s_176 < 2) {
      Y_local[(i_2_1_s_176 * 8)] = (Y_local[(i_2_1_s_176 * 8)] + (A_shared_dyn_local[i_2_1_s_176] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_177 = 0; i_2_1_s_177 < 4; ++i_2_1_s_177) {
    if (i_2_1_s_177 < 2) {
      Y_local[((i_2_1_s_177 * 8) + 1)] = (Y_local[((i_2_1_s_177 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_177] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_178 = 0; i_2_1_s_178 < 4; ++i_2_1_s_178) {
    if (i_2_1_s_178 < 2) {
      Y_local[((i_2_1_s_178 * 8) + 2)] = (Y_local[((i_2_1_s_178 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_178] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_179 = 0; i_2_1_s_179 < 4; ++i_2_1_s_179) {
    if (i_2_1_s_179 < 2) {
      Y_local[((i_2_1_s_179 * 8) + 3)] = (Y_local[((i_2_1_s_179 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_179] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_180 = 0; i_2_1_s_180 < 4; ++i_2_1_s_180) {
    if (i_2_1_s_180 < 2) {
      Y_local[((i_2_1_s_180 * 8) + 4)] = (Y_local[((i_2_1_s_180 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_180] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_181 = 0; i_2_1_s_181 < 4; ++i_2_1_s_181) {
    if (i_2_1_s_181 < 2) {
      Y_local[((i_2_1_s_181 * 8) + 5)] = (Y_local[((i_2_1_s_181 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_181] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_182 = 0; i_2_1_s_182 < 4; ++i_2_1_s_182) {
    if (i_2_1_s_182 < 2) {
      Y_local[((i_2_1_s_182 * 8) + 6)] = (Y_local[((i_2_1_s_182 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_182] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_183 = 0; i_2_1_s_183 < 4; ++i_2_1_s_183) {
    if (i_2_1_s_183 < 2) {
      Y_local[((i_2_1_s_183 * 8) + 7)] = (Y_local[((i_2_1_s_183 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_183] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_24 = 0; ax0_1_s_24 < 4; ++ax0_1_s_24) {
    if (ax0_1_s_24 < 2) {
      A_shared_dyn_local[ax0_1_s_24] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_24 * 20)) + 4480)];
    }
  }
  for (int ax1_0_24 = 0; ax1_0_24 < 2; ++ax1_0_24) {
    *(float4*)(B_shared_dyn_local + (ax1_0_24 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_24 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1536));
  }
  for (int i_2_1_s_184 = 0; i_2_1_s_184 < 4; ++i_2_1_s_184) {
    if (i_2_1_s_184 < 2) {
      Y_local[(i_2_1_s_184 * 8)] = (Y_local[(i_2_1_s_184 * 8)] + (A_shared_dyn_local[(i_2_1_s_184 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_185 = 0; i_2_1_s_185 < 4; ++i_2_1_s_185) {
    if (i_2_1_s_185 < 2) {
      Y_local[((i_2_1_s_185 * 8) + 1)] = (Y_local[((i_2_1_s_185 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_185 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_186 = 0; i_2_1_s_186 < 4; ++i_2_1_s_186) {
    if (i_2_1_s_186 < 2) {
      Y_local[((i_2_1_s_186 * 8) + 2)] = (Y_local[((i_2_1_s_186 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_186 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_187 = 0; i_2_1_s_187 < 4; ++i_2_1_s_187) {
    if (i_2_1_s_187 < 2) {
      Y_local[((i_2_1_s_187 * 8) + 3)] = (Y_local[((i_2_1_s_187 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_187 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_188 = 0; i_2_1_s_188 < 4; ++i_2_1_s_188) {
    if (i_2_1_s_188 < 2) {
      Y_local[((i_2_1_s_188 * 8) + 4)] = (Y_local[((i_2_1_s_188 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_188 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_189 = 0; i_2_1_s_189 < 4; ++i_2_1_s_189) {
    if (i_2_1_s_189 < 2) {
      Y_local[((i_2_1_s_189 * 8) + 5)] = (Y_local[((i_2_1_s_189 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_189 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_190 = 0; i_2_1_s_190 < 4; ++i_2_1_s_190) {
    if (i_2_1_s_190 < 2) {
      Y_local[((i_2_1_s_190 * 8) + 6)] = (Y_local[((i_2_1_s_190 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_190 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_191 = 0; i_2_1_s_191 < 4; ++i_2_1_s_191) {
    if (i_2_1_s_191 < 2) {
      Y_local[((i_2_1_s_191 * 8) + 7)] = (Y_local[((i_2_1_s_191 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_191 + 2)] * B_shared_dyn_local[15]));
    }
  }
__asm__ __volatile__("cp.async.wait_group 0;");

  __syncthreads();
  for (int ax0_1_s_25 = 0; ax0_1_s_25 < 4; ++ax0_1_s_25) {
    if (ax0_1_s_25 < 2) {
      A_shared_dyn_local[(ax0_1_s_25 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_25 * 20)) + 4481)];
    }
  }
  for (int ax1_0_25 = 0; ax1_0_25 < 2; ++ax1_0_25) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_25 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_25 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1600));
  }
  for (int i_2_1_s_192 = 0; i_2_1_s_192 < 4; ++i_2_1_s_192) {
    if (i_2_1_s_192 < 2) {
      Y_local[(i_2_1_s_192 * 8)] = (Y_local[(i_2_1_s_192 * 8)] + (A_shared_dyn_local[i_2_1_s_192] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_193 = 0; i_2_1_s_193 < 4; ++i_2_1_s_193) {
    if (i_2_1_s_193 < 2) {
      Y_local[((i_2_1_s_193 * 8) + 1)] = (Y_local[((i_2_1_s_193 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_193] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_194 = 0; i_2_1_s_194 < 4; ++i_2_1_s_194) {
    if (i_2_1_s_194 < 2) {
      Y_local[((i_2_1_s_194 * 8) + 2)] = (Y_local[((i_2_1_s_194 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_194] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_195 = 0; i_2_1_s_195 < 4; ++i_2_1_s_195) {
    if (i_2_1_s_195 < 2) {
      Y_local[((i_2_1_s_195 * 8) + 3)] = (Y_local[((i_2_1_s_195 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_195] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_196 = 0; i_2_1_s_196 < 4; ++i_2_1_s_196) {
    if (i_2_1_s_196 < 2) {
      Y_local[((i_2_1_s_196 * 8) + 4)] = (Y_local[((i_2_1_s_196 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_196] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_197 = 0; i_2_1_s_197 < 4; ++i_2_1_s_197) {
    if (i_2_1_s_197 < 2) {
      Y_local[((i_2_1_s_197 * 8) + 5)] = (Y_local[((i_2_1_s_197 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_197] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_198 = 0; i_2_1_s_198 < 4; ++i_2_1_s_198) {
    if (i_2_1_s_198 < 2) {
      Y_local[((i_2_1_s_198 * 8) + 6)] = (Y_local[((i_2_1_s_198 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_198] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_199 = 0; i_2_1_s_199 < 4; ++i_2_1_s_199) {
    if (i_2_1_s_199 < 2) {
      Y_local[((i_2_1_s_199 * 8) + 7)] = (Y_local[((i_2_1_s_199 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_199] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_26 = 0; ax0_1_s_26 < 4; ++ax0_1_s_26) {
    if (ax0_1_s_26 < 2) {
      A_shared_dyn_local[ax0_1_s_26] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_26 * 20)) + 4482)];
    }
  }
  for (int ax1_0_26 = 0; ax1_0_26 < 2; ++ax1_0_26) {
    *(float4*)(B_shared_dyn_local + (ax1_0_26 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_26 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1664));
  }
  for (int i_2_1_s_200 = 0; i_2_1_s_200 < 4; ++i_2_1_s_200) {
    if (i_2_1_s_200 < 2) {
      Y_local[(i_2_1_s_200 * 8)] = (Y_local[(i_2_1_s_200 * 8)] + (A_shared_dyn_local[(i_2_1_s_200 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_201 = 0; i_2_1_s_201 < 4; ++i_2_1_s_201) {
    if (i_2_1_s_201 < 2) {
      Y_local[((i_2_1_s_201 * 8) + 1)] = (Y_local[((i_2_1_s_201 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_201 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_202 = 0; i_2_1_s_202 < 4; ++i_2_1_s_202) {
    if (i_2_1_s_202 < 2) {
      Y_local[((i_2_1_s_202 * 8) + 2)] = (Y_local[((i_2_1_s_202 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_202 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_203 = 0; i_2_1_s_203 < 4; ++i_2_1_s_203) {
    if (i_2_1_s_203 < 2) {
      Y_local[((i_2_1_s_203 * 8) + 3)] = (Y_local[((i_2_1_s_203 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_203 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_204 = 0; i_2_1_s_204 < 4; ++i_2_1_s_204) {
    if (i_2_1_s_204 < 2) {
      Y_local[((i_2_1_s_204 * 8) + 4)] = (Y_local[((i_2_1_s_204 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_204 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_205 = 0; i_2_1_s_205 < 4; ++i_2_1_s_205) {
    if (i_2_1_s_205 < 2) {
      Y_local[((i_2_1_s_205 * 8) + 5)] = (Y_local[((i_2_1_s_205 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_205 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_206 = 0; i_2_1_s_206 < 4; ++i_2_1_s_206) {
    if (i_2_1_s_206 < 2) {
      Y_local[((i_2_1_s_206 * 8) + 6)] = (Y_local[((i_2_1_s_206 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_206 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_207 = 0; i_2_1_s_207 < 4; ++i_2_1_s_207) {
    if (i_2_1_s_207 < 2) {
      Y_local[((i_2_1_s_207 * 8) + 7)] = (Y_local[((i_2_1_s_207 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_207 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_27 = 0; ax0_1_s_27 < 4; ++ax0_1_s_27) {
    if (ax0_1_s_27 < 2) {
      A_shared_dyn_local[(ax0_1_s_27 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_27 * 20)) + 4483)];
    }
  }
  for (int ax1_0_27 = 0; ax1_0_27 < 2; ++ax1_0_27) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_27 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_27 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1728));
  }
  for (int i_2_1_s_208 = 0; i_2_1_s_208 < 4; ++i_2_1_s_208) {
    if (i_2_1_s_208 < 2) {
      Y_local[(i_2_1_s_208 * 8)] = (Y_local[(i_2_1_s_208 * 8)] + (A_shared_dyn_local[i_2_1_s_208] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_209 = 0; i_2_1_s_209 < 4; ++i_2_1_s_209) {
    if (i_2_1_s_209 < 2) {
      Y_local[((i_2_1_s_209 * 8) + 1)] = (Y_local[((i_2_1_s_209 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_209] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_210 = 0; i_2_1_s_210 < 4; ++i_2_1_s_210) {
    if (i_2_1_s_210 < 2) {
      Y_local[((i_2_1_s_210 * 8) + 2)] = (Y_local[((i_2_1_s_210 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_210] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_211 = 0; i_2_1_s_211 < 4; ++i_2_1_s_211) {
    if (i_2_1_s_211 < 2) {
      Y_local[((i_2_1_s_211 * 8) + 3)] = (Y_local[((i_2_1_s_211 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_211] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_212 = 0; i_2_1_s_212 < 4; ++i_2_1_s_212) {
    if (i_2_1_s_212 < 2) {
      Y_local[((i_2_1_s_212 * 8) + 4)] = (Y_local[((i_2_1_s_212 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_212] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_213 = 0; i_2_1_s_213 < 4; ++i_2_1_s_213) {
    if (i_2_1_s_213 < 2) {
      Y_local[((i_2_1_s_213 * 8) + 5)] = (Y_local[((i_2_1_s_213 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_213] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_214 = 0; i_2_1_s_214 < 4; ++i_2_1_s_214) {
    if (i_2_1_s_214 < 2) {
      Y_local[((i_2_1_s_214 * 8) + 6)] = (Y_local[((i_2_1_s_214 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_214] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_215 = 0; i_2_1_s_215 < 4; ++i_2_1_s_215) {
    if (i_2_1_s_215 < 2) {
      Y_local[((i_2_1_s_215 * 8) + 7)] = (Y_local[((i_2_1_s_215 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_215] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_28 = 0; ax0_1_s_28 < 4; ++ax0_1_s_28) {
    if (ax0_1_s_28 < 2) {
      A_shared_dyn_local[ax0_1_s_28] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_28 * 20)) + 4496)];
    }
  }
  for (int ax1_0_28 = 0; ax1_0_28 < 2; ++ax1_0_28) {
    *(float4*)(B_shared_dyn_local + (ax1_0_28 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_28 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1792));
  }
  for (int i_2_1_s_216 = 0; i_2_1_s_216 < 4; ++i_2_1_s_216) {
    if (i_2_1_s_216 < 2) {
      Y_local[(i_2_1_s_216 * 8)] = (Y_local[(i_2_1_s_216 * 8)] + (A_shared_dyn_local[(i_2_1_s_216 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_217 = 0; i_2_1_s_217 < 4; ++i_2_1_s_217) {
    if (i_2_1_s_217 < 2) {
      Y_local[((i_2_1_s_217 * 8) + 1)] = (Y_local[((i_2_1_s_217 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_217 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_218 = 0; i_2_1_s_218 < 4; ++i_2_1_s_218) {
    if (i_2_1_s_218 < 2) {
      Y_local[((i_2_1_s_218 * 8) + 2)] = (Y_local[((i_2_1_s_218 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_218 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_219 = 0; i_2_1_s_219 < 4; ++i_2_1_s_219) {
    if (i_2_1_s_219 < 2) {
      Y_local[((i_2_1_s_219 * 8) + 3)] = (Y_local[((i_2_1_s_219 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_219 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_220 = 0; i_2_1_s_220 < 4; ++i_2_1_s_220) {
    if (i_2_1_s_220 < 2) {
      Y_local[((i_2_1_s_220 * 8) + 4)] = (Y_local[((i_2_1_s_220 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_220 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_221 = 0; i_2_1_s_221 < 4; ++i_2_1_s_221) {
    if (i_2_1_s_221 < 2) {
      Y_local[((i_2_1_s_221 * 8) + 5)] = (Y_local[((i_2_1_s_221 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_221 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_222 = 0; i_2_1_s_222 < 4; ++i_2_1_s_222) {
    if (i_2_1_s_222 < 2) {
      Y_local[((i_2_1_s_222 * 8) + 6)] = (Y_local[((i_2_1_s_222 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_222 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_223 = 0; i_2_1_s_223 < 4; ++i_2_1_s_223) {
    if (i_2_1_s_223 < 2) {
      Y_local[((i_2_1_s_223 * 8) + 7)] = (Y_local[((i_2_1_s_223 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_223 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_29 = 0; ax0_1_s_29 < 4; ++ax0_1_s_29) {
    if (ax0_1_s_29 < 2) {
      A_shared_dyn_local[(ax0_1_s_29 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_29 * 20)) + 4497)];
    }
  }
  for (int ax1_0_29 = 0; ax1_0_29 < 2; ++ax1_0_29) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_29 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_29 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1856));
  }
  for (int i_2_1_s_224 = 0; i_2_1_s_224 < 4; ++i_2_1_s_224) {
    if (i_2_1_s_224 < 2) {
      Y_local[(i_2_1_s_224 * 8)] = (Y_local[(i_2_1_s_224 * 8)] + (A_shared_dyn_local[i_2_1_s_224] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_225 = 0; i_2_1_s_225 < 4; ++i_2_1_s_225) {
    if (i_2_1_s_225 < 2) {
      Y_local[((i_2_1_s_225 * 8) + 1)] = (Y_local[((i_2_1_s_225 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_225] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_226 = 0; i_2_1_s_226 < 4; ++i_2_1_s_226) {
    if (i_2_1_s_226 < 2) {
      Y_local[((i_2_1_s_226 * 8) + 2)] = (Y_local[((i_2_1_s_226 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_226] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_227 = 0; i_2_1_s_227 < 4; ++i_2_1_s_227) {
    if (i_2_1_s_227 < 2) {
      Y_local[((i_2_1_s_227 * 8) + 3)] = (Y_local[((i_2_1_s_227 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_227] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_228 = 0; i_2_1_s_228 < 4; ++i_2_1_s_228) {
    if (i_2_1_s_228 < 2) {
      Y_local[((i_2_1_s_228 * 8) + 4)] = (Y_local[((i_2_1_s_228 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_228] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_229 = 0; i_2_1_s_229 < 4; ++i_2_1_s_229) {
    if (i_2_1_s_229 < 2) {
      Y_local[((i_2_1_s_229 * 8) + 5)] = (Y_local[((i_2_1_s_229 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_229] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_230 = 0; i_2_1_s_230 < 4; ++i_2_1_s_230) {
    if (i_2_1_s_230 < 2) {
      Y_local[((i_2_1_s_230 * 8) + 6)] = (Y_local[((i_2_1_s_230 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_230] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_231 = 0; i_2_1_s_231 < 4; ++i_2_1_s_231) {
    if (i_2_1_s_231 < 2) {
      Y_local[((i_2_1_s_231 * 8) + 7)] = (Y_local[((i_2_1_s_231 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_231] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_30 = 0; ax0_1_s_30 < 4; ++ax0_1_s_30) {
    if (ax0_1_s_30 < 2) {
      A_shared_dyn_local[ax0_1_s_30] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_30 * 20)) + 4498)];
    }
  }
  for (int ax1_0_30 = 0; ax1_0_30 < 2; ++ax1_0_30) {
    *(float4*)(B_shared_dyn_local + (ax1_0_30 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_30 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1920));
  }
  for (int i_2_1_s_232 = 0; i_2_1_s_232 < 4; ++i_2_1_s_232) {
    if (i_2_1_s_232 < 2) {
      Y_local[(i_2_1_s_232 * 8)] = (Y_local[(i_2_1_s_232 * 8)] + (A_shared_dyn_local[(i_2_1_s_232 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_233 = 0; i_2_1_s_233 < 4; ++i_2_1_s_233) {
    if (i_2_1_s_233 < 2) {
      Y_local[((i_2_1_s_233 * 8) + 1)] = (Y_local[((i_2_1_s_233 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_233 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_234 = 0; i_2_1_s_234 < 4; ++i_2_1_s_234) {
    if (i_2_1_s_234 < 2) {
      Y_local[((i_2_1_s_234 * 8) + 2)] = (Y_local[((i_2_1_s_234 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_234 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_235 = 0; i_2_1_s_235 < 4; ++i_2_1_s_235) {
    if (i_2_1_s_235 < 2) {
      Y_local[((i_2_1_s_235 * 8) + 3)] = (Y_local[((i_2_1_s_235 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_235 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_236 = 0; i_2_1_s_236 < 4; ++i_2_1_s_236) {
    if (i_2_1_s_236 < 2) {
      Y_local[((i_2_1_s_236 * 8) + 4)] = (Y_local[((i_2_1_s_236 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_236 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_237 = 0; i_2_1_s_237 < 4; ++i_2_1_s_237) {
    if (i_2_1_s_237 < 2) {
      Y_local[((i_2_1_s_237 * 8) + 5)] = (Y_local[((i_2_1_s_237 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_237 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_238 = 0; i_2_1_s_238 < 4; ++i_2_1_s_238) {
    if (i_2_1_s_238 < 2) {
      Y_local[((i_2_1_s_238 * 8) + 6)] = (Y_local[((i_2_1_s_238 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_238 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_239 = 0; i_2_1_s_239 < 4; ++i_2_1_s_239) {
    if (i_2_1_s_239 < 2) {
      Y_local[((i_2_1_s_239 * 8) + 7)] = (Y_local[((i_2_1_s_239 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_239 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_31 = 0; ax0_1_s_31 < 4; ++ax0_1_s_31) {
    if (ax0_1_s_31 < 2) {
      A_shared_dyn_local[(ax0_1_s_31 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_31 * 20)) + 4499)];
    }
  }
  for (int ax1_0_31 = 0; ax1_0_31 < 2; ++ax1_0_31) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_31 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_31 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 1984));
  }
  for (int i_2_1_s_240 = 0; i_2_1_s_240 < 4; ++i_2_1_s_240) {
    if (i_2_1_s_240 < 2) {
      Y_local[(i_2_1_s_240 * 8)] = (Y_local[(i_2_1_s_240 * 8)] + (A_shared_dyn_local[i_2_1_s_240] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_241 = 0; i_2_1_s_241 < 4; ++i_2_1_s_241) {
    if (i_2_1_s_241 < 2) {
      Y_local[((i_2_1_s_241 * 8) + 1)] = (Y_local[((i_2_1_s_241 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_241] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_242 = 0; i_2_1_s_242 < 4; ++i_2_1_s_242) {
    if (i_2_1_s_242 < 2) {
      Y_local[((i_2_1_s_242 * 8) + 2)] = (Y_local[((i_2_1_s_242 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_242] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_243 = 0; i_2_1_s_243 < 4; ++i_2_1_s_243) {
    if (i_2_1_s_243 < 2) {
      Y_local[((i_2_1_s_243 * 8) + 3)] = (Y_local[((i_2_1_s_243 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_243] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_244 = 0; i_2_1_s_244 < 4; ++i_2_1_s_244) {
    if (i_2_1_s_244 < 2) {
      Y_local[((i_2_1_s_244 * 8) + 4)] = (Y_local[((i_2_1_s_244 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_244] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_245 = 0; i_2_1_s_245 < 4; ++i_2_1_s_245) {
    if (i_2_1_s_245 < 2) {
      Y_local[((i_2_1_s_245 * 8) + 5)] = (Y_local[((i_2_1_s_245 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_245] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_246 = 0; i_2_1_s_246 < 4; ++i_2_1_s_246) {
    if (i_2_1_s_246 < 2) {
      Y_local[((i_2_1_s_246 * 8) + 6)] = (Y_local[((i_2_1_s_246 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_246] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_247 = 0; i_2_1_s_247 < 4; ++i_2_1_s_247) {
    if (i_2_1_s_247 < 2) {
      Y_local[((i_2_1_s_247 * 8) + 7)] = (Y_local[((i_2_1_s_247 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_247] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_32 = 0; ax0_1_s_32 < 4; ++ax0_1_s_32) {
    if (ax0_1_s_32 < 2) {
      A_shared_dyn_local[ax0_1_s_32] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_32 * 20)) + 5120)];
    }
  }
  for (int ax1_0_32 = 0; ax1_0_32 < 2; ++ax1_0_32) {
    *(float4*)(B_shared_dyn_local + (ax1_0_32 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_32 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2048));
  }
  for (int i_2_1_s_248 = 0; i_2_1_s_248 < 4; ++i_2_1_s_248) {
    if (i_2_1_s_248 < 2) {
      Y_local[(i_2_1_s_248 * 8)] = (Y_local[(i_2_1_s_248 * 8)] + (A_shared_dyn_local[(i_2_1_s_248 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_249 = 0; i_2_1_s_249 < 4; ++i_2_1_s_249) {
    if (i_2_1_s_249 < 2) {
      Y_local[((i_2_1_s_249 * 8) + 1)] = (Y_local[((i_2_1_s_249 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_249 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_250 = 0; i_2_1_s_250 < 4; ++i_2_1_s_250) {
    if (i_2_1_s_250 < 2) {
      Y_local[((i_2_1_s_250 * 8) + 2)] = (Y_local[((i_2_1_s_250 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_250 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_251 = 0; i_2_1_s_251 < 4; ++i_2_1_s_251) {
    if (i_2_1_s_251 < 2) {
      Y_local[((i_2_1_s_251 * 8) + 3)] = (Y_local[((i_2_1_s_251 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_251 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_252 = 0; i_2_1_s_252 < 4; ++i_2_1_s_252) {
    if (i_2_1_s_252 < 2) {
      Y_local[((i_2_1_s_252 * 8) + 4)] = (Y_local[((i_2_1_s_252 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_252 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_253 = 0; i_2_1_s_253 < 4; ++i_2_1_s_253) {
    if (i_2_1_s_253 < 2) {
      Y_local[((i_2_1_s_253 * 8) + 5)] = (Y_local[((i_2_1_s_253 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_253 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_254 = 0; i_2_1_s_254 < 4; ++i_2_1_s_254) {
    if (i_2_1_s_254 < 2) {
      Y_local[((i_2_1_s_254 * 8) + 6)] = (Y_local[((i_2_1_s_254 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_254 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_255 = 0; i_2_1_s_255 < 4; ++i_2_1_s_255) {
    if (i_2_1_s_255 < 2) {
      Y_local[((i_2_1_s_255 * 8) + 7)] = (Y_local[((i_2_1_s_255 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_255 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_33 = 0; ax0_1_s_33 < 4; ++ax0_1_s_33) {
    if (ax0_1_s_33 < 2) {
      A_shared_dyn_local[(ax0_1_s_33 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_33 * 20)) + 5121)];
    }
  }
  for (int ax1_0_33 = 0; ax1_0_33 < 2; ++ax1_0_33) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_33 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_33 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2112));
  }
  for (int i_2_1_s_256 = 0; i_2_1_s_256 < 4; ++i_2_1_s_256) {
    if (i_2_1_s_256 < 2) {
      Y_local[(i_2_1_s_256 * 8)] = (Y_local[(i_2_1_s_256 * 8)] + (A_shared_dyn_local[i_2_1_s_256] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_257 = 0; i_2_1_s_257 < 4; ++i_2_1_s_257) {
    if (i_2_1_s_257 < 2) {
      Y_local[((i_2_1_s_257 * 8) + 1)] = (Y_local[((i_2_1_s_257 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_257] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_258 = 0; i_2_1_s_258 < 4; ++i_2_1_s_258) {
    if (i_2_1_s_258 < 2) {
      Y_local[((i_2_1_s_258 * 8) + 2)] = (Y_local[((i_2_1_s_258 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_258] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_259 = 0; i_2_1_s_259 < 4; ++i_2_1_s_259) {
    if (i_2_1_s_259 < 2) {
      Y_local[((i_2_1_s_259 * 8) + 3)] = (Y_local[((i_2_1_s_259 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_259] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_260 = 0; i_2_1_s_260 < 4; ++i_2_1_s_260) {
    if (i_2_1_s_260 < 2) {
      Y_local[((i_2_1_s_260 * 8) + 4)] = (Y_local[((i_2_1_s_260 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_260] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_261 = 0; i_2_1_s_261 < 4; ++i_2_1_s_261) {
    if (i_2_1_s_261 < 2) {
      Y_local[((i_2_1_s_261 * 8) + 5)] = (Y_local[((i_2_1_s_261 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_261] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_262 = 0; i_2_1_s_262 < 4; ++i_2_1_s_262) {
    if (i_2_1_s_262 < 2) {
      Y_local[((i_2_1_s_262 * 8) + 6)] = (Y_local[((i_2_1_s_262 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_262] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_263 = 0; i_2_1_s_263 < 4; ++i_2_1_s_263) {
    if (i_2_1_s_263 < 2) {
      Y_local[((i_2_1_s_263 * 8) + 7)] = (Y_local[((i_2_1_s_263 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_263] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_34 = 0; ax0_1_s_34 < 4; ++ax0_1_s_34) {
    if (ax0_1_s_34 < 2) {
      A_shared_dyn_local[ax0_1_s_34] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_34 * 20)) + 5122)];
    }
  }
  for (int ax1_0_34 = 0; ax1_0_34 < 2; ++ax1_0_34) {
    *(float4*)(B_shared_dyn_local + (ax1_0_34 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_34 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2176));
  }
  for (int i_2_1_s_264 = 0; i_2_1_s_264 < 4; ++i_2_1_s_264) {
    if (i_2_1_s_264 < 2) {
      Y_local[(i_2_1_s_264 * 8)] = (Y_local[(i_2_1_s_264 * 8)] + (A_shared_dyn_local[(i_2_1_s_264 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_265 = 0; i_2_1_s_265 < 4; ++i_2_1_s_265) {
    if (i_2_1_s_265 < 2) {
      Y_local[((i_2_1_s_265 * 8) + 1)] = (Y_local[((i_2_1_s_265 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_265 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_266 = 0; i_2_1_s_266 < 4; ++i_2_1_s_266) {
    if (i_2_1_s_266 < 2) {
      Y_local[((i_2_1_s_266 * 8) + 2)] = (Y_local[((i_2_1_s_266 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_266 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_267 = 0; i_2_1_s_267 < 4; ++i_2_1_s_267) {
    if (i_2_1_s_267 < 2) {
      Y_local[((i_2_1_s_267 * 8) + 3)] = (Y_local[((i_2_1_s_267 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_267 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_268 = 0; i_2_1_s_268 < 4; ++i_2_1_s_268) {
    if (i_2_1_s_268 < 2) {
      Y_local[((i_2_1_s_268 * 8) + 4)] = (Y_local[((i_2_1_s_268 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_268 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_269 = 0; i_2_1_s_269 < 4; ++i_2_1_s_269) {
    if (i_2_1_s_269 < 2) {
      Y_local[((i_2_1_s_269 * 8) + 5)] = (Y_local[((i_2_1_s_269 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_269 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_270 = 0; i_2_1_s_270 < 4; ++i_2_1_s_270) {
    if (i_2_1_s_270 < 2) {
      Y_local[((i_2_1_s_270 * 8) + 6)] = (Y_local[((i_2_1_s_270 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_270 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_271 = 0; i_2_1_s_271 < 4; ++i_2_1_s_271) {
    if (i_2_1_s_271 < 2) {
      Y_local[((i_2_1_s_271 * 8) + 7)] = (Y_local[((i_2_1_s_271 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_271 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_35 = 0; ax0_1_s_35 < 4; ++ax0_1_s_35) {
    if (ax0_1_s_35 < 2) {
      A_shared_dyn_local[(ax0_1_s_35 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_35 * 20)) + 5123)];
    }
  }
  for (int ax1_0_35 = 0; ax1_0_35 < 2; ++ax1_0_35) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_35 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_35 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2240));
  }
  for (int i_2_1_s_272 = 0; i_2_1_s_272 < 4; ++i_2_1_s_272) {
    if (i_2_1_s_272 < 2) {
      Y_local[(i_2_1_s_272 * 8)] = (Y_local[(i_2_1_s_272 * 8)] + (A_shared_dyn_local[i_2_1_s_272] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_273 = 0; i_2_1_s_273 < 4; ++i_2_1_s_273) {
    if (i_2_1_s_273 < 2) {
      Y_local[((i_2_1_s_273 * 8) + 1)] = (Y_local[((i_2_1_s_273 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_273] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_274 = 0; i_2_1_s_274 < 4; ++i_2_1_s_274) {
    if (i_2_1_s_274 < 2) {
      Y_local[((i_2_1_s_274 * 8) + 2)] = (Y_local[((i_2_1_s_274 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_274] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_275 = 0; i_2_1_s_275 < 4; ++i_2_1_s_275) {
    if (i_2_1_s_275 < 2) {
      Y_local[((i_2_1_s_275 * 8) + 3)] = (Y_local[((i_2_1_s_275 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_275] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_276 = 0; i_2_1_s_276 < 4; ++i_2_1_s_276) {
    if (i_2_1_s_276 < 2) {
      Y_local[((i_2_1_s_276 * 8) + 4)] = (Y_local[((i_2_1_s_276 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_276] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_277 = 0; i_2_1_s_277 < 4; ++i_2_1_s_277) {
    if (i_2_1_s_277 < 2) {
      Y_local[((i_2_1_s_277 * 8) + 5)] = (Y_local[((i_2_1_s_277 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_277] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_278 = 0; i_2_1_s_278 < 4; ++i_2_1_s_278) {
    if (i_2_1_s_278 < 2) {
      Y_local[((i_2_1_s_278 * 8) + 6)] = (Y_local[((i_2_1_s_278 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_278] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_279 = 0; i_2_1_s_279 < 4; ++i_2_1_s_279) {
    if (i_2_1_s_279 < 2) {
      Y_local[((i_2_1_s_279 * 8) + 7)] = (Y_local[((i_2_1_s_279 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_279] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_36 = 0; ax0_1_s_36 < 4; ++ax0_1_s_36) {
    if (ax0_1_s_36 < 2) {
      A_shared_dyn_local[ax0_1_s_36] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_36 * 20)) + 5136)];
    }
  }
  for (int ax1_0_36 = 0; ax1_0_36 < 2; ++ax1_0_36) {
    *(float4*)(B_shared_dyn_local + (ax1_0_36 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_36 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2304));
  }
  for (int i_2_1_s_280 = 0; i_2_1_s_280 < 4; ++i_2_1_s_280) {
    if (i_2_1_s_280 < 2) {
      Y_local[(i_2_1_s_280 * 8)] = (Y_local[(i_2_1_s_280 * 8)] + (A_shared_dyn_local[(i_2_1_s_280 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_281 = 0; i_2_1_s_281 < 4; ++i_2_1_s_281) {
    if (i_2_1_s_281 < 2) {
      Y_local[((i_2_1_s_281 * 8) + 1)] = (Y_local[((i_2_1_s_281 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_281 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_282 = 0; i_2_1_s_282 < 4; ++i_2_1_s_282) {
    if (i_2_1_s_282 < 2) {
      Y_local[((i_2_1_s_282 * 8) + 2)] = (Y_local[((i_2_1_s_282 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_282 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_283 = 0; i_2_1_s_283 < 4; ++i_2_1_s_283) {
    if (i_2_1_s_283 < 2) {
      Y_local[((i_2_1_s_283 * 8) + 3)] = (Y_local[((i_2_1_s_283 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_283 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_284 = 0; i_2_1_s_284 < 4; ++i_2_1_s_284) {
    if (i_2_1_s_284 < 2) {
      Y_local[((i_2_1_s_284 * 8) + 4)] = (Y_local[((i_2_1_s_284 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_284 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_285 = 0; i_2_1_s_285 < 4; ++i_2_1_s_285) {
    if (i_2_1_s_285 < 2) {
      Y_local[((i_2_1_s_285 * 8) + 5)] = (Y_local[((i_2_1_s_285 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_285 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_286 = 0; i_2_1_s_286 < 4; ++i_2_1_s_286) {
    if (i_2_1_s_286 < 2) {
      Y_local[((i_2_1_s_286 * 8) + 6)] = (Y_local[((i_2_1_s_286 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_286 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_287 = 0; i_2_1_s_287 < 4; ++i_2_1_s_287) {
    if (i_2_1_s_287 < 2) {
      Y_local[((i_2_1_s_287 * 8) + 7)] = (Y_local[((i_2_1_s_287 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_287 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_37 = 0; ax0_1_s_37 < 4; ++ax0_1_s_37) {
    if (ax0_1_s_37 < 2) {
      A_shared_dyn_local[(ax0_1_s_37 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_37 * 20)) + 5137)];
    }
  }
  for (int ax1_0_37 = 0; ax1_0_37 < 2; ++ax1_0_37) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_37 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_37 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2368));
  }
  for (int i_2_1_s_288 = 0; i_2_1_s_288 < 4; ++i_2_1_s_288) {
    if (i_2_1_s_288 < 2) {
      Y_local[(i_2_1_s_288 * 8)] = (Y_local[(i_2_1_s_288 * 8)] + (A_shared_dyn_local[i_2_1_s_288] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_289 = 0; i_2_1_s_289 < 4; ++i_2_1_s_289) {
    if (i_2_1_s_289 < 2) {
      Y_local[((i_2_1_s_289 * 8) + 1)] = (Y_local[((i_2_1_s_289 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_289] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_290 = 0; i_2_1_s_290 < 4; ++i_2_1_s_290) {
    if (i_2_1_s_290 < 2) {
      Y_local[((i_2_1_s_290 * 8) + 2)] = (Y_local[((i_2_1_s_290 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_290] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_291 = 0; i_2_1_s_291 < 4; ++i_2_1_s_291) {
    if (i_2_1_s_291 < 2) {
      Y_local[((i_2_1_s_291 * 8) + 3)] = (Y_local[((i_2_1_s_291 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_291] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_292 = 0; i_2_1_s_292 < 4; ++i_2_1_s_292) {
    if (i_2_1_s_292 < 2) {
      Y_local[((i_2_1_s_292 * 8) + 4)] = (Y_local[((i_2_1_s_292 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_292] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_293 = 0; i_2_1_s_293 < 4; ++i_2_1_s_293) {
    if (i_2_1_s_293 < 2) {
      Y_local[((i_2_1_s_293 * 8) + 5)] = (Y_local[((i_2_1_s_293 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_293] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_294 = 0; i_2_1_s_294 < 4; ++i_2_1_s_294) {
    if (i_2_1_s_294 < 2) {
      Y_local[((i_2_1_s_294 * 8) + 6)] = (Y_local[((i_2_1_s_294 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_294] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_295 = 0; i_2_1_s_295 < 4; ++i_2_1_s_295) {
    if (i_2_1_s_295 < 2) {
      Y_local[((i_2_1_s_295 * 8) + 7)] = (Y_local[((i_2_1_s_295 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_295] * B_shared_dyn_local[7]));
    }
  }
  for (int ax0_1_s_38 = 0; ax0_1_s_38 < 4; ++ax0_1_s_38) {
    if (ax0_1_s_38 < 2) {
      A_shared_dyn_local[ax0_1_s_38] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_38 * 20)) + 5138)];
    }
  }
  for (int ax1_0_38 = 0; ax1_0_38 < 2; ++ax1_0_38) {
    *(float4*)(B_shared_dyn_local + (ax1_0_38 * 4)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_38 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2432));
  }
  for (int i_2_1_s_296 = 0; i_2_1_s_296 < 4; ++i_2_1_s_296) {
    if (i_2_1_s_296 < 2) {
      Y_local[(i_2_1_s_296 * 8)] = (Y_local[(i_2_1_s_296 * 8)] + (A_shared_dyn_local[(i_2_1_s_296 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_297 = 0; i_2_1_s_297 < 4; ++i_2_1_s_297) {
    if (i_2_1_s_297 < 2) {
      Y_local[((i_2_1_s_297 * 8) + 1)] = (Y_local[((i_2_1_s_297 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_297 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_298 = 0; i_2_1_s_298 < 4; ++i_2_1_s_298) {
    if (i_2_1_s_298 < 2) {
      Y_local[((i_2_1_s_298 * 8) + 2)] = (Y_local[((i_2_1_s_298 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_298 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_299 = 0; i_2_1_s_299 < 4; ++i_2_1_s_299) {
    if (i_2_1_s_299 < 2) {
      Y_local[((i_2_1_s_299 * 8) + 3)] = (Y_local[((i_2_1_s_299 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_299 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_300 = 0; i_2_1_s_300 < 4; ++i_2_1_s_300) {
    if (i_2_1_s_300 < 2) {
      Y_local[((i_2_1_s_300 * 8) + 4)] = (Y_local[((i_2_1_s_300 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_300 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_301 = 0; i_2_1_s_301 < 4; ++i_2_1_s_301) {
    if (i_2_1_s_301 < 2) {
      Y_local[((i_2_1_s_301 * 8) + 5)] = (Y_local[((i_2_1_s_301 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_301 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_302 = 0; i_2_1_s_302 < 4; ++i_2_1_s_302) {
    if (i_2_1_s_302 < 2) {
      Y_local[((i_2_1_s_302 * 8) + 6)] = (Y_local[((i_2_1_s_302 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_302 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_303 = 0; i_2_1_s_303 < 4; ++i_2_1_s_303) {
    if (i_2_1_s_303 < 2) {
      Y_local[((i_2_1_s_303 * 8) + 7)] = (Y_local[((i_2_1_s_303 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_303 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax0_1_s_39 = 0; ax0_1_s_39 < 4; ++ax0_1_s_39) {
    if (ax0_1_s_39 < 2) {
      A_shared_dyn_local[(ax0_1_s_39 + 2)] = ((float*)buf_dyn_shmem)[((((((((int)threadIdx.x) & 63) >> 3) * 80) + ((((int)threadIdx.x) & 1) * 40)) + (ax0_1_s_39 * 20)) + 5139)];
    }
  }
  for (int ax1_0_39 = 0; ax1_0_39 < 2; ++ax1_0_39) {
    *(float4*)(B_shared_dyn_local + ((ax1_0_39 * 4) + 8)) = *(float4*)(((float*)buf_dyn_shmem) + ((((ax1_0_39 * 32) + ((((int)threadIdx.x) >> 6) * 16)) + (((((int)threadIdx.x) & 7) >> 1) * 4)) + 2496));
  }
  for (int i_2_1_s_304 = 0; i_2_1_s_304 < 4; ++i_2_1_s_304) {
    if (i_2_1_s_304 < 2) {
      Y_local[(i_2_1_s_304 * 8)] = (Y_local[(i_2_1_s_304 * 8)] + (A_shared_dyn_local[i_2_1_s_304] * B_shared_dyn_local[0]));
    }
  }
  for (int i_2_1_s_305 = 0; i_2_1_s_305 < 4; ++i_2_1_s_305) {
    if (i_2_1_s_305 < 2) {
      Y_local[((i_2_1_s_305 * 8) + 1)] = (Y_local[((i_2_1_s_305 * 8) + 1)] + (A_shared_dyn_local[i_2_1_s_305] * B_shared_dyn_local[1]));
    }
  }
  for (int i_2_1_s_306 = 0; i_2_1_s_306 < 4; ++i_2_1_s_306) {
    if (i_2_1_s_306 < 2) {
      Y_local[((i_2_1_s_306 * 8) + 2)] = (Y_local[((i_2_1_s_306 * 8) + 2)] + (A_shared_dyn_local[i_2_1_s_306] * B_shared_dyn_local[2]));
    }
  }
  for (int i_2_1_s_307 = 0; i_2_1_s_307 < 4; ++i_2_1_s_307) {
    if (i_2_1_s_307 < 2) {
      Y_local[((i_2_1_s_307 * 8) + 3)] = (Y_local[((i_2_1_s_307 * 8) + 3)] + (A_shared_dyn_local[i_2_1_s_307] * B_shared_dyn_local[3]));
    }
  }
  for (int i_2_1_s_308 = 0; i_2_1_s_308 < 4; ++i_2_1_s_308) {
    if (i_2_1_s_308 < 2) {
      Y_local[((i_2_1_s_308 * 8) + 4)] = (Y_local[((i_2_1_s_308 * 8) + 4)] + (A_shared_dyn_local[i_2_1_s_308] * B_shared_dyn_local[4]));
    }
  }
  for (int i_2_1_s_309 = 0; i_2_1_s_309 < 4; ++i_2_1_s_309) {
    if (i_2_1_s_309 < 2) {
      Y_local[((i_2_1_s_309 * 8) + 5)] = (Y_local[((i_2_1_s_309 * 8) + 5)] + (A_shared_dyn_local[i_2_1_s_309] * B_shared_dyn_local[5]));
    }
  }
  for (int i_2_1_s_310 = 0; i_2_1_s_310 < 4; ++i_2_1_s_310) {
    if (i_2_1_s_310 < 2) {
      Y_local[((i_2_1_s_310 * 8) + 6)] = (Y_local[((i_2_1_s_310 * 8) + 6)] + (A_shared_dyn_local[i_2_1_s_310] * B_shared_dyn_local[6]));
    }
  }
  for (int i_2_1_s_311 = 0; i_2_1_s_311 < 4; ++i_2_1_s_311) {
    if (i_2_1_s_311 < 2) {
      Y_local[((i_2_1_s_311 * 8) + 7)] = (Y_local[((i_2_1_s_311 * 8) + 7)] + (A_shared_dyn_local[i_2_1_s_311] * B_shared_dyn_local[7]));
    }
  }
  for (int i_2_1_s_312 = 0; i_2_1_s_312 < 4; ++i_2_1_s_312) {
    if (i_2_1_s_312 < 2) {
      Y_local[(i_2_1_s_312 * 8)] = (Y_local[(i_2_1_s_312 * 8)] + (A_shared_dyn_local[(i_2_1_s_312 + 2)] * B_shared_dyn_local[8]));
    }
  }
  for (int i_2_1_s_313 = 0; i_2_1_s_313 < 4; ++i_2_1_s_313) {
    if (i_2_1_s_313 < 2) {
      Y_local[((i_2_1_s_313 * 8) + 1)] = (Y_local[((i_2_1_s_313 * 8) + 1)] + (A_shared_dyn_local[(i_2_1_s_313 + 2)] * B_shared_dyn_local[9]));
    }
  }
  for (int i_2_1_s_314 = 0; i_2_1_s_314 < 4; ++i_2_1_s_314) {
    if (i_2_1_s_314 < 2) {
      Y_local[((i_2_1_s_314 * 8) + 2)] = (Y_local[((i_2_1_s_314 * 8) + 2)] + (A_shared_dyn_local[(i_2_1_s_314 + 2)] * B_shared_dyn_local[10]));
    }
  }
  for (int i_2_1_s_315 = 0; i_2_1_s_315 < 4; ++i_2_1_s_315) {
    if (i_2_1_s_315 < 2) {
      Y_local[((i_2_1_s_315 * 8) + 3)] = (Y_local[((i_2_1_s_315 * 8) + 3)] + (A_shared_dyn_local[(i_2_1_s_315 + 2)] * B_shared_dyn_local[11]));
    }
  }
  for (int i_2_1_s_316 = 0; i_2_1_s_316 < 4; ++i_2_1_s_316) {
    if (i_2_1_s_316 < 2) {
      Y_local[((i_2_1_s_316 * 8) + 4)] = (Y_local[((i_2_1_s_316 * 8) + 4)] + (A_shared_dyn_local[(i_2_1_s_316 + 2)] * B_shared_dyn_local[12]));
    }
  }
  for (int i_2_1_s_317 = 0; i_2_1_s_317 < 4; ++i_2_1_s_317) {
    if (i_2_1_s_317 < 2) {
      Y_local[((i_2_1_s_317 * 8) + 5)] = (Y_local[((i_2_1_s_317 * 8) + 5)] + (A_shared_dyn_local[(i_2_1_s_317 + 2)] * B_shared_dyn_local[13]));
    }
  }
  for (int i_2_1_s_318 = 0; i_2_1_s_318 < 4; ++i_2_1_s_318) {
    if (i_2_1_s_318 < 2) {
      Y_local[((i_2_1_s_318 * 8) + 6)] = (Y_local[((i_2_1_s_318 * 8) + 6)] + (A_shared_dyn_local[(i_2_1_s_318 + 2)] * B_shared_dyn_local[14]));
    }
  }
  for (int i_2_1_s_319 = 0; i_2_1_s_319 < 4; ++i_2_1_s_319) {
    if (i_2_1_s_319 < 2) {
      Y_local[((i_2_1_s_319 * 8) + 7)] = (Y_local[((i_2_1_s_319 * 8) + 7)] + (A_shared_dyn_local[(i_2_1_s_319 + 2)] * B_shared_dyn_local[15]));
    }
  }
  for (int ax1_0_40 = 0; ax1_0_40 < 2; ++ax1_0_40) {
    *(float4*)(Y + ((((((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) >> 6) * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + (ax1_0_40 * 4))) = *(float4*)(Y_local + (ax1_0_40 * 4));
  }
  for (int ax1_0_41 = 0; ax1_0_41 < 2; ++ax1_0_41) {
    *(float4*)(Y + (((((((((((int)blockIdx.x) >> 2) * 8192) + (((((int)threadIdx.x) & 63) >> 3) * 1024)) + ((((int)threadIdx.x) & 1) * 512)) + ((((int)blockIdx.x) & 3) * 64)) + ((((int)threadIdx.x) >> 6) * 32)) + (((((int)threadIdx.x) & 7) >> 1) * 8)) + (ax1_0_41 * 4)) + 256)) = *(float4*)(Y_local + ((ax1_0_41 * 4) + 8));
  }
}


