
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(512) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  __shared__ float A_shared[4096];
  __shared__ float B_shared[8192];
  for (int i_3_init = 0; i_3_init < 4; ++i_3_init) {
    for (int i_4_init = 0; i_4_init < 2; ++i_4_init) {
      for (int j_4_init = 0; j_4_init < 4; ++j_4_init) {
        Y_local[(((i_3_init * 8) + (i_4_init * 4)) + j_4_init)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 4)) + j_4_init) + 32)] = 0.000000e+00f;
      }
    }
  }
  for (int k_0 = 0; k_0 < 128; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 2; ++ax0_ax1_fused_0) {
      *(float4*)(A_shared + ((ax0_ax1_fused_0 * 2048) + (((int)threadIdx.x) * 4))) = *(float4*)(A + ((((((((int)blockIdx.x) >> 4) * 524288) + (ax0_ax1_fused_0 * 262144)) + ((((int)threadIdx.x) >> 3) * 4096)) + (k_0 * 32)) + ((((int)threadIdx.x) & 7) * 4)));
    }
    for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 4; ++ax0_ax1_fused_0_1) {
      *(float4*)(B_shared + ((ax0_ax1_fused_0_1 * 2048) + (((int)threadIdx.x) * 4))) = *(float4*)(B + (((((k_0 * 131072) + (ax0_ax1_fused_0_1 * 32768)) + ((((int)threadIdx.x) >> 6) * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 63) * 4)));
    }
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {
      for (int i_3 = 0; i_3 < 4; ++i_3) {
        for (int k_2 = 0; k_2 < 16; ++k_2) {
          for (int i_4 = 0; i_4 < 2; ++i_4) {
            for (int j_4 = 0; j_4 < 4; ++j_4) {
              Y_local[(((i_3 * 8) + (i_4 * 4)) + j_4)] = (Y_local[(((i_3 * 8) + (i_4 * 4)) + j_4)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 256) + (i_3 * 64)) + (i_4 * 32)) + (k_1 * 16)) + k_2)] * B_shared[((((k_1 * 4096) + (k_2 * 256)) + ((((int)threadIdx.x) & 31) * 4)) + j_4)]));
              Y_local[((((i_3 * 8) + (i_4 * 4)) + j_4) + 32)] = (Y_local[((((i_3 * 8) + (i_4 * 4)) + j_4) + 32)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 256) + (i_3 * 64)) + (i_4 * 32)) + (k_1 * 16)) + k_2)] * B_shared[(((((k_1 * 4096) + (k_2 * 256)) + ((((int)threadIdx.x) & 31) * 4)) + j_4) + 128)]));
            }
          }
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 8; ++ax0) {
    for (int ax1 = 0; ax1 < 4; ++ax1) {
      Y[(((((((((int)blockIdx.x) >> 4) * 524288) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax0 * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + ax1)] = Y_local[((ax0 * 4) + ax1)];
      Y[((((((((((int)blockIdx.x) >> 4) * 524288) + ((((int)threadIdx.x) >> 5) * 32768)) + (ax0 * 4096)) + ((((int)blockIdx.x) & 15) * 256)) + ((((int)threadIdx.x) & 31) * 4)) + ax1) + 128)] = Y_local[(((ax0 * 4) + ax1) + 32)];
    }
  }
}


