
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[128];
  __shared__ float A_shared[4096];
  __shared__ float B_shared[4096];
  for (int i_3_init = 0; i_3_init < 2; ++i_3_init) {
    Y_local[(i_3_init * 32)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 64)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 1)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 65)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 2)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 66)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 3)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 67)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 4)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 68)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 5)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 69)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 6)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 70)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 7)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 71)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 8)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 72)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 9)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 73)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 10)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 74)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 11)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 75)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 12)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 76)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 13)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 77)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 14)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 78)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 15)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 79)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 16)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 80)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 17)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 81)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 18)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 82)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 19)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 83)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 20)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 84)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 21)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 85)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 22)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 86)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 23)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 87)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 24)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 88)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 25)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 89)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 26)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 90)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 27)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 91)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 28)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 92)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 29)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 93)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 30)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 94)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 31)] = 0.000000e+00f;
    Y_local[((i_3_init * 32) + 95)] = 0.000000e+00f;
  }
  for (int k_0 = 0; k_0 < 96; ++k_0) {
    __syncthreads();
    *(float2*)(A_shared + (((int)threadIdx.x) * 2)) = *(float2*)(A + (((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 24576));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 49152));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 73728));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 98304));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 122880));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 147456));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 172032));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2048)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 196608));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2304)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 221184));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2560)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 245760));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2816)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 270336));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3072)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 294912));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3328)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 319488));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3584)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 344064));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3840)) = *(float2*)(A + ((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 3072)) + (k_0 * 32)) + ((((int)threadIdx.x) & 15) * 2)) + 368640));
    *(float4*)(B_shared + (((int)threadIdx.x) * 4)) = *(float4*)(B + ((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 12288));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 24576));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 36864));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 49152));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 61440));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 73728));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(B + (((((k_0 * 98304) + ((((int)threadIdx.x) >> 5) * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 86016));
    __syncthreads();
    for (int k_1 = 0; k_1 < 16; ++k_1) {
      for (int i_3 = 0; i_3 < 2; ++i_3) {
        for (int k_2 = 0; k_2 < 2; ++k_2) {
          Y_local[(i_3 * 32)] = (Y_local[(i_3 * 32)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 64)] = (Y_local[((i_3 * 32) + 64)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 1)] = (Y_local[((i_3 * 32) + 1)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 65)] = (Y_local[((i_3 * 32) + 65)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 2)] = (Y_local[((i_3 * 32) + 2)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 66)] = (Y_local[((i_3 * 32) + 66)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 3)] = (Y_local[((i_3 * 32) + 3)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 67)] = (Y_local[((i_3 * 32) + 67)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 4)] = (Y_local[((i_3 * 32) + 4)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 68)] = (Y_local[((i_3 * 32) + 68)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 5)] = (Y_local[((i_3 * 32) + 5)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 69)] = (Y_local[((i_3 * 32) + 69)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 6)] = (Y_local[((i_3 * 32) + 6)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 70)] = (Y_local[((i_3 * 32) + 70)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 7)] = (Y_local[((i_3 * 32) + 7)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 71)] = (Y_local[((i_3 * 32) + 71)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 32)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 8)] = (Y_local[((i_3 * 32) + 8)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 72)] = (Y_local[((i_3 * 32) + 72)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 9)] = (Y_local[((i_3 * 32) + 9)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 73)] = (Y_local[((i_3 * 32) + 73)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 10)] = (Y_local[((i_3 * 32) + 10)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 74)] = (Y_local[((i_3 * 32) + 74)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 11)] = (Y_local[((i_3 * 32) + 11)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 75)] = (Y_local[((i_3 * 32) + 75)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 64)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 12)] = (Y_local[((i_3 * 32) + 12)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 76)] = (Y_local[((i_3 * 32) + 76)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 13)] = (Y_local[((i_3 * 32) + 13)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 77)] = (Y_local[((i_3 * 32) + 77)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 14)] = (Y_local[((i_3 * 32) + 14)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 78)] = (Y_local[((i_3 * 32) + 78)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 15)] = (Y_local[((i_3 * 32) + 15)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 79)] = (Y_local[((i_3 * 32) + 79)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 96)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 16)] = (Y_local[((i_3 * 32) + 16)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 80)] = (Y_local[((i_3 * 32) + 80)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 17)] = (Y_local[((i_3 * 32) + 17)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 81)] = (Y_local[((i_3 * 32) + 81)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 18)] = (Y_local[((i_3 * 32) + 18)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 82)] = (Y_local[((i_3 * 32) + 82)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 19)] = (Y_local[((i_3 * 32) + 19)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 83)] = (Y_local[((i_3 * 32) + 83)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 128)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 20)] = (Y_local[((i_3 * 32) + 20)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 84)] = (Y_local[((i_3 * 32) + 84)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 21)] = (Y_local[((i_3 * 32) + 21)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 85)] = (Y_local[((i_3 * 32) + 85)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 22)] = (Y_local[((i_3 * 32) + 22)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 86)] = (Y_local[((i_3 * 32) + 86)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 23)] = (Y_local[((i_3 * 32) + 23)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 87)] = (Y_local[((i_3 * 32) + 87)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 160)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 24)] = (Y_local[((i_3 * 32) + 24)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 88)] = (Y_local[((i_3 * 32) + 88)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 25)] = (Y_local[((i_3 * 32) + 25)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 89)] = (Y_local[((i_3 * 32) + 89)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 26)] = (Y_local[((i_3 * 32) + 26)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 90)] = (Y_local[((i_3 * 32) + 90)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 27)] = (Y_local[((i_3 * 32) + 27)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 91)] = (Y_local[((i_3 * 32) + 91)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 192)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
          Y_local[((i_3 * 32) + 28)] = (Y_local[((i_3 * 32) + 28)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[(((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4))]));
          Y_local[((i_3 * 32) + 92)] = (Y_local[((i_3 * 32) + 92)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)]));
          Y_local[((i_3 * 32) + 29)] = (Y_local[((i_3 * 32) + 29)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)]));
          Y_local[((i_3 * 32) + 93)] = (Y_local[((i_3 * 32) + 93)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)]));
          Y_local[((i_3 * 32) + 30)] = (Y_local[((i_3 * 32) + 30)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)]));
          Y_local[((i_3 * 32) + 94)] = (Y_local[((i_3 * 32) + 94)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)]));
          Y_local[((i_3 * 32) + 31)] = (Y_local[((i_3 * 32) + 31)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)]));
          Y_local[((i_3 * 32) + 95)] = (Y_local[((i_3 * 32) + 95)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 512) + (i_3 * 256)) + (k_1 * 2)) + k_2) + 224)] * B_shared[((((k_1 * 256) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)]));
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 16; ++ax0) {
    Y[((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4))] = Y_local[(ax0 * 4)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 64)] = Y_local[((ax0 * 4) + 64)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 1)] = Y_local[((ax0 * 4) + 1)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 65)] = Y_local[((ax0 * 4) + 65)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 2)] = Y_local[((ax0 * 4) + 2)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 66)] = Y_local[((ax0 * 4) + 66)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 3)] = Y_local[((ax0 * 4) + 3)];
    Y[(((((((((int)blockIdx.x) / 24) * 393216) + ((((int)threadIdx.x) >> 4) * 49152)) + (ax0 * 3072)) + ((((int)blockIdx.x) % 24) * 128)) + ((((int)threadIdx.x) & 15) * 4)) + 67)] = Y_local[((ax0 * 4) + 67)];
  }
}


