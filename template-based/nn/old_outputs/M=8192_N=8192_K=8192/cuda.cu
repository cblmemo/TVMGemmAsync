
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[128];
  __shared__ float A_shared[2048];
  __shared__ float B_shared[2048];
  for (int i_3_init = 0; i_3_init < 2; ++i_3_init) {
    for (int j_3_init = 0; j_3_init < 2; ++j_3_init) {
      for (int i_4_init = 0; i_4_init < 4; ++i_4_init) {
        Y_local[(((i_3_init * 8) + (i_4_init * 2)) + j_3_init)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 16)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 32)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 48)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 64)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 80)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 96)] = 0.000000e+00f;
        Y_local[((((i_3_init * 8) + (i_4_init * 2)) + j_3_init) + 112)] = 0.000000e+00f;
      }
    }
  }
  for (int k_0 = 0; k_0 < 512; ++k_0) {
    __syncthreads();
    *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (k_0 * 16)) + ((((int)threadIdx.x) & 3) * 4)));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(A + ((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (k_0 * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 262144));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(A + ((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (k_0 * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 524288));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(A + ((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 2) * 8192)) + (k_0 * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 786432));
    *(float4*)(B_shared + (((int)threadIdx.x) * 4)) = *(float4*)(B + ((((k_0 * 131072) + ((((int)threadIdx.x) >> 5) * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 31) * 4)));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(B + (((((k_0 * 131072) + ((((int)threadIdx.x) >> 5) * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 32768));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(B + (((((k_0 * 131072) + ((((int)threadIdx.x) >> 5) * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 65536));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(B + (((((k_0 * 131072) + ((((int)threadIdx.x) >> 5) * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 31) * 4)) + 98304));
    __syncthreads();
    for (int k_1 = 0; k_1 < 4; ++k_1) {
      for (int i_3 = 0; i_3 < 2; ++i_3) {
        for (int j_3 = 0; j_3 < 2; ++j_3) {
          for (int k_2 = 0; k_2 < 4; ++k_2) {
            for (int i_4 = 0; i_4 < 4; ++i_4) {
              Y_local[(((i_3 * 8) + (i_4 * 2)) + j_3)] = (Y_local[(((i_3 * 8) + (i_4 * 2)) + j_3)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2)] * B_shared[((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 16)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 16)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 32)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 32)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 32)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 64)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 48)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 48)] + (A_shared[((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 96)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 64)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 64)] + (A_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2) + 1024)] * B_shared[((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 80)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 80)] + (A_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2) + 1024)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 32)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 96)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 96)] + (A_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2) + 1024)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 64)]));
              Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 112)] = (Y_local[((((i_3 * 8) + (i_4 * 2)) + j_3) + 112)] + (A_shared[(((((((((int)threadIdx.x) >> 4) * 128) + (i_3 * 64)) + (i_4 * 16)) + (k_1 * 4)) + k_2) + 1024)] * B_shared[(((((k_1 * 512) + (k_2 * 128)) + ((((int)threadIdx.x) & 15) * 2)) + j_3) + 96)]));
            }
          }
        }
      }
    }
  }
  for (int ax0 = 0; ax0 < 8; ++ax0) {
    Y[((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2))] = Y_local[(ax0 * 2)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 32)] = Y_local[((ax0 * 2) + 16)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 64)] = Y_local[((ax0 * 2) + 32)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 96)] = Y_local[((ax0 * 2) + 48)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524288)] = Y_local[((ax0 * 2) + 64)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524320)] = Y_local[((ax0 * 2) + 80)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524352)] = Y_local[((ax0 * 2) + 96)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524384)] = Y_local[((ax0 * 2) + 112)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = Y_local[((ax0 * 2) + 1)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 33)] = Y_local[((ax0 * 2) + 17)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 65)] = Y_local[((ax0 * 2) + 33)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 97)] = Y_local[((ax0 * 2) + 49)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524289)] = Y_local[((ax0 * 2) + 65)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524321)] = Y_local[((ax0 * 2) + 81)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524353)] = Y_local[((ax0 * 2) + 97)];
    Y[(((((((((int)blockIdx.x) >> 6) * 1048576) + ((((int)threadIdx.x) >> 4) * 65536)) + (ax0 * 8192)) + ((((int)blockIdx.x) & 63) * 128)) + ((((int)threadIdx.x) & 15) * 2)) + 524385)] = Y_local[((ax0 * 2) + 113)];
  }
}


