
#include <hip/hip_runtime.h>

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Y) {
  float Y_local[64];
  __shared__ float A_shared[4096];
  __shared__ float B_shared[2048];
  Y_local[0] = 0.000000e+00f;
  Y_local[32] = 0.000000e+00f;
  Y_local[1] = 0.000000e+00f;
  Y_local[33] = 0.000000e+00f;
  Y_local[2] = 0.000000e+00f;
  Y_local[34] = 0.000000e+00f;
  Y_local[3] = 0.000000e+00f;
  Y_local[35] = 0.000000e+00f;
  Y_local[4] = 0.000000e+00f;
  Y_local[36] = 0.000000e+00f;
  Y_local[5] = 0.000000e+00f;
  Y_local[37] = 0.000000e+00f;
  Y_local[6] = 0.000000e+00f;
  Y_local[38] = 0.000000e+00f;
  Y_local[7] = 0.000000e+00f;
  Y_local[39] = 0.000000e+00f;
  Y_local[8] = 0.000000e+00f;
  Y_local[40] = 0.000000e+00f;
  Y_local[9] = 0.000000e+00f;
  Y_local[41] = 0.000000e+00f;
  Y_local[10] = 0.000000e+00f;
  Y_local[42] = 0.000000e+00f;
  Y_local[11] = 0.000000e+00f;
  Y_local[43] = 0.000000e+00f;
  Y_local[12] = 0.000000e+00f;
  Y_local[44] = 0.000000e+00f;
  Y_local[13] = 0.000000e+00f;
  Y_local[45] = 0.000000e+00f;
  Y_local[14] = 0.000000e+00f;
  Y_local[46] = 0.000000e+00f;
  Y_local[15] = 0.000000e+00f;
  Y_local[47] = 0.000000e+00f;
  Y_local[16] = 0.000000e+00f;
  Y_local[48] = 0.000000e+00f;
  Y_local[17] = 0.000000e+00f;
  Y_local[49] = 0.000000e+00f;
  Y_local[18] = 0.000000e+00f;
  Y_local[50] = 0.000000e+00f;
  Y_local[19] = 0.000000e+00f;
  Y_local[51] = 0.000000e+00f;
  Y_local[20] = 0.000000e+00f;
  Y_local[52] = 0.000000e+00f;
  Y_local[21] = 0.000000e+00f;
  Y_local[53] = 0.000000e+00f;
  Y_local[22] = 0.000000e+00f;
  Y_local[54] = 0.000000e+00f;
  Y_local[23] = 0.000000e+00f;
  Y_local[55] = 0.000000e+00f;
  Y_local[24] = 0.000000e+00f;
  Y_local[56] = 0.000000e+00f;
  Y_local[25] = 0.000000e+00f;
  Y_local[57] = 0.000000e+00f;
  Y_local[26] = 0.000000e+00f;
  Y_local[58] = 0.000000e+00f;
  Y_local[27] = 0.000000e+00f;
  Y_local[59] = 0.000000e+00f;
  Y_local[28] = 0.000000e+00f;
  Y_local[60] = 0.000000e+00f;
  Y_local[29] = 0.000000e+00f;
  Y_local[61] = 0.000000e+00f;
  Y_local[30] = 0.000000e+00f;
  Y_local[62] = 0.000000e+00f;
  Y_local[31] = 0.000000e+00f;
  Y_local[63] = 0.000000e+00f;
  for (int k_0 = 0; k_0 < 32; ++k_0) {
    __syncthreads();
    *(float2*)(A_shared + (((int)threadIdx.x) * 2)) = *(float2*)(A + ((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 2048));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 4096));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 6144));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 8192));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 10240));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 12288));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 14336));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2048)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 16384));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2304)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 18432));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2560)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 20480));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 2816)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 22528));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3072)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 24576));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3328)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 26624));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3584)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 28672));
    *(float2*)(A_shared + ((((int)threadIdx.x) * 2) + 3840)) = *(float2*)(A + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 6) * 1024)) + ((((int)blockIdx.x) >> 4) * 128)) + ((((int)threadIdx.x) & 63) * 2)) + 30720));
    *(float2*)(B_shared + (((int)threadIdx.x) * 2)) = *(float2*)(B + ((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 256)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 4096));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 512)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 8192));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 768)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 12288));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 1024)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 16384));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 1280)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 20480));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 1536)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 24576));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 1792)) = *(float2*)(B + (((((k_0 * 32768) + ((((int)threadIdx.x) >> 5) * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 31) * 2)) + 28672));
    __syncthreads();
    for (int k_1 = 0; k_1 < 2; ++k_1) {
      for (int k_2 = 0; k_2 < 16; ++k_2) {
        Y_local[0] = (Y_local[0] + (A_shared[(((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16))] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[32] = (Y_local[32] + (A_shared[(((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16))] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[1] = (Y_local[1] + (A_shared[(((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16))] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[33] = (Y_local[33] + (A_shared[(((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16))] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[2] = (Y_local[2] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 1)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[34] = (Y_local[34] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 1)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[3] = (Y_local[3] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 1)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[35] = (Y_local[35] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 1)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[4] = (Y_local[4] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 2)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[36] = (Y_local[36] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 2)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[5] = (Y_local[5] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 2)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[37] = (Y_local[37] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 2)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[6] = (Y_local[6] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 3)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[38] = (Y_local[38] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 3)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[7] = (Y_local[7] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 3)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[39] = (Y_local[39] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 3)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[8] = (Y_local[8] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 4)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[40] = (Y_local[40] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 4)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[9] = (Y_local[9] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 4)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[41] = (Y_local[41] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 4)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[10] = (Y_local[10] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 5)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[42] = (Y_local[42] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 5)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[11] = (Y_local[11] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 5)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[43] = (Y_local[43] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 5)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[12] = (Y_local[12] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 6)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[44] = (Y_local[44] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 6)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[13] = (Y_local[13] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 6)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[45] = (Y_local[45] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 6)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[14] = (Y_local[14] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 7)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[46] = (Y_local[46] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 7)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[15] = (Y_local[15] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 7)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[47] = (Y_local[47] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 7)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[16] = (Y_local[16] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 8)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[48] = (Y_local[48] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 8)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[17] = (Y_local[17] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 8)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[49] = (Y_local[49] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 8)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[18] = (Y_local[18] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 9)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[50] = (Y_local[50] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 9)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[19] = (Y_local[19] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 9)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[51] = (Y_local[51] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 9)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[20] = (Y_local[20] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 10)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[52] = (Y_local[52] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 10)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[21] = (Y_local[21] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 10)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[53] = (Y_local[53] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 10)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[22] = (Y_local[22] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 11)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[54] = (Y_local[54] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 11)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[23] = (Y_local[23] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 11)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[55] = (Y_local[55] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 11)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[24] = (Y_local[24] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 12)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[56] = (Y_local[56] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 12)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[25] = (Y_local[25] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 12)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[57] = (Y_local[57] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 12)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[26] = (Y_local[26] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 13)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[58] = (Y_local[58] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 13)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[27] = (Y_local[27] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 13)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[59] = (Y_local[59] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 13)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[28] = (Y_local[28] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 14)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[60] = (Y_local[60] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 14)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[29] = (Y_local[29] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 14)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[61] = (Y_local[61] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 14)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
        Y_local[30] = (Y_local[30] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 15)] * B_shared[(((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2))]));
        Y_local[62] = (Y_local[62] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 15)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)]));
        Y_local[31] = (Y_local[31] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 15)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)]));
        Y_local[63] = (Y_local[63] + (A_shared[((((k_1 * 2048) + (k_2 * 128)) + ((((int)threadIdx.x) >> 4) * 16)) + 15)] * B_shared[((((k_1 * 1024) + (k_2 * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)]));
      }
    }
  }
  Y[(((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2))] = Y_local[0];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 32)] = Y_local[32];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1)] = Y_local[1];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 33)] = Y_local[33];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1024)] = Y_local[2];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1056)] = Y_local[34];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1025)] = Y_local[3];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 1057)] = Y_local[35];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 2048)] = Y_local[4];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 2080)] = Y_local[36];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 2049)] = Y_local[5];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 2081)] = Y_local[37];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 3072)] = Y_local[6];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 3104)] = Y_local[38];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 3073)] = Y_local[7];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 3105)] = Y_local[39];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 4096)] = Y_local[8];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 4128)] = Y_local[40];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 4097)] = Y_local[9];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 4129)] = Y_local[41];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 5120)] = Y_local[10];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 5152)] = Y_local[42];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 5121)] = Y_local[11];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 5153)] = Y_local[43];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 6144)] = Y_local[12];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 6176)] = Y_local[44];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 6145)] = Y_local[13];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 6177)] = Y_local[45];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 7168)] = Y_local[14];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 7200)] = Y_local[46];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 7169)] = Y_local[15];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 7201)] = Y_local[47];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 8192)] = Y_local[16];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 8224)] = Y_local[48];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 8193)] = Y_local[17];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 8225)] = Y_local[49];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 9216)] = Y_local[18];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 9248)] = Y_local[50];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 9217)] = Y_local[19];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 9249)] = Y_local[51];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 10240)] = Y_local[20];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 10272)] = Y_local[52];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 10241)] = Y_local[21];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 10273)] = Y_local[53];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 11264)] = Y_local[22];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 11296)] = Y_local[54];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 11265)] = Y_local[23];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 11297)] = Y_local[55];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 12288)] = Y_local[24];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 12320)] = Y_local[56];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 12289)] = Y_local[25];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 12321)] = Y_local[57];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 13312)] = Y_local[26];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 13344)] = Y_local[58];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 13313)] = Y_local[27];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 13345)] = Y_local[59];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 14336)] = Y_local[28];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 14368)] = Y_local[60];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 14337)] = Y_local[29];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 14369)] = Y_local[61];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 15360)] = Y_local[30];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 15392)] = Y_local[62];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 15361)] = Y_local[31];
  Y[((((((((int)blockIdx.x) >> 4) * 131072) + ((((int)threadIdx.x) >> 4) * 16384)) + ((((int)blockIdx.x) & 15) * 64)) + ((((int)threadIdx.x) & 15) * 2)) + 15393)] = Y_local[63];
}


