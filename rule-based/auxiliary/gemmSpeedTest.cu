#include "hip/hip_runtime.h"
#include <stdio.h>

#include "include/inc.h"

class CUDAMatrix {
private:
    float *mat, *dev_mat;
    int bat, row, col;
    size_t size;

public:
    CUDAMatrix(int b, int r, int c) : bat(b), row(r), col(c), size(sizeof(float) * b * r * c) { mat = (float *) malloc(size); hipMalloc((void **) &dev_mat, size); }
    ~CUDAMatrix() { free(mat), hipFree(dev_mat); }
    float *devPtr() { return dev_mat; }
    void cpyToDevice() { hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice); }
    void cpyToHost() { hipMemcpy(mat, dev_mat, size, hipMemcpyDeviceToHost); }
    float &at(int b, int i, int j) { return mat[b * row * col + i * col + j]; }
    const float &at(int b, int i, int j) const { return mat[b * row * col + i * col + j]; }
    // void print() {
    //     for (int i = 0; i < row; i++) {
    //         for (int j = 0; j < col; j++) {
    //             printf("%.2f ", mat[i * col + j]);
    //         }
    //         printf("\n");
    //     }
    // }
    void fillRand() {
        for (int b = 0; b < bat; b++)
            for (int i = 0; i < row; i++)
                for (int j = 0; j < col; j++)
                    at(b, i, j) = rand() % 100;
    }
    void fill(float val) {
        for (int b = 0; b < bat; b++)
            for (int i = 0; i < row; i++)
                for (int j = 0; j < col; j++)
                    at(b, i, j) = val;
    }
};

float equal(float a, float b) { return abs(a - b) < 0.001; }

float multiTest(void (*calc)(float *, float *, float *), int round = 100) {
    srand(time(nullptr));
    // fprintf(stderr, "srand seed: %ld\n", time(nullptr));
    // srand(1666362747);
    static const int warmupGoal = round / 10;
    CUDAMatrix AM(B, K, M), BM(B, K, N), CM(B, M, N), ans(B, M, N);
    double totTime = 0;
    AM.fillRand(), BM.fillRand(); 
    AM.cpyToDevice(), BM.cpyToDevice();
    for (int i = 0; i < warmupGoal; i++) calc(AM.devPtr(), BM.devPtr(), CM.devPtr());
    for (int i = 0; i < round; i++) {
        // A.fillRand(), B.fillRand();
        // A.cpyToDevice(), B.cpyToDevice();
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        calc(AM.devPtr(), BM.devPtr(), CM.devPtr());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float ti;
        hipEventElapsedTime(&ti, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        totTime += ti;
        // C.cpyToHost();
        // printf("%f\n", ti);
    }
    CM.cpyToHost();
    double FLOPs = (((long long int) (M)) * N * K + M * N) * 2 * B;
    double runtime = totTime / round;
    float ret = double(FLOPs) / runtime / 1.0e6;
    if (ret > 25000) fprintf(stderr, "Cuda Error: %s\n", hipGetErrorString(hipGetLastError()));
    return ret;
}

// void run_tune_old(float *A, float *B, float *C) {
//     if (SHARED_SIZE > (48 << 10)) hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel0), hipFuncAttributeMaxDynamicSharedMemorySize, SHARED_SIZE);
//     main_kernel0<<<BLOCK_NUM, THREAD_NUM, SHARED_SIZE>>>(A, B, C);
// }

void run_tune(float *AM, float *BM, float *CM) {
    main_kernel0<<<BLOCK_NUM, THREAD_NUM>>>(AM, BM, CM);
}

int main() {
    float tune = multiTest(run_tune, 100);
    printf("tune GFLOPs: %f\n", tune);
    return 0;
}