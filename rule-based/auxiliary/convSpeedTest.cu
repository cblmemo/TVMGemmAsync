#include "hip/hip_runtime.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/layout/tensor.h"
#include "cutlass/conv/kernel/default_conv2d_fprop.h"
#include "cutlass/conv/device/implicit_gemm_convolution.h"
#include "cutlass/util/device_memory.h"
#include "include/inc.h"
#include <unistd.h>
#include <vector>

// using CutlassBestPerf = cutlass::gemm::device::Gemm<
//     float, cutlass::layout::ColumnMajor,
//     float, cutlass::layout::RowMajor,
//     float, cutlass::layout::RowMajor,
//     float,
//     cutlass::arch::OpClassSimt,
//     cutlass::arch::Sm80,
//     cutlass::gemm::GemmShape<64, 128, 8>,
//     cutlass::gemm::GemmShape<32, 64, 8>,
//     cutlass::gemm::GemmShape<1, 1, 1>,
//     cutlass::epilogue::thread::LinearCombination<
//         float,
//         1,
//         float,
//         float
//     >,
//     cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
//     5,
//     1,
//     1,
//     true,
//     cutlass::arch::OpMultiplyAdd
//     >;

// void run_bestPerf(float *IT, float *WT, float *OT) {
// #ifdef NT_LAYOUT
//     int lda = M;
//     int ldb = N;
//     int ldc = N;
// #elif defined(NN_LAYOUT)
//     int lda = M;
//     int ldb = N;
//     int ldc = M;
// #endif
//     CutlassBestPerf gemm_operator;
//     CutlassBestPerf::Arguments args(
//         {M, N, K},
//         {A, lda},
//         {B, ldb},
//         {C, ldc},
//         {C, ldc},
//         {1, 0}
//     );
//     gemm_operator(args);
// }

/// Conv operation element types for the Gemm equivalent (ImplicitGemm)
using ElementA           = float;
using ElementB           = float;
using ElementC           = float;
using ElementAccumulator = float;
using ElementCompute     = float;


/// Device-level Conv2d instance
using Conv2dFpropKernel = typename cutlass::conv::kernel::DefaultConv2dFprop<
    ElementA, 
    cutlass::layout::TensorNHWC,
    ElementB, 
    cutlass::layout::TensorNHWC,
    ElementC, 
    cutlass::layout::TensorNHWC,
    ElementAccumulator,
    cutlass::arch::OpClassSimt,
    cutlass::arch::Sm80,
    cutlass::gemm::GemmShape<64, 64, 8>,
    cutlass::gemm::GemmShape<32, 64, 8>, 
    cutlass::gemm::GemmShape<1, 1, 1>,
    cutlass::epilogue::thread::LinearCombination<
        ElementC,
        1,
        ElementAccumulator,
        ElementCompute
    >,
    cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<4>,
    5,
    cutlass::arch::OpMultiplyAdd,
    cutlass::conv::IteratorAlgorithm::kOptimized,
    cutlass::conv::StrideSupport::kStrided,
    1,
    1
>::Kernel;

using Conv2dFprop = cutlass::conv::device::ImplicitGemmConvolution<Conv2dFpropKernel>;

cutlass::conv::Conv2dProblemSize problem_size(      
    cutlass::Tensor4DCoord(N, H, W, C),
    cutlass::Tensor4DCoord(R, S, C, K),
    cutlass::Tensor4DCoord(1, 1, 1, 1),
    cutlass::MatrixCoord(1, 1),
    cutlass::MatrixCoord(1, 1),
    cutlass::Tensor4DCoord(N, P, Q, K),
    cutlass::conv::Mode::kCrossCorrelation,
    1
);

size_t workspace_size;
cutlass::device_memory::allocation<uint8_t> workspace;

void preprocess() {
    Conv2dFprop::Arguments arguments {
        problem_size,
        {nullptr, cutlass::layout::TensorNHWC::packed({N, H, W, C})},
        {nullptr, cutlass::layout::TensorNHWC::packed({R, S, C, K})},
        {nullptr, cutlass::layout::TensorNHWC::packed({N, P, Q, K})},
        {nullptr, cutlass::layout::TensorNHWC::packed({N, P, Q, K})},
        {1, 0}
    };
    Conv2dFprop implicit_gemm_op;
    workspace_size = implicit_gemm_op.get_workspace_size(arguments);
    workspace.reset(workspace_size);
}

void run_cutlass(float *IT, float *WT, float *OT) {
    Conv2dFprop::Arguments arguments {
        problem_size,
        {IT, cutlass::layout::TensorNHWC::packed({N, H, W, C})},
        {WT, cutlass::layout::TensorNHWC::packed({R, S, C, K})},
        {OT, cutlass::layout::TensorNHWC::packed({N, P, Q, K})},
        {OT, cutlass::layout::TensorNHWC::packed({N, P, Q, K})},
        {1, 0}
    };
    Conv2dFprop implicit_gemm_op;
    implicit_gemm_op(arguments, workspace.get());
}

void run_tune(float *IT, float *WT, float *OT) {
    main_kernel0<<<BLOCK_NUM, THREAD_NUM>>>(IT, WT, OT);
}

class CUDATensor {
private:
    float *tensor_, *dev_tensor_;
    std::vector<int> shape_;
    size_t size_;

public:
    CUDATensor(std::vector<int> shape) {
        size_ = 1;
        for (int i = 0; i < shape.size(); i++) {
            shape_.push_back(shape[i]);
            size_ *= shape[i];
        }
        tensor_ = (float *) malloc(size_ * sizeof(float));
        hipMalloc((void **) &dev_tensor_, size_ * sizeof(float));
    }
    ~CUDATensor() { free(tensor_), hipFree(dev_tensor_); }
    float *devPtr() { return dev_tensor_; }
    void cpyToDevice() { hipMemcpy(dev_tensor_, tensor_, size_, hipMemcpyHostToDevice); }
    void cpyToHost() { hipMemcpy(tensor_, dev_tensor_, size_, hipMemcpyDeviceToHost); }
    void fillRand() { for (int i = 0; i < size_; i++) tensor_[i] = rand() % 100; }
    void fill(float val) { for (int i = 0; i < size_; i++) tensor_[i] = val; }
};

float multiTest(void (*calc)(float *, float *, float *), int round = 100) {
    srand(time(nullptr));
    // fprintf(stderr, "srand seed: %ld\n", time(nullptr));
    // srand(1666362747);
    static const int warmupGoal = round / 10;
    CUDATensor IT({N, H, W, C}), WT({R, S, C, K}), OT({N, P, Q, K});
    double totTime = 0;
    IT.fillRand(), WT.fillRand();
    IT.cpyToDevice(), WT.cpyToDevice();
    for (int i = 0; i < warmupGoal; i++) calc(IT.devPtr(), WT.devPtr(), OT.devPtr());
    for (int i = 0; i < round; i++) {
        // A.fillRand(), B.fillRand();
        // A.cpyToDevice(), B.cpyToDevice();
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        calc(IT.devPtr(), WT.devPtr(), OT.devPtr());

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float ti;
        hipEventElapsedTime(&ti, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
        totTime += ti;
        // C.cpyToHost();
        // printf("%f\n", ti);
    }
    // printf("%f\n", totTime);
    OT.cpyToHost();
    int gemmM = N * P * Q, gemmN = K, gemmK = R * S * C;
    double FLOPs = (((long long int) (gemmM)) * gemmN * gemmK + gemmM * gemmN) * 2;
    double runtime = totTime / round;
    float ret = double(FLOPs) / runtime / 1.0e6;
    if (ret > 25000) fprintf(stderr, "Cuda Error: %s\n", hipGetErrorString(hipGetLastError()));
    return ret;
}

int main() {
    preprocess();
    float cutlass = multiTest(run_cutlass, 100);
    float tune = multiTest(run_tune, 100);
    printf("tune: %f, cutlass: %f\n", tune, cutlass);
    return 0;
}