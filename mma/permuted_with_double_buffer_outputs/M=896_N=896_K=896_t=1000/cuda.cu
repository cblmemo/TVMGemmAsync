#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(half* __restrict__ A, half* __restrict__ B, half* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  uint1 Y_reindex_m16n8k8_matrixC[64];
  uint4 A_reindex_shared_dyn_local[20];
  uint4 B_reindex_shared_dyn_local[68];
  half A_reindex_shared_dyn_m16n8k8_matrixA[16];
  half B_reindex_shared_dyn_m16n8k8_matrixB[64];
  for (int ax1_0_3_init = 0; ax1_0_3_init < 2; ++ax1_0_3_init) {
    for (int ax0_0_4_init = 0; ax0_0_4_init < 2; ++ax0_0_4_init) {
      for (int ax1_0_4_init = 0; ax1_0_4_init < 8; ++ax1_0_4_init) {
        for (int b = 0; b < 2; ++b) {
          Y_reindex_m16n8k8_matrixC[((((ax0_0_4_init * 32) + (ax1_0_3_init * 16)) + (ax1_0_4_init * 2)) + b)] = make_uint1(__pack_half2(__float2half_rn(0.000000e+00f), __float2half_rn(0.000000e+00f)));
        }
      }
    }
  }
  for (int ax0_ax1_fused_0_cache = 0; ax0_ax1_fused_0_cache < 4; ++ax0_ax1_fused_0_cache) {
    A_reindex_shared_dyn_local[(ax0_ax1_fused_0_cache * 5)] = *(uint4*)(A + ((((((((int)blockIdx.y) / 7) * 114688) + (ax0_ax1_fused_0_cache * 28672)) + (((int)threadIdx.y) * 7168)) + ((((int)threadIdx.x) >> 2) * 896)) + ((((int)threadIdx.x) & 3) * 8)));
  }
  for (int ax0_ax1_fused_0_cache_1 = 0; ax0_ax1_fused_0_cache_1 < 4; ++ax0_ax1_fused_0_cache_1) {
    B_reindex_shared_dyn_local[(ax0_ax1_fused_0_cache_1 * 17)] = *(uint4*)(B + (((((ax0_ax1_fused_0_cache_1 * 7168) + (((int)threadIdx.y) * 1792)) + ((((int)threadIdx.x) >> 4) * 896)) + ((((int)blockIdx.y) % 7) * 128)) + ((((int)threadIdx.x) & 15) * 8)));
  }
  for (int ax0_ax1_fused_0 = 0; ax0_ax1_fused_0 < 4; ++ax0_ax1_fused_0) {
    *(uint4*)(((half*)buf_dyn_shmem) + (((((ax0_ax1_fused_0 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8)) + 8192)) = A_reindex_shared_dyn_local[(ax0_ax1_fused_0 * 5)];
  }
  for (int ax0_ax1_fused_0_1 = 0; ax0_ax1_fused_0_1 < 4; ++ax0_ax1_fused_0_1) {
    *(uint4*)(((half*)buf_dyn_shmem) + ((((ax0_ax1_fused_0_1 * 1024) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + (((((int)threadIdx.x) & 7) ^ ((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 4))) * 8))) = B_reindex_shared_dyn_local[(ax0_ax1_fused_0_1 * 17)];
  }
  __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[8192])) + (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 32)) + ((0 ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[0]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[1]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[2]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[3])
      : "r"(addr)
    );
  }
  for (int ax1_0 = 0; ax1_0 < 4; ++ax1_0) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[0])) + ((((((int)threadIdx.x) & 7) * 128) + ((ax1_0 >> 1) * 64)) + (((((ax1_0 & 1) * 4) + (((int)threadIdx.x) >> 3)) ^ (((int)threadIdx.x) & 7)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0 * 8)))[0]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0 * 8)))[1]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0 * 8)))[2]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0 * 8)))[3])
      : "r"(addr)
    );
  }
  }
  for (int ax2_0_0 = 0; ax2_0_0 < 27; ++ax2_0_0) {
    for (int ax0_ax1_fused_0_cache_2 = 0; ax0_ax1_fused_0_cache_2 < 4; ++ax0_ax1_fused_0_cache_2) {
      A_reindex_shared_dyn_local[(ax0_ax1_fused_0_cache_2 * 5)] = *(uint4*)(A + ((((((((((int)blockIdx.y) / 7) * 114688) + (ax0_ax1_fused_0_cache_2 * 28672)) + (((int)threadIdx.y) * 7168)) + ((((int)threadIdx.x) >> 2) * 896)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 32));
    }
    for (int ax0_ax1_fused_0_cache_3 = 0; ax0_ax1_fused_0_cache_3 < 4; ++ax0_ax1_fused_0_cache_3) {
      B_reindex_shared_dyn_local[(ax0_ax1_fused_0_cache_3 * 17)] = *(uint4*)(B + (((((((ax2_0_0 * 28672) + (ax0_ax1_fused_0_cache_3 * 7168)) + (((int)threadIdx.y) * 1792)) + ((((int)threadIdx.x) >> 4) * 896)) + ((((int)blockIdx.y) % 7) * 128)) + ((((int)threadIdx.x) & 15) * 8)) + 28672));
    }
    for (int ax2_0_1 = 0; ax2_0_1 < 3; ++ax2_0_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[(((ax2_0_0 & 1) * 4096) + 8192)])) + (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 32)) + (((ax2_0_1 + 1) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 + 1) & 1) * 8)))[0]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 + 1) & 1) * 8)))[1]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 + 1) & 1) * 8)))[2]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 + 1) & 1) * 8)))[3])
      : "r"(addr)
    );
  }
      for (int ax1_0_1 = 0; ax1_0_1 < 4; ++ax1_0_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[((ax2_0_0 & 1) * 4096)])) + (((((ax2_0_1 * 1024) + ((((int)threadIdx.x) & 7) * 128)) + ((ax1_0_1 >> 1) * 64)) + (((((ax1_0_1 & 1) * 4) + (((int)threadIdx.x) >> 3)) ^ (((int)threadIdx.x) & 7)) * 8)) + 1024)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1 + 1) & 1) * 32) + (ax1_0_1 * 8))))[0]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1 + 1) & 1) * 32) + (ax1_0_1 * 8))))[1]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1 + 1) & 1) * 32) + (ax1_0_1 * 8))))[2]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1 + 1) & 1) * 32) + (ax1_0_1 * 8))))[3])
      : "r"(addr)
    );
  }
      }
      for (int ax1_0_3 = 0; ax1_0_3 < 2; ++ax1_0_3) {
        for (int ax0_0_4 = 0; ax0_0_4 < 2; ++ax0_0_4) {
          for (int ax1_0_4 = 0; ax1_0_4 < 8; ++ax1_0_4) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3}, {%4}, {%5, %6};\n"
      :  "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4 * 32) + (ax1_0_3 * 16)) + (ax1_0_4 * 2))))[0]), "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4 * 32) + (ax1_0_3 * 16)) + (ax1_0_4 * 2))))[1])
      : "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 & 1) * 8) + (ax0_0_4 * 4))))[0]), "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1 & 1) * 8) + (ax0_0_4 * 4))))[1]), "r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1 & 1) * 32) + (ax1_0_3 * 16)) + (ax1_0_4 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4 * 32) + (ax1_0_3 * 16)) + (ax1_0_4 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4 * 32) + (ax1_0_3 * 16)) + (ax1_0_4 * 2))))[1]));
  }
          }
        }
      }
    }
    __syncthreads();
    for (int ax0_ax1_fused_0_2 = 0; ax0_ax1_fused_0_2 < 4; ++ax0_ax1_fused_0_2) {
      *(uint4*)(((half*)buf_dyn_shmem) + ((((((((ax2_0_0 + 1) & 1) * 4096) + (ax0_ax1_fused_0_2 * 1024)) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 2) * 32)) + (((((int)threadIdx.x) & 3) ^ (((int)threadIdx.x) >> 3)) * 8)) + 8192)) = A_reindex_shared_dyn_local[(ax0_ax1_fused_0_2 * 5)];
    }
    for (int ax0_ax1_fused_0_3 = 0; ax0_ax1_fused_0_3 < 4; ++ax0_ax1_fused_0_3) {
      *(uint4*)(((half*)buf_dyn_shmem) + (((((((ax2_0_0 + 1) & 1) * 4096) + (ax0_ax1_fused_0_3 * 1024)) + (((int)threadIdx.y) * 256)) + ((((int)threadIdx.x) >> 3) * 64)) + (((((int)threadIdx.x) & 7) ^ ((((int)threadIdx.y) * 2) + (((int)threadIdx.x) >> 4))) * 8))) = B_reindex_shared_dyn_local[(ax0_ax1_fused_0_3 * 17)];
    }
    __syncthreads();

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[((((ax2_0_0 + 1) & 1) * 4096) + 8192)])) + (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 32)) + ((0 ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[0]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[1]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[2]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + 0))[3])
      : "r"(addr)
    );
  }
    for (int ax1_0_2 = 0; ax1_0_2 < 4; ++ax1_0_2) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[(((ax2_0_0 + 1) & 1) * 4096)])) + ((((((int)threadIdx.x) & 7) * 128) + ((ax1_0_2 >> 1) * 64)) + (((((ax1_0_2 & 1) * 4) + (((int)threadIdx.x) >> 3)) ^ (((int)threadIdx.x) & 7)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0_2 * 8)))[0]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0_2 * 8)))[1]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0_2 * 8)))[2]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (ax1_0_2 * 8)))[3])
      : "r"(addr)
    );
  }
    }
    for (int ax1_0_3_1 = 0; ax1_0_3_1 < 2; ++ax1_0_3_1) {
      for (int ax0_0_4_1 = 0; ax0_0_4_1 < 2; ++ax0_0_4_1) {
        for (int ax1_0_4_1 = 0; ax1_0_4_1 < 8; ++ax1_0_4_1) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3}, {%4}, {%5, %6};\n"
      :  "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_1 * 32) + (ax1_0_3_1 * 16)) + (ax1_0_4_1 * 2))))[0]), "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_1 * 32) + (ax1_0_3_1 * 16)) + (ax1_0_4_1 * 2))))[1])
      : "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + ((ax0_0_4_1 * 4) + 8)))[0]), "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + ((ax0_0_4_1 * 4) + 8)))[1]), "r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (((ax1_0_3_1 * 16) + (ax1_0_4_1 * 2)) + 32)))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_1 * 32) + (ax1_0_3_1 * 16)) + (ax1_0_4_1 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_1 * 32) + (ax1_0_3_1 * 16)) + (ax1_0_4_1 * 2))))[1]));
  }
        }
      }
    }
  }
  for (int ax2_0_1_1 = 0; ax2_0_1_1 < 3; ++ax2_0_1_1) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[12288])) + (((((int)threadIdx.y) * 1024) + (((int)threadIdx.x) * 32)) + (((ax2_0_1_1 + 1) ^ ((((int)threadIdx.x) & 7) >> 1)) * 8))))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 + 1) & 1) * 8)))[0]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 + 1) & 1) * 8)))[1]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 + 1) & 1) * 8)))[2]), "=r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 + 1) & 1) * 8)))[3])
      : "r"(addr)
    );
  }
    for (int ax1_0_5 = 0; ax1_0_5 < 4; ++ax1_0_5) {

  {
    unsigned int addr;
    __asm__ __volatile__(
      "{ .reg .u64 addr; cvta.to.shared.u64 addr, %1; cvt.u32.u64 %0, addr; }\n"
      : "=r"(addr)
      : "l"((void *)((&(((half*)buf_dyn_shmem)[4096])) + (((((ax2_0_1_1 * 1024) + ((((int)threadIdx.x) & 7) * 128)) + ((ax1_0_5 >> 1) * 64)) + (((((ax1_0_5 & 1) * 4) + (((int)threadIdx.x) >> 3)) ^ (((int)threadIdx.x) & 7)) * 8)) + 1024)))
    );
    __asm__ __volatile__(
      "ldmatrix.sync.aligned.m8n8.x4.trans.shared.b16"
      "{%0, %1, %2, %3}, [%4];\n"
      : "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1_1 + 1) & 1) * 32) + (ax1_0_5 * 8))))[0]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1_1 + 1) & 1) * 32) + (ax1_0_5 * 8))))[1]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1_1 + 1) & 1) * 32) + (ax1_0_5 * 8))))[2]), "=r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1_1 + 1) & 1) * 32) + (ax1_0_5 * 8))))[3])
      : "r"(addr)
    );
  }
    }
    for (int ax1_0_3_2 = 0; ax1_0_3_2 < 2; ++ax1_0_3_2) {
      for (int ax0_0_4_2 = 0; ax0_0_4_2 < 2; ++ax0_0_4_2) {
        for (int ax1_0_4_2 = 0; ax1_0_4_2 < 8; ++ax1_0_4_2) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3}, {%4}, {%5, %6};\n"
      :  "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_2 * 32) + (ax1_0_3_2 * 16)) + (ax1_0_4_2 * 2))))[0]), "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_2 * 32) + (ax1_0_3_2 * 16)) + (ax1_0_4_2 * 2))))[1])
      : "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 & 1) * 8) + (ax0_0_4_2 * 4))))[0]), "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + (((ax2_0_1_1 & 1) * 8) + (ax0_0_4_2 * 4))))[1]), "r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + ((((ax2_0_1_1 & 1) * 32) + (ax1_0_3_2 * 16)) + (ax1_0_4_2 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_2 * 32) + (ax1_0_3_2 * 16)) + (ax1_0_4_2 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_2 * 32) + (ax1_0_3_2 * 16)) + (ax1_0_4_2 * 2))))[1]));
  }
        }
      }
    }
  }
  for (int ax1_0_3_3 = 0; ax1_0_3_3 < 2; ++ax1_0_3_3) {
    for (int ax0_0_4_3 = 0; ax0_0_4_3 < 2; ++ax0_0_4_3) {
      for (int ax1_0_4_3 = 0; ax1_0_4_3 < 8; ++ax1_0_4_3) {

  {
    __asm__ __volatile__(
      "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16"
      "{%0, %1}, {%2, %3}, {%4}, {%5, %6};\n"
      :  "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_3 * 32) + (ax1_0_3_3 * 16)) + (ax1_0_4_3 * 2))))[0]), "=r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_3 * 32) + (ax1_0_3_3 * 16)) + (ax1_0_4_3 * 2))))[1])
      : "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + ((ax0_0_4_3 * 4) + 8)))[0]), "r"(((unsigned *)(A_reindex_shared_dyn_m16n8k8_matrixA + ((ax0_0_4_3 * 4) + 8)))[1]), "r"(((unsigned *)(B_reindex_shared_dyn_m16n8k8_matrixB + (((ax1_0_3_3 * 16) + (ax1_0_4_3 * 2)) + 32)))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_3 * 32) + (ax1_0_3_3 * 16)) + (ax1_0_4_3 * 2))))[0]), "r"(((unsigned *)(Y_reindex_m16n8k8_matrixC + (((ax0_0_4_3 * 32) + (ax1_0_3_3 * 16)) + (ax1_0_4_3 * 2))))[1]));
  }
      }
    }
  }
  for (int ax0_0 = 0; ax0_0 < 4; ++ax0_0) {
    __syncthreads();
    for (int ax1_0_6 = 0; ax1_0_6 < 16; ++ax1_0_6) {
      *(uint1*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 1024) + (ax1_0_6 * 64)) + (((int)threadIdx.x) * 2)) + 8192)) = Y_reindex_m16n8k8_matrixC[((((ax0_0 >> 1) * 32) + (ax1_0_6 * 2)) + (ax0_0 & 1))];
    }
    __syncthreads();
    for (int ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0 = 0; ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0 < 32; ++ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0) {
      Y[((((((((((((int)blockIdx.y) / 7) * 114688) + ((ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0 >> 3) * 28672)) + (ax0_0 * 7168)) + ((((int)threadIdx.y) & 1) * 3584)) + ((((int)threadIdx.x) >> 3) * 896)) + ((((int)blockIdx.y) % 7) * 128)) + ((ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0 & 7) * 16)) + ((((int)threadIdx.y) >> 1) * 8)) + (((int)threadIdx.x) & 7))] = ((half*)buf_dyn_shmem)[((((ax0_0_2_ax1_0_2_fused_cache_ax1_0_cache_ax0_1_cache_ax1_1_cache_fused_0 * 128) + (((int)threadIdx.y) * 32)) + ((int)threadIdx.x)) + 8192)];
    }
  }
}


