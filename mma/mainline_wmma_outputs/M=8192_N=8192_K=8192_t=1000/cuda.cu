#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif
#include <mma.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(half* __restrict__ A, half* __restrict__ B, half* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> Y_reindex_shared_dyn_wmma_accumulator[32];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_reindex_shared_dyn_wmma_matrix_a[4];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> B_reindex_shared_dyn_wmma_matrix_b[8];
  for (int ax0_0_3_init = 0; ax0_0_3_init < 4; ++ax0_0_3_init) {
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[(ax0_0_3_init * 8)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 1)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 2)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 3)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 4)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 5)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 6)], 0.000000e+00f);
    nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3_init * 8) + 7)], 0.000000e+00f);
  }
  for (int ax2_0_0 = 0; ax2_0_0 < 256; ++ax2_0_0) {
    __syncthreads();
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 4352)) = *(uint4*)(A + (((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 5632)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 262144));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 6912)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 524288));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 8192)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 786432));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 9472)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 1048576));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 10752)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 1310720));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 12032)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 1572864));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 13312)) = *(uint4*)(A + ((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + (((int)threadIdx.y) * 65536)) + ((((int)threadIdx.x) >> 2) * 8192)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 1835008));
    *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4))) = *(uint2*)(B + ((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 544)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 32768));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 1088)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 65536));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 1632)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 98304));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 2176)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 131072));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 2720)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 163840));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 3264)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 196608));
    *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 136) + (((int)threadIdx.x) * 4)) + 3808)) = *(uint2*)(B + (((((ax2_0_0 * 262144) + (((int)threadIdx.y) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + (((int)threadIdx.x) * 4)) + 229376));
    __syncthreads();
    for (int ax2_0_1 = 0; ax2_0_1 < 2; ++ax2_0_1) {
      nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[0], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) * 2560) + (ax2_0_1 * 16)) + 4352)])), 40);
      nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[1], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) * 2560) + (ax2_0_1 * 16)) + 4992)])), 40);
      nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[2], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) * 2560) + (ax2_0_1 * 16)) + 5632)])), 40);
      nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[3], (&(((half*)buf_dyn_shmem)[(((((int)threadIdx.y) * 2560) + (ax2_0_1 * 16)) + 6272)])), 40);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[0], (&(((half*)buf_dyn_shmem)[(ax2_0_1 * 2176)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[1], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 16)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[2], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 32)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[3], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 48)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[4], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 64)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[5], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 80)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[6], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 96)])), 136);
      nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[7], (&(((half*)buf_dyn_shmem)[((ax2_0_1 * 2176) + 112)])), 136);
      for (int ax0_0_3 = 0; ax0_0_3 < 4; ++ax0_0_3) {
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[(ax0_0_3 * 8)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 1)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 2)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[3], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 3)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[4], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 4)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[5], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 5)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[6], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 6)]);
        nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)], A_reindex_shared_dyn_wmma_matrix_a[ax0_0_3], B_reindex_shared_dyn_wmma_matrix_b[7], Y_reindex_shared_dyn_wmma_accumulator[((ax0_0_3 * 8) + 7)]);
      }
    }
  }
  for (int ax2 = 0; ax2 < 4; ++ax2) {
    __syncthreads();
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 4352)])), Y_reindex_shared_dyn_wmma_accumulator[(ax2 * 8)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 4608)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 1)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 4864)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 2)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 5120)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 3)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 5376)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 4)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 5632)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 5)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 5888)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 6)], 16, nvcuda::wmma::mem_row_major);
    nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 2048) + 6144)])), Y_reindex_shared_dyn_wmma_accumulator[((ax2 * 8) + 7)], 16, nvcuda::wmma::mem_row_major);
    __syncthreads();
    for (int ax0_ax1_ax3_ax4_ax5_fused_0 = 0; ax0_ax1_ax3_ax4_ax5_fused_0 < 32; ++ax0_ax1_ax3_ax4_ax5_fused_0) {
      *(uint1*)(Y + ((((((((((((int)blockIdx.y) >> 6) * 8388608) + (((int)blockIdx.x) * 2097152)) + ((ax0_ax1_ax3_ax4_ax5_fused_0 >> 3) * 524288)) + (ax2 * 131072)) + (((int)threadIdx.y) * 32768)) + ((((int)threadIdx.x) >> 3) * 8192)) + ((((int)blockIdx.y) & 63) * 128)) + ((ax0_ax1_ax3_ax4_ax5_fused_0 & 7) * 16)) + ((((int)threadIdx.x) & 7) * 2))) = *(uint1*)(((half*)buf_dyn_shmem) + ((((ax0_ax1_ax3_ax4_ax5_fused_0 * 256) + (((int)threadIdx.y) * 64)) + (((int)threadIdx.x) * 2)) + 4352));
    }
  }
}


