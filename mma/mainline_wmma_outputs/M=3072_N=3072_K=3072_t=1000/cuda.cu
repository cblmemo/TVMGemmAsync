#include "hip/hip_runtime.h"
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#include <hip/hip_fp16.h>
__device__ half max(half a, half b)
{
  return __hgt(__half(a), __half(b)) ? a : b;
}
__device__ half min(half a, half b)
{
  return __hlt(__half(a), __half(b)) ? a : b;
}
#else

typedef unsigned short uint16_t;
typedef unsigned char uint8_t;
typedef signed char int8_t;
typedef int int32_t;
typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

#define TVM_FORCE_INLINE inline __attribute__((always_inline))
#define TVM_XINLINE TVM_FORCE_INLINE __device__ __host__
#define TVM_ALIGNED(x) __attribute__ ((aligned(x)))
#define TVM_HALF_OPERATOR(RTYPE, OP)                              \
  TVM_XINLINE RTYPE operator OP (half a, half b) {                \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (half a, T b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE RTYPE operator OP (T a, half b) {                   \
    return RTYPE(float(a) OP float(b));                           \
  }

#define TVM_HALF_ASSIGNOP(AOP, OP)                                \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const T& a) {                    \
    return *this = half(float(*this) OP float(a));                \
  }                                                               \
  template<typename T>                                            \
  TVM_XINLINE half operator AOP (const volatile T& a) volatile {  \
    return *this = half(float(*this) OP float(a));                \
  }

class TVM_ALIGNED(2) half {
 public:
  uint16_t half_;

  static TVM_XINLINE half Binary(uint16_t value) {
    half res;
    res.half_ = value;
    return res;
  }

  TVM_XINLINE half() {}

  TVM_XINLINE half(const float& value) { constructor(value); }
  TVM_XINLINE explicit half(const double& value) { constructor(value); }
  TVM_XINLINE explicit half(const int8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint8_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const int32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint32_t& value) { constructor(value); }
  TVM_XINLINE explicit half(const long long& value) { constructor(value); }
  TVM_XINLINE explicit half(const uint64_t& value) { constructor(value); }

  TVM_XINLINE operator float() const {                          \
    return float(half2float(half_));                            \
  }                                                             \
  TVM_XINLINE operator float() const volatile {                 \
    return float(half2float(half_));                            \
  }


  TVM_HALF_ASSIGNOP(+=, +)
  TVM_HALF_ASSIGNOP(-=, -)
  TVM_HALF_ASSIGNOP(*=, *)
  TVM_HALF_ASSIGNOP(/=, /)

  TVM_XINLINE half operator+() {
    return *this;
  }

  TVM_XINLINE half operator-() {
    return half(-float(*this));
  }

  TVM_XINLINE half operator=(const half& a) {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) {
    return *this = half(a);
  }

  TVM_XINLINE half operator=(const half& a) volatile {
    half_ = a.half_;
    return a;
  }

  template<typename T>
  TVM_XINLINE half operator=(const T& a) volatile {
    return *this = half(a);
  }

 private:
  union Bits {
    float f;
    int32_t si;
    uint32_t ui;
  };

  static int const fp16FractionBits = 10;
  static int const fp32FractionBits = 23;
  static int32_t const fp32FractionMask = ~(~0u << fp32FractionBits);   // == 0x7fffff
  static int32_t const fp32HiddenBit = 1 << fp32FractionBits;   // == 0x800000
  static int const shift = fp32FractionBits - fp16FractionBits;   // == 13
  static int const shiftSign = 16;
  static int32_t const expAdjust = 127 - 15;   // exp32-127 = exp16-15, so exp16 = exp32 - (127-15)

  static int32_t const infN = 0x7F800000;   // flt32 infinity
  static int32_t const maxN = 0x477FFFFF;   // max flt32 that's a flt16 normal after >> by shift
  static int32_t const minN = 0x38800000;   // min flt16 normal as a flt32
  static int32_t const maxZ = 0x33000000;   // max fp32 number that's still rounded to zero in fp16
  static int32_t const signN = 0x80000000;  // flt32 sign bit

  static int32_t const infC = infN >> shift;
  static int32_t const nanN = (infC + 1) << shift;   // minimum flt16 nan as a flt32
  static int32_t const maxC = maxN >> shift;
  static int32_t const minC = minN >> shift;
  static int32_t const signC = signN >> shiftSign;  // flt16 sign bit

  static int32_t const mulN = 0x52000000;  // (1 << 23) / minN
  static int32_t const mulC = 0x33800000;  // minN / (1 << (23 - shift))

  static int32_t const subC = 0x003FF;  // max flt32 subnormal down shifted
  static int32_t const norC = 0x00400;  // min flt32 normal down shifted

  static int32_t const maxD = infC - maxC - 1;
  static int32_t const minD = minC - subC - 1;

  TVM_XINLINE uint16_t float2half(const float& value) const {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  // Same as above routine, except for addition of volatile keyword
  TVM_XINLINE uint16_t float2half(
    const volatile float& value) const volatile {
    Bits v;
    v.f = value;
    uint32_t sign = v.si & signN;    // grab sign bit
    v.si ^= sign;                    // clear sign bit from v
    sign >>= shiftSign;              // logical shift sign to fp16 position

    if (v.si <= maxZ) {
      // Handle eventual zeros here to ensure
      // vshift will not exceed 32 below.
      v.ui = 0;
    } else if (v.si < minN) {
      // Handle denorms
      uint32_t exp32 = v.ui >> fp32FractionBits;
      int32_t exp16 = exp32 - expAdjust;
      // If exp16 == 0 (just into the denorm range), then significant should be shifted right 1.
      // Smaller (so negative) exp16 values should result in greater right shifts.
      uint32_t vshift = 1 - exp16;
      uint32_t significand = fp32HiddenBit | (v.ui & fp32FractionMask);
      v.ui = significand >> vshift;
      v.ui += (v.ui & 0x3fff) != 0x1000 || (significand & 0x7ff) ? 0x1000 : 0;
    } else if (v.si <= maxN) {
      // Handle norms
      v.ui += (v.ui & 0x3fff) != 0x1000 ? 0x1000 : 0;
      v.ui -= expAdjust << fp32FractionBits;
    } else if (v.si <= infN) {
      v.si = infN;
    } else if (v.si < nanN) {
      v.si = nanN;
    }

    v.ui >>= shift;
    return sign | (v.ui & 0x7fff);
  }

  TVM_XINLINE float half2float(const uint16_t& value) const {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  TVM_XINLINE float half2float(
    const volatile uint16_t& value) const volatile {
    Bits v;
    v.ui = value;
    int32_t sign = v.si & signC;
    v.si ^= sign;
    sign <<= shiftSign;
    v.si ^= ((v.si + minD) ^ v.si) & -(v.si > subC);
    v.si ^= ((v.si + maxD) ^ v.si) & -(v.si > maxC);
    Bits s;
    s.si = mulC;
    s.f *= v.si;
    int32_t mask = -(norC > v.si);
    v.si <<= shift;
    v.si ^= (s.si ^ v.si) & mask;
    v.si |= sign;
    return v.f;
  }

  template<typename T>
  TVM_XINLINE void constructor(const T& value) {
    half_ = float2half(float(value));
  }
};

TVM_HALF_OPERATOR(half, +)
TVM_HALF_OPERATOR(half, -)
TVM_HALF_OPERATOR(half, *)
TVM_HALF_OPERATOR(half, /)
TVM_HALF_OPERATOR(bool, >)
TVM_HALF_OPERATOR(bool, <)
TVM_HALF_OPERATOR(bool, >=)
TVM_HALF_OPERATOR(bool, <=)

TVM_XINLINE half __float2half_rn(const float a) {
  return half(a);
}
#endif


// Pack two half values.
static inline __device__ __host__ unsigned
__pack_half2(const half x, const half y) {
  unsigned v0 = *((unsigned short *)&x);
  unsigned v1 = *((unsigned short *)&y);
  return (v1 << 16) | v0;
}

// Some fp16 math functions are not supported in hip/hip_fp16.h,
// so we define them here to make sure the generated CUDA code
// is valid.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 530)
#define CUDA_UNSUPPORTED_HALF_MATH_BINARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x, half y) {   \
  float tmp_x = __half2float(x);                                          \
  float tmp_y = __half2float(y);                                          \
  float result = FP32_MATH_NAME(tmp_x, tmp_y);                            \
  return __float2half(result);                                            \
}

#define CUDA_UNSUPPORTED_HALF_MATH_UNARY(HALF_MATH_NAME, FP32_MATH_NAME) \
static inline __device__ __host__ half HALF_MATH_NAME(half x) {          \
  float tmp_x = __half2float(x);                                         \
  float result = FP32_MATH_NAME(tmp_x);                                  \
  return __float2half(result);                                           \
}

CUDA_UNSUPPORTED_HALF_MATH_BINARY(hpow, powf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htanh, tanhf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(htan, tanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(hatan, atanf)
CUDA_UNSUPPORTED_HALF_MATH_UNARY(herf, erf)

#undef CUDA_UNSUPPORTED_HALF_MATH_BINARY
#undef CUDA_UNSUPPORTED_HALF_MATH_UNARY

#endif
#include <mma.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel0(half* __restrict__ A, half* __restrict__ B, half* __restrict__ Y) {
  extern __shared__ uchar buf_dyn_shmem[];
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, half> Y_reindex_shared_dyn_wmma_accumulator[36];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, half, nvcuda::wmma::row_major> A_reindex_shared_dyn_wmma_matrix_a[12];
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, half, nvcuda::wmma::row_major> B_reindex_shared_dyn_wmma_matrix_b[3];
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[0], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[1], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[2], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[3], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[4], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[5], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[6], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[7], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[8], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[9], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[10], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[11], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[12], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[13], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[14], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[15], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[16], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[17], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[18], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[19], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[20], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[21], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[22], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[23], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[24], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[25], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[26], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[27], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[28], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[29], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[30], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[31], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[32], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[33], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[34], 0.000000e+00f);
  nvcuda::wmma::fill_fragment(Y_reindex_shared_dyn_wmma_accumulator[35], 0.000000e+00f);
  for (int ax2_0_0 = 0; ax2_0_0 < 96; ++ax2_0_0) {
    __syncthreads();
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 6400)) = *(uint4*)(A + (((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 7680)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 98304));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 8960)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 196608));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 10240)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 294912));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 11520)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 393216));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((int)threadIdx.y) * 320) + ((((int)threadIdx.x) >> 2) * 40)) + ((((int)threadIdx.x) & 3) * 8)) + 12800)) = *(uint4*)(A + ((((((((int)blockIdx.y) * 589824) + (((int)threadIdx.y) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (ax2_0_0 * 32)) + ((((int)threadIdx.x) & 3) * 8)) + 491520));
    *(uint4*)(((half*)buf_dyn_shmem) + (((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) / 192) * 200) + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) % 192))) = *(uint4*)(B + ((((ax2_0_0 * 98304) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) % 192)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 1024) / 192) * 200) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 64) % 192))) = *(uint4*)(B + ((((ax2_0_0 * 98304) + (((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 1024) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 64) % 192)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 2048) / 192) * 200) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 128) % 192))) = *(uint4*)(B + ((((ax2_0_0 * 98304) + (((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 2048) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 128) % 192)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) / 192) * 200) + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) % 192)) + 3200)) = *(uint4*)(B + (((((ax2_0_0 * 98304) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + (((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 8)) % 192)) + 49152));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 4096) / 192) * 200) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 64) % 192))) = *(uint4*)(B + ((((ax2_0_0 * 98304) + (((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 4096) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 64) % 192)));
    *(uint4*)(((half*)buf_dyn_shmem) + ((((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 5120) / 192) * 200) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 128) % 192))) = *(uint4*)(B + ((((ax2_0_0 * 98304) + (((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 5120) / 192) * 3072)) + (((int)blockIdx.x) * 192)) + ((((((int)threadIdx.y) * 256) + (((int)threadIdx.x) * 8)) + 128) % 192)));
    __syncthreads();
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[0], (&(((half*)buf_dyn_shmem)[6400])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[1], (&(((half*)buf_dyn_shmem)[7040])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[2], (&(((half*)buf_dyn_shmem)[7680])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[3], (&(((half*)buf_dyn_shmem)[8320])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[4], (&(((half*)buf_dyn_shmem)[8960])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[5], (&(((half*)buf_dyn_shmem)[9600])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[6], (&(((half*)buf_dyn_shmem)[10240])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[7], (&(((half*)buf_dyn_shmem)[10880])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[8], (&(((half*)buf_dyn_shmem)[11520])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[9], (&(((half*)buf_dyn_shmem)[12160])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[10], (&(((half*)buf_dyn_shmem)[12800])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[11], (&(((half*)buf_dyn_shmem)[13440])), 40);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[0], (&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 48)])), 200);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[1], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 48) + 16)])), 200);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[2], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 48) + 32)])), 200);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[0], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[0]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[1], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[1]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[2], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[2]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[3], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[3]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[4], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[4]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[5], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[5]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[6], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[6]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[7], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[7]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[8], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[8]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[9], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[9]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[10], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[10]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[11], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[11]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[12], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[12]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[13], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[13]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[14], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[14]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[15], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[15]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[16], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[16]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[17], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[17]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[18], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[18]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[19], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[19]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[20], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[20]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[21], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[21]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[22], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[22]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[23], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[23]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[24], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[24]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[25], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[25]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[26], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[26]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[27], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[27]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[28], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[28]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[29], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[29]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[30], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[30]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[31], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[31]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[32], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[32]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[33], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[33]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[34], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[34]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[35], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[35]);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[0], (&(((half*)buf_dyn_shmem)[6416])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[1], (&(((half*)buf_dyn_shmem)[7056])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[2], (&(((half*)buf_dyn_shmem)[7696])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[3], (&(((half*)buf_dyn_shmem)[8336])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[4], (&(((half*)buf_dyn_shmem)[8976])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[5], (&(((half*)buf_dyn_shmem)[9616])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[6], (&(((half*)buf_dyn_shmem)[10256])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[7], (&(((half*)buf_dyn_shmem)[10896])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[8], (&(((half*)buf_dyn_shmem)[11536])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[9], (&(((half*)buf_dyn_shmem)[12176])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[10], (&(((half*)buf_dyn_shmem)[12816])), 40);
    nvcuda::wmma::load_matrix_sync(A_reindex_shared_dyn_wmma_matrix_a[11], (&(((half*)buf_dyn_shmem)[13456])), 40);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[0], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 48) + 3200)])), 200);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[1], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 48) + 3216)])), 200);
    nvcuda::wmma::load_matrix_sync(B_reindex_shared_dyn_wmma_matrix_b[2], (&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 48) + 3232)])), 200);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[0], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[0]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[1], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[1]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[2], A_reindex_shared_dyn_wmma_matrix_a[0], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[2]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[3], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[3]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[4], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[4]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[5], A_reindex_shared_dyn_wmma_matrix_a[1], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[5]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[6], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[6]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[7], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[7]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[8], A_reindex_shared_dyn_wmma_matrix_a[2], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[8]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[9], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[9]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[10], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[10]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[11], A_reindex_shared_dyn_wmma_matrix_a[3], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[11]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[12], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[12]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[13], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[13]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[14], A_reindex_shared_dyn_wmma_matrix_a[4], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[14]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[15], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[15]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[16], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[16]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[17], A_reindex_shared_dyn_wmma_matrix_a[5], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[17]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[18], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[18]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[19], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[19]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[20], A_reindex_shared_dyn_wmma_matrix_a[6], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[20]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[21], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[21]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[22], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[22]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[23], A_reindex_shared_dyn_wmma_matrix_a[7], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[23]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[24], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[24]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[25], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[25]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[26], A_reindex_shared_dyn_wmma_matrix_a[8], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[26]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[27], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[27]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[28], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[28]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[29], A_reindex_shared_dyn_wmma_matrix_a[9], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[29]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[30], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[30]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[31], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[31]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[32], A_reindex_shared_dyn_wmma_matrix_a[10], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[32]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[33], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[0], Y_reindex_shared_dyn_wmma_accumulator[33]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[34], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[1], Y_reindex_shared_dyn_wmma_accumulator[34]);
    nvcuda::wmma::mma_sync(Y_reindex_shared_dyn_wmma_accumulator[35], A_reindex_shared_dyn_wmma_matrix_a[11], B_reindex_shared_dyn_wmma_matrix_b[2], Y_reindex_shared_dyn_wmma_accumulator[35]);
  }
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[0], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[1], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[2], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + ((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4))) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 32)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 64)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 96)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 128)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 160)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[3], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[4], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[5], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49152)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49184)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49216)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49248)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49280)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 49312)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[6], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[7], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[8], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98304)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98336)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98368)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98400)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98432)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 98464)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[9], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[10], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[11], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147456)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147488)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147520)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147552)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147584)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 147616)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[12], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[13], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[14], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196608)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196640)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196672)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196704)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196736)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 196768)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[15], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[16], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[17], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245760)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245792)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245824)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245856)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245888)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 245920)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[18], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[19], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[20], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 294912)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 294944)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 294976)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 295008)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 295040)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 295072)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[21], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[22], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[23], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344064)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344096)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344128)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344160)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344192)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 344224)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[24], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[25], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[26], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393216)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393248)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393280)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393312)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393344)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 393376)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[27], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[28], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[29], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442368)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442400)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442432)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442464)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442496)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 442528)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[30], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[31], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[32], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491520)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491552)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491584)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491616)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491648)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 491680)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
  __syncthreads();
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[(((int)threadIdx.y) * 768)])), Y_reindex_shared_dyn_wmma_accumulator[33], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 256)])), Y_reindex_shared_dyn_wmma_accumulator[34], 16, nvcuda::wmma::mem_row_major);
  nvcuda::wmma::store_matrix_sync((&(((half*)buf_dyn_shmem)[((((int)threadIdx.y) * 768) + 512)])), Y_reindex_shared_dyn_wmma_accumulator[35], 16, nvcuda::wmma::mem_row_major);
  __syncthreads();
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540672)) = *(uint2*)(((half*)buf_dyn_shmem) + ((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540704)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 512));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540736)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1024));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540768)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 1536));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540800)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2048));
  *(uint2*)(Y + (((((((((int)blockIdx.y) * 589824) + ((((int)threadIdx.y) & 1) * 24576)) + ((((int)threadIdx.x) >> 2) * 3072)) + (((int)blockIdx.x) * 192)) + ((((int)threadIdx.y) >> 1) * 16)) + ((((int)threadIdx.x) & 3) * 4)) + 540832)) = *(uint2*)(((half*)buf_dyn_shmem) + (((((int)threadIdx.y) * 128) + (((int)threadIdx.x) * 4)) + 2560));
}


