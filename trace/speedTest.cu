#include "hip/hip_runtime.h"
#include "simplifiedCutlassMultiStage.cu"
#include "bin/tvm_cuda_with_async.cu"

class CUDAMatrix {
private:
    float *mat, *dev_mat;
    int length;
    size_t size;

public:
    CUDAMatrix(int l) : length(l), size(sizeof(float) * l * l) {
        mat = (float *) malloc(size);
        hipMalloc((void **) &dev_mat, size);
    }

    ~CUDAMatrix() {
        free(mat);
        hipFree(dev_mat);
    }

    float *devPtr() {
        return dev_mat;
    }

    void cpyToDevice() {
        hipMemcpy(dev_mat, mat, size, hipMemcpyHostToDevice);
    }

    void cpyToHost() {
        hipMemcpy(mat, dev_mat, size, hipMemcpyDeviceToHost);
    }

    float &at(int i, int j) {
        return mat[i * length + j];
    }

    const float &at(int i, int j) const {
        return mat[i * length + j];
    }

    void print() {
        for (int i = 0; i < length; i++) {
            for (int j = 0; j < length; j++) {
                printf("%.2f ", mat[i * length + j]);
            }
            printf("\n");
        }
    }
};

void test() {
    CUDAMatrix A(1024), B(1024), C(1024);
    for (int i = 0; i < 1024; i++) {
        for (int j = 0; j < 1024; j++) {
            A.at(i, j) = j == 0 ? 1 : 0;
            B.at(i, j) = i == 0 ? 1 : 0;
            C.at(i, j) = 3;
        }
    }
    A.cpyToDevice(), B.cpyToDevice();
    dim3 grid(64, 1, 1), block(256, 1, 1);
    int smem_size = int(sizeof(SharedStorage));
    kernel<<<grid, block, smem_size, nullptr>>>(A.devPtr(), B.devPtr(), C.devPtr());
    auto err = hipGetLastError();
    if (err != hipSuccess) printf("CUDA Error: %s\n", hipGetErrorString(err));
    C.cpyToHost();
    C.print();
}

float equal(float a, float b) { return abs(a - b) < 0.001; }

// #define USE_CUTLASS_ORIGINAL

void multiTest(int round = 300, int maxCheckThreshold = 1) {
    srand(time(nullptr));
    // fprintf(stderr, "srand seed: %ld\n", time(nullptr));
    // srand(1666362747);
    static const int l = 1024, warmupGoal = round / 5;
    CUDAMatrix A(l), B(l), C(l), ans(l);
    int cur = round;
    float totGFLOPs = 0, totTime = 0;
    int verifiedCnt = 0;
    while (cur--) {
        // fprintf(stderr, "\r                                                                                                             ");
        fprintf(stderr, "\rRound %d ", round - cur);
        // fprintf(stderr, "Initializing A/B matrix...\n");
        for (int i = 0; i < l; i++) {
            for (int j = 0; j < l; j++) {
                // A.at(i, j) = i <= j ? rand() % 100 : A.at(j, i);
                // B.at(i, j) = i <= j ? rand() % 100 : B.at(j, i);
                // A.at(i, j) = 1;
                // B.at(i, j) = 1;
                // A.at(i, j) = rand() * 337 + 28;
                // B.at(i, j) = rand() * 359 + 76;
                A.at(i, j) = rand() % 100;
                B.at(i, j) = rand() % 100;
                ans.at(i, j) = 0.0f;
            }
        }
        A.cpyToDevice(), B.cpyToDevice();

        if (verifiedCnt < maxCheckThreshold) {
            // fprintf(stderr, "Calculating std...\n");
            for (int i = 0; i < l; i++)
                for (int j = 0; j < l; j++)
                    for (int k = 0; k < l; k++)
                        // i == 0 && j == 0 ? fprintf(stdout, "k = %d, ans[i][j] = %f, A.at(k, i) * B.at(k, j) = %f\n", k, ans[i][j], A.at(k, i) * B.at(k, j)) : 0, 
#ifdef USE_CUTLASS_ORIGINAL
                        ans.at(i, j) += A.at(k, i) * B.at(k, j); // A: ColumnMajor, B: RowMajor
#else
                        ans.at(i, j) += A.at(k, i) * B.at(k, j); // A: ColumnMajor, B: RowMajor
#endif
        }
        
        // fprintf(stderr, "Calculating in cuda...\n");
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

#ifdef USE_CUTLASS_ORIGINAL
        dim3 grid(64, 1, 1), block(256, 1, 1);
        int smem_size = int(sizeof(SharedStorage)); // 32768
        kernel<<<grid, block, smem_size, nullptr>>>(A.devPtr(), B.devPtr(), C.devPtr());
#else
        dim3 grid(64), block(256);
        int smem_size = int(sizeof(SharedStorage));
        // hipFuncSetAttribute(reinterpret_cast<const void*>(main_kernel0), hipFuncAttributeMaxDynamicSharedMemorySize, 81920);
        // smem_size = 81920;
        main_kernel0<<<grid, block, smem_size>>>(A.devPtr(), B.devPtr(), C.devPtr());
#endif

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float ti, GFLOPs;
        hipEventElapsedTime(&ti, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);

        auto err = hipGetLastError();
        if (err != hipSuccess) fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));

    //    if (verifiedCnt < maxCheckThreshold) sleep(1);

        C.cpyToHost();

        if (verifiedCnt < maxCheckThreshold) {
            // fprintf(stderr, "Verifying...\n");
            for (int i = 0; i < l; i++)
                for (int j = 0; j < l; j++)
                    if (!equal(ans.at(i, j), C.at(i, j)))
                        return 
                            fprintf(stderr, "Error with (%.2f - %.2f) = %.2f at (%d, %d)\n", ans.at(i, j), C.at(i, j), (ans.at(i, j) - C.at(i, j)), i, j),
                            // printf("\n\nA:\n\n"), A.print(),
                            // printf("\n\nB:\n\n"), B.print(),
                            printf("\n\nC:\n\n"), C.print(),
                            printf("\n\nans:\n\n"), ans.print(),
                            void();
            verifiedCnt++;
            fprintf(stderr, "Verifiy passed this round. ");
        }

        GFLOPs = 2149580800.0 / ti / 1e6; // totTime is in microsecond
        fprintf(stderr, "Total time this round: %.5fms, speed this round: %.2f GFLOPs, ", ti, GFLOPs);
        if (round - cur > warmupGoal) {
            totGFLOPs += GFLOPs;
            totTime += ti;
            float avgGFLOPs = totGFLOPs / (round - cur - warmupGoal), avgTime = totTime / (round - cur - warmupGoal);
            fprintf(stderr, "average time: %.5f, average speed: %.2f GFLOPs", avgTime, avgGFLOPs);
        } else fprintf(stderr, "warmuping...");
    }
}

int main() {
    // test();
    multiTest(300);
    return 0;
}