// #pragma once


#include <hip/hip_runtime.h>
#include <type_traits>
#include <cstdint>
#include <cstdio>

#define CUTLASS_HOST_DEVICE __forceinline__ __device__ __host__
#define CUTLASS_DEVICE __forceinline__ __device__
// #define CUTLASS_PRAGMA_UNROLL #pragma unroll

#define CUTLASS_ENABLE_L2_PREFETCH 1
#define CUDA_CP_ASYNC_ACTIVATED 1

struct IntPair { // pair
    int x; // continuous / row
    int y; // strided / column

    CUTLASS_HOST_DEVICE
    IntPair swap() const {
        return {y, x};
    }

    CUTLASS_HOST_DEVICE
    IntPair operator+(const IntPair &o) const {
        return {x + o.x, y + o.y};
    }

    CUTLASS_HOST_DEVICE
    IntPair &operator+=(const IntPair &o) {
        x += o.x, y += o.y;
        return *this;
    }
};

template<int X, int Y>
struct IntPairTemplate {
    static constexpr int x = X;
    static constexpr int y = Y;
};

template <typename T>
struct sizeof_bits {
    static int const value = int(sizeof(T) * 8);
};

template<typename T, int N, int Align = 16>
struct AlignedBuffer {
    using Storage = uint8_t;
    static int const kBytes = (sizeof_bits<T>::value * N + 7) / 8;
    alignas(Align) Storage storage[kBytes];

    CUTLASS_HOST_DEVICE
    T *data() {
        return reinterpret_cast<T *>(storage);
    }
};

template<typename T, int N>
class Array {
public:
    T storage[N];

    CUTLASS_HOST_DEVICE
    void fill(const T &value) {
        #pragma unroll
        for (int i = 0; i < N; ++i) {
            storage[i] = value; // static_cast<Storage>(value) ???
        }
    }

    CUTLASS_HOST_DEVICE
    void clear() {
        fill(T(0));
    }

    CUTLASS_HOST_DEVICE
    void print() const {
        for (int i = 0; i < N; i++) printf("%f ", storage[i]);
        printf("\n");
    }
};

template <typename T, int N, int Alignment = sizeof_bits<T>::value * N / 8>
class alignas(Alignment) AlignedArray: public Array<T, N> {};

using LongIndex = int64_t;
using Index = int32_t;

enum LayoutType {
    ColumnMajorType, RowMajorType
};

template<typename T, LayoutType Layout>
class TensorRef {
private:
    T *ptr_;
    LongIndex stride_; // Layout's stride

public:
    CUTLASS_HOST_DEVICE
    TensorRef() : ptr_(nullptr) {}

    CUTLASS_HOST_DEVICE
    TensorRef(T *ptr, LongIndex stride) : ptr_(ptr), stride_(stride) {}

    CUTLASS_HOST_DEVICE
    LongIndex stride() const { return stride_; }

    CUTLASS_HOST_DEVICE
    LongIndex offset(const IntPair &coord) const {
        if (Layout == RowMajorType) return LongIndex(coord.x) * stride_ + coord.y;
        else return LongIndex(coord.y) * stride_ + coord.x;
    }

    CUTLASS_HOST_DEVICE
    T *data() const { return ptr_; }

    CUTLASS_HOST_DEVICE
    T &data(LongIndex idx) {
        return ptr_[idx];
    }

    CUTLASS_HOST_DEVICE
    void add_coord_offset(const IntPair &coord) {
        ptr_ += offset(coord);
    }

    CUTLASS_HOST_DEVICE
    void reset(T *ptr, LongIndex stride) {
        ptr_ = ptr, stride_ = stride;
    }
    
    CUTLASS_HOST_DEVICE
    T &at(IntPair coord) {
        return data(offset(coord));
    }
};

template<LayoutType Layout>
class GlobalMemoryIterator {
public:
    using Pointer = float *;
    using NonConstPointer = typename std::remove_const<float>::type *;
    using BytePointer = char *;
    using Fragment = Array<float, 4>;
    using AccessType = AlignedArray<float, 1>;

private:
    uint32_t predicate_;
    BytePointer pointer_;
    // int iteration_vector_, iteration_contiguous_;
    int iteration_strided_;
    bool is_residue_tile_;
    IntPair thread_offset_, residue_offset_;

private:
    CUTLASS_DEVICE
    void compute_predicates_(IntPair extent, bool is_steady_state = false) {
        predicate_ = 0b1111u;
        return;

        predicate_ = 0u;
        #pragma unroll
        for (int s = 0; s < 4; ++s) {
            IntPair iteration_coord = {0, s * 2};
            IntPair coord = thread_offset_ + iteration_coord;
            bool guard;
            if (is_steady_state) {
                // if (AdvanceRank == 0) guard = (coord.y < extent.y);
                // else 
                guard = (coord.x < extent.x);
            } else guard = (coord.x < extent.x && coord.y < extent.y);
            predicate_ |= (unsigned(guard) << s);
        }
    }

    CUTLASS_HOST_DEVICE
    void set_predicates(int thread_id, const IntPair &threadblock_offset) {
        // IntPair residue_extent;
        // if (AdvanceRank) {
            // int residue_size = (1024 - threadblock_offset.y) % 8; // = 0
            // if (!residue_size) residue_size = 8;
            // residue_offset_ = {0, residue_size};
            // residue_extent = {1024, threadblock_offset.y + residue_size};
        // } else {
        //     int residue_size = (1024 - threadblock_offset.x) % Shape::x; // = 0
        //     if (!residue_size) residue_size = Shape::x;
        //     residue_offset_ = {residue_size, 0};
        //     residue_extent = {threadblock_offset.x + residue_size, 1024};
        // }
        residue_offset_ = {0, 8};
        thread_offset_ = threadblock_offset + IntPair({thread_id % 128, thread_id / 128});
        compute_predicates_({1024, 8}, false);
        set_iteration_index(0);
    }

    CUTLASS_HOST_DEVICE
    void add_pointer_offset(LongIndex pointer_offset) {
        pointer_ += 4 * pointer_offset;
    }

public:
    CUTLASS_HOST_DEVICE
    GlobalMemoryIterator(
        Pointer pointer,
        int thread_id,
        IntPair threadblock_offset
    ) : 
        pointer_(reinterpret_cast<BytePointer>(const_cast<NonConstPointer>(pointer))),
        is_residue_tile_(true) {
        if (Layout == RowMajorType) threadblock_offset = threadblock_offset.swap();
        set_predicates(thread_id, threadblock_offset);
        add_pointer_offset(LongIndex(thread_offset_.x) + LongIndex(thread_offset_.y) * LongIndex(1024));
    }

    CUTLASS_HOST_DEVICE
    void clear_mask(bool enable) {
        predicate_ = enable ? 0u : predicate_;
    }

    CUTLASS_HOST_DEVICE
    void set_iteration_index(int index) {
        // iteration_vector_ = 0;
        // iteration_contiguous_ = 0;
        iteration_strided_ = index;
    }

    CUTLASS_HOST_DEVICE
    bool valid() const {
        return (predicate_ & (1u << iteration_strided_));

        int pred_idx = iteration_strided_;
        if (pred_idx / 16 != 0) printf("Error!\n");
        int residual = pred_idx % 16;
        int byte_idx = residual / 4;
        int bit_idx  = residual % 4;
        return (predicate_ & (1u << (byte_idx * 8 + bit_idx))) != 0;
    }

    CUTLASS_HOST_DEVICE
    AccessType *get() const {
        // return reinterpret_cast<AccessType *>(
        //     pointer_ +
        //     iteration_contiguous_ * 4
        // ) + iteration_vector_;
        return reinterpret_cast<AccessType *>(pointer_);
    }

    CUTLASS_HOST_DEVICE
    void increment() {
        // ++iteration_vector_;
        // if (iteration_vector_ < 1) return;
        // iteration_vector_ = 0;
        // ++iteration_contiguous_;
        // if (iteration_contiguous_ < 1) return;
        // iteration_contiguous_ = 0;
        ++iteration_strided_;
        if (iteration_strided_ < 4) {
            pointer_ += 8192;
            return;
        }
        iteration_strided_ = 0;
        pointer_ += 8192;
        pointer_ -= 32768;
    }

    CUTLASS_DEVICE
    void add_tile_offset(IntPair tile_offset) {
        if (Layout == RowMajorType) tile_offset = tile_offset.swap();
        if (is_residue_tile_) {
            thread_offset_ += residue_offset_;
            compute_predicates_({1024, 1024}, true);
            add_pointer_offset(LongIndex(residue_offset_.x) + LongIndex(residue_offset_.y) * LongIndex(1024));
            // if (AdvanceRank) {
                pointer_ += LongIndex(32768) * LongIndex(tile_offset.y - 1);
                pointer_ += 128 * tile_offset.x;
            // } else {
            //     pointer_ += LongIndex(32768) * LongIndex(tile_offset.x - 1);
            //     pointer_ +=   8 * tile_offset.y;
            // }
        } else {
            // if (AdvanceRank) {
                pointer_ += LongIndex(32768) * LongIndex(tile_offset.y);
                pointer_ += 128 * tile_offset.x;
            // } else {
            //     pointer_ += LongIndex(32768) * LongIndex(tile_offset.x);
            //     pointer_ +=   8 * tile_offset.y;
            // }
        }
        is_residue_tile_ = false;
    }
};

template<LayoutType Layout>
class SharedMemoryStoreIterator {
private:
    using AccessType = Array<float, 1>;

    AccessType *pointer_;
    LongIndex stride_;

    int iteration_contiguous_, iteration_strided_;
    Index byte_offset_;

public:
    CUTLASS_HOST_DEVICE
    SharedMemoryStoreIterator(TensorRef<float, Layout> ref, int thread_id) : stride_(ref.stride()), byte_offset_(0) {
        pointer_ = reinterpret_cast<AccessType *>(ref.data() + thread_id);
        set_iteration_index(0);
    }

    CUTLASS_HOST_DEVICE
    void set_iteration_index(int index) {
        iteration_contiguous_ = 0;
        iteration_strided_ = index;
    }

    CUTLASS_DEVICE
    AccessType *get() const {
        int access_offset = iteration_strided_ * 2 * stride_ + iteration_contiguous_;
        char * access_byte_ptr = reinterpret_cast<char *>(pointer_ + access_offset);
        return reinterpret_cast<AccessType *>(access_byte_ptr + byte_offset_);
    }

    CUTLASS_HOST_DEVICE
    void increment() {
        // ++iteration_contiguous_;
        // if (iteration_contiguous_ < 1) return;
        // iteration_contiguous_ = 0;
        ++iteration_strided_;
        if (iteration_strided_ < 4) return;
        iteration_strided_ = 0;
        return;
    }

    CUTLASS_DEVICE
    void add_tile_offset(IntPair coord) {
        if (Layout == RowMajorType) coord = coord.swap();
        byte_offset_ += sizeof(float) * (coord.x * 128 + coord.y * 8 * stride_);
    }
};

// TODO __nv_cvta_generic_to_shared_impl not found
// currently just use the google version
inline __device__ unsigned cutlass_get_smem_pointer(void *ptr) {
    return static_cast<unsigned>(__cvta_generic_to_shared(ptr));
}

inline __device__ unsigned cutlass_get_smem_pointer(void const *ptr) {
    return cutlass_get_smem_pointer(const_cast<void *>(ptr));
}

CUTLASS_DEVICE
void shared_load(Array<float, 4> &D, void const *ptr) {
    unsigned addr = cutlass_get_smem_pointer(ptr);
    uint4 v;
    asm volatile ("ld.shared.v4.b32 {%0, %1, %2, %3}, [%4];" : 
    "=r"(v.x), "=r"(v.y), "=r"(v.z), "=r"(v.w) : "r"(addr));
    D = reinterpret_cast<Array<float, 4> const &>(v);
}

template<typename Shape, LayoutType Layout>
class SharedMemoryLoadIterator {
public:
    using Fragment = Array<float, 8>;

private:
    TensorRef<Array<float, 4>, Layout> ref_;

    CUTLASS_DEVICE
    IntPair layout_inverse(int offset) {
        int row_major = offset / 16;
        int residual = offset % 16;
        int column = residual / 2;
        int row_minor = residual % 2;
        return {row_major * 2 + row_minor, column};
    }

public:
    CUTLASS_DEVICE
    SharedMemoryLoadIterator(TensorRef<float, Layout> ref, int lane_id) {
        IntPair lane_offset;
        if (Layout == ColumnMajorType) lane_offset = {layout_inverse(lane_id).x * 4, 0};
        else lane_offset = {0, layout_inverse(lane_id).y * 4};
        ref.add_coord_offset(lane_offset);
        ref_.reset(reinterpret_cast<Array<float, 4> *>(ref.data()), ref.stride() / 4);
    }

    CUTLASS_DEVICE
    void add_tile_offset(IntPair coord) {
        if (Layout == RowMajorType) ref_.add_coord_offset({coord.x * Shape::x, coord.y * Shape::y / 4});
        else ref_.add_coord_offset({coord.x * Shape::x / 4, coord.y * Shape::y});
    }

    CUTLASS_DEVICE
    void set_kgroup_index(int k_group) {
        // no operation here
        return;
    }

    CUTLASS_DEVICE
    void load(Fragment &frag) const {
        Array<float, 4> *dst_ptr = reinterpret_cast<Array<float, 4> *>(&frag);
        for (int mn = 0; mn < 2; ++mn) {
            void const * ptr;
            if (Layout == RowMajorType) ptr = ref_.data() + ref_.offset({0, mn * 8});
            else ptr = ref_.data() + ref_.offset({mn * 4, 0});
            shared_load(dst_ptr[mn], ptr);
        }
    }

    CUTLASS_DEVICE
    void increment() {
        if (Layout == RowMajorType) ref_.add_coord_offset({Shape::x, 0});
        else ref_.add_coord_offset({0, Shape::y});
    }
};

using GlobalMemoryIteratorA = GlobalMemoryIterator<ColumnMajorType>;
using GlobalMemoryIteratorB = GlobalMemoryIterator<RowMajorType>;
using SharedMemoryStoreIteratorA = SharedMemoryStoreIterator<ColumnMajorType>;
using SharedMemoryStoreIteratorB = SharedMemoryStoreIterator<RowMajorType>;
using SharedMemoryLoadIteratorA = SharedMemoryLoadIterator<IntPairTemplate<32, 1>, ColumnMajorType>;
using SharedMemoryLoadIteratorB = SharedMemoryLoadIterator<IntPairTemplate<1, 64>, RowMajorType>;

template<int SizeInBytes = 4>
CUTLASS_DEVICE
void cp_async(void *smem_ptr, void const *global_ptr, bool pred_guard = true) {
#if CUDA_CP_ASYNC_ACTIVATED
    unsigned smem_int_ptr = cutlass_get_smem_pointer(smem_ptr);
    asm volatile(
        "{\n"
        "  .reg .pred p;\n"
        "  setp.ne.b32 p, %0, 0;\n"
#if CUTLASS_ENABLE_L2_PREFETCH
        "  @p cp.async.ca.shared.global.L2::128B [%1], [%2], %3;\n"
#else
        "  @p cp.async.ca.shared.global [%1], [%2], %3;\n"
#endif
        "}\n" ::"r"((int)pred_guard),
        "r"(smem_int_ptr), "l"(global_ptr), "n"(SizeInBytes)
    );
#else
    using AccessType = Array<uint8_t, SizeInBytes>;
    if (pred_guard) *static_cast<AccessType *>(smem_ptr) = *static_cast<AccessType const *>(global_ptr);
#endif
}

template<int SizeInBytes = 4>
CUTLASS_DEVICE
void cp_async_zfill(void *smem_ptr, void const *global_ptr, bool pred_guard = true) {
#if CUDA_CP_ASYNC_ACTIVATED
    unsigned smem_int_ptr = cutlass_get_smem_pointer(smem_ptr);
    int src_in_bytes = (pred_guard ? SizeInBytes : 0);
    asm volatile(
#if CUTLASS_ENABLE_L2_PREFETCH
        "cp.async.ca.shared.global.L2::128B [%0], [%1], %2, %3;\n" ::"r"(smem_int_ptr),
#else
        "cp.async.ca.shared.global [%0], [%1], %2, %3;\n" ::"r"(smem_int_ptr),
#endif
        "l"(global_ptr), "n"(SizeInBytes), "r"(src_in_bytes)
    );
#else
    using AccessType = Array<uint8_t, SizeInBytes>;
    if (pred_guard) *static_cast<AccessType *>(smem_ptr) = *static_cast<AccessType const *>(global_ptr);
    else {
        AccessType zeros;
        zeros.clear();
        *static_cast<AccessType *>(smem_ptr) = zeros;
    }
#endif
}

CUTLASS_DEVICE
void cp_async_fence() {
#if CUDA_CP_ASYNC_ACTIVATED
    asm volatile("cp.async.commit_group;\n" ::);
#endif
}

template<int N>
CUTLASS_DEVICE
void cp_async_wait() {
#if CUDA_CP_ASYNC_ACTIVATED
    asm volatile("cp.async.wait_group %0;\n" ::"n"(N));
#endif
}

class Mma {
private:
    // iterator responsible for store to smem
    SharedMemoryStoreIteratorA smem_iterator_A_;
    SharedMemoryStoreIteratorB smem_iterator_B_;
    // iterator responsible for load from smem
    SharedMemoryLoadIteratorA warp_tile_iterator_A_;
    SharedMemoryLoadIteratorB warp_tile_iterator_B_;

    CUTLASS_HOST_DEVICE
    void warp_mma(Array<float, 64> &D, const Array<float, 8> &A, const Array<float, 8> &B, const Array<float, 64> &C) {
        TensorRef<float const, ColumnMajorType> a_ref(reinterpret_cast<float const *>(&A), 8);
        TensorRef<float const, RowMajorType> b_ref(reinterpret_cast<float const *>(&B), 8);
        TensorRef<float, RowMajorType> d_ref(reinterpret_cast<float *>(&D), 8);
        D = C;
        int k = 0;
        #pragma unroll
        for (int n = 0; n < 8; n += 2) {
            #pragma unroll
            for (int m = 0; m < 8; m += 2) {
                int m_serpentine = (n % 4) ? (6 - m) : m;
                {
                    IntPair mn = {m_serpentine, n};
                    IntPair mk = {m_serpentine, k};
                    IntPair kn = {k, n};
                    d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
                }
                {
                    IntPair mn = {m_serpentine + 1, n};
                    IntPair mk = {m_serpentine + 1, k};
                    IntPair kn = {k, n};
                    d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
                }
                {
                    IntPair mn = {m_serpentine + 1, n + 1};
                    IntPair mk = {m_serpentine + 1, k};
                    IntPair kn = {k, n + 1};
                    d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
                }
                {
                    IntPair mn = {m_serpentine, n + 1};
                    IntPair mk = {m_serpentine, k};
                    IntPair kn = {k, n + 1};
                    d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
                }
            }
        }
    }

public:
    class SharedStorage {
    private:
        AlignedBuffer<float, 4096> operand_A;
        AlignedBuffer<float, 4096> operand_B;

    public:
        CUTLASS_HOST_DEVICE
        TensorRef<float, ColumnMajorType> operand_A_ref() {
            return TensorRef<float, ColumnMajorType>(operand_A.data(), 128); // ShapeA::kRow
        }

        CUTLASS_HOST_DEVICE
        TensorRef<float, RowMajorType>    operand_B_ref() {
            return TensorRef<float, RowMajorType>   (operand_B.data(), 128); // ShapeB::kColumn
        }
    };

    CUTLASS_DEVICE
    Mma(SharedStorage &shared_storage, int thread_idx, int warp_idx, int lane_idx) :
        smem_iterator_A_(shared_storage.operand_A_ref(), thread_idx),
        smem_iterator_B_(shared_storage.operand_B_ref(), thread_idx),
        warp_tile_iterator_A_(shared_storage.operand_A_ref(), lane_idx),
        warp_tile_iterator_B_(shared_storage.operand_B_ref(), lane_idx) {
        int warp_idx_mn = warp_idx % 8;
        // int warp_idx_k = warp_idx / 8; // == 0
        int warp_idx_m = warp_idx_mn % 4;
        int warp_idx_n = warp_idx_mn / 4;
        warp_tile_iterator_A_.add_tile_offset({warp_idx_m, 0});
        warp_tile_iterator_B_.add_tile_offset({0, warp_idx_n});
    }

    CUTLASS_DEVICE
    void copy_tiles_and_advance(GlobalMemoryIteratorA &iterator_A, GlobalMemoryIteratorB &iterator_B, int group_start_A = 0, int group_start_B = 0) {
        iterator_A.set_iteration_index(group_start_A);
        smem_iterator_A_.set_iteration_index(group_start_A);
        if (group_start_A < 4) {
            typename GlobalMemoryIteratorA::AccessType *dst_ptr = reinterpret_cast<typename GlobalMemoryIteratorA::AccessType *>(smem_iterator_A_.get());
            auto gmem_ptr = iterator_A.get();
            cp_async(dst_ptr, gmem_ptr, iterator_A.valid());
            iterator_A.increment();
            smem_iterator_A_.increment();
        }

        iterator_B.set_iteration_index(group_start_B);
        smem_iterator_B_.set_iteration_index(group_start_B);
        if (group_start_B < 4) {
            typename GlobalMemoryIteratorB::AccessType *dst_ptr = reinterpret_cast<typename GlobalMemoryIteratorB::AccessType *>(smem_iterator_B_.get());
            auto gmem_ptr = iterator_B.get();
            cp_async(dst_ptr, gmem_ptr, iterator_B.valid());
            iterator_B.increment();
            smem_iterator_B_.increment();
        }
    }

    CUTLASS_DEVICE
    void operator()(
        int gemm_k_iterations, Array<float, 64> &accum,
        GlobalMemoryIteratorA iterator_A, GlobalMemoryIteratorB iterator_B, const Array<float, 64> &src_accum
    ) {
        #pragma unroll
        for (int stage = 0; stage < 3; ++stage, --gemm_k_iterations) {
            // iterator_A.clear_mask(gemm_k_iterations == 0);
            // iterator_B.clear_mask(gemm_k_iterations == 0);

            iterator_A.set_iteration_index(0);
            smem_iterator_A_.set_iteration_index(0);
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                typename GlobalMemoryIteratorA::AccessType *dst_ptr = reinterpret_cast<typename GlobalMemoryIteratorA::AccessType *>(smem_iterator_A_.get());
                int src_bytes = (iterator_A.valid() ? 4 : 0);
                cp_async_zfill<4>(dst_ptr, iterator_A.get(), iterator_A.valid());
                iterator_A.increment();
                smem_iterator_A_.increment();
            }

            iterator_B.set_iteration_index(0);
            smem_iterator_B_.set_iteration_index(0);
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                typename GlobalMemoryIteratorB::AccessType *dst_ptr = reinterpret_cast<typename GlobalMemoryIteratorB::AccessType *>(smem_iterator_B_.get());
                int src_bytes = (iterator_B.valid() ? 4 : 0);
                cp_async_zfill<4>(dst_ptr, iterator_B.get(), iterator_B.valid());
                iterator_B.increment();
                smem_iterator_B_.increment();
            }

            iterator_A.add_tile_offset({0, 1});
            iterator_B.add_tile_offset({1, 0});
            smem_iterator_A_.add_tile_offset({0, 1});
            smem_iterator_B_.add_tile_offset({1, 0});

            cp_async_fence();
        }

        accum = src_accum;

        cp_async_wait<2>();
        __syncthreads();

        Array<float, 8> warp_loaded_frag_A[2];
        Array<float, 8> warp_loaded_frag_B[2];
        // Array<float, 8> warp_transformed_frag_A[2];
        // Array<float, 8> warp_transformed_frag_B[2];

        warp_tile_iterator_A_.load(warp_loaded_frag_A[0]);
        warp_tile_iterator_B_.load(warp_loaded_frag_B[0]);
        warp_tile_iterator_A_.increment();
        warp_tile_iterator_B_.increment();
        // iterator_A.clear_mask(gemm_k_iterations == 0);
        // iterator_B.clear_mask(gemm_k_iterations == 0);
        int smem_write_stage_idx = 3;
        int smem_read_stage_idx = 0;
        // warp_transformed_frag_A[0] = warp_loaded_frag_A[0];
        // warp_transformed_frag_B[0] = warp_loaded_frag_B[0];

        #pragma unroll 1
        for(; gemm_k_iterations > -3; ) {
            #pragma unroll
            for (int warp_mma_k = 0; warp_mma_k < 8; ++warp_mma_k) {
                warp_tile_iterator_A_.load(warp_loaded_frag_A[(warp_mma_k + 1) % 2]);
                warp_tile_iterator_B_.load(warp_loaded_frag_B[(warp_mma_k + 1) % 2]);
                warp_tile_iterator_A_.increment();
                warp_tile_iterator_B_.increment();

                warp_mma(accum, warp_loaded_frag_A[warp_mma_k % 2], warp_loaded_frag_B[warp_mma_k % 2], accum);

                if (warp_mma_k < 7) {
                    int group_start_iteration_A = warp_mma_k;
                    int group_start_iteration_B = warp_mma_k;
                    copy_tiles_and_advance(iterator_A, iterator_B, group_start_iteration_A, group_start_iteration_B);
                }
                if (warp_mma_k + 2 == 8) {
                    int group_start_iteration_A = warp_mma_k + 1;
                    int group_start_iteration_B = warp_mma_k + 1;
                    copy_tiles_and_advance(iterator_A, iterator_B, group_start_iteration_A, group_start_iteration_B);

                    cp_async_fence();

                    cp_async_wait<2>();
                    __syncthreads();

                    iterator_A.add_tile_offset({0, 1});
                    iterator_B.add_tile_offset({1, 0});
                    smem_iterator_A_.add_tile_offset({0, 1});
                    smem_iterator_B_.add_tile_offset({1, 0});

                    if (smem_write_stage_idx == 3) {
                        smem_iterator_A_.add_tile_offset({0, -4});
                        smem_iterator_B_.add_tile_offset({-4, 0});
                        smem_write_stage_idx = 0;
                    } else ++smem_write_stage_idx;
                    if (smem_read_stage_idx == 3) {
                        warp_tile_iterator_A_.add_tile_offset({0, -32});
                        warp_tile_iterator_B_.add_tile_offset({-32, 0});
                        smem_read_stage_idx = 0;
                    } else ++smem_read_stage_idx;

                    --gemm_k_iterations;
                    // !! this is not useless
                    iterator_A.clear_mask(gemm_k_iterations == 0);
                    iterator_B.clear_mask(gemm_k_iterations == 0);
                }
            }
        }
    }
};

class FragmentIterator {
private:
    using AccessType = Array<float, 4>;

    AccessType const *accumulators_;
    int index_;

public:
    CUTLASS_HOST_DEVICE
    FragmentIterator(const Array<float, 64> &accum) : accumulators_(reinterpret_cast<AccessType const *>(&accum)), index_(0) {}

    CUTLASS_HOST_DEVICE
    void increment() {
        ++index_;
    }

    CUTLASS_HOST_DEVICE
    void load(Array<float, 8> &frag) {
        AccessType *frag_ptr = reinterpret_cast<AccessType *>(&frag);
        #pragma unroll
        for (int n = 0; n < 2; ++n) {
            frag_ptr[n] = accumulators_[index_ * 2 + n];
        }
    }
};

class EpilogueSharedMemoryStoreIterator {
private:
    using AccessType = AlignedArray<float, 1>;

    AccessType *pointer_;
    LongIndex layout_stride_;

    CUTLASS_DEVICE
    LongIndex layout_get(const IntPair &coord) {
        return LongIndex(coord.x) * LongIndex(layout_stride_) + LongIndex(coord.y);
    }

public:
    CUTLASS_DEVICE
    EpilogueSharedMemoryStoreIterator(const TensorRef<float, RowMajorType> &ref, int lane_id) :
        pointer_(reinterpret_cast<AccessType *>(ref.data())), layout_stride_(ref.stride()) {
        // RowMajorInterleaved<2>
        int residual = lane_id % 16;
        IntPair lane_offset = {(lane_id / 16) * 2 + residual % 2, residual / 2};
        pointer_ += layout_get({lane_offset.x, lane_offset.y * 4});
    }

    CUTLASS_DEVICE
    void store(const Array<float, 8> &frag) {
        AccessType const *scalar_frag_ptr = reinterpret_cast<AccessType const *>(&frag);
        #pragma unroll
        for (int n = 0; n < 2; ++n) {
            #pragma unroll
            for (int s = 0; s < 4; ++s) {
                pointer_[n * 32 + s] = scalar_frag_ptr[n * 4 + s];
            }
        }
    }

    CUTLASS_DEVICE
    void add_tile_offset(const IntPair tile_offset) {
        pointer_ += layout_get({tile_offset.x * 4, tile_offset.y * 64});
    }
};

class EpilogueSharedMemoryLoadIterator {
private:
    using LoadType = AlignedArray<float, 1, 4>;

    uint8_t *byte_pointer_;
    int stride_;

public:
    CUTLASS_DEVICE
    EpilogueSharedMemoryLoadIterator(const TensorRef<float, RowMajorType> &ref, int thread_idx, int warp_idx, int lane_idx) :
        byte_pointer_(reinterpret_cast<uint8_t *>(ref.data())), stride_(ref.stride() * 4) {
        IntPair thread_offset = {(warp_idx / 2) * 4 + (warp_idx % 2), lane_idx % 32};
        byte_pointer_ += thread_offset.x * stride_ + thread_offset.y * 4;
    }

    CUTLASS_DEVICE
    void load(Array<float, 8> &frag) {
        #pragma unroll
        for (int group = 0; group < 2; ++group) {
            uint8_t const *byte_pointer = byte_pointer_ + group * 2 * stride_;
            LoadType *frag_ptr = reinterpret_cast<LoadType *>(&frag);
            LoadType const *memory_pointer = reinterpret_cast<LoadType const *>(byte_pointer);
            #pragma unroll
            for (int column = 0; column < 4; ++column) {
                frag_ptr[group * 4 + column] = memory_pointer[column * 32];
            }
        }
    }
};

CUTLASS_DEVICE
void global_store(const AlignedArray<float, 1> &D, void *ptr, bool pred_guard) {
    uint32_t const &data = reinterpret_cast<uint32_t const &>(D);
    asm volatile(
        "{\n"
        "  .reg .pred p;\n"
        "  setp.ne.b32 p, %2, 0;\n"
        "  @p st.global.u32 [%0], %1;\n"
        "}\n"
        :
        : "l"(ptr), "r"(data), "r"((int)pred_guard)
    );
}

class OutputTileIterator {
private:
    using AccessType = AlignedArray<float, 1>;

    bool predicates_[4];
    uint8_t *byte_pointer_;
    int state_[3];
    Index thread_start_row_, thread_start_column_;

public:
    CUTLASS_DEVICE
    OutputTileIterator(float *pointer, int thread_idx, int warp_idx, int lane_idx, IntPair threadblock_offset) {
        IntPair thread_offset = IntPair({(warp_idx / 2) * 32 + (warp_idx % 2) * 4, lane_idx % 32}) + threadblock_offset;
        thread_start_row_ = thread_offset.x;
        thread_start_column_ = thread_offset.y;
        #pragma unroll
        for (int c = 0; c < 4; ++c) {
            predicates_[c] = ((thread_offset.y + c * 32) < 1024);
        }
        byte_pointer_ = reinterpret_cast<uint8_t *>(pointer) + LongIndex(thread_offset.x) * LongIndex(4096) + LongIndex(thread_offset.y) * sizeof(AccessType);
        state_[0] = state_[1] = state_[2] = 0;
    }

    CUTLASS_DEVICE
    void store(const Array<float, 8> &frag) {
        uint8_t *byte_pointer = byte_pointer_;
        AccessType const *frag_ptr = reinterpret_cast<AccessType const *>(&frag);
        #pragma unroll
        for (int group = 0; group < 2; ++group) {
            int frag_row_idx = group;
            int row_offset = group * 2;
            bool row_guard = ((row_offset + thread_start_row_) < 1024);
            AccessType *memory_pointer = reinterpret_cast<AccessType *>(byte_pointer);
            #pragma unroll
            for (int column = 0; column < 4; ++column) {
                bool guard = row_guard && predicates_[column];
                global_store(frag_ptr[frag_row_idx * 4 + column], (void *)&memory_pointer[0], guard);
                memory_pointer += 32;
            }
            if (group + 1 < 2) byte_pointer += 32768;
        }
    }

    CUTLASS_DEVICE
    void increment() {
        ++state_[0];
        byte_pointer_ += 4096; // advance row
        thread_start_row_ += 1;
        if (state_[0] == 4) {
            state_[0] = 0;
            ++state_[1];
            byte_pointer_ += 49152; // advance group
            thread_start_row_ += 12;
            if (state_[1] == 2) {
                state_[1] = 0;
                ++state_[2];
                byte_pointer_ += 131072; // advance cluster
                thread_start_row_ += 32;
                if (state_[2] == 1) {
                    state_[2] = 0;
                    byte_pointer_ += 65536; // advance tile
                }
            }
        }
    }
};

class Epilogue {
public:
    class SharedStorage {
    private:
        AlignedBuffer<float, 2320> storage; // <16, 145>

    public:
        CUTLASS_DEVICE
        TensorRef<float, RowMajorType> reference() {
            return TensorRef<float, RowMajorType>(storage.data(), 145);
        }
    };

private:
    SharedStorage &shared_storage_;
    EpilogueSharedMemoryStoreIterator warp_tile_iterator_;
    EpilogueSharedMemoryLoadIterator shared_load_iterator_;

    CUTLASS_DEVICE
    static void acc2smem_source_not_needed(int Advance, FragmentIterator accum_fragment_iterator, EpilogueSharedMemoryStoreIterator &warp_tile_iterator) {
        #pragma unroll
        for (int i = 0; i < Advance; ++i) {
            accum_fragment_iterator.increment();
        }
        Array<float, 8> accum_fragment;
        accum_fragment_iterator.load(accum_fragment);
        warp_tile_iterator.store(accum_fragment);
    }

public:
    CUTLASS_DEVICE
    Epilogue(SharedStorage &shared_storage, int thread_idx, int warp_idx, int lane_idx) : 
        shared_storage_(shared_storage),
        warp_tile_iterator_(shared_storage.reference(), lane_idx),
        shared_load_iterator_(shared_storage.reference(), thread_idx, warp_idx, lane_idx) {
        int warp_k = warp_idx / 8;
        int warp_mn = warp_idx % 8;
        int warp_m = warp_mn % 4;
        int warp_n = warp_mn / 4;
        IntPair warp_offset = {warp_k * 4 + warp_m, warp_n};
        warp_tile_iterator_.add_tile_offset(warp_offset);
    }

    CUTLASS_DEVICE
    void operator()(OutputTileIterator destination_iterator, const Array<float, 64> accumulators) {
        FragmentIterator accum_fragment_iterator(accumulators);
        #pragma unroll(8)
        for (int iter = 0; iter < 8; ++iter) {
            __syncthreads();
            acc2smem_source_not_needed(iter, accum_fragment_iterator, warp_tile_iterator_);
            __syncthreads();
            Array<float, 8> aligned_accum_fragment;
            shared_load_iterator_.load(aligned_accum_fragment);
            // for(int i=0;i<8;i++)if(aligned_accum_fragment.storage[i]!=1024.0)printf("%.2f ", aligned_accum_fragment.storage[i]);

            // Array<float, 8> output_fragment;
            // output_fragment = aligned_accum_fragment; // linear combination for alpha = 1, beta = 0
            // destination_iterator.store(output_fragment);
            destination_iterator.store(aligned_accum_fragment);
            destination_iterator.increment();
        }
    }
};

class GemmKernel {
public:
    union SharedStorage {
        typename Mma::SharedStorage main_loop;
        typename Epilogue::SharedStorage epilogue;
    };
};

__global__ 
void kernel(float * A, float * B, float * C) { // C = A * B
    extern __shared__ int SharedStorageBase[];
    typename GemmKernel::SharedStorage *shared_storage_ptr = reinterpret_cast<typename GemmKernel::SharedStorage *>(SharedStorageBase);
    typename GemmKernel::SharedStorage &shared_storage = *shared_storage_ptr;

    IntPair threadblock_tile_offset = {int(blockIdx.x / 8), int(blockIdx.x % 8)};
    IntPair threadblock_offset = {threadblock_tile_offset.x * 128, threadblock_tile_offset.y * 128};
    int thread_idx = threadIdx.x;
    IntPair tb_offset_A = {threadblock_offset.x, 0};
    IntPair tb_offset_B = {0, threadblock_offset.y};
    GlobalMemoryIteratorA iterator_A(A, thread_idx, tb_offset_A);
    GlobalMemoryIteratorB iterator_B(B, thread_idx, tb_offset_B);
// if(blockIdx.x == 0) printf("%d %d %d\n", int(blockIdx.x), int(threadIdx.x), int(reinterpret_cast<float*>(iterator_A.get()) - A));
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;
    Mma mma(shared_storage.main_loop, thread_idx, warp_idx, lane_idx);
    Array<float, 64> accumulators;
    accumulators.clear();
    int gemm_k_iterations = 128;
    mma(gemm_k_iterations, accumulators, iterator_A, iterator_B, accumulators);
    // accumulators.print();

    OutputTileIterator iterator_C(C, thread_idx, warp_idx, lane_idx, threadblock_offset);
    Epilogue epilogue(shared_storage.epilogue, thread_idx, warp_idx, lane_idx);
    epilogue(iterator_C, accumulators);
}

void run_cutlassMultiStage(int M, int N, int K, float * A, float * B, float * C, float alpha, float beta) {
    dim3 grid(64, 1, 1), block(256, 1, 1);
    int smem_size = int(sizeof(typename GemmKernel::SharedStorage));
    kernel<<<grid, block, smem_size, nullptr>>>(A, B, C);
}

using SharedStorage = typename GemmKernel::SharedStorage;

// class CUDAMatrix {
// private:
//     float *mat, *dev_mat;
//     int length;
//     size_t size;

// public:
//     CUDAMatrix(int l) : length(l), size(sizeof(float) * l * l) {
//         mat = (float *) malloc(size);
//         cudaMalloc((void **) &dev_mat, size);
//     }

//     ~CUDAMatrix() {
//         free(mat);
//         cudaFree(dev_mat);
//     }

//     float *devPtr() {
//         return dev_mat;
//     }

//     void cpyToDevice() {
//         cudaMemcpy(dev_mat, mat, size, cudaMemcpyHostToDevice);
//     }

//     void cpyToHost() {
//         cudaMemcpy(mat, dev_mat, size, cudaMemcpyDeviceToHost);
//     }

//     float &at(int i, int j) {
//         return mat[i * length + j];
//     }

//     const float &at(int i, int j) const {
//         return mat[i * length + j];
//     }

//     void print() {
//         for (int i = 0; i < length; i++) {
//             for (int j = 0; j < length; j++) {
//                 printf("%.2f ", mat[i * length + j]);
//             }
//             printf("\n");
//         }
//     }
// };

// void test() {
//     CUDAMatrix A(1024), B(1024), C(1024);
//     for (int i = 0; i < 1024; i++) {
//         for (int j = 0; j < 1024; j++) {
//             A.at(i, j) = j == 0 ? 1 : 0;
//             B.at(i, j) = i == 0 ? 1 : 0;
//             C.at(i, j) = 3;
//         }
//     }
//     A.cpyToDevice(), B.cpyToDevice();
//     dim3 grid(64, 1, 1), block(256, 1, 1);
//     int smem_size = int(sizeof(typename GemmKernel::SharedStorage));
//     kernel<<<grid, block, smem_size, nullptr>>>(A.devPtr(), B.devPtr(), C.devPtr());
//     auto err = cudaGetLastError();
//     if (err != cudaSuccess) printf("CUDA Error: %s\n", cudaGetErrorString(err));
//     C.cpyToHost();
//     C.print();
// }

// float equal(float a, float b) { return abs(a - b) < 0.001; }

// void correctnessTest(int round = 100) {
//     static const int l = 1024;
//     CUDAMatrix A(l), B(l), C(l), ans(l);
//     int cur = round;
//     while (cur--) {
//         fprintf(stderr, "Round %d\n", round - cur);
//         for (int i = 0; i < l; i++) {
//             for (int j = 0; j < l; j++) {
//                 // A.at(i, j) = i <= j ? rand() : A.at(j, i);
//                 // B.at(i, j) = i <= j ? rand() : B.at(j, i);
//                 A.at(i, j) = rand();
//                 B.at(i, j) = rand();
//                 ans.at(i, j) = 0;
//             }
//         }
//         A.cpyToDevice(), B.cpyToDevice();
//         dim3 grid(64, 1, 1), block(256, 1, 1);
//         int smem_size = int(sizeof(typename GemmKernel::SharedStorage));
//         kernel<<<grid, block, smem_size, nullptr>>>(A.devPtr(), B.devPtr(), C.devPtr());
//         auto err = cudaGetLastError();
//         if (err != cudaSuccess) printf("CUDA Error: %s\n", cudaGetErrorString(err));
//         C.cpyToHost();
//         for (int i = 0; i < l; i++)
//             for (int j = 0; j < l; j++)
//                 for (int k = 0; k < l; k++)
//                     ans.at(i, j) += A.at(k, i) * B.at(k, j); // A: ColumnMajor, B: RowMajor
//         for (int i = 0; i < l; i++)
//             for (int j = 0; j < l; j++)
//                 if (!equal(ans.at(i, j), C.at(i, j)))
//                     return fprintf(stderr, "Error with %.2f at (%d, %d)\n", abs(ans.at(i, j) - C.at(i, j)), i, j),
//                         printf("\n\nA:\n\n"), A.print(),
//                         printf("\n\nB:\n\n"), B.print(),
//                         printf("\n\nC:\n\n"), C.print(),
//                         printf("\n\nans:\n\n"), ans.print(),
//                         void();
//     }
// }

// int main() {
//     // test();
//     correctnessTest();
//     return 0;
// }
