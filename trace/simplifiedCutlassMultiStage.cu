// #pragma once


#include <hip/hip_runtime.h>
#include <type_traits>
#include <cstdint>
#include <cstdio>
#include <unistd.h>

#define CUTLASS_HOST_DEVICE __forceinline__ __device__ __host__
#define CUTLASS_DEVICE __forceinline__ __device__

#define CUTLASS_ENABLE_L2_PREFETCH 1
#define CUDA_CP_ASYNC_ACTIVATED 1

struct IntPair { // pair
    int x; // continuous / row
    int y; // strided / column

    CUTLASS_HOST_DEVICE
    IntPair swap() const {
        return {y, x};
    }

    CUTLASS_HOST_DEVICE
    IntPair operator+(const IntPair &o) const {
        return {x + o.x, y + o.y};
    }

    CUTLASS_HOST_DEVICE
    IntPair &operator+=(const IntPair &o) {
        x += o.x, y += o.y;
        return *this;
    }
};

template <typename T>
struct sizeof_bits {
    static int const value = int(sizeof(T) * 8);
};

template<typename T, int N, int Align = 16>
struct AlignedBuffer {
    using Storage = uint8_t;
    static int const kBytes = (sizeof_bits<T>::value * N + 7) / 8;
    alignas(Align) Storage storage[kBytes];

    CUTLASS_HOST_DEVICE
    T *data() {
        return reinterpret_cast<T *>(storage);
    }
};

template<typename T, int N>
class Array {
public:
    T storage[N];

    CUTLASS_HOST_DEVICE
    void fill(const T &value) {
        #pragma unroll
        for (int i = 0; i < N; ++i) {
            storage[i] = value; // static_cast<Storage>(value) ???
        }
    }

    CUTLASS_HOST_DEVICE
    void clear() {
        fill(T(0));
    }

    CUTLASS_HOST_DEVICE
    void print() const {
        for (int i = 0; i < N; i++) printf("%f ", storage[i]);
        printf("\n");
    }
};

template <typename T, int N, int Alignment = sizeof_bits<T>::value * N / 8>
class alignas(Alignment) AlignedArray: public Array<T, N> {};

using LongIndex = int64_t;
using Index = int32_t;

enum LayoutType {
    ColumnMajorType, RowMajorType
};

template<typename T, LayoutType Layout>
class TensorRef {
private:
    T *ptr_;
    LongIndex stride_;

public:
    CUTLASS_HOST_DEVICE
    TensorRef(T *ptr, LongIndex stride) : ptr_(ptr), stride_(stride) {}

    CUTLASS_HOST_DEVICE
    LongIndex offset(const IntPair &coord) const {
        if (Layout == RowMajorType) return LongIndex(coord.x) * stride_ + coord.y;
        else return coord.x + LongIndex(coord.y) * stride_;
    }
    
    CUTLASS_HOST_DEVICE
    T &at(IntPair coord) {
        return ptr_[offset(coord)];
    }
};

inline __device__ unsigned cutlass_get_smem_pointer(void *ptr) {
    return static_cast<unsigned>(__cvta_generic_to_shared(ptr));
}

inline __device__ unsigned cutlass_get_smem_pointer(void const *ptr) {
    return cutlass_get_smem_pointer(const_cast<void *>(ptr));
}

CUTLASS_DEVICE
void shared_load(Array<float, 4> &D, void const *ptr) {
    unsigned addr = cutlass_get_smem_pointer(ptr);
    uint4 v;
    asm volatile ("ld.shared.v4.b32 {%0, %1, %2, %3}, [%4];" : 
    "=r"(v.x), "=r"(v.y), "=r"(v.z), "=r"(v.w) : "r"(addr));
    D = reinterpret_cast<Array<float, 4> const &>(v);
}

template<int SizeInBytes = 4>
CUTLASS_DEVICE
void cp_async(void *smem_ptr, void const *global_ptr, bool pred_guard = true) {
#if CUDA_CP_ASYNC_ACTIVATED
    unsigned smem_int_ptr = cutlass_get_smem_pointer(smem_ptr);
    asm volatile(
        "{\n"
        "  .reg .pred p;\n"
        "  setp.ne.b32 p, %0, 0;\n"
#if CUTLASS_ENABLE_L2_PREFETCH
        "  @p cp.async.ca.shared.global.L2::128B [%1], [%2], %3;\n"
#else
        "  @p cp.async.ca.shared.global [%1], [%2], %3;\n"
#endif
        "}\n" ::"r"((int)pred_guard),
        "r"(smem_int_ptr), "l"(global_ptr), "n"(SizeInBytes)
    );
#else
    using AccessType = Array<uint8_t, SizeInBytes>;
    if (pred_guard) *static_cast<AccessType *>(smem_ptr) = *static_cast<AccessType const *>(global_ptr);
#endif
}

template<int SizeInBytes = 4>
CUTLASS_DEVICE
void cp_async_zfill(void *smem_ptr, void const *global_ptr, bool pred_guard = true) {
#if CUDA_CP_ASYNC_ACTIVATED
    unsigned smem_int_ptr = cutlass_get_smem_pointer(smem_ptr);
    int src_in_bytes = (pred_guard ? SizeInBytes : 0);
    asm volatile(
#if CUTLASS_ENABLE_L2_PREFETCH
        "cp.async.ca.shared.global.L2::128B [%0], [%1], %2, %3;\n" ::"r"(smem_int_ptr),
#else
        "cp.async.ca.shared.global [%0], [%1], %2, %3;\n" ::"r"(smem_int_ptr),
#endif
        "l"(global_ptr), "n"(SizeInBytes), "r"(src_in_bytes)
    );
#else
    using AccessType = Array<uint8_t, SizeInBytes>;
    if (pred_guard) *static_cast<AccessType *>(smem_ptr) = *static_cast<AccessType const *>(global_ptr);
    else {
        AccessType zeros;
        zeros.clear();
        *static_cast<AccessType *>(smem_ptr) = zeros;
    }
#endif
}

CUTLASS_DEVICE
void cp_async_fence() {
#if CUDA_CP_ASYNC_ACTIVATED
    asm volatile("cp.async.commit_group;\n" ::);
#endif
}

template<int N>
CUTLASS_DEVICE
void cp_async_wait() {
#if CUDA_CP_ASYNC_ACTIVATED
    asm volatile("cp.async.wait_group %0;\n" ::"n"(N));
#endif
}

CUTLASS_HOST_DEVICE
void warp_mma(Array<float, 64> &D, const Array<float, 8> &A, const Array<float, 8> &B, const Array<float, 64> &C) {
    TensorRef<float const, ColumnMajorType> a_ref(reinterpret_cast<float const *>(&A), 8);
    TensorRef<float const, RowMajorType> b_ref(reinterpret_cast<float const *>(&B), 8);
    TensorRef<float, RowMajorType> d_ref(reinterpret_cast<float *>(&D), 8);
    D = C;
    int k = 0;
    #pragma unroll
    for (int n = 0; n < 8; n += 2) {
        #pragma unroll
        for (int m = 0; m < 8; m += 2) {
            int m_serpentine = (n % 4) ? (6 - m) : m;
            {
                IntPair mn = {m_serpentine, n};
                IntPair mk = {m_serpentine, k};
                IntPair kn = {k, n};
                d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
            }
            {
                IntPair mn = {m_serpentine + 1, n};
                IntPair mk = {m_serpentine + 1, k};
                IntPair kn = {k, n};
                d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
            }
            {
                IntPair mn = {m_serpentine, n + 1};
                IntPair mk = {m_serpentine, k};
                IntPair kn = {k, n + 1};
                d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
            }
            {
                IntPair mn = {m_serpentine + 1, n + 1};
                IntPair mk = {m_serpentine + 1, k};
                IntPair kn = {k, n + 1};
                d_ref.at(mn) = a_ref.at(mk) * b_ref.at(kn) + d_ref.at(mn);
            }
        }
    }
}

CUTLASS_DEVICE
void load_warp_frag(Array<float, 8> &frag, Array<float, 4> *pointer_, int inc) {
    Array<float, 4> *dst_ptr = reinterpret_cast<Array<float, 4> *>(&frag);
    #pragma unroll
    for (int n = 0; n < 2; n++) {
        void const *ptr = pointer_ + n * inc;
        shared_load(dst_ptr[n], ptr);
    }
}

class MmaSharedStorage {
private:
    AlignedBuffer<float, 4096> operand_A;
    AlignedBuffer<float, 4096> operand_B;

public:
    CUTLASS_HOST_DEVICE
    float *operand_A_ptr() {
        return operand_A.data();
    }

    CUTLASS_HOST_DEVICE
    float *operand_B_ptr() {
        return operand_B.data();
    }
};

CUTLASS_DEVICE
void global_store(const AlignedArray<float, 1> &D, void *ptr, bool pred_guard) {
    uint32_t const &data = reinterpret_cast<uint32_t const &>(D);
    asm volatile(
        "{\n"
        "  .reg .pred p;\n"
        "  setp.ne.b32 p, %2, 0;\n"
        "  @p st.global.u32 [%0], %1;\n"
        "}\n"
        :
        : "l"(ptr), "r"(data), "r"((int)pred_guard)
    );
}

class EpilogueSharedStorage {
private:
    AlignedBuffer<float, 2320> storage; // <16, 145>

public:
    CUTLASS_DEVICE
    float *data() {
        return storage.data();
    }
};

union SharedStorage {
    MmaSharedStorage main_loop;
    EpilogueSharedStorage epilogue;
};

__global__ 
void kernel(float * A, float * B, float * C) { // C = A * B
    extern __shared__ int SharedStorageBase[];
    SharedStorage *shared_storage_ptr = reinterpret_cast<SharedStorage *>(SharedStorageBase);
    SharedStorage &shared_storage = *shared_storage_ptr;

    IntPair threadblock_tile_offset = {int(blockIdx.x / 8), int(blockIdx.x % 8)};
    IntPair threadblock_offset = {threadblock_tile_offset.x * 128, threadblock_tile_offset.y * 128};
    __syncthreads();
    int thread_idx = threadIdx.x;
    AlignedArray<float, 1> *global_memory_A = reinterpret_cast<AlignedArray<float, 1> *>(A) + threadblock_offset.x + thread_idx % 128 + 1024 * (thread_idx / 128);
    AlignedArray<float, 1> *global_memory_B = reinterpret_cast<AlignedArray<float, 1> *>(B) + threadblock_offset.y + thread_idx % 128 + 1024 * (thread_idx / 128);
    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);
    int lane_idx = threadIdx.x % 32;

    float *smem_iterator_A_ = shared_storage.main_loop.operand_A_ptr() + thread_idx;
    float *smem_iterator_B_ = shared_storage.main_loop.operand_B_ptr() + thread_idx;
    Array<float, 4> *warp_tile_iterator_A_ = reinterpret_cast<Array<float, 4> *>(shared_storage.main_loop.operand_A_ptr() + ((lane_idx / 16) * 2 + lane_idx % 2) * 4) + (warp_idx % 4) * 8;
    Array<float, 4> *warp_tile_iterator_B_ = reinterpret_cast<Array<float, 4> *>(shared_storage.main_loop.operand_B_ptr() + ((lane_idx % 16) / 2) * 4) + (warp_idx / 4) * 16;
    Array<float, 64> accumulators;
    accumulators.clear();
    int gemm_k_iterations = 128;
    int smem_write_stage_idx = 0;
    int smem_read_stage_idx = 0;

    #pragma unroll
    for (int stage = 0; stage < 3; ++stage) {
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            AlignedArray<float, 1> *dst_ptr = reinterpret_cast<AlignedArray<float, 1> *>(smem_iterator_A_ + 256 * j + 1024 * smem_write_stage_idx);
            cp_async_zfill<4>(dst_ptr, global_memory_A + (128 - gemm_k_iterations) * 8192 + j * 2048);
        }
        #pragma unroll
        for (int j = 0; j < 4; ++j) {
            AlignedArray<float, 1> *dst_ptr = reinterpret_cast<AlignedArray<float, 1> *>(smem_iterator_B_ + 256 * j + 1024 * smem_write_stage_idx);
            cp_async_zfill<4>(dst_ptr, global_memory_B + (128 - gemm_k_iterations) * 8192 + j * 2048);
        }
        ++smem_write_stage_idx;
        --gemm_k_iterations;
        cp_async_fence();
    }
    cp_async_wait<2>();
    __syncthreads();

    Array<float, 8> warp_loaded_frag_A[2];
    Array<float, 8> warp_loaded_frag_B[2];
    load_warp_frag(warp_loaded_frag_A[0], warp_tile_iterator_A_, 4);
    load_warp_frag(warp_loaded_frag_B[0], warp_tile_iterator_B_, 8);
    #pragma unroll 1
    for ( ; gemm_k_iterations > -3; ) {
        #pragma unroll
        for (int warp_mma_k = 0; warp_mma_k < 8; ++warp_mma_k) {
            load_warp_frag(warp_loaded_frag_A[(warp_mma_k + 1) % 2], warp_tile_iterator_A_ + ((warp_mma_k + 1) % 8 + smem_read_stage_idx * 8) * 32, 4);
            load_warp_frag(warp_loaded_frag_B[(warp_mma_k + 1) % 2], warp_tile_iterator_B_ + ((warp_mma_k + 1) % 8 + smem_read_stage_idx * 8) * 32, 8);

            warp_mma(accumulators, warp_loaded_frag_A[warp_mma_k % 2], warp_loaded_frag_B[warp_mma_k % 2], accumulators);

            if (warp_mma_k < 4 && gemm_k_iterations > 0) {
                AlignedArray<float, 1> *dst_ptr_A = reinterpret_cast<AlignedArray<float, 1> *>(smem_iterator_A_ + 256 * warp_mma_k + 1024 * smem_write_stage_idx);
                cp_async<4>(dst_ptr_A, global_memory_A + (128 - gemm_k_iterations) * 8192 + warp_mma_k * 2048);
                AlignedArray<float, 1> *dst_ptr_B = reinterpret_cast<AlignedArray<float, 1> *>(smem_iterator_B_ + 256 * warp_mma_k + 1024 * smem_write_stage_idx);
                cp_async<4>(dst_ptr_B, global_memory_B + (128 - gemm_k_iterations) * 8192 + warp_mma_k * 2048);
            }
            if (warp_mma_k == 6) {
                cp_async_fence();
                cp_async_wait<2>();
                __syncthreads();

                if (smem_write_stage_idx == 3) smem_write_stage_idx = 0;
                else ++smem_write_stage_idx;
                if (smem_read_stage_idx == 3) smem_read_stage_idx = 0;
                else ++smem_read_stage_idx;
                --gemm_k_iterations;
            }
        }
    }



    AlignedArray<float, 1> *warp_tile_iterator_ = reinterpret_cast<AlignedArray<float, 1> *>(shared_storage.epilogue.data())
        + ((lane_idx / 16) * 2 + lane_idx % 2) * 145 + ((lane_idx % 16) / 2) * 4 + (warp_idx % 4) * 4 * 145 + (warp_idx / 4) * 64;
    AlignedArray<float, 1> const *shared_load_iterator_ = reinterpret_cast<AlignedArray<float, 1> const *>(shared_storage.epilogue.data())
        + ((warp_idx / 2) * 4 + (warp_idx % 2)) * 145 + lane_idx;
    AlignedArray<float, 1> *global_memory_C = reinterpret_cast<AlignedArray<float, 1> *>(
        C + ((warp_idx / 2) * 32 + (warp_idx % 2) * 4 + threadblock_offset.x) * 1024 + lane_idx + threadblock_offset.y);
    Array<float, 4> const * accum_fragment_iterator = reinterpret_cast<Array<float, 4> const *>(&accumulators);

    #pragma unroll(8)
    for (int iter = 0; iter < 8; ++iter) {
        __syncthreads();
        Array<float, 8> accum_fragment;
        Array<float, 4> *frag_ptr_0 = reinterpret_cast<Array<float, 4> *>(&accum_fragment);
        #pragma unroll
        for (int n = 0; n < 2; n++) {
            frag_ptr_0[n] = accum_fragment_iterator[iter * 2 + n];
        }
        AlignedArray<float, 1> const *scalar_frag_ptr = reinterpret_cast<AlignedArray<float, 1> const *>(&accum_fragment);
        #pragma unroll
        for (int n = 0; n < 2; ++n) {
            #pragma unroll
            for (int s = 0; s < 4; ++s) {
                warp_tile_iterator_[n * 32 + s] = scalar_frag_ptr[n * 4 + s];
            }
        }

        __syncthreads();
        Array<float, 8> aligned_accum_fragment;
        AlignedArray<float, 1> *frag_ptr_1 = reinterpret_cast<AlignedArray<float, 1> *>(&aligned_accum_fragment);
        #pragma unroll
        for (int group = 0; group < 2; ++group) {
            AlignedArray<float, 1> const *memory_pointer = shared_load_iterator_ + group * 2 * 145;
            #pragma unroll
            for (int column = 0; column < 4; ++column) {
                frag_ptr_1[group * 4 + column] = memory_pointer[column * 32];
            }
        }
        AlignedArray<float, 1> *pointer = global_memory_C + (iter % 4) * 1024 + (iter / 4) * 16384;
        AlignedArray<float, 1> const *frag_ptr_2 = reinterpret_cast<AlignedArray<float, 1> const *>(&aligned_accum_fragment);
        #pragma unroll
        for (int group = 0; group < 2; ++group) {
            AlignedArray<float, 1> *memory_pointer = reinterpret_cast<AlignedArray<float, 1> *>(pointer + group * 8192);
            #pragma unroll
            for (int column = 0; column < 4; ++column) {
                global_store(frag_ptr_2[group * 4 + column], (void *)&memory_pointer[0], true);
                memory_pointer += 32;
            }
        }
    }
}

void run_cutlassMultiStage(int M, int N, int K, float * A, float * B, float * C, float alpha, float beta) {
    dim3 grid(64, 1, 1), block(256, 1, 1);
    int smem_size = int(sizeof(SharedStorage));
    kernel<<<grid, block, smem_size, nullptr>>>(A, B, C);
}

